#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/aggregation.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/copying.hpp>
#include <rolling/rolling_detail.hpp>
#include <cudf/rolling.hpp>

#include <rmm/device_scalar.hpp>

#include <memory>

namespace cudf {
namespace experimental {

namespace detail {

namespace { // anonymous

template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, bool has_nulls>
std::enable_if_t<std::is_same<agg_op, cudf::DeviceCount>::value, bool>
__device__
specific_rolling_kernel(column_device_view input,
                        mutable_column_device_view output,
                        size_type start_index,
                        size_type end_index,
                        size_type current_index,
                        size_type min_periods,
                        InputType identity) {

    // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
    // for CUDA 10.0 and below (fixed in CUDA 10.1)
    volatile cudf::size_type count = 0;
    
    for (size_type j = start_index; j < end_index; j++) {
        if (!has_nulls || input.is_valid(j)) {
            count++;
        }
    }
   
    bool output_is_valid = (count >= min_periods);
    output.element<OutputType>(current_index) = count;

    return output_is_valid;
}

template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, bool has_nulls>
std::enable_if_t<(std::is_same<agg_op, cudf::DeviceMin>::value or std::is_same<agg_op, cudf::DeviceMax>::value) and
                 (op == aggregation::ARGMIN  or op == aggregation::ARGMAX) and
                 std::is_same<InputType, cudf::string_view>::value, bool>
__device__
specific_rolling_kernel(column_device_view input,
                        mutable_column_device_view output,
                        size_type start_index,
                        size_type end_index,
                        size_type current_index,
                        size_type min_periods,
                        InputType identity) {

    // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
    // for CUDA 10.0 and below (fixed in CUDA 10.1)
    volatile cudf::size_type count = 0;
    InputType val = identity;
    OutputType val_index = (op == aggregation::ARGMIN)? ARGMIN_SENTINEL : ARGMAX_SENTINEL;

    for (size_type j = start_index; j < end_index; j++) {
        if (!has_nulls || input.is_valid(j)) {
            InputType element = input.element<InputType>(j);
            val = agg_op{}(element, val);
            if (val == element) {
                val_index = j;
            }
            count++;
        }
    }

    bool output_is_valid = (count >= min_periods);
    // -1 will help identify null elements while gathering for Min and Max
    // In case of count, this would be null, so doesn't matter.
    output.element<OutputType>(current_index) = (output_is_valid)? val_index : -1;

    return output_is_valid;
}

template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, bool has_nulls>
std::enable_if_t<!std::is_same<InputType, cudf::string_view>::value and !(op == aggregation::COUNT), bool>
__device__
specific_rolling_kernel(column_device_view input,
                        mutable_column_device_view output,
                        size_type start_index,
                        size_type end_index,
                        size_type current_index,
                        size_type min_periods,
                        InputType identity) {

    // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
    // for CUDA 10.0 and below (fixed in CUDA 10.1)
    volatile cudf::size_type count = 0;
    OutputType val = agg_op::template identity<OutputType>();

    for (size_type j = start_index; j < end_index; j++) {
        if (!has_nulls || input.is_valid(j)) {
            OutputType element = input.element<InputType>(j);
            val = agg_op{}(element, val);
            count++;
        }
    }

    bool output_is_valid = (count >= min_periods);

    // store the output value, one per thread
    if (output_is_valid)
        cudf::detail::store_output_functor<OutputType, op == aggregation::MEAN>{}(output.element<OutputType>(current_index),
                val, count);

    return output_is_valid;
}

/**
 * @brief Computes the rolling window function
 *
 * @tparam ColumnType  Datatype of values pointed to by the pointers
 * @tparam agg_op  A functor that defines the aggregation operation
 * @tparam is_mean Compute mean=sum/count across all valid elements in the window
 * @tparam block_size CUDA block size for the kernel
 * @tparam has_nulls true if the input column has nulls
 * @tparam WindowIterator iterator type (inferred)
 * @param input Input column device view
 * @param output Output column device view
 * @param preceding_window_begin[in] Rolling window size iterator, accumulates from
 *                in_col[i-preceding_window] to in_col[i] inclusive
 * @param following_window_begin[in] Rolling window size iterator in the forward
 *                direction, accumulates from in_col[i] to
 *                in_col[i+following_window] inclusive
 * @param min_periods[in]  Minimum number of observations in window required to
 *                have a value, otherwise 0 is stored in the valid bit mask
 */
template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, 
         int block_size, bool arg_min_max, bool has_nulls, typename WindowIterator>
__launch_bounds__(block_size)
__global__
void gpu_rolling(column_device_view input,
                 mutable_column_device_view output,
                 size_type * __restrict__ output_valid_count,
                 WindowIterator preceding_window_begin,
                 WindowIterator following_window_begin,
                 size_type min_periods,
                 InputType identity)
{
  size_type i = blockIdx.x * block_size + threadIdx.x;
  size_type stride = block_size * gridDim.x;

  size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffffffff, i < input.size());
  while(i < input.size())
  {

    size_type preceding_window = preceding_window_begin[i];
    size_type following_window = following_window_begin[i];

    // compute bounds
    size_type start_index = max(0, i - preceding_window);
    size_type end_index = min(input.size(), i + following_window + 1); // exclusive

    // aggregate
    // TODO: We should explore using shared memory to avoid redundant loads.
    //       This might require separating the kernel into a special version
    //       for dynamic and static sizes.

    bool output_is_valid = specific_rolling_kernel<InputType, OutputType, agg_op,
                           op, has_nulls>(input, output, start_index, end_index, i, min_periods, identity); 

    // set the mask
    // We can't have gather map being created for Min and Max for string_view to be null
    cudf::bitmask_type result_mask{__ballot_sync(active_threads, arg_min_max? true : output_is_valid)};

    // only one thread writes the mask
    if (0 == threadIdx.x % cudf::experimental::detail::warp_size) {
      output.set_mask_word(cudf::word_index(i), result_mask);
      warp_valid_count += __popc(result_mask);
    }

    // process next element 
    i += stride;
    active_threads = __ballot_sync(active_threads, i < input.size());
  }

  // sum the valid counts across the whole block  
  size_type block_valid_count = 
    cudf::experimental::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);
  
  if(threadIdx.x == 0) {
    atomicAdd(output_valid_count, block_valid_count);
  }
}

template <typename InputType>
struct rolling_window_launcher
{

  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator, bool op_argmin_agrmax=false>
  void kernel_launcher(column_view const& input,
                       mutable_column_view& output,
                       WindowIterator preceding_window_begin,
                       WindowIterator following_window_begin,
                       size_type min_periods,
                       std::unique_ptr<aggregation> const& aggr,
                       T identity,
                       hipStream_t stream) {
      
      cudf::nvtx::range_push("CUDF_ROLLING_WINDOW", cudf::nvtx::color::ORANGE);

      constexpr cudf::size_type block_size = 256;
      cudf::experimental::detail::grid_1d grid(input.size(), block_size);

      auto input_device_view = column_device_view::create(input, stream);
      auto output_device_view = mutable_column_device_view::create(output, stream);

      rmm::device_scalar<size_type> device_valid_count{0, stream};

      if (input.has_nulls()) {
          gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, op_argmin_agrmax, true><<<grid.num_blocks, block_size, 0, stream>>>
              (*input_device_view, *output_device_view, device_valid_count.data(),
               preceding_window_begin, following_window_begin, min_periods, identity);
      } else {
          gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, op_argmin_agrmax, false><<<grid.num_blocks, block_size, 0, stream>>>
              (*input_device_view, *output_device_view, device_valid_count.data(),
               preceding_window_begin, following_window_begin, min_periods, identity);
      }

      output.set_null_count(output.size() - device_valid_count.value(stream));

      // check the stream for debugging
      CHECK_CUDA(stream);
      
      cudf::nvtx::range_pop();

  }

  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<(cudf::detail::is_supported<T, agg_op,
                                  op, op == aggregation::MEAN>()) and
                   !(cudf::detail::is_string_supported<T, agg_op, op>()), std::unique_ptr<column>>
  launch(column_view const& input,
         WindowIterator preceding_window_begin,
         WindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& aggr,
         rmm::mr::device_memory_resource *mr,
         hipStream_t stream) {

      if (input.is_empty()) return empty_like(input);

      auto output = make_fixed_width_column(target_type(input.type(), op), input.size(),
              UNINITIALIZED, stream, mr);

      cudf::mutable_column_view output_view = output->mutable_view();
      kernel_launcher<T, agg_op, op, WindowIterator>(input, output_view, preceding_window_begin,
              following_window_begin, min_periods, aggr, agg_op::template identity<T>(), stream);

      return output;
  }

  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<!(cudf::detail::is_supported<T, agg_op,
                                  op, op == aggregation::MEAN>()) and
                   (cudf::detail::is_string_supported<T, agg_op, op>()), std::unique_ptr<column>>
  launch(column_view const& input,
         WindowIterator preceding_window_begin,
         WindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& aggr,
         rmm::mr::device_memory_resource *mr,
         hipStream_t stream) {

      if (input.is_empty()) return empty_like(input);

      auto output = make_numeric_column(cudf::data_type{cudf::experimental::type_to_id<size_type>()},
            input.size(), cudf::UNINITIALIZED, stream, mr);

      cudf::mutable_column_view output_view = output->mutable_view();

      if(op == aggregation::MIN) {
          kernel_launcher<T, DeviceMin, aggregation::ARGMIN, WindowIterator, true>(input, output_view, preceding_window_begin,
                  following_window_begin, min_periods, aggr, DeviceMin::template identity<T>(), stream);
      } else if(op == aggregation::MAX) {
          kernel_launcher<T, DeviceMax, aggregation::ARGMAX, WindowIterator, true>(input, output_view, preceding_window_begin,
                  following_window_begin, min_periods, aggr, DeviceMax::template identity<T>(), stream);
      } else {
          kernel_launcher<T, DeviceCount, aggregation::COUNT, WindowIterator>(input, output_view, preceding_window_begin,
                  following_window_begin, min_periods, aggr, string_view{}, stream);
      }

      // If aggregation operation is MIN or MAX, then the output we got is a scatter map
      if((op == aggregation::MIN) or (op == aggregation::MAX)) {
          // The rows that represent null elements will be having negative values in gather map,
          // and that's why nullify_out_of_bounds/ignore_out_of_bounds is true.
          auto output_table = detail::gather(table_view{{input}}, output->view(), false, true, false, mr, stream);
          return std::make_unique<cudf::column>(std::move(output_table->get_column(0)));;
      }

      return output;
  }

  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<!(cudf::detail::is_supported<T, agg_op,
                                  op, op == aggregation::MEAN>()) and
                   !(cudf::detail::is_string_supported<T, agg_op, op>()), std::unique_ptr<column>>
  launch(column_view const& input,
         WindowIterator preceding_window_begin,
         WindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& aggr,
         rmm::mr::device_memory_resource *mr,
         hipStream_t stream) {

      CUDF_FAIL("Aggregation operator and/or input type combination is invalid");
  }


  template<aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<!(op == aggregation::MEAN), std::unique_ptr<column>>
  operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& aggr,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream)
  {
      return launch <InputType, typename corresponding_operator<op>::type, op, WindowIterator> (
              input,
              preceding_window_begin,
              following_window_begin,
              min_periods,
              aggr,
              mr,
              stream);
  }

  template<aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<(op == aggregation::MEAN), std::unique_ptr<column>>
  operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& aggr,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream) {

      return launch <InputType, cudf::DeviceSum, op, WindowIterator> (
              input,
              preceding_window_begin,
              following_window_begin,
              min_periods,
              aggr,
              mr,
              stream);
  }


};

struct dispatch_rolling {
    template <typename T, typename WindowIterator>
    std::unique_ptr<column> operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& aggr,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream) {

        return aggregation_dispatcher(aggr->kind, rolling_window_launcher<T>{},
                                      input,
                                      preceding_window_begin, following_window_begin,
                                      min_periods, aggr, mr, stream);
    }
};

} // namespace anonymous

// Applies a rolling window function to the values in a column.
template <typename WindowIterator>
std::unique_ptr<column> rolling_window(column_view const& input,
                                       WindowIterator preceding_window_begin,
                                       WindowIterator following_window_begin,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& aggr,
                                       rmm::mr::device_memory_resource* mr,
                                       hipStream_t stream = 0)
{
  return cudf::experimental::type_dispatcher(input.type(),
                                             dispatch_rolling{},
                                             input, preceding_window_begin, following_window_begin,
                                             min_periods, aggr, mr, stream);
}

} // namespace detail

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& aggr,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS((preceding_window >= 0) && (following_window >= 0) && (min_periods >= 0),
               "Window sizes and min periods must be non-negative");

  auto preceding_window_begin = thrust::make_constant_iterator(preceding_window);
  auto following_window_begin = thrust::make_constant_iterator(following_window);

  return cudf::experimental::detail::rolling_window(input, preceding_window_begin,
                                                    following_window_begin, min_periods, aggr, mr, 0);
}

// Applies a variable-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& preceding_window,
                                       column_view const& following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& aggr,
                                       rmm::mr::device_memory_resource* mr)
{
  if (preceding_window.size() == 0 || following_window.size() == 0) return empty_like(input);

  CUDF_EXPECTS(preceding_window.type().id() == INT32 && following_window.type().id() == INT32,
               "preceding_window/following_window must have INT32 type");

  CUDF_EXPECTS(preceding_window.size() == input.size() && following_window.size() == input.size(),
               "preceding_window/following_window size must match input size");

  return cudf::experimental::detail::rolling_window(input, preceding_window.begin<size_type>(),
                                                    following_window.begin<size_type>(),
                                                    min_periods, aggr, mr, 0);
}

} // namespace experimental 
} // namespace cudf

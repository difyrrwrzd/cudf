#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file parsing_utils.cu Utility functions for parsing plain-text files
 *
 */


#include "parsing_utils.cuh"

#include <hip/hip_runtime.h>

#include <vector>
#include <memory>
#include <iostream>

#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "utilities/error_utils.hpp"
#include "io/utilities/wrapper_utils.hpp"

// When processing the input in chunks, this is the maximum size of each chunk.
// Only one chunk is loaded on the GPU at a time, so this value is chosen to
// be small enough to fit on the GPU in most cases.
constexpr size_t max_chunk_bytes = 256*1024*1024; // 256MB

constexpr int bytes_per_find_thread = 64;

using pos_key_pair = thrust::pair<uint64_t,char>;

template <typename T>
constexpr T divCeil(T dividend, T divisor) noexcept { return (dividend + divisor - 1) / divisor; }

/**---------------------------------------------------------------------------*
 * @brief Sets the specified element of the array to the passed value
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(T* array, gdf_size_type idx, const T& t, const V& v){
	array[idx] = t;
}

/**---------------------------------------------------------------------------*
 * @brief Sets the specified element of the array of pairs using the two passed
 * parameters.
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(thrust::pair<T, V>* array, gdf_size_type idx, const T& t, const V& v) {
	array[idx] = {t, v};
}

/**---------------------------------------------------------------------------*
 * @brief Overloads the setElement() functions for void* arrays.
 * Does not do anything, indexing is not allowed with void* arrays.
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(void* array, gdf_size_type idx, const T& t, const V& v) {
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that finds all occurrences of a character in the given 
 * character array. If the 'positions' parameter is not void*,
 * positions of all occurrences are stored in the output array.
 * 
 * @param[in] data Pointer to the input character array
 * @param[in] size Number of bytes in the input array
 * @param[in] offset Offset to add to the output positions
 * @param[in] key Character to find in the array
 * @param[in,out] count Pointer to the number of found occurrences
 * @param[out] positions Array containing the output positions
 * 
 * @return void
 *---------------------------------------------------------------------------**/
template<class T>
 __global__ 
 void countAndSetPositions(char *data, uint64_t size, uint64_t offset, const char key, gdf_size_type* count,
	T* positions) {

	// thread IDs range per block, so also need the block id
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t did = tid * bytes_per_find_thread;
	
	const char *raw = (data + did);

	const long byteToProcess = ((did + bytes_per_find_thread) < size) ?
									bytes_per_find_thread :
									(size - did);

	// Process the data
	for (long i = 0; i < byteToProcess; i++) {
		if (raw[i] == key) {
			const auto idx = atomicAdd(count, (gdf_size_type)1);
			setElement(positions, idx, did + offset + i, key);
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set.
 * Sums up the number of occurrences. If the 'positions' parameter is not void*,
 * positions of all occurrences are stored in the output device array.
 * 
 * Does not load the entire file into the GPU memory at any time, so it can 
 * be used to parse large files. Output array needs to be preallocated.
 * 
 * @param[in] h_data Pointer to the input character array
 * @param[in] h_size Number of bytes in the input array
 * @param[in] keys Vector containing the keys to count in the buffer
 * @param[in] result_offset Offset to add to the output positions
 * @param[out] positions Array containing the output positions
 * 
 * @return gdf_size_type total number of occurrences
 *---------------------------------------------------------------------------**/
template<class T>
gdf_size_type findAllFromSet(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	T *positions) {

	char* d_chunk = nullptr;
	RMM_TRY(RMM_ALLOC (&d_chunk, min(max_chunk_bytes, h_size), 0));
	device_ptr<char> chunk_deleter(d_chunk);

	gdf_size_type*	d_count;
	RMM_TRY(RMM_ALLOC((void**)&d_count, sizeof(gdf_size_type), 0) );
	device_ptr<gdf_size_type> count_deleter(d_count);
	CUDA_TRY(hipMemsetAsync(d_count, 0ull, sizeof(gdf_size_type)));

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, countAndSetPositions<T>) );

	const size_t chunk_count = divCeil(h_size, max_chunk_bytes);
	for (size_t ci = 0; ci < chunk_count; ++ci) {	
		const auto chunk_offset = ci * max_chunk_bytes;	
		const auto h_chunk = h_data + chunk_offset;
		const int chunk_bytes = std::min((size_t)(h_size - ci * max_chunk_bytes), max_chunk_bytes);
		const auto chunk_bits = divCeil(chunk_bytes, bytes_per_find_thread);
		const int gridSize = divCeil(chunk_bits, blockSize);

		// Copy chunk to device
		CUDA_TRY(hipMemcpyAsync(d_chunk, h_chunk, chunk_bytes, hipMemcpyDefault));

		for (char key: keys) {
			countAndSetPositions<T> <<< gridSize, blockSize >>> (
				d_chunk, chunk_bytes, chunk_offset + result_offset, key,
				d_count, positions);
		}
	}

	gdf_size_type h_count = 0;
	CUDA_TRY(hipMemcpy(&h_count, d_count, sizeof(gdf_size_type), hipMemcpyDefault));
	return h_count;
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set
 * and sums up the number of occurrences.
 *
 * Does not load the entire buffer into the GPU memory at any time, so it can 
 * be used with buffers of any size.
 *
 * @param[in] h_data Pointer to the data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in] keys Vector containing the keys to count in the buffer
 *
 * @return gdf_size_type total number of occurrences
 *---------------------------------------------------------------------------**/
gdf_size_type countAllFromSet(const char *h_data, size_t h_size, const std::vector<char>& keys) {
	return findAllFromSet<void>(h_data, h_size, keys, 0, nullptr);
}

template gdf_size_type findAllFromSet<uint64_t>(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	uint64_t *positions);

template gdf_size_type findAllFromSet<pos_key_pair>(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	pos_key_pair *positions);

/**
 * @brief A class representing an array of partial sums, stored on the GPU memory.
 *
 * The object is just a reference to the device memory,
 * it does not own the device memory.
 **/
struct BlockSumArray {
		int16_t* d_sums = nullptr;	///< Array of partial sums
		uint64_t length = 0;		///< Length of the array
		uint64_t block_size;		///< The number of elements aggregated into each partial sum

		BlockSumArray(uint64_t len, uint64_t bsize): length(len), block_size(bsize){}
};

/**
 * @brief A class that stores a pyramid of aggregated sums, in the GPU memory.
 *
 * Each level of the pyramid is aggregation_rate times smaller than the previous, rounded down.
 * Objects of this type own the allocated memory.
 **/
class BlockSumPyramid {
	const uint16_t aggregation_rate_ = 32;	///< Aggregation rate between each level of the pyramid
	BlockSumArray* d_levels_ = nullptr;		///< Device array of partial sums, largest to smallest
	std::vector<BlockSumArray> levels_;		///< Host array of the partial sums on device, largest to smallest

public:
	BlockSumPyramid(int count){
		int prev_lvl_cnt = count;
		int prev_lvl_block_size = 1;
		while (prev_lvl_cnt >= aggregation_rate_) {
			levels_.emplace_back(prev_lvl_cnt/aggregation_rate_, prev_lvl_block_size*aggregation_rate_);
			RMM_ALLOC(&levels_.back().d_sums, levels_.back().length*sizeof(int16_t), 0);
			prev_lvl_cnt = levels_.back().length;
			prev_lvl_block_size = levels_.back().block_size;
		}

		if (!levels_.empty()) {	
			RMM_ALLOC(&d_levels_, levels_.size()*sizeof(BlockSumArray), 0);
			hipMemcpyAsync(d_levels_, levels_.data(), levels_.size()*sizeof(BlockSumArray), hipMemcpyDefault);
		}
	}

	auto operator[](int lvl) const {return levels_[lvl];}
	auto deviceGetLevels() const noexcept {return d_levels_;}
	size_t getHeight() const noexcept {return levels_.size();}
	constexpr auto getAggregationRate() const {return aggregation_rate_;}

	// disable copying
	BlockSumPyramid(BlockSumPyramid&) = delete;
	BlockSumPyramid& operator=(BlockSumPyramid&) = delete;

	~BlockSumPyramid() {
		for (auto& lvl: levels_) {
			RMM_FREE(lvl.d_sums, 0);
		}
		RMM_FREE(d_levels_, 0);
	}
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that aggregates bracket nesting levels for each block
 * in the input array.
 *
 * Each sum is the level difference between the first bracket in the block,
 * and the first bracket in the next block (if any). For example, "[[]]" = 0,
 * because all open brackets are closed. "[[]" = 1, because the one unmatched
 * open bracket would raise the level of all subsequent lemenets.
 * 
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] bracket_count Number of brackets
 * @param[in] open_chars Array of characters to treat as open brackets
 * @param[in] close_chars Array of characters to treat as open brackets
 * @param[in] bracket_char_cnt Number of bracket character pairs
 * @param[in, out] sum_array Array of partial sums
 * 
 * @return void
 *---------------------------------------------------------------------------**/
__global__
void sumBracketsKernel(
	pos_key_pair* brackets, int bracket_count,
	const char* open_chars, const char* close_chars, int bracket_char_cnt,
	BlockSumArray sum_array) {
	const uint64_t sum_idx = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t first_in_idx = sum_idx * sum_array.block_size;

	if (sum_idx >= sum_array.length)
		return;

	int16_t sum = 0;
	for (uint64_t in_idx = first_in_idx; in_idx < first_in_idx + sum_array.block_size; ++in_idx) {
		for (int bchar_idx = 0; bchar_idx < bracket_char_cnt; ++bchar_idx) {
			if (brackets[in_idx].second == open_chars[bchar_idx]) {
				++sum; 
				break;
			}
			if (brackets[in_idx].second == close_chars[bchar_idx]) {
				--sum; 
				break;
			}
		}
	}
	sum_array.d_sums[sum_idx] = sum;
}

/**---------------------------------------------------------------------------*
 * @brief Wrapper around sumBracketsKernel
 *
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] bracket_count Number of brackets
 * @param[in] open_chars Array of characters to treat as open brackets
 * @param[in] close_chars Array of characters to treat as close brackets
 * @param[in] bracket_char_cnt Number of bracket character pairs
 * @param[in, out] sum_array Array of partial sums
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void sumBrackets(
	pos_key_pair* brackets, int bracket_count,
	char* open_chars, char* close_chars, int bracket_char_cnt,
	const BlockSumArray& sum_array) {
	int blockSize = 0;
	int minGridSize = 0;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
		sumBracketsKernel));

	const int gridSize = divCeil(sum_array.length, (uint64_t)blockSize);

	sumBracketsKernel<<<gridSize, blockSize>>>(
		brackets, bracket_count,
		open_chars, close_chars, bracket_char_cnt,
		sum_array);
	CUDA_TRY(hipGetLastError());
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that computes partial sums of the input elements
 * 
 * @param[in] elements Array of input elements to sum
 * @param[in, out] aggregate Array of partial sums
 * 
 * @return void
 *---------------------------------------------------------------------------**/
__global__
void aggregateSumKernel(BlockSumArray elements, BlockSumArray aggregate){
	const uint64_t aggregate_idx = threadIdx.x + (blockDim.x * blockIdx.x);
	const int aggregate_group_size = aggregate.block_size / elements.block_size;
	const uint64_t first_in_idx = aggregate_idx * aggregate_group_size;

	if (aggregate_idx >= aggregate.length)
		return;

	int16_t sum = 0;
	for (int in_idx = first_in_idx; in_idx < first_in_idx + aggregate_group_size; ++in_idx) {
		sum += elements.d_sums[in_idx];
	}

	aggregate.d_sums[aggregate_idx] = sum;
}

/**---------------------------------------------------------------------------*
 * @brief Wrapper around aggregateSumKernel
 * 
 * @param[in] elements Array of input elements to sum
 * @param[in, out] aggregate Array of partial sums
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void aggregateSum(const BlockSumArray& elements, const BlockSumArray& aggregate){
	int blockSize = 0;
	int minGridSize = 0;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
		aggregateSumKernel));

	const int gridSize = divCeil(aggregate.length, (uint64_t)blockSize);

	aggregateSumKernel<<<gridSize, blockSize>>>(elements, aggregate);
	CUDA_TRY(hipGetLastError());
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that assigns levels to each bracket,
 * with 1 being the top level
 *
 * The algorithm uses the pyramid of partial sums to compute the levels
 * in parallel, in log(n) time per block of elements.
 * 
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] count Number of brackets
 * @param[in] sum_pyramid 
 * @param[in] pyramid_height 
 * @param[in] open_chars Array of characters to treat as open brackets
 * @param[in] close_chars Array of characters to treat as close brackets
 * @param[in] bracket_char_cnt Number of bracket character pairs
 * @param[out] levels Array of outout levels
 * 
 * @return void
 *---------------------------------------------------------------------------**/
__global__
void assignLevelsKernel(
	pos_key_pair* brackets, uint64_t count,
	BlockSumArray* sum_pyramid, int pyramid_height,
	char* open_chars, char* close_chars, int bracket_char_cnt,
	int16_t* levels) {
	// Process the number of elements equal to the aggregation rate, if the pyramid is used
	// Process all elements otherwise
	const auto to_process = pyramid_height != 0 ? sum_pyramid[0].block_size : count;
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t first_bracket_idx = tid * to_process;

	if (first_bracket_idx >= count)
		return;

	// Find the total sum of levels before the current block
	int sum = 0;
	if (pyramid_height != 0) {
		const auto aggregation_rate = sum_pyramid[0].block_size;
		int level = pyramid_height - 1;
		int block_idx = 0;
		int offset = first_bracket_idx;
		while(offset) {
			while(offset < sum_pyramid[level].block_size && level > 0) {
				--level; block_idx *= aggregation_rate;
			}
			while(offset >= sum_pyramid[level].block_size) {
				offset -= sum_pyramid[level].block_size;
				sum += sum_pyramid[level].d_sums[block_idx];
				++block_idx;
			}
		}
	}

	const auto last_bracket_idx = min(first_bracket_idx + to_process, count) - 1;
	for (uint64_t bracket_idx = first_bracket_idx; bracket_idx <= last_bracket_idx; ++bracket_idx){
		for (int bchar_idx = 0; bchar_idx < bracket_char_cnt; ++bchar_idx) {
			if (brackets[bracket_idx].second == open_chars[bchar_idx]) {
				levels[bracket_idx] = ++sum;
				break;
			}
			else if (brackets[bracket_idx].second == close_chars[bchar_idx]) {
				levels[bracket_idx] = sum--;
				break;
			}
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Wrapper around assignLevelsKernel
 * 
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] count Number of brackets
 * @param[in] sum_pyramid 
 * @param[in] pyramid_height 
 * @param[in] open_chars Array of characters to treat as open brackets
 * @param[in] close_chars Array of characters to treat as close brackets
 * @param[in] bracket_char_cnt Number of bracket character pairs
 * @param[out] levels Array of outout levels
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void assignLevels(pos_key_pair* brackets, uint64_t count,
	const BlockSumPyramid& sum_pyramid,
	char* open_chars, char* close_chars, int bracket_char_cnt,
	int16_t* levels) {
	int blockSize;
	int minGridSize;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
		assignLevelsKernel));

	const int threadCnt = divCeil(count, (uint64_t)sum_pyramid.getAggregationRate());
	const int gridSize = divCeil(threadCnt, blockSize);

	assignLevelsKernel<<<gridSize, blockSize>>>(
		brackets, count,
		sum_pyramid.deviceGetLevels(), sum_pyramid.getHeight(),
		open_chars, close_chars, bracket_char_cnt,
		levels);
	CUDA_TRY(hipGetLastError());
};

/**---------------------------------------------------------------------------*
 * @brief Computes nested level for each of the brackets in the input array
 * 
 * The input array of brackets is sorted before levels are computed. 
 * Brackets at the top level are assigned level 1.
 * 
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] count Number of brackets
 * @param[in] open_chars string of characters to treat as open brackets
 * @param[in] close_chars string of characters to treat as close brackets
 * 
 * @return device_ptr<int16_t> Device memory array of levels
 *---------------------------------------------------------------------------**/
device_ptr<int16_t> getBracketLevels(
	pos_key_pair* brackets, int count,
	const std::string& open_chars, const std::string& close_chars){
	// TODO: consider moving sort() out of this function
	thrust::sort(rmm::exec_policy()->on(0), brackets, brackets + count);

	// Total bracket level difference within each segment of brackets
	BlockSumPyramid aggregated_sums(count);
	
	assert(open_chars.size() == open_chars.size());

	// Copy the open/close chars to device
	char* d_open_chars = nullptr;
	RMM_ALLOC(&d_open_chars, open_chars.size() * sizeof(char), 0);
	device_ptr<char> d_open_chars_deleter(d_open_chars);
	CUDA_TRY(hipMemcpyAsync(
		d_open_chars, open_chars.c_str(),
		open_chars.size() * sizeof(char), hipMemcpyDefault));

	char* d_close_chars = nullptr;
	RMM_ALLOC(&d_close_chars, close_chars.size() * sizeof(char), 0);
	device_ptr<char> d_close_chars_deleter(d_open_chars);
	CUDA_TRY(hipMemcpyAsync(
		d_close_chars, close_chars.c_str(),
		close_chars.size() * sizeof(char), hipMemcpyDefault));

	if (aggregated_sums.getHeight() != 0) {
		sumBrackets(brackets, count, d_open_chars, d_close_chars, open_chars.size(), aggregated_sums[0]);
		for (size_t level_idx = 1; level_idx < aggregated_sums.getHeight(); ++level_idx)
			aggregateSum(aggregated_sums[level_idx - 1], aggregated_sums[level_idx]);
	}

	int16_t* d_levels = nullptr;
	RMM_ALLOC(&d_levels, sizeof(int16_t) * count, 0);
	assignLevels(brackets, count, aggregated_sums, d_open_chars, d_close_chars, open_chars.size(), d_levels);

	return device_ptr<int16_t>(d_levels);
}

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file parsing_utils.cu Utility functions for parsing plain-text files
 *
 */

#include "parsing_utils.cuh"

#include <hip/hip_runtime.h>

#include <vector>
#include <memory>

#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "utilities/error_utils.hpp"

// When processing the input in chunks, this is the maximum size of each chunk.
// Only one chunk is loaded on the GPU at a time, so this value is chosen to
// be small enough to fit on the GPU in most cases.
constexpr size_t max_chunk_bytes = 256*1024*1024; // 256MB
constexpr int bytes_per_find_thread = 64;

template <typename T>
struct rmm_deleter {
 void operator()(T *ptr) { RMM_FREE(ptr, 0); }
};
template <typename T>
using device_ptr = std::unique_ptr<T, rmm_deleter<T>>;

using position_key_pair = thrust::pair<uint64_t,char>;

//doxy
template<class P, class K>
__device__ __forceinline__
void updatePosition(P* positions, long idx, P position, K key){
	positions[idx] = position;
}

template<class P, class K>
__device__ __forceinline__
void updatePosition(thrust::pair<P, K>* positions, long idx, P position, K key) {
	positions[idx] = {position, key};
}

template<class P, class K>
__device__ __forceinline__
void updatePosition(void* positions, long idx, P position, K key) {
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that finds all occurrences of a character in the given 
 * character array. The positions are stored in the output array.
 * 
 * @param[in] data Pointer to the input character array
 * @param[in] size Number of bytes in the input array
 * @param[in] offset Offset to add to the output positions
 * @param[in] key Character to find in the array
 * @param[in,out] count Pointer to the number of found occurrences
 * @param[out] positions Array containing the output positions
 * 
 * @return void
 *---------------------------------------------------------------------------**/
template<class T>
 __global__ 
 void countAndSetPositions(char *data, uint64_t size, uint64_t offset, const char key, gdf_size_type* count,
	T* positions) {

	// thread IDs range per block, so also need the block id
	const long tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const long did = tid * bytes_per_find_thread;
	
	const char *raw = (data + did);

	const long byteToProcess = ((did + bytes_per_find_thread) < size) ?
									bytes_per_find_thread :
									(size - did);

	// Process the data
	for (long i = 0; i < byteToProcess; i++) {
		if (raw[i] == key) {
			const auto idx = atomicAdd(count, (gdf_size_type)1);
			updatePosition(positions, idx, did + offset + i, key);
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set
 * and sums up the number of occurrences.
 *
 * Does not load the entire buffer into the GPU memory at any time, so it can 
 * be used with buffers of any size.
 *
 * @param[in] h_data Pointer to the data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in] keys Vector containing the keys to count in the buffer
 * @param[out] count Total number of occurrences of all keys
 *
 * @return gdf_error
 *---------------------------------------------------------------------------**/
gdf_size_type countAllFromSet(const char *h_data, size_t h_size, std::vector<char> keys)
{
	char* d_chunk = nullptr;
	RMM_TRY(RMM_ALLOC (&d_chunk, min(max_chunk_bytes, h_size), 0));
	device_ptr<char> chunk_data(d_chunk);
	
	gdf_size_type*	d_count;
	RMM_TRY(RMM_ALLOC((void**)&d_count, sizeof(gdf_size_type), 0) );
	device_ptr<gdf_size_type> count_data(d_count);
	CUDA_TRY(hipMemsetAsync(d_count, 0ull, sizeof(gdf_size_type)));
 
	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, countAndSetPositions<void>) );
 
	const size_t chunk_count = (h_size + max_chunk_bytes - 1) / max_chunk_bytes;
	for (size_t ci = 0; ci < chunk_count; ++ci) {	
		const auto chunk_offset = ci * max_chunk_bytes;	
		const auto h_chunk = h_data + chunk_offset;
		const auto chunk_bytes = std::min((size_t)(h_size - ci * max_chunk_bytes), max_chunk_bytes);
		const auto chunk_bits = (chunk_bytes + bytes_per_find_thread - 1) / bytes_per_find_thread;
		const int gridSize = (chunk_bits + blockSize - 1) / blockSize;
 
		// Copy chunk to device
		CUDA_TRY(hipMemcpyAsync(d_chunk, h_chunk, chunk_bytes, hipMemcpyDefault));
 
		for (char key: keys) {
		 countAndSetPositions<void> <<< gridSize, blockSize >>> (
				d_chunk, chunk_bytes, 0, key,
				d_count, nullptr);
		}
	}
 
	gdf_size_type h_count = 0;
	CUDA_TRY(hipMemcpy(&h_count, d_count, sizeof(gdf_size_type), hipMemcpyDefault));


	return h_count;
 }

/**---------------------------------------------------------------------------*
 * @brief For each of the characters in the input set, find and saves all 
 * occurrences in the input host array of characters.
 * The positions are stored in the output device array.
 * 
 * Does not load the entire file into the GPU memory at any time, so it can 
 * be used to parse large files. Output array needs to be preallocated.
 * 
 * @param[in] h_data Pointer to the input character array
 * @param[in] h_size Number of bytes in the input array
 * @param[in] keys Vector containing the keys to count in the buffer
 * @param[in] result_offset Offset to add to the output positions
 * @param[out] positions Array containing the output positions
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
template<class T>
gdf_error findAllFromSet(const char *h_data, size_t h_size, std::vector<char> keys, uint64_t result_offset,
	T *positions) {

	char* d_chunk = nullptr;
	RMM_TRY(RMM_ALLOC (&d_chunk, min(max_chunk_bytes, h_size), 0)); 
	
	gdf_size_type*	d_count;
	RMM_TRY(RMM_ALLOC((void**)&d_count, sizeof(gdf_size_type), 0) );
	CUDA_TRY(hipMemsetAsync(d_count, 0ull, sizeof(gdf_size_type)));

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, countAndSetPositions<T>) );

	const size_t chunk_count = (h_size + max_chunk_bytes - 1) / max_chunk_bytes;
	for (size_t ci = 0; ci < chunk_count; ++ci) {	
		const auto chunk_offset = ci * max_chunk_bytes;	
		const auto h_chunk = h_data + chunk_offset;
		const auto chunk_bytes = std::min((size_t)(h_size - ci * max_chunk_bytes), max_chunk_bytes);
		const auto chunk_bits = (chunk_bytes + bytes_per_find_thread - 1) / bytes_per_find_thread;
		const int gridSize = (chunk_bits + blockSize - 1) / blockSize;

		// Copy chunk to device
		CUDA_TRY(hipMemcpyAsync(d_chunk, h_chunk, chunk_bytes, hipMemcpyDefault));

		for (char key: keys) {
			countAndSetPositions<T> <<< gridSize, blockSize >>> (
				d_chunk, chunk_bytes, chunk_offset + result_offset, key,
				d_count, positions);
		}
	}

	gdf_size_type h_count = 0;
	CUDA_TRY(hipMemcpy(&h_count, d_count, sizeof(gdf_size_type), hipMemcpyDefault));
	thrust::sort(rmm::exec_policy()->on(0), positions, positions + h_count);

	RMM_TRY(RMM_FREE(d_count, 0)); 
	RMM_TRY(RMM_FREE(d_chunk, 0));

	CUDA_TRY(hipGetLastError());

	return GDF_SUCCESS;
}

template gdf_error findAllFromSet<uint64_t>(const char *h_data, size_t h_size, std::vector<char> keys, uint64_t result_offset,
	uint64_t *positions);

template gdf_error findAllFromSet<thrust::pair<uint64_t,char>>(const char *h_data, size_t h_size, std::vector<char> keys, uint64_t result_offset,
	thrust::pair<uint64_t,char> *positions);

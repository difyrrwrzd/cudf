#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file parsing_utils.cu Utility functions for parsing plain-text files
 *
 */


#include "parsing_utils.cuh"

#include <hip/hip_runtime.h>

#include <vector>
#include <memory>
#include <iostream>

#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "utilities/error_utils.hpp"

// When processing the input in chunks, this is the maximum size of each chunk.
// Only one chunk is loaded on the GPU at a time, so this value is chosen to
// be small enough to fit on the GPU in most cases.
constexpr size_t max_chunk_bytes = 256*1024*1024; // 256MB

constexpr int bytes_per_find_thread = 64;

using pos_key_pair = thrust::pair<uint64_t,char>;

template <typename T>
struct rmm_deleter {
 void operator()(T *ptr) { RMM_FREE(ptr, 0); }
};
template <typename T>
using device_ptr = std::unique_ptr<T, rmm_deleter<T>>;

/**---------------------------------------------------------------------------*
 * @brief Sets the specified element of the array to the passed value
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(T* array, gdf_size_type idx, const T& t, const V& v){
	array[idx] = t;
}

/**---------------------------------------------------------------------------*
 * @brief Sets the specified element of the array of pairs using the two passed
 * parameters.
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(thrust::pair<T, V>* array, gdf_size_type idx, const T& t, const V& v) {
	array[idx] = {t, v};
}

/**---------------------------------------------------------------------------*
 * @brief Overloads the setElement() functions for void* arrays.
 * Does not do anything, indexing is not allowed with void* arrays.
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(void* array, gdf_size_type idx, const T& t, const V& v) {
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that finds all occurrences of a character in the given 
 * character array. If the 'positions' parameter is not void*,
 * positions of all occurrences are stored in the output array.
 * 
 * @param[in] data Pointer to the input character array
 * @param[in] size Number of bytes in the input array
 * @param[in] offset Offset to add to the output positions
 * @param[in] key Character to find in the array
 * @param[in,out] count Pointer to the number of found occurrences
 * @param[out] positions Array containing the output positions
 * 
 * @return void
 *---------------------------------------------------------------------------**/
template<class T>
 __global__ 
 void countAndSetPositions(char *data, uint64_t size, uint64_t offset, const char key, gdf_size_type* count,
	T* positions) {

	// thread IDs range per block, so also need the block id
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t did = tid * bytes_per_find_thread;
	
	const char *raw = (data + did);

	const long byteToProcess = ((did + bytes_per_find_thread) < size) ?
									bytes_per_find_thread :
									(size - did);

	// Process the data
	for (long i = 0; i < byteToProcess; i++) {
		if (raw[i] == key) {
			const auto idx = atomicAdd(count, (gdf_size_type)1);
			setElement(positions, idx, did + offset + i, key);
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set.
 * Sums up the number of occurrences. If the 'positions' parameter is not void*,
 * positions of all occurrences are stored in the output device array.
 * 
 * Does not load the entire file into the GPU memory at any time, so it can 
 * be used to parse large files. Output array needs to be preallocated.
 * 
 * @param[in] h_data Pointer to the input character array
 * @param[in] h_size Number of bytes in the input array
 * @param[in] keys Vector containing the keys to count in the buffer
 * @param[in] result_offset Offset to add to the output positions
 * @param[out] positions Array containing the output positions
 * 
 * @return gdf_size_type total number of occurrences
 *---------------------------------------------------------------------------**/
template<class T>
gdf_size_type findAllFromSet(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	T *positions) {

	char* d_chunk = nullptr;
	RMM_TRY(RMM_ALLOC (&d_chunk, min(max_chunk_bytes, h_size), 0));
	device_ptr<char> chunk_deleter(d_chunk);

	gdf_size_type*	d_count;
	RMM_TRY(RMM_ALLOC((void**)&d_count, sizeof(gdf_size_type), 0) );
	device_ptr<gdf_size_type> count_deleter(d_count);
	CUDA_TRY(hipMemsetAsync(d_count, 0ull, sizeof(gdf_size_type)));

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, countAndSetPositions<T>) );

	const size_t chunk_count = (h_size + max_chunk_bytes - 1) / max_chunk_bytes;
	for (size_t ci = 0; ci < chunk_count; ++ci) {	
		const auto chunk_offset = ci * max_chunk_bytes;	
		const auto h_chunk = h_data + chunk_offset;
		const auto chunk_bytes = std::min((size_t)(h_size - ci * max_chunk_bytes), max_chunk_bytes);
		const auto chunk_bits = (chunk_bytes + bytes_per_find_thread - 1) / bytes_per_find_thread;
		const int gridSize = (chunk_bits + blockSize - 1) / blockSize;

		// Copy chunk to device
		CUDA_TRY(hipMemcpyAsync(d_chunk, h_chunk, chunk_bytes, hipMemcpyDefault));

		for (char key: keys) {
			countAndSetPositions<T> <<< gridSize, blockSize >>> (
				d_chunk, chunk_bytes, chunk_offset + result_offset, key,
				d_count, positions);
		}
	}

	gdf_size_type h_count = 0;
	CUDA_TRY(hipMemcpy(&h_count, d_count, sizeof(gdf_size_type), hipMemcpyDefault));
	return h_count;
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set
 * and sums up the number of occurrences.
 *
 * Does not load the entire buffer into the GPU memory at any time, so it can 
 * be used with buffers of any size.
 *
 * @param[in] h_data Pointer to the data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in] keys Vector containing the keys to count in the buffer
 *
 * @return gdf_size_type total number of occurrences
 *---------------------------------------------------------------------------**/
gdf_size_type countAllFromSet(const char *h_data, size_t h_size, const std::vector<char>& keys) {
	return findAllFromSet<void>(h_data, h_size, keys, 0, nullptr);
 }

template gdf_size_type findAllFromSet<uint64_t>(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	uint64_t *positions);

template gdf_size_type findAllFromSet<pos_key_pair>(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	pos_key_pair *positions);

struct BlockSumArray {
		int16_t* d_sums = nullptr;
		uint64_t length;
		uint64_t block_size;
		BlockSumArray(uint64_t len, uint64_t bsize): length(len), block_size(bsize){}
};

class BlockSumPyramid {
	const uint16_t aggregation_rate = 32;
	std::vector<BlockSumArray> levels;
	BlockSumArray* d_levels;

public:
	BlockSumPyramid(int count){
		levels.emplace_back(count/aggregation_rate, aggregation_rate);
		RMM_ALLOC(&levels.back().d_sums, levels.back().length*sizeof(BlockSumArray), 0);

		while (levels.back().length >= aggregation_rate) {
			const auto& prev_level = levels.back();
			levels.emplace_back(prev_level.length/aggregation_rate, prev_level.block_size*aggregation_rate);
			RMM_ALLOC(&levels.back().d_sums, levels.back().length*sizeof(BlockSumArray), 0);
		}
	
		RMM_ALLOC(&d_levels, levels.size()*sizeof(BlockSumArray), 0);
		hipMemcpyAsync(d_levels, levels.data(), levels.size()*sizeof(BlockSumArray), hipMemcpyDefault);
	}

	auto operator[](int lvl) const {return levels[lvl];}
	auto deviceGetLevels() const noexcept {return d_levels;}
	size_t getHeigth() const noexcept {return levels.size();}
	constexpr int getAggregationRate() const noexcept {return aggregation_rate;}

	~BlockSumPyramid() {
		for (auto& lvl: levels) {
			RMM_FREE(lvl.d_sums, 0);
		}
		RMM_FREE(d_levels, 0);
	}
};


__global__
void sumBracketsKernel(
	pos_key_pair* brackets, int bracket_count,
	char open_bracket, char closed_bracket,
	BlockSumArray sum_array) {
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t did = tid * sum_array.block_size;


	if (tid >= sum_array.length)
		return;

	auto* start = brackets + did;
	int16_t csum = 0;
	for (int i = 0; i < sum_array.block_size; ++i) {
		if ((start + i)->second == open_bracket) ++csum;
		if ((start + i)->second == closed_bracket) --csum;
	}
	sum_array.d_sums[tid] = csum;
}

void sumBrackets(
	pos_key_pair* brackets, int bracket_count,
	char open_bracket, char closed_bracket,
	const BlockSumArray& sum_array) {
	int blockSize;
	int minGridSize;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
		sumBracketsKernel));

	// Calculate actual block count to use based on records count
	int gridSize = (sum_array.length + blockSize - 1) / blockSize;

	sumBracketsKernel<<<gridSize, blockSize>>>(brackets, bracket_count, open_bracket, closed_bracket, sum_array);

	CUDA_TRY(hipGetLastError());
};

__global__
void aggregateSumKernel(BlockSumArray in, BlockSumArray aggregate){
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const int aggregate_group_size = aggregate.block_size / in.block_size;
	const uint64_t did = tid * aggregate_group_size;

	if (tid >= aggregate.length)
		return;

	int16_t sum = 0;
	for (int i = did; i < did + aggregate_group_size; ++i)
		sum += in.d_sums[i];

	aggregate.d_sums[tid] = sum;
}

void aggregateSum(const BlockSumArray& in, const BlockSumArray& aggregate){
	int blockSize;
	int minGridSize;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
		aggregateSumKernel));

	// Calculate actual block count to use based on records count
	int gridSize = (aggregate.length + blockSize - 1) / blockSize;

	aggregateSumKernel<<<gridSize, blockSize>>>(in, aggregate);

	CUDA_TRY(hipGetLastError());
};

__global__
void assignLevelsKernel(
	pos_key_pair* brackets, uint64_t count,
	BlockSumArray* sum_pyramid, int pyramid_height,
	char open_bracket, char closed_bracket,
	int16_t* lvls) {
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const auto aggregation_rate = sum_pyramid[0].block_size;
	const uint64_t did = tid * aggregation_rate;

	if (did >= count)
		return;

	// find the previous sum
	int lvl = pyramid_height - 1;
	int sum = 0;
	int block_idx = 0;
	int offset = did;
	while(offset) {
		while(offset < sum_pyramid[lvl].block_size && lvl > 0) {
			--lvl; block_idx *= aggregation_rate;
		}
		while(offset >= sum_pyramid[lvl].block_size) {
			offset -= sum_pyramid[lvl].block_size;
			sum += sum_pyramid[lvl].d_sums[block_idx];
			++block_idx;
		}
	}

	for (int i = did; i < min(did + aggregation_rate, count); ++i){
		if (brackets[i].second == open_bracket)
			lvls[i] = ++sum;
		else if (brackets[i].second == closed_bracket)
			lvls[i] = sum--;
	}
}

void assignLevels(pos_key_pair* brackets, uint64_t count,
	const BlockSumPyramid& sum_pyramid,
	char open_bracket, char closed_bracket,
	int16_t* lvls) {
	int blockSize;
	int minGridSize;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
		assignLevelsKernel));

	// Calculate actual block count to use based on records count
	const int threadCnt = (count + sum_pyramid.getAggregationRate() - 1) / sum_pyramid.getAggregationRate();
	const int gridSize = (threadCnt + blockSize - 1) / blockSize;

	assignLevelsKernel<<<gridSize, blockSize>>>(
		brackets, count,
		sum_pyramid.deviceGetLevels(), sum_pyramid.getHeigth(),
		open_bracket, closed_bracket,
		lvls);

	CUDA_TRY(hipGetLastError());
};


// return a unique_ptr, once they are merged
int16_t* getBracketLevels(pos_key_pair* brackets, int count, char open_bracket, char closed_bracket){
	// Probably should be done outside of this function
	thrust::sort(rmm::exec_policy()->on(0), brackets, brackets + count);

	// total level difference for each segment of brackets in the file
	BlockSumPyramid aggregated_sums(count);
	
	// aggregate sums
	sumBrackets(brackets, count, open_bracket, closed_bracket, aggregated_sums[0]);
	for (size_t level_idx = 1; level_idx < aggregated_sums.getHeigth(); ++level_idx)
		aggregateSum(aggregated_sums[level_idx - 1], aggregated_sums[level_idx]);

	// assign levels
	int16_t* d_levels = nullptr;
	RMM_ALLOC(&d_levels, sizeof(int16_t) * count, 0);
	assignLevels(brackets, count, aggregated_sums, open_bracket, closed_bracket, d_levels);

	return d_levels;
}
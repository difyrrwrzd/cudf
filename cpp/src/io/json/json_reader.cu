#include "hip/hip_runtime.h"
#include "json_reader.hpp"

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>
#include <memory>

#include <stdio.h>
#include <stdlib.h>

#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>

#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <thrust/host_vector.h>

#include "cudf.h"
#include "utilities/error_utils.hpp"
#include "utilities/trie.cuh"
#include "utilities/type_dispatcher.hpp"
#include "utilities/cudf_utils.h" 

#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "io/comp/io_uncomp.h"

#include "io/utilities/parsing_utils.cuh"
#include "io/utilities/wrapper_utils.hpp"

using string_pair = std::pair<const char*,size_t>;

gdf_error read_json(json_read_arg *args) {
  JsonReader reader(args);
  // TODO validate arguments

  reader.parse();

  reader.storeColumns(args);

  return GDF_SUCCESS;
}

/*
 * Convert dtype strings into gdf_dtype enum
 */
gdf_dtype convertStringToDtype(const std::string &dtype) {
  if (dtype.compare( "str") == 0) return GDF_STRING;
  if (dtype.compare( "date") == 0) return GDF_DATE64;
  if (dtype.compare( "date32") == 0) return GDF_DATE32;
  if (dtype.compare( "date64") == 0) return GDF_DATE64;
  if (dtype.compare( "timestamp") == 0) return GDF_TIMESTAMP;
  if (dtype.compare( "category") == 0) return GDF_CATEGORY;
  if (dtype.compare( "float") == 0) return GDF_FLOAT32;
  if (dtype.compare( "float32") == 0) return GDF_FLOAT32;
  if (dtype.compare( "float64") == 0) return GDF_FLOAT64;
  if (dtype.compare( "double") == 0) return GDF_FLOAT64;
  if (dtype.compare( "short") == 0) return GDF_INT16;
  if (dtype.compare( "int") == 0) return GDF_INT32;
  if (dtype.compare( "int32") == 0) return GDF_INT32;
  if (dtype.compare( "int64") == 0) return GDF_INT64;
  if (dtype.compare( "long") == 0) return GDF_INT64;
  return GDF_invalid;
}

void JsonReader::parse(){
  // no file input and compression support for now
  h_uncomp_data_ = args_->source;
  h_uncomp_size_ = strlen(h_uncomp_data_);

  // Currently, ignoring lineterminations within quotes is handled by recording
  // the records of both, and then filtering out the records that is a quotechar
  // or a linetermination within a quotechar pair.
  rec_starts_ = filterNewlines(enumerateNewlinesAndQuotes());

  uploadDataToDevice();
  
  // TODO Determine column names - only when lines are objects
  for (int col = 0; col < args_->num_cols; ++col) {
    column_names_.emplace_back(std::to_string(col));
  }

  determineDataTypes();

  // Allocate columns
  for (size_t col = 0; col < dtypes_.size(); ++col) {
    columns_.emplace_back(rec_starts_.size(), dtypes_[col], gdf_dtype_extra_info{TIME_UNIT_NONE}, column_names_[col]);
    CUDF_EXPECTS(columns_.back().allocate() == GDF_SUCCESS, "Cannot allocate columns");
  }

  convertDataToColumns();
}

device_buffer<uint64_t> JsonReader::enumerateNewlinesAndQuotes() {
  std::vector<char> chars_to_count{'\n'};
  if (allow_newlines_in_strings_) {
    chars_to_count.push_back('\"');
  }
  auto count = countAllFromSet(h_uncomp_data_, h_uncomp_size_, chars_to_count);
  // If not starting at an offset, add an extra row to account for the first row in the file
  if (byte_range_offset_ == 0) {
    ++count;
  }

  // Allocate space to hold the record starting points
  device_buffer<uint64_t> rec_starts(count); 
  auto* find_result_ptr = rec_starts.data();
  if (byte_range_offset_ == 0) {
    find_result_ptr++;
    CUDA_TRY(hipMemsetAsync(rec_starts.data(), 0ull, sizeof(uint64_t)));
  }

  std::vector<char> chars_to_find{'\n'};
  if (allow_newlines_in_strings_) {
    chars_to_find.push_back('\"');
  }
  // Passing offset = 1 to return positions AFTER the found character
  findAllFromSet(h_uncomp_data_, h_uncomp_size_, chars_to_find, 1, find_result_ptr);

  // Previous call stores the record pinput_file.typeositions as encountered by all threads
  // Sort the record positions as subsequent processing may require filtering
  // certain rows or other processing on specific records
  thrust::sort(rmm::exec_policy()->on(0), rec_starts.data(), rec_starts.data() + count);

  return std::move(rec_starts);
}

device_buffer<uint64_t> JsonReader::filterNewlines(device_buffer<uint64_t> newlines_and_quotes) {
  const int prefilter_count = newlines_and_quotes.size();
  auto filtered_count = prefilter_count;

  if (allow_newlines_in_strings_) {
    std::vector<uint64_t> h_rec_starts(prefilter_count);
    const size_t prefilter_size = sizeof(uint64_t) * (prefilter_count);
    CUDA_TRY(hipMemcpy(h_rec_starts.data(), newlines_and_quotes.data(), prefilter_size, hipMemcpyDeviceToHost));
    for (auto elem: h_rec_starts)
      std::cout << elem << ' ';
    std::cout << '\n';

    bool quotation = false;
    for (gdf_size_type i = 1; i < prefilter_count; ++i) {
      if (h_uncomp_data_[h_rec_starts[i] - 1] == '\"') {
        quotation = !quotation;
        h_rec_starts[i] = h_uncomp_size_;
        filtered_count--;
      }
      else if (quotation) {
        h_rec_starts[i] = h_uncomp_size_;
        filtered_count--;
      }
    }

    CUDA_TRY(hipMemcpy(newlines_and_quotes.data(), h_rec_starts.data(), prefilter_count, hipMemcpyHostToDevice));
    thrust::sort(rmm::exec_policy()->on(0), newlines_and_quotes.data(), newlines_and_quotes.data() + prefilter_count);
  }
  if (h_uncomp_data_[h_uncomp_size_ - 1] == '\n') {
    filtered_count--;
  }

  newlines_and_quotes.resize(filtered_count);
  
  return newlines_and_quotes;
}

void JsonReader::uploadDataToDevice() {
  CUDF_EXPECTS(rec_starts_.size() > 0, "No data to process");
  size_t start_offset = 0;
  size_t bytes_to_upload = h_uncomp_size_;

  // Trim lines that are outside range
  if (byte_range_size_ != 0) {
    std::vector<uint64_t> h_rec_starts(rec_starts_.size());
    CUDA_TRY(hipMemcpy(h_rec_starts.data(), rec_starts_.data(),
                        sizeof(uint64_t) * h_rec_starts.size(),
                        hipMemcpyDefault));

    auto it = h_rec_starts.end() - 1;
    while (it >= h_rec_starts.begin() && *it > byte_range_size_) {
      --it;
    }
    const auto end_offset = *(it + 1);
    h_rec_starts.erase(it + 1, h_rec_starts.end());

    start_offset = h_rec_starts.front();
    bytes_to_upload = end_offset - start_offset;
    CUDF_EXPECTS(bytes_to_upload <= h_uncomp_size_,
      "Error finding the record within the specified byte range.");

    // Resize to exclude rows outside of the range; adjust row start positions to account for the data subcopy
    rec_starts_.resize(h_rec_starts.size());
    thrust::transform(rmm::exec_policy()->on(0), rec_starts_.data(),
                      rec_starts_.data() + rec_starts_.size(),
                      thrust::make_constant_iterator(start_offset),
                      rec_starts_.data(), thrust::minus<uint64_t>());
  }

  // Upload the raw data that is within the rows of interest
  d_uncomp_data_ = device_buffer<char>(bytes_to_upload);
  CUDA_TRY(hipMemcpy(d_uncomp_data_.data(), h_uncomp_data_ + start_offset,
                      bytes_to_upload, hipMemcpyHostToDevice));
}

void JsonReader::convertDataToColumns(){
  const auto num_columns = columns_.size();

  thrust::host_vector<gdf_dtype> h_dtypes(num_columns);
  thrust::host_vector<void*> h_data(num_columns);
  thrust::host_vector<gdf_valid_type*> h_valid(num_columns);

  for (size_t i = 0; i < num_columns; ++i) {
    h_dtypes[i] = columns_[i]->dtype;
    h_data[i] = columns_[i]->data;
    h_valid[i] = columns_[i]->valid;
  }

  rmm::device_vector<gdf_dtype> d_dtypes = h_dtypes;
  rmm::device_vector<void*> d_data = h_data;
  rmm::device_vector<gdf_valid_type*> d_valid = h_valid;
  rmm::device_vector<gdf_size_type> d_valid_counts(num_columns, 0);

  convertJsonToColumns(d_dtypes.data().get(), d_data.data().get(),
                       d_valid.data().get(), d_valid_counts.data().get());
  CUDA_TRY(hipDeviceSynchronize());
  CUDA_TRY(hipGetLastError());

  thrust::host_vector<gdf_size_type> h_valid_counts = d_valid_counts;
  for (size_t i = 0; i < num_columns; ++i) {
    columns_[i]->null_count = columns_[i]->size - h_valid_counts[i];
  }

  // Handle string columns
  for (size_t i = 0; i < num_columns; ++i) {
    if (columns_[i]->dtype == GDF_STRING) {
      auto str_list = static_cast<string_pair *>(columns_[i]->data);
      auto str_data = NVStrings::create_from_index(str_list, columns_[i]->size);
      RMM_FREE(std::exchange(columns_[i]->data, str_data), 0);
    }
  }
}

void JsonReader::storeColumns(json_read_arg *out_args){

  // Transfer ownership to raw pointer output arguments
  out_args->data = (gdf_column **)malloc(sizeof(gdf_column *) * columns_.size());
  for (size_t i = 0; i < columns_.size(); ++i) {
    out_args->data[i] = columns_[i].release();
  }
  out_args->num_cols_out = columns_.size();
  out_args->num_rows_out = rec_starts_.size();
}

/**---------------------------------------------------------------------------*
 * @brief Functor for converting plain text data to cuDF data type value.
 *---------------------------------------------------------------------------**/
struct ConvertFunctor {
  /**---------------------------------------------------------------------------*
   * @brief Default template operator() dispatch
   *---------------------------------------------------------------------------**/
  template <typename T>
  __host__ __device__ __forceinline__ void operator()(
      const char *data, void *gdf_columns, long row, long start,
      long end, const ParseOptions &opts) {
    T &value{static_cast<T *>(gdf_columns)[row]};
    value = convertStrToValue<T>(data, start, end, opts);
  }
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel iterates over the data until the end of the current field
 * 
 * Also iterates over (one or more) delimiter characters after the field.
 *
 * @param[in] data The entire plain text data to read
 * @param[in] opts A set of parsing options
 * @param[in] pos Offset to start the seeking from 
 * @param[in] stop Offset of the end of the row
 *
 * @return long position of the last character in the field, including the 
 *  delimiter(s) folloing the field data
 *---------------------------------------------------------------------------**/
__inline__ __device__ 
long seekFieldEnd(const char *data, const ParseOptions opts, long pos, long stop) {
  bool quotation  = false;
  while(true){
    // Use simple logic to ignore control chars between any quote seq
    // Handles nominal cases including doublequotes within quotes, but
    // may not output exact failures as PANDAS for malformed fields
    if(data[pos] == opts.quotechar){
      quotation = !quotation;
    }
    else if(quotation==false){
      if(data[pos] == opts.delimiter){
        while (opts.multi_delimiter &&
             pos < stop &&
             data[pos + 1] == opts.delimiter) {
          ++pos;
        }
        break;
      }
      else if(data[pos] == opts.terminator){
        break;
      }
      else if(data[pos] == '\r' && (pos + 1 < stop && data[pos + 1] == '\n')){
        stop--;
        break;
      }
    }
    if(pos>=stop)
      break;
    pos++;
  }
  return pos;
}

// TODO move to a common location instead of duplicating the code
__inline__ __device__ long whichBitmap(long record) { return (record/8);  }
__inline__ __device__ int whichBit(long record) { return (record % 8);  }

__inline__ __device__ void validAtomicOR(gdf_valid_type* address, gdf_valid_type val)
{
  int32_t *base_address = (int32_t*)((gdf_valid_type*)address - ((size_t)address & 3));
  int32_t int_val = (int32_t)val << (((size_t) address & 3) * 8);

  atomicOr(base_address, int_val);
}

__inline__ __device__ void setBit(gdf_valid_type* address, int bit) {
  gdf_valid_type bitMask[8]     = {1, 2, 4, 8, 16, 32, 64, 128};
  validAtomicOR(address, bitMask[bit]);
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts plain text data into cuDF column data.
 * 
 * Data is processed one record at a time
 *
 * @param[in] data The entire data to read
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] rec_starts The start of each data record
 * @param[in] num_records The number of lines/rows
 * @param[in] dtypes The data type of each column
 * @param[in] opts A set of parsing options
 * @param[out] gdf_columns The output column data
 * @param[in] num_columns The number of columns
 * @param[out] valid_fields The bitmaps indicating whether column fields are valid
 * @param[out] num_valid_fields The numbers of valid fields in columns
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
__global__ void convertJsonToGdf(char * const data, size_t data_size,
                                uint64_t * const rec_starts, gdf_size_type num_records,
                                gdf_dtype * const dtypes, ParseOptions opts,
                                void ** gdf_columns, int num_columns, 
                                gdf_valid_type **valid_fields, gdf_size_type *num_valid_fields) {
  const long  rec_id  = threadIdx.x + (blockDim.x * blockIdx.x);
  if ( rec_id >= num_records)
    return;

  long start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  long stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  // Adjust for brackets
  while(data[start++] != '[');
  while(data[--stop] != ']');

  for (int col = 0; col < num_columns; col++){

    if(start >= stop)
      return;

    // field_end is at the next delimiter/newline
    const long field_end = seekFieldEnd(data, opts, start, stop);
    long field_data_last = field_end - 1;
    // Modify start & end to ignore whitespace and quotechars
    adjustForWhitespaceAndQuotes(data, &start, &field_data_last, opts.quotechar);
    // Empty fields are not legal values
    if(start <= field_data_last) {
      // Type dispatcher does not handle GDF_STRINGS
      if (dtypes[col] == gdf_dtype::GDF_STRING) {
        auto str_list = static_cast<string_pair*>(gdf_columns[col]);
        str_list[rec_id].first = data + start;
        str_list[rec_id].second = field_data_last - start + 1;
      } else {
        cudf::type_dispatcher(
          dtypes[col], ConvertFunctor{}, data,
          gdf_columns[col], rec_id, start, field_data_last, opts);
      }

      // set the valid bitmap - all bits were set to 0 to start
      long bitmapIdx = whichBitmap(rec_id);
      long bitIdx = whichBit(rec_id);
      setBit(valid_fields[col] + bitmapIdx, bitIdx);
      atomicAdd(&num_valid_fields[col], 1);
    }
    else if(dtypes[col] == gdf_dtype::GDF_STRING){
      auto str_list = static_cast<string_pair*>(gdf_columns[col]);
      str_list[rec_id].first = nullptr;
      str_list[rec_id].second = 0;
    }
    start = field_end + 1;
  }
}

/**---------------------------------------------------------------------------*
 * @brief Helper function to setup and launch JSON parsing CUDA kernel.
 * 
 * @param[in] dtypes The data type of each column
 * @param[out] gdf_columns The output column data
 * @param[out] valid_fields The bitmaps indicating whether column fields are valid
 * @param[out] num_valid_fields The numbers of valid fields in columns
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
void JsonReader::convertJsonToColumns(gdf_dtype * const dtypes,
                                      void **gdf_columns,
                                      gdf_valid_type **valid_fields, gdf_size_type *num_valid_fields) {
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, convertJsonToGdf));

  const int grid_size = (rec_starts_.size() + block_size - 1)/block_size;

  convertJsonToGdf <<< grid_size, block_size >>> (
    d_uncomp_data_.data(), d_uncomp_data_.size(),
    rec_starts_.data(), rec_starts_.size(),
    dtypes, opts_,
    gdf_columns, columns_.size(),
    valid_fields, num_valid_fields);

  CUDA_TRY(hipGetLastError());
}

/**
* @brief Returns true is the input character is a valid digit.
* Supports both decimal and hexadecimal digits (uppercase and lowercase).
*/
__device__ __forceinline__
bool isDigit(char c, bool is_hex){
  if (c >= '0' && c <= '9') return true;
  if (is_hex) {
    if (c >= 'A' && c <= 'F') return true;
    if (c >= 'a' && c <= 'f') return true;
  }
  return false;
}

/**
* @brief Returns true if the counters indicate a potentially valid float.
* False positives are possible because positions are not taken into account.
* For example, field "e.123-" would match the pattern.
*/
__device__ __forceinline__
bool isLikeFloat(long len, long digit_cnt, long decimal_cnt, long dash_cnt, long exponent_cnt) {
  // Can't have more than one exponent and one decimal point
  if (decimal_cnt > 1) return false;
  if (exponent_cnt > 1) return false;
  // Without the exponent or a decimal point, this is an integer, not a float
  if (decimal_cnt == 0 && exponent_cnt == 0) return false;

  // Can only have one '-' per component
  if (dash_cnt > 1 + exponent_cnt) return false;

  // If anything other than these characters is present, it's not a float
  if (digit_cnt + decimal_cnt + dash_cnt + exponent_cnt != len) return false;

  // Needs at least 1 digit, 2 if exponent is present
  if (digit_cnt < 1 + exponent_cnt) return false;

  return true;
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts data into cuDF column data.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param[in] data The entire plain text data to read
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] opts A set of parsing options
 * @param[in] num_columns The number of columns of input data
 * @param[in] rec_starts The start the input data of interest
 * @param[in] num_records The number of lines/rows of input data
 * @param[out] column_infos The count for each column data type
 *
 * @returns GDF_SUCCESS upon successful computation
 *---------------------------------------------------------------------------**/
__global__
void detectJsonDataTypes(char *data, size_t data_size,
                         const ParseOptions opts, int num_columns,
                         uint64_t *rec_starts, gdf_size_type num_records,
                         JsonReader::ColumnInfo *column_infos)
{
  long  rec_id  = threadIdx.x + (blockDim.x * blockIdx.x); 
  if ( rec_id >= num_records)
    return;

  long start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  long stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  // Adjust for brackets
  while(data[start++] != '[');
  while(data[--stop] != ']');

  for (int col = 0; col < num_columns; col++) {
    const long field_end = seekFieldEnd(data, opts, start, stop);
    long field_data_last = field_end - 1;
    adjustForWhitespaceAndQuotes(data, &start, &field_data_last);

    // Checking if the field is empty
    if(start > field_data_last){
      atomicAdd(&column_infos[col].null_count, 1);
      start = field_end + 1;
      continue;
    }

    int digit_count = 0;
    int decimal_count = 0;
    int slash_count = 0;
    int dash_count = 0;
    int colon_count = 0;
    int exponent_count = 0;
    int other_count = 0;

    const int field_len = field_data_last - start + 1;
    const bool maybe_hex = ((field_len > 2 && data[start] == '0' && data[start + 1] == 'x') ||
      (field_len > 3 && data[start] == '-' && data[start + 1] == '0' && data[start + 2] == 'x'));
    for(long pos = start; pos <= field_data_last; pos++){
      if(isDigit(data[pos], maybe_hex)){
        digit_count++;
        continue;
      }
      // Looking for unique characters that will help identify column types
      switch (data[pos]){
        case '.':
          decimal_count++; break;
        case '-':
          dash_count++; break;
        case '/':
          slash_count++; break;
        case ':':
          colon_count++; break;
        case 'e':
        case 'E':
          if (!maybe_hex && pos > start && pos < field_data_last) 
            exponent_count++; break;
        default:
        other_count++; break;
      }
    }

    // Integers have to have the length of the string
    int int_req_number_cnt = field_len;
    // Off by one if they start with a minus sign
    if(data[start] == '-' && field_len > 1){
      --int_req_number_cnt;
    }
    // Off by one if they are a hexadecimal number
    if(maybe_hex) {
      --int_req_number_cnt;
    }
    if(digit_count == int_req_number_cnt){
        atomicAdd(&column_infos[col].int_count, 1);
    }
    else if(isLikeFloat(field_len, digit_count, decimal_count, dash_count, exponent_count)){
        atomicAdd(&column_infos[col].float_count, 1);
    }
    // A date-time field cannot have more than 3 non-special characters
    // A number field cannot have more than one decimal point
    else if(other_count > 3 || decimal_count > 1){
      atomicAdd(&column_infos[col].string_count, 1);
    }
    else {
      // A date field can have either one or two '-' or '\'; A legal combination will only have one of them
      // To simplify the process of auto column detection, we are not covering all the date-time formation permutations
      if((dash_count > 0 && dash_count <= 2 && slash_count == 0) || 
         (dash_count == 0 && slash_count > 0 && slash_count <= 2)){
        if(colon_count <= 2){
          atomicAdd(&column_infos[col].datetime_count, 1);
        }
        else{
          atomicAdd(&column_infos[col].string_count, 1);
        }
      }
      else{
        // Default field type is string
        atomicAdd(&column_infos[col].string_count, 1);
      }
    }
    start = field_end + 1;
  }
}

/**---------------------------------------------------------------------------*
 * @brief Set up and launches JSON data type detect CUDA kernel.
 * 
 * @param[out] column_infos The count for each column data type
 *
 * @return void
 *---------------------------------------------------------------------------**/
void JsonReader::detectDataTypes(ColumnInfo *column_infos) {
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, detectJsonDataTypes));

  // Calculate actual block count to use based on records count
  const int grid_size = (rec_starts_.size() + block_size - 1) / block_size;

  detectJsonDataTypes <<< grid_size, block_size >>> (
      d_uncomp_data_.data(), d_uncomp_data_.size(),
      opts_, column_names_.size(),
      rec_starts_.data(), rec_starts_.size(),
      column_infos);

  CUDA_TRY(hipGetLastError());
}

/**---------------------------------------------------------------------------*
 * @brief Set the data type array data member
 * 
 * If user does not pass the data types, deduces types from the file content
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void JsonReader::setDataTypes() {
  if (args_->dtype != nullptr) {
    CUDF_EXPECTS(args_->num_cols != 0, "Number of columns must be greated than zero.");
    for (int col = 0; col < args_->num_cols; ++col) {
      dtypes_.push_back(convertStringToDtype(args_->dtype[col]));
    }
  }
  else {
    CUDF_EXPECTS(rec_starts_.size() != 0, "No data available for data type inference");
    const auto num_columns = column_names_.size();

    rmm::device_vector<ColumnInfo> d_column_infos(num_columns, ColumnInfo{});
    detectDataTypes(d_column_infos.data().get());
    thrust::host_vector<ColumnInfo> h_column_infos = d_column_infos;

    for(const auto& cinfo: h_column_infos){
      CUDF_EXPECTS(cinfo.null_count == 0, "All fields must contain valid objects");

      if(cinfo.string_count > 0){
        dtypes_.push_back(GDF_STRING);
      } else if(cinfo.datetime_count > 0){
        dtypes_.push_back(GDF_DATE64);
      } else if(cinfo.float_count > 0) {
        dtypes_.push_back(GDF_FLOAT64);
      } else if(cinfo.int_count > 0) {
        dtypes_.push_back(GDF_INT64);
      }
      else {
        CUDF_FAIL("Data type detection failed");
      }
    }
  }
}
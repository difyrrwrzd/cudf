#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "json_common.h"
#include "json_gpu.h"

#include <thrust/find.h>
#include <rmm/device_buffer.hpp>

#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/trie.cuh>

#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <cudf/lists/list_view.cuh>
#include <cudf/strings/string_view.cuh>

#include <io/csv/datetime.cuh>
#include <io/utilities/parsing_utils.cuh>

namespace cudf {
namespace io {
namespace json {
namespace gpu {
using namespace ::cudf;

using string_pair = std::pair<const char *, size_t>;

namespace {
/**
 * @brief CUDA Kernel that modifies the start and stop offsets to exclude
 * the sections outside of the top level brackets.
 *
 * The top level brackets characters are excluded from the resulting range.
 * Parameter stop has the same semantics as end() in STL containers
 * (one past the last element)
 *
 * @param[in] data Pointer to the device buffer containing the data to process
 * @param[in,out] start Offset of the first character in the range
 * @param[in,out] stop Offset of the first character after the range
 *
 */
__device__ std::pair<char const *, char const *> limit_range_to_brackets(char const *begin,
                                                                         char const *end)
{
  while (begin < end && *begin != '[' && *begin != '{') { begin++; }
  begin++;

  while (begin < end && *(end - 1) != ']' && *(end - 1) != '}') { end--; }
  end--;
  return {begin, end};
}

/**
 * @brief Computes the JSON object key hash and moves the start offset past the key.
 *
 * @param[in] data Pointer to the device buffer containing the data to process
 * @param[in] quotechar TODO
 * @param[in,out] start Offset of the first character in the range. The offset is updated to the
 * first character after the key.
 * @param[in] stop Offset of the first character after the range
 *
 * @return uint32_t Hash value of the key; zero if parsing failed
 */
__device__ std::pair<char const *, char const *> parse_next_key(const char *begin,
                                                                const char *end,
                                                                char quotechar)
{
  // Key string starts after the first quote
  auto const key_begin = thrust::find(thrust::seq, begin, end, quotechar) + 1;

  // Key ends after the next unescaped quote
  auto prev_ch       = ' ';
  auto const key_end = thrust::find_if(thrust::seq, key_begin, end, [&] __device__(auto ch) {
    auto res = (ch == quotechar && prev_ch != '\\');
    prev_ch  = ch;
    return res;
  });

  return {key_begin, key_end};
}  // namespace

/**
 * @brief Decodes a numeric value base on templated cudf type T with specified
 * base.
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed numeric value
 */
template <typename T, int base>
__inline__ __device__ T
decode_value(const char *data, uint64_t start, uint64_t end, ParseOptions const &opts)
{
  return cudf::io::gpu::parse_numeric<T, base>(data, start, end, opts);
}

/**
 * @brief Decodes a numeric value base on templated cudf type T
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed numeric value
 */
template <typename T>
__inline__ __device__ T
decode_value(const char *data, uint64_t start, uint64_t end, ParseOptions const &opts)
{
  return cudf::io::gpu::parse_numeric<T>(data, start, end, opts);
}

/**
 * @brief Decodes a timestamp_D
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_D
 */
template <>
__inline__ __device__ cudf::timestamp_D decode_value(const char *data,
                                                     uint64_t start,
                                                     uint64_t end,
                                                     ParseOptions const &opts)
{
  return parseDateFormat(data, start, end, opts.dayfirst);
}

/**
 * @brief Decodes a timestamp_s
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_s
 */
template <>
__inline__ __device__ cudf::timestamp_s decode_value(const char *data,
                                                     uint64_t start,
                                                     uint64_t end,
                                                     ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli / 1000;
}

/**
 * @brief Decodes a timestamp_ms
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_ms
 */
template <>
__inline__ __device__ cudf::timestamp_ms decode_value(const char *data,
                                                      uint64_t start,
                                                      uint64_t end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli;
}

/**
 * @brief Decodes a timestamp_us
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_us
 */
template <>
__inline__ __device__ cudf::timestamp_us decode_value(const char *data,
                                                      uint64_t start,
                                                      uint64_t end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli * 1000;
}

/**
 * @brief Decodes a timestamp_ns
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_ns
 */
template <>
__inline__ __device__ cudf::timestamp_ns decode_value(const char *data,
                                                      uint64_t start,
                                                      uint64_t end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli * 1000000;
}

// The purpose of this is merely to allow compilation ONLY
// TODO : make this work for json
#ifndef DURATION_DECODE_VALUE
#define DURATION_DECODE_VALUE(Type)                                           \
  template <>                                                                 \
  __inline__ __device__ Type decode_value(                                    \
    const char *data, uint64_t start, uint64_t end, ParseOptions const &opts) \
  {                                                                           \
    return Type{};                                                            \
  }
#endif
DURATION_DECODE_VALUE(duration_D)
DURATION_DECODE_VALUE(duration_s)
DURATION_DECODE_VALUE(duration_ms)
DURATION_DECODE_VALUE(duration_us)
DURATION_DECODE_VALUE(duration_ns)

// The purpose of these is merely to allow compilation ONLY
template <>
__inline__ __device__ cudf::string_view decode_value(const char *data,
                                                     uint64_t start,
                                                     uint64_t end,
                                                     ParseOptions const &opts)
{
  return cudf::string_view{};
}
template <>
__inline__ __device__ cudf::dictionary32 decode_value(const char *data,
                                                      uint64_t start,
                                                      uint64_t end,
                                                      ParseOptions const &opts)
{
  return cudf::dictionary32{};
}
template <>
__inline__ __device__ cudf::list_view decode_value(const char *data,
                                                   uint64_t start,
                                                   uint64_t end,
                                                   ParseOptions const &opts)
{
  return cudf::list_view{};
}

/**
 * @brief Functor for converting plain text data to cuDF data type value.
 */
struct ConvertFunctor {
  /**
   * @brief Template specialization for operator() for types whose values can be
   * convertible to a 0 or 1 to represent false/true. The converting is done by
   * checking against the default and user-specified true/false values list.
   *
   * It is handled here rather than within convertStrToValue() as that function
   * is used by other types (ex. timestamp) that aren't 'booleable'.
   */
  template <typename T, typename std::enable_if_t<std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(char const *begin,
                                                      char const *end,
                                                      void *output_columns,
                                                      cudf::size_type row,
                                                      const ParseOptions &opts)
  {
    T &value{static_cast<T *>(output_columns)[row]};

    // Check for user-specified true/false values first, where the output is
    // replaced with 1/0 respectively
    const size_t field_len = end - begin;
    if (serializedTrieContains(opts.trueValuesTrie, begin, field_len)) {
      value = 1;
    } else if (serializedTrieContains(opts.falseValuesTrie, begin, field_len)) {
      value = 0;
    } else {
      value = decode_value<T>(begin, 0, field_len - 1, opts);  // TODO: refactor this too
    }

    return true;
  }

  /**
   * @brief Dispatch for floating points, which are set to NaN if the input
   * is not valid. In such case, the validity mask is set to zero too.
   */
  template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(
    char const *begin, char const *end, void *out_buffer, size_t row, ParseOptions const &opts)
  {
    auto &value{static_cast<T *>(out_buffer)[row]};
    value = decode_value<T>(begin, 0, end - begin - 1, opts);
    return !std::isnan(value);
  }

  /**
   * @brief Default template operator() dispatch specialization all data types
   * (including wrapper types) that is not covered by above.
   */
  template <typename T,
            typename std::enable_if_t<!std::is_floating_point<T>::value and
                                      !std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(char const *begin,
                                                      char const *end,
                                                      void *output_columns,
                                                      cudf::size_type row,
                                                      const ParseOptions &opts)
  {
    T &value{static_cast<T *>(output_columns)[row]};
    value = decode_value<T>(begin, 0, end - begin - 1, opts);

    return true;
  }
};

/**
 * @brief Checks whether the given character is a whitespace character.
 *
 * @param[in] ch The character to check
 *
 * @return True if the input is whitespace, False otherwise
 */
__inline__ __device__ bool is_whitespace(char ch) { return ch == '\t' || ch == ' '; }

/**
 * @brief Scans a character stream within a range, and adjusts the start and end
 * indices of the range to ignore whitespace and quotation characters.
 *
 * @param[in] start TODO
 * @param[in] end TODO
 * @param[in] quotechar The character used to denote quotes
 *
 * @return std::pair<char const *, char const *>
 */
__inline__ __device__ std::pair<char const *, char const *> trim_whitespaces_quotes(
  char const *begin, char const *end, char quotechar = '\0')
{
  auto first = begin;
  auto last  = end - 1;
  while ((first < last) && is_whitespace(*first)) { first++; }
  if ((first < last) && *first == quotechar) { first++; }
  while ((first <= last) && is_whitespace(*last)) { last--; }
  if ((first <= last) && *last == quotechar) { last--; }
  return {first, last + 1};
}

/**
 * @brief Returns true is the input character is a valid digit.
 * Supports both decimal and hexadecimal digits (uppercase and lowercase).
 *
 * @param c Character to check
 * @param is_hex Whether to check as a hexadecimal
 *
 * @return `true` if it is digit-like, `false` otherwise
 */
__device__ __inline__ bool is_digit(char c, bool is_hex = false)
{
  if (c >= '0' && c <= '9') return true;

  if (is_hex) {
    if (c >= 'A' && c <= 'F') return true;
    if (c >= 'a' && c <= 'f') return true;
  }

  return false;
}

/**
 * @brief Returns true if the counters indicate a potentially valid float.
 * False positives are possible because positions are not taken into account.
 * For example, field "e.123-" would match the pattern.
 */
__device__ __inline__ bool is_like_float(
  long len, long digit_cnt, long decimal_cnt, long dash_cnt, long exponent_cnt)
{
  // Can't have more than one exponent and one decimal point
  if (decimal_cnt > 1) return false;
  if (exponent_cnt > 1) return false;
  // Without the exponent or a decimal point, this is an integer, not a float
  if (decimal_cnt == 0 && exponent_cnt == 0) return false;

  // Can only have one '-' per component
  if (dash_cnt > 1 + exponent_cnt) return false;

  // If anything other than these characters is present, it's not a float
  if (digit_cnt + decimal_cnt + dash_cnt + exponent_cnt != len) return false;

  // Needs at least 1 digit, 2 if exponent is present
  if (digit_cnt < 1 + exponent_cnt) return false;

  return true;
}

struct field_descriptor {
  cudf::size_type column;
  char const *value_begin;
  char const *value_end;
};

__device__ field_descriptor next_field_descriptor(const char *begin,
                                                  const char *end,
                                                  ParseOptions const &opts,
                                                  cudf::size_type field_idx,
                                                  col_map_type *col_map)
{
  auto const desc_pre_trim =
    !col_map
      // No key - column and begin are trivial
      ? field_descriptor{field_idx, begin, cudf::io::gpu::seek_field_end(begin, end, opts)}
      : [&]() {
          auto const key_range = parse_next_key(begin, end, opts.quotechar);
          auto const key_hash  = MurmurHash3_32<cudf::string_view>{}(
            cudf::string_view(key_range.first, key_range.second - key_range.first));
          auto const hash_col = col_map->find(key_hash);
          // fall back to field index if not found (parsing error)
          auto const column = (hash_col != col_map->end()) ? (*hash_col).second : field_idx;

          // Skip the colon between the key and the value
          auto const value_begin = thrust::find(thrust::seq, key_range.second, end, ':') + 1;
          return field_descriptor{
            column, value_begin, cudf::io::gpu::seek_field_end(value_begin, end, opts)};
        }();

  // Modify start & end to ignore whitespace and quotechars
  auto const trimmed_value_range =
    trim_whitespaces_quotes(desc_pre_trim.value_begin, desc_pre_trim.value_end, opts.quotechar);
  return {desc_pre_trim.column, trimmed_value_range.first, trimmed_value_range.second};
}  // namespace

/**
 * @brief CUDA kernel that parses and converts plain text data into cuDF column data.
 *
 * Data is processed one record at a time
 *
 * @param[in] data The entire data to read
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] rec_starts The start of each data record
 * @param[in] num_records The number of lines/rows
 * @param[in] dtypes The data type of each column
 * @param[in] opts A set of parsing options
 * @param[in] col_map Pointer to the (column name hash -> solumn index) map in device memory
 * @param[out] output_columns The output column data
 * @param[in] num_columns The number of columns
 * @param[out] valid_fields The bitmaps indicating whether column fields are valid
 * @param[out] num_valid_fields The numbers of valid fields in columns
 *
 */
__global__ void convert_data_to_columns_kernel(const char *data,
                                               size_t data_size,
                                               const uint64_t *rec_starts,
                                               cudf::size_type num_records,
                                               const data_type *dtypes,
                                               ParseOptions opts,
                                               col_map_type *col_map,
                                               void *const *output_columns,
                                               cudf::size_type num_columns,
                                               bitmask_type *const *valid_fields,
                                               cudf::size_type *num_valid_fields)
{
  const auto rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records) return;

  auto const row_data_range = [&]() {
    auto const row_begin = data + rec_starts[rec_id];
    auto const row_end   = data + ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);
    return limit_range_to_brackets(row_begin, row_end);
  }();

  auto current = row_data_range.first;
  for (int input_field_index = 0;
       input_field_index < num_columns && current < row_data_range.second;
       input_field_index++) {
    auto const desc =
      next_field_descriptor(current, row_data_range.second, opts, input_field_index, col_map);
    auto const value_len = desc.value_end - desc.value_begin;

    current = desc.value_end + 1;

    // Empty fields are not legal values
    if (value_len > 0 && !serializedTrieContains(opts.naValuesTrie, desc.value_begin, value_len)) {
      // Type dispatcher does not handle strings
      if (dtypes[desc.column].id() == type_id::STRING) {
        auto str_list           = static_cast<string_pair *>(output_columns[desc.column]);
        str_list[rec_id].first  = desc.value_begin;
        str_list[rec_id].second = value_len;

        // set the valid bitmap - all bits were set to 0 to start
        set_bit(valid_fields[desc.column], rec_id);
        atomicAdd(&num_valid_fields[desc.column], 1);
      } else {
        if (cudf::type_dispatcher(dtypes[desc.column],
                                  ConvertFunctor{},
                                  desc.value_begin,
                                  desc.value_end,
                                  output_columns[desc.column],
                                  rec_id,
                                  opts)) {
          // set the valid bitmap - all bits were set to 0 to start
          set_bit(valid_fields[desc.column], rec_id);
          atomicAdd(&num_valid_fields[desc.column], 1);
        }
      }
    } else if (dtypes[desc.column].id() == type_id::STRING) {
      auto str_list           = static_cast<string_pair *>(output_columns[desc.column]);
      str_list[rec_id].first  = nullptr;
      str_list[rec_id].second = 0;
    }
  }
}

/**
 * @brief CUDA kernel that processes a buffer of data and determines information about the
 * column types within.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param[in] data Input data buffer
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] opts A set of parsing options
 * @param[in] col_map Pointer to the (column name hash -> solumn index) map in device memory
 * @param[in] num_columns The number of columns of input data
 * @param[in] rec_starts The offset of each row in the input
 * @param[in] num_records The number of rows
 * @param[out] column_infos The count for each column data type
 *
 */
__global__ void detect_data_types_kernel(const char *data,
                                         size_t data_size,
                                         const ParseOptions opts,
                                         col_map_type *col_map,
                                         int num_columns,
                                         const uint64_t *rec_starts,
                                         cudf::size_type num_records,
                                         ColumnInfo *column_infos)
{
  auto const rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records) return;

  auto const are_rows_objects = col_map != nullptr;
  auto const row_data_range   = [&]() {
    auto const row_begin = data + rec_starts[rec_id];
    auto const row_end   = data + ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);
    return limit_range_to_brackets(row_begin, row_end);
  }();

  int input_field_index = 0;
  for (auto current = row_data_range.first;
       input_field_index < num_columns && current < row_data_range.second;
       input_field_index++) {
    auto const desc =
      next_field_descriptor(current, row_data_range.second, opts, input_field_index, col_map);
    auto const value_len = desc.value_end - desc.value_begin;

    // Advance to the next field; +1 to skip the delimiter
    current = desc.value_end + 1;

    // Checking if the field is empty/valid
    if (value_len <= 0 || serializedTrieContains(opts.naValuesTrie, desc.value_begin, value_len)) {
      // Increase the null count for array rows, where the null count is initialized to zero.
      if (!are_rows_objects) { atomicAdd(&column_infos[desc.column].null_count, 1); }
      continue;
    } else if (are_rows_objects) {
      // For files with object rows, null count is initialized to row count. The value is decreased
      // here for every valid field.
      atomicAdd(&column_infos[desc.column].null_count, -1);
    }
    // Don't need counts to detect strings, any field in quotes is deduced to be a string
    if (*(desc.value_begin - 1) == opts.quotechar && *desc.value_end == opts.quotechar) {
      atomicAdd(&column_infos[desc.column].string_count, 1);
      continue;
    }

    int digit_count    = 0;
    int decimal_count  = 0;
    int slash_count    = 0;
    int dash_count     = 0;
    int colon_count    = 0;
    int exponent_count = 0;
    int other_count    = 0;

    const bool maybe_hex =
      ((value_len > 2 && *desc.value_begin == '0' && *(desc.value_begin + 1) == 'x') ||
       (value_len > 3 && *desc.value_begin == '-' && *(desc.value_begin + 1) == '0' &&
        *(desc.value_begin + 2) == 'x'));
    for (auto pos = desc.value_begin; pos < desc.value_end; ++pos) {
      if (is_digit(*pos, maybe_hex)) {
        digit_count++;
        continue;
      }
      // Looking for unique characters that will help identify column types
      switch (*pos) {
        case '.': decimal_count++; break;
        case '-': dash_count++; break;
        case '/': slash_count++; break;
        case ':': colon_count++; break;
        case 'e':
        case 'E':
          if (!maybe_hex && pos > desc.value_begin && pos < desc.value_end - 1) exponent_count++;
          break;
        default: other_count++; break;
      }
    }

    // Integers have to have the length of the string
    int int_req_number_cnt = value_len;
    // Off by one if they start with a minus sign
    if (*desc.value_begin == '-' && value_len > 1) { --int_req_number_cnt; }
    // Off by one if they are a hexadecimal number
    if (maybe_hex) { --int_req_number_cnt; }
    if (serializedTrieContains(opts.trueValuesTrie, desc.value_begin, value_len) ||
        serializedTrieContains(opts.falseValuesTrie, desc.value_begin, value_len)) {
      atomicAdd(&column_infos[desc.column].bool_count, 1);
    } else if (digit_count == int_req_number_cnt) {
      atomicAdd(&column_infos[desc.column].int_count, 1);
    } else if (is_like_float(value_len, digit_count, decimal_count, dash_count, exponent_count)) {
      atomicAdd(&column_infos[desc.column].float_count, 1);
    }
    // A date-time field cannot have more than 3 non-special characters
    // A number field cannot have more than one decimal point
    else if (other_count > 3 || decimal_count > 1) {
      atomicAdd(&column_infos[desc.column].string_count, 1);
    } else {
      // A date field can have either one or two '-' or '\'; A legal combination will only have one
      // of them To simplify the process of auto column detection, we are not covering all the
      // date-time formation permutations
      if ((dash_count > 0 && dash_count <= 2 && slash_count == 0) ||
          (dash_count == 0 && slash_count > 0 && slash_count <= 2)) {
        if (colon_count <= 2) {
          atomicAdd(&column_infos[desc.column].datetime_count, 1);
        } else {
          atomicAdd(&column_infos[desc.column].string_count, 1);
        }
      } else {
        // Default field type is string
        atomicAdd(&column_infos[desc.column].string_count, 1);
      }
    }
  }
  if (!are_rows_objects) {
    // For array rows, mark missing fields as null
    for (; input_field_index < num_columns; ++input_field_index)
      atomicAdd(&column_infos[input_field_index].null_count, 1);
  }
}

/**
 * @brief Enumerator for states when parsing JSON object keys.
 */
enum class key_parse_state { PRE_KEY, KEY, POST_KEY, POST_KEY_QUOTE };

/**
 * @brief Cuda kernel that collects information about JSON object keys in the file.
 *
 * @param[in] data Input data buffer
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] opts A set of parsing options
 * @param[in] rec_starts The offset of each row in the input
 * @param[in] num_records The number of rows
 * @param[out] keys_cnt Number of found keys in the file
 * @param[out] keys_info Information (offset, length, hash) for each found key
 *
 */
__global__ void collect_keys_info_kernel(const char *data,
                                         size_t data_size,
                                         const ParseOptions opts,
                                         const uint64_t *rec_starts,
                                         cudf::size_type num_records,
                                         unsigned long long int *keys_cnt,
                                         mutable_table_device_view *keys_info)
{
  auto const rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records) return;

  auto const start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  auto const stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  auto st              = key_parse_state::PRE_KEY;
  auto last_name_start = start;
  for (auto pos = start; pos < stop; ++pos) {
    if (st == key_parse_state::PRE_KEY && data[pos] == opts.quotechar) {
      st              = key_parse_state::KEY;
      last_name_start = pos + 1;
    } else if (st == key_parse_state::KEY && data[pos] == opts.quotechar && data[pos - 1] != '\\') {
      st       = key_parse_state::POST_KEY;
      auto idx = atomicAdd(keys_cnt, 1);
      if (nullptr != keys_info) {
        auto len                                    = pos - last_name_start;
        keys_info->column(0).element<uint64_t>(idx) = last_name_start;
        keys_info->column(1).element<uint16_t>(idx) = len;
        keys_info->column(2).element<uint32_t>(idx) =
          MurmurHash3_32<cudf::string_view>{}(cudf::string_view(data + last_name_start, len));
      }
    } else if (st == key_parse_state::POST_KEY && data[pos] == opts.quotechar) {
      st = key_parse_state::POST_KEY_QUOTE;
    } else if (st == key_parse_state::POST_KEY_QUOTE && data[pos] == opts.quotechar &&
               data[pos - 1] != '\\') {
      st = key_parse_state::POST_KEY;
    } else if (st == key_parse_state::POST_KEY && data[pos] == opts.delimiter) {
      st = key_parse_state::PRE_KEY;
    }
  }
}

}  // namespace

/**
 * @copydoc cudf::io::json::gpu::convert_json_to_columns
 *
 */
void convert_json_to_columns(rmm::device_buffer const &input_data,
                             data_type *const dtypes,
                             void *const *output_columns,
                             cudf::size_type num_records,
                             cudf::size_type num_columns,
                             const uint64_t *rec_starts,
                             bitmask_type *const *valid_fields,
                             cudf::size_type *num_valid_fields,
                             ParseOptions const &opts,
                             col_map_type *col_map,
                             hipStream_t stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, convert_data_to_columns_kernel));

  const int grid_size = (num_records + block_size - 1) / block_size;

  convert_data_to_columns_kernel<<<grid_size, block_size, 0, stream>>>(
    static_cast<const char *>(input_data.data()),
    input_data.size(),
    rec_starts,
    num_records,
    dtypes,
    opts,
    col_map,
    output_columns,
    num_columns,
    valid_fields,
    num_valid_fields);

  CUDA_TRY(hipGetLastError());
}

/**
 * @copydoc cudf::io::json::gpu::detect_data_types
 *
 */
void detect_data_types(ColumnInfo *column_infos,
                       const char *data,
                       size_t data_size,
                       const ParseOptions &options,
                       col_map_type *col_map,
                       int num_columns,
                       const uint64_t *rec_starts,
                       cudf::size_type num_records,
                       hipStream_t stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, detect_data_types_kernel));

  // Calculate actual block count to use based on records count
  const int grid_size = (num_records + block_size - 1) / block_size;

  detect_data_types_kernel<<<grid_size, block_size, 0, stream>>>(
    data, data_size, options, col_map, num_columns, rec_starts, num_records, column_infos);

  CUDA_TRY(hipGetLastError());
}

/**
 * @copydoc cudf::io::json::gpu::gpu_collect_keys_info
 */
void collect_keys_info(const char *data,
                       size_t data_size,
                       const ParseOptions &options,
                       const uint64_t *rec_starts,
                       cudf::size_type num_records,
                       unsigned long long int *keys_cnt,
                       mutable_table_device_view *keys_info,
                       hipStream_t stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, collect_keys_info_kernel));

  // Calculate actual block count to use based on records count
  const int grid_size = (num_records + block_size - 1) / block_size;

  collect_keys_info_kernel<<<grid_size, block_size, 0, stream>>>(
    data, data_size, options, rec_starts, num_records, keys_cnt, keys_info);

  CUDA_TRY(hipGetLastError());
}

}  // namespace gpu
}  // namespace json
}  // namespace io
}  // namespace cudf

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "json_common.h"
#include "json_gpu.h"

#include <rmm/device_buffer.hpp>

#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/trie.cuh>

#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <cudf/lists/list_view.cuh>
#include <cudf/strings/string_view.cuh>

#include <io/csv/datetime.cuh>
#include <io/utilities/parsing_utils.cuh>

namespace cudf {
namespace io {
namespace json {
namespace gpu {
using namespace ::cudf;

using string_pair = std::pair<const char *, size_t>;

namespace {
/**
 * @brief CUDA Kernel that modifies the start and stop offsets to exclude
 * the sections outside of the top level brackets.
 *
 * The top level brackets characters are excluded from the resulting range.
 * Parameter stop has the same semantics as end() in STL containers
 * (one past the last element)
 *
 * @param[in] data Pointer to the device buffer containing the data to process
 * @param[in,out] start Offset of the first character in the range
 * @param[in,out] stop Offset of the first character after the range
 *
 * @return void
 **/
__device__ void limit_range_to_brackets(const char *data, long &start, long &stop)
{
  while (start < stop && data[start] != '[' && data[start] != '{') { start++; }
  start++;

  while (start < stop && data[stop - 1] != ']' && data[stop - 1] != '}') { stop--; }
  stop--;
}

enum parse_state { PRE_NAME, NAME, POST_NAME, POST_NAME_QUOTE };

/**
 * @brief CUDA kernel that finds the end position of the next field name,
 * including the colon that separates the name from the field value.
 *
 * Returns the position after the colon that precedes the value token.
 *
 * @param[in] data Pointer to the device buffer containing the data to process
 * @param[in] opts Parsing options (e.g. delimiter and quotation character)
 * @param[in,out] start Offset of the first character in the range
 * @param[in] stop Offset of the first character after the range
 *
 * @return uint32_t Hash value of the field name
 **/
__device__ uint32_t parse_field_name(const char *data,
                                     const ParseOptions opts,
                                     long &start,
                                     long stop)
{
  parse_state state = PRE_NAME;
  auto name_start   = start;
  uint32_t hash_val = 0;
  for (auto pos = start; pos < stop; ++pos) {
    if (state == PRE_NAME && data[pos] == opts.quotechar) {
      state      = NAME;
      name_start = pos + 1;
    } else if (state == NAME && data[pos] == opts.quotechar && data[pos - 1] != '\\') {
      state = POST_NAME;
      hash_val =
        MurmurHash3_32<cudf::string_view>{}(cudf::string_view(data + name_start, pos - name_start));
    } else if (state == POST_NAME && data[pos] == ':') {
      start = pos + 1;
      break;
    }
  }
  return hash_val;
}

/**
 * @brief Decodes a numeric value base on templated cudf type T with specified
 * base.
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed numeric value
 **/
template <typename T, int base>
__inline__ __device__ T
decode_value(const char *data, long start, long end, ParseOptions const &opts)
{
  return cudf::io::gpu::parse_numeric<T, base>(data, start, end, opts);
}

/**
 * @brief Decodes a numeric value base on templated cudf type T
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed numeric value
 **/
template <typename T>
__inline__ __device__ T
decode_value(const char *data, long start, long end, ParseOptions const &opts)
{
  return cudf::io::gpu::parse_numeric<T>(data, start, end, opts);
}

/**
 * @brief Decodes a timestamp_D
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_D
 **/
template <>
__inline__ __device__ cudf::timestamp_D decode_value(const char *data,
                                                     long start,
                                                     long end,
                                                     ParseOptions const &opts)
{
  return parseDateFormat(data, start, end, opts.dayfirst);
}

/**
 * @brief Decodes a timestamp_s
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_s
 **/
template <>
__inline__ __device__ cudf::timestamp_s decode_value(const char *data,
                                                     long start,
                                                     long end,
                                                     ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli / 1000;
}

/**
 * @brief Decodes a timestamp_ms
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_ms
 **/
template <>
__inline__ __device__ cudf::timestamp_ms decode_value(const char *data,
                                                      long start,
                                                      long end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli;
}

/**
 * @brief Decodes a timestamp_us
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_us
 **/
template <>
__inline__ __device__ cudf::timestamp_us decode_value(const char *data,
                                                      long start,
                                                      long end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli * 1000;
}

/**
 * @brief Decodes a timestamp_ns
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_ns
 **/
template <>
__inline__ __device__ cudf::timestamp_ns decode_value(const char *data,
                                                      long start,
                                                      long end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli * 1000000;
}

// The purpose of this is merely to allow compilation ONLY
// TODO : make this work for json
#ifndef DURATION_DECODE_VALUE
#define DURATION_DECODE_VALUE(Type)                                   \
  template <>                                                         \
  __inline__ __device__ Type decode_value(                            \
    const char *data, long start, long end, ParseOptions const &opts) \
  {                                                                   \
    return Type{};                                                    \
  }
#endif
DURATION_DECODE_VALUE(duration_D)
DURATION_DECODE_VALUE(duration_s)
DURATION_DECODE_VALUE(duration_ms)
DURATION_DECODE_VALUE(duration_us)
DURATION_DECODE_VALUE(duration_ns)

// The purpose of these is merely to allow compilation ONLY
template <>
__inline__ __device__ cudf::string_view decode_value(const char *data,
                                                     long start,
                                                     long end,
                                                     ParseOptions const &opts)
{
  return cudf::string_view{};
}
template <>
__inline__ __device__ cudf::dictionary32 decode_value(const char *data,
                                                      long start,
                                                      long end,
                                                      ParseOptions const &opts)
{
  return cudf::dictionary32{};
}
template <>
__inline__ __device__ cudf::list_view decode_value(const char *data,
                                                   long start,
                                                   long end,
                                                   ParseOptions const &opts)
{
  return cudf::list_view{};
}

/**
 * @brief Functor for converting plain text data to cuDF data type value.
 **/
struct ConvertFunctor {
  /**
   * @brief Template specialization for operator() for types whose values can be
   * convertible to a 0 or 1 to represent false/true. The converting is done by
   * checking against the default and user-specified true/false values list.
   *
   * It is handled here rather than within convertStrToValue() as that function
   * is used by other types (ex. timestamp) that aren't 'booleable'.
   **/
  template <typename T, typename std::enable_if_t<std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(const char *data,
                                                      void *output_columns,
                                                      long row,
                                                      long start,
                                                      long end,
                                                      const ParseOptions &opts)
  {
    T &value{static_cast<T *>(output_columns)[row]};

    // Check for user-specified true/false values first, where the output is
    // replaced with 1/0 respectively
    const size_t field_len = end - start + 1;
    if (serializedTrieContains(opts.trueValuesTrie, data + start, field_len)) {
      value = 1;
    } else if (serializedTrieContains(opts.falseValuesTrie, data + start, field_len)) {
      value = 0;
    } else {
      value = decode_value<T>(data, start, end, opts);
    }

    return true;
  }

  /**
   * @brief Dispatch for floating points, which are set to NaN if the input
   * is not valid. In such case, the validity mask is set to zero too.
   */
  template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(
    const char *data, void *out_buffer, size_t row, long start, long end, ParseOptions const &opts)
  {
    auto &value{static_cast<T *>(out_buffer)[row]};
    value = decode_value<T>(data, start, end, opts);
    return !std::isnan(value);
  }

  /**
   * @brief Default template operator() dispatch specialization all data types
   * (including wrapper types) that is not covered by above.
   **/
  template <typename T,
            typename std::enable_if_t<!std::is_floating_point<T>::value and
                                      !std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(const char *data,
                                                      void *output_columns,
                                                      long row,
                                                      long start,
                                                      long end,
                                                      const ParseOptions &opts)
  {
    T &value{static_cast<T *>(output_columns)[row]};
    value = decode_value<T>(data, start, end, opts);

    return true;
  }
};

/**
 * @brief Checks whether the given character is a whitespace character.
 *
 * @param[in] ch The character to check
 *
 * @return True if the input is whitespace, False otherwise
 **/
__inline__ __device__ bool is_whitespace(char ch) { return ch == '\t' || ch == ' '; }

/**
 * @brief Scans a character stream within a range, and adjusts the start and end
 * indices of the range to ignore whitespace and quotation characters.
 *
 * @param[in] data The character stream to scan
 * @param[in,out] start The start index to adjust
 * @param[in,out] end The end index to adjust
 * @param[in] quotechar The character used to denote quotes
 *
 * @return Adjusted or unchanged start_idx and end_idx
 **/
__inline__ __device__ void trim_field_start_end(const char *data,
                                                long *start,
                                                long *end,
                                                char quotechar = '\0')
{
  while ((*start < *end) && is_whitespace(data[*start])) { (*start)++; }
  if ((*start < *end) && data[*start] == quotechar) { (*start)++; }
  while ((*start <= *end) && is_whitespace(data[*end])) { (*end)--; }
  if ((*start <= *end) && data[*end] == quotechar) { (*end)--; }
}

/**
 * @brief Returns true is the input character is a valid digit.
 * Supports both decimal and hexadecimal digits (uppercase and lowercase).
 *
 * @param c Character to check
 * @param is_hex Whether to check as a hexadecimal
 *
 * @return `true` if it is digit-like, `false` otherwise
 */
__device__ __inline__ bool is_digit(char c, bool is_hex = false)
{
  if (c >= '0' && c <= '9') return true;

  if (is_hex) {
    if (c >= 'A' && c <= 'F') return true;
    if (c >= 'a' && c <= 'f') return true;
  }

  return false;
}

/**
 * @brief Returns true if the counters indicate a potentially valid float.
 * False positives are possible because positions are not taken into account.
 * For example, field "e.123-" would match the pattern.
 */
__device__ __inline__ bool is_like_float(
  long len, long digit_cnt, long decimal_cnt, long dash_cnt, long exponent_cnt)
{
  // Can't have more than one exponent and one decimal point
  if (decimal_cnt > 1) return false;
  if (exponent_cnt > 1) return false;
  // Without the exponent or a decimal point, this is an integer, not a float
  if (decimal_cnt == 0 && exponent_cnt == 0) return false;

  // Can only have one '-' per component
  if (dash_cnt > 1 + exponent_cnt) return false;

  // If anything other than these characters is present, it's not a float
  if (digit_cnt + decimal_cnt + dash_cnt + exponent_cnt != len) return false;

  // Needs at least 1 digit, 2 if exponent is present
  if (digit_cnt < 1 + exponent_cnt) return false;

  return true;
}

/**
 * @brief CUDA kernel that parses and converts plain text data into cuDF column data.
 *
 * Data is processed one record at a time
 *
 * @param[in] data The entire data to read
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] rec_starts The start of each data record
 * @param[in] num_records The number of lines/rows
 * @param[in] dtypes The data type of each column
 * @param[in] opts A set of parsing options
 * @param[out] output_columns The output column data
 * @param[in] num_columns The number of columns
 * @param[out] valid_fields The bitmaps indicating whether column fields are valid
 * @param[out] num_valid_fields The numbers of valid fields in columns
 *
 * @return void
 **/
__global__ void convert_json_to_columns_kernel(const char *data,
                                               size_t data_size,
                                               const uint64_t *rec_starts,
                                               cudf::size_type num_records,
                                               const data_type *dtypes,
                                               ParseOptions opts,
                                               void *const *output_columns,
                                               int num_columns,
                                               bitmask_type *const *valid_fields,
                                               cudf::size_type *num_valid_fields)
{
  const long rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records) return;

  long start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  long stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  limit_range_to_brackets(data, start, stop);
  const bool is_object = (data[start - 1] == '{');

  for (int col = 0; col < num_columns && start < stop; col++) {
    auto dst_col = col;
    if (is_object) { parse_field_name(data, opts, start, stop); }
    // field_end is at the next delimiter/newline
    const long field_end = cudf::io::gpu::seek_field_end(data, opts, start, stop);
    long field_data_last = field_end - 1;
    // Modify start & end to ignore whitespace and quotechars
    trim_field_start_end(data, &start, &field_data_last, opts.quotechar);
    // Empty fields are not legal values
    if (start <= field_data_last &&
        !serializedTrieContains(opts.naValuesTrie, data + start, field_end - start)) {
      // Type dispatcher does not handle strings
      if (dtypes[dst_col].id() == type_id::STRING) {
        auto str_list           = static_cast<string_pair *>(output_columns[dst_col]);
        str_list[rec_id].first  = data + start;
        str_list[rec_id].second = field_data_last - start + 1;

        // set the valid bitmap - all bits were set to 0 to start
        set_bit(valid_fields[dst_col], rec_id);
        atomicAdd(&num_valid_fields[dst_col], 1);
      } else {
        if (cudf::type_dispatcher(dtypes[dst_col],
                                  ConvertFunctor{},
                                  data,
                                  output_columns[dst_col],
                                  rec_id,
                                  start,
                                  field_data_last,
                                  opts)) {
          // set the valid bitmap - all bits were set to 0 to start
          set_bit(valid_fields[dst_col], rec_id);
          atomicAdd(&num_valid_fields[dst_col], 1);
        }
      }
    } else if (dtypes[dst_col].id() == type_id::STRING) {
      auto str_list           = static_cast<string_pair *>(output_columns[dst_col]);
      str_list[rec_id].first  = nullptr;
      str_list[rec_id].second = 0;
    }
    start = field_end + 1;
  }
}

/**
 * @brief CUDA kernel that processes a buffer of data and determines information about the
 * column types within.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param[in] data Input data buffer
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] opts A set of parsing options
 * @param[in] num_columns The number of columns of input data
 * @param[in] rec_starts The start the input data of interest
 * @param[in] num_records The number of lines/rows of input data
 * @param[out] column_infos The count for each column data type
 *
 * @returns void
 **/
__global__ void detect_json_data_types(const char *data,
                                       size_t data_size,
                                       const ParseOptions opts,
                                       int num_columns,
                                       const uint64_t *rec_starts,
                                       cudf::size_type num_records,
                                       ColumnInfo *column_infos)
{
  long rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records) return;

  long start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  long stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  limit_range_to_brackets(data, start, stop);
  const bool is_object = (data[start - 1] == '{');

  for (int col = 0; col < num_columns; col++) {
    if (is_object) { parse_field_name(data, opts, start, stop); }
    auto field_start     = start;
    const long field_end = cudf::io::gpu::seek_field_end(data, opts, field_start, stop);
    long field_data_last = field_end - 1;
    trim_field_start_end(data, &field_start, &field_data_last);
    const int field_len = field_data_last - field_start + 1;
    // Advance the start offset
    start = field_end + 1;

    // Checking if the field is empty
    if (field_start > field_data_last ||
        serializedTrieContains(opts.naValuesTrie, data + field_start, field_len)) {
      atomicAdd(&column_infos[col].null_count, 1);
      continue;
    }
    // Don't need counts to detect strings, any field in quotes is deduced to be a string
    if (data[field_start] == opts.quotechar && data[field_data_last] == opts.quotechar) {
      atomicAdd(&column_infos[col].string_count, 1);
      continue;
    }

    int digit_count    = 0;
    int decimal_count  = 0;
    int slash_count    = 0;
    int dash_count     = 0;
    int colon_count    = 0;
    int exponent_count = 0;
    int other_count    = 0;

    const bool maybe_hex =
      ((field_len > 2 && data[field_start] == '0' && data[field_start + 1] == 'x') ||
       (field_len > 3 && data[field_start] == '-' && data[field_start + 1] == '0' &&
        data[field_start + 2] == 'x'));
    for (long pos = field_start; pos <= field_data_last; pos++) {
      if (is_digit(data[pos], maybe_hex)) {
        digit_count++;
        continue;
      }
      // Looking for unique characters that will help identify column types
      switch (data[pos]) {
        case '.': decimal_count++; break;
        case '-': dash_count++; break;
        case '/': slash_count++; break;
        case ':': colon_count++; break;
        case 'e':
        case 'E':
          if (!maybe_hex && pos > field_start && pos < field_data_last) exponent_count++;
          break;
        default: other_count++; break;
      }
    }

    // Integers have to have the length of the string
    int int_req_number_cnt = field_len;
    // Off by one if they start with a minus sign
    if (data[field_start] == '-' && field_len > 1) { --int_req_number_cnt; }
    // Off by one if they are a hexadecimal number
    if (maybe_hex) { --int_req_number_cnt; }
    if (serializedTrieContains(opts.trueValuesTrie, data + field_start, field_len) ||
        serializedTrieContains(opts.falseValuesTrie, data + field_start, field_len)) {
      atomicAdd(&column_infos[col].bool_count, 1);
    } else if (digit_count == int_req_number_cnt) {
      atomicAdd(&column_infos[col].int_count, 1);
    } else if (is_like_float(field_len, digit_count, decimal_count, dash_count, exponent_count)) {
      atomicAdd(&column_infos[col].float_count, 1);
    }
    // A date-time field cannot have more than 3 non-special characters
    // A number field cannot have more than one decimal point
    else if (other_count > 3 || decimal_count > 1) {
      atomicAdd(&column_infos[col].string_count, 1);
    } else {
      // A date field can have either one or two '-' or '\'; A legal combination will only have one
      // of them To simplify the process of auto column detection, we are not covering all the
      // date-time formation permutations
      if ((dash_count > 0 && dash_count <= 2 && slash_count == 0) ||
          (dash_count == 0 && slash_count > 0 && slash_count <= 2)) {
        if (colon_count <= 2) {
          atomicAdd(&column_infos[col].datetime_count, 1);
        } else {
          atomicAdd(&column_infos[col].string_count, 1);
        }
      } else {
        // Default field type is string
        atomicAdd(&column_infos[col].string_count, 1);
      }
    }
  }
}

/**
 * @brief TODO
 *
 * @param[in] data Input data buffer
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] opts A set of parsing options
 * @param[in] rec_starts The start the input data of interest
 * @param[in] num_records The number of lines/rows of input data
 * @param[out] tag_cnt TODO
 *
 * @returns void
 **/
__global__ void collect_field_names_info_kernel(const char *data,
                                                size_t data_size,
                                                const ParseOptions opts,
                                                const uint64_t *rec_starts,
                                                cudf::size_type num_records,
                                                unsigned long long int *names_cnt,
                                                mutable_table_device_view *names_info)
{
  long rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records) return;

  auto const start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  auto const stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  parse_state st       = PRE_NAME;
  auto last_name_start = start;
  for (auto pos = start; pos < stop; ++pos) {
    if (st == PRE_NAME && data[pos] == opts.quotechar) {
      st              = NAME;
      last_name_start = pos + 1;
    } else if (st == NAME && data[pos] == opts.quotechar && data[pos - 1] != '\\') {
      st       = POST_NAME;
      auto idx = atomicAdd(names_cnt, 1);
      if (nullptr != names_info) {
        auto len                                     = pos - last_name_start;
        names_info->column(0).element<uint64_t>(idx) = last_name_start;
        names_info->column(1).element<uint16_t>(idx) = len;
        names_info->column(2).element<uint32_t>(idx) =
          MurmurHash3_32<cudf::string_view>{}(cudf::string_view(data + last_name_start, len));
      }
    } else if (st == POST_NAME && data[pos] == opts.quotechar) {
      st = POST_NAME_QUOTE;
    } else if (st == POST_NAME_QUOTE && data[pos] == opts.quotechar && data[pos - 1] != '\\') {
      st = POST_NAME;
    } else if (st == POST_NAME && data[pos] == opts.delimiter) {
      st = PRE_NAME;
    }
  }
}

}  // namespace

/**
 * @copydoc cudf::io::json::gpu::convert_json_to_columns
 *
 **/
void convert_json_to_columns(rmm::device_buffer const &input_data,
                             data_type *const dtypes,
                             void *const *output_columns,
                             cudf::size_type num_records,
                             cudf::size_type num_columns,
                             const uint64_t *rec_starts,
                             bitmask_type *const *valid_fields,
                             cudf::size_type *num_valid_fields,
                             ParseOptions const &opts,
                             hipStream_t stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, convert_json_to_columns_kernel));

  const int grid_size = (num_records + block_size - 1) / block_size;

  convert_json_to_columns_kernel<<<grid_size, block_size, 0, stream>>>(
    static_cast<const char *>(input_data.data()),
    input_data.size(),
    rec_starts,
    num_records,
    dtypes,
    opts,
    output_columns,
    num_columns,
    valid_fields,
    num_valid_fields);

  CUDA_TRY(hipGetLastError());
}

/**
 * @copydoc cudf::io::json::gpu::detect_data_types
 *
 **/
void detect_data_types(ColumnInfo *column_infos,
                       const char *data,
                       size_t data_size,
                       const ParseOptions &options,
                       int num_columns,
                       const uint64_t *rec_starts,
                       cudf::size_type num_records,
                       hipStream_t stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, detect_json_data_types));

  // Calculate actual block count to use based on records count
  const int grid_size = (num_records + block_size - 1) / block_size;

  detect_json_data_types<<<grid_size, block_size, 0, stream>>>(
    data, data_size, options, num_columns, rec_starts, num_records, column_infos);

  CUDA_TRY(hipGetLastError());
}

/**
 * @copydoc cudf::io::json::gpu::gpu_collect_field_names_info
 */
void collect_field_names_info(const char *data,
                              size_t data_size,
                              const ParseOptions &options,
                              const uint64_t *rec_starts,
                              cudf::size_type num_records,
                              unsigned long long int *names_cnt,
                              mutable_table_device_view *names_info,
                              hipStream_t stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, collect_field_names_info_kernel));

  // Calculate actual block count to use based on records count
  const int grid_size = (num_records + block_size - 1) / block_size;

  collect_field_names_info_kernel<<<grid_size, block_size, 0, stream>>>(
    data, data_size, options, rec_starts, num_records, names_cnt, names_info);

  CUDA_TRY(hipGetLastError());
}

}  // namespace gpu
}  // namespace json
}  // namespace io
}  // namespace cudf

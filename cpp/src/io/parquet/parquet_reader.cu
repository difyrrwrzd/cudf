#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudf.h"
#include "io/comp/gpuinflate.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.hpp"

#include <hip/hip_runtime.h>
#include "NVStrings.h"
#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"

#include "parquet.h"
#include "parquet_gpu.h"

#include <array>
#include <cstring>
#include <iostream>
#include <numeric>
#include <utility>
#include <vector>

#include <fcntl.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#define CUDF_TRY(call) CUDF_EXPECTS(call == GDF_SUCCESS, "Failed call")

#if 0
#define LOG_PRINTF(...) std::printf(__VA_ARGS__)
#else
#define LOG_PRINTF(...) (void)0
#endif

/**
 * @brief Helper class for memory mapping a file source
 **/
class DataSource {
 public:
  explicit DataSource(const char *filepath) {
    fd = open(filepath, O_RDONLY);
    CUDF_EXPECTS(fd > 0, "Failed file open");

    struct stat st {};
    CUDF_EXPECTS(fstat(fd, &st) == 0, "Failed file size query");

    mapped_size = st.st_size;
    CUDF_EXPECTS(mapped_size > 0, "Found zero-sized file");

    mapped_data = mmap(NULL, mapped_size, PROT_READ, MAP_PRIVATE, fd, 0);
    CUDF_EXPECTS(mapped_data != MAP_FAILED, "Failed memory mapping file");
  }

  ~DataSource() {
    if (mapped_data) {
      munmap(mapped_data, mapped_size);
    }
    if (fd) {
      close(fd);
    }
  }

  const uint8_t *data() const { return static_cast<uint8_t *>(mapped_data); }
  size_t size() const { return mapped_size; }

 private:
  void *mapped_data = nullptr;
  size_t mapped_size = 0;
  int fd = 0;
};

/**
 * @brief Function that translates Parquet datatype to GDF dtype
 **/
constexpr std::pair<gdf_dtype, gdf_dtype_extra_info> to_dtype(
    parquet::Type physical, parquet::ConvertedType logical) {
  // Logical type used for actual data interpretation; the legacy converted type
  // is superceded by 'logical' type whenever available.
  switch (logical) {
    case parquet::UINT_8:
    case parquet::INT_8:
      return std::make_pair(GDF_INT8, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::UINT_16:
    case parquet::INT_16:
      return std::make_pair(GDF_INT16, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::DATE:
      return std::make_pair(GDF_DATE32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::TIMESTAMP_MILLIS:
      return std::make_pair(GDF_DATE64, gdf_dtype_extra_info{TIME_UNIT_ms});
    case parquet::TIMESTAMP_MICROS:
      return std::make_pair(GDF_DATE64, gdf_dtype_extra_info{TIME_UNIT_us});
    default:
      break;
  }

  // Physical storage type supported by Parquet; controls the on-disk storage
  // format in combination with the encoding type.
  switch (physical) {
    case parquet::BOOLEAN:
      return std::make_pair(GDF_INT8, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::INT32:
      return std::make_pair(GDF_INT32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::INT64:
      return std::make_pair(GDF_INT64, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::FLOAT:
      return std::make_pair(GDF_FLOAT32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::DOUBLE:
      return std::make_pair(GDF_FLOAT64, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::BYTE_ARRAY:
    case parquet::FIXED_LEN_BYTE_ARRAY:
      // Can be mapped to GDF_CATEGORY (32-bit hash) or GDF_STRING (nvstring)
      return std::make_pair(GDF_CATEGORY, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case parquet::INT96:
      // deprecated, only used by legacy implementations
    default:
      break;
  }

  return std::make_pair(GDF_invalid, gdf_dtype_extra_info{TIME_UNIT_NONE});
}

/**
 * @brief Function that requires the number of bits to store a given value
 **/
template <typename T = uint8_t>
T required_bits(uint32_t max_level) {
  return static_cast<T>(parquet::CPReader::NumRequiredBits(max_level));
}

/**
 * @brief A helper class that wraps a gdf_column and any associated memory.
 *
 * This abstraction provides functionality for initializing and managing a
 * gdf_column (its fields and its memory) while still allowing direct access.
 * Any free memory is automatically deallocated unless ownership is transferred
 * via releasing and assigning the raw pointer to the underlying gdf_column.
 **/
class gdf_column_wrapper {
 public:
  gdf_column_wrapper(gdf_size_type size, gdf_dtype dtype,
                     gdf_dtype_extra_info dtype_info, const std::string name) {
    col = (gdf_column *)malloc(sizeof(gdf_column));
    col->col_name = (char *)malloc(name.length() + 1);
    strcpy(col->col_name, name.c_str());
    gdf_column_view_augmented(col, nullptr, nullptr, size, dtype, 0, dtype_info);
  }

  ~gdf_column_wrapper() {
    if (col) {
      RMM_FREE(col->data, 0);
      RMM_FREE(col->valid, 0);
      free(col->col_name);
    }
    free(col);
  };

  gdf_column_wrapper(const gdf_column_wrapper &other) = delete;
  gdf_column_wrapper(gdf_column_wrapper &&other) : col(other.col) {
    other.col = nullptr;
  }

  gdf_error allocate() {
    // For strings, just store the startpos + length for now
    const auto num_rows = std::max(col->size, 1);
    const auto column_byte_width = (col->dtype == GDF_STRING)
                                       ? sizeof(parquet::gpu::nvstrdesc_s)
                                       : gdf_dtype_size(col->dtype);

    RMM_TRY(RMM_ALLOC(&col->data, num_rows * column_byte_width, 0));
    RMM_TRY(RMM_ALLOC(&col->valid, gdf_valid_allocation_size(num_rows), 0));
    CUDA_TRY(hipMemset(col->valid, 0, gdf_valid_allocation_size(num_rows)));

    return GDF_SUCCESS;
  }

  gdf_column *operator->() const { return col; }
  gdf_column *get() const { return col; }
  gdf_column *release() {
    auto temp = col;
    col = nullptr;
    return temp;
  }

 private:
  gdf_column *col = nullptr;
};

/**
 * @brief A type-templated helper class that wraps fixed-length device memory,
 * and a complementary host pinned memory of the same size.
 *
 * This abstraction allocates a specified fixed chunk of device memory that can
 * initialized upfront, or gradually initialized as required.
 * The host-side memory can be used to manipulate data on the CPU before and
 * after operating on the same data on the GPU.
 **/
template <typename T>
class hostdevice_vector {
 public:
  using value_type = T;

  explicit hostdevice_vector(size_t initial_size, size_t max_size)
      : num_elements(initial_size), max_elements(max_size) {
    CUDA_TRY(hipHostMalloc(&h_data, sizeof(T) * max_elements));
    RMM_ALLOC(&d_data, sizeof(T) * max_elements, 0);
  }

  ~hostdevice_vector() {
    RMM_FREE(d_data, 0);
    CUDA_TRY(hipHostFree(h_data));
  }

  bool insert(const T &data) {
    if (num_elements < max_elements) {
      h_data[num_elements] = data;
      num_elements++;
      return true;
    }
    return false;
  }

  size_t max_size() const { return max_elements; }
  size_t size() const { return num_elements; }
  size_t memory_size() const { return sizeof(T) * num_elements; }

  T &operator[](size_t i) const { return h_data[i]; }
  T *host_ptr(size_t offset = 0) const { return h_data + offset; }
  T *device_ptr(size_t offset = 0) const { return d_data + offset; }

 private:
  size_t max_elements = 0;
  size_t num_elements = 0;
  T *h_data = nullptr;
  T *d_data = nullptr;
};

/**
 * @brief A unique_ptr with a custom deleter that frees the associated device
 * memory back to RMM. Used to help automatically release device memory of
 * manually allocated pointers.
 **/
template <typename T>
struct rmm_deleter {
  void operator()(T *ptr) { RMM_FREE(ptr, 0); }
};
template <typename T>
using device_ptr = std::unique_ptr<T, rmm_deleter<T>>;

/**
 * @brief A helper wrapper class for the Parquet file metadata
 **/
class ParquetMetadata : public parquet::FileMetaData {
  static std::string to_dot_string(
      std::vector<std::string> const &path_in_schema) {
    size_t n = path_in_schema.size();
    std::string s = (n > 0) ? path_in_schema[0] : "";
    for (size_t i = 1; i < n; i++) {
      s += '.';
      s += path_in_schema[i];
    }
    return s;
  }

 public:
  explicit ParquetMetadata(const uint8_t *data, size_t len) {
    constexpr auto header_len = sizeof(parquet::file_header_s);
    constexpr auto ender_len = sizeof(parquet::file_ender_s);
    const auto header = (const parquet::file_header_s *)data;
    const auto ender = (const parquet::file_ender_s *)(data + len - ender_len);
    CUDF_EXPECTS(
        data && len > header_len + ender_len,
        "Incorrect data source");
    CUDF_EXPECTS(
        header->magic == PARQUET_MAGIC && ender->magic == PARQUET_MAGIC,
        "Corrupted header or footer");
    CUDF_EXPECTS(
        ender->footer_len != 0 && ender->footer_len <= len - header_len - ender_len,
        "Incorrect footer length");

    parquet::CPReader cp;
    cp.init(data + len - ender->footer_len - ender_len, ender->footer_len);
    CUDF_EXPECTS(cp.read(this), "Cannot parse metadata");
    CUDF_EXPECTS(cp.InitSchema(this), "Cannot parse metadata schema");

    print_metadata();
  }

  inline int get_total_rows() const { return num_rows; }
  inline int get_num_rowgroups() const { return row_groups.size(); }
  inline int get_num_columns() const { return row_groups[0].columns.size(); }

  std::vector<std::string> get_column_names() {
    std::vector<std::string> col_names;
    for (auto &col : row_groups[0].columns) {
      col_names.push_back(to_dot_string(col.meta_data.path_in_schema));
    }
    return col_names;
  }
  std::string get_column_name(const std::vector<std::string> &path_in_schema) {
    return to_dot_string(path_in_schema);
  }
  std::string get_index_column_name() {
    auto it =
        std::find_if(key_value_metadata.begin(), key_value_metadata.end(),
                     [](const auto &item) { return item.key == "pandas"; });

    if (it != key_value_metadata.end()) {
      const auto pos = it->value.find("index_columns");

      if (pos != std::string::npos) {
        const auto begin = it->value.find('[', pos);
        const auto end = it->value.find(']', begin);
        if ((end - begin) > 4) {
          return it->value.substr(begin + 2, end - begin - 3);
        }
      }
    }
    return "";
  }

  void print_metadata() {
    LOG_PRINTF("\n[+] Metadata:\n");
    LOG_PRINTF(" version = %d\n", version);
    LOG_PRINTF(" created_by = \"%s\"\n", created_by.c_str());
    LOG_PRINTF(" schema (%zd entries):\n", schema.size());
    for (size_t i = 0; i < schema.size(); i++) {
      LOG_PRINTF(
          "  [%zd] type=%d, name=\"%s\", num_children=%d, rep_type=%d, "
          "max_def_lvl=%d, max_rep_lvl=%d\n",
          i, schema[i].type, schema[i].name.c_str(), schema[i].num_children,
          schema[i].repetition_type, schema[i].max_definition_level,
          schema[i].max_repetition_level);
    }
    LOG_PRINTF(" num rows = %zd\n", (size_t)num_rows);
    LOG_PRINTF(" num row groups = %zd\n", row_groups.size());
    LOG_PRINTF(" num columns = %zd\n", row_groups[0].columns.size());
  }
};

/**
 * @brief Returns the number of total pages from the given column chunks
 * 
 * @param[in] chunks List of column chunk descriptors
 * @param[in,out] total_pages Total number of pages making up the column chunks
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error count_page_headers(
    const hostdevice_vector<parquet::gpu::ColumnChunkDesc> &chunks,
    size_t *total_pages) {

  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                           chunks.memory_size(), hipMemcpyHostToDevice));
  CUDA_TRY(parquet::gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size()));
  CUDA_TRY(hipMemcpyAsync(chunks.host_ptr(), chunks.device_ptr(),
                           chunks.memory_size(), hipMemcpyDeviceToHost));
  CUDA_TRY(hipStreamSynchronize(0));

  LOG_PRINTF("[+] Chunk Information\n");
  for (size_t c = 0; c < chunks.size(); c++) {
    LOG_PRINTF(
        " %2zd: comp_data=%ld, comp_size=%zd, num_values=%zd\n     "
        "start_row=%zd num_rows=%d max_def_level=%d max_rep_level=%d\n     "
        "data_type=%d def_level_bits=%d rep_level_bits=%d\n     "
        "num_data_pages=%d num_dict_pages=%d max_num_pages=%d\n",
        c, (uint64_t)chunks[c].compressed_data, chunks[c].compressed_size,
        chunks[c].num_values, chunks[c].start_row, chunks[c].num_rows,
        chunks[c].max_def_level, chunks[c].max_rep_level, chunks[c].data_type,
        chunks[c].def_level_bits, chunks[c].rep_level_bits,
        chunks[c].num_data_pages, chunks[c].num_dict_pages,
        chunks[c].max_num_pages);
    *total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return GDF_SUCCESS;
}

/**
 * @brief Returns the page information from the given column chunks
 *
 * @param[in] chunks List of column chunk descriptors
 * @param[in] pages List of page information
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error decode_page_headers(
    const hostdevice_vector<parquet::gpu::ColumnChunkDesc> &chunks,
    const hostdevice_vector<parquet::gpu::PageInfo> &pages) {

  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    chunks[c].max_num_pages = chunks[c].num_data_pages + chunks[c].num_dict_pages;
    chunks[c].page_info = pages.device_ptr(page_count);
    page_count += chunks[c].max_num_pages;
  }

  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                           chunks.memory_size(), hipMemcpyHostToDevice));
  CUDA_TRY(parquet::gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size()));
  CUDA_TRY(hipMemcpyAsync(pages.host_ptr(), pages.device_ptr(),
                           pages.memory_size(), hipMemcpyDeviceToHost));
  CUDA_TRY(hipStreamSynchronize(0));

  LOG_PRINTF("[+] Page Header Information\n");
  for (size_t i = 0; i < pages.size(); i++) {
    LOG_PRINTF(
        " %2zd: comp_size=%d, uncomp_size=%d, num_values=%d, chunk_row=%d, "
        "num_rows=%d\n     chunk_idx=%d, flags=%d, encoding=%d, def_level=%d "
        "rep_level=%d, valid_count=%d\n",
        i, pages[i].compressed_page_size, pages[i].uncompressed_page_size,
        pages[i].num_values, pages[i].chunk_row, pages[i].num_rows,
        pages[i].chunk_idx, pages[i].flags, pages[i].encoding,
        pages[i].definition_level_encoding, pages[i].repetition_level_encoding,
        pages[i].valid_count);
  }

  return GDF_SUCCESS;
}

/**
 * @brief Decompresses the page data, at page granularity
 *
 * @param[in] chunks List of column chunk descriptors
 * @param[in] pages List of page information
 * @param[in,out] page_data List of outstanding page data device allocations
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error decompress_page_data(
    const hostdevice_vector<parquet::gpu::ColumnChunkDesc> &chunks,
    const hostdevice_vector<parquet::gpu::PageInfo> &pages,
    std::vector<device_ptr<void>> *page_data) {

  auto for_each_codec_page = [&](parquet::Compression codec,
                                 const std::function<void(size_t)> &f) {
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      const auto page_stride = chunks[c].max_num_pages;
      if (chunks[c].codec == codec) {
        for (int k = 0; k < page_stride; k++) {
          f(page_count + k);
        }
      }
      page_count += page_stride;
    }
  };

  // Brotli scratch memory for decoding
  rmm::device_vector<uint8_t> debrotli_scratch;

  // Count the exact number of compressed pages
  size_t num_compressed_pages = 0;
  size_t total_decompressed_size = 0;
  std::array<std::pair<parquet::Compression, size_t>, 3> codecs{
      std::make_pair(parquet::GZIP, 0), std::make_pair(parquet::SNAPPY, 0),
      std::make_pair(parquet::BROTLI, 0)};

  for (auto &codec : codecs) {
    for_each_codec_page(codec.first, [&](size_t page) {
      total_decompressed_size += pages[page].uncompressed_page_size;
      codec.second++;
      num_compressed_pages++;
    });
    if (codec.first == parquet::BROTLI && codec.second > 0) {
      debrotli_scratch.resize(get_gpu_debrotli_scratch_size(codec.second > 0));
    }
  }

  LOG_PRINTF(
      "[+] Compression\n Total compressed size: %zd\n Number of "
      "compressed pages: %zd\n  gzip:    %zd \n  snappy: %zd\n",
      total_decompressed_size, num_compressed_pages, codecs[0].second,
      codecs[1].second);

  // Dispatch batches of pages to decompress for each codec
  uint8_t *decompressed_pages = nullptr;
  RMM_TRY(RMM_ALLOC(&decompressed_pages, total_decompressed_size, 0));
  page_data->emplace_back(decompressed_pages);

  hostdevice_vector<gpu_inflate_input_s> inflate_in(0, num_compressed_pages);
  hostdevice_vector<gpu_inflate_status_s> inflate_out(0, num_compressed_pages);

  size_t decompressed_ofs = 0;
  int32_t argc = 0;
  for (const auto &codec : codecs) {
    if (codec.second > 0) {
      int32_t start_pos = argc;

      for_each_codec_page(codec.first, [&](size_t page) {
        inflate_in[argc].srcDevice = pages[page].page_data;
        inflate_in[argc].srcSize = pages[page].compressed_page_size;
        inflate_in[argc].dstDevice = decompressed_pages + decompressed_ofs;
        inflate_in[argc].dstSize = pages[page].uncompressed_page_size;

        inflate_out[argc].bytes_written = 0;
        inflate_out[argc].status = static_cast<uint32_t>(-1000);
        inflate_out[argc].reserved = 0;

        pages[page].page_data = (uint8_t *)inflate_in[argc].dstDevice;
        decompressed_ofs += inflate_in[argc].dstSize;
        argc++;
      });

      CUDA_TRY(hipMemcpyAsync(
          inflate_in.device_ptr(start_pos), inflate_in.host_ptr(start_pos),
          sizeof(decltype(inflate_in)::value_type) * (argc - start_pos),
          hipMemcpyHostToDevice));
      CUDA_TRY(hipMemcpyAsync(
          inflate_out.device_ptr(start_pos),
          inflate_out.host_ptr(start_pos),
          sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
          hipMemcpyHostToDevice));
      switch (codec.first) {
        case parquet::GZIP:
          CUDA_TRY(gpuinflate(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos, 1))
          break;
        case parquet::SNAPPY:
          CUDA_TRY(gpu_unsnap(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos));
          break;
        case parquet::BROTLI:
          CUDA_TRY(gpu_debrotli(inflate_in.device_ptr(start_pos),
                                inflate_out.device_ptr(start_pos),
                                debrotli_scratch.data().get(),
                                debrotli_scratch.size(), argc - start_pos));
          break;
        default:
          std::cerr << "This is a bug" << std::endl;
          break;
      }
      CUDA_TRY(hipMemcpyAsync(
          inflate_out.host_ptr(start_pos),
          inflate_out.device_ptr(start_pos),
          sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
          hipMemcpyDeviceToHost));
    }
  }
  CUDA_TRY(hipStreamSynchronize(0));

  // Update the page information in device memory with the updated value of
  // page_data; it now points to the uncompressed data buffer
  CUDA_TRY(hipMemcpyAsync(pages.device_ptr(), pages.host_ptr(),
                           pages.memory_size(), hipMemcpyHostToDevice));

  return GDF_SUCCESS;
}

/**
 * @brief Converts the page data and outputs to gdf_columns
 *
 * @param[in] chunks List of column chunk descriptors
 * @param[in] pages List of page information
 * @param[in] chunk_map Mapping between column chunk and gdf_column
 * @param[in] total_rows Total number of rows to output
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error decode_page_data(
    const hostdevice_vector<parquet::gpu::ColumnChunkDesc> &chunks,
    const hostdevice_vector<parquet::gpu::PageInfo> &pages,
    const std::vector<gdf_column *> &chunk_map, size_t total_rows) {

  auto is_dict_chunk = [](const parquet::gpu::ColumnChunkDesc &chunk) {
    return (chunk.data_type & 0x7) == parquet::BYTE_ARRAY &&
           chunk.num_dict_pages > 0;
  };

  // Count the number of string dictionary entries
  // NOTE: Assumes first page in the chunk is always the dictionary page
  size_t total_str_dict_indexes = 0;
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) {
      total_str_dict_indexes += pages[page_count].num_values;
    }
    page_count += chunks[c].max_num_pages;
  }

  // Build index for string dictionaries since they can't be indexed
  // directly due to variable-sized elements
  rmm::device_vector<parquet::gpu::nvstrdesc_s> str_dict_index;
  if (total_str_dict_indexes > 0) {
    str_dict_index.resize(total_str_dict_indexes);
  }

  // Update chunks with pointers to column data
  for (size_t c = 0, page_count = 0, str_ofs = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) {
      chunks[c].str_dict_index = str_dict_index.data().get() + str_ofs;
      str_ofs += pages[page_count].num_values;
    }
    chunks[c].valid_map_base = (uint32_t *)chunk_map[c]->valid;
    chunks[c].column_data_base = chunk_map[c]->data;
    page_count += chunks[c].max_num_pages;
  }
  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                           chunks.memory_size(), hipMemcpyHostToDevice));
  if (total_str_dict_indexes > 0) {
    CUDA_TRY(BuildStringDictionaryIndex(chunks.device_ptr(), chunks.size()));
  }
  CUDA_TRY(DecodePageData(pages.device_ptr(), pages.size(), chunks.device_ptr(),
                          chunks.size(), total_rows));
  CUDA_TRY(hipMemcpyAsync(pages.host_ptr(), pages.device_ptr(),
                           pages.memory_size(), hipMemcpyDeviceToHost));
  CUDA_TRY(hipStreamSynchronize(0));

  LOG_PRINTF("[+] Page Data Information\n");
  for (size_t i = 0; i < pages.size(); i++) {
    if (pages[i].num_rows > 0) {
      LOG_PRINTF(" %2zd: valid_count=%d/%d\n", i, pages[i].valid_count,
                 pages[i].num_rows);
      const size_t c = pages[i].chunk_idx;
      if (c < chunks.size()) {
        chunk_map[c]->null_count += pages[i].num_rows - pages[i].valid_count;
      }
    }
  }

  return GDF_SUCCESS;
}

/**
 * @brief Reads Apache Parquet data and returns an array of gdf_columns.
 *
 * @param[in,out] args Structure containing input and output args
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error read_parquet(pq_read_arg *args) {

  std::vector<gdf_column_wrapper> columns;
  int num_columns = 0;
  int num_rows = 0;
  int index_col = -1;

  DataSource input(args->source);
  const auto raw = input.data();
  const auto raw_size = input.size();

  // Init schema and metadata
  ParquetMetadata md(raw, raw_size);
  CUDF_EXPECTS(md.get_num_rowgroups() > 0, "No row groups found");
  CUDF_EXPECTS(md.get_num_columns() > 0, "No columns found");

  // Obtain the index column if available
  std::string index_col_name = md.get_index_column_name();

  // Select only columns required (if it exists), otherwise select all
  // For PANDAS behavior, always return index column unless there are no rows
  std::vector<std::pair<int, std::string>> col_names;
  if (args->use_cols) {
    std::vector<std::string> use_names(args->use_cols,
                                       args->use_cols + args->use_cols_len);
    if (md.get_total_rows() > 0) {
      use_names.push_back(index_col_name);
    }
    for (const auto &use_name : use_names) {
      size_t index = 0;
      for (const auto name : md.get_column_names()) {
        if (name == use_name) {
          col_names.emplace_back(index, name);
        }
        index++;
      }
    }
  } else {
    for (const auto& name : md.get_column_names()) {
      if (md.get_total_rows() > 0 || name != index_col_name) {
        col_names.emplace_back(col_names.size(), name);
      }
    }
  }
  CUDF_EXPECTS(not col_names.empty(), "Filterd out all columns");
  num_columns = col_names.size();

  // Initialize gdf_columns
  LOG_PRINTF("[+] Selected columns: %d\n", num_columns);
  for (const auto &name : col_names) {
    auto &col_schema = md.schema[md.row_groups[0].columns[name.first].schema_idx];
    auto dtype_info = to_dtype(col_schema.type, col_schema.converted_type);

    columns.emplace_back(static_cast<gdf_size_type>(md.get_total_rows()),
                         dtype_info.first, dtype_info.second, name.second);

    LOG_PRINTF(" %2zd: name=%s size=%zd type=%d data=%lx valid=%lx\n",
               columns.size() - 1, columns.back()->col_name,
               (size_t)columns.back()->size, columns.back()->dtype,
               (uint64_t)columns.back()->data, (uint64_t)columns.back()->valid);

    if (name.second == index_col_name) {
      index_col = columns.size() - 1;
    }
  }

  // Allocate column chunk descriptors
  const auto num_column_chunks = md.get_num_rowgroups() * num_columns;
  hostdevice_vector<parquet::gpu::ColumnChunkDesc> chunks(0, num_column_chunks);
  std::vector<gdf_column *> chunk_map(num_column_chunks);
  std::vector<device_ptr<void>> page_data;

  // Initialize column chunk info
  size_t total_decompressed_size = 0;
  LOG_PRINTF("[+] Column Chunk Description\n");
  for (const auto &rowgroup : md.row_groups) {
    for (size_t i = 0; i < col_names.size(); i++) {
      auto name = col_names[i];
      auto &col_meta = rowgroup.columns[name.first].meta_data;
      auto &col_schema = md.schema[rowgroup.columns[name.first].schema_idx];
      auto &gdf_column = columns[i];

      // Spec requires each row group to contain exactly one chunk for every
      // column. If there are too many or too few, continue with best effort
      if (name.second != md.get_column_name(col_meta.path_in_schema)) {
        std::cerr << "Detected mismatched column chunk" << std::endl;
        continue;
      }
      if (chunks.size() >= chunks.max_size()) {
        std::cerr << "Detected too many column chunks" << std::endl;
        continue;
      }

      int32_t type_width = (col_schema.type == parquet::FIXED_LEN_BYTE_ARRAY)
                               ? (col_schema.type_length << 3)
                               : 0;
      if (gdf_column->dtype == GDF_INT8)
        type_width = 1;  // I32 -> I8
      else if (gdf_column->dtype == GDF_INT16)
        type_width = 2;  // I32 -> I16
      else if (gdf_column->dtype == GDF_CATEGORY)
        type_width = 4;  // str -> hash32

      uint8_t *d_data = nullptr;
      if (col_meta.total_compressed_size != 0) {
        const auto offset = (col_meta.dictionary_page_offset != 0)
                                ? std::min(col_meta.data_page_offset,
                                           col_meta.dictionary_page_offset)
                                : col_meta.data_page_offset;
        RMM_TRY(RMM_ALLOC(&d_data, col_meta.total_compressed_size, 0));
        page_data.emplace_back(d_data);
        CUDA_TRY(hipMemcpyAsync(d_data, raw + offset,
                                 col_meta.total_compressed_size,
                                 hipMemcpyHostToDevice));
      }
      chunks.insert(parquet::gpu::ColumnChunkDesc(
          col_meta.total_compressed_size, d_data, col_meta.num_values,
          col_schema.type, type_width, num_rows, rowgroup.num_rows,
          col_schema.max_definition_level, col_schema.max_repetition_level,
          required_bits(col_schema.max_definition_level),
          required_bits(col_schema.max_repetition_level), col_meta.codec));

      LOG_PRINTF(
          " %2d: %s start_row=%d, num_rows=%ld, codec=%d, "
          "num_values=%ld\n",
          name.first, name.second.c_str(), num_rows, rowgroup.num_rows,
          col_meta.codec, col_meta.num_values);
      LOG_PRINTF("     total_compressed_size=%ld total_uncompressed_size=%ld\n",
          col_meta.total_compressed_size, col_meta.total_uncompressed_size);
      LOG_PRINTF(
          "     schema_idx=%d, type=%d, type_width=%d, max_def_level=%d, "
          "max_rep_level=%d\n",
          rowgroup.columns[name.first].schema_idx, chunks[chunks.size()-1].data_type, type_width,
          col_schema.max_definition_level, col_schema.max_repetition_level);
      LOG_PRINTF(
          "     data_page_offset=%zd, index_page_offset=%zd, "
          "dict_page_offset=%zd\n",
          (size_t)col_meta.data_page_offset,
          (size_t)col_meta.index_page_offset,
          (size_t)col_meta.dictionary_page_offset);

      // Map each column chunk to its output gdf_column
      chunk_map[chunks.size() - 1] = gdf_column.get();

      if (col_meta.codec != parquet::Compression::UNCOMPRESSED) {
        total_decompressed_size += col_meta.total_uncompressed_size;
      }
    }
    num_rows += rowgroup.num_rows;
  }

  // Determine how many page headers to allocate
  size_t total_pages = 0;
  CUDF_TRY(count_page_headers(chunks, &total_pages));

  if (total_pages > 0) {
    hostdevice_vector<parquet::gpu::PageInfo> pages(total_pages, total_pages);

    CUDF_TRY(decode_page_headers(chunks, pages));
    if (total_decompressed_size > 0) {
      CUDF_TRY(decompress_page_data(chunks, pages, &page_data));
    }
    for (auto &column : columns) {
      CUDF_TRY(column.allocate());
    }
    CUDF_TRY(decode_page_data(chunks, pages, chunk_map, num_rows));
  } else {
    // Columns are still expected to be allocated for an empty dataframe
    for (auto &column : columns) {
      CUDF_TRY(column.allocate());
    }
  }

  // Transfer ownership to raw pointer output arguments
  args->data = (gdf_column **)malloc(sizeof(gdf_column *) * num_columns);
  for (int i = 0; i < num_columns; ++i) {
    args->data[i] = columns[i].release();

    // For string dtype, allocate and return in an NvStrings container instance
    // The container takes a list of string pointers and lengths, and copies
    // into its own memory so the source memory must not be released yet
    if (args->data[i]->dtype == GDF_STRING) {
      using str_pair = std::pair<const char *, size_t>;
      static_assert(sizeof(str_pair) == sizeof(parquet::gpu::nvstrdesc_s));

      auto str_list = static_cast<str_pair *>(args->data[i]->data);
      args->data[i]->data = NVStrings::create_from_index(str_list, num_rows);
    }
  }
  args->num_cols_out = num_columns;
  args->num_rows_out = num_rows;
  if (index_col != -1) {
    args->index_col = (int *)malloc(sizeof(int));
    *args->index_col = index_col;
  }

  return GDF_SUCCESS;
}

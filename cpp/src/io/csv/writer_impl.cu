/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO CSV writer class implementation
 */

#include "writer_impl.hpp"

#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>

#include <algorithm>
#include <cstring>
#include <utility>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

namespace cudf {
namespace experimental {
namespace io {
namespace detail {
namespace csv {

void writer::impl::write(table_view const &table,
                         const table_metadata *metadata,
                         hipStream_t stream) {
  //TODO!
}

void writer::write_all(table_view const &table, const table_metadata *metadata, hipStream_t stream) {
  _impl->write(table, metadata, stream);
}



}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace experimental
}  // namespace cudf


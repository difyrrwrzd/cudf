/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO CSV writer class implementation
 */

#include "writer_impl.hpp"

#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>

#include <algorithm>
#include <cstring>
#include <utility>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

namespace cudf {
namespace experimental {
namespace io {
namespace detail {
namespace csv {

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               writer_options const& options,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mr)) {}


void writer::impl::write(table_view const &table,
                         const table_metadata *metadata,
                         hipStream_t stream) {
  //TODO!
  //chunked behavior(?)
}

void writer::write_all(table_view const &table, const table_metadata *metadata, hipStream_t stream) {
  _impl->write(table, metadata, stream);
}



}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace experimental
}  // namespace cudf


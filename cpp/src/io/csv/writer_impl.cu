/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO CSV writer class implementation
 */

#include "writer_impl.hpp"

#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/traits.hpp>

#include <cudf/strings/convert/convert_booleans.hpp>
#include <cudf/strings/convert/convert_integers.hpp>
#include <cudf/strings/convert/convert_floats.hpp>
#include <cudf/strings/convert/convert_datetime.hpp>

#include <cudf/strings/replace.hpp>


#include <algorithm>
#include <cstring>
#include <utility>
#include <type_traits>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

namespace cudf {
namespace experimental {
namespace io {
namespace detail {
namespace csv {

namespace {//unnammed:
//helpers:
  
struct column_to_strings_fn
{
  //negates all conditions used for
  //instantiations of individual converters
  //in strings/convert/convert_*.hpp
  //(this should have been a `variable template`,
  // instead of a static function, but nvcc (10.0)
  // fails to compile var-templs);
  //
  template<typename column_type>
  constexpr static bool is_not_handled(void)
  {
    return( //(!std::is_same<column_type, bool>::value) && <- case covered by is_integral
            (!std::is_same<column_type, cudf::string_view>::value) &&
            (!std::is_integral<column_type>::value) &&
            (!std::is_floating_point<column_type>::value) &&
            (!cudf::is_timestamp<column_type>()) );
  }
  
  explicit column_to_strings_fn(writer_options const& options,
                                rmm::mr::device_memory_resource* mr = nullptr):
    options_(options),
    mr_(mr)
  {
  }

  //bools:
  //
  template<typename column_type>
  std::enable_if_t<std::is_same<column_type, bool>::value,
                   strings_column_view>
  operator()(column_view const& column) const
  {
    auto conv_col_ptr = cudf::strings::from_booleans(column,
                                                     options_.true_value(),
                                                     options_.false_value(),
                                                     mr_);

    strings_column_view strings_converted{std::move(*conv_col_ptr)};
    auto converted_nulls_replaced = cudf::strings::replace_nulls(strings_converted,
                                                                 options_.na_rep() ,
                                                                 mr_);
    
    return strings_column_view{std::move(*converted_nulls_replaced)};
  }

  //strings:
  //
  template<typename column_type>
  std::enable_if_t<std::is_same<column_type, cudf::string_view>::value,
                   strings_column_view>
  operator()(column_view const& column) const
  {
    auto converted_nulls_replaced = cudf::strings::replace_nulls(column,
                                                                 options_.na_rep() ,
                                                                 mr_);
    
    return strings_column_view{std::move(*converted_nulls_replaced)};
  }

  template<typename column_type>
  std::enable_if_t<std::is_integral<column_type>::value && !std::is_same<column_type, bool>::value,
                   strings_column_view>
  operator()(column_view const& column) const
  {
    //ints...
    //
    auto conv_col_ptr = cudf::strings::from_integers(column,
                                                     mr_);
    //TODO: replace nulls by options.na_;
    //
    return strings_column_view{std::move(*conv_col_ptr)};
  }

  template<typename column_type>
  std::enable_if_t<std::is_floating_point<column_type>::value,
                   strings_column_view>
  operator()(column_view const& column) const
  {
    //floats...
    //
    auto conv_col_ptr = cudf::strings::from_floats(column,
                                                   mr_);
    //TODO: replace nulls by options.na_;
    //
    return strings_column_view{std::move(*conv_col_ptr)};
  }

  template<typename column_type>
  std::enable_if_t<cudf::is_timestamp<column_type>(),
                   strings_column_view>
  operator()(column_view const& column) const
  {
    //timestamps...
    //
    std::string format{"%Y-%m-%dT%H:%M:%SZ"};
    auto conv_col_ptr = cudf::strings::from_timestamps(column,
                                                       format,
                                                       mr_);
    //TODO: replace nulls by options.na_;
    //
    return strings_column_view{std::move(*conv_col_ptr)};
  }


  template<typename column_type>
  std::enable_if_t<is_not_handled<column_type>(),
                   strings_column_view>
  operator()(column_view const& column) const
  {
    //not to be called...
    //
    CUDF_FAIL("Unsupported column type.");
    //silence the compiler: no return
  }
private:
  writer_options const& options_;
  rmm::mr::device_memory_resource* mr_;
};


/**
 * @brief Helper function for write_csv.
 *
 * @param column The column to be converted.
 * @param options ...
 * @param mr...
 * @return strings_column_view instance formated for CSV column output.
**/
strings_column_view column_to_strings_csv(column_view const& column,
                                          writer_options const& options,
                                          rmm::mr::device_memory_resource* mr = nullptr) {
  //TODO;
  //
  column_to_strings_fn col2str{options, mr};
  auto ret = col2str.template operator()<bool>(column); // check instantiation: okay
  
  return strings_column_view{column}; // for now
}

} // unnamed namespace

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               writer_options const& options,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mr))
{
}

// Destructor within this translation unit
writer::~writer() = default;


writer::impl::impl(std::unique_ptr<data_sink> sink,
                   writer_options const &options,
                   rmm::mr::device_memory_resource *mr):
  out_sink_(std::move(sink)),
  mr_(mr),
  options_(options)
{
}

void writer::impl::write_chunked_begin(table_view const& table,
                                       const table_metadata *metadata,
                                       hipStream_t stream)
{
}


void writer::impl::write_chunked(table_view const& table,
                                 const table_metadata *metadata,
                                 hipStream_t stream)
{
}

  
void writer::impl::write(table_view const &table,
                         const table_metadata *metadata,
                         hipStream_t stream) {
  //TODO: chunked behavior / decision making (?)

  CUDF_EXPECTS( table.num_columns() > 0 && table.num_rows() > 0, "Empty table." );

  //no need to check same-size columns constraint; auto-enforced by table_view
  auto rows_chunk = options_.rows_per_chunk();
  //
  // This outputs the CSV in row chunks to save memory.
  // Maybe we can use the total_rows*count calculation and a memory threshold
  // instead of an arbitrary chunk count.
  // The entire CSV chunk must fit in CPU memory before writing it out.
  //
  if( rows_chunk % 8 ) // must be divisible by 8
    rows_chunk += 8 - (rows_chunk % 8);
  CUDF_EXPECTS( rows_chunk>0, "write_csv: invalid chunk_rows; must be at least 8" );

  auto exec = rmm::exec_policy(stream);

  //vts = split(table_view, row_offset, nrows);
  //loop v: vts{
  //  loop crt_col_v: v.columns{
  //    str_col_v = column_to_strings_csv(crt_col_v);
  //  }
  //}
}

void writer::write_all(table_view const &table, const table_metadata *metadata, hipStream_t stream) {
  _impl->write(table, metadata, stream);
}



}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace experimental
}  // namespace cudf


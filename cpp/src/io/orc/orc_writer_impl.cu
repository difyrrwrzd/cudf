#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "orc_writer_impl.hpp"

#include <cstring>

#include "orc.h"
#include "orc_gpu.h"

#include <io/utilities/wrapper_utils.hpp>
#include <nvstrings/NVCategory.h>
#include <nvstrings/NVStrings.h>

namespace {

template <typename T>
using pinned_buffer = std::unique_ptr<T, decltype(&hipHostFree)>;

}  // namespace

namespace cudf {
namespace io {
namespace orc {

/**
 * @brief Function that translates GDF dtype to ORC datatype
 **/
constexpr TypeKind to_orckind(gdf_dtype dtype) {
  switch (dtype) {
    case GDF_INT8:
      return BYTE;
    case GDF_INT16:
      return SHORT;
    case GDF_INT32:
      return INT;
    case GDF_INT64:
      return LONG;
    case GDF_FLOAT32:
      return FLOAT;
    case GDF_FLOAT64:
      return DOUBLE;
    case GDF_BOOL8:
      return BOOLEAN;
    case GDF_DATE32:
      return DATE;
    case GDF_DATE64:
    case GDF_TIMESTAMP:
      return TIMESTAMP;
    case GDF_CATEGORY:
      return INT;
    case GDF_STRING:
    case GDF_STRING_CATEGORY:
      return STRING;
    default:
      return INVALID_TYPE_KIND;
  }
}

/**
 * @brief Function that copies all chunks belonging to a stream
 **/
size_t gather_stripe_stream(uint8_t *dst, const gpu::EncChunk *chunks,
                            gpu::StreamIndexType strm_type,
                            size_t num_rowgroups, size_t num_columns) {
  size_t dst_pos = 0;
  for (size_t g = 0; g < num_rowgroups; g++) {
    const gpu::EncChunk *ck = &chunks[g * num_columns];
    uint32_t chunk_len = ck->strm_len[strm_type];
    if (ck->streams[strm_type]) {
      CUDA_TRY(hipMemcpyAsync(dst + dst_pos, ck->streams[strm_type], chunk_len,
                               hipMemcpyDeviceToHost));
    } else {
      memset(dst + dst_pos, 0, chunk_len);
    }
    dst_pos += chunk_len;
  }
  CUDA_TRY(hipStreamSynchronize(0));

  return dst_pos;
}

writer::Impl::Impl(std::string filepath, writer_options const& options) {
  outfile_.open(filepath, std::ios::out | std::ios::binary | std::ios::trunc);
  CUDF_EXPECTS(outfile_.is_open(), "Cannot open output file");
}

void writer::Impl::write(const cudf::table& table) {
    auto columns = table.begin();
    const int num_columns = table.num_columns();

    std::vector<uint8_t> buf;
    ProtobufWriter pbw(&buf);
    PostScript ps;
    FileFooter ff;
    StripeFooter sf;
    size_t ps_length;
    std::vector<int32_t> stream_ids;
    std::vector<size_t> strm_offsets;
    std::vector<int> str_col_ids;
    std::vector<int> str_col_map;
    std::vector<uint32_t> stripe_list;
    size_t num_rowgroups, num_chunks, num_string_columns, num_dict_chunks, rleout_bfr_size, strdata_bfr_size;
    bool has_timestamp_column = false;
    int32_t max_stream_size;
    pinned_buffer<uint8_t> stream_io_buf{nullptr, hipHostFree};

    // PostScript
    ps.compression = NONE;
    ps.compressionBlockSize = 256 * 1024; // TODO: Pick smaller values if too few compression blocks
    ps.version = {0,12};
    ps.metadataLength = 0; // TODO: Write stripe statistics
    ps.magic = "ORC";
    // File Footer
    ff.headerLength = ps.magic.length();
    ff.numberOfRows = 0;
    ff.rowIndexStride = 10000;
    ff.types.resize(1 + num_columns);
    ff.types[0].kind = STRUCT;
    ff.types[0].subtypes.resize(num_columns);
    ff.types[0].fieldNames.resize(num_columns);
    stream_ids.resize(num_columns * gpu::CI_NUM_STREAMS, -1);
    sf.columns.resize(num_columns + 1);
    sf.streams.resize(num_columns + 1);
    sf.streams[0].column = 0;
    sf.streams[0].kind = ROW_INDEX;
    sf.streams[0].length = 0;
    str_col_map.resize(num_columns);
    num_string_columns = 0;
    for (int i = 0; i < num_columns; i++)
    {
        ff.numberOfRows = std::max(ff.numberOfRows, (uint64_t)columns[i]->size);
        sf.streams[1+i].column = 1 + i;
        sf.streams[1+i].kind = ROW_INDEX;
        sf.streams[1+i].length = 0;
        str_col_map[i] = (int)num_string_columns;
        if (columns[i]->dtype == GDF_STRING || columns[i]->dtype == GDF_STRING_CATEGORY)
        {
            str_col_ids.push_back(i);
            num_string_columns++;
        }
    }
    num_rowgroups = (ff.numberOfRows + ff.rowIndexStride - 1) / ff.rowIndexStride;
    num_chunks = num_rowgroups * num_columns;
    num_dict_chunks = num_rowgroups * num_string_columns;
    hostdevice_vector<gpu::DictionaryChunk> dict(num_dict_chunks);
    device_buffer<uint32_t> dict_index(num_string_columns * ff.numberOfRows);
    device_buffer<uint32_t> dict_data(num_string_columns * ff.numberOfRows);
    std::vector<device_buffer<std::pair<const char*,size_t>>> str_indices(num_string_columns);
    if (num_dict_chunks != 0)
    {
        // Create initial per-rowgroup string dictionaries
        for (int i = 0; i < (int)num_string_columns; i++)
        {
            const gdf_column *col = columns[str_col_ids[i]];

            str_indices[i].resize(col->size);
            if (col->dtype == GDF_STRING) {
              auto *str = static_cast<NVStrings *>(col->data);
              CUDF_EXPECTS(str->create_index(str_indices[i].data()) == 0,
                           "Cannot retrieve nvcategory string pairs");
            } else if (col->dtype == GDF_STRING_CATEGORY) {
              auto *cat = static_cast<NVCategory *>(col->dtype_info.category);
              CUDF_EXPECTS(cat->create_index(str_indices[i].data()) == 0,
                           "Cannot retrieve nvcategory string pairs");
            } else {
              CUDF_FAIL("Expected a string-type column");
            }

            for (int g = 0; g < (int)num_rowgroups; g++)
            {
                gpu::DictionaryChunk *ck = &dict[g * num_string_columns + i];
                ck->valid_map_base = (col->null_count != 0) ? reinterpret_cast<const uint32_t *>(col->valid) : nullptr;
                ck->column_data_base = str_indices[i].data();
                ck->dict_data = dict_data.data() + i * ff.numberOfRows + g * ff.rowIndexStride;
                ck->dict_index = dict_index.data() + i * ff.numberOfRows; // Indexed by absolute row
                ck->start_row = g * ff.rowIndexStride;
                ck->num_rows = std::min(ff.rowIndexStride, (uint32_t)std::max(col->size - (int)ck->start_row, 0));
                ck->num_strings = 0;
                ck->string_char_count = 0;
                ck->num_dict_strings = 0;
                ck->dict_char_count = 0;
            }
        }
        // Build string dictionaries and update character data and dictionary sizes
        CUDA_TRY(hipMemcpyAsync(dict.device_ptr(), dict.host_ptr(),
                                 dict.memory_size(), hipMemcpyHostToDevice));
        CUDA_TRY(InitDictionaryIndices(dict.device_ptr(),
                                       (uint32_t)num_string_columns,
                                       (uint32_t)num_rowgroups));
        CUDA_TRY(hipMemcpyAsync(dict.host_ptr(), dict.device_ptr(),
                                 dict.memory_size(), hipMemcpyDeviceToHost));
        CUDA_TRY(hipStreamSynchronize(0));
    }
    // Decide stripe boundaries early on, based on uncompressed size
    for (size_t g = 0, stripe_start = 0, stripe_size = 0; g < num_rowgroups; g++)
    {
        const unsigned int kMaxStripeSize = 64 * 1024 * 1024;   // TBD: Stripe size hardcoded to 64MB
        const unsigned int max_stripe_rows = (num_string_columns) ? 1000000 : 5000000; // Limits dictionary size
        size_t rowgroup_size = 0;
        for (size_t i = 0; i < (size_t)num_columns; i++)
        {
            size_t dtype_len = 0;
            switch (columns[i]->dtype)
            {
            default:
            case GDF_INT8:
            case GDF_BOOL8:
                dtype_len = 1;
                break;
            case GDF_INT16:
                dtype_len = 2;
                break;
            case GDF_INT32:
            case GDF_FLOAT32:
            case GDF_DATE32:
            case GDF_CATEGORY:
                dtype_len = 4;
                break;
            case GDF_INT64:
            case GDF_FLOAT64:
            case GDF_DATE64:
            case GDF_TIMESTAMP:
                dtype_len = 8;
                break;
            case GDF_STRING:
            case GDF_STRING_CATEGORY:
                dtype_len = 1; // Count 1 byte for length
                rowgroup_size += dict[g * num_string_columns + str_col_map[i]].string_char_count;
                break;
            }
            rowgroup_size += dtype_len * ff.rowIndexStride;
        }
        if (g > stripe_start && (stripe_size + rowgroup_size > kMaxStripeSize || (g + 1 - stripe_start) * ff.rowIndexStride > max_stripe_rows) )
        {
            stripe_list.push_back((uint32_t)(g - stripe_start));
            stripe_start = g;
            stripe_size = 0;
        }
        stripe_size += rowgroup_size;
        if (g + 1 == num_rowgroups)
        {
            stripe_list.push_back((uint32_t)(num_rowgroups - stripe_start));
        }
    }

    // Build stripe-level dictionaries
    size_t num_stripes = stripe_list.size();
    size_t num_stripe_dict = num_stripes * num_string_columns;
    hostdevice_vector<gpu::StripeDictionary> stripe_dict(num_stripe_dict);
    if (num_dict_chunks != 0)
    {
        for (size_t i = 0; i < num_string_columns; i++)
        {
            size_t direct_cost = 0, dictionary_cost = 0;
            for (size_t j = 0, g = 0; j < num_stripes; j++)
            {
                uint32_t num_chunks = stripe_list[j];
                gpu::StripeDictionary *sd = &stripe_dict[j * num_string_columns + i];
                sd->column_data_base = dict[i].column_data_base;
                sd->dict_data = dict[g * num_string_columns + i].dict_data;
                sd->dict_index = dict_index.data() + i * ff.numberOfRows; // Indexed by absolute row
                sd->column_id = str_col_ids[i];
                sd->start_chunk = (uint32_t)g;
                sd->num_chunks = num_chunks;
                sd->num_strings = 0;
                for (size_t k = g; k < g + num_chunks; k++)
                {
                    direct_cost += dict[k * num_string_columns + i].string_char_count;
                    dictionary_cost += dict[k * num_string_columns + i].dict_char_count;
                    sd->num_strings += dict[k * num_string_columns + i].num_dict_strings;
                }
                dictionary_cost += sd->num_strings;
                sd->dict_char_count = 0;
                g += num_chunks;
            }
            // Early decision to disable dictionary if it doesn't look good at the chunk level
            if (dictionary_cost >= direct_cost)
            {
                for (size_t j = 0; j < num_stripes; j++)
                    stripe_dict[j * num_string_columns + i].dict_data = nullptr;
            }
        }
        CUDA_TRY(
            hipMemcpyAsync(stripe_dict.device_ptr(), stripe_dict.host_ptr(),
                            stripe_dict.memory_size(), hipMemcpyHostToDevice));
        CUDA_TRY(BuildStripeDictionaries(
            stripe_dict.device_ptr(), stripe_dict.host_ptr(), dict.device_ptr(),
            (uint32_t)num_stripes, (uint32_t)num_rowgroups,
            (uint32_t)num_string_columns));
        CUDA_TRY(
            hipMemcpyAsync(stripe_dict.host_ptr(), stripe_dict.device_ptr(),
                            stripe_dict.memory_size(), hipMemcpyDeviceToHost));
        CUDA_TRY(hipStreamSynchronize(0));
    }
    // Initialize streams
    sf.columns[0].kind = DIRECT;
    sf.columns[0].dictionarySize = 0;
    strdata_bfr_size = 0;
    for (int i = 0; i < num_columns; i++)
    {
        TypeKind kind = to_orckind(columns[i]->dtype);
        StreamKind data_kind = DATA, data2_kind = LENGTH;
        ColumnEncodingKind encoding_kind = DIRECT;
        int64_t present_stream_size = 0, data_stream_size = 0, data2_stream_size = 0, dict_stream_size = 0;

        ff.types[1 + i].kind = kind;
        if (columns[i]->null_count != 0 || (uint64_t)columns[i]->size != ff.numberOfRows)
        {
            present_stream_size = ((ff.rowIndexStride + 7) >> 3);
            present_stream_size += (present_stream_size + 0x7f) >> 7;
        }
        switch(kind)
        {
        case BOOLEAN:
            data_stream_size = ((ff.rowIndexStride + 0x3ff) >> 10) * (128 + 1);
            break;
        case BYTE:
            data_stream_size = ((ff.rowIndexStride + 0x7f) >> 7) * (128 + 1);
            break;
        case SHORT:
            data_stream_size = ((ff.rowIndexStride + 0x1ff) >> 9) * (512 * 2 + 2);
            encoding_kind = DIRECT_V2;
            break;
        case FLOAT:
            // Pass through if no nulls (no RLE encoding for floating point)
            data_stream_size = (columns[i]->null_count) ? ((ff.rowIndexStride + 0x1ff) >> 9) * (512 * 4 + 2) : INT64_C(-1);
            break;
        case INT:
        case DATE:
            data_stream_size = ((ff.rowIndexStride + 0x1ff) >> 9) * (512 * 4 + 2);
            encoding_kind = DIRECT_V2;
            break;
        case DOUBLE:
            // Pass through if no nulls (no RLE encoding for floating point)
            data_stream_size = (columns[i]->null_count) ? ((ff.rowIndexStride + 0x1ff) >> 9) * (512 * 8 + 2) : INT64_C(-1);
            break;
        case LONG:
            data_stream_size = ((ff.rowIndexStride + 0x1ff) >> 9) * (512 * 8 + 2);
            encoding_kind = DIRECT_V2;
            break;
        case STRING: {
            uint32_t scol = str_col_map[i], dict_bits;
            size_t direct_data_size = 0, dict_data_size = 0, dict_strings = 0, dict_lengths_div512 = 0, dict_overhead;
            bool enable_dictionary = true;
            for (size_t stripe_id = 0, g = 0; stripe_id < stripe_list.size(); stripe_id++)
            {
                const gpu::StripeDictionary *sd = &stripe_dict[stripe_id * num_string_columns + scol];
                enable_dictionary = (enable_dictionary && sd->dict_data != nullptr);
                dict_strings += sd->num_strings;
                dict_lengths_div512 += (sd->num_strings + 0x1ff) >> 9;
                dict_data_size += sd->dict_char_count;
                for (uint32_t k = 0; k < stripe_list[stripe_id]; k++, g++)
                    direct_data_size += dict[g * num_string_columns + scol].string_char_count;
            }
            for (dict_bits = 1; dict_bits < 32; dict_bits <<= 1)
            {
                if (dict_strings <= (1ull << dict_bits))
                    break;
            }
            dict_overhead = (dict_bits * (columns[i]->size - columns[i]->null_count) + 7) >> 3;
            //printf("col%d: dict_data_size(%zd strings) = %zd+%zd, direct_size = %zd\n", i, dict_strings, dict_data_size, dict_overhead, direct_data_size);
            if (enable_dictionary && dict_data_size + dict_overhead < direct_data_size)
            {
                // Dictionary encoding
                data_stream_size = ((ff.rowIndexStride + 0x1ff) >> 9) * (512 * 4 + 2);
                data2_stream_size = dict_lengths_div512 * (512 * 4 + 2);
                dict_stream_size = std::max(dict_data_size, (size_t)1);
                encoding_kind = DICTIONARY_V2;
            }
            else
            {
                // Direct encoding
                data_stream_size = std::max(direct_data_size, (size_t)1);
                data2_stream_size = ((ff.rowIndexStride + 0x1ff) >> 9) * (512 * 4 + 2);
                encoding_kind = DIRECT_V2;
            }
            break;
          }
        case TIMESTAMP:
            data2_stream_size = data_stream_size = ((ff.rowIndexStride + 0x1ff) >> 9) * (512*4 + 2);
            data2_kind = SECONDARY;
            has_timestamp_column = true;
            encoding_kind = DIRECT_V2;
            break;
        default:
            break;
        }
        if (present_stream_size != 0)
        {
            uint32_t present_stream_id = (uint32_t)sf.streams.size();
            sf.streams.resize(present_stream_id + 1);
            sf.streams[present_stream_id].column = 1 + i;
            sf.streams[present_stream_id].kind = PRESENT;
            sf.streams[present_stream_id].length = present_stream_size;
            stream_ids[i * gpu::CI_NUM_STREAMS + gpu::CI_PRESENT] = (int32_t)present_stream_id;
        }
        if (data_stream_size != 0)
        {
            uint32_t data_stream_id = (uint32_t)sf.streams.size();
            sf.streams.resize(data_stream_id + 1);
            sf.streams[data_stream_id].column = 1 + i;
            sf.streams[data_stream_id].kind = data_kind;
            sf.streams[data_stream_id].length = std::max<int64_t>(data_stream_size, 0);
            stream_ids[i * gpu::CI_NUM_STREAMS + gpu::CI_DATA] = (int32_t)data_stream_id;
        }
        if (data2_stream_size != 0)
        {
            uint32_t data_stream_id = (uint32_t)sf.streams.size();
            sf.streams.resize(data_stream_id + 1);
            sf.streams[data_stream_id].column = 1 + i;
            sf.streams[data_stream_id].kind = data2_kind;
            sf.streams[data_stream_id].length = std::max<int64_t>(data2_stream_size, 0);
            stream_ids[i * gpu::CI_NUM_STREAMS + gpu::CI_DATA2] = (int32_t)data_stream_id;
        }
        if (dict_stream_size != 0)
        {
            uint32_t dict_stream_id = (uint32_t)sf.streams.size();
            sf.streams.resize(dict_stream_id + 1);
            sf.streams[dict_stream_id].column = 1 + i;
            sf.streams[dict_stream_id].kind = DICTIONARY_DATA;
            sf.streams[dict_stream_id].length = dict_stream_size;
            stream_ids[i * gpu::CI_NUM_STREAMS + gpu::CI_DICTIONARY] = (int32_t)dict_stream_id;
        }
        ff.types[0].subtypes[i] = 1 + i;
        if (columns[i]->col_name)
            ff.types[0].fieldNames[i].assign(columns[i]->col_name);
        else
            ff.types[0].fieldNames[i] = "_col" + std::to_string(i);
        sf.columns[1 + i].kind = encoding_kind;
        sf.columns[1 + i].dictionarySize = 0;
    }
    sf.writerTimezone = (has_timestamp_column) ? "UTC" : "";
    strm_offsets.resize(sf.streams.size());
    rleout_bfr_size = 0;
    for (size_t i = 0; i < sf.streams.size(); i++)
    {
        if (((sf.streams[i].kind == DICTIONARY_DATA || sf.streams[i].kind == LENGTH) && sf.columns[sf.streams[i].column].kind == DICTIONARY_V2)
         || (sf.streams[i].kind == DATA && ff.types[sf.streams[i].column].kind == STRING && sf.columns[sf.streams[i].column].kind == DIRECT_V2))
        {
            strm_offsets[i] = strdata_bfr_size;
            strdata_bfr_size += sf.streams[i].length;
        }
        else
        {
            strm_offsets[i] = rleout_bfr_size;
            rleout_bfr_size += (sf.streams[i].length * num_rowgroups + 7) & ~7;
        }
    }
    strdata_bfr_size = (strdata_bfr_size + 7) & ~7;
    hostdevice_vector<gpu::EncChunk> chunks(num_chunks);
    device_buffer<uint8_t> rleout_bfr_dev(rleout_bfr_size + strdata_bfr_size);
    for (size_t j = 0, stripe_start = 0, stripe_id = 0; j < num_rowgroups; j++)
    {
        for (size_t i = 0; i < (size_t)num_columns; i++)
        {
            gpu::EncChunk *ck = &chunks[j * num_columns + i];
            ck->valid_map_base = (const uint32_t *)columns[i]->valid;
            ck->column_data_base = columns[i]->data;
            ck->start_row = (uint32_t)(j * ff.rowIndexStride);
            ck->num_rows = (uint32_t)std::min((uint32_t)ff.rowIndexStride, (uint32_t)(ff.numberOfRows - ck->start_row));
            ck->valid_rows = columns[i]->size;
            ck->encoding_kind = (uint8_t)sf.columns[1+i].kind;
            ck->type_kind = (uint8_t)ff.types[1+i].kind;
            ck->dtype_len = 0;
            switch(columns[i]->dtype_info.time_unit)
            {
            case TIME_UNIT_s:   ck->scale = 9; break;
            case TIME_UNIT_ms:  ck->scale = 6; break;
            case TIME_UNIT_us:  ck->scale = 3; break;
            case TIME_UNIT_ns:  ck->scale = 0; break;
            default:            ck->scale = 0; break;
            }
            switch (ck->type_kind)
            {
            case SHORT:
                ck->dtype_len = 2;
                break;
            case INT:
            case FLOAT:
            case DATE:
                ck->dtype_len = 4;
                break;
            case LONG:
            case DOUBLE:
            case TIMESTAMP:
                ck->dtype_len = 8;
                break;
            case STRING:
                ck->column_data_base = str_indices[str_col_map[i]].data();
                // fall-through
            default:
                ck->dtype_len = 1;
                if (ck->encoding_kind == DICTIONARY_V2)
                    ck->column_data_base = stripe_dict[stripe_id * num_string_columns + str_col_map[i]].dict_index;
                break;
            }
            for (int k = 0; k < gpu::CI_NUM_STREAMS; k++)
            {
                int32_t strm_id = stream_ids[i * gpu::CI_NUM_STREAMS + k];
                ck->strm_id[k] = strm_id;
                if (strm_id >= 0)
                {
                    if (k == gpu::CI_DICTIONARY || (k == gpu::CI_DATA2 && ck->encoding_kind == DICTIONARY_V2))
                    {
                        if (j == stripe_start)
                        {
                            const gpu::StripeDictionary *stripe = &stripe_dict[stripe_id * num_string_columns + str_col_map[i]];
                            ck->strm_len[k] = (k == gpu::CI_DICTIONARY) ? stripe->dict_char_count : (((stripe->num_strings + 0x1ff) >> 9) * (512 * 4 + 2));
                            if (stripe_id == 0)
                            {
                                ck->streams[k] = rleout_bfr_dev.data() + strm_offsets[strm_id];
                            }
                            else
                            {
                                const gpu::EncChunk *ck_up = &chunks[stripe[-(int32_t)num_string_columns].start_chunk * num_columns + i];
                                ck->streams[k] = ck_up->streams[k] + ck_up->strm_len[k];
                            }
                        }
                        else
                        {
                            ck->strm_len[k] = 0;
                            ck->streams[k] = ck[-num_columns].streams[k];
                        }
                    }
                    else if (k == gpu::CI_DATA && ck->type_kind == STRING && ck->encoding_kind == DIRECT_V2)
                    {
                        ck->strm_len[k] = dict[j * num_string_columns + str_col_map[i]].string_char_count;
                        ck->streams[k] = (j == 0) ? rleout_bfr_dev.data() + strm_offsets[strm_id] : (ck[-num_columns].streams[k] + ck[-num_columns].strm_len[k]);
                    }
                    else if (k == gpu::CI_DATA && sf.streams[strm_id].length == 0 && (ck->type_kind == DOUBLE || ck->type_kind == FLOAT))
                    {
                        // Pass-through
                        ck->streams[k] = nullptr;
                        ck->strm_len[k] = ck->num_rows * ck->dtype_len;
                    }
                    else
                    {
                        ck->streams[k] = rleout_bfr_dev.data() + strdata_bfr_size + strm_offsets[strm_id] + sf.streams[strm_id].length * j;
                        ck->strm_len[k] = (uint32_t)sf.streams[strm_id].length;
                    }
                }
                else
                {
                    ck->strm_len[k] = 0;
                    ck->streams[k] = nullptr;
                }
            }
        }
        if (j + 1 == stripe_start + stripe_list[stripe_id])
        {
            stripe_start = j + 1;
            stripe_id++;
        }
    }
    CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                             chunks.memory_size(), hipMemcpyHostToDevice));
    // Encode string dictionaries
    if (num_dict_chunks != 0) {
      CUDA_TRY(EncodeStripeDictionaries(
          stripe_dict.device_ptr(), chunks.device_ptr(),
          (uint32_t)num_string_columns, (uint32_t)num_columns,
          (uint32_t)stripe_list.size()));
    }
    // Encode column data
    CUDA_TRY(EncodeOrcColumnData(chunks.device_ptr(), (uint32_t)num_columns,
                                 (uint32_t)num_rowgroups));
    CUDA_TRY(hipMemcpyAsync(chunks.host_ptr(), chunks.device_ptr(),
                             chunks.memory_size(), hipMemcpyDeviceToHost));
    CUDA_TRY(hipStreamSynchronize(0));
    // Initialize stripe data in file footer
    ff.stripes.resize(stripe_list.size());
    for (size_t group = 0, stripe_id = 0, stripe_start = 0; stripe_id < stripe_list.size(); stripe_id++)
    {
        size_t stripe_data_length = 0, stripe_end;
        for (uint32_t g = 0; g < stripe_list[stripe_id]; g++)
        {
            for (int i = 0; i < num_columns; i++)
            {
                gpu::EncChunk *ck = &chunks[group * num_columns + i];
                for (int k = 0; k < gpu::CI_NUM_STREAMS; k++)
                {
                    stripe_data_length += ck->strm_len[k];
                }
            }
            group++;
        }
        stripe_end = std::min((uint64_t)group * ff.rowIndexStride, ff.numberOfRows);
        ff.stripes[stripe_id].dataLength = stripe_data_length;
        ff.stripes[stripe_id].numberOfRows = (uint32_t)(stripe_end - stripe_start);
        stripe_start = stripe_end;
    }

    // Write file header
    outfile_.write(ps.magic.c_str(), ps.magic.length());

    // Write stripe data
    max_stream_size = 0;
    for (size_t stripe_id = 0, group = 0; stripe_id < ff.stripes.size(); stripe_id++)
    {
        size_t groups_in_stripe = (ff.stripes[stripe_id].numberOfRows + ff.rowIndexStride - 1) / ff.rowIndexStride;
        int max_size = 0;
        ff.stripes[stripe_id].offset = outfile_.tellp();
        // Write index streams
        ff.stripes[stripe_id].indexLength = 0;
        for (size_t strm = 0; strm <= (size_t)num_columns; strm++)
        {
            TypeKind kind = ff.types[strm].kind;
            int32_t present_blk = -1, present_pos = -1, present_size = 0;
            int32_t data_blk = -1, data_pos = -1, data_size = 0;
            int32_t data2_blk = -1, data2_pos = -1, data2_size = 0;

            buf.resize(0);
            // TBD: Not sure we need an empty index stream for record column 0
            if (strm != 0)
            {
                gpu::EncChunk *ck = &chunks[strm - 1];
                if (ck->strm_id[gpu::CI_PRESENT] > 0)
                {
                    present_pos = 0;
                }
                if (ck->strm_id[gpu::CI_DATA] > 0)
                {
                    data_pos = 0;
                }
                if (ck->strm_id[gpu::CI_DATA2] > 0)
                {
                    data2_pos = 0;
                }
                max_size = std::max(max_size, (int)chunks[group * num_columns + strm - 1].strm_len[gpu::CI_DICTIONARY]);
            }
            if (kind == STRING && sf.columns[strm].kind == DICTIONARY_V2)
            {
                kind = INT; // Change string dictionary to int from index point of view
            }
            for (size_t g = group; g < group + groups_in_stripe; g++)
            {
                pbw.put_row_index_entry(present_blk, present_pos, data_blk, data_pos, data2_blk, data2_pos, kind);
                if (strm != 0)
                {
                    gpu::EncChunk *ck = &chunks[g * num_columns + strm - 1];
                    if (present_pos >= 0)
                    {
                        present_pos += ck->strm_len[gpu::CI_PRESENT];
                        present_size += ck->strm_len[gpu::CI_PRESENT];
                    }
                    if (data_pos >= 0)
                    {
                        data_pos += ck->strm_len[gpu::CI_DATA];
                        data_size += ck->strm_len[gpu::CI_DATA];
                    }
                    if (data2_pos >= 0)
                    {
                        data2_pos += ck->strm_len[gpu::CI_DATA2];
                        data2_size += ck->strm_len[gpu::CI_DATA2];
                    }
                }
            }
            max_size = std::max(max_size, present_size);
            max_size = std::max(max_size, data_size);
            max_size = std::max(max_size, data2_size);
            sf.streams[strm].length = buf.size();
            outfile_.write(reinterpret_cast<char*>(buf.data()), buf.size());
            ff.stripes[stripe_id].indexLength += buf.size();
        }
        if (max_size > max_stream_size)
        {
            max_stream_size = max_size;
            stream_io_buf =
                pinned_buffer<uint8_t>{[](size_t size) {
                                         uint8_t *ptr = nullptr;
                                         CUDA_TRY(hipHostMalloc(&ptr, size));
                                         return ptr;
                                       }(max_stream_size),
                                       hipHostFree};
        }
        // Write data streams
        ff.stripes[stripe_id].dataLength = 0;
        for (int i = 0; i < num_columns; i++)
        {
            gpu::EncChunk *ck = &chunks[group * num_columns + i];
            if (ck->strm_id[gpu::CI_PRESENT] > 0)
            {
                size_t len = gather_stripe_stream(stream_io_buf.get(), ck, gpu::CI_PRESENT, groups_in_stripe, num_columns);
                outfile_.write(reinterpret_cast<char*>(stream_io_buf.get()), len);
                ff.stripes[stripe_id].dataLength += len;
                sf.streams[ck->strm_id[gpu::CI_PRESENT]].length = len;
            }
            if (ck->strm_id[gpu::CI_DATA] > 0)
            {
                size_t len = gather_stripe_stream(stream_io_buf.get(), ck, gpu::CI_DATA, groups_in_stripe, num_columns);
                outfile_.write(reinterpret_cast<char*>(stream_io_buf.get()), len);
                ff.stripes[stripe_id].dataLength += len;
                sf.streams[ck->strm_id[gpu::CI_DATA]].length = len;
            }
            if (ck->strm_id[gpu::CI_DATA2] > 0)
            {
                size_t len = gather_stripe_stream(stream_io_buf.get(), ck, gpu::CI_DATA2, groups_in_stripe, num_columns);
                outfile_.write(reinterpret_cast<char*>(stream_io_buf.get()), len);
                ff.stripes[stripe_id].dataLength += len;
                sf.streams[ck->strm_id[gpu::CI_DATA2]].length = len;
            }
            if (ck->strm_id[gpu::CI_DICTIONARY] > 0)
            {
                size_t len = gather_stripe_stream(stream_io_buf.get(), ck, gpu::CI_DICTIONARY, groups_in_stripe, num_columns);
                outfile_.write(reinterpret_cast<char*>(stream_io_buf.get()), len);
                ff.stripes[stripe_id].dataLength += len;
                sf.streams[ck->strm_id[gpu::CI_DICTIONARY]].length = len;
            }
            sf.columns[1 + i].dictionarySize = (sf.columns[1 + i].kind == DICTIONARY_V2) ? stripe_dict[stripe_id * num_string_columns + str_col_map[i]].num_strings : 0;
        }
        // Write stripe footer
        buf.resize(0);
        ff.stripes[stripe_id].footerLength = (uint32_t)pbw.write(&sf);
        outfile_.write(reinterpret_cast<char*>(buf.data()), ff.stripes[stripe_id].footerLength);
        group += groups_in_stripe;
    }

    // TBD: We may want to add pandas or spark column metadata strings here
    ff.contentLength = outfile_.tellp();
    buf.resize(0);
    ps.footerLength = pbw.write(&ff);
    ps_length = pbw.write(&ps);
    buf.push_back((uint8_t)ps_length);

    // Write metadata
    outfile_.write(reinterpret_cast<char*>(buf.data()), buf.size());
    outfile_.flush();
}

writer::writer(std::string filepath, writer_options const& options)
    : impl_(std::make_unique<Impl>(filepath, options)) {}

void writer::write_all(const cudf::table& table) { impl_->write(table); }

writer::~writer() = default;

}  // namespace orc
}  // namespace io
}  // namespace cudf

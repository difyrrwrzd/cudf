#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "orc_common.h"
#include "orc_gpu.h"
#include <io/utilities/block_utils.cuh>

namespace cudf {
namespace io {
namespace orc {
namespace gpu {

/**
 * @brief Initializes statistics groups
 *
 * @param[out] groups Statistics groups
 * @param[in] cols Column descriptors
 * @param[in] num_columns Number of columns
 * @param[in] num_rowgroups Number of rowgroups
 * @param[in] row_index_stride Rowgroup size in rows
 *
 **/
// blockDim {128,1,1}
__global__ void __launch_bounds__(128)
gpuInitStatisticsGroups(statistics_group *groups, const stats_column_desc *cols,
                        uint32_t num_columns, uint32_t num_rowgroups, uint32_t row_index_stride) {
  __shared__ __align__(4) volatile statistics_group grp_g[4];
  uint32_t col_id = blockIdx.y;
  uint32_t ck_id = (blockIdx.x * 4) + (threadIdx.x >> 5);
  uint32_t t = threadIdx.x & 0x1f;
  volatile statistics_group *grp = &grp_g[threadIdx.x >> 5];
  if (ck_id < num_rowgroups) {
    if (!t) {
      uint32_t num_rows = cols[col_id].num_rows;
      grp->col = &cols[col_id];
      grp->start_row = ck_id * row_index_stride;
      grp->num_rows = min(num_rows - min(ck_id * row_index_stride, num_rows), row_index_stride);
      __threadfence_block();
    }
    SYNCWARP();
    if (t < sizeof(statistics_group) / sizeof(uint32_t)) {
      reinterpret_cast<uint32_t *>(&groups[col_id * num_rowgroups + ck_id])[t] = reinterpret_cast<volatile uint32_t *>(grp)[t];
    }
  } 
}


/**
 * @brief Get the buffer size and offsets of encoded statistics
 *
 * @param[in,out] groups Statistics merge groups
 * @param[in] statistics_count Number of statistics buffers
 *
 **/
// blockDim {1024,1,1}
__global__ void __launch_bounds__(1024, 1)
gpuInitStatisticsBufferSize(statistics_merge_group *groups, const statistics_chunk *chunks, uint32_t statistics_count) {
  __shared__ volatile uint32_t scratch_red[32];
  __shared__ volatile uint32_t stats_size;
  uint32_t t = threadIdx.x;
  if (!t) {
    stats_size = 0;
  }
  __syncthreads();
  for (uint32_t start = 0; start < statistics_count; start += 1024) {
    uint32_t stats_len = 0, stats_pos;
    uint32_t idx = start + t;
    if (idx < statistics_count) {
      const stats_column_desc *col = groups[idx].col;
      statistics_dtype dtype = col->stats_dtype;
      switch(dtype) {
      case dtype_bool8:
        stats_len = 2 + 2 * 5;
        break;
      case dtype_int8:
      case dtype_int16:
      case dtype_int32:
      case dtype_date32:
      case dtype_int64:
      case dtype_timestamp64:
        stats_len = 2 + 3 * (2 + 10);
        break;
      case dtype_float32:
      case dtype_float64:
        stats_len = 2 + 3 * (2 + 8);
        break;
      case dtype_decimal64:
      case dtype_decimal128:
        stats_len = 2 + 3 * (2 + 40);
        break;
      case dtype_string:
        stats_len = 5 + 10 + chunks[idx].min_value.str_val.length + chunks[idx].max_value.str_val.length;
        break;
      default: break;
      }
    }
    stats_pos = WarpReducePos32(stats_len, t);
    if ((t & 0x1f) == 0x1f) {
      scratch_red[t >> 5] = stats_pos;
    }
    __syncthreads();
    if (t < 32) {
      scratch_red[t] = WarpReducePos32(scratch_red[t], t);
    }
    __syncthreads();
    if (t >= 32) {
      stats_pos += scratch_red[(t >> 5) - 1];
    }
    stats_pos += stats_size;
    if (idx < statistics_count) {
      groups[idx].start_chunk = stats_pos - stats_len;
      groups[idx].num_chunks = stats_len;
    }
    __syncthreads();
    if (t == 1023) {
      stats_size = stats_pos;
    }
  }
}


struct stats_state_s {
  uint8_t *base;              ///< Output buffer start
  uint8_t *cur;               ///< Output buffer current write position
  uint8_t *end;               ///< Output buffer end
  statistics_chunk ck;
  statistics_merge_group grp;
  stats_column_desc col;
  // ORC stats
  uint64_t numberOfValues;
  uint8_t hasNull;
};


// Protobuf varint encoding for unsigned int
__device__ inline uint8_t *pb_encode_uint(uint8_t *p, uint64_t v) {
  while (v > 0x7f) {
    *p++ = ((uint32_t)v | 0x80);
    v >>= 7;
  }
  *p++ = v;
  return p;
}

// Protobuf field encoding for unsigned int
__device__ inline uint8_t *pb_put_uint(uint8_t *p, uint32_t id, uint64_t v) {
  p[0] = id * 8 + PB_TYPE_VARINT; // NOTE: Assumes id < 16
  return pb_encode_uint(p + 1, v);
}

// Protobuf field encoding for signed int
__device__ inline uint8_t *pb_put_int(uint8_t *p, uint32_t id, int64_t v) {
  int64_t s = (v < 0);
  return pb_put_uint(p, id, (v ^ -s) * 2 + s);
}

// Protobuf field encoding for binary/string
__device__ inline uint8_t *pb_put_binary(uint8_t *p, uint32_t id, const uint8_t *bytes, uint32_t len) {
  p[0] = id * 8 + PB_TYPE_FIXEDLEN;
  p = pb_encode_uint(p + 1, len);
  memcpy(p, bytes, len);
  return p + len;
}


/**
 * @brief Encode statistics in ORC protobuf format
 *
 * @param[in,out] groups Statistics merge groups
 * @param[in,out] chunks Statistics data
 * @param[in] statistics_count Number of statistics buffers
 *
 **/
// blockDim {128,1,1}
__global__ void __launch_bounds__(128)
gpuEncodeStatistics(uint8_t *blob_bfr, statistics_merge_group *groups, const statistics_chunk *chunks,
                    uint32_t statistics_count) {
  __shared__ __align__(8) stats_state_s state_g[4];
  uint32_t t = threadIdx.x & 0x1f;
  uint32_t idx = blockIdx.x * 4 + (threadIdx.x >> 5);
  stats_state_s * const s = &state_g[threadIdx.x >> 5];
  if (idx < statistics_count) {
    if (t < sizeof(statistics_chunk) / sizeof(uint32_t)) {
      reinterpret_cast<uint32_t *>(&s->ck)[t] = reinterpret_cast<const uint32_t *>(&chunks[idx])[t];
    }
    if (t < sizeof(statistics_merge_group) / sizeof(uint32_t)) {
      reinterpret_cast<uint32_t *>(&s->grp)[t] = reinterpret_cast<uint32_t *>(&groups[idx])[t];
    }
  }
  __syncthreads();
  if (idx < statistics_count) {
    if (t < sizeof(stats_column_desc) / sizeof(uint32_t)) {
      reinterpret_cast<uint32_t *>(&s->col)[t] = reinterpret_cast<const uint32_t *>(s->grp.col)[t];
    }
    if (t == 0) {
      s->cur = s->base = blob_bfr + s->grp.start_chunk;
      s->end = blob_bfr + s->grp.start_chunk + s->grp.num_chunks;
    }
  }
  __syncthreads();
  // Update actual bfr size
  if (idx < statistics_count && t == 0) {
    groups[idx].num_chunks = static_cast<uint32_t>(s->cur - s->base);
  }
}


/**
 * @brief Launches kernels to initialize statistics collection
 *
 * @param[out] groups Statistics groups (rowgroup-level)
 * @param[in] cols Column descriptors
 * @param[in] num_columns Number of columns
 * @param[in] num_rowgroups Number of rowgroups
 * @param[in] row_index_stride Rowgroup size in rows
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t OrcInitStatisticsGroups(statistics_group *groups, const stats_column_desc *cols, uint32_t num_columns,
                                    uint32_t num_rowgroups, uint32_t row_index_stride, hipStream_t stream)
{
    dim3 dim_groups((num_rowgroups+3) >> 2, num_columns);
    gpuInitStatisticsGroups <<< dim_groups, 128, 0, stream >>>(groups, cols, num_columns, num_rowgroups, row_index_stride);

    return hipSuccess;
}


/**
 * @brief Launches kernels to return statistics buffer offsets and sizes
 *
 * @param[in,out] groups Statistics merge groups
 * @param[in] chunks Satistics chunks
 * @param[in] statistics_count Number of statistics buffers to encode
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t OrcInitStatisticsBufferSize(statistics_merge_group *groups, const statistics_chunk *chunks,
                                        uint32_t statistics_count, hipStream_t stream)
{
    gpuInitStatisticsBufferSize <<< 1, 1024, 0, stream >>>(groups, chunks, statistics_count);
    return hipSuccess;
}


/**
 * @brief Launches kernel to encode statistics in ORC protobuf format
 *
 * @param[out] blob_bfr Output buffer for statistics blobs
 * @param[in,out] groups Statistics merge groups
 * @param[in,out] chunks Statistics data
 * @param[in] statistics_count Number of statistics buffers
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t OrcEncodeStatistics(uint8_t *blob_bfr, statistics_merge_group *groups, const statistics_chunk *chunks,
                                uint32_t statistics_count, hipStream_t stream)
{
    gpuEncodeStatistics <<< (statistics_count + 3) >> 2, 128, 0, stream >>>(blob_bfr, groups, chunks, statistics_count);
    return hipSuccess;
}


} // namespace gpu
} // namespace orc
} // namespace io
} // namespace cudf

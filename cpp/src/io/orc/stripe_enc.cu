/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "orc_common.h"
#include "orc_gpu.h"

namespace cudf {
namespace io {
namespace orc {
namespace gpu {



/**
 * @brief Launches kernel for encoding column data
 *
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[in] num_columns Number of columns
 * @param[in] num_rowgroups Number of row groups
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t EncodeOrcColumnData(EncChunk *chunks, uint32_t num_columns, uint32_t num_rowgroups, hipStream_t stream)
{
    return hipSuccess;
}


} // namespace gpu
} // namespace orc
} // namespace io
} // namespace cudf

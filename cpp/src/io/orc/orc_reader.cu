#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "orc.hpp"

#include "cudf.h"
#include "io/comp/gpuinflate.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.hpp"

#include <hip/hip_runtime.h>
#include <nvstrings/NVStrings.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include <array>
#include <cstring>
#include <iostream>
#include <numeric>
#include <vector>

#include <fcntl.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

/**
 * @brief Helper class for memory mapping a file source
 **/
class DataSource {
 public:
  explicit DataSource(const char *filepath) {
    fd = open(filepath, O_RDONLY);
    CUDF_EXPECTS(fd > 0, "Cannot open file");

    struct stat st {};
    CUDF_EXPECTS(fstat(fd, &st) == 0, "Cannot query file size");

    mapped_size = st.st_size;
    CUDF_EXPECTS(mapped_size > 0, "Unexpected zero-sized file");

    mapped_data = mmap(NULL, mapped_size, PROT_READ, MAP_PRIVATE, fd, 0);
    CUDF_EXPECTS(mapped_data != MAP_FAILED, "Cannot memory-mapping file");
  }

  ~DataSource() {
    if (mapped_data) {
      munmap(mapped_data, mapped_size);
    }
    if (fd) {
      close(fd);
    }
  }

  const uint8_t *data() const { return static_cast<uint8_t *>(mapped_data); }
  size_t size() const { return mapped_size; }

 private:
  void *mapped_data = nullptr;
  size_t mapped_size = 0;
  int fd = 0;
};

/**
 * @brief A helper class that wraps a gdf_column and any associated memory.
 *
 * This abstraction initializes and manages a gdf_column (fields and memory)
 * while still allowing direct access. Memory is automatically deallocated
 * unless ownership is transferred via releasing and assigning the raw pointer.
 **/
class gdf_column_wrapper {
 public:
  gdf_column_wrapper(gdf_size_type size, gdf_dtype dtype,
                     gdf_dtype_extra_info dtype_info, const std::string name) {
    col = (gdf_column *)malloc(sizeof(gdf_column));
    col->col_name = (char *)malloc(name.length() + 1);
    strcpy(col->col_name, name.c_str());
    gdf_column_view_augmented(col, nullptr, nullptr, size, dtype, 0, dtype_info);
  }

  ~gdf_column_wrapper() {
    if (col) {
      RMM_FREE(col->data, 0);
      RMM_FREE(col->valid, 0);
      free(col->col_name);
    }
    free(col);
  };

  gdf_column_wrapper(const gdf_column_wrapper &other) = delete;
  gdf_column_wrapper(gdf_column_wrapper &&other) : col(other.col) {
    other.col = nullptr;
  }

  gdf_error allocate() {
    // For strings, just store the ptr + length. Eventually, column's data ptr
    // is replaced with an NvString instance created from these pairs.
    const auto num_rows = std::max(col->size, 1);
    const auto column_byte_width = (col->dtype == GDF_STRING)
                                       ? sizeof(std::pair<const char*, int>)
                                       : gdf_dtype_size(col->dtype);

    RMM_TRY(RMM_ALLOC(&col->data, num_rows * column_byte_width, 0));
    RMM_TRY(RMM_ALLOC(&col->valid, gdf_valid_allocation_size(num_rows), 0));
    CUDA_TRY(hipMemset(col->valid, 0, gdf_valid_allocation_size(num_rows)));

    return GDF_SUCCESS;
  }

  gdf_column *operator->() const { return col; }
  gdf_column *get() const { return col; }
  gdf_column *release() {
    auto temp = col;
    col = nullptr;
    return temp;
  }

 private:
  gdf_column *col = nullptr;
};

/**
 * @brief Reads Apache ORC data and returns an array of gdf_columns.
 *
 * @param[in,out] args Structure containing input and output args
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error read_orc(orc_read_arg *args) {

  std::vector<gdf_column_wrapper> columns;
  int num_columns = 0;
  int num_rows = 0;
  int index_col = -1;

  DataSource input(args->source);

  // Select columns

  // Obtain list of stripe chunks

  // Transfer ownership to raw pointer output arguments
  args->data = (gdf_column **)malloc(sizeof(gdf_column *) * num_columns);
  for (int i = 0; i < num_columns; ++i) {
    args->data[i] = columns[i].release();

    // For string dtype, allocate and return an NvStrings container instance,
    // deallocating the original string list memory in the process.
    // This container takes a list of string pointers and lengths, and copies
    // into its own memory so the source memory must not be released yet.
    if (args->data[i]->dtype == GDF_STRING) {
      using str_pair = std::pair<const char *, size_t>;

      auto str_list = static_cast<str_pair *>(args->data[i]->data);
      auto str_data = NVStrings::create_from_index(str_list, num_rows);
      RMM_FREE(std::exchange(args->data[i]->data, str_data), 0);
    }
  }
  args->num_cols_out = num_columns;
  args->num_rows_out = num_rows;
  if (index_col != -1) {
    args->index_col = (int *)malloc(sizeof(int));
    *args->index_col = index_col;
  }

  return GDF_SUCCESS;
}

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "orc.h"
#include "orc_gpu.h"

#include "cudf.h"
#include "io/comp/gpuinflate.h"
#include "io/utilities/wrapper_utils.hpp"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.hpp"

#include <hip/hip_runtime.h>
#include <nvstrings/NVStrings.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include <array>
#include <cstring>
#include <iostream>
#include <numeric>
#include <vector>

#include <fcntl.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#if 1
#define LOG_PRINTF(...) std::printf(__VA_ARGS__)
#else
#define LOG_PRINTF(...) (void)0
#endif

/**
 * @brief Helper class for memory mapping a file source
 **/
class DataSource {
 public:
  explicit DataSource(const char *filepath) {
    fd = open(filepath, O_RDONLY);
    CUDF_EXPECTS(fd > 0, "Cannot open file");

    struct stat st {};
    CUDF_EXPECTS(fstat(fd, &st) == 0, "Cannot query file size");

    mapped_size = st.st_size;
    CUDF_EXPECTS(mapped_size > 0, "Unexpected zero-sized file");

    mapped_data = mmap(NULL, mapped_size, PROT_READ, MAP_PRIVATE, fd, 0);
    CUDF_EXPECTS(mapped_data != MAP_FAILED, "Cannot memory-mapping file");
  }

  ~DataSource() {
    if (mapped_data) {
      munmap(mapped_data, mapped_size);
    }
    if (fd) {
      close(fd);
    }
  }

  const uint8_t *data() const { return static_cast<uint8_t *>(mapped_data); }
  size_t size() const { return mapped_size; }

 private:
  void *mapped_data = nullptr;
  size_t mapped_size = 0;
  int fd = 0;
};

/**
 * @brief Function that translates ORC datatype to GDF dtype
 **/
constexpr std::pair<gdf_dtype, gdf_dtype_extra_info> to_dtype(
    const orc::SchemaType &schema) {
  switch (schema.kind) {
    case orc::BOOLEAN:
    case orc::BYTE:
      return std::make_pair(GDF_INT8, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::SHORT:
      return std::make_pair(GDF_INT16, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::INT:
      return std::make_pair(GDF_INT32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::LONG:
      return std::make_pair(GDF_INT64, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::FLOAT:
      return std::make_pair(GDF_FLOAT32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::DOUBLE:
      return std::make_pair(GDF_FLOAT64, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::STRING:
    case orc::BINARY:
    case orc::VARCHAR:
    case orc::CHAR:
      // Variable-length types can all be mapped to GDF_STRING
      return std::make_pair(GDF_STRING, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::TIMESTAMP:
      return std::make_pair(GDF_TIMESTAMP, gdf_dtype_extra_info{TIME_UNIT_ns});
    case orc::DATE:
      return std::make_pair(GDF_DATE32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::DECIMAL:
      // Currently unhandled as there isn't an explicit mapping
    default:
      break;
  }

  return std::make_pair(GDF_invalid, gdf_dtype_extra_info{TIME_UNIT_NONE});
}

/**
 * @brief A helper class for ORC file metadata. Provides some additional
 * convenience methods for initializing and accessing metadata.
 **/
class OrcMetadata {
 public:
  explicit OrcMetadata(const uint8_t *data_, size_t len_)
      : data(data_), len(len_) {
    const auto ps_length = data[len - 1];
    const auto ps_data = &data[len - ps_length - 1];

    // Read uncompressed postscript section
    orc::ProtobufReader pb;
    pb.init(ps_data, ps_length);
    CUDF_EXPECTS(pb.read(&ps, ps_length), "Cannot read postscript");
    CUDF_EXPECTS(ps.footerLength + ps_length < len, "Invalid footer length");

    LOG_PRINTF("\n[+] PostScript:\n");
    LOG_PRINTF(" postscriptLength = %d\n", ps_length);
    LOG_PRINTF(" footerLength = %zd\n", (size_t)ps.footerLength);
    LOG_PRINTF(" compression = %d\n", ps.compression);
    LOG_PRINTF(" compressionBlockSize = %d\n", ps.compressionBlockSize);
    LOG_PRINTF(" version(%zd) = {%d,%d}\n", ps.version.size(),
               (ps.version.size() > 0) ? (int32_t)ps.version[0] : -1,
               (ps.version.size() > 1) ? (int32_t)ps.version[1] : -1);
    LOG_PRINTF(" metadataLength = %zd\n", (size_t)ps.metadataLength);
    LOG_PRINTF(" magic = \"%s\"\n", ps.magic.c_str());

    // If compression is used, all the rest of the metadata is compressed
    // If no compressed is used, the decompressor is simply a pass-through
    decompressor = std::make_unique<orc::OrcDecompressor>(
        ps.compression, ps.compressionBlockSize);

    // Read compressed filefooter section
    size_t ff_length = 0;
    auto ff_data = decompressor->Decompress(ps_data - ps.footerLength,
                                            ps.footerLength, &ff_length);
    pb.init(ff_data, ff_length);
    CUDF_EXPECTS(pb.read(&ff, ff_length), "Cannot read filefooter");

    LOG_PRINTF("\n[+] FileFooter:\n");
    LOG_PRINTF(" headerLength = %zd\n", (size_t)ff.headerLength);
    LOG_PRINTF(" contentLength = %zd\n", (size_t)ff.contentLength);
    for (size_t i = 0; i < ff.stripes.size(); i++) {
      LOG_PRINTF(
          " stripe #%zd @ %zd: %d rows, index+data+footer: %zd+%zd+%d bytes\n",
          i, (size_t)ff.stripes[i].offset, ff.stripes[i].numberOfRows,
          (size_t)ff.stripes[i].indexLength, (size_t)ff.stripes[i].dataLength,
          ff.stripes[i].footerLength);
    }
    for (size_t i = 0; i < ff.types.size(); i++) {
      LOG_PRINTF(" column %zd: kind=%d, parent=%d\n", i, ff.types[i].kind,
                 ff.types[i].parent_idx);
      if (ff.types[i].subtypes.size() > 0) {
        LOG_PRINTF("   subtypes = ");
        for (int j = 0; j < (int)ff.types[i].subtypes.size(); j++) {
          LOG_PRINTF("%c%d", (j) ? ',' : '{', ff.types[i].subtypes[j]);
        }
        printf("}\n");
      }
      if (ff.types[i].fieldNames.size() > 0) {
        printf("   fieldNames = ");
        for (int j = 0; j < (int)ff.types[i].fieldNames.size(); j++) {
          printf("%c\"%s\"", (j) ? ',' : '{',
                 ff.types[i].fieldNames[j].c_str());
        }
        printf("}\n");
      }
    }
    for (size_t i = 0; i < ff.metadata.size(); i++) {
      printf(" metadata: \"%s\" = \"%s\"\n", ff.metadata[i].name.c_str(),
             ff.metadata[i].value.c_str());
    }
    printf(" numberOfRows = %zd\n", (size_t)ff.numberOfRows);
    printf(" rowIndexStride = %d\n", ff.rowIndexStride);
  }

  void select_stripes(uint64_t min_row, uint64_t num_rows) {
    // Exclude non-needed stripes
    while (ff.stripes.size() > 0 && ff.stripes[0].numberOfRows <= min_row) {
      ff.numberOfRows -= ff.stripes[0].numberOfRows;
      min_row -= ff.stripes[0].numberOfRows;
      ff.stripes.erase(ff.stripes.begin());
    }
    num_rows = std::min(num_rows,
                        ff.numberOfRows - std::min(min_row, ff.numberOfRows));
    if (ff.numberOfRows > num_rows) {
      uint64_t row = 0;
      for (size_t i = 0; i < ff.stripes.size(); i++) {
        if (row >= num_rows) {
          ff.stripes.resize(i);
          ff.numberOfRows = row;
          break;
        }
        row += ff.stripes[i].numberOfRows;
      }
    }

    // Read stripefooter metadata
    sf.resize(ff.stripes.size());
    for (size_t i = 0; i < ff.stripes.size(); ++i) {
      const auto stripe = ff.stripes[i];
      const auto sf_comp_offset =
          stripe.offset + stripe.indexLength + stripe.dataLength;
      const auto sf_comp_length = stripe.footerLength;
      CUDF_EXPECTS(sf_comp_offset + sf_comp_length < len,
                   "Invalid stripe information");

      size_t sf_length = 0;
      auto sf_data = decompressor->Decompress(data + sf_comp_offset,
                                              sf_comp_length, &sf_length);

      orc::ProtobufReader pb;
      pb.init(sf_data, sf_length);
      CUDF_EXPECTS(pb.read(&sf[i], sf_length), "Cannot read stripefooter");

#if VERBOSE_OUTPUT
      printf("StripeFooter(%d/%zd):\n", 1 + i, ff.stripes.size());
      printf(" %d streams:\n", (int)sf.streams.size());
      for (int j = 0; j < (int)sf.streams.size(); j++) {
        printf(" [%d] column=%d, kind=%d, len=%zd\n", j, sf.streams[j].column,
               sf.streams[j].kind, (size_t)sf.streams[j].length);
      }
      printf(" %d columns:\n", (int)sf.columns.size());
      for (int j = 0; j < (int)sf.columns.size(); j++) {
        printf(" [%d] kind=%d, dictionarySize=%d\n", j, sf.columns[j].kind,
               sf.columns[j].dictionarySize);
      }
#endif
    }
  }

  inline int get_total_rows() const { return ff.numberOfRows; }
  inline int get_num_rowgroups() const { return ff.stripes.size(); }
  inline int get_num_columns() const { return ff.types.size(); }

 public:
  orc::PostScript ps;
  orc::FileFooter ff;
  std::vector<orc::StripeFooter> sf;
  std::unique_ptr<orc::OrcDecompressor> decompressor;

 private:
  const uint8_t *const data;
  const size_t len;
};

/**
 * @brief Struct that maps ORC streams to columns
 **/
struct OrcStreamInfo {
  OrcStreamInfo() = default;
  explicit OrcStreamInfo(uint64_t offset_, size_t dst_pos_, uint32_t length_,
                         uint32_t gdf_idx_, uint32_t stripe_idx_)
      : offset(offset_),
        dst_pos(dst_pos_),
        length(length_),
        gdf_idx(gdf_idx_),
        stripe_idx(stripe_idx_) {}
  uint64_t offset;      // offset in file
  size_t dst_pos;       // offset in memory relative to the beginning of the compressed stripe data
  uint32_t length;      // length in file
  uint32_t gdf_idx;     // gdf column index
  uint32_t stripe_idx;  // stripe index
};

/**
 * @brief Decompresses the stripe data, at stream granularity
 *
 * @param[in] chunks List of column chunk descriptors
 * @param[in] pages List of page information
 *
 * @return uint8_t* Device pointer to decompressed page data
 **/
uint8_t *decompress_stripe_data(
    const hostdevice_vector<orc::gpu::ColumnDesc> &chunks,
    const std::vector<device_ptr<uint8_t>> &stripe_data,
    const orc::OrcDecompressor *decompressor,
    std::vector<OrcStreamInfo> &stream_info, size_t num_stripes) {

  // Parse the columns' compressed info
  hostdevice_vector<orc::gpu::CompressedStreamInfo> streams(0,
                                                            stream_info.size());
  for (size_t i = 0; i < streams.max_size(); ++i) {
    streams.insert(orc::gpu::CompressedStreamInfo(
        stripe_data[stream_info[i].stripe_idx].get() + stream_info[i].dst_pos,
        stream_info[i].length));
  }
  CUDA_TRY(hipMemcpyAsync(streams.device_ptr(), streams.host_ptr(),
                           streams.memory_size(), hipMemcpyHostToDevice));
  CUDA_TRY(ParseCompressedStripeData(
      streams.device_ptr(), streams.size(), decompressor->GetBlockSize(),
      decompressor->GetLog2MaxCompressionRatio()));
  CUDA_TRY(hipMemcpyAsync(streams.host_ptr(), streams.device_ptr(),
                           streams.memory_size(), hipMemcpyDeviceToHost));
  CUDA_TRY(hipStreamSynchronize(0));

  // Count the exact number of compressed blocks
  size_t num_compressed_blocks = 0;
  size_t total_decompressed_size = 0;
  for (size_t i = 0; i < streams.size(); ++i) {
    num_compressed_blocks += streams[i].num_compressed_blocks;
    total_decompressed_size += streams[i].max_uncompressed_size;
  }
  CUDF_EXPECTS(total_decompressed_size > 0, "No decompressible data found");

  LOG_PRINTF(
      "[+] Compression\n Total compressed size: %zd\n Number of "
      "compressed blocks: %zd\n Codec: %d\n",
      total_decompressed_size, num_compressed_blocks, decompressor->GetKind());

  uint8_t *decompressed_data = nullptr;
  RMM_ALLOC(&decompressed_data, total_decompressed_size, 0);
  rmm::device_vector<gpu_inflate_input_s> inflate_in(num_compressed_blocks);
  rmm::device_vector<gpu_inflate_status_s> inflate_out(num_compressed_blocks);

  // Parse again to populate the decompression input/output buffers
  size_t decompressed_ofs = 0;
  uint32_t start_pos = 0;
  for (size_t i = 0; i < streams.size(); ++i) {
    streams[i].uncompressed_data = decompressed_data + decompressed_ofs;
    streams[i].decctl = inflate_in.data().get() + start_pos;
    streams[i].decstatus = inflate_out.data().get() + start_pos;
    streams[i].max_compressed_blocks = streams[i].num_compressed_blocks;

    stream_info[i].dst_pos = decompressed_ofs;
    decompressed_ofs += streams[i].max_uncompressed_size;
    start_pos += streams[i].num_compressed_blocks;
  }
  CUDA_TRY(hipMemcpyAsync(streams.device_ptr(), streams.host_ptr(),
                           streams.memory_size(), hipMemcpyHostToDevice));
  CUDA_TRY(ParseCompressedStripeData(
      streams.device_ptr(), streams.size(), decompressor->GetBlockSize(),
      decompressor->GetLog2MaxCompressionRatio()));

  // Dispatch batches of blocks to decompress
  switch (decompressor->GetKind()) {
    case orc::ZLIB:
      CUDA_TRY(gpuinflate(inflate_in.data().get(), inflate_out.data().get(),
                          num_compressed_blocks, 0));
      break;
    case orc::SNAPPY:
      CUDA_TRY(gpu_unsnap(inflate_in.data().get(), inflate_out.data().get(),
                          num_compressed_blocks));
      break;
    default:
      CUDF_EXPECTS(false, "Unexpected decompression dispatch");
      break;
  }
  CUDA_TRY(PostDecompressionReassemble(streams.device_ptr(), streams.size()));

  // Update the stream information with the updated uncompressed info
  // TBD: We could update the value from the information we already
  // have in stream_info[], but using the gpu results also updates
  // max_uncompressed_size to the actual uncompressed size, or zero if
  // decompression failed.
  CUDA_TRY(hipMemcpyAsync(streams.host_ptr(), streams.device_ptr(),
                           streams.memory_size(), hipMemcpyDeviceToHost));
  CUDA_TRY(hipStreamSynchronize(0));

  // const auto num_stripes = md.ff.stripes.size();
  const size_t num_columns = chunks.size() / num_stripes;

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto &desc = chunks[i * num_columns + j];

      using IndexType = std::underlying_type<orc::gpu::StreamType>::type;
      for (IndexType k = 0; k < orc::gpu::CI_NUM_STREAMS; ++k) {
        uint32_t strm_id = desc.strm_id[k];
        if (desc.strm_len[k] > 0 && strm_id < streams.size()) {
          desc.streams[k] = streams[strm_id].uncompressed_data;
          desc.strm_len[k] = (uint32_t)streams[strm_id].max_uncompressed_size;
        }
      }
    }
  }

  return decompressed_data;
}

/**
 * @brief Converts the stripe column data and outputs to gdf_columns
 *
 * @param[in] chunks List of column chunk descriptors
 * @param[in] num_dicts Number of dictionary entries required
 * @param[in,out] columns List of gdf_columns
 **/
void decode_stream_data(const hostdevice_vector<orc::gpu::ColumnDesc> &chunks,
                        size_t num_dicts,
                        const std::vector<gdf_column_wrapper> &columns) {

  const size_t num_columns = columns.size();
  const size_t num_stripes = chunks.size() / columns.size();
  const size_t num_rows = columns[0]->size;

  // Update chunks with pointers to column data
  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto &chunk = chunks[i * num_columns + j];
      chunk.valid_map_base = reinterpret_cast<uint32_t *>(columns[j]->valid);
      chunk.column_data_base = columns[j]->data;
      chunk.dtype_len = (columns[j]->dtype == GDF_STRING)
                           ? sizeof(std::pair<const char *, size_t>)
                           : gdf_dtype_size(columns[j]->dtype);
    }
  }

  // Allocate global dictionary for deserializing
  rmm::device_vector<orc::gpu::DictionaryEntry> global_dict(num_dicts);

  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                           chunks.memory_size(), hipMemcpyHostToDevice));
  CUDA_TRY(DecodeNullsAndStringDictionaries(
      chunks.device_ptr(), global_dict.data().get(), num_columns, num_stripes,
      num_rows, 0));
  CUDA_TRY(DecodeOrcColumnData(chunks.device_ptr(), global_dict.data().get(),
                               num_columns, num_stripes, num_rows, 0));
  CUDA_TRY(hipMemcpyAsync(chunks.host_ptr(), chunks.device_ptr(),
                           chunks.memory_size(), hipMemcpyDeviceToHost));
  CUDA_TRY(hipStreamSynchronize(0));

  LOG_PRINTF("[+] Decoded Column Information\n");
  for (size_t i = 0; i < num_columns; ++i) {
    for (size_t j = 0; j < num_stripes; ++j) {
      columns[i]->null_count += chunks[j * num_columns + i].null_count;
    }
    LOG_PRINTF(
        "columns[%zd].null_count = %d/%d (start_row=%d, nrows=%d, "
        "strm_len=%d)\n",
        i, columns[i]->null_count, columns[i]->size, chunks[i].start_row,
        chunks[i].num_rows, chunks[i].strm_len[orc::gpu::CI_PRESENT]);
  }
}

/**
 * @brief Reads Apache ORC data and returns an array of gdf_columns.
 *
 * @param[in,out] args Structure containing input and output args
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error read_orc(orc_read_arg *args) {

  int num_columns = 0;
  int num_rows = 0;

  DataSource input(args->source);

  OrcMetadata md(input.data(), input.size());
  CUDF_EXPECTS(md.get_num_columns() > 0, "No columns found");

  static_assert(sizeof(orc::gpu::CompressedStreamInfo) <= 256 &&
                    !(sizeof(orc::gpu::CompressedStreamInfo) & 7),
                "Unexpected sizeof(CompressedStreamInfo)");
  static_assert(sizeof(orc::gpu::ColumnDesc) <= 256 &&
                    !(sizeof(orc::gpu::ColumnDesc) & 7),
                "Unexpected sizeof(ColumnDesc)");

  // Select only rowgroups required
  md.select_stripes(0, 0x7fffffff);

  // Select only columns required (if it exists), otherwise select all
  std::vector<int32_t> gdf2orc;                           // Map gdf columns to orc columns
  std::vector<int32_t> orc2gdf(md.get_num_columns(), -1); // Map orc columns to gdf columns
  if (args->use_cols) {
    std::vector<std::string> use_names(args->use_cols,
                                       args->use_cols + args->use_cols_len);
    int index = 0;
    for (const auto &use_name : use_names) {
      for (int i = 0; i < md.get_num_columns(); ++i, ++index) {
        if (index >= md.get_num_columns()) {
          index = 0;
        }
        if (md.ff.GetColumnName(index) == use_name) {
          orc2gdf[index] = gdf2orc.size();
          gdf2orc.emplace_back(index);
          index++;
        }
      }
    }
  } else {
    // For now, only select all leaf nodes
    for (int i = 0; i < md.get_num_columns(); ++i) {
      if (md.ff.types[i].subtypes.size() == 0) {
        orc2gdf[i] = gdf2orc.size();
        gdf2orc.emplace_back(i);
      }
    }
  }

  // Initialize gdf_columns, but hold off on allocating storage space
  std::vector<gdf_column_wrapper> columns;
  LOG_PRINTF("[+] Selected columns: %d\n", num_columns);
  for (const auto &col : gdf2orc) {
    auto dtype_info = to_dtype(md.ff.types[col]);

    columns.emplace_back(static_cast<gdf_size_type>(md.ff.numberOfRows),
                         dtype_info.first, dtype_info.second,
                         md.ff.GetColumnName(col));

    LOG_PRINTF(" %2zd: name=%s size=%zd type=%d data=%lx valid=%lx\n",
               columns.size() - 1, columns.back()->col_name,
               (size_t)columns.back()->size, columns.back()->dtype,
               (uint64_t)columns.back()->data, (uint64_t)columns.back()->valid);
  }

  num_rows = md.get_total_rows();
  num_columns = (int)gdf2orc.size();

  // Logically view streams as columns
  std::vector<OrcStreamInfo> stream_info;

  // Tracker for eventually deallocating compressed and uncompressed data
  std::vector<device_ptr<uint8_t>> stripe_data;

  if (num_rows > 0 && num_columns > 0) {
    const auto num_column_chunks = md.ff.stripes.size() * num_columns;
    hostdevice_vector<orc::gpu::ColumnDesc> chunks(num_column_chunks);

    // Read stripe footers
    size_t total_compressed_size = 0;
    size_t stripe_start_row = 0;
    uint32_t num_dictionary_entries = 0;

    for (size_t i = 0; i < md.sf.size(); i++) {
      size_t strm_count;
      uint64_t src_offset, dst_offset, index_length;
      uint8_t *data_dev = nullptr;

      // Read stream data
      src_offset = 0;
      dst_offset = 0;
      index_length = md.ff.stripes[i].indexLength;
      strm_count = stream_info.size();
      for (int j = 0; j < (int)md.sf[i].streams.size(); j++) {
        uint32_t strm_length = (uint32_t)md.sf[i].streams[j].length;
        uint32_t column_id = md.sf[i].streams[j].column;
        int32_t gdf_idx = -1;
        if (column_id < orc2gdf.size()) {
          gdf_idx = orc2gdf[column_id];
          if (gdf_idx < 0 && md.ff.types[column_id].subtypes.size() != 0) {
            // This column may be a parent column, in which case the PRESENT
            // stream may be needed
            bool needed = (md.ff.types[column_id].kind == orc::STRUCT &&
                           md.sf[i].streams[j].kind == orc::PRESENT);
            if (needed) {
              for (int k = 0; k < (int)md.ff.types[column_id].subtypes.size();
                   k++) {
                uint32_t idx = md.ff.types[column_id].subtypes[k];
                int32_t child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
                if (child_idx >= 0) {
                  gdf_idx = child_idx;
                  chunks[i * num_columns + gdf_idx]
                      .strm_id[orc::gpu::CI_PRESENT] =
                      (uint32_t)stream_info.size();
                  chunks[i * num_columns + gdf_idx]
                      .strm_len[orc::gpu::CI_PRESENT] = strm_length;
                }
              }
            }
          }
        }
        if (src_offset >= index_length && gdf_idx >= 0) {
          int ci_kind = orc::gpu::CI_NUM_STREAMS;
          switch (md.sf[i].streams[j].kind) {
            case orc::DATA:
              ci_kind = orc::gpu::CI_DATA;
              break;
            case orc::LENGTH:
            case orc::SECONDARY:
              ci_kind = orc::gpu::CI_DATA2;
              break;
            case orc::DICTIONARY_DATA:
              ci_kind = orc::gpu::CI_DICTIONARY;
              chunks[i * num_columns + gdf_idx].dictionary_start =
                  num_dictionary_entries;
              chunks[i * num_columns + gdf_idx].dict_len =
                  md.sf[i].columns[column_id].dictionarySize;
              num_dictionary_entries +=
                  md.sf[i].columns[column_id].dictionarySize;
              break;
            case orc::PRESENT:
              ci_kind = orc::gpu::CI_PRESENT;
              break;
            default:
              // TBD: Could skip loading this stream
              break;
          }
          if (ci_kind < orc::gpu::CI_NUM_STREAMS) {
            chunks[i * num_columns + gdf_idx].strm_id[ci_kind] =
                (uint32_t)stream_info.size();
            chunks[i * num_columns + gdf_idx].strm_len[ci_kind] = strm_length;
          }
        }
        if (gdf_idx >= 0) {
          stream_info.emplace_back(md.ff.stripes[i].offset + src_offset,
                                   dst_offset, strm_length, gdf_idx, i);
          dst_offset += strm_length;
        }
        src_offset += strm_length;
      }
      if (dst_offset > 0) {
        RMM_TRY(RMM_ALLOC((void **)&data_dev, dst_offset, 0));

        while (strm_count < stream_info.size()) {
          // Coalesce consecutive streams into one read
          uint64_t len = stream_info[strm_count].length;
          uint64_t offset = stream_info[strm_count].offset;
          void *dst = data_dev + stream_info[strm_count].dst_pos;
          strm_count++;
          while (strm_count < stream_info.size() &&
                 stream_info[strm_count].offset == offset + len) {
            len += stream_info[strm_count].length;
            strm_count++;
          }
          hipMemcpyAsync(dst, input.data() + offset, len,
                          hipMemcpyHostToDevice,
                          0);  // TODO: datasource::gpuread
          total_compressed_size += len;
        }
        // Update stream pointers
        for (int j = 0; j < num_columns; j++) {
          for (int k = 0; k < orc::gpu::CI_NUM_STREAMS; k++) {
            if (chunks[i * num_columns + j].strm_len[k] > 0) {
              uint32_t strm_id = chunks[i * num_columns + j].strm_id[k];
              chunks[i * num_columns + j].streams[k] =
                  data_dev + stream_info[strm_id].dst_pos;
            }
          }
          chunks[i * num_columns + j].start_row = (uint32_t)stripe_start_row;
          chunks[i * num_columns + j].num_rows = md.ff.stripes[i].numberOfRows;
          chunks[i * num_columns + j].encoding_kind =
              md.sf[i].columns[gdf2orc[j]].kind;
          chunks[i * num_columns + j].type_kind = md.ff.types[gdf2orc[j]].kind;
        }
      }
      stripe_data.emplace_back(data_dev);
      stripe_start_row += md.ff.stripes[i].numberOfRows;
    }

    // Deallocate and replace compressed data with decompressed data
    if (md.ps.compression != orc::NONE) {
      uint8_t *d_decomp_data =
          decompress_stripe_data(chunks, stripe_data, md.decompressor.get(),
                                 stream_info, md.ff.stripes.size());
      stripe_data.clear();
      stripe_data.emplace_back(d_decomp_data);
    }

    for (auto &column : columns) {
      CUDF_EXPECTS(column.allocate() == GDF_SUCCESS, "Cannot allocate columns");
    }
    decode_stream_data(chunks, num_dictionary_entries, columns);
  } else {
    // Columns' data's memory is still expected for an empty dataframe
    for (auto &column : columns) {
      CUDF_EXPECTS(column.allocate() == GDF_SUCCESS, "Cannot allocate columns");
    }
  }

  // Transfer ownership to raw pointer output arguments
  args->data = (gdf_column **)malloc(sizeof(gdf_column *) * num_columns);
  for (int i = 0; i < num_columns; ++i) {
    args->data[i] = columns[i].release();

    // For string dtype, allocate and return an NvStrings container instance,
    // deallocating the original string list memory in the process.
    // This container takes a list of string pointers and lengths, and copies
    // into its own memory so the source memory must not be released yet.
    if (args->data[i]->dtype == GDF_STRING) {
      using str_pair = std::pair<const char *, size_t>;

      auto str_list = static_cast<str_pair *>(args->data[i]->data);
      auto str_data = NVStrings::create_from_index(str_list, num_rows);
      RMM_FREE(std::exchange(args->data[i]->data, str_data), 0);
    }
  }
  args->num_cols_out = num_columns;
  args->num_rows_out = num_rows;
  args->index_col = nullptr;

  return GDF_SUCCESS;
}

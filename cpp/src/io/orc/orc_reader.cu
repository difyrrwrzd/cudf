#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "orc.h"
#include "orc_gpu.h"

#include "cudf.h"
#include "io/comp/gpuinflate.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.hpp"

#include <hip/hip_runtime.h>
#include <nvstrings/NVStrings.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include <array>
#include <cstring>
#include <iostream>
#include <numeric>
#include <vector>

#include <fcntl.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#if 1
#define LOG_PRINTF(...) std::printf(__VA_ARGS__)
#else
#define LOG_PRINTF(...) (void)0
#endif

/**
 * @brief Helper class for memory mapping a file source
 **/
class DataSource {
 public:
  explicit DataSource(const char *filepath) {
    fd = open(filepath, O_RDONLY);
    CUDF_EXPECTS(fd > 0, "Cannot open file");

    struct stat st {};
    CUDF_EXPECTS(fstat(fd, &st) == 0, "Cannot query file size");

    mapped_size = st.st_size;
    CUDF_EXPECTS(mapped_size > 0, "Unexpected zero-sized file");

    mapped_data = mmap(NULL, mapped_size, PROT_READ, MAP_PRIVATE, fd, 0);
    CUDF_EXPECTS(mapped_data != MAP_FAILED, "Cannot memory-mapping file");
  }

  ~DataSource() {
    if (mapped_data) {
      munmap(mapped_data, mapped_size);
    }
    if (fd) {
      close(fd);
    }
  }

  const uint8_t *data() const { return static_cast<uint8_t *>(mapped_data); }
  size_t size() const { return mapped_size; }

 private:
  void *mapped_data = nullptr;
  size_t mapped_size = 0;
  int fd = 0;
};

size_t GetGDFTypeLength(gdf_dtype dtype)
{
    size_t dtype_len = 0;
    switch (dtype)
    {
    case GDF_INT8:
        dtype_len = 1;
        break;
    case GDF_INT16:
        dtype_len = 2;
        break;
    case GDF_INT32:
    case GDF_FLOAT32:
    case GDF_DATE32:
    case GDF_CATEGORY: // NOTE: Category type converts the underlying string type into a 32-bit hash
        dtype_len = 4;
        break;
    case GDF_INT64:
    case GDF_FLOAT64:
    case GDF_DATE64:
    case GDF_TIMESTAMP:
        dtype_len = 8;
        break;
    // NOTE: String returns the size of the std::pair needed to create the nvStrings array
    case GDF_STRING:
        dtype_len = sizeof(std::pair<char *,size_t>); // For now, just the index
        break;
    default:
        return 0;
    }
    return dtype_len;
}

/**
 * @brief A helper class that wraps a gdf_column and any associated memory.
 *
 * This abstraction initializes and manages a gdf_column (fields and memory)
 * while still allowing direct access. Memory is automatically deallocated
 * unless ownership is transferred via releasing and assigning the raw pointer.
 **/
class gdf_column_wrapper {
 public:
  gdf_column_wrapper(gdf_size_type size, gdf_dtype dtype,
                     gdf_dtype_extra_info dtype_info, const std::string name) {
    col = (gdf_column *)malloc(sizeof(gdf_column));
    col->col_name = (char *)malloc(name.length() + 1);
    strcpy(col->col_name, name.c_str());
    gdf_column_view_augmented(col, nullptr, nullptr, size, dtype, 0, dtype_info);
  }

  ~gdf_column_wrapper() {
    if (col) {
      RMM_FREE(col->data, 0);
      RMM_FREE(col->valid, 0);
      free(col->col_name);
    }
    free(col);
  };

  gdf_column_wrapper(const gdf_column_wrapper &other) = delete;
  gdf_column_wrapper(gdf_column_wrapper &&other) : col(other.col) {
    other.col = nullptr;
  }

  gdf_error allocate() {
    // For strings, just store the ptr + length. Eventually, column's data ptr
    // is replaced with an NvString instance created from these pairs.
    const auto num_rows = std::max(col->size, 1);
    const auto column_byte_width = (col->dtype == GDF_STRING)
                                       ? sizeof(std::pair<const char*, int>)
                                       : gdf_dtype_size(col->dtype);

    RMM_TRY(RMM_ALLOC(&col->data, num_rows * column_byte_width, 0));
    RMM_TRY(RMM_ALLOC(&col->valid, gdf_valid_allocation_size(num_rows), 0));
    CUDA_TRY(hipMemset(col->valid, 0, gdf_valid_allocation_size(num_rows)));

    return GDF_SUCCESS;
  }

  gdf_column *operator->() const { return col; }
  gdf_column *get() const { return col; }
  gdf_column *release() {
    auto temp = col;
    col = nullptr;
    return temp;
  }

 private:
  gdf_column *col = nullptr;
};

/**
 * @brief Function that translates Parquet datatype to GDF dtype
 **/
constexpr std::pair<gdf_dtype, gdf_dtype_extra_info> to_dtype(
    const orc::SchemaType &schema) {
  switch (schema.kind) {
    case orc::BOOLEAN:
    case orc::BYTE:
      return std::make_pair(GDF_INT8, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::SHORT:
      return std::make_pair(GDF_INT16, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::INT:
      return std::make_pair(GDF_INT32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::LONG:
      return std::make_pair(GDF_INT64, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::FLOAT:
      return std::make_pair(GDF_FLOAT32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::DOUBLE:
      return std::make_pair(GDF_FLOAT64, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::STRING:
    case orc::BINARY:
    case orc::VARCHAR:
    case orc::CHAR:
      // Variable-length types can all be mapped to GDF_STRING
      return std::make_pair(GDF_STRING, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::TIMESTAMP:
      return std::make_pair(GDF_TIMESTAMP, gdf_dtype_extra_info{TIME_UNIT_ns});
    case orc::DATE:
      return std::make_pair(GDF_DATE32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::DECIMAL:
      // Currently unhandled as there isn't an explicit mapping
    default:
      break;
  }

  return std::make_pair(GDF_invalid, gdf_dtype_extra_info{TIME_UNIT_NONE});
}

// Map orc streams to columns
struct OrcStrmInfo
{
    uint64_t offset;        // offset in file
    size_t dst_pos;         // offset in memory relative to the beginning of the compressed stripe data
    uint32_t length;        // length in file
    uint32_t gdf_idx;       // gdf column index
    uint32_t stripe_idx;    // stripe index
};

/**
 * @brief Reads Apache ORC data and returns an array of gdf_columns.
 *
 * @param[in,out] args Structure containing input and output args
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error read_orc(orc_read_arg *args) {

  std::vector<gdf_column_wrapper> columns;
  //int num_columns = 0;
  //int num_rows = 0;
  int index_col = -1;

  DataSource input(args->source);

  // Select columns
  int postscript_length = input.data()[input.size() - 1];
  const uint8_t *postscript = &input.data()[input.size() - postscript_length - 1];
  orc::PostScript ps;
  orc::FileFooter ff;
  orc::ProtobufReader pb;
  const uint8_t *uncompressed_footer;
  size_t footer_length, total_compressed_size;

  int num_streams, num_columns;
  orc::gpu::CompressedStreamInfo *strm_desc = nullptr, *strm_desc_dev = nullptr;
  std::vector<OrcStrmInfo> stream_info;
  std::vector<uint8_t *> compressed_stripe_data, uncompressed_stripe_data;
  std::vector<int32_t> gdf2orc;       // Map gdf columns to orc columns
  std::vector<int32_t> orc2gdf;       // Map orc columns to gdf columns
  orc::gpu::ColumnDesc *chunks = nullptr, *chunks_dev = nullptr;
  orc::gpu::DictionaryEntry *global_dictionary = nullptr;
  size_t stripe_start_row;
  uint32_t num_dictionary_entries;
  uint64_t first_row = 0;
  uint64_t num_rows = 0x7fffffff;

  static_assert(sizeof(orc::gpu::CompressedStreamInfo) <= 256 && !(sizeof(orc::gpu::CompressedStreamInfo) & 7), "Unexpected sizeof(CompressedStreamInfo)");
  static_assert(sizeof(orc::gpu::ColumnDesc) <= 256 && !(sizeof(orc::gpu::ColumnDesc) & 7), "Unexpected sizeof(ColumnDesc)");
  printf("postscript length = %d\n", postscript_length);
  pb.init(postscript, postscript_length);
  CUDF_EXPECTS(pb.read(&ps, postscript_length),
              "Failed to read postscript metadata");
  printf("PostScript:\n");
  printf(" footerLength = %zd\n", (size_t)ps.footerLength);
  printf(" compression = %d\n", ps.compression);
  printf(" compressionBlockSize = %d\n", ps.compressionBlockSize);
  printf(" version(%zd) = {%d,%d}\n", ps.version.size(), (ps.version.size() > 0) ? (int32_t)ps.version[0] : -1, (ps.version.size() > 1) ? (int32_t)ps.version[1] : -1);
  printf(" metadataLength = %zd\n", (size_t)ps.metadataLength);
  printf(" magic = \"%s\"\n", ps.magic.c_str());
  CUDF_EXPECTS(ps.footerLength + postscript_length < input.size(), "Invalid footer length");

  orc::OrcDecompressor decompressor(ps.compression, ps.compressionBlockSize);
  uncompressed_footer = decompressor.Decompress(postscript - ps.footerLength, ps.footerLength, &footer_length);
  CUDF_EXPECTS(uncompressed_footer, "Failed to uncompress file footer");
  pb.init(uncompressed_footer, footer_length);
  CUDF_EXPECTS(pb.read(&ff, footer_length), "Failed to read file footer");
  printf("FileFooter:\n");
  printf(" headerLength = %zd\n", (size_t)ff.headerLength);
  printf(" contentLength = %zd\n", (size_t)ff.contentLength);
  for (int i = 0; i < (int)ff.stripes.size(); i++)
  {
      printf(" stripe #%d @ %zd: %d rows, index+data+footer: %zd+%zd+%d bytes\n", i, (size_t)ff.stripes[i].offset, ff.stripes[i].numberOfRows, (size_t)ff.stripes[i].indexLength, (size_t)ff.stripes[i].dataLength, ff.stripes[i].footerLength);
  }
  for (int i = 0; i < (int)ff.types.size(); i++)
  {
      printf(" column %d: kind=%d, parent=%d\n", i, ff.types[i].kind, ff.types[i].parent_idx);
      if (ff.types[i].subtypes.size() > 0)
      {
          printf("   subtypes = ");
          for (int j = 0; j < (int)ff.types[i].subtypes.size(); j++)
          {
              printf("%c%d", (j) ? ',' : '{', ff.types[i].subtypes[j]);
          }
          printf("}\n");
      }
      if (ff.types[i].fieldNames.size() > 0)
      {
          printf("   fieldNames = ");
          for (int j = 0; j < (int)ff.types[i].fieldNames.size(); j++)
          {
              printf("%c\"%s\"", (j) ? ',' : '{', ff.types[i].fieldNames[j].c_str());
          }
          printf("}\n");
      }
  }
  for (int i = 0; i < (int)ff.metadata.size(); i++)
  {
      printf(" metadata: \"%s\" = \"%s\"\n", ff.metadata[i].name.c_str(), ff.metadata[i].value.c_str());
  }
  printf(" numberOfRows = %zd\n", (size_t)ff.numberOfRows);
  printf(" rowIndexStride = %d\n", ff.rowIndexStride);
  // Modify the footer to exclude non-needed stripes
  while (ff.stripes.size() > 0 && ff.stripes[0].numberOfRows <= first_row)
  {
      ff.numberOfRows -= ff.stripes[0].numberOfRows;
      first_row -= ff.stripes[0].numberOfRows;
      ff.stripes.erase(ff.stripes.begin());
  }
  num_rows = std::min(num_rows, ff.numberOfRows - std::min(first_row, ff.numberOfRows));
  if (ff.numberOfRows > num_rows)
  {
      uint64_t row = 0;
      for (size_t i = 0; i < ff.stripes.size(); i++)
      {
          if (row >= num_rows)
          {
              ff.stripes.resize(i);
              ff.numberOfRows = row;
              break;
          }
          row += ff.stripes[i].numberOfRows;
      }
  }
  // Select columns
  orc2gdf.resize(ff.types.size(), -1);
  if (args->use_cols_len > 0)
  {
      // Find columns by name
      gdf2orc.resize(args->use_cols_len);
      for (int i = 0, column_id = 0; i < args->use_cols_len; i++)
      {
          int num_orc_columns = (int)ff.types.size();
          gdf2orc[i] = -1;
          for (int j = 0; j < num_orc_columns; j++, column_id++)
          {
              if (column_id >= num_orc_columns)
              {
                  column_id = 0;
              }
              if (ff.GetColumnName(column_id) == args->use_cols[i])
              {
                  gdf2orc[i] = column_id;
                  orc2gdf[column_id] = i;
                  column_id++;
                  break;
              }
          }
          if (gdf2orc[i] < 0)
          {
              printf("Column not found: \"%s\"\n", args->use_cols[i]);
              return GDF_FILE_ERROR;
          }
      }
  }
  else
  {
      // Select all columns
      for (int i = 0; i < (int)ff.types.size(); i++)
      {
          bool col_en = (ff.types[i].subtypes.size() == 0); // For now, select all leaf nodes in the schema
          if (col_en)
          {
              int32_t gdf_idx = (int32_t)gdf2orc.size();
              gdf2orc.resize(gdf_idx + 1);
              gdf2orc[gdf_idx] = i;
              orc2gdf[i] = gdf_idx;
          }
      }
  }
  // Allocate gdf columns
  num_columns = (int)gdf2orc.size();
  for (int i = 0; i < num_columns; i++)
  {
    auto dtype_info = to_dtype(ff.types[gdf2orc[i]]);

    columns.emplace_back(static_cast<gdf_size_type>(ff.numberOfRows),
                         dtype_info.first, dtype_info.second,
                         ff.GetColumnName(gdf2orc[i]));

    LOG_PRINTF(" %2zd: name=%s size=%zd type=%d data=%lx valid=%lx\n",
               columns.size() - 1, columns.back()->col_name,
               (size_t)columns.back()->size, columns.back()->dtype,
               (uint64_t)columns.back()->data, (uint64_t)columns.back()->valid);
  }

  if (num_rows > 0 && num_columns > 0) {
    // Allocate row index: essentially 2D array indexed by stripe id & gdf column index
    hipHostMalloc((void **)&chunks, ff.stripes.size() * num_columns * sizeof(chunks[0]));
    RMM_ALLOC((void **)&chunks_dev, ff.stripes.size() * num_columns * sizeof(chunks_dev[0]), 0);
    if (!(chunks && chunks_dev))
        goto error_exit;
    memset(chunks, 0, ff.stripes.size() * num_columns * sizeof(chunks[0]));
    // Read stripe footers
    total_compressed_size = 0;
    stripe_start_row = 0;
    num_dictionary_entries = 0;
    for (int i = 0; i < (int)ff.stripes.size(); i++)
    {
        size_t sfooter_offset = ff.stripes[i].offset + ff.stripes[i].indexLength + ff.stripes[i].dataLength;
        size_t sfooter_length = ff.stripes[i].footerLength;
        orc::StripeFooter sf;
        const uint8_t *uncomp;
        size_t uncomp_len = 0, strm_count;
        uint64_t src_offset, dst_offset, index_length;
        uint8_t *data_dev = nullptr;

        if (sfooter_offset + sfooter_length >= input.size())
        {
            printf("Invalid stripe information\n");
            return GDF_CUDA_ERROR;
        }
        uncomp = decompressor.Decompress(input.data() + sfooter_offset, sfooter_length, &uncomp_len);
        pb.init(uncomp, uncomp_len);
        pb.read(&sf, uncomp_len);
    #if VERBOSE_OUTPUT
        printf("StripeFooter(%d/%zd):\n", 1+i, ff.stripes.size());
        printf(" %d streams:\n", (int)sf.streams.size());
        for (int j = 0; j < (int)sf.streams.size(); j++)
        {
            printf(" [%d] column=%d, kind=%d, len=%zd\n", j, sf.streams[j].column, sf.streams[j].kind, (size_t)sf.streams[j].length);
        }
        printf(" %d columns:\n", (int)sf.columns.size());
        for (int j = 0; j < (int)sf.columns.size(); j++)
        {
            printf(" [%d] kind=%d, dictionarySize=%d\n", j, sf.columns[j].kind, sf.columns[j].dictionarySize);
        }
    #endif
        // Read stream data
        src_offset = 0;
        dst_offset = 0;
        index_length = ff.stripes[i].indexLength;
        strm_count = stream_info.size();
        for (int j = 0; j < (int)sf.streams.size(); j++)
        {
            uint32_t strm_length = (uint32_t)sf.streams[j].length;
            uint32_t column_id = sf.streams[j].column;
            int32_t gdf_idx = -1;
            if (column_id < orc2gdf.size())
            {
                gdf_idx = orc2gdf[column_id];
                if (gdf_idx < 0 && ff.types[column_id].subtypes.size() != 0)
                {
                    // This column may be a parent column, in which case the PRESENT stream may be needed
                    bool needed = (ff.types[column_id].kind == orc::STRUCT && sf.streams[j].kind == orc::PRESENT);
                    if (needed)
                    {
                        for (int k = 0; k < (int)ff.types[column_id].subtypes.size(); k++)
                        {
                            uint32_t idx = ff.types[column_id].subtypes[k];
                            int32_t child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
                            if (child_idx >= 0)
                            {
                                gdf_idx = child_idx;
                                chunks[i * num_columns + gdf_idx].strm_id[orc::gpu::CI_PRESENT] = (uint32_t)stream_info.size();
                                chunks[i * num_columns + gdf_idx].strm_len[orc::gpu::CI_PRESENT] = strm_length;
                            }
                        }
                    }
                }
            }
            if (src_offset >= index_length && gdf_idx >= 0)
            {
                int ci_kind = orc::gpu::CI_NUM_STREAMS;
                switch (sf.streams[j].kind)
                {
                case orc::DATA:
                    ci_kind = orc::gpu::CI_DATA;
                    break;
                case orc::LENGTH:
                case orc::SECONDARY:
                    ci_kind = orc::gpu::CI_DATA2;
                    break;
                case orc::DICTIONARY_DATA:
                    ci_kind = orc::gpu::CI_DICTIONARY;
                    chunks[i * num_columns + gdf_idx].dictionary_start = num_dictionary_entries;
                    chunks[i * num_columns + gdf_idx].dict_len = sf.columns[column_id].dictionarySize;
                    num_dictionary_entries += sf.columns[column_id].dictionarySize;
                    break;
                case orc::PRESENT:
                    ci_kind = orc::gpu::CI_PRESENT;
                    break;
                default:
                    // TBD: Could skip loading this stream
                    break;
                }
                if (ci_kind < orc::gpu::CI_NUM_STREAMS)
                {
                    chunks[i * num_columns + gdf_idx].strm_id[ci_kind] = (uint32_t)stream_info.size();
                    chunks[i * num_columns + gdf_idx].strm_len[ci_kind] = strm_length;
                }
            }
            if (gdf_idx >= 0)
            {
                OrcStrmInfo info;
                info.offset = ff.stripes[i].offset + src_offset;
                info.length = strm_length;
                info.dst_pos = dst_offset;
                info.gdf_idx = gdf_idx;
                info.stripe_idx = i;
                stream_info.push_back(info); // FIXME: Use emplace_back
                dst_offset += strm_length;
            }
            src_offset += strm_length;
        }
        if (dst_offset > 0)
        {
            RMM_ALLOC((void **)&data_dev, dst_offset, 0);
            if (!data_dev)
                goto error_exit;
            while (strm_count < stream_info.size())
            {
                // Coalesce consecutive streams into one read
                uint64_t len = stream_info[strm_count].length;
                uint64_t offset = stream_info[strm_count].offset;
                void *dst = data_dev + stream_info[strm_count].dst_pos;
                strm_count++;
                while (strm_count < stream_info.size() && stream_info[strm_count].offset == offset + len)
                {
                    len += stream_info[strm_count].length;
                    strm_count++;
                }
                hipMemcpyAsync(dst, input.data() + offset, len, hipMemcpyHostToDevice, 0); // TODO: datasource::gpuread
                total_compressed_size += len;
            }
            // Update stream pointers
            for (int j = 0; j < num_columns; j++)
            {
                for (int k = 0; k < orc::gpu::CI_NUM_STREAMS; k++)
                {
                    if (chunks[i * num_columns + j].strm_len[k] > 0)
                    {
                        uint32_t strm_id = chunks[i * num_columns + j].strm_id[k];
                        chunks[i * num_columns + j].streams[k] = data_dev + stream_info[strm_id].dst_pos;
                    }
                }
                chunks[i * num_columns + j].start_row = (uint32_t)stripe_start_row;
                chunks[i * num_columns + j].num_rows = ff.stripes[i].numberOfRows;
                chunks[i * num_columns + j].encoding_kind = sf.columns[gdf2orc[j]].kind;
                chunks[i * num_columns + j].type_kind = ff.types[gdf2orc[j]].kind;
            }
        }
        compressed_stripe_data.push_back(data_dev);
        stripe_start_row += ff.stripes[i].numberOfRows;
    }

    printf("[CPU] Read %zd bytes\n", total_compressed_size);
    // Allocate global dictionary
    if (num_dictionary_entries > 0)
    {
        RMM_ALLOC((void **)&global_dictionary, num_dictionary_entries * sizeof(orc::gpu::DictionaryEntry), 0);
    }
    // Setup decompression
    num_streams = (int)stream_info.size();
    printf(" %d data streams, %d dictionary entries\n", num_streams, num_dictionary_entries);
    if (ps.compression != orc::NONE)
    {
        uint32_t total_compressed_blocks;
        size_t total_uncompressed_size;
        double decompression_time = 0;
        hipHostMalloc((void **)&strm_desc, num_streams * sizeof(orc::gpu::CompressedStreamInfo));
        RMM_ALLOC((void **)&strm_desc_dev, num_streams * sizeof(orc::gpu::CompressedStreamInfo), 0);
        if (!(strm_desc && strm_desc_dev))
            goto error_exit;
        for (int i = 0; i < num_streams; i++)
        {
            strm_desc[i].compressed_data = compressed_stripe_data[stream_info[i].stripe_idx] + stream_info[i].dst_pos;
            strm_desc[i].uncompressed_data = nullptr;
            strm_desc[i].compressed_data_size = stream_info[i].length;
            strm_desc[i].decctl = nullptr;
            strm_desc[i].max_compressed_blocks = 0;
            strm_desc[i].num_compressed_blocks = 0;
            strm_desc[i].max_uncompressed_size = 0;
        }
        hipMemcpyAsync(strm_desc_dev, strm_desc, num_streams * sizeof(orc::gpu::CompressedStreamInfo), hipMemcpyHostToDevice);
        ParseCompressedStripeData(strm_desc_dev, num_streams, ps.compressionBlockSize, decompressor.GetLog2MaxCompressionRatio());
        hipMemcpyAsync(strm_desc, strm_desc_dev, num_streams * sizeof(orc::gpu::CompressedStreamInfo), hipMemcpyDeviceToHost);
        hipStreamSynchronize(0);
        total_compressed_blocks = 0;
        total_uncompressed_size = 0;
        for (int i = 0; i < num_streams; i++)
        {
            total_compressed_blocks += strm_desc[i].num_compressed_blocks;
            total_uncompressed_size += strm_desc[i].max_uncompressed_size;
        }
        if (total_uncompressed_size > 0)
        {
            uint8_t *uncompressed_data = nullptr;
            gpu_inflate_input_s *inflate_in = nullptr;
            gpu_inflate_status_s *inflate_out = nullptr;
            size_t uncomp_ofs;
            printf("%d compressed blocks, max_uncompressed_size=%zd\n", total_compressed_blocks, total_uncompressed_size);
            RMM_ALLOC((void **)&uncompressed_data, total_uncompressed_size, 0);
            if (!uncompressed_data)
                goto error_exit;
            uncompressed_stripe_data.push_back(uncompressed_data);
            if (total_compressed_blocks > 0)
            {
                RMM_ALLOC((void **)&inflate_in, total_compressed_blocks * (sizeof(gpu_inflate_input_s) + sizeof(gpu_inflate_status_s)), 0);
                if (!inflate_in)
                    goto error_exit;
                inflate_out = reinterpret_cast<gpu_inflate_status_s *>(inflate_in + total_compressed_blocks);
                hipMemsetAsync(inflate_out, 0, total_compressed_blocks * sizeof(gpu_inflate_status_s));
            }
            uncomp_ofs = 0;
            for (int i = 0, pos = 0; i < num_streams; i++)
            {
                strm_desc[i].uncompressed_data = uncompressed_data + uncomp_ofs;
                strm_desc[i].decctl = inflate_in + pos;
                strm_desc[i].decstatus = inflate_out + pos;
                strm_desc[i].max_compressed_blocks = strm_desc[i].num_compressed_blocks;
                stream_info[i].dst_pos = uncomp_ofs; // Now indicates the offset relative to base uncompressed data
                uncomp_ofs += strm_desc[i].max_uncompressed_size;
                pos += strm_desc[i].num_compressed_blocks;
            }
            // Parse again, this time populating the decompression input/output buffers
            hipMemcpyAsync(strm_desc_dev, strm_desc, num_streams * sizeof(orc::gpu::CompressedStreamInfo), hipMemcpyHostToDevice);
            ParseCompressedStripeData(strm_desc_dev, num_streams, ps.compressionBlockSize, decompressor.GetLog2MaxCompressionRatio());
            switch (ps.compression)
            {
            case orc::ZLIB:
                gpuinflate(inflate_in, inflate_out, total_compressed_blocks, 0);
                break;
            case orc::SNAPPY:
                gpu_unsnap(inflate_in, inflate_out, total_compressed_blocks);
                break;
            default:
                printf("Unsupported GPU compression\n");
                goto error_exit;
            }
            PostDecompressionReassemble(strm_desc_dev, num_streams);
            // Update pointers to uncompressed data
            // TBD: We could update the value from the information we already have in stream_info[], but using the gpu results also updates max_uncompressed_size
            // to the actual uncompressed size, or zero if decompression failed.
            hipMemcpyAsync(strm_desc, strm_desc_dev, num_streams * sizeof(orc::gpu::CompressedStreamInfo), hipMemcpyDeviceToHost);
            hipStreamSynchronize(0);
            for (int i = 0; i < (int)ff.stripes.size(); i++)
            {
                for (int j = 0; j < num_columns; j++)
                {
                    orc::gpu::ColumnDesc *ck = &chunks[i * num_columns + j];
                    for (uint32_t k = 0; k < orc::gpu::CI_NUM_STREAMS; k++)
                    {
                        uint32_t len = ck->strm_len[k];
                        uint32_t strm_id = ck->strm_id[k];
                        if (len > 0 && strm_id < (uint32_t)num_streams)
                        {
                            ck->streams[k] = strm_desc[strm_id].uncompressed_data;
                            ck->strm_len[k] = (uint32_t)strm_desc[strm_id].max_uncompressed_size;
                        }
                    }
                }
            }
            printf("[GPU] Decompressed %zd bytes in %.1fms (%.2fMB/s)\n", total_uncompressed_size, decompression_time * 1000.0, 1.e-6 * total_uncompressed_size / decompression_time);
            RMM_FREE(inflate_in, 0);
        }
        // Free compressed slice data after decompression (not needed any further)
        for (size_t i = 0; i < compressed_stripe_data.size(); i++)
        {
            RMM_FREE(compressed_stripe_data[i], 0);
            compressed_stripe_data[i] = nullptr;
        }
        hipHostFree(strm_desc);
        RMM_FREE(strm_desc_dev, 0);
        strm_desc = strm_desc_dev = nullptr;
    }
    else
    {
        for (size_t i = 0; i < compressed_stripe_data.size(); i++)
        {
            uncompressed_stripe_data.push_back(compressed_stripe_data[i]);
            compressed_stripe_data[i] = nullptr;
        }
    }

    // Allocate column data
    for (auto &column : columns) {
      CUDF_EXPECTS(column.allocate() == GDF_SUCCESS, "Cannot allocate columns");
    }

    // Finalize column chunk initialization
    for (int i = 0; i < (int)ff.stripes.size(); i++)
    {
        for (int j = 0; j < num_columns; j++)
        {
            orc::gpu::ColumnDesc *ck = &chunks[i * num_columns + j];
            ck->valid_map_base = reinterpret_cast<uint32_t *>(columns[j]->valid);
            ck->column_data_base = columns[j]->data;
            ck->dtype_len = (uint8_t)GetGDFTypeLength(columns[j]->dtype);
        }
    }

    // Copy column chunk data to device
    hipMemcpyAsync(chunks_dev, chunks, num_columns * ff.stripes.size() * sizeof(orc::gpu::ColumnDesc), hipMemcpyHostToDevice);
    DecodeNullsAndStringDictionaries(chunks_dev, global_dictionary, num_columns, (uint32_t)ff.stripes.size(), num_rows, first_row);
    DecodeOrcColumnData(chunks_dev, global_dictionary, num_columns, (uint32_t)ff.stripes.size(), num_rows, first_row);
    hipMemcpyAsync(chunks, chunks_dev, num_columns * ff.stripes.size() * sizeof(orc::gpu::ColumnDesc), hipMemcpyDeviceToHost);
    hipStreamSynchronize(0);
    printf("[GPU] Decoded bytes\n");
#if 0
    if (num_dictionary_entries > 0)
    {
        orc::gpu::DictionaryEntry *host_dictionary = nullptr;
        RMM_ALLOC_HOST((void **)&host_dictionary, num_dictionary_entries * sizeof(orc::gpu::DictionaryEntry));
        if (!host_dictionary)
            goto error_exit;
        hipMemcpyAsync(host_dictionary, global_dictionary, num_dictionary_entries * sizeof(orc::gpu::DictionaryEntry), hipMemcpyDeviceToHost);
        hipStreamSynchronize(0);
        printf("global dictionary:\n");
        for (uint32_t i = 0; i < 1000; i++)
        {
            if (i < num_dictionary_entries /*&& (i < 50 || !host_dictionary[i].len)*/)
            {
                printf("[%d] %d bytes @ %d\n", i, host_dictionary[i].len, host_dictionary[i].pos);
            }
        }
        RMM_FREE_HOST(host_dictionary);
    }
#endif
    for (int i = 0; i < num_columns; i++)
    {
        gdf_size_type null_count = 0;
        for (int j = 0; j < (int)ff.stripes.size(); j++)
        {
            null_count += (gdf_size_type)chunks[j * num_columns + i].null_count;
        }
        columns[i]->null_count = null_count;
        printf("columns[%d].null_count = %d/%d (start_row=%d, nrows=%d, strm_len=%d)\n", i, null_count, columns[i]->size, chunks[i].start_row, chunks[i].num_rows, chunks[i].strm_len[orc::gpu::CI_PRESENT]);
    }
  } else {
    // Allocate column data
    for (auto &column : columns) {
      CUDF_EXPECTS(column.allocate() == GDF_SUCCESS, "Cannot allocate columns");
    }
  }

  // Transfer ownership to raw pointer output arguments
  args->data = (gdf_column **)malloc(sizeof(gdf_column *) * num_columns);
  for (int i = 0; i < num_columns; ++i) {
    args->data[i] = columns[i].release();

    // For string dtype, allocate and return an NvStrings container instance,
    // deallocating the original string list memory in the process.
    // This container takes a list of string pointers and lengths, and copies
    // into its own memory so the source memory must not be released yet.
    if (args->data[i]->dtype == GDF_STRING) {
      using str_pair = std::pair<const char *, size_t>;

      auto str_list = static_cast<str_pair *>(args->data[i]->data);
      auto str_data = NVStrings::create_from_index(str_list, num_rows);
      RMM_FREE(std::exchange(args->data[i]->data, str_data), 0);
    }
  }
  args->num_cols_out = num_columns;
  args->num_rows_out = num_rows;
  if (index_col != -1) {
    args->index_col = (int *)malloc(sizeof(int));
    *args->index_col = index_col;
  } else {
    args->index_col = nullptr;
  }

error_exit:
  for (size_t i = 0; i < compressed_stripe_data.size(); i++)
  {
      RMM_FREE(compressed_stripe_data[i], 0);
      compressed_stripe_data[i] = nullptr;
  }
  for (size_t i = 0; i < uncompressed_stripe_data.size(); i++)
  {
      RMM_FREE(uncompressed_stripe_data[i], 0);
      uncompressed_stripe_data[i] = nullptr;
  }
  hipHostFree(chunks);
  hipHostFree(strm_desc);
  RMM_FREE(global_dictionary, 0);
  RMM_FREE(strm_desc_dev, 0);
  RMM_FREE(chunks_dev, 0);

  return GDF_SUCCESS;
}

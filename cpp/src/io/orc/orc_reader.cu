#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "orc.h"
#include "orc_gpu.h"

#include "cudf.h"
#include "io/comp/gpuinflate.h"
#include "io/utilities/wrapper_utils.hpp"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.hpp"

#include <hip/hip_runtime.h>
#include <nvstrings/NVStrings.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include <array>
#include <cstring>
#include <iostream>
#include <numeric>
#include <vector>

#include <fcntl.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#if 1
#define LOG_PRINTF(...) std::printf(__VA_ARGS__)
#else
#define LOG_PRINTF(...) (void)0
#endif

/**
 * @brief Helper class for memory mapping a file source
 **/
class DataSource {
 public:
  explicit DataSource(const char *filepath) {
    fd = open(filepath, O_RDONLY);
    CUDF_EXPECTS(fd > 0, "Cannot open file");

    struct stat st {};
    CUDF_EXPECTS(fstat(fd, &st) == 0, "Cannot query file size");

    mapped_size = st.st_size;
    CUDF_EXPECTS(mapped_size > 0, "Unexpected zero-sized file");

    mapped_data = mmap(NULL, mapped_size, PROT_READ, MAP_PRIVATE, fd, 0);
    CUDF_EXPECTS(mapped_data != MAP_FAILED, "Cannot memory-mapping file");
  }

  ~DataSource() {
    if (mapped_data) {
      munmap(mapped_data, mapped_size);
    }
    if (fd) {
      close(fd);
    }
  }

  const uint8_t *data() const { return static_cast<uint8_t *>(mapped_data); }
  size_t size() const { return mapped_size; }

 private:
  void *mapped_data = nullptr;
  size_t mapped_size = 0;
  int fd = 0;
};

/**
 * @brief Function that translates ORC datatype to GDF dtype
 **/
constexpr std::pair<gdf_dtype, gdf_dtype_extra_info> to_dtype(
    const orc::SchemaType &schema) {
  switch (schema.kind) {
    case orc::BOOLEAN:
    case orc::BYTE:
      return std::make_pair(GDF_INT8, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::SHORT:
      return std::make_pair(GDF_INT16, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::INT:
      return std::make_pair(GDF_INT32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::LONG:
      return std::make_pair(GDF_INT64, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::FLOAT:
      return std::make_pair(GDF_FLOAT32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::DOUBLE:
      return std::make_pair(GDF_FLOAT64, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::STRING:
    case orc::BINARY:
    case orc::VARCHAR:
    case orc::CHAR:
      // Variable-length types can all be mapped to GDF_STRING
      return std::make_pair(GDF_STRING, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::TIMESTAMP:
      return std::make_pair(GDF_TIMESTAMP, gdf_dtype_extra_info{TIME_UNIT_ns});
    case orc::DATE:
      return std::make_pair(GDF_DATE32, gdf_dtype_extra_info{TIME_UNIT_NONE});
    case orc::DECIMAL:
      // Currently unhandled as there isn't an explicit mapping
    default:
      break;
  }

  return std::make_pair(GDF_invalid, gdf_dtype_extra_info{TIME_UNIT_NONE});
}

/**
 * @brief A helper class for ORC file metadata. Provides some additional
 * convenience methods for initializing and accessing metadata.
 **/
class OrcMetadata {
 public:
  explicit OrcMetadata(const uint8_t *data_, size_t len_)
      : data(data_), len(len_) {
    const auto ps_length = data[len - 1];
    const auto ps_data = &data[len - ps_length - 1];

    // Read uncompressed postscript section
    orc::ProtobufReader pb;
    pb.init(ps_data, ps_length);
    CUDF_EXPECTS(pb.read(&ps, ps_length), "Cannot read postscript");
    CUDF_EXPECTS(ps.footerLength + ps_length < len, "Invalid footer length");

    LOG_PRINTF("\n[+] PostScript:\n");
    LOG_PRINTF(" postscriptLength = %d\n", ps_length);
    LOG_PRINTF(" footerLength = %zd\n", (size_t)ps.footerLength);
    LOG_PRINTF(" compression = %d\n", ps.compression);
    LOG_PRINTF(" compressionBlockSize = %d\n", ps.compressionBlockSize);
    LOG_PRINTF(" version(%zd) = {%d,%d}\n", ps.version.size(), (ps.version.size() > 0) ? (int32_t)ps.version[0] : -1, (ps.version.size() > 1) ? (int32_t)ps.version[1] : -1);
    LOG_PRINTF(" metadataLength = %zd\n", (size_t)ps.metadataLength);
    LOG_PRINTF(" magic = \"%s\"\n", ps.magic.c_str());

    // If compression is used, all the rest of the metadata is compressed
    // If no compressed is used, the decompressor is simply a pass-through
    decompressor = std::make_unique<orc::OrcDecompressor>(
        ps.compression, ps.compressionBlockSize);

    // Read compressed filefooter section
    size_t ff_length = 0;
    auto ff_data = decompressor->Decompress(ps_data - ps.footerLength,
                                            ps.footerLength, &ff_length);
    pb.init(ff_data, ff_length);
    CUDF_EXPECTS(pb.read(&ff, ff_length), "Cannot read filefooter");

    LOG_PRINTF("\n[+] FileFooter:\n");
    LOG_PRINTF(" headerLength = %zd\n", (size_t)ff.headerLength);
    LOG_PRINTF(" contentLength = %zd\n", (size_t)ff.contentLength);
    for (size_t i = 0; i < ff.stripes.size(); i++) {
      LOG_PRINTF(
          " stripe #%zd @ %zd: %d rows, index+data+footer: %zd+%zd+%d bytes\n",
          i, (size_t)ff.stripes[i].offset, ff.stripes[i].numberOfRows,
          (size_t)ff.stripes[i].indexLength, (size_t)ff.stripes[i].dataLength,
          ff.stripes[i].footerLength);
    }
    for (size_t i = 0; i < ff.types.size(); i++) {
      LOG_PRINTF(" column %zd: kind=%d, parent=%d\n", i, ff.types[i].kind,
                 ff.types[i].parent_idx);
      if (ff.types[i].subtypes.size() > 0) {
        LOG_PRINTF("   subtypes = ");
        for (int j = 0; j < (int)ff.types[i].subtypes.size(); j++) {
          LOG_PRINTF("%c%d", (j) ? ',' : '{', ff.types[i].subtypes[j]);
        }
        printf("}\n");
      }
      if (ff.types[i].fieldNames.size() > 0) {
        printf("   fieldNames = ");
        for (int j = 0; j < (int)ff.types[i].fieldNames.size(); j++) {
          printf("%c\"%s\"", (j) ? ',' : '{',
                 ff.types[i].fieldNames[j].c_str());
        }
        printf("}\n");
      }
    }
    for (size_t i = 0; i < ff.metadata.size(); i++) {
      printf(" metadata: \"%s\" = \"%s\"\n", ff.metadata[i].name.c_str(),
             ff.metadata[i].value.c_str());
    }
    printf(" numberOfRows = %zd\n", (size_t)ff.numberOfRows);
    printf(" rowIndexStride = %d\n", ff.rowIndexStride);
  }

  void select_stripes(uint64_t min_row, uint64_t num_rows) {
    // Exclude non-needed stripes
    while (ff.stripes.size() > 0 && ff.stripes[0].numberOfRows <= min_row) {
      ff.numberOfRows -= ff.stripes[0].numberOfRows;
      min_row -= ff.stripes[0].numberOfRows;
      ff.stripes.erase(ff.stripes.begin());
    }
    num_rows = std::min(num_rows,
                        ff.numberOfRows - std::min(min_row, ff.numberOfRows));
    if (ff.numberOfRows > num_rows) {
      uint64_t row = 0;
      for (size_t i = 0; i < ff.stripes.size(); i++) {
        if (row >= num_rows) {
          ff.stripes.resize(i);
          ff.numberOfRows = row;
          break;
        }
        row += ff.stripes[i].numberOfRows;
      }
    }

    // Read stripefooter metadata
    sf.resize(ff.stripes.size());
    for (size_t i = 0; i < ff.stripes.size(); ++i)
    {
      const auto stripe = ff.stripes[i];
      const auto sf_comp_offset =
          stripe.offset + stripe.indexLength + stripe.dataLength;
      const auto sf_comp_length = stripe.footerLength;
      CUDF_EXPECTS(sf_comp_offset + sf_comp_length < len,
                   "Invalid stripe information");

      size_t sf_length = 0;
      auto sf_data = decompressor->Decompress(data + sf_comp_offset,
                                              sf_comp_length, &sf_length);

      orc::ProtobufReader pb;
      pb.init(sf_data, sf_length);
      CUDF_EXPECTS(pb.read(&sf[i], sf_length), "Cannot read stripefooter");

#if VERBOSE_OUTPUT
        printf("StripeFooter(%d/%zd):\n", 1+i, ff.stripes.size());
        printf(" %d streams:\n", (int)sf.streams.size());
        for (int j = 0; j < (int)sf.streams.size(); j++)
        {
            printf(" [%d] column=%d, kind=%d, len=%zd\n", j, sf.streams[j].column, sf.streams[j].kind, (size_t)sf.streams[j].length);
        }
        printf(" %d columns:\n", (int)sf.columns.size());
        for (int j = 0; j < (int)sf.columns.size(); j++)
        {
            printf(" [%d] kind=%d, dictionarySize=%d\n", j, sf.columns[j].kind, sf.columns[j].dictionarySize);
        }
#endif
    }
  }

  inline int get_total_rows() const { return ff.numberOfRows; }
  inline int get_num_rowgroups() const { return ff.stripes.size(); }
  inline int get_num_columns() const { return ff.types.size(); }

public:
  orc::PostScript ps;
  orc::FileFooter ff;
  std::vector<orc::StripeFooter> sf;
  std::unique_ptr<orc::OrcDecompressor> decompressor;

 private:
  const uint8_t *const data;
  const size_t len;
};

/**
 * @brief Struct that maps ORC streams to columns
 **/
struct OrcStrmInfo
{
    uint64_t offset;        // offset in file
    size_t dst_pos;         // offset in memory relative to the beginning of the compressed stripe data
    uint32_t length;        // length in file
    uint32_t gdf_idx;       // gdf column index
    uint32_t stripe_idx;    // stripe index
};

/**
 * @brief Reads Apache ORC data and returns an array of gdf_columns.
 *
 * @param[in,out] args Structure containing input and output args
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 **/
gdf_error read_orc(orc_read_arg *args) {

  int num_columns = 0;
  int num_rows = 0;

  DataSource input(args->source);

  OrcMetadata md(input.data(), input.size());
  CUDF_EXPECTS(md.get_num_columns() > 0, "No columns found");

  static_assert(sizeof(orc::gpu::CompressedStreamInfo) <= 256 &&
                    !(sizeof(orc::gpu::CompressedStreamInfo) & 7),
                "Unexpected sizeof(CompressedStreamInfo)");
  static_assert(sizeof(orc::gpu::ColumnDesc) <= 256 &&
                    !(sizeof(orc::gpu::ColumnDesc) & 7),
                "Unexpected sizeof(ColumnDesc)");

  // Select only rowgroups required
  md.select_stripes(0, 0x7fffffff);

  // Select only columns required (if it exists), otherwise select all
  std::vector<int32_t> gdf2orc;                           // Map gdf columns to orc columns
  std::vector<int32_t> orc2gdf(md.get_num_columns(), -1); // Map orc columns to gdf columns
  if (args->use_cols) {
    std::vector<std::string> use_names(args->use_cols,
                                       args->use_cols + args->use_cols_len);
    int index = 0;
    for (const auto &use_name : use_names) {
      for (int i = 0; i < md.get_num_columns(); ++i, ++index) {
        if (index >= md.get_num_columns()) {
          index = 0;
        }
        if (md.ff.GetColumnName(index) == use_name) {
          orc2gdf[index] = gdf2orc.size();
          gdf2orc.emplace_back(index);
          index++;
        }
      }
    }
  } else {
    // For now, only select all leaf nodes
    for (int i = 0; i < md.get_num_columns(); ++i) {
      if (md.ff.types[i].subtypes.size() == 0) {
        orc2gdf[i] = gdf2orc.size();
        gdf2orc.emplace_back(i);
      }
    }
  }

  // Initialize gdf_columns, but hold off on allocating storage space
  std::vector<gdf_column_wrapper> columns;
  LOG_PRINTF("[+] Selected columns: %d\n", num_columns);
  for (const auto &col : gdf2orc) {
    auto dtype_info = to_dtype(md.ff.types[col]);

    columns.emplace_back(static_cast<gdf_size_type>(md.ff.numberOfRows),
                         dtype_info.first, dtype_info.second,
                         md.ff.GetColumnName(col));

    LOG_PRINTF(" %2zd: name=%s size=%zd type=%d data=%lx valid=%lx\n",
               columns.size() - 1, columns.back()->col_name,
               (size_t)columns.back()->size, columns.back()->dtype,
               (uint64_t)columns.back()->data, (uint64_t)columns.back()->valid);
  }

  num_rows = md.get_total_rows();
  num_columns = (int)gdf2orc.size();

  // Select columns
  size_t total_compressed_size;
  int num_streams;
  std::vector<OrcStrmInfo> stream_info;
  std::vector<uint8_t *> compressed_stripe_data, uncompressed_stripe_data;
  orc::gpu::DictionaryEntry *global_dictionary = nullptr;
  size_t stripe_start_row;
  uint32_t num_dictionary_entries;

  if (num_rows > 0 && num_columns > 0) {
    // Allocate row index: essentially 2D array indexed by stripe id & gdf column index
    const auto num_column_chunks = md.ff.stripes.size() * num_columns;
    hostdevice_vector<orc::gpu::ColumnDesc> chunks(num_column_chunks);

    // Read stripe footers
    total_compressed_size = 0;
    stripe_start_row = 0;
    num_dictionary_entries = 0;
    for (int i = 0; i < (int)md.sf.size(); i++)
    {
        size_t strm_count;
        uint64_t src_offset, dst_offset, index_length;
        uint8_t *data_dev = nullptr;

        // Read stream data
        src_offset = 0;
        dst_offset = 0;
        index_length = md.ff.stripes[i].indexLength;
        strm_count = stream_info.size();
        for (int j = 0; j < (int)md.sf[i].streams.size(); j++)
        {
            uint32_t strm_length = (uint32_t)md.sf[i].streams[j].length;
            uint32_t column_id = md.sf[i].streams[j].column;
            int32_t gdf_idx = -1;
            if (column_id < orc2gdf.size())
            {
                gdf_idx = orc2gdf[column_id];
                if (gdf_idx < 0 && md.ff.types[column_id].subtypes.size() != 0)
                {
                    // This column may be a parent column, in which case the PRESENT stream may be needed
                    bool needed = (md.ff.types[column_id].kind == orc::STRUCT && md.sf[i].streams[j].kind == orc::PRESENT);
                    if (needed)
                    {
                        for (int k = 0; k < (int)md.ff.types[column_id].subtypes.size(); k++)
                        {
                            uint32_t idx = md.ff.types[column_id].subtypes[k];
                            int32_t child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
                            if (child_idx >= 0)
                            {
                                gdf_idx = child_idx;
                                chunks[i * num_columns + gdf_idx].strm_id[orc::gpu::CI_PRESENT] = (uint32_t)stream_info.size();
                                chunks[i * num_columns + gdf_idx].strm_len[orc::gpu::CI_PRESENT] = strm_length;
                            }
                        }
                    }
                }
            }
            if (src_offset >= index_length && gdf_idx >= 0)
            {
                int ci_kind = orc::gpu::CI_NUM_STREAMS;
                switch (md.sf[i].streams[j].kind)
                {
                case orc::DATA:
                    ci_kind = orc::gpu::CI_DATA;
                    break;
                case orc::LENGTH:
                case orc::SECONDARY:
                    ci_kind = orc::gpu::CI_DATA2;
                    break;
                case orc::DICTIONARY_DATA:
                    ci_kind = orc::gpu::CI_DICTIONARY;
                    chunks[i * num_columns + gdf_idx].dictionary_start = num_dictionary_entries;
                    chunks[i * num_columns + gdf_idx].dict_len = md.sf[i].columns[column_id].dictionarySize;
                    num_dictionary_entries += md.sf[i].columns[column_id].dictionarySize;
                    break;
                case orc::PRESENT:
                    ci_kind = orc::gpu::CI_PRESENT;
                    break;
                default:
                    // TBD: Could skip loading this stream
                    break;
                }
                if (ci_kind < orc::gpu::CI_NUM_STREAMS)
                {
                    chunks[i * num_columns + gdf_idx].strm_id[ci_kind] = (uint32_t)stream_info.size();
                    chunks[i * num_columns + gdf_idx].strm_len[ci_kind] = strm_length;
                }
            }
            if (gdf_idx >= 0)
            {
                OrcStrmInfo info;
                info.offset = md.ff.stripes[i].offset + src_offset;
                info.length = strm_length;
                info.dst_pos = dst_offset;
                info.gdf_idx = gdf_idx;
                info.stripe_idx = i;
                stream_info.push_back(info); // FIXME: Use emplace_back
                dst_offset += strm_length;
            }
            src_offset += strm_length;
        }
        if (dst_offset > 0)
        {
            RMM_ALLOC((void **)&data_dev, dst_offset, 0);
            if (!data_dev)
                goto error_exit;
            while (strm_count < stream_info.size())
            {
                // Coalesce consecutive streams into one read
                uint64_t len = stream_info[strm_count].length;
                uint64_t offset = stream_info[strm_count].offset;
                void *dst = data_dev + stream_info[strm_count].dst_pos;
                strm_count++;
                while (strm_count < stream_info.size() && stream_info[strm_count].offset == offset + len)
                {
                    len += stream_info[strm_count].length;
                    strm_count++;
                }
                hipMemcpyAsync(dst, input.data() + offset, len, hipMemcpyHostToDevice, 0); // TODO: datasource::gpuread
                total_compressed_size += len;
            }
            // Update stream pointers
            for (int j = 0; j < num_columns; j++)
            {
                for (int k = 0; k < orc::gpu::CI_NUM_STREAMS; k++)
                {
                    if (chunks[i * num_columns + j].strm_len[k] > 0)
                    {
                        uint32_t strm_id = chunks[i * num_columns + j].strm_id[k];
                        chunks[i * num_columns + j].streams[k] = data_dev + stream_info[strm_id].dst_pos;
                    }
                }
                chunks[i * num_columns + j].start_row = (uint32_t)stripe_start_row;
                chunks[i * num_columns + j].num_rows = md.ff.stripes[i].numberOfRows;
                chunks[i * num_columns + j].encoding_kind = md.sf[i].columns[gdf2orc[j]].kind;
                chunks[i * num_columns + j].type_kind = md.ff.types[gdf2orc[j]].kind;
            }
        }
        compressed_stripe_data.push_back(data_dev);
        stripe_start_row += md.ff.stripes[i].numberOfRows;
    }

    printf("[CPU] Read %zd bytes\n", total_compressed_size);
    // Allocate global dictionary
    if (num_dictionary_entries > 0)
    {
        RMM_ALLOC((void **)&global_dictionary, num_dictionary_entries * sizeof(orc::gpu::DictionaryEntry), 0);
    }

    // Setup decompression
    num_streams = (int)stream_info.size();
    printf(" %d data streams, %d dictionary entries\n", num_streams, num_dictionary_entries);
    if (md.ps.compression != orc::NONE)
    {
        uint32_t total_compressed_blocks;
        size_t total_uncompressed_size;
        double decompression_time = 0;

        hostdevice_vector<orc::gpu::CompressedStreamInfo> streams(0, num_streams);
        for (int i = 0; i < num_streams; i++) {
          streams.insert(orc::gpu::CompressedStreamInfo(
              compressed_stripe_data[stream_info[i].stripe_idx] +
                  stream_info[i].dst_pos,
              stream_info[i].length));
        }
        CUDA_TRY(hipMemcpyAsync(streams.device_ptr(), streams.host_ptr(),
                                 streams.memory_size(),
                                 hipMemcpyHostToDevice));
        CUDA_TRY(ParseCompressedStripeData(streams.device_ptr(), streams.size(),
                                           md.ps.compressionBlockSize,
                                           md.decompressor->GetLog2MaxCompressionRatio()));
        CUDA_TRY(hipMemcpyAsync(streams.host_ptr(), streams.device_ptr(),
                                 streams.memory_size(),
                                 hipMemcpyDeviceToHost));
        CUDA_TRY(hipStreamSynchronize(0));

        total_compressed_blocks = 0;
        total_uncompressed_size = 0;
        for (int i = 0; i < num_streams; i++)
        {
            total_compressed_blocks += streams[i].num_compressed_blocks;
            total_uncompressed_size += streams[i].max_uncompressed_size;
        }
        if (total_uncompressed_size > 0)
        {
            uint8_t *uncompressed_data = nullptr;

            size_t uncomp_ofs;
            printf("%d compressed blocks, max_uncompressed_size=%zd\n", total_compressed_blocks, total_uncompressed_size);
            RMM_ALLOC((void **)&uncompressed_data, total_uncompressed_size, 0);
            if (!uncompressed_data)
                goto error_exit;
            uncompressed_stripe_data.push_back(uncompressed_data);

            rmm::device_vector<gpu_inflate_input_s> inflate_in(total_compressed_blocks);
            rmm::device_vector<gpu_inflate_status_s> inflate_out(total_compressed_blocks);

            uncomp_ofs = 0;
            for (int i = 0, pos = 0; i < num_streams; i++)
            {
                streams[i].uncompressed_data = uncompressed_data + uncomp_ofs;
                streams[i].decctl = inflate_in.data().get() + pos;
                streams[i].decstatus = inflate_out.data().get() + pos;
                streams[i].max_compressed_blocks = streams[i].num_compressed_blocks;
                stream_info[i].dst_pos = uncomp_ofs; // Now indicates the offset relative to base uncompressed data
                uncomp_ofs += streams[i].max_uncompressed_size;
                pos += streams[i].num_compressed_blocks;
            }

            // Parse again, this time populating the decompression input/output buffers
            CUDA_TRY(hipMemcpyAsync(streams.device_ptr(), streams.host_ptr(),
                                     streams.memory_size(),
                                     hipMemcpyHostToDevice));
            CUDA_TRY(ParseCompressedStripeData(
                streams.device_ptr(), streams.size(),
                md.ps.compressionBlockSize,
                md.decompressor->GetLog2MaxCompressionRatio()));
            switch (md.ps.compression) {
              case orc::ZLIB:
                CUDA_TRY(gpuinflate(inflate_in.data().get(),
                                    inflate_out.data().get(),
                                    total_compressed_blocks, 0));
                break;
              case orc::SNAPPY:
                CUDA_TRY(gpu_unsnap(inflate_in.data().get(),
                                    inflate_out.data().get(),
                                    total_compressed_blocks));
                break;
              default:
                CUDF_EXPECTS(false, "Unexpected decompression dispatch");
                break;
            }
            CUDA_TRY(PostDecompressionReassemble(streams.device_ptr(),
                                                 streams.size()));

            // Update the stream information in device memory with the updated
            // pointers to the uncompressed data buffer
            // TBD: We could update the value from the information we already
            // have in stream_info[], but using the gpu results also updates
            // max_uncompressed_size to the actual uncompressed size, or zero if
            // decompression failed.
            CUDA_TRY(hipMemcpyAsync(streams.host_ptr(), streams.device_ptr(),
                                     streams.memory_size(),
                                     hipMemcpyDeviceToHost));
            CUDA_TRY(hipStreamSynchronize(0));
            for (int i = 0; i < (int)md.ff.stripes.size(); i++)
            {
                for (int j = 0; j < num_columns; j++)
                {
                    orc::gpu::ColumnDesc *ck = &chunks[i * num_columns + j];
                    for (uint32_t k = 0; k < orc::gpu::CI_NUM_STREAMS; k++)
                    {
                        uint32_t len = ck->strm_len[k];
                        uint32_t strm_id = ck->strm_id[k];
                        if (len > 0 && strm_id < (uint32_t)num_streams)
                        {
                            ck->streams[k] = streams[strm_id].uncompressed_data;
                            ck->strm_len[k] = (uint32_t)streams[strm_id].max_uncompressed_size;
                        }
                    }
                }
            }
            printf("[GPU] Decompressed %zd bytes in %.1fms (%.2fMB/s)\n", total_uncompressed_size, decompression_time * 1000.0, 1.e-6 * total_uncompressed_size / decompression_time);

        }
        // Free compressed slice data after decompression (not needed any further)
        for (size_t i = 0; i < compressed_stripe_data.size(); i++)
        {
            RMM_FREE(compressed_stripe_data[i], 0);
            compressed_stripe_data[i] = nullptr;
        }
    }
    else
    {
        for (size_t i = 0; i < compressed_stripe_data.size(); i++)
        {
            uncompressed_stripe_data.push_back(compressed_stripe_data[i]);
            compressed_stripe_data[i] = nullptr;
        }
    }

    // Allocate column data
    for (auto &column : columns) {
      CUDF_EXPECTS(column.allocate() == GDF_SUCCESS, "Cannot allocate columns");
    }

    // Finalize column chunk initialization
    for (int i = 0; i < (int)md.ff.stripes.size(); i++) {
      for (int j = 0; j < num_columns; j++) {
        auto &chunk = chunks[i * num_columns + j];
        chunk.valid_map_base = reinterpret_cast<uint32_t *>(columns[j]->valid);
        chunk.column_data_base = columns[j]->data;
        chunk.dtype_len = (columns[j]->dtype == GDF_STRING)
                              ? sizeof(std::pair<const char *, size_t>)
                              : gdf_dtype_size(columns[j]->dtype);
      }
    }

    CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                             chunks.memory_size(), hipMemcpyHostToDevice));
    CUDA_TRY(DecodeNullsAndStringDictionaries(chunks.device_ptr(),
                                              global_dictionary, num_columns,
                                              md.get_num_rowgroups(), num_rows,
                                              0));
    CUDA_TRY(DecodeOrcColumnData(chunks.device_ptr(), global_dictionary,
                                 num_columns, md.get_num_rowgroups(), num_rows,
                                 0));
    CUDA_TRY(hipMemcpyAsync(chunks.host_ptr(), chunks.device_ptr(),
                             chunks.memory_size(), hipMemcpyDeviceToHost));
    CUDA_TRY(hipStreamSynchronize(0));

    printf("[GPU] Decoded bytes\n");
    for (int i = 0; i < num_columns; i++) {
      for (int j = 0; j < (int)md.ff.stripes.size(); j++) {
        columns[i]->null_count += chunks[j * num_columns + i].null_count;
      }
      LOG_PRINTF(
          "columns[%d].null_count = %d/%d (start_row=%d, nrows=%d, "
          "strm_len=%d)\n",
          i, columns[i]->null_count, columns[i]->size, chunks[i].start_row,
          chunks[i].num_rows, chunks[i].strm_len[orc::gpu::CI_PRESENT]);
    }
  } else {
    // Columns' data's memory is still expected for an empty dataframe
    for (auto &column : columns) {
      CUDF_EXPECTS(column.allocate() == GDF_SUCCESS, "Cannot allocate columns");
    }
  }

  // Transfer ownership to raw pointer output arguments
  args->data = (gdf_column **)malloc(sizeof(gdf_column *) * num_columns);
  for (int i = 0; i < num_columns; ++i) {
    args->data[i] = columns[i].release();

    // For string dtype, allocate and return an NvStrings container instance,
    // deallocating the original string list memory in the process.
    // This container takes a list of string pointers and lengths, and copies
    // into its own memory so the source memory must not be released yet.
    if (args->data[i]->dtype == GDF_STRING) {
      using str_pair = std::pair<const char *, size_t>;

      auto str_list = static_cast<str_pair *>(args->data[i]->data);
      auto str_data = NVStrings::create_from_index(str_list, num_rows);
      RMM_FREE(std::exchange(args->data[i]->data, str_data), 0);
    }
  }
  args->num_cols_out = num_columns;
  args->num_rows_out = num_rows;
  args->index_col = nullptr;

error_exit:
  for (size_t i = 0; i < compressed_stripe_data.size(); i++)
  {
      RMM_FREE(compressed_stripe_data[i], 0);
      compressed_stripe_data[i] = nullptr;
  }
  for (size_t i = 0; i < uncompressed_stripe_data.size(); i++)
  {
      RMM_FREE(uncompressed_stripe_data[i], 0);
      uncompressed_stripe_data[i] = nullptr;
  }
  RMM_FREE(global_dictionary, 0);

  return GDF_SUCCESS;
}

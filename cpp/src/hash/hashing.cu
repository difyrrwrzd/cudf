#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/detail/hashing.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/detail/utilities/cuda.cuh>

#include <hash/hash_functions.cuh>

#include <thrust/tabulate.h>

namespace cudf {

namespace {

constexpr int BLOCK_SIZE = 256;
constexpr int ROWS_PER_THREAD = 1;

template <template <typename> class hash_function>
struct row_hasher_initial_values {
  row_hasher_initial_values(table_device_view const& table_to_hash,
                            hash_value_type *initial_hash)
      : _table{table_to_hash}, _initial_hash(initial_hash) {}

  __device__ hash_value_type operator()(cudf::size_type row_index) const {
    return 0; // TODO
    //return hash_row<true, hash_function>(_table, row_index, _initial_hash);
  }

  table_device_view const& _table;
  hash_value_type *_initial_hash{nullptr};
};

template <template <typename> class hash_function>
struct row_hasher {
  row_hasher(table_device_view const& table_to_hash) : _table{table_to_hash} {}

  __device__ hash_value_type operator()(cudf::size_type row_index) const {
    return 0; // TODO
    //return hash_row<true, hash_function>(_table, row_index);
  }

  table_device_view const& _table;
};

/** 
 * @brief  Functor to map a hash value to a particular 'bin' or partition number
 * that uses the modulo operation.
 */
template <typename hash_value_t>
struct modulo_partitioner
{
  modulo_partitioner(size_type num_partitions) : divisor{num_partitions}{}

  __host__ __device__
  size_type operator()(hash_value_t hash_value) const 
  {
    return hash_value % divisor;
  }

  const size_type divisor;
};

/** 
 * @brief  Functor to map a hash value to a particular 'bin' or partition number
 * that uses bitshifts. Only works when num_partitions is a power of 2.
 *
 * For n % d, if d is a power of two, then it can be computed more efficiently via 
 * a single bitwise AND as:
 * n & (d - 1)
 */
template <typename hash_value_t>
struct bitwise_partitioner
{
  bitwise_partitioner(size_type num_partitions) : divisor{(num_partitions - 1)}
  {
    assert( is_power_two(num_partitions) );
  }

  __host__ __device__
  size_type operator()(hash_value_t hash_value) const 
  {
    return hash_value & (divisor);
  }

  const size_type divisor;
};

/** 
 * @brief Computes which partition each row of a device_table will belong to based
   on hashing each row, and applying a partition function to the hash value. 
   Records the size of each partition for each thread block as well as the global
   size of each partition across all thread blocks.
 */
template <template <typename> class hash_function,
          typename partitioner_type>
__global__ 
void compute_row_partition_numbers(table_device_view the_table, 
                                   const size_type num_rows,
                                   const size_type num_partitions,
                                   const partitioner_type the_partitioner,
                                   size_type *row_partition_numbers,
                                   size_type *block_partition_sizes,
                                   size_type *global_partition_sizes)
{
  // Accumulate histogram of the size of each partition in shared memory
  extern __shared__ size_type shared_partition_sizes[];

  size_type row_number = threadIdx.x + blockIdx.x * blockDim.x;

  // Initialize local histogram
  size_type partition_number = threadIdx.x;
  while (partition_number < num_partitions) {
    shared_partition_sizes[partition_number] = 0;
    partition_number += blockDim.x;
  }

  __syncthreads();

  // Compute the hash value for each row, store it to the array of hash values
  // and compute the partition to which the hash value belongs and increment
  // the shared memory counter for that partition
  while (row_number < num_rows) {
    const hash_value_type row_hash_value =
        0; // TODO
        //hash_row<true, hash_function>(the_table, row_number);

    const size_type partition_number = the_partitioner(row_hash_value);

    row_partition_numbers[row_number] = partition_number;

    atomicAdd(&(shared_partition_sizes[partition_number]), size_type(1));

    row_number += blockDim.x * gridDim.x;
  }

  __syncthreads();

  // Flush shared memory histogram to global memory
  partition_number = threadIdx.x;
  while (partition_number < num_partitions) {
    const size_type block_partition_size = shared_partition_sizes[partition_number];

    // Update global size of each partition
    atomicAdd(&global_partition_sizes[partition_number], block_partition_size);

    // Record the size of this partition in this block
    const size_type write_location = partition_number * gridDim.x + blockIdx.x;
    block_partition_sizes[write_location] = block_partition_size;
    partition_number += blockDim.x;
  }
}

template <template <typename> class hash_function>
std::vector<std::unique_ptr<experimental::table>>
hash_partition_table(table_view const& input,
                     table_view const &table_to_hash,
                     const cudf::size_type num_partitions,
                     hipStream_t stream)
{
  const cudf::size_type num_rows = table_to_hash.num_rows();
  constexpr cudf::size_type rows_per_block = BLOCK_SIZE * ROWS_PER_THREAD;
  const cudf::size_type grid_size = util::div_rounding_up_safe(num_rows, rows_per_block);

  auto device_input = table_device_view::create(input, stream);
  auto row_partition_numbers = rmm::device_vector<size_type>(num_rows);
  auto block_partition_sizes = rmm::device_vector<size_type>(grid_size * num_partitions);
  auto global_partition_sizes = rmm::device_vector<size_type>(num_partitions);
  CUDA_TRY(hipMemsetAsync(global_partition_sizes.data().get(), 0, num_partitions * sizeof(size_type), stream));

  // If the number of partitions is a power of two, we can compute the partition 
  // number of each row more efficiently with bitwise operations
  if (is_power_two(num_partitions)) {
    // Determines how the mapping between hash value and partition number is computed
    using partitioner_type = bitwise_partitioner<hash_value_type>;

    // Computes which partition each row belongs to by hashing the row and performing
    // a partitioning operator on the hash value. Also computes the number of
    // rows in each partition both for each thread block as well as across all blocks
    compute_row_partition_numbers<hash_function>
        <<<grid_size, BLOCK_SIZE, num_partitions * sizeof(cudf::size_type), stream>>>(
            *device_input, num_rows, num_partitions,
            partitioner_type(num_partitions), row_partition_numbers.data().get(),
            block_partition_sizes.data().get(), global_partition_sizes.data().get());

  } else {
    // Determines how the mapping between hash value and partition number is computed
    using partitioner_type = modulo_partitioner<hash_value_type>;

    // Computes which partition each row belongs to by hashing the row and performing
    // a partitioning operator on the hash value. Also computes the number of
    // rows in each partition both for each thread block as well as across all blocks
    compute_row_partition_numbers<hash_function>
        <<<grid_size, BLOCK_SIZE, num_partitions * sizeof(cudf::size_type), stream>>>(
            *device_input, num_rows, num_partitions,
            partitioner_type(num_partitions), row_partition_numbers.data().get(),
            block_partition_sizes.data().get(), global_partition_sizes.data().get());
  }

  // TODO

  // build output tables from partitioned row indices
  std::vector<std::unique_ptr<experimental::table>> output(num_partitions);

  return output;
}

}  // namespace

namespace detail {

std::vector<std::unique_ptr<experimental::table>>
hash_partition(table_view const& input,
               std::vector<size_type> const& columns_to_hash,
               int num_partitions,
               hash_func::Type hash,
               rmm::mr::device_memory_resource* mr,
               hipStream_t stream)
{
  CUDF_EXPECTS(columns_to_hash.size() > 0, "Need at least one column to hash");
  CUDF_EXPECTS(num_partitions > 0, "Need at least one partition");

  auto table_to_hash = input.select(columns_to_hash);
  std::vector<std::unique_ptr<experimental::table>> output;

  cudf::nvtx::range_push("CUDF_HASH_PARTITION", cudf::nvtx::PARTITION_COLOR);

  switch (hash) {
    case hash_func::MURMUR3:
      output = hash_partition_table<MurmurHash3_32>(
          input, table_to_hash, num_partitions, stream);
      break;
    case hash_func::IDENTITY:
      output = hash_partition_table<IdentityHash>(
          input, table_to_hash, num_partitions, stream);
      break;
    default:
      CUDF_FAIL("Invalid hash function");
  }

  cudf::nvtx::range_pop();

  return output;
}

std::unique_ptr<column> hash(table_view const& input,
                             hash_func::Type hash,
                             std::vector<uint32_t> const& initial_hash,
                             rmm::mr::device_memory_resource* mr,
                             hipStream_t stream)
{
  // TODO this should be UINT32
  auto hash_column = make_numeric_column(data_type(INT32), input.num_rows());
  auto hash_view = hash_column->mutable_view();

  // Return early if there's nothing to hash
  if (input.num_columns() == 0 || input.num_rows() == 0) {
    return hash_column;
  }

  auto device_input = table_device_view::create(input, stream);

  // Compute the hash value for each row depending on the specified hash function
  if (!initial_hash.empty()) {
    CUDF_EXPECTS(initial_hash.size() == input.num_columns(),
      "Expected same size of initial hash values as number of columns");
    auto device_initial_hash = rmm::device_vector<uint32_t>(initial_hash);

    switch (hash) {
      case hash_func::MURMUR3:
        thrust::tabulate(rmm::exec_policy(stream)->on(stream),
                         hash_view.begin<int32_t>(), hash_view.end<int32_t>(),
                         row_hasher_initial_values<MurmurHash3_32>(
                             *device_input, device_initial_hash.data().get()));
        break;
      case hash_func::IDENTITY:
        thrust::tabulate(rmm::exec_policy(stream)->on(stream),
                         hash_view.begin<int32_t>(), hash_view.end<int32_t>(),
                         row_hasher_initial_values<IdentityHash>(
                             *device_input, device_initial_hash.data().get()));
        break;
      default:
        CUDF_FAIL("Invalid hash function");
    }
  } else {
    switch (hash) {
      case hash_func::MURMUR3:
        thrust::tabulate(rmm::exec_policy(stream)->on(stream),
                         hash_view.begin<int32_t>(), hash_view.end<int32_t>(),
                         row_hasher<MurmurHash3_32>(*device_input));
        break;
      case hash_func::IDENTITY:
        thrust::tabulate(rmm::exec_policy(stream)->on(stream),
                         hash_view.begin<int32_t>(), hash_view.end<int32_t>(),
                         row_hasher<IdentityHash>(*device_input));
        break;
      default:
        CUDF_FAIL("Invalid hash function");
    }
  }

  return hash_column;
}

}  // namespace detail

std::vector<std::unique_ptr<experimental::table>>
hash_partition(table_view const& input,
               std::vector<size_type> const& columns_to_hash,
               int num_partitions,
               hash_func::Type hash,
               rmm::mr::device_memory_resource* mr)
{
  return detail::hash_partition(input, columns_to_hash, num_partitions, hash, mr);
}

std::unique_ptr<column> hash(table_view const& input,
                             hash_func::Type hash,
                             std::vector<uint32_t> const& initial_hash,
                             rmm::mr::device_memory_resource* mr)
{
  return detail::hash(input, hash, initial_hash, mr);
}

}  // namespace cudf

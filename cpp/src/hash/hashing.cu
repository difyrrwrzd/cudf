#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/detail/hashing.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/table/row_operators.cuh>

#include <thrust/tabulate.h>

namespace cudf {

namespace {

/** 
 * @brief  Functor to map a hash value to a particular 'bin' or partition number
 * that uses the modulo operation.
 */
template <typename hash_value_t>
struct modulo_partitioner
{
  modulo_partitioner(size_type num_partitions) : divisor{num_partitions} {}

  __host__ __device__
  size_type operator()(hash_value_t hash_value) const {
    return hash_value % divisor;
  }

  const size_type divisor;
};

template <typename T>
bool is_power_two(T number) {
  return (0 == (number & (number - 1)));
}

/** 
 * @brief  Functor to map a hash value to a particular 'bin' or partition number
 * that uses bitshifts. Only works when num_partitions is a power of 2.
 *
 * For n % d, if d is a power of two, then it can be computed more efficiently via 
 * a single bitwise AND as:
 * n & (d - 1)
 */
template <typename hash_value_t>
struct bitwise_partitioner
{
  bitwise_partitioner(size_type num_partitions) : divisor{(num_partitions - 1)} {
    assert(is_power_two(num_partitions));
  }

  __host__ __device__
  size_type operator()(hash_value_t hash_value) const {
    return hash_value & (divisor);
  }

  const size_type divisor;
};

/** 
 * @brief Computes which partition each row of a device_table will belong to based
   on hashing each row, and applying a partition function to the hash value. 
   Records the size of each partition for each thread block as well as the global
   size of each partition across all thread blocks.
 */
template <bool has_nulls, typename partitioner_type>
__global__ 
void compute_row_partition_numbers(table_device_view the_table, 
                                   const size_type num_rows,
                                   const size_type num_partitions,
                                   const partitioner_type the_partitioner,
                                   size_type *row_partition_numbers,
                                   size_type *block_partition_sizes,
                                   size_type *global_partition_sizes)
{
  // Accumulate histogram of the size of each partition in shared memory
  extern __shared__ size_type shared_partition_sizes[];

  size_type row_number = threadIdx.x + blockIdx.x * blockDim.x;

  // Initialize local histogram
  size_type partition_number = threadIdx.x;
  while (partition_number < num_partitions) {
    shared_partition_sizes[partition_number] = 0;
    partition_number += blockDim.x;
  }

  __syncthreads();

  auto hasher = experimental::row_hasher<MurmurHash3_32, has_nulls>(the_table);

  // Compute the hash value for each row, store it to the array of hash values
  // and compute the partition to which the hash value belongs and increment
  // the shared memory counter for that partition
  while (row_number < num_rows) {
    const hash_value_type row_hash_value = hasher(row_number);

    const size_type partition_number = the_partitioner(row_hash_value);

    row_partition_numbers[row_number] = partition_number;

    atomicAdd(&(shared_partition_sizes[partition_number]), size_type(1));

    row_number += blockDim.x * gridDim.x;
  }

  __syncthreads();

  // Flush shared memory histogram to global memory
  partition_number = threadIdx.x;
  while (partition_number < num_partitions) {
    const size_type block_partition_size = shared_partition_sizes[partition_number];

    // Update global size of each partition
    atomicAdd(&global_partition_sizes[partition_number], block_partition_size);

    // Record the size of this partition in this block
    const size_type write_location = partition_number * gridDim.x + blockIdx.x;
    block_partition_sizes[write_location] = block_partition_size;
    partition_number += blockDim.x;
  }
}

/** 
 * @brief  Given an array of partition numbers, computes the final output location
   for each element in the output such that all rows with the same partition are 
   contiguous in memory.
 */
__global__ 
void compute_row_output_locations(size_type *row_partition_numbers, 
                                  const size_type num_rows,
                                  const size_type num_partitions,
                                  size_type *block_partition_offsets)
{
  // Shared array that holds the offset of this blocks partitions in 
  // global memory
  extern __shared__ size_type shared_partition_offsets[];

  // Initialize array of this blocks offsets from global array
  size_type partition_number= threadIdx.x;
  while (partition_number < num_partitions) {
    shared_partition_offsets[partition_number] = block_partition_offsets[partition_number * gridDim.x + blockIdx.x];
    partition_number += blockDim.x;
  }
  __syncthreads();

  size_type row_number = threadIdx.x + blockIdx.x * blockDim.x;

  // Get each row's partition number, and get it's output location by 
  // incrementing block's offset counter for that partition number
  // and store the row's output location in-place
  while (row_number < num_rows) {
    // Get partition number of this row
    const size_type partition_number = row_partition_numbers[row_number];

    // Get output location based on partition number by incrementing the corresponding
    // partition offset for this block
    const size_type row_output_location = atomicAdd(&(shared_partition_offsets[partition_number]), size_type(1));

    // Store the row's output location in-place
    row_partition_numbers[row_number] = row_output_location;

    row_number += blockDim.x * gridDim.x;
  }
}

template <bool has_nulls>
std::vector<std::unique_ptr<experimental::table>>
hash_partition_table(table_view const& input,
                     table_view const &table_to_hash,
                     const size_type num_partitions,
                     hipStream_t stream)
{
  constexpr size_type block_size = 256;
  constexpr size_type rows_per_thread = 1;
  constexpr size_type rows_per_block = block_size * rows_per_thread;

  const size_type num_rows = table_to_hash.num_rows();
  const size_type grid_size = util::div_rounding_up_safe(num_rows, rows_per_block);

  auto device_input = table_device_view::create(input, stream);
  auto row_partition_numbers = rmm::device_vector<size_type>(num_rows);
  auto block_partition_sizes = rmm::device_vector<size_type>(grid_size * num_partitions);
  auto global_partition_sizes = rmm::device_vector<size_type>(num_partitions);
  CUDA_TRY(hipMemsetAsync(global_partition_sizes.data().get(), 0, num_partitions * sizeof(size_type), stream));

  // If the number of partitions is a power of two, we can compute the partition 
  // number of each row more efficiently with bitwise operations
  if (is_power_two(num_partitions)) {
    // Determines how the mapping between hash value and partition number is computed
    using partitioner_type = bitwise_partitioner<hash_value_type>;

    // Computes which partition each row belongs to by hashing the row and performing
    // a partitioning operator on the hash value. Also computes the number of
    // rows in each partition both for each thread block as well as across all blocks
    compute_row_partition_numbers<has_nulls>
        <<<grid_size, block_size, num_partitions * sizeof(size_type), stream>>>(
            *device_input, num_rows, num_partitions,
            partitioner_type(num_partitions), row_partition_numbers.data().get(),
            block_partition_sizes.data().get(), global_partition_sizes.data().get());

  } else {
    // Determines how the mapping between hash value and partition number is computed
    using partitioner_type = modulo_partitioner<hash_value_type>;

    // Computes which partition each row belongs to by hashing the row and performing
    // a partitioning operator on the hash value. Also computes the number of
    // rows in each partition both for each thread block as well as across all blocks
    compute_row_partition_numbers<has_nulls>
        <<<grid_size, block_size, num_partitions * sizeof(size_type), stream>>>(
            *device_input, num_rows, num_partitions,
            partitioner_type(num_partitions), row_partition_numbers.data().get(),
            block_partition_sizes.data().get(), global_partition_sizes.data().get());
  }

  // Compute in-place exclusive scan of all blocks' partition sizes to determine 
  // the starting point for each blocks portion of each partition in the output
  thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream),
                         block_partition_sizes.begin(),
                         block_partition_sizes.end(),
                         block_partition_sizes.begin());

  // Compute in-place exclusive scan of size of each partition to determine
  // offset location of each partition in final output.
  thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream),
                         global_partition_sizes.begin(),
                         global_partition_sizes.end(),
                         global_partition_sizes.begin());

  // Copy the result of the exlusive scan to the output offsets array
  // to indicate the starting point for each partition in the output
  std::vector<size_type> partition_offsets(num_partitions);
  CUDA_TRY(hipMemcpyAsync(partition_offsets.data(),
                           global_partition_sizes.data().get(),
                           num_partitions * sizeof(size_type),
                           hipMemcpyDeviceToHost,
                           stream));

  // Compute in-place the output location for each row based on it's 
  // partition number such that each partition will be contiguous in memory
  compute_row_output_locations
    <<<grid_size, block_size, num_partitions * sizeof(size_type), stream>>>
    (row_partition_numbers.data().get(), num_rows, num_partitions, block_partition_sizes.data().get());


  // TODO build output tables from partitioned row indices
  std::vector<std::unique_ptr<experimental::table>> output(num_partitions);

  // Creates the partitioned output table by scattering the rows of
  // the input table to rows of the output table based on each rows
  // output location
  // TODO need scatter from PR 3296
  //cudf::detail::scatter(&input_table, row_partition_numbers.data().get(), &partitioned_output);

  return output;
}

}  // namespace

namespace detail {

std::vector<std::unique_ptr<experimental::table>>
hash_partition(table_view const& input,
               std::vector<size_type> const& columns_to_hash,
               int num_partitions,
               rmm::mr::device_memory_resource* mr,
               hipStream_t stream)
{
  CUDF_EXPECTS(columns_to_hash.size() > 0, "Need at least one column to hash");
  CUDF_EXPECTS(num_partitions > 0, "Need at least one partition");

  auto table_to_hash = input.select(columns_to_hash);
  bool const nullable = has_nulls(table_to_hash);

  cudf::nvtx::range_push("CUDF_HASH_PARTITION", cudf::nvtx::PARTITION_COLOR);

  std::vector<std::unique_ptr<experimental::table>> output;
  if (nullable) {
    output = hash_partition_table<true>(
        input, table_to_hash, num_partitions, stream);
  } else {
    output = hash_partition_table<false>(
        input, table_to_hash, num_partitions, stream);
  }

  cudf::nvtx::range_pop();

  return output;
}

std::unique_ptr<column> hash(table_view const& input,
                             std::vector<uint32_t> const& initial_hash,
                             rmm::mr::device_memory_resource* mr,
                             hipStream_t stream)
{
  // TODO this should be UINT32
  auto output = make_numeric_column(data_type(INT32), input.num_rows());

  // Return early if there's nothing to hash
  if (input.num_columns() == 0 || input.num_rows() == 0) {
    return output;
  }

  bool const nullable = has_nulls(input);
  auto const device_input = table_device_view::create(input, stream);
  auto output_view = output->mutable_view();

  // Compute the hash value for each row depending on the specified hash function
  if (!initial_hash.empty()) {
    CUDF_EXPECTS(initial_hash.size() == size_t(input.num_columns()),
      "Expected same size of initial hash values as number of columns");
    auto device_initial_hash = rmm::device_vector<uint32_t>(initial_hash);

    if (nullable) {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher_initial_values<MurmurHash3_32, true>(
              *device_input, device_initial_hash.data().get()));
    } else {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher_initial_values<MurmurHash3_32, false>(
              *device_input, device_initial_hash.data().get()));
    }
  } else {
    if (nullable) {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher<MurmurHash3_32, true>(*device_input));
    } else {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher<MurmurHash3_32, false>(*device_input));
    }
  }

  return output;
}

}  // namespace detail

std::vector<std::unique_ptr<experimental::table>>
hash_partition(table_view const& input,
               std::vector<size_type> const& columns_to_hash,
               int num_partitions,
               rmm::mr::device_memory_resource* mr)
{
  return detail::hash_partition(input, columns_to_hash, num_partitions, mr);
}

std::unique_ptr<column> hash(table_view const& input,
                             std::vector<uint32_t> const& initial_hash,
                             rmm::mr::device_memory_resource* mr)
{
  return detail::hash(input, initial_hash, mr);
}

}  // namespace cudf

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/cudf.h>
#include <cudf/types.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/cuda_utils.hpp>
#include <utilities/column_utils.hpp>
#include <bitmask/legacy/bit_mask.cuh>
#include <cudf/filling.hpp>

using bit_mask::bit_mask_t;
static constexpr int BLOCK_SIZE = 256;


namespace cudf {

namespace detail {

namespace unary {

__global__
void null_ops_kernel(bit_mask_t const* __restrict__ valid,
	    bool* output, bool nulls_are_false)
{
    gdf_size_type i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = nulls_are_false == bit_mask::is_valid(valid, i);
}
}// unary

gdf_column null_op(gdf_column const& input, bool nulls_are_false = true, hipStream_t stream = 0) {
    auto output = cudf::allocate_column(GDF_BOOL8, input.size, false, 
		  gdf_dtype_extra_info{}, stream);

    if (not cudf::is_nullable(input)) {
	gdf_scalar value {nulls_are_false, GDF_BOOL8, true}; 
	cudf::fill(&output, value, 0, output.size);
    }
    else {
        const bit_mask_t* __restrict__ typed_input_valid = reinterpret_cast<bit_mask_t*>(input.valid); 
	cudf::util::cuda::grid_config_1d grid{output.size, BLOCK_SIZE, 1};
	bool* out_data = static_cast<bool*>(output.data);

	unary::null_ops_kernel<<<grid.num_blocks, BLOCK_SIZE, 0, stream>>>(
			                     typed_input_valid,
					     out_data,
					     nulls_are_false);
    }

    return output;
}
}// detail

gdf_column is_null(gdf_column const& input) {

    return detail::null_op(input, false, 0);
}

gdf_column is_not_null(gdf_column const& input) {

    return detail::null_op(input, true, 0);
}

}// cudf

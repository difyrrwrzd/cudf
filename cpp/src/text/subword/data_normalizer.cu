#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <text/subword/detail/cp_data.h>
#include <cudf/utilities/error.hpp>
#include <text/subword/detail/codepoint_metadata.ah>
#include <text/subword/detail/data_normalizer.hpp>
#include <text/subword/detail/tokenizer_utils.cuh>

#include <>
#include <thrust/fill.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <cub/device/device_scan.cuh>
#include <cub/device/device_select.cuh>
#include <string>
#include <vector>

namespace nvtext {
namespace detail {
namespace {

#define SORT_BIT 22
#define THREADS_PER_BLOCK 64

__device__ __forceinline__ uint32_t get_first_cp(uint32_t metadata)
{
  return metadata & NEW_CP_MASK;
}

__device__ __forceinline__ uint32_t extract_token_cat(uint32_t metadata)
{
  return (metadata >> TOKEN_CAT_SHIFT) & TOKEN_CAT_MASK;
}

__device__ __forceinline__ bool should_remove_cp(uint32_t metadata, bool lower_case)
{
  auto const cat = extract_token_cat(metadata);
  return (cat == TOKEN_CAT_REMOVE_CHAR) || (lower_case && (cat == TOKEN_CAT_REMOVE_CHAR_IF_LOWER));
}

__device__ __forceinline__ bool should_add_spaces(uint32_t metadata, bool lower_case)
{
  auto const cat = extract_token_cat(metadata);
  return (cat == TOKEN_CAT_ADD_SPACE) || (lower_case && (cat == TOKEN_CAT_ADD_SPACE_IF_LOWER));
}

__device__ __forceinline__ bool always_replace(uint32_t metadata)
{
  return extract_token_cat(metadata) == TOKEN_CAT_ALWAYS_REPLACE;
}

__device__ __forceinline__ uint32_t is_multi_char_transform(uint32_t metadata)
{
  return (metadata >> MULTICHAR_SHIFT) & MULTICHAR_MASK;
}

/**
 * @brief Returns true if the byte passed in could be a valid head byte for
 * a utf8 character. That is, not binary `10xxxxxx`
 */
__device__ __forceinline__ bool is_head_byte(unsigned char utf8_byte)
{
  return (utf8_byte >> 6) != 2;
}

/**
 * @brief Converts a UTF-8 character into a unicode code point value.
 *
 * If the byte at start_byte_for_thread is the first byte of a UTF-8 character (head byte),
 * the UTF-8 character is converted to a unicode code point and returned.
 *
 * If the byte at start_byte_for_thread is not a head byte, 0 is returned.
 *
 * All threads start reading bytes from the pointer denoted by strings.
 *
 * @param strings A pointer to the start of the sequence of characters to be analyzed.
 * @param start_byte_for_thread Which byte to start analyzing
 * @return New code point value for this byte.
 */
__device__ __forceinline__ uint32_t
extract_code_points_from_utf8(const unsigned char* strings, const uint32_t start_byte_for_thread)
{
  constexpr uint8_t max_utf8_blocks_for_char = 4;
  uint8_t utf8_blocks[max_utf8_blocks_for_char];

#pragma unroll
  for (int i = 0; i < max_utf8_blocks_for_char; ++i) {
    utf8_blocks[i] = strings[start_byte_for_thread + i];
  }

  // We can have at most 5 bits encoding the length. We check those bits to infer the actual length
  const uint8_t length_encoding_bits = utf8_blocks[0] >> 3;

  // Set the number of characters and the top masks based on the
  // length encoding bits.
  uint8_t char_encoding_length = 0, top_mask = 0;
  if (length_encoding_bits < 16) {
    char_encoding_length = 1;
    top_mask             = 0x7F;
  } else if (length_encoding_bits >= 24 && length_encoding_bits <= 27) {
    char_encoding_length = 2;
    top_mask             = 0x1F;
  } else if (length_encoding_bits == 28 || length_encoding_bits == 29) {
    char_encoding_length = 3;
    top_mask             = 0x0F;
  } else if (length_encoding_bits == 30) {
    char_encoding_length = 4;
    top_mask             = 0x07;
  }

  // Now pack up the bits into a uint32_t. All threads will process 4 bytes
  // to reduce divergence.
  uint32_t code_point = (utf8_blocks[0] & top_mask) << 18;

#pragma unroll
  for (int i = 1; i < max_utf8_blocks_for_char; ++i) {
    code_point |= ((utf8_blocks[i] & 0x3F) << (18 - 6 * i));
  }

  // Zero out the bottom of code points with extra reads
  const uint8_t shift_amt = 24 - 6 * char_encoding_length;
  code_point >>= shift_amt;

  return code_point;
}

}  // namespace

__global__ void kernel_data_normalizer(unsigned char const* strings,
                                       uint32_t const* device_strings_offsets,
                                       size_t const total_bytes,
                                       uint32_t const* cp_metadata,
                                       uint64_t const* aux_table,
                                       bool const do_lower_case,
                                       uint32_t const num_strings,
                                       uint32_t* code_points,
                                       uint32_t* chars_per_thread)
{
  constexpr uint32_t init_val                     = (1 << SORT_BIT);
  uint32_t replacement_code_points[MAX_NEW_CHARS] = {init_val, init_val, init_val};

  const uint32_t char_for_thread = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t num_new_chars         = 0;

  if (char_for_thread < total_bytes) {
    auto const code_point = extract_code_points_from_utf8(strings, char_for_thread);
    auto const metadata   = cp_metadata[code_point];

    if (is_head_byte(strings[char_for_thread]) && !should_remove_cp(metadata, do_lower_case)) {
      num_new_chars = 1;
      // Apply lower cases and accent stripping if necessary
      auto const new_cp =
        do_lower_case || always_replace(metadata) ? get_first_cp(metadata) : code_point;
      replacement_code_points[0] = new_cp == 0 ? code_point : new_cp;

      if (do_lower_case && is_multi_char_transform(metadata)) {
        auto const next_cps          = aux_table[code_point];
        replacement_code_points[1]   = static_cast<uint32_t>(next_cps >> 32);
        auto const potential_next_cp = static_cast<uint32_t>(next_cps);
        replacement_code_points[2] =
          potential_next_cp != 0 ? potential_next_cp : replacement_code_points[2];
        num_new_chars = 2 + (potential_next_cp != 0);
      }

      if (should_add_spaces(metadata, do_lower_case)) {
        // Need to shift all existing code-points up one
        // TODO: see if there is algorithm for this
        for (int loc = num_new_chars; loc > 0; --loc) {
          replacement_code_points[loc] = replacement_code_points[loc - 1];
        }

        // Write the required spaces at the end
        replacement_code_points[0]                 = SPACE_CODE_POINT;
        replacement_code_points[num_new_chars + 1] = SPACE_CODE_POINT;
        num_new_chars += 2;
      }
    }
  }

  chars_per_thread[char_for_thread] = num_new_chars;

  typedef cub::
    BlockStore<uint32_t, THREADS_PER_BLOCK, MAX_NEW_CHARS, hipcub::BLOCK_STORE_WARP_TRANSPOSE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage temp_storage;

  // Now we perform coalesced writes back to global memory using cub.
  uint32_t* block_base = code_points + blockIdx.x * blockDim.x * MAX_NEW_CHARS;
  BlockStore(temp_storage).Store(block_base, replacement_code_points);
}

data_normalizer::data_normalizer(uint32_t max_num_strings,
                                 uint32_t max_num_chars,
                                 bool do_lower_case,
                                 hipStream_t stream)
  : do_lower_case(do_lower_case),
    device_strings_offsets(max_num_strings + 1),
    device_strings(max_num_chars),
    device_cp_metadata(codepoint_metadata_size),
    device_aux_table(aux_codepoint_data_size)
{
  // build the code point metadata tables in device memory
  // using the vector pieces from codepoint_metadata.ah
  auto execpol = rmm::exec_policy(stream);
  thrust::fill(execpol->on(stream),
               device_cp_metadata.begin() + cp_section1_end,
               device_cp_metadata.end(),
               codepoint_metadata_default_value);
  CUDA_TRY(hipMemcpyAsync(device_cp_metadata.data().get(),
                           codepoint_metadata,
                           cp_section1_end * sizeof(codepoint_metadata[0]),  // 1st section
                           hipMemcpyHostToDevice,
                           stream));
  CUDA_TRY(hipMemcpyAsync(
    device_cp_metadata.data().get() + cp_section2_begin,
    cp_metadata_917505_917999,
    (cp_section2_end - cp_section2_begin + 1) * sizeof(codepoint_metadata[0]),  // 2nd section
    hipMemcpyHostToDevice,
    stream));

  thrust::fill(execpol->on(stream),
               device_aux_table.begin() + aux_section1_end,
               device_aux_table.end(),
               aux_codepoint_default_value);
  CUDA_TRY(hipMemcpyAsync(device_aux_table.data().get(),
                           aux_codepoint_data,
                           aux_section1_end * sizeof(aux_codepoint_data[0]),  // 1st section
                           hipMemcpyHostToDevice,
                           stream));
  CUDA_TRY(hipMemcpyAsync(
    device_aux_table.data().get() + aux_section2_begin,
    aux_cp_data_44032_55203,
    (aux_section2_end - aux_section2_begin + 1) * sizeof(aux_codepoint_data[0]),  // 2nd section
    hipMemcpyHostToDevice,
    stream));
  CUDA_TRY(hipMemcpyAsync(
    device_aux_table.data().get() + aux_section3_begin,
    aux_cp_data_70475_71099,
    (aux_section3_end - aux_section3_begin + 1) * sizeof(aux_codepoint_data[0]),  // 3rd section
    hipMemcpyHostToDevice,
    stream));
  CUDA_TRY(hipMemcpyAsync(
    device_aux_table.data().get() + aux_section4_begin,
    aux_cp_data_119134_119232,
    (aux_section4_end - aux_section4_begin + 1) * sizeof(aux_codepoint_data[0]),  // 4th section
    hipMemcpyHostToDevice,
    stream));

  size_t max_BLOCKS               = (max_num_chars + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  size_t max_threads_on_device    = max_BLOCKS * THREADS_PER_BLOCK;
  const size_t max_new_char_total = MAX_NEW_CHARS * max_threads_on_device;
  device_code_points.resize(max_new_char_total);
  device_chars_per_thread.resize(max_threads_on_device);

  // TODO: using thrust instead would alleviate this extra processing
  // Determine temporary device storage requirements for cub
  //  size_t temp_storage_scan_bytes    = 0;
  //  uint32_t* device_chars_per_thread = nullptr;
  //  hipcub::DeviceScan::InclusiveSum(nullptr,
  //                                temp_storage_scan_bytes,
  //                                device_chars_per_thread,
  //                                device_chars_per_thread,
  //                                max_threads_on_device,
  //                                stream);
  //  size_t temp_storage_select_bytes = 0;
  //  static NotEqual select_op((1 << SORT_BIT));
  //  hipcub::DeviceSelect::If(nullptr,
  //                        temp_storage_select_bytes,
  //                        device_code_points.data().get(),
  //                        device_code_points.data().get(),
  //                        device_num_selected.data().get(),
  //                        max_new_char_total,
  //                        select_op,
  //                        stream);
  //  max_cub_storage_bytes = std::max(temp_storage_scan_bytes, temp_storage_select_bytes);
  //  cub_temp_storage.resize(max_cub_storage_bytes);
  //  device_num_selected.resize(1);
}

std::pair<ptr_length_pair, ptr_length_pair> data_normalizer::normalize(const char* d_strings,
                                                                       const uint32_t* d_offsets,
                                                                       uint32_t num_strings,
                                                                       hipStream_t stream)
{
  ptr_length_pair cp_and_length;
  ptr_length_pair offset_and_length;

  // copy offsets to working memory
  size_t num_offsets = std::min(size_t{num_strings + 1}, device_strings_offsets.size());
  CUDA_TRY(hipMemcpyAsync(device_strings_offsets.data().get(),
                           d_offsets,
                           sizeof(uint32_t) * num_offsets,
                           hipMemcpyDeviceToDevice,
                           stream));
  uint32_t bytes_count = device_strings_offsets[num_offsets - 1];

  size_t BLOCKS                   = (bytes_count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  const size_t max_new_char_total = MAX_NEW_CHARS * BLOCKS * THREADS_PER_BLOCK;
  size_t threads_on_device        = BLOCKS * THREADS_PER_BLOCK;

  kernel_data_normalizer<<<BLOCKS, THREADS_PER_BLOCK, 0, stream>>>(
    reinterpret_cast<const unsigned char*>(d_strings),
    device_strings_offsets.data().get(),
    bytes_count,
    device_cp_metadata.data().get(),
    device_aux_table.data().get(),
    do_lower_case,
    num_strings,
    device_code_points.data().get(),
    device_chars_per_thread.data().get());
  CHECK_CUDA(stream);

  // TODO: see if thrust remove-if and inclusive_scan (below) performance is equivalent
  // static NotEqual select_op((1 << SORT_BIT));
  // hipcub::DeviceSelect::If(cub_temp_storage.data().get(),
  //                      max_cub_storage_bytes,
  //                      device_code_points.data().get(),
  //                      device_code_points.data().get(),
  //                      device_num_selected.data().get(),
  //                      max_new_char_total,
  //                      select_op,
  //                      stream);
  // CHECK_CUDA(stream);
  auto execpol = rmm::exec_policy(stream);
  thrust::remove_if(execpol->on(stream),
                    device_code_points.begin(),
                    device_code_points.begin() + max_new_char_total,
                    [] __device__(auto value) { return value == (1 << SORT_BIT); });

  // We also need to prefix sum the number of characters up to an including the current character in
  // order to get the new strings lengths.
  // hipcub::DeviceScan::InclusiveSum(cub_temp_storage.data().get(),
  //                              max_cub_storage_bytes,
  //                              device_chars_per_thread.data().get(),
  //                              device_chars_per_thread.data().get(),
  //                              threads_on_device,
  //                              stream);
  // CHECK_CUDA(stream);
  thrust::inclusive_scan(execpol->on(stream),
                         device_chars_per_thread.begin(),
                         device_chars_per_thread.begin() + threads_on_device,
                         device_chars_per_thread.begin());

  constexpr uint16_t SENTENCE_UPDATE_THREADS = 64;
  size_t SEN_KERNEL_BLOCKS = (num_strings + SENTENCE_UPDATE_THREADS - 1) / SENTENCE_UPDATE_THREADS;
  update_strings_lengths<<<SEN_KERNEL_BLOCKS, SENTENCE_UPDATE_THREADS, 0, stream>>>(
    device_strings_offsets.data().get(), device_chars_per_thread.data().get(), num_strings);
  CHECK_CUDA(stream);

  offset_and_length.gpu_ptr = device_strings_offsets.data().get();
  offset_and_length.length  = num_strings + 1;

  uint32_t num_chars = 0;
  CUDA_TRY(hipMemcpyAsync(&num_chars,
                           offset_and_length.gpu_ptr + num_strings,
                           sizeof(num_chars),
                           hipMemcpyDeviceToHost,
                           stream));
  cp_and_length.gpu_ptr = device_code_points.data().get();
  cp_and_length.length  = num_chars;

  return std::make_pair(cp_and_length, offset_and_length);
}

}  // namespace detail
}  // namespace nvtext

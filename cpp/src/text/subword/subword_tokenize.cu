#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/utilities/error.hpp>
#include <nvtext/detail/load_hash_file.hpp>
#include <nvtext/subword_tokenize.hpp>
#include <text/subword/detail/wordpiece_tokenizer.hpp>

#include <thrust/for_each.h>
#include <thrust/transform_scan.h>
#include <fstream>
#include <iostream>
#include <vector>

namespace nvtext {
namespace detail {
namespace {

/**
 * @brief Convert tokens and row2tensor map to final tensor data.
 *
 * @param[in] token_ids Tokens from tokenizer
 * @param[in] offsets Offsets to each string's output row of tokens
 * @param[in] row2tensor String to tensor token counts
 * @param[in] row2row_within_tensor Token counts within sub-rows of the output
 * @param[in] max_sequence_length Maximum number of tokens in a row
 * @param[in] stride Number of tokens in sub-rows
 * @param[in] do_truncate True if tokens should not spill into sub-rows in the output
 * @param[out] final_tensor Output vector of token-ids
 * @param[out] attn_mask Identifies valid token id entries
 * @param[out] metadata Additional data per row
 */
__global__ void kernel_compute_tensor_metadata(
  // input
  uint32_t const* token_ids,
  uint32_t const* offsets,
  uint32_t const* row2tensor,
  uint32_t const* row2row_within_tensor,
  uint32_t max_sequence_length,
  uint32_t stride,
  bool do_truncate,
  // output
  uint32_t* final_tensor,
  uint32_t* attn_mask,
  uint32_t* metadata)
{
  uint32_t const absolute_row_id         = blockIdx.x;
  uint32_t const tensor_id               = row2tensor[absolute_row_id];
  uint32_t const row_within_tensor       = row2row_within_tensor[absolute_row_id];
  uint32_t const offset_token_ids_tensor = offsets[tensor_id];
  uint32_t const n_tokens_tensor         = offsets[tensor_id + 1] - offset_token_ids_tensor;
  bool const last_row_of_tensor =
    (absolute_row_id == gridDim.x - 1) || (row2tensor[absolute_row_id + 1] != tensor_id);

  uint32_t const row_offset_token_ids =
    offset_token_ids_tensor +
    (row_within_tensor ? (max_sequence_length + (stride * (row_within_tensor - 1))) : 0);

  auto const output_idx = absolute_row_id * max_sequence_length + threadIdx.x;

  if (row_within_tensor == 0) {
    if (threadIdx.x < n_tokens_tensor) {
      // copy token ids
      final_tensor[output_idx] = token_ids[row_offset_token_ids + threadIdx.x];
      attn_mask[output_idx]    = 1;
    } else {
      // pad with 0
      final_tensor[output_idx] = 0;
      attn_mask[output_idx]    = 0;
    }
  } else {
    uint32_t const n_replicates = max_sequence_length - stride;
    if ((row_offset_token_ids - n_replicates + threadIdx.x) <
        (offset_token_ids_tensor + n_tokens_tensor)) {
      // replicate elements or copy new tokens
      final_tensor[output_idx] = token_ids[row_offset_token_ids - n_replicates + threadIdx.x];
      attn_mask[output_idx]    = 1;
    } else {
      // pad with 0
      final_tensor[output_idx] = 0;
      attn_mask[output_idx]    = 0;
    }
  }

  // write metadata
  if (threadIdx.x == 0) {
    metadata[absolute_row_id * 3] = tensor_id;
    if (row_within_tensor == 0)
      metadata[absolute_row_id * 3 + 1] = 0;
    else
      metadata[absolute_row_id * 3 + 1] = (max_sequence_length - stride) / 2;
    if (last_row_of_tensor) {
      if (n_tokens_tensor < max_sequence_length)
        metadata[absolute_row_id * 3 + 2] = n_tokens_tensor - 1;
      else {
        if (!do_truncate)
          metadata[absolute_row_id * 3 + 2] =
            (max_sequence_length - stride) + (n_tokens_tensor - max_sequence_length) % stride - 1;
        else
          // truncate
          metadata[absolute_row_id * 3 + 2] = (max_sequence_length - 1);
      }
    } else
      metadata[absolute_row_id * 3 + 2] =
        max_sequence_length - (max_sequence_length - stride) / 2 - 1;
  }
}

}  // namespace

tokenizer_result subword_tokenize(cudf::strings_column_view const& strings,
                                  hashed_vocabulary const& vocab_table,
                                  uint32_t max_sequence_length,
                                  uint32_t stride,
                                  bool do_lower_case,
                                  bool do_truncate,
                                  uint32_t max_num_strings,
                                  uint32_t max_num_chars,
                                  uint32_t max_rows_tensor,
                                  hipStream_t stream,
                                  rmm::mr::device_memory_resource* mr)
{
  auto const strings_count = strings.size();
  auto const offsets       = strings.offsets();
  auto const d_offsets     = offsets.data<uint32_t>() + strings.offset();
  auto const offset        = cudf::detail::get_value<int32_t>(offsets, strings.offset(), stream);
  auto const chars_bytes =
    cudf::detail::get_value<int32_t>(offsets, strings.offset() + strings_count, stream) - offset;
  auto const d_chars = strings.chars().data<char>() + offset;

  // Create tokenizer
  wordpiece_tokenizer tokenizer(vocab_table,
                                max_num_strings,
                                max_num_chars,
                                max_rows_tensor,
                                max_sequence_length,
                                stride,
                                do_truncate,
                                do_lower_case,
                                stream);
  // Run tokenizer
  auto const tokens = tokenizer.tokenize(d_chars, d_offsets, strings_count, stream);
  // assign output components
  uint32_t const* device_token_ids = tokens.first;
  uint32_t const* device_offsets   = tokens.second;

  // Format output from tokenizer
  // each string can create 1 or more tensor entries
  // compute the string-per-tensor offsets values by scanning over the number of tokens for each
  // string
  rmm::device_uvector<uint32_t> offsets_per_tensor(strings_count + 1, stream);
  auto d_offsets_per_tensor = offsets_per_tensor.data();
  auto const execpol        = rmm::exec_policy(stream);
  thrust::transform_exclusive_scan(
    execpol->on(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    thrust::make_counting_iterator<cudf::size_type>(strings_count + 1),
    offsets_per_tensor.begin(),
    [device_offsets, do_truncate, max_sequence_length, stride] __device__(cudf::size_type idx) {
      uint32_t num_tokens = device_offsets[idx + 1] - device_offsets[idx];
      if (do_truncate || num_tokens <= max_sequence_length) return uint32_t{1};
      return 1 + ((num_tokens - max_sequence_length + stride - 1) / stride);
    },
    uint32_t{0},
    thrust::plus<uint32_t>());
  // last element is the total number of tokens
  uint32_t const nrows_tensor_token_ids = offsets_per_tensor.element(strings_count, stream);

  // compute global_row to tensor, and global_row to within_tensor_row correspondence
  rmm::device_uvector<uint32_t> row2tensor(nrows_tensor_token_ids, stream);
  auto d_row2tensor = row2tensor.data();
  rmm::device_uvector<uint32_t> row2row_within_tensor(nrows_tensor_token_ids, stream);
  auto d_row2row_within_tensor = row2row_within_tensor.data();
  thrust::for_each_n(
    execpol->on(stream),
    thrust::make_counting_iterator<uint32_t>(0),
    strings_count,
    [d_offsets_per_tensor, d_row2tensor, d_row2row_within_tensor] __device__(auto idx) {
      uint32_t offset = d_offsets_per_tensor[idx];
      uint32_t nrows  = d_offsets_per_tensor[idx + 1] - offset;
      for (uint32_t jdx = 0; jdx < nrows; ++jdx) {
        d_row2tensor[jdx + offset]            = idx;
        d_row2row_within_tensor[jdx + offset] = jdx;
      }
    });

  // create output data columns
  auto tensor_token_ids = cudf::make_numeric_column(cudf::data_type{cudf::type_id::UINT32},
                                                    nrows_tensor_token_ids * max_sequence_length,
                                                    cudf::mask_state::UNALLOCATED,
                                                    stream,
                                                    mr);
  auto tensor_attention_mask =
    cudf::make_numeric_column(cudf::data_type{cudf::type_id::UINT32},
                              nrows_tensor_token_ids * max_sequence_length,
                              cudf::mask_state::UNALLOCATED,
                              stream,
                              mr);
  auto tensor_metadata = cudf::make_numeric_column(cudf::data_type{cudf::type_id::UINT32},
                                                   nrows_tensor_token_ids * 3,
                                                   cudf::mask_state::UNALLOCATED,
                                                   stream,
                                                   mr);

  // compute final-tensor, mask, and metadata
  kernel_compute_tensor_metadata<<<nrows_tensor_token_ids, max_sequence_length, 0, stream>>>(
    device_token_ids,
    device_offsets,
    d_row2tensor,
    d_row2row_within_tensor,
    max_sequence_length,
    stride,
    do_truncate,
    tensor_token_ids->mutable_view().data<uint32_t>(),
    tensor_attention_mask->mutable_view().data<uint32_t>(),
    tensor_metadata->mutable_view().data<uint32_t>());

  return tokenizer_result{nrows_tensor_token_ids,
                          max_sequence_length,
                          std::move(tensor_token_ids),
                          std::move(tensor_attention_mask),
                          std::move(tensor_metadata)};
}

}  // namespace detail

tokenizer_result subword_tokenize(cudf::strings_column_view const& strings,
                                  std::string const& filename_hashed_vocabulary,
                                  uint32_t max_sequence_length,
                                  uint32_t stride,
                                  bool do_lower_case,
                                  bool do_truncate,
                                  uint32_t max_num_strings,
                                  uint32_t max_num_chars,
                                  uint32_t max_rows_tensor,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  hashed_vocabulary vocab_table = load_vocabulary_file(filename_hashed_vocabulary, mr);
  return detail::subword_tokenize(strings,
                                  vocab_table,
                                  max_sequence_length,
                                  stride,
                                  do_lower_case,
                                  do_truncate,
                                  max_num_strings,
                                  max_num_chars,
                                  max_rows_tensor,
                                  0,
                                  mr);
}

tokenizer_result subword_tokenize(cudf::strings_column_view const& strings,
                                  hashed_vocabulary const& vocabulary_table,
                                  uint32_t max_sequence_length,
                                  uint32_t stride,
                                  bool do_lower_case,
                                  bool do_truncate,
                                  uint32_t max_num_strings,
                                  uint32_t max_num_chars,
                                  uint32_t max_rows_tensor,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::subword_tokenize(strings,
                                  vocabulary_table,
                                  max_sequence_length,
                                  stride,
                                  do_lower_case,
                                  do_truncate,
                                  max_num_strings,
                                  max_num_chars,
                                  max_rows_tensor,
                                  0,
                                  mr);
}

}  // namespace nvtext

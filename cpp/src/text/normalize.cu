/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/text/normalize.hpp>
#include <strings/utilities.cuh>
#include <strings/utilities.hpp>

#include <text/utilities/tokenize_ops.cuh>

#include <thrust/for_each.h>

namespace cudf
{
namespace nvtext
{
namespace detail
{
namespace
{

/**
 * @brief Normalize spaces in a strings column.
 *
 * Repeated whitespace is replaced with a single space.
 * Also, whitespace is trimmed from the beginning and end of each string.
 *
 * This functor can be called to compute the output size in bytes
 * of each string and then called again to fill in the allocated buffer.
 */
struct normalize_spaces_fn : base_tokenator
{
    column_device_view d_strings;
    int32_t const* d_offsets{}; // offsets into d_buffer
    char* d_buffer{};           // output buffer for characters

    normalize_spaces_fn( column_device_view d_strings,
                         int32_t const* d_offsets = nullptr,
                         char* d_buffer = nullptr )
        : d_strings(d_strings), d_offsets(d_offsets), d_buffer(d_buffer) {}
    //
    __device__ int32_t operator()(unsigned int idx)
    {
        if( d_strings.is_null(idx) )
            return 0;
        string_view single_space(" ",1);
        string_view d_str = d_strings.element<string_view>(idx);
        char* buffer = d_offsets ? d_buffer + d_offsets[idx] : nullptr;
        char* optr = buffer; // running output pointer
        int32_t nbytes = 0;  // holds the number of bytes per output string
        size_type spos = 0;  // start position of current token
        size_type epos = d_str.length();  // end position of current token
        bool spaces = true;  // true to trim whitespace from the beginning
        auto itr = d_str.begin();
        // this will retrieve tokens automatically skipping runs of whitespace
        while( next_token(d_str,spaces,itr,spos,epos) )
        {
            auto spos_bo = d_str.byte_offset(spos); // convert character position
            auto epos_bo = d_str.byte_offset(epos); // values to byte offsets
            nbytes += epos_bo - spos_bo + 1; // token size plus a single space
            if( optr )
            {
                string_view token( d_str.data() + spos_bo, epos_bo - spos_bo );
                if( optr != buffer ) // prepend space unless we are at the beginning
                    optr = strings::detail::copy_string(optr,single_space);
                // write token to output buffer
                optr = strings::detail::copy_string(optr,token); // copy token to output
            }
            spos = epos + 1;
            ++itr; // next character
        }
        return (nbytes>0) ? nbytes-1:0; // remove trailing space
    }
};

} // namspace

// details API
std::unique_ptr<column> normalize_spaces( strings_column_view const& strings,
                                          rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                          hipStream_t stream = 0 )
{
    size_type strings_count = strings.size();
    if( strings_count == 0 )
        return make_empty_column(data_type{STRING});
    // create device column
    auto strings_column = column_device_view::create(strings.parent(), stream);
    auto d_strings = *strings_column;
    // copy bitmask
    rmm::device_buffer null_mask = copy_bitmask( strings.parent(), stream, mr );
    // create offsets by calculating size of each string for output
    auto offsets_transformer_itr = thrust::make_transform_iterator( thrust::make_counting_iterator<int32_t>(0),
        normalize_spaces_fn{d_strings} ); // this does size-only calc
    auto offsets_column = strings::detail::make_offsets_child_column(offsets_transformer_itr,
                                                                     offsets_transformer_itr+strings_count,
                                                                     mr, stream);
    auto d_offsets = offsets_column->view().data<int32_t>();
    // build the chars column
    size_type bytes = thrust::device_pointer_cast(d_offsets)[strings_count];
    auto chars_column = strings::detail::create_chars_child_column( strings_count, strings.null_count(), bytes, mr, stream );
    auto d_chars = chars_column->mutable_view().data<char>();
    // copy tokens to the chars buffer
    thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
        thrust::make_counting_iterator<size_type>(0), strings_count,
        normalize_spaces_fn{d_strings, d_offsets, d_chars} );
    chars_column->set_null_count(0); // reset null count for child column
    //
    return make_strings_column(strings_count, std::move(offsets_column), std::move(chars_column),
                               strings.null_count(), std::move(null_mask), stream, mr);
}

} // namespace detail

// external APIs

std::unique_ptr<column> normalize_spaces( strings_column_view const& strings,
                                          rmm::mr::device_memory_resource* mr )
{
    return detail::normalize_spaces( strings, mr );
}

} // namespace nvtext
} // namespace cudf

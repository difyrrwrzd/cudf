/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/types.hpp>
#include <utilities/error_utils.hpp>

#include <rmm/rmm_api.h>
#include <rmm/thrust_rmm_allocator.h>

namespace cudf {

// Trivially copy all members but the children
column_device_view::column_device_view(column_view source)
    : detail::column_device_view_base{source.type(),       source.size(),
                                      source.head(),       source.null_mask(),
                                      source.null_count(), source.offset()},
      _num_children{source.num_children()} {}

// Free device memory allocated for children
void column_device_view::destroy() {
  // TODO Needs to handle grand-children
  if( d_children )
    RMM_FREE(d_children,0);
  delete this;
}

// For use with inplace-new to pre-fill memory to be copied to device
column_device_view::column_device_view( column_view source, ptrdiff_t h_ptr, ptrdiff_t d_ptr )
    : detail::column_device_view_base{source.type(),       source.size(),
                                      source.head(),       source.null_mask(),
                                      source.null_count(), source.offset()},
      _num_children{source.num_children()}
{
  if( _num_children > 0 )
  {
    column_device_view* h_column = reinterpret_cast<column_device_view*>(h_ptr);
    column_device_view* d_column = reinterpret_cast<column_device_view*>(d_ptr);
    d_children = d_column;
    for( size_type idx=0; idx < _num_children; ++idx )
    { // inplace-new each child
      column_view child = source.child(idx);
      CUDF_EXPECTS( child.num_children()==0, "column grand-children not currently supported");
      new(h_column) column_device_view(child);
      h_column++;
      //d_column++;
    }
  }
}

// For use with inplace-new to pre-fill memory to be copied to device
mutable_column_device_view::mutable_column_device_view( mutable_column_view source, ptrdiff_t h_ptr, ptrdiff_t d_ptr )
    : detail::column_device_view_base{source.type(),       source.size(),
                                      source.head(),       source.null_mask(),
                                      source.null_count(), source.offset()}
{}

// Construct a unique_ptr that invokes `destroy()` as it's deleter
std::unique_ptr<column_device_view, std::function<void(column_device_view*)>> column_device_view::create(column_view source, hipStream_t stream) {
  //size_type num_descendants{count_descendants(source)};
  //if( num_descendants > 0 )   {
  //  CUDF_FAIL("Columns with children are not currently supported.");
  // }
  auto deleter = [](column_device_view* v) { v->destroy(); };
  std::unique_ptr<column_device_view, decltype(deleter)> p{
      new column_device_view(source), deleter};
  size_type num_children = source.num_children();
  if( num_children > 0 )
  {
    // ignore grand-children right now
    RMM_ALLOC(&p->d_children, sizeof(column_device_view)*num_children, stream);
    for( size_type idx=0; idx < num_children; ++idx )
    {
      column_device_view child(source.child(idx));
      CUDF_EXPECTS( child._num_children==0, "column grand-children not currently supported");
      CUDA_TRY(hipMemcpyAsync(p->d_children+idx, &child, sizeof(column_device_view),
                               hipMemcpyHostToDevice, stream));
    }
    p->_num_children = num_children;
    hipStreamSynchronize(stream);
  }
  return p;
}

size_type column_device_view::extent(column_view source) {
  size_type data_size = sizeof(column_device_view);
  for( size_type idx=0; idx < source.num_children(); ++idx )
    data_size += extent(source.child(idx));
  return data_size;
}

size_type mutable_column_device_view::extent(column_view source) {
  size_type data_size = sizeof(column_device_view);
  for( size_type idx=0; idx < source.num_children(); ++idx )
    data_size += extent(source.child(idx));
  return data_size;
}


}  // namespace cudf
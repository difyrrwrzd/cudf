/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/types.hpp>
#include <utilities/error_utils.hpp>

#include <rmm/rmm_api.h>
#include <rmm/thrust_rmm_allocator.h>

namespace cudf {

// Trivially copy all members but the children
column_device_view::column_device_view(column_view source)
    : detail::column_device_view_base{source.type(),       source.size(),
                                      source.head(),       source.null_mask(),
                                      source.null_count(), source.offset()},
      _num_children{source.num_children()} {}

// Free device memory allocated for children
void column_device_view::destroy() {
  // TODO Needs to handle grand-children
  if( d_children )
    RMM_FREE(d_children,0);
  delete this;
}

// Construct a unique_ptr that invokes `destroy()` as it's deleter
std::unique_ptr<column_device_view, std::function<void(column_device_view*)>> column_device_view::create(column_view source, hipStream_t stream) {
  size_type num_descendants{count_descendants(source)};
  //if( num_descendants > 0 )   {
  //  CUDF_FAIL("Columns with children are not currently supported.");
  // }
  auto deleter = [](column_device_view* v) { v->destroy(); };
  std::unique_ptr<column_device_view, decltype(deleter)> p{
      new column_device_view(source), deleter};
  if( num_descendants > 0 )
  {
    // ignore grand-children right now
    RMM_ALLOC(&p->d_children, sizeof(column_device_view)*num_descendants, stream);
    for( size_type idx=0; idx < num_descendants; ++idx )
    {
      column_device_view child(source.child(idx));
      hipMemcpyAsync(p->d_children+idx, &child, sizeof(column_device_view), hipMemcpyHostToDevice, stream);
    }
    p->_num_children = num_descendants;
    hipStreamSynchronize(stream);
  }
  return p;
}

}  // namespace cudf
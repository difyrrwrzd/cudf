#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy_if.cuh>
#include <cudf/stream_compaction.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/detail/gather.hpp>
#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <algorithm>
#include <cmath>

namespace cudf {
namespace experimental {
namespace detail {

/*
 * unique_copy copies elements from the range [first, last) to a range beginning
 * with output, except that in a consecutive group of duplicate elements only
 * depending on last argument keep, only the first one is copied, or the last
 * one is copied or neither is copied. The return value is the end of the range
 * to which the elements are copied.
 */
template<typename Exec,
         typename InputIterator,
         typename OutputIterator,
         typename BinaryPredicate,
    typename IndexType = typename
  thrust::iterator_difference<InputIterator>::type>
  OutputIterator unique_copy(Exec&& exec,
                             InputIterator first,
                             InputIterator last,
                             OutputIterator output,
                             BinaryPredicate comp,
                             const duplicate_keep_option keep)
{
  IndexType n = (last-first)-1;
  if (keep == duplicate_keep_option::KEEP_FIRST) {
      return thrust::copy_if(exec,
              first,
              last,
              thrust::counting_iterator<IndexType>(0),
              output, 
              [first, comp, n] __device__ (IndexType i) mutable {
              return (i == 0 || !comp(first[i], first[i-1]));
              }); 
  } else if (keep == duplicate_keep_option::KEEP_LAST) {
      return thrust::copy_if(exec,
              first,
              last,
              thrust::counting_iterator<IndexType>(0),
              output, 
              [first, comp, n] __device__ (IndexType i) mutable {
              return (i == n || !comp(first[i], first[i+1]));
              });
  } else {
      return thrust::copy_if(exec,
              first,
              last,
              thrust::counting_iterator<IndexType>(0),
              output, 
              [first, comp, n] __device__ (IndexType i) mutable {
              return (i == 0 || !comp(first[i], first[i-1])) 
                  && (i == n || !comp(first[i], first[i+1]));
              });
  }
}

auto get_unique_ordered_indices(cudf::table_view const& keys,
                                cudf::column &unique_indices,
                                duplicate_keep_option const& keep,
                                bool const& nulls_are_equal = true,
                                rmm::mr::device_memory_resource* mr =
                                rmm::mr::get_default_resource(),
                                hipStream_t stream=0)
{
  // sort only indices
  auto sorted_indices = sorted_order(keys,
                                     std::vector<order>{},
                                     std::vector<null_order>{},
                                     stream,
                                     mr);


  // extract unique indices 
  auto device_input_table = cudf::table_device_view::create(keys, stream);

  if(cudf::has_nulls(keys)) {
    auto comp = row_equality_comparator<true>(*device_input_table,
                                              *device_input_table,
                                               nulls_are_equal);
    auto result_end = unique_copy(rmm::exec_policy(stream)->on(stream),
        sorted_indices->view().begin<cudf::size_type>(),
        sorted_indices->view().end<cudf::size_type>(),
        unique_indices.mutable_view().begin<cudf::size_type>(),
        comp,
        keep);
  
    return cudf::detail::slice(unique_indices.view(), 0, thrust::distance(unique_indices.mutable_view().begin<cudf::size_type>(), result_end));
  } else {
    auto comp = row_equality_comparator<false>(*device_input_table,
                                               *device_input_table,
                                               nulls_are_equal);
    auto result_end = unique_copy(rmm::exec_policy(stream)->on(stream),
        sorted_indices->view().begin<cudf::size_type>(),
        sorted_indices->view().end<cudf::size_type>(),
        unique_indices.mutable_view().begin<cudf::size_type>(),
        comp,
        keep);
  
    return cudf::detail::slice(unique_indices.view(), 0, thrust::distance(unique_indices.mutable_view().begin<cudf::size_type>(), result_end));
  }
  
}

cudf::size_type unique_count(table_view const& keys,
                             bool const& nulls_are_equal = true,
                             rmm::mr::device_memory_resource* mr 
                                 = rmm::mr::get_default_resource(),
                             hipStream_t stream=0)
{
  // sort only indices
  auto sorted_indices = sorted_order(keys,
                                     std::vector<order>{},
                                     std::vector<null_order>{},
                                     stream,
                                     mr);
  
  // count unique elements
  cudf::size_type* sorted_row_index = const_cast<cudf::size_type *>(sorted_indices->view().data<cudf::size_type>());
  auto device_input_table = cudf::table_device_view::create(keys, stream);

  if(cudf::has_nulls(keys)) {
    row_equality_comparator<true> comp (*device_input_table,
                                              *device_input_table,
                                              nulls_are_equal);
    return thrust::count_if(rmm::exec_policy(stream)->on(stream),
              thrust::counting_iterator<cudf::size_type>(0),
              thrust::counting_iterator<cudf::size_type>(keys.num_rows()),
              [sorted_row_index, comp]
              __device__ (cudf::size_type i) mutable {
              return (i == 0 || not comp(sorted_row_index[i], sorted_row_index[i-1]));
              });
  } else {
    row_equality_comparator<false> comp(*device_input_table,
                                              *device_input_table,
                                              nulls_are_equal);
    return thrust::count_if(rmm::exec_policy(stream)->on(stream),
              thrust::counting_iterator<cudf::size_type>(0),
              thrust::counting_iterator<cudf::size_type>(keys.num_rows()),
              [sorted_row_index, comp]
              __device__ (cudf::size_type i) mutable {
              return (i == 0 || not comp(sorted_row_index[i], sorted_row_index[i-1]));
              });
  }
}

std::unique_ptr<experimental::table>
  drop_duplicates(table_view const& input,
                  table_view const& keys,
                  duplicate_keep_option const& keep,
                  bool const& nulls_are_equal,
                  rmm::mr::device_memory_resource* mr,
                  hipStream_t stream)
{
  if (0 == input.num_rows() || 
      0 == input.num_columns() ||
      0 == keys.num_columns()
      ) {
      std::vector<std::unique_ptr<column>> out_columns(input.num_columns());
      std::transform(input.begin(), input.end(), out_columns.begin(),
                [&stream] (auto col_view){
                return detail::empty_like(col_view, stream);
                });

    return std::make_unique<experimental::table>(std::move(out_columns));
  }
  
  CUDF_EXPECTS( input.num_rows() == keys.num_rows(), "number of \
rows in input table should be equal to number of rows in key colums table");

  // The values will be filled into this column
  auto unique_indices = 
        cudf::make_numeric_column(data_type{INT32}, 
                                  keys.num_rows(), UNALLOCATED, stream, mr);
  // This is just slice of `unique_indices` but with different size as per the
  // keys has been processed in `get_unique_ordered_indices`
  auto unique_indices_view = 
      detail::get_unique_ordered_indices(keys, 
                                         *unique_indices,
                                         keep, nulls_are_equal);
 
  // run gather operation to establish new order
  return detail::gather(input, unique_indices_view, false, false, true, mr, stream);
}

template <typename T>
struct check_for_nan
{
    check_for_nan(cudf::column_device_view input) :_input{input}{}
  __device__
  bool operator()(size_type index)
  {
    return std::isnan(_input.data<T>()[index]) and _input.is_valid(index);
  }

protected:
  cudf::column_device_view _input;
};

cudf::size_type unique_count(column_view const& input,
                             bool const& ignore_nulls,
                             bool const& nan_as_null,
                             rmm::mr::device_memory_resource *mr,
                             hipStream_t stream)
{
  if (0 == input.size() || input.null_count() == input.size()) {
    return 0;
  }

  cudf::size_type nrows = input.size();
 
  bool has_nans = false;
  // Check for Nans
  if (input.type().id() == FLOAT32 and input.has_nulls() and nan_as_null) {

      auto input_device_view = cudf::column_device_view::create(input, stream);
      auto device_view = *input_device_view;
      has_nans = thrust::any_of(rmm::exec_policy(stream)->on(stream), 
                                thrust::counting_iterator<cudf::size_type>(0),
                                thrust::counting_iterator<cudf::size_type>(nrows),
                                check_for_nan<float>(device_view));

  }
  else if (input.type().id() == FLOAT64 and input.has_nulls() and nan_as_null) {

      auto input_device_view = cudf::column_device_view::create(input, stream);
      auto device_view = *input_device_view;
      has_nans = thrust::any_of(rmm::exec_policy(stream)->on(stream),
                                thrust::counting_iterator<cudf::size_type>(0),
                                thrust::counting_iterator<cudf::size_type>(nrows),
                                check_for_nan<double>(device_view));
      
  }

  auto count = detail::unique_count(table_view{{input}}, true, mr, stream);


  // if nan is considered null and there are already null values
  if (nan_as_null and has_nans and input.has_nulls())
    --count;

  if(ignore_nulls and input.has_nulls())
    return --count;
  else
    return count;
}

}// namespace detail

std::unique_ptr<experimental::table>
  drop_duplicates(table_view const& input,
                  table_view const& keys,
                  duplicate_keep_option const& keep,
                  bool const& nulls_are_equal,
                  rmm::mr::device_memory_resource* mr) {

    return detail::drop_duplicates(input, keys, keep, nulls_are_equal, mr);
}

cudf::size_type unique_count(column_view const& input,
                             bool const& ignore_nulls,
                             bool const& nan_as_null,
                             rmm::mr::device_memory_resource *mr) {

    return detail::unique_count(input, ignore_nulls, nan_as_null, mr);
}

}// namespace experimental
}// namespace cudf

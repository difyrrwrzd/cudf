/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "copy_if.cuh"
#include <cudf/legacy/table.hpp>
#include <thrust/logical.h>
 
namespace {

// Returns false if the valid mask is false for index i in ANY/ALL columns of
// table indicated by column_indices, where ANY/ALL is the value of drop_if.
// Columns not indexed by column_indices are not checked
struct valid_table_filter
{
  valid_table_filter(bit_mask_t **masks,
                     gdf_size_type num_columns,
                     cudf::any_or_all drop_if) 
  : drop_if(drop_if),
    num_columns(num_columns),
    d_masks(masks) {}

  __device__ inline 
  bool operator()(gdf_index_type i)
  {
    auto valid = [i](auto mask) { 
      return (mask == nullptr) || bit_mask::is_valid(mask, i);
    };

    if (drop_if == cudf::ALL) // drop rows that have a null in all columns
      return thrust::any_of(thrust::seq, d_masks, d_masks + num_columns, valid);
    else // drop_if == cudf::ANY => drop rows that have any nulls
      return thrust::all_of(thrust::seq, d_masks, d_masks + num_columns, valid); 
  }

  cudf::any_or_all drop_if;
  gdf_size_type num_columns;
  bit_mask_t **d_masks;
};

bit_mask_t** get_bitmasks(cudf::table const &table, hipStream_t stream = 0) {
  bit_mask_t** h_masks = new bit_mask_t*[table.num_columns()];
  
  int i = 0;
  for (auto col : table) {
    h_masks[i++] = reinterpret_cast<bit_mask_t*>(col->valid);
  }

  size_t masks_size = sizeof(bit_mask_t*) * table.num_columns();

  bit_mask_t **d_masks = nullptr;
  RMM_TRY(RMM_ALLOC(&d_masks, masks_size, stream));
  CUDA_TRY(hipMemcpyAsync(d_masks, h_masks, masks_size,
                           hipMemcpyHostToDevice, stream));
  CHECK_STREAM(stream);

  return d_masks;
}

valid_table_filter make_valid_table_filter(cudf::table const &table,
                                           cudf::any_or_all drop_if,
                                           hipStream_t stream=0)
{
  return valid_table_filter(get_bitmasks(table, stream),
                            table.num_columns(),
                            drop_if);
}

void destroy_valid_table_filter(valid_table_filter const& filter,
                                hipStream_t stream = 0) {
  RMM_FREE(filter.d_masks, stream);
}

}  // namespace

namespace cudf {

/*
 * Filters a table to remove null elements.
 */
table drop_nulls(table const &input,
                 table const &keys,
                 any_or_all drop_if) {
  if (keys.num_columns() == 0 || keys.num_rows() == 0 ||
      not cudf::has_nulls(keys))
    return cudf::copy(input);

  CUDF_EXPECTS(keys.num_rows() <= input.num_rows(), 
               "Column size mismatch");
  
  valid_table_filter filter =
    make_valid_table_filter(keys, drop_if);
  table result = detail::copy_if(input, filter);
  destroy_valid_table_filter(filter);
  return result;
}

}  // namespace cudf

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "copy_if.cuh"
#include <cudf/table.hpp>
 
namespace {

// Returns true if the valid mask is true for index i
// Note we use a functor here so we can cast to a bitmask_t __restrict__
// pointer on the host side, which we can't do with a lambda.
struct valid_column_filter
{
  valid_column_filter(gdf_column const & column) :
    size{column.size},
    bitmask{reinterpret_cast<bit_mask_t *>(column.valid)}
    {}

  __device__ inline 
  bool operator()(gdf_index_type i)
  {
    if (bitmask == nullptr) return true;
    if (i < size) {
      bool valid = bit_mask::is_valid(bitmask, i);
      return valid;
    }
    return false;
  }

  bool all_true;
  gdf_size_type size;
  bit_mask_t const  * __restrict__ bitmask;
};


// Returns false if the valid mask is false for index i in ANY/ALL columns of
// table indicated by column_indices, where ANY/ALL is the value of drop_if.
// Columns not indexed by column_indices are not checked
struct valid_table_filter
{
  valid_table_filter(bit_mask_t **masks,
                     gdf_size_type num_columns,
                     gdf_size_type num_rows,
                     cudf::any_or_all drop_if) 
  : drop_if(drop_if),
    num_columns(num_columns),
    num_rows(num_rows),
    d_masks(masks) {}

  __device__ inline 
  bool operator()(gdf_index_type i)
  {
    if (i < num_rows) {
      int c = 0;
      if (drop_if == cudf::ALL) {
        while (c < num_columns) {
          bit_mask_t *mask = d_masks[c++];
          if (mask == nullptr || bit_mask::is_valid(mask, i)) return true;
        }
        return false;
      }
      else { // drop_if == cudf::ANY => all columns must be valid
        while (c < num_columns) {
          bit_mask_t *mask = d_masks[c++];
          if (mask != nullptr && !bit_mask::is_valid(mask, i)) return false;
        }
        return true;
      }
    }
    return false;
  }

  cudf::any_or_all drop_if;
  gdf_size_type num_columns;
  gdf_size_type num_rows;
  bit_mask_t **d_masks;
};

bit_mask_t** get_bitmasks(cudf::table const &table,
                          std::vector<gdf_index_type> const &column_indices,
                          hipStream_t stream = 0) {
  bit_mask_t** h_masks = new bit_mask_t*[column_indices.size()];
  
  int i = 0;
  for (auto index : column_indices) {
    h_masks[i++] = reinterpret_cast<bit_mask_t*>(table.get_column(index)->valid);
  }

  size_t masks_size = sizeof(bit_mask_t*) * column_indices.size();

  bit_mask_t **d_masks = nullptr;
  RMM_ALLOC(&d_masks, masks_size, stream);
  hipMemcpyAsync(d_masks, h_masks, masks_size, hipMemcpyHostToDevice, stream);
  CHECK_STREAM(stream);

  return d_masks;
}

valid_table_filter make_valid_table_filter(cudf::table const &table,
                                           std::vector<gdf_index_type> const &column_indices,
                                           cudf::any_or_all drop_if,
                                           hipStream_t stream=0)
{
  return valid_table_filter(get_bitmasks(table, column_indices, stream),
                            column_indices.size(), table.num_rows(),
                            drop_if);
}

void destroy_valid_table_filter(valid_table_filter const& filter,
                                hipStream_t stream = 0) {
  RMM_FREE(filter.d_masks, stream);
}

}  // namespace

namespace cudf {

/*
 * Filters a column to remove null elements.
 */
gdf_column drop_nulls(gdf_column const &input) {
  if (input.valid != nullptr && input.null_count != 0)
    return detail::copy_if(input, valid_column_filter{input});
  else // no null bitmask, so just copy
    return cudf::copy(input);
}

/*
 * Filters a table to remove null elements.
 */
table drop_nulls(table const &input, 
                 std::vector<gdf_index_type> const& column_indices,
                 any_or_all drop_if) {
  if (cudf::has_nulls(input)) {
    valid_table_filter filter =
      make_valid_table_filter(input, column_indices, drop_if);
    table result = detail::copy_if(input, filter);
    destroy_valid_table_filter(filter);
    return result;
  }
  else
    return cudf::copy(input);
}

}  // namespace cudf

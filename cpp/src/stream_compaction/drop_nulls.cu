/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "copy_if.cuh"
#include <cudf/legacy/table.hpp>
#include <thrust/logical.h>
#include <thrust/count.h>
 
namespace {

using bit_mask_t = bit_mask::bit_mask_t;

// Returns false if the valid mask is false for index i in ANY/ALL columns of
// table indicated by column_indices, where ANY/ALL is the value of drop_if.
// Columns not indexed by column_indices are not checked
struct valid_table_filter
{
  __device__ inline 
  bool operator()(gdf_index_type i)
  {
    auto valid = [i](auto mask) { 
      return (mask == nullptr) || bit_mask::is_valid(mask, i);
    };

    if (valid_threshold > 0) {
      auto count =
        thrust::count_if(thrust::seq, d_masks, d_masks + num_columns, valid);
      return count >= valid_threshold;
    }
    else if (drop_if == cudf::ALL) // drop rows that have a null in all columns
      return thrust::any_of(thrust::seq, d_masks, d_masks + num_columns, valid);
    else // drop_if == cudf::ANY => drop rows that have any nulls
      return thrust::all_of(thrust::seq, d_masks, d_masks + num_columns, valid); 
  }

  static auto create(cudf::table const &table,
                     cudf::any_or_all drop_if,
                     gdf_size_type valid_threshold,
                     hipStream_t stream = 0)
  {
    std::vector<bit_mask_t*> h_masks(table.num_columns());

    std::transform(std::cbegin(table), std::cend(table), std::begin(h_masks),
      [](auto col) { return reinterpret_cast<bit_mask_t*>(col->valid); }
    );    
    
    size_t masks_size = sizeof(bit_mask_t*) * table.num_columns();

    bit_mask_t **device_masks = nullptr;
    RMM_TRY(RMM_ALLOC(&device_masks, masks_size, stream));
    CUDA_TRY(hipMemcpyAsync(device_masks, h_masks.data(), masks_size,
                            hipMemcpyHostToDevice, stream));
    CHECK_STREAM(stream);

    auto deleter = [stream](valid_table_filter* f) { f->destroy(stream); };
    std::unique_ptr<valid_table_filter, decltype(deleter)> p {
      new valid_table_filter(device_masks, table.num_columns(),
                             drop_if, valid_threshold),
      deleter
    };

    CHECK_STREAM(stream);

    return p;
  }

  __host__ void destroy(hipStream_t stream = 0) {
    RMM_FREE(d_masks, stream);
    delete this;
  }

  valid_table_filter() = delete;
  ~valid_table_filter() = default;

protected:

  valid_table_filter(bit_mask_t **masks,
                     gdf_size_type num_columns,
                     cudf::any_or_all drop_if,
                     gdf_size_type valid_threshold) 
  : drop_if(drop_if),
    valid_threshold(valid_threshold),
    num_columns(num_columns),
    d_masks(masks) {}

  cudf::any_or_all drop_if;
  gdf_size_type valid_threshold;
  gdf_size_type num_columns;
  bit_mask_t **d_masks;
};

}  // namespace

namespace cudf {

/*
 * Filters a table to remove null elements.
 */
table drop_nulls(table const &input,
                 table const &keys,
                 any_or_all drop_if,
                 gdf_size_type valid_threshold) {
  if (keys.num_columns() == 0 || keys.num_rows() == 0 ||
      not cudf::has_nulls(keys))
    return cudf::copy(input);

  CUDF_EXPECTS(keys.num_rows() <= input.num_rows(), 
               "Column size mismatch");
  
  auto filter = valid_table_filter::create(keys, drop_if, valid_threshold);

  return detail::copy_if(input, *filter.get());
}

}  // namespace cudf


#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/merge.h>
#include <algorithm>
#include <utility>
#include <vector>
#include <memory>
#include <type_traits>
#include <nvstrings/NVCategory.h>

#include <cudf/cudf.h>
#include <cudf/types.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/thrust_rmm_allocator.h>
#include <utilities/cuda_utils.hpp>

namespace {

/**
 * @brief Source table identifier to copy data from.
 */
enum class side : bool { LEFT, RIGHT };
  
using index_type = thrust::tuple<side, cudf::size_type>; // `thrust::get<0>` indicates left/right side, `thrust::get<1>` indicates the row index

  /*
rmm::device_vector<index_type>
generate_merged_indices(device_table const& left_table,
                        device_table const& right_table,
                        rmm::device_vector<int8_t> const& asc_desc,
                        bool nulls_are_smallest,
                        hipStream_t stream) {

    const cudf::size_type left_size  = left_table.num_rows();
    const cudf::size_type right_size = right_table.num_rows();
    const cudf::size_type total_size = left_size + right_size;

    thrust::constant_iterator<side> left_side(side::LEFT);
    thrust::constant_iterator<side> right_side(side::RIGHT);

    auto left_indices = thrust::make_counting_iterator(static_cast<cudf::size_type>(0));
    auto right_indices = thrust::make_counting_iterator(static_cast<cudf::size_type>(0));

    auto left_begin_zip_iterator = thrust::make_zip_iterator(thrust::make_tuple(left_side, left_indices));
    auto right_begin_zip_iterator = thrust::make_zip_iterator(thrust::make_tuple(right_side, right_indices));

    auto left_end_zip_iterator = thrust::make_zip_iterator(thrust::make_tuple(left_side + left_size, left_indices + left_size));
    auto right_end_zip_iterator = thrust::make_zip_iterator(thrust::make_tuple(right_side + right_size, right_indices + right_size));

    rmm::device_vector<index_type> merged_indices(total_size);
    bool nullable = left_table.has_nulls() || right_table.has_nulls();
    auto exec_pol = rmm::exec_policy(stream);
    if (nullable){
        auto ineq_op = row_inequality_comparator<true>(right_table, left_table, nulls_are_smallest, asc_desc.data().get()); 
        thrust::merge(exec_pol->on(stream),
                    left_begin_zip_iterator,
                    left_end_zip_iterator,
                    right_begin_zip_iterator,
                    right_end_zip_iterator,
                    merged_indices.begin(),
                    [=] __device__ (thrust::tuple<side, cudf::size_type> const & right_tuple,
                                    thrust::tuple<side, cudf::size_type> const & left_tuple) {
                        return ineq_op(thrust::get<1>(right_tuple), thrust::get<1>(left_tuple));
                    });			        
    } else {
        auto ineq_op = row_inequality_comparator<false>(right_table, left_table, nulls_are_smallest, asc_desc.data().get()); 
        thrust::merge(exec_pol->on(stream),
                    left_begin_zip_iterator,
                    left_end_zip_iterator,
                    right_begin_zip_iterator,
                    right_end_zip_iterator,
                    merged_indices.begin(),
                    [=] __device__ (thrust::tuple<side, cudf::size_type> const & right_tuple,
                                    thrust::tuple<side, cudf::size_type> const & left_tuple) {
                        return ineq_op(thrust::get<1>(right_tuple), thrust::get<1>(left_tuple));
                    });					        
    }

    CHECK_STREAM(stream);

    return merged_indices;
}
  */

} // namespace

namespace cudf {
namespace experimental { 
namespace detail {

struct ColumnMerger
{
  ColumnMerger(cudf::size_type const& key_col,
               cudf::order const& asc_desc,
               cudf::null_order const& null_p)
  {
  }
  
  // type_dispatcher() _can_ dispatch host functors:
  //
  template<typename Element>//required: column type
  std::unique_ptr<column>
  operator()(cudf::column_view const& lcol, cudf::column_view const& rcol)
  {
    return nullptr;//for now...
  }
  //see `class element_relational_comparator` in `cpp/include/cudf/table/row_operators.cuh` as a model;
};
  

std::unique_ptr<cudf::experimental::table> merge(table_view const& left_table,
                                   table_view const& right_table,
                                   std::vector<cudf::size_type> const& key_cols,
                                   std::vector<cudf::order> const& asc_desc,
                                   std::vector<cudf::null_order> const& null_precedence) {
    auto n_cols = left_table.num_columns();
    CUDF_EXPECTS( n_cols == right_table.num_columns(), "Mismatched number of columns");
    if (left_table.num_columns() == 0) {
        return nullptr;
    }

    // TODO: replace / drop;
    //no replacement, yet:
    //{
    //proposal for a replacement: bool have_same_types(table_view const& lhs, table_view const& rhs);
    //
    //std::vector<gdf_dtype> left_table_dtypes = cudf::column_dtypes(left_table);
    //std::vector<gdf_dtype> right_table_dtypes = cudf::column_dtypes(right_table);
    //CUDF_EXPECTS(std::equal(left_table_dtypes.cbegin(), left_table_dtypes.cend(), right_table_dtypes.cbegin(), right_table_dtypes.cend()), "Mismatched column dtypes");
    //}
    
    CUDF_EXPECTS(key_cols.size() > 0, "Empty key_cols");
    CUDF_EXPECTS(key_cols.size() <= static_cast<size_t>(left_table.num_columns()), "Too many values in key_cols");
    CUDF_EXPECTS(asc_desc.size() > 0, "Empty asc_desc");
    CUDF_EXPECTS(asc_desc.size() <= static_cast<size_t>(left_table.num_columns()), "Too many values in asc_desc");
    CUDF_EXPECTS(key_cols.size() == asc_desc.size(), "Mismatched size between key_cols and asc_desc");


    using column_rep_t = cudf::column; // or column_view?
    using col_ptr_t = typename std::unique_ptr<column_rep_t>;

    std::vector<col_ptr_t> v_merged_cols;
    v_merged_cols.reserve(n_cols);

    static_assert(std::is_same<decltype(v_merged_cols), std::vector<std::unique_ptr<cudf::column>> >::value, "ERROR: unexpected type.");

    for(auto i=0;i<n_cols;++i)
      {
        const auto& left_col = left_table.column(i);
        const auto& right_col= right_table.column(i);

        //not clear yet what must be done for STRING:
        //
        //if( left_col.type().id() != STRING )
        //  continue;//?

        ColumnMerger merger{key_cols[i],
                            asc_desc[i],
                            null_precedence[i]};

        col_ptr_t merged = cudf::experimental::type_dispatcher(left_col.type(),
                                                               merger,
                                                               left_col,
                                                               right_col);
        v_merged_cols.emplace_back(std::move(merged));
      }
    //cudf::table tbl{std::move(v_merged_cols)};//nope...
    
    return std::unique_ptr<cudf::experimental::table>{new cudf::experimental::table(std::move(v_merged_cols))};
}

}  // namespace detail

std::unique_ptr<cudf::experimental::table> merge(table_view const& left_table,
                                   table_view const& right_table,
                                   std::vector<cudf::size_type> const& key_cols,
                                   std::vector<cudf::order> const& asc_desc,
                                   std::vector<cudf::null_order> const& null_precedence){
  return detail::merge(left_table, right_table, key_cols, asc_desc, null_precedence);
}

}  // namespace experimental
}  // namespace cudf

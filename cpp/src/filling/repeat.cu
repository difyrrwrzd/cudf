/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/error_utils.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <cudf/copying.hpp>

#include <cudf/cudf.h>
#include <cudf/types.h>
#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>

template <typename T>
void print(rmm::device_vector<T> const& d_vec, std::string label = "") {
  thrust::host_vector<T> h_vec = d_vec;
  printf("%s \t", label.c_str());
  for (auto &&i : h_vec)  std::cout << i << " ";
  printf("\n");
}

struct printer
{
  template <typename T>
  void operator()(gdf_column const& col, std::string label = "") {
    auto col_data = reinterpret_cast<T*>(col.data);
    auto d_vec = rmm::device_vector<T>(col_data, col_data+col.size);
    print(d_vec, label);
  }
};

void print(gdf_column const& col, std::string label = "") {
  cudf::type_dispatcher(col.dtype, printer{}, col, label);
}

namespace cudf {

namespace detail {

gdf_column repeat(const gdf_column &in, const gdf_column& count, hipStream_t stream = 0) {
  CUDF_EXPECTS(count.dtype == gdf_dtype_of<gdf_size_type>(),
    "Count column should be of index type");
  CUDF_EXPECTS(count.null_count == 0, "Couns cannot be null");
  
  auto exec_policy = rmm::exec_policy(stream)->on(stream);
  rmm::device_vector<gdf_size_type> offset(count.size);
  auto count_data = static_cast <gdf_size_type*> (count.data);
  
  thrust::inclusive_scan(exec_policy, count_data, count_data + count.size, offset.begin());
  // print(offset);

  gdf_size_type output_size = offset[count.size - 1];

  rmm::device_vector<gdf_size_type> indices(output_size);
  thrust::upper_bound(exec_policy,
                      offset.begin(), offset.end(),
                      thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(output_size),
                      indices.begin());
  // print(indices);

  // Allocate `output` with output_size elements
  gdf_column output = cudf::allocate_like(in, output_size);

  cudf::table in_table{const_cast<gdf_column*>(&in)};
  cudf::table out_table{&output};
  cudf::gather(&in_table, indices.data().get(), &out_table);
  // print(output);

  return output;
}

} // namespace detail


gdf_column repeat(const gdf_column &in, const gdf_column& count) {
  return detail::repeat(in, count);
}

} // namespace cudf

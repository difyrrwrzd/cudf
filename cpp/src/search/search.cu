/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iterator/iterator.cuh>
#include <utilities/wrapper_types.hpp>
#include <utilities/column_utils.hpp>
#include <cudf/search.hpp>
#include <cudf/copying.hpp>
#include <rmm/thrust_rmm_allocator.h>

#include <thrust/binary_search.h>

#include <iostream>

namespace cudf {

namespace {

struct search_functor {
private:
  template <typename T>
  static constexpr bool is_supported() {
    // TODO: allow for all types which can be compared and have std::numeric_limits defined for
    return std::is_arithmetic<T>::value;
  }

public:
  template <typename T,
            typename std::enable_if_t<is_supported<T>()>* = nullptr>
  void operator()(gdf_column const& column,
                  gdf_column const& values,
                  bool find_first,
                  hipStream_t stream,
                  gdf_column& result)
  {
    // TODO: handle nulls
    // if ( is_nullable(column) ) {
    //   auto it_dev = cudf::make_iterator<true, T>(column,
    //     std::numeric_limits<T>::lowest());
    // }
    // else {

      auto it_col = cudf::make_iterator<false, T>(column);
      auto it_val = cudf::make_iterator<false, T>(values);
      if (find_first)
      {
        thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                            it_col, it_col + column.size,
                            it_val, it_val + values.size,
                            static_cast<gdf_index_type*>(result.data));
      }
      
      // result.data = thrust::raw_pointer_cast(&result1[0]);

    // }

  }

  template <typename T,
            typename std::enable_if_t<!is_supported<T>()>* = nullptr>
  void operator()(gdf_column const& column,
                  gdf_column const& values,
                  bool find_first,
                  hipStream_t stream,
                  gdf_column& result)
  {
    CUDF_FAIL("Unsupported datatype for search_sorted");
  }

};

} // namespace

namespace detail {

gdf_column search_sorted(gdf_column const& column,
                         gdf_column const& values,
                         bool find_first,
                         hipStream_t stream = 0)
{
  validate(column);

  // Allocate result column
  gdf_column result_like{};
  result_like.dtype = GDF_INT32;
  result_like.size = values.size;
  result_like.data = values.data;
  // TODO: let result have nulls? this could be used for records not found
  auto result = allocate_like(result_like);

  // TODO: find out whether nulls_before_values
  // Just need to check if nullmask exists and then first value is null or not
  // before that, maybe check if sorted or not
  
  type_dispatcher(column.dtype,
                  search_functor{},
                  column, values, find_first, stream, result);

  return result;
}

} // namespace detail

gdf_column search_sorted(gdf_column const& column,
                         gdf_column const& values,
                         bool find_first)
{
  return detail::search_sorted(column, values, find_first);
}

} // namespace cudf

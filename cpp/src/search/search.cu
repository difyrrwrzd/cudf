#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/search.hpp>
#include <cudf/legacy/copying.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/logical.h>

namespace cudf {
namespace experimental {

namespace {

template <typename DataIterator, typename ValuesIterator,
          typename OutputIterator, typename Comparator>
void launch_search(DataIterator it_data,
                   ValuesIterator it_vals,
                   size_type data_size,
                   size_type values_size,
                   OutputIterator it_output,
                   Comparator comp,
                   bool find_first,
                   hipStream_t stream)
{
  if (find_first) {
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                        it_data, it_data + data_size,
                        it_vals, it_vals + values_size,
                        it_output, comp);
  }
  else {
    thrust::upper_bound(rmm::exec_policy(stream)->on(stream),
                        it_data, it_data + data_size,
                        it_vals, it_vals + values_size,
                        it_output, comp);
  }
}

} // namespace

namespace detail {

std::unique_ptr<column> search_ordered(table_view const& t,
                                       table_view const& values,
                                       bool find_first,
                                       std::vector<order> const& column_order,
                                       null_order null_precedence,
                                       hipStream_t stream,
                                       rmm::mr::device_memory_resource *mr)
{
  // Allocate result column
  std::unique_ptr<column> result = make_numeric_column(data_type{experimental::type_to_id<size_type>()}, values.num_rows(),
                                                       mask_state::UNALLOCATED, stream, mr);

  mutable_column_view result_view = result.get()->mutable_view();

  // Handle empty inputs
  if (t.num_rows() == 0) {
    CUDA_TRY(hipMemset(result_view.data<size_type>(), 0, values.num_rows() * sizeof(size_type)));
    return result;
  }

  if (not column_order.empty()) {
    CUDF_EXPECTS(
        static_cast<std::size_t>(t.num_columns()) == column_order.size(),
        "Mismatch between number of columns and column order.");
  }

  auto d_t      = table_device_view::create(t, stream);
  auto d_values = table_device_view::create(values, stream);
  auto count_it = thrust::make_counting_iterator<size_type>(0);

  //  Need an order*
  rmm::device_vector<order> d_column_order(column_order.begin(), column_order.end());

  if (has_nulls(t)) {
    auto ineq_op = (find_first)
                 ? row_lexicographic_comparator<true>(*d_t, *d_values, null_precedence, d_column_order.data().get())
                 : row_lexicographic_comparator<true>(*d_values, *d_t, null_precedence, d_column_order.data().get());

    launch_search(count_it, count_it, t.num_rows(), values.num_rows(),
                  result_view.data<size_type>(), ineq_op, find_first, stream);
  } else {
    auto ineq_op = (find_first)
                 ? row_lexicographic_comparator<false>(*d_t, *d_values, null_precedence, d_column_order.data().get())
                 : row_lexicographic_comparator<false>(*d_values, *d_t, null_precedence, d_column_order.data().get());

    launch_search(count_it, count_it, t.num_rows(), values.num_rows(),
                  result_view.data<size_type>(), ineq_op, find_first, stream);
  }

  return result;
}

template <bool nullable = true>
struct compare_with_value{
  compare_with_value(table_device_view t, table_device_view val, bool nulls_are_equal = true)
    : compare(t, val, nulls_are_equal) {}

  __device__ bool operator()(size_type i){
    return compare(i, 0);
  }
  row_equality_comparator<nullable> compare;
};

bool contains(column_view const& col,
              scalar const& value,
              hipStream_t stream,
              rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(col.type() == value.type(), "DTYPE mismatch");

  if (col.size() == 0) {
    return false;
  }

  if (not value.is_valid()) {
    return col.has_nulls();
  }

  std::unique_ptr<column> scalar_as_column = make_numeric_column(col.type(), 1, mask_state::UNALLOCATED, stream, mr);
  cudf::experimental::fill(scalar_as_column, size_type{0}, size_type{1}, value, mr);

  auto d_t = cudf::table_device_view::create(cudf::table_view{{col}}, stream);
  auto d_value = cudf::table_device_view::create(cudf::table_view{{*scalar_as_column}}, stream);

  auto data_it = thrust::make_counting_iterator<size_type>(0);

  if (col.has_nulls()) {
    auto eq_op = compare_with_value<true>(*d_t, *d_value, true);

    return thrust::any_of(rmm::exec_policy(stream)->on(stream),
                          data_it, data_it + col.size(),
                          eq_op);
  } else {
    auto eq_op = compare_with_value<false>(*d_t, *d_value, true);

    return thrust::any_of(rmm::exec_policy(stream)->on(stream),
                          data_it, data_it + col.size(),
                          eq_op);
  }
}
} // namespace detail

std::unique_ptr<column> lower_bound(table_view const& t,
                                    table_view const& values,
                                    std::vector<order> const& column_order,
                                    null_order null_precedence,
                                    rmm::mr::device_memory_resource *mr)
{
  hipStream_t stream = 0;
  return detail::search_ordered(t, values, true, column_order, null_precedence, stream, mr);
}

std::unique_ptr<column> upper_bound(table_view const& t,
                                    table_view const& values,
                                    std::vector<order> const& column_order,
                                    null_order null_precedence,
                                    rmm::mr::device_memory_resource *mr)
{
  hipStream_t stream = 0;
  return detail::search_ordered(t, values, false, column_order, null_precedence, stream, mr);
}

bool contains(column const& col, scalar const& value, rmm::mr::device_memory_resource *mr)
{
  hipStream_t stream = 0;
  return detail::contains(col, value, stream, mr);
}

} // namespace exp
} // namespace cudf

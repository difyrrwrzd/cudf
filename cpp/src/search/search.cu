#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iterator/iterator.cuh>
#include <table/device_table.cuh>
#include <table/device_table_row_operators.cuh>
#include <utilities/wrapper_types.hpp>
#include <utilities/column_utils.hpp>

#include <cudf/search.hpp>
#include <cudf/copying.hpp>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/binary_search.h>


namespace cudf {

namespace {

struct search_functor {
private:
  template <typename T>
  static constexpr bool is_supported() {
    // TODO: allow for all types which can be compared and have std::numeric_limits defined for
    return std::is_arithmetic<T>::value;
  }

public:
  template <typename T,
            typename std::enable_if_t<is_supported<T>()>* = nullptr>
  void operator()(gdf_column const& column,
                  gdf_column const& values,
                  bool find_first,
                  bool nulls_as_largest,
                  hipStream_t stream,
                  gdf_column& result)
  {
    if ( is_nullable(column) ) {
      T null_substitute = (nulls_as_largest) 
                        ? std::numeric_limits<T>::max()
                        : std::numeric_limits<T>::lowest();

      auto it_col = cudf::make_iterator<true, T>(column, null_substitute);
      auto it_val = cudf::make_iterator<true, T>(values, null_substitute);

      if (find_first) {
        thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                            it_col, it_col + column.size,
                            it_val, it_val + values.size,
                            static_cast<gdf_index_type*>(result.data));
      }
      else {
        thrust::upper_bound(rmm::exec_policy(stream)->on(stream),
                            it_col, it_col + column.size,
                            it_val, it_val + values.size,
                            static_cast<gdf_index_type*>(result.data));
      }
    }
    else {
      auto it_col = cudf::make_iterator<false, T>(column);
      auto it_val = cudf::make_iterator<false, T>(values);

      if (find_first) {
        thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                            it_col, it_col + column.size,
                            it_val, it_val + values.size,
                            static_cast<gdf_index_type*>(result.data));
      }
      else {
        thrust::upper_bound(rmm::exec_policy(stream)->on(stream),
                            it_col, it_col + column.size,
                            it_val, it_val + values.size,
                            static_cast<gdf_index_type*>(result.data));
      }
    }

  }

  template <typename T,
            typename... Args,
            typename std::enable_if_t<!is_supported<T>()>* = nullptr>
  void operator()(Args&&... args)
  {
    CUDF_FAIL("Unsupported datatype for search_ordered");
  }

};

} // namespace

namespace detail {

gdf_column search_ordered(gdf_column const& column,
                          gdf_column const& values,
                          bool find_first,
                          bool nulls_as_largest,
                          hipStream_t stream = 0)
{
  // TODO: allow empty input
  validate(column);
  validate(values);

  // Allocate result column
  gdf_column result_like{};
  result_like.dtype = GDF_INT32;
  result_like.size = values.size;
  result_like.data = values.data;
  // TODO: let result have nulls? this could be used for records not found
  auto result = allocate_like(result_like);

  type_dispatcher(column.dtype,
                  search_functor{},
                  column, values, find_first, nulls_as_largest, stream, result);

  return result;
}

gdf_column search_ordered(table const& t,
                          table const& values,
                          bool find_first,
                          bool nulls_as_largest,
                          hipStream_t stream = 0)
{
  // TODO: validate input table and values
  // TODO: allow empty input

  // Allocate result column
  gdf_column result_like{};
  result_like.dtype = GDF_INT32;
  result_like.size = values.num_rows();
  result_like.data = values.get_column(0)->data;
  // TODO: let result have nulls? this could be used for records not found
  auto result = allocate_like(result_like);

  auto d_t      = device_table::create(t, stream);
  auto d_values = device_table::create(values, stream);
  if (find_first) {
    auto ineq_op  = row_inequality_comparator<false>(*d_t, *d_values, !nulls_as_largest);
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                        thrust::make_counting_iterator(0), 
                        thrust::make_counting_iterator(t.num_rows()),
                        thrust::make_counting_iterator(0),
                        thrust::make_counting_iterator(values.num_rows()),
                        static_cast<gdf_index_type*>(result.data),
                        ineq_op);
  }
  else {
    auto ineq_op  = row_inequality_comparator<false>(*d_values, *d_t, !nulls_as_largest);
    thrust::upper_bound(rmm::exec_policy(stream)->on(stream),
                        thrust::make_counting_iterator(0), 
                        thrust::make_counting_iterator(t.num_rows()),
                        thrust::make_counting_iterator(0),
                        thrust::make_counting_iterator(values.num_rows()),
                        static_cast<gdf_index_type*>(result.data),
                        ineq_op);
  }

  return result;
}

} // namespace detail

gdf_column search_ordered(gdf_column const& column,
                          gdf_column const& values,
                          bool find_first,
                          bool nulls_as_largest)
{
  return detail::search_ordered(column, values, find_first, nulls_as_largest);
}

gdf_column search_ordered(table const& t,
                          table const& values,
                          bool find_first,
                          bool nulls_as_largest)
{
  return detail::search_ordered(t, values, find_first, nulls_as_largest);
}

} // namespace cudf

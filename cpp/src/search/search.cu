/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iterator/iterator.cuh>
#include <table/device_table.cuh>
#include <table/device_table_row_operators.cuh>
#include <utilities/wrapper_types.hpp>
#include <utilities/column_utils.hpp>

#include <cudf/search.hpp>
#include <cudf/copying.hpp>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>

namespace cudf {

namespace {

template <typename DataIterator, typename ValuesIterator, typename Comparator>
void launch_search(DataIterator it_data,
                    ValuesIterator it_vals,
                    gdf_size_type data_size,
                    gdf_size_type values_size,
                    void* output,
                    Comparator comp,
                    bool find_first,
                    hipStream_t stream)
{
  if (find_first) {
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                        it_data, it_data + data_size,
                        it_vals, it_vals + values_size,
                        static_cast<gdf_index_type*>(output),
                        comp);
  }
  else {
    thrust::upper_bound(rmm::exec_policy(stream)->on(stream),
                        it_data, it_data + data_size,
                        it_vals, it_vals + values_size,
                        static_cast<gdf_index_type*>(output),
                        comp);
  }
}

} // namespace

namespace detail {

gdf_column search_ordered(table const& t,
                          table const& values,
                          bool find_first,
                          std::vector<bool> const& desc_flags,
                          bool nulls_as_largest,
                          hipStream_t stream = 0)
{
  // TODO: validate input table and values

  // Allocate result column
  gdf_column result_like{};
  result_like.dtype = GDF_INT32;
  result_like.size = values.num_rows();
  result_like.data = values.get_column(0)->data;
  // TODO: let result have nulls? this could be used for records not found
  auto result = allocate_like(result_like);

  // Handle empty inputs
  if (t.num_rows() == 0) {
    CUDA_TRY(hipMemset(result.data, 0, values.num_rows()));
    if (is_nullable(result)) {
      CUDA_TRY(hipMemset(result.valid, 0, values.num_rows()));
    }
  }

  auto d_t      = device_table::create(t, stream);
  auto d_values = device_table::create(values, stream);
  auto count_it = thrust::make_counting_iterator(0);

  rmm::device_vector<int8_t> dv_desc_flags(desc_flags);
  auto d_desc_flags = dv_desc_flags.data().get();
  
  if ( has_nulls(t) ) {
    auto ineq_op = (find_first)
                 ? row_inequality_comparator<true>(*d_t, *d_values, !nulls_as_largest, d_desc_flags)
                 : row_inequality_comparator<true>(*d_values, *d_t, !nulls_as_largest, d_desc_flags);

    launch_search(count_it, count_it, t.num_rows(), values.num_rows(), result.data,
                  ineq_op, find_first, stream);
  }
  else {
    auto ineq_op = (find_first)
                 ? row_inequality_comparator<false>(*d_t, *d_values, !nulls_as_largest, d_desc_flags)
                 : row_inequality_comparator<false>(*d_values, *d_t, !nulls_as_largest, d_desc_flags);

    launch_search(count_it, count_it, t.num_rows(), values.num_rows(), result.data,
                  ineq_op, find_first, stream);
  }

  return result;
}

gdf_column search_ordered(gdf_column const& column,
                          gdf_column const& values,
                          bool find_first,
                          bool descending,
                          bool nulls_as_largest,
                          hipStream_t stream = 0)
{
  const table t{const_cast<gdf_column*>(&column)};
  const table val{const_cast<gdf_column*>(&values)};
  std::vector<bool> desc_flags{descending};

  return search_ordered(t, val, find_first, desc_flags, nulls_as_largest, stream);
}

} // namespace detail

gdf_column lower_bound(gdf_column const& column,
                       gdf_column const& values,
                       bool ascending,
                       bool nulls_as_largest)
{
  return detail::search_ordered(column, values, true, ascending, nulls_as_largest);
}

gdf_column upper_bound(gdf_column const& column,
                       gdf_column const& values,
                       bool ascending,
                       bool nulls_as_largest)
{
  return detail::search_ordered(column, values, false, ascending, nulls_as_largest);
}

gdf_column lower_bound(table const& t,
                       table const& values,
                       std::vector<bool> const& desc_flags,
                       bool nulls_as_largest)
{
  return detail::search_ordered(t, values, true, desc_flags, nulls_as_largest);
}

gdf_column upper_bound(table const& t,
                       table const& values,
                       std::vector<bool> const& desc_flags,
                       bool nulls_as_largest)
{
  return detail::search_ordered(t, values, false, desc_flags, nulls_as_largest);
}

} // namespace cudf

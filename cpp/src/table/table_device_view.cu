/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <utilities/error_utils.hpp>

#include <rmm/rmm.h>

#include <algorithm>

namespace cudf {

namespace detail {
template <typename ColumnDeviceView>
void table_device_view_base<ColumnDeviceView>::destroy() {
  RMM_TRY(RMM_FREE(_columns, _stream));
  delete this;
}
}  // namespace detail

auto table_device_view::create(table_view source_view, hipStream_t stream) {
  size_type total_descendants =
      std::accumulate(source_view.begin(), source_view.end(), 0,
                      [](size_type init, column_view col) {
                        return init + count_descendants(col);
                      });
  CUDF_EXPECTS(0 == total_descendants,
               "Columns with descendants are not yet supported.");

  auto deleter = [](table_device_view* t) { t->destroy(); };

  return std::unique_ptr<table_device_view, decltype(deleter)>{
      new table_device_view(source_view, stream), deleter};
}

table_device_view::table_device_view(table_view source_view,
                                     hipStream_t stream)
    : detail::table_device_view_base<column_device_view>(
          source_view.num_rows(), source_view.num_columns(), stream) {
  auto views_size_bytes =
      source_view.num_columns() * sizeof(column_device_view);
  RMM_TRY(RMM_ALLOC(_columns, views_size_bytes, stream));
  CUDA_TRY(hipMemcpy(_columns, source_view.begin(), views_size_bytes));
}

auto mutable_table_device_view::create(mutable_table_view source_view,
                                       hipStream_t stream) {
  size_type total_descendants =
      std::accumulate(source_view.begin(), source_view.end(), 0,
                      [](size_type init, column_view col) {
                        return init + count_descendants(col);
                      });
  CUDF_EXPECTS(0 == total_descendants,
               "Columns with descendants are not yet supported.");

  auto deleter = [](mutable_table_device_view* t) { t->destroy(); };

  return std::unique_ptr<mutable_table_device_view, decltype(deleter)>{
      new mutable_table_device_view(source_view, stream), deleter};
}

mutable_table_device_view::mutable_table_device_view(
    mutable_table_view source_view, hipStream_t stream)
    : detail::table_device_view_base<mutable_column_device_view>(
          source_view.num_rows(), source_view.num_columns(), stream) {
  auto views_size_bytes =
      source_view.num_columns() * sizeof(mutable_column_device_view);
  RMM_TRY(RMM_ALLOC(_columns, views_size_bytes, stream));
  CUDA_TRY(hipMemcpy(_columns, source_view.begin(), views_size_bytes));
}

}  // namespace cudf

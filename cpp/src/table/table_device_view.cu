/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <utilities/error_utils.hpp>

#include <rmm/rmm.h>

#include <vector>
#include <algorithm>
#include <numeric>

namespace cudf {
namespace detail {

template <typename ColumnDeviceView, typename HostTableView>
void table_device_view_base<ColumnDeviceView, HostTableView>::destroy() {
  RMM_TRY(RMM_FREE(_columns, _stream));
  delete this;
}

template <typename ColumnDeviceView, typename HostTableView>
table_device_view_base<ColumnDeviceView, HostTableView>::table_device_view_base(
    HostTableView source_view, hipStream_t stream)
    : _num_rows{source_view.num_rows()},
      _num_columns{source_view.num_columns()},
      _stream{stream} {
  if (source_view.num_columns() > 0) {
    //
    size_type views_size_bytes =
        std::accumulate(source_view.begin(), source_view.end(), 0,
            [](size_type init, column_view col) {
                return init + ColumnDeviceView::extent(col);
            });
    
    //CUDA_TRY(hipMemcpyAsync(_columns, &(*source_view.begin()),
    //                         views_size_bytes, hipMemcpyDefault, stream));

    std::vector<int8_t> h_buffer(views_size_bytes);
    ColumnDeviceView* h_column = reinterpret_cast<ColumnDeviceView*>(h_buffer.data());
    int8_t* h_end = (int8_t*)(h_column + _num_columns);
    RMM_TRY(RMM_ALLOC(&_columns, views_size_bytes, stream));
    ColumnDeviceView* d_column = _columns;
    int8_t* d_end = (int8_t*)(d_column + _num_columns);
    for( size_type idx=0; idx < _num_columns; ++idx )
    {
      auto col = source_view.column(idx);
      new(h_column) ColumnDeviceView(col,(ptrdiff_t)h_end,(ptrdiff_t)d_end);
      h_column++;
      h_end += (ColumnDeviceView::extent(col));
      d_end += (ColumnDeviceView::extent(col));
    }
    
    CUDA_TRY(hipMemcpyAsync(_columns, h_buffer.data(),
                             views_size_bytes, hipMemcpyDefault, stream));
  }
}

// Explicit instantiation for a device table of immutable views
template class table_device_view_base<column_device_view, table_view>;

// Explicit instantiation for a device table of mutable views
template class table_device_view_base<mutable_column_device_view,
                                      mutable_table_view>;

}  // namespace detail
}  // namespace cudf

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */



#include <cudf/types.hpp>
#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <cudf/copying.hpp>
#include <utilities/column_utils.hpp>
#include <utilities/error_utils.hpp>
#include <utilities/type_dispatcher.hpp>
#include <utilities/nvtx/nvtx_utils.h>
#include <string/nvcategory_util.hpp>
#include <nvstrings/NVCategory.h>
#include <copying/gather.hpp>
#include "joining.h"

#include <limits>
#include <set>
#include <vector>

// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
using output_index_type = gdf_index_type;
constexpr output_index_type MAX_JOIN_SIZE{std::numeric_limits<output_index_type>::max()};

namespace cudf {
/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a repeated value
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @param[in] value The value to be filled into the buffer
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocValueBuffer(data_type ** buffer,
                           const gdf_size_type buffer_length,
                           const data_type value) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::fill(thrust::device, *buffer, *buffer + buffer_length, value);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a sequence
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocSequenceBuffer(data_type ** buffer,
                              const gdf_size_type buffer_length) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::sequence(thrust::device, *buffer, *buffer + buffer_length);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Trivially computes full join of two tables if one of the tables
 * are empty
 * 
 * @param[in] left_size The size of the left table
 * @param[in] right_size The size of the right table
 * @param[in] rightcol The right set of columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 */
/* ----------------------------------------------------------------------------*/
void trivial_full_join(
        const gdf_size_type left_size,
        const gdf_size_type right_size,
        gdf_column *left_result,
        gdf_column *right_result) {
    // Deduce the type of the output gdf_columns
    gdf_dtype dtype;
    switch(sizeof(output_index_type))
    {
      case 1 : dtype = GDF_INT8;  break;
      case 2 : dtype = GDF_INT16; break;
      case 4 : dtype = GDF_INT32; break;
      case 8 : dtype = GDF_INT64; break;
    }

    output_index_type *l_ptr{nullptr};
    output_index_type *r_ptr{nullptr};
    gdf_size_type result_size{0};
    CUDF_EXPECTS((left_size != 0) || (right_size != 0), "Dataset is empty");
    if (left_size == 0) {
        allocValueBuffer(&l_ptr, right_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&r_ptr, right_size);
        result_size = right_size;
    } else if (right_size == 0) {
        allocValueBuffer(&r_ptr, left_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&l_ptr, left_size);
        result_size = left_size;
    }
    gdf_column_view( left_result, l_ptr, nullptr, result_size, dtype);
    gdf_column_view(right_result, r_ptr, nullptr, result_size, dtype);
    CUDA_CHECK_LAST();
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Computes the join operation between two sets of columns
 * 
 * @param[in] left_table  cudf table of left columns to join
 * @param[in] right_table cudf table of right  columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 * @param[in] join_context A structure that determines various run parameters, such as
 *                         whether to perform a hash or sort based join
 * @tparam join_type The type of join to be performed
 * 
 * @returns void
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
void join_call(cudf::table const& left_table, cudf::table const& right_table,
                     gdf_column *left_result, gdf_column *right_result,
                     gdf_context *join_context)
{
  CUDF_EXPECTS( 0 != left_table.num_columns(), "Left Dataset is empty");
  CUDF_EXPECTS( 0 != right_table.num_columns(), "Right Dataset is empty");
  CUDF_EXPECTS( nullptr != join_context, "Invalid join context");

  const auto left_col_size = left_table.num_rows();
  const auto right_col_size = right_table.num_rows();
  
  CUDF_EXPECTS( left_col_size < MAX_JOIN_SIZE, "left column size is too big");
  CUDF_EXPECTS( right_col_size < MAX_JOIN_SIZE, "right column size is too big");

  // If both frames are empty, return immediately
  if((0 == left_col_size ) && (0 == right_col_size)) {
    return;
  }

  // If left join and the left table is empty, return immediately
  if( (JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)){
    return;
  }

  // If Inner Join and either table is empty, return immediately
  if( (JoinType::INNER_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return;
  }

  // If Full Join and either table is empty, compute trivial full join
  if( (JoinType::FULL_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return trivial_full_join(left_col_size, right_col_size, left_result, right_result);
  }

  // check that the columns data are not null, have matching types, 
  // and the same number of rows
  for (int i = 0; i < left_table.num_columns(); i++) {
    CUDF_EXPECTS (!((left_col_size > 0) && (nullptr == left_table.get_column(i)->data)), "Null column data in left set");
    CUDF_EXPECTS (!((right_col_size > 0) && (nullptr == right_table.get_column(i)->data)), "Null column data in right set");
    CUDF_EXPECTS (right_table.get_column(i)->dtype == left_table.get_column(i)->dtype, "DTYPE mismatch");

    // Ensure GDF_TIMESTAMP columns have the same resolution
    if (GDF_TIMESTAMP == right_table.get_column(i)->dtype) {
      CUDF_EXPECTS(
          right_table.get_column(i)->dtype_info.time_unit == left_table.get_column(i)->dtype_info.time_unit,
          "Timestamp resolution mismatch");
    }
  }

  gdf_method join_method = join_context->flag_method; 
  gdf_error gdf_error_code{GDF_SUCCESS};

  PUSH_RANGE("LIBGDF_JOIN", JOIN_COLOR);

  switch(join_method)
  {
    case GDF_HASH:
      {
        gdf_error_code = join_hash<join_type, output_index_type>(left_table, right_table, left_result, right_result);
        CUDF_EXPECTS(gdf_error_code == GDF_SUCCESS, "GDF Error");
        break;
      }
    case GDF_SORT:
      {
        // Sort based joins only support single column joins
        if(1 == left_table.num_columns())
        {
          gdf_error_code =  sort_join<join_type, output_index_type>(const_cast <gdf_column*> (left_table.get_column(0)), 
                            const_cast <gdf_column*> (right_table.get_column(0)), left_result, right_result);
          CUDF_EXPECTS(gdf_error_code == GDF_SUCCESS, "GDF Error");
        }
        else
        {
          CUDF_EXPECTS(false, "Too many columns to join");
        }

        break;
      }
    default:
      CUDF_EXPECTS(false, "Unsupported Method");
  }

  POP_RANGE();
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Computes the resulting merged table
 * 
 * @param[in] ljoincols  Table contatining left join columns 
 * @param[in] rjoincols  Table contatining right join columns
 * @param[in] left_cols  Table containing updated left columns
 * @param[in] right_cols  Table contating updated right columns
 * @param[in] join_cols Table containing left and right columns having join indices respectively
 * @param[in] merging_join_cols Table containing left and right columns having join indices with same name respectively
 * @param[in] left_indices 
 * @param[in] right_indicess  Table contatining right join columns
 * @tparam join_type The type of join to be performed
 * 
 * @returns void
 */
/* ----------------------------------------------------------------------------*/

template <JoinType join_type, typename index_type>
std::pair<cudf::table, cudf::table> construct_join_output_df(
        cudf::table const& ljoincols,
        cudf::table const& rjoincols,
        cudf::table const& left_cols, 
        cudf::table const& right_cols,
        cudf::table const& join_cols,
        cudf::table const& merging_join_cols,
        gdf_column * left_indices,
        gdf_column * right_indices) {


    PUSH_RANGE("LIBGDF_JOIN_OUTPUT", JOIN_COLOR);
    //create left and right input table with columns not joined on
    std::vector<int> l_col_ind(left_cols.num_columns());
    std::vector<int> r_col_ind(right_cols.num_columns());
    std::vector<int> left_j_cols (join_cols.num_rows());
    std::vector<int> l_common_name_join_ind (merging_join_cols.num_rows());
    std::vector<int> r_common_name_join_ind (merging_join_cols.num_rows());
    std::vector<int> l_nonjoin_ind (left_cols.num_columns() - l_common_name_join_ind.size());
    std::vector<int> r_nonjoin_ind (right_cols.num_columns() - r_common_name_join_ind.size());

   
    std::cout <<"RGSL : Created basics"<<std::endl; 

    if (join_cols.num_rows() > 0)
    {
        CUDA_TRY (hipMemcpy(left_j_cols.data(), join_cols.get_column(0)->data, 
                             sizeof(int)*join_cols.num_rows() , hipMemcpyDeviceToHost));
    }
    if (merging_join_cols.num_rows() > 0)
    {
        CUDA_TRY (hipMemcpy(l_common_name_join_ind.data(), merging_join_cols.get_column(0)->data,
                               sizeof(int)*merging_join_cols.num_rows() , hipMemcpyDeviceToHost));
        CUDA_TRY (hipMemcpy(r_common_name_join_ind.data(), merging_join_cols.get_column(1)->data,
                               sizeof(int)*merging_join_cols.num_rows() , hipMemcpyDeviceToHost));
    }
    
  std::vector <int> tmp_l_join_ind = l_common_name_join_ind;
  std::vector <int> tmp_r_join_ind = r_common_name_join_ind;
  
    std::iota(std::begin(l_col_ind), std::end(l_col_ind), 0);
    std::iota(std::begin(r_col_ind), std::end(r_col_ind), 0);
    std::sort(std::begin(tmp_l_join_ind), std::end(tmp_l_join_ind));
    std::sort(std::begin(tmp_r_join_ind), std::end(tmp_r_join_ind));

    // Gathering the indices that are not in join
     
    std::cout <<"RGSL : Checking differnece and size of join ind "<<tmp_l_join_ind.size()<<std::endl; 
    std::cout<<"RGSL : lnon join ind size "<<l_nonjoin_ind.size()<<std::endl;
 
    std::cout<<"l col ind"<<std::endl; 
    for (std::vector<int>::iterator it = l_col_ind.begin() ; it != l_col_ind.end(); ++it) {std::cout<<*it<<std::endl;}
    std::cout<<"tmp_l_join_ind"<<std::endl; 
    for (std::vector<int>::iterator it = tmp_l_join_ind.begin() ; it != tmp_l_join_ind.end(); ++it) {std::cout<<*it<<std::endl;}
 
    if (l_nonjoin_ind.size() > 0)
        std::set_difference(std::cbegin(l_col_ind), std::cend(l_col_ind),
                                 std::cbegin(tmp_l_join_ind), std::cend(tmp_l_join_ind),
                                 std::begin(l_nonjoin_ind));
    
    std::cout <<"RGSL : Checking differnece 2 and size "<<tmp_r_join_ind.size()<<std::endl; 
    std::cout<<"RGSL : rnon join ind size "<<r_nonjoin_ind.size()<<std::endl;
    if (r_nonjoin_ind.size() > 0)
        std::set_difference(std::cbegin(r_col_ind), std::cend(r_col_ind),
                                 std::cbegin(tmp_r_join_ind), std::cend(tmp_r_join_ind),
                                 std::begin(r_nonjoin_ind));

    gdf_size_type join_size = left_indices->size;
    std::vector <gdf_dtype> rdtypes;
    std::vector <gdf_dtype_extra_info> rdtype_infos;

    std::vector<gdf_column*> lnonjoincol;
    std::vector<gdf_column*> rnonjoincol;

    // Gathering all the left table columns not in join
    for (std::vector<int>::iterator it = l_nonjoin_ind.begin() ; it != l_nonjoin_ind.end(); ++it)
    {
        std::cout <<"Non join left index "<<*it<<std::endl;
        lnonjoincol.push_back(const_cast<gdf_column*>(left_cols.get_column(*it)));
    }
    // Gathering all the right table columns not in join
    for (std::vector<int>::iterator it = r_nonjoin_ind.begin() ; it != r_nonjoin_ind.end(); ++it){
        std::cout <<"Non join right index "<<*it<<std::endl;
        rnonjoincol.push_back(const_cast<gdf_column*>(right_cols.get_column(*it)));
        rdtypes.push_back(right_cols.get_column(*it)->dtype);
        rdtype_infos.push_back(right_cols.get_column(*it)->dtype_info);
    }
    std::cout<<"RGSL : rnon join col size "<<rnonjoincol.size()<<std::endl;
    std::cout<<"RGSL : rnon join ind size "<<r_nonjoin_ind.size()<<std::endl;

    cudf::table result_left(join_size, cudf::column_dtypes(left_cols), cudf::column_dtype_infos(left_cols), true);
    cudf::table result_right(join_size, rdtypes, rdtype_infos, true);
    
    std::vector<gdf_column*> result_lnonjoincol;
    std::vector<gdf_column*> result_rnonjoincol;
    std::vector<gdf_column*> result_joincol;

    // Gather the left non-join col of result
    for (std::vector<int>::iterator it = l_nonjoin_ind.begin() ; it != l_nonjoin_ind.end(); ++it)
    {
        std::cout <<"Non join left result index "<<*it<<std::endl;
        result_lnonjoincol.push_back(result_left.get_column(*it));
    }
    std::cout<<"RGSL : size of left result "<<result_lnonjoincol.size()<<std::endl;
        
    // Gather join-col of result 
    for (int i=0; i < merging_join_cols.num_rows(); ++i)
    {
        std::cout <<"join left result index "<<i<<std::endl;
        result_joincol.push_back(result_left.get_column(l_common_name_join_ind[i]));
    }
    
    // Accumalate the right non-join col of result
    for (int i=0 ; i < result_right.num_columns(); ++i)
    {
        std::cout <<"right join left result index "<<i<<std::endl;
        result_rnonjoincol.push_back(result_right.get_column(i));
    }
    std::cout<<"RGSL : Created result "<<result_rnonjoincol.size()<<std::endl;
 
    bool const check_bounds{ join_type != JoinType::INNER_JOIN };

    // Construct the left columns
    if (0 != lnonjoincol.size()) {
      cudf::table left_source_table(lnonjoincol);
      cudf::table left_destination_table(result_lnonjoincol);

      cudf::detail::gather(&left_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &left_destination_table, check_bounds);

      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(left_source_table,left_destination_table);
      CHECK_STREAM(0);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table throwing a GDF error");
    }
    
    std::cout<<"RGSL : Construct right column"<<std::endl;
    // Construct the right columns
    if (0 != rnonjoincol.size()) {
       std::cout<<"RGSL : Before src"<<std::endl;
      cudf::table right_source_table(rnonjoincol);
       std::cout<<"RGSL : Before dst"<<std::endl;
      cudf::table right_destination_table(result_rnonjoincol);
      std::cout<<"RGSL : Before gather"<<std::endl;

      cudf::detail::gather(&right_source_table,
                           static_cast<index_type const *>(right_indices->data),
                           &right_destination_table, check_bounds);
      std::cout<<"RGSL : After gatherc"<<std::endl;
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(right_source_table,right_destination_table);
      CHECK_STREAM(0);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table throwing a GDF error");
    }

    std::cout<<"RGSL : Construct joined column"<<std::endl;
    // Construct the joined columns
    if (0 != ljoincols.num_columns() && merging_join_cols.num_rows() > 0) {

      std::vector <gdf_column *> l_join;
      std::vector <gdf_column *> r_join;
      // Gather the columns which merge into single column
      for (int join_ind = 0; join_ind < merging_join_cols.num_rows(); ++join_ind)
      {
          std::vector<int>::iterator itr = std::find(left_j_cols.begin(), left_j_cols.end(),
               l_common_name_join_ind[join_ind]);

          int index = std::distance(left_j_cols.begin(), itr);

          l_join.push_back(const_cast<gdf_column*>(ljoincols.get_column(index)));

          if (JoinType::FULL_JOIN == join_type)
          {
              r_join.push_back(const_cast<gdf_column*>(rjoincols.get_column(index)));
          }
      }
      cudf::table join_source_table(l_join);
      cudf::table join_destination_table(result_joincol);

      // Gather valid rows from the right table
      // TODO: Revisit this, because it probably can be done more efficiently
      if (JoinType::FULL_JOIN == join_type) {
        cudf::table right_source_table(r_join);

        cudf::detail::gather(
            &right_source_table,
            static_cast<index_type const *>(right_indices->data),
            &join_destination_table, check_bounds);
        CHECK_STREAM(0);
      }

      cudf::detail::gather(&join_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &join_destination_table, check_bounds);
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(join_source_table,join_destination_table);
      CHECK_STREAM(0);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table error");
    }
     
    POP_RANGE();
    return std::pair<cudf::table, cudf::table>(result_left, result_right);
}

template <JoinType join_type, typename index_type>
std::pair<cudf::table, cudf::table> join_call_compute_df(
                         cudf::table const& left_cols, 
                         cudf::table const& right_cols,
                         cudf::table const& join_cols,
                         cudf::table const& merging_join_cols,
                         cudf::table const& out_indices, 
                         gdf_context *join_context) {
 
  std::cout <<"RGSL : Entering the join_call_compute_df"<<std::endl; 
  CUDF_EXPECTS (0 != left_cols.num_columns(), "Left table is empty");
  CUDF_EXPECTS (0 != right_cols.num_columns(), "Right table is empty");
  CUDF_EXPECTS (nullptr != join_context, "Join context is invalid");

  std::vector<int> left_join_cols (join_cols.num_rows());
  std::vector<int> right_join_cols (join_cols.num_rows());
  std::vector<int> l_common_name_join_ind (merging_join_cols.num_rows());
  std::vector<int> r_common_name_join_ind (merging_join_cols.num_rows());

  std::cout <<"l_common_name_join_ind size "<< l_common_name_join_ind.size()<<std::endl;
  std::cout <<"left_join_cols size "<< left_join_cols.size()<<std::endl;

  if (join_cols.num_rows() > 0)
  {
      CUDA_TRY (hipMemcpy((void *)left_join_cols.data(), (void *)join_cols.get_column(0)->data, 
                           sizeof(int)*join_cols.num_rows(), hipMemcpyDeviceToHost));
      CUDA_TRY (hipMemcpy((void *)right_join_cols.data(), (void *)join_cols.get_column(1)->data, 
                           sizeof(int)*join_cols.num_rows(), hipMemcpyDeviceToHost));
  }
  if (merging_join_cols.num_rows() > 0)
  {
      CUDA_TRY (hipMemcpy(l_common_name_join_ind.data(), merging_join_cols.get_column(0)->data, 
                           sizeof(int)*merging_join_cols.num_rows(), hipMemcpyDeviceToHost));
      CUDA_TRY (hipMemcpy(r_common_name_join_ind.data(), merging_join_cols.get_column(1)->data, 
                           sizeof(int)*merging_join_cols.num_rows(), hipMemcpyDeviceToHost));
  }

  std::vector <gdf_column*> tmp_right_cols;
  std::vector<int> r_col_ind(right_cols.num_columns());
  std::vector <int> tmp_r_join_ind = r_common_name_join_ind;
  std::iota(std::begin(r_col_ind), std::end(r_col_ind), 0);
  std::sort(std::begin(tmp_r_join_ind), std::end(tmp_r_join_ind));
  std::vector <int> r_nonjoin_ind (right_cols.num_columns() - r_common_name_join_ind.size());

  // Gathering the indices that are not in join 
  std::set_difference(std::cbegin(r_col_ind), std::cend(r_col_ind),
                      std::cbegin(tmp_r_join_ind), std::cend(tmp_r_join_ind),
                      std::begin(r_nonjoin_ind));

  for (std::vector<int>::iterator it = r_nonjoin_ind.begin() ; it != r_nonjoin_ind.end(); ++it)
      tmp_right_cols.push_back(const_cast<gdf_column *> (right_cols.get_column(*it)));

  cudf::table tmp_right_table = (tmp_right_cols.size()>0)? cudf::table (tmp_right_cols) : cudf::table{};


  if (0 == join_cols.num_rows())
  {
      return std::pair <cudf::table, cudf::table>(cudf::empty_like(left_cols), cudf::empty_like(tmp_right_table));
  }

  for(int column_index = 0; column_index  < left_cols.num_columns(); column_index++){
    CUDF_EXPECTS(left_cols.get_column(column_index)->dtype != GDF_invalid, "Unsupported Dtype in Left column");
  }
  for(int column_index = 0; column_index  < right_cols.num_columns(); column_index++){
    CUDF_EXPECTS(right_cols.get_column(column_index)->dtype != GDF_invalid, "Unsupported Dtype in right column");
  }

  auto const left_col_size = left_cols.get_column(0)->size;
  auto const right_col_size = right_cols.get_column(0)->size;

  // If the inputs are empty, immediately return
  if ((0 == left_col_size) && (0 == right_col_size)) {
      return std::pair <cudf::table, cudf::table>(cudf::empty_like(left_cols), cudf::empty_like(tmp_right_table));
  }

  // If left join and the left table is empty, return immediately
  if ((JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)) {
      return std::pair <cudf::table, cudf::table>(cudf::empty_like(left_cols), cudf::empty_like(tmp_right_table));
  }

  // If Inner Join and either table is empty, return immediately
  if ((JoinType::INNER_JOIN == join_type) &&
      ((0 == left_col_size) || (0 == right_col_size))) {
      return std::pair <cudf::table, cudf::table>(cudf::empty_like(left_cols), cudf::empty_like(tmp_right_table));
  }


  //if the inputs are nvcategory we need to make the dictionaries comparable
  bool at_least_one_category_column = false;
  for(int join_column_index = 0; join_column_index < join_cols.num_rows(); join_column_index++){
    at_least_one_category_column |= left_cols.get_column(left_join_cols[join_column_index])->dtype == GDF_STRING_CATEGORY;
  }

  std::cout <<"RGSL : After first set of things"<<std::endl; 
  std::vector<gdf_column*> new_left_cols;
  std::vector<gdf_column*> new_right_cols;

  for (int i = 0; i < left_cols.num_columns(); i++)
      new_left_cols.push_back (const_cast<gdf_column*>(left_cols.get_column(i)));
  for (int i = 0; i < right_cols.num_columns(); i++)
      new_right_cols.push_back (const_cast<gdf_column*>(right_cols.get_column(i)));

  std::vector<gdf_column *> temp_columns_to_free;
  if(at_least_one_category_column){
    for(int join_column_index = 0; join_column_index < join_cols.num_rows(); join_column_index++){
      if(left_cols.get_column(left_join_cols[join_column_index])->dtype == GDF_STRING_CATEGORY){
        CUDF_EXPECTS(right_cols.get_column(right_join_cols[join_column_index])->dtype == GDF_STRING_CATEGORY, "GDF type mismatch");

        gdf_column * left_original_column = new_left_cols[left_join_cols[join_column_index]];
        gdf_column * right_original_column = new_right_cols[right_join_cols[join_column_index]];

        gdf_column * new_left_column_ptr = new gdf_column{};
        gdf_column * new_right_column_ptr = new gdf_column{};

        temp_columns_to_free.push_back(new_left_column_ptr);
        temp_columns_to_free.push_back(new_right_column_ptr);

        gdf_column * input_join_columns_merge[2] = {left_original_column, right_original_column};
        gdf_column * new_join_columns[2] = {new_left_column_ptr,
            new_right_column_ptr};
        gdf_column_view(new_left_column_ptr, nullptr, nullptr, left_original_column->size, GDF_STRING_CATEGORY);
        gdf_column_view(new_right_column_ptr, nullptr, nullptr, right_original_column->size, GDF_STRING_CATEGORY);

        int col_width = cudf::byte_width(*new_left_column_ptr);
        RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->data), col_width * left_original_column->size, 0) ); // TODO: non-default stream?
        if(left_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(left_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_left_column_ptr->valid, left_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(left_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_left_column_ptr->valid = nullptr;
        }
        new_left_column_ptr->null_count = left_original_column->null_count;


        RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->data), col_width * right_original_column->size, 0) ); // TODO: non-default stream?
        if(right_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(right_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_right_column_ptr->valid, right_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(right_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_right_column_ptr->valid = nullptr;
        }
        new_right_column_ptr->null_count = right_original_column->null_count;
        gdf_error err = sync_column_categories(input_join_columns_merge,
            new_join_columns,
            2);

        CUDF_EXPECTS(GDF_SUCCESS == err, "GDF_ERROR");

        new_left_cols[left_join_cols[join_column_index]] = new_join_columns[0];
        new_right_cols[right_join_cols[join_column_index]] = new_join_columns[1];
        CHECK_STREAM(0);
      }
    }
  }

  cudf::table  updated_left_cols(new_left_cols);
  cudf::table  updated_right_cols(new_right_cols);

  using gdf_col_pointer =
      typename std::unique_ptr<gdf_column, std::function<void(gdf_column *)>>;
  auto gdf_col_deleter = [](gdf_column *col) {
    col->size = 0;
    if (col->data) {
      RMM_FREE(col->data, 0);
    }
    if (col->valid) {
      RMM_FREE(col->valid, 0);
    }
  };

  gdf_col_pointer l_index_temp, r_index_temp;
  gdf_column *left_index_out = nullptr;
  gdf_column *right_index_out = nullptr;

  if (out_indices.num_columns () > 0)
  {
      left_index_out = const_cast <gdf_column*>(out_indices.get_column(0));
      right_index_out = const_cast <gdf_column*>(out_indices.get_column(1));
  }
  else
  {
      l_index_temp = {new gdf_column{}, gdf_col_deleter};
      left_index_out = l_index_temp.get();

      r_index_temp = {new gdf_column{}, gdf_col_deleter};
      right_index_out = r_index_temp.get();
  }

  //get column pointers to join on
  std::vector<gdf_column*> ljoincol;
  std::vector<gdf_column*> rjoincol;
  for (int i = 0; i < join_cols.num_rows(); ++i) {
      ljoincol.push_back(updated_left_cols.get_column(left_join_cols[i]));
      rjoincol.push_back(updated_right_cols.get_column(right_join_cols[i]));
  }
  std::cout <<"RGSL : Before Join call"<<std::endl; 
  cudf::table ljoin_cols(ljoincol);
  cudf::table rjoin_cols(rjoincol);
  join_call<join_type>(ljoin_cols, rjoin_cols,
            left_index_out, right_index_out,
            join_context);
  CHECK_STREAM(0);

  std::cout <<"RGSL : After Join call"<<std::endl; 
  std::pair<cudf::table, cudf::table> merged_result =
      construct_join_output_df<join_type, index_type>(
          ljoin_cols, rjoin_cols,
          updated_left_cols, updated_right_cols, 
          join_cols, merging_join_cols, 
          left_index_out, right_index_out);
  CHECK_STREAM(0);
  l_index_temp.reset(nullptr);
  r_index_temp.reset(nullptr);

  //freeing up the temp column used to synch categories between columns
  for(unsigned int column_to_free = 0; column_to_free < temp_columns_to_free.size(); column_to_free++){
      gdf_column_free(temp_columns_to_free[column_to_free]);
      delete temp_columns_to_free[column_to_free];
  }

  CHECK_STREAM(0);
    
  return merged_result;
}

std::pair<cudf::table, cudf::table> left_join(
                         cudf::table const& left_cols,
                         cudf::table const& right_cols,
                         cudf::table const& join_cols,
                         cudf::table const& merging_join_cols,
                         cudf::table const& out_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::LEFT_JOIN, output_index_type>(
                     left_cols,
                     right_cols,
                     join_cols,
                     merging_join_cols,
                     out_indices,
                     join_context);
}

std::pair<cudf::table, cudf::table> inner_join(
                         cudf::table const& left_cols,
                         cudf::table const& right_cols,
                         cudf::table const& join_cols,
                         cudf::table const& merging_join_cols,
                         cudf::table const& out_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::INNER_JOIN, output_index_type>(
                     left_cols,
                     right_cols,
                     join_cols,
                     merging_join_cols,
                     out_indices,
                     join_context);
}

std::pair<cudf::table, cudf::table> full_join(
                         cudf::table const& left_cols,
                         cudf::table const& right_cols,
                         cudf::table const& join_cols,
                         cudf::table const& merging_join_cols,
                         cudf::table const& out_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::FULL_JOIN, output_index_type>(
                     left_cols,
                     right_cols,
                     join_cols,
                     merging_join_cols,
                     out_indices,
                     join_context);
}
}

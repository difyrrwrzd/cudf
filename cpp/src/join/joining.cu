#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */



#include <cudf/types.hpp>
#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <cudf/copying.hpp>
#include <utilities/column_utils.hpp>
#include <utilities/error_utils.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/nvtx/nvtx_utils.h>
#include <cudf/utilities/legacy/nvcategory_util.hpp>
#include <nvstrings/NVCategory.h>
#include <copying/gather.hpp>
#include "joining.h"

#include <limits>
#include <set>
#include <vector>
#include <numeric>
#include <algorithm>

// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
using output_index_type = gdf_index_type;
constexpr output_index_type MAX_JOIN_SIZE{std::numeric_limits<output_index_type>::max()};

namespace cudf {
/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a repeated value
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @param[in] value The value to be filled into the buffer
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocValueBuffer(data_type ** buffer,
                           const gdf_size_type buffer_length,
                           const data_type value) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::fill(thrust::device, *buffer, *buffer + buffer_length, value);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a sequence
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocSequenceBuffer(data_type ** buffer,
                              const gdf_size_type buffer_length) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::sequence(thrust::device, *buffer, *buffer + buffer_length);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Trivially computes full join of two tables if one of the tables
 * are empty
 *
 * @throws cudf::logic_error
 * "Dataset is empty" if both left_dataframe and right_dataframe is empty
 * 
 * @param[in] left_size The size of the left table
 * @param[in] right_size The size of the right table
 * @param[in] rightcol The right set of columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 */
/* ----------------------------------------------------------------------------*/
void trivial_full_join(
        const gdf_size_type left_size,
        const gdf_size_type right_size,
        gdf_column *left_result,
        gdf_column *right_result) {
    // Deduce the type of the output gdf_columns
    gdf_dtype dtype;
    switch(sizeof(output_index_type))
    {
      case 1 : dtype = GDF_INT8;  break;
      case 2 : dtype = GDF_INT16; break;
      case 4 : dtype = GDF_INT32; break;
      case 8 : dtype = GDF_INT64; break;
    }

    output_index_type *l_ptr{nullptr};
    output_index_type *r_ptr{nullptr};
    gdf_size_type result_size{0};
    CUDF_EXPECTS((left_size != 0) || (right_size != 0), "Dataset is empty");
    if (left_size == 0) {
        allocValueBuffer(&l_ptr, right_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&r_ptr, right_size);
        result_size = right_size;
    } else if (right_size == 0) {
        allocValueBuffer(&r_ptr, left_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&l_ptr, left_size);
        result_size = left_size;
    }

    gdf_column_view_augmented(left_result,
                              l_ptr, nullptr,
                              result_size, dtype, 0,
                              left_result->dtype_info,
                              left_result->col_name);

    gdf_column_view_augmented(right_result,
                              r_ptr, nullptr,
                              result_size, dtype, 0,
                              right_result->dtype_info,
                              right_result->col_name);

    CUDA_CHECK_LAST();
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Computes the join operation between two sets of columns
 *
 * @throws cudf::logic_error
 * 
 * @param[in] left  Table of left columns to join
 * @param[in] right Table of right  columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 * @param[in] join_context A structure that determines various run parameters, such as
 *                         whether to perform a hash or sort based join
 * @tparam join_type The type of join to be performed
 * 
 * @returns void
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
void join_call(cudf::table const& left, cudf::table const& right,
                     gdf_column *left_result, gdf_column *right_result,
                     gdf_context *join_context)
{
  CUDF_EXPECTS( 0 != left.num_columns(), "Left Dataset is empty");
  CUDF_EXPECTS( 0 != right.num_columns(), "Right Dataset is empty");
  CUDF_EXPECTS( nullptr != join_context, "Invalid join context");
  CUDF_EXPECTS( left.num_rows() < MAX_JOIN_SIZE, "left column size is too big");
  CUDF_EXPECTS( right.num_rows() < MAX_JOIN_SIZE, "right column size is too big");

  // If both frames are empty, return immediately
  if((0 == left.num_rows() ) && (0 == right.num_rows())) {
    return;
  }

  // If left join and the left table is empty, return immediately
  if( (JoinType::LEFT_JOIN == join_type) && (0 == left.num_rows())){
    return;
  }

  // If Inner Join and either table is empty, return immediately
  if( (JoinType::INNER_JOIN == join_type) && 
      ((0 == left.num_rows()) || (0 == right.num_rows())) ){
    return;
  }

  // If Full Join and either table is empty, compute trivial full join
  if( (JoinType::FULL_JOIN == join_type) && 
      ((0 == left.num_rows()) || (0 == right.num_rows())) ){
    return trivial_full_join(left.num_rows(), right.num_rows(), left_result, right_result);
  }

  // check that the columns data are not null, have matching types, 
  // and the same number of rows
  for (int i = 0; i < left.num_columns(); i++) {
    CUDF_EXPECTS (!((left.num_rows() > 0) && (nullptr == left.get_column(i)->data)), "Null column data in left set");
    CUDF_EXPECTS (!((right.num_rows() > 0) && (nullptr == right.get_column(i)->data)), "Null column data in right set");
    CUDF_EXPECTS (right.get_column(i)->dtype == left.get_column(i)->dtype, "DTYPE mismatch");

    // Ensure GDF_TIMESTAMP columns have the same resolution
    if (GDF_TIMESTAMP == right.get_column(i)->dtype) {
      CUDF_EXPECTS(
          right.get_column(i)->dtype_info.time_unit == left.get_column(i)->dtype_info.time_unit,
          "Timestamp resolution mismatch");
    }
  }

  gdf_method join_method = join_context->flag_method; 
  gdf_error gdf_error_code{GDF_SUCCESS};

  PUSH_RANGE("LIBGDF_JOIN", JOIN_COLOR);

  switch(join_method)
  {
    case GDF_HASH:
      {
        gdf_error_code = join_hash<join_type, output_index_type>(left, right, left_result, right_result);
        CUDF_EXPECTS(gdf_error_code == GDF_SUCCESS, "GDF Error");
        break;
      }
    case GDF_SORT:
      {
        // Sort based joins only support single column joins
        if(1 == left.num_columns())
        {
          gdf_error_code =  sort_join<join_type, output_index_type>(const_cast <gdf_column*> (left.get_column(0)), 
                            const_cast <gdf_column*> (right.get_column(0)), left_result, right_result);
          CUDF_EXPECTS(gdf_error_code != GDF_VALIDITY_UNSUPPORTED, "GDF Validity is unsupported by sort_join");
          CUDF_EXPECTS(gdf_error_code == GDF_SUCCESS, "GDF Error");
        }
        else
        {
          CUDF_EXPECTS(false, "Too many columns to join");
        }

        break;
      }
    default:
      CUDF_EXPECTS(false, "Unsupported Method");
  }

  POP_RANGE();
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Computes the resulting joined table
 *
 * @throws cudf::logic_error
 * 
 * @param[in] ljoin  Table of left join columns 
 * @param[in] rjoin  Table of right join columns
 * @param[in] left   Updated left dataframe
 * @param[in] right  Updated right dataframe
 * @param[in] left_on The table containing two columns,
 * first column - indices of left table provided for join
 * second column - indices of right table provided for join
 * @param[in] joining_ind is a vector of pairs of left and right
 * join indcies derived from left_on and right_on. This contains
 * the indices with the same name which evetually result into a 
 * single column.
 * @param[in] left_indices 
 * @param[in] right_indicess  Table contatining right join columns
 * @tparam join_type The type of join to be performed
 * 
 * @returns void
 */
/* ----------------------------------------------------------------------------*/

template <JoinType join_type, typename index_type>
std::pair<cudf::table, cudf::table> construct_join_output_df(
        cudf::table const& ljoin,
        cudf::table const& rjoin,
        cudf::table const& left, 
        cudf::table const& right,
        cudf::table const& left_on,
        std::vector<std::pair<int, int>> const& joining_ind,
        gdf_column * left_indices,
        gdf_column * right_indices) {

    PUSH_RANGE("LIBGDF_JOIN_OUTPUT", JOIN_COLOR);
    //create left and right input table with columns not joined on
    std::vector<int> l_col_ind(left.num_columns());
    std::vector<int> r_col_ind(right.num_columns());
    std::vector<int> left_j_cols (left_on.num_rows());
    std::vector<int> l_joining_ind (joining_ind.size());
    std::vector<int> r_joining_ind (joining_ind.size());
    std::vector<int> l_nonjoin_ind (left.num_columns() - l_joining_ind.size());
    std::vector<int> r_nonjoin_ind (right.num_columns() - r_joining_ind.size());

    if (left_on.num_rows() > 0)
    {
        CUDA_TRY (hipMemcpy(left_j_cols.data(), left_on.get_column(0)->data, 
                             sizeof(int)*left_on.num_rows() , hipMemcpyDeviceToHost));
    }

    for (unsigned int i = 0; i < joining_ind.size(); ++i)
    {
        l_joining_ind[i] = joining_ind[i].first;
        r_joining_ind[i] = joining_ind[i].second;
    }
    
    std::vector <int> tmp_l_join_ind = l_joining_ind;
    std::vector <int> tmp_r_join_ind = r_joining_ind;
  
    std::iota(std::begin(l_col_ind), std::end(l_col_ind), 0);
    std::iota(std::begin(r_col_ind), std::end(r_col_ind), 0);
    std::sort(std::begin(tmp_l_join_ind), std::end(tmp_l_join_ind));
    std::sort(std::begin(tmp_r_join_ind), std::end(tmp_r_join_ind));

    // Gathering the indices that are not in join
    if (l_nonjoin_ind.size() > 0)
        std::set_difference(std::cbegin(l_col_ind), std::cend(l_col_ind),
                                 std::cbegin(tmp_l_join_ind), std::cend(tmp_l_join_ind),
                                 std::begin(l_nonjoin_ind));
    
    if (r_nonjoin_ind.size() > 0)
        std::set_difference(std::cbegin(r_col_ind), std::cend(r_col_ind),
                                 std::cbegin(tmp_r_join_ind), std::cend(tmp_r_join_ind),
                                 std::begin(r_nonjoin_ind));

    gdf_size_type join_size = left_indices->size;
    std::vector <gdf_dtype> rdtypes;
    std::vector <gdf_dtype_extra_info> rdtype_infos;

    std::vector<gdf_column*> lnonjoincol;
    std::vector<gdf_column*> rnonjoincol;

    // Gathering all the left table columns not in joining indices 
    for (std::vector<int>::iterator it = l_nonjoin_ind.begin() ; it != l_nonjoin_ind.end(); ++it)
    {
        lnonjoincol.push_back(const_cast<gdf_column*>(left.get_column(*it)));
    }
    // Gathering all the right table columns not in joining indices
    for (std::vector<int>::iterator it = r_nonjoin_ind.begin() ; it != r_nonjoin_ind.end(); ++it){
        rnonjoincol.push_back(const_cast<gdf_column*>(right.get_column(*it)));
        rdtypes.push_back(right.get_column(*it)->dtype);
        rdtype_infos.push_back(right.get_column(*it)->dtype_info);
    }
  
    cudf::table result_left(join_size, cudf::column_dtypes(left), cudf::column_dtype_infos(left), true);
    cudf::table result_right(join_size, rdtypes, rdtype_infos, true);
    
    std::vector<gdf_column*> result_lnonjoincol;
    std::vector<gdf_column*> result_rnonjoincol;
    std::vector<gdf_column*> result_joincol;

    // Gather the left non-join col of result
    for (std::vector<int>::iterator it = l_nonjoin_ind.begin(); it != l_nonjoin_ind.end(); ++it)
    {
        result_lnonjoincol.push_back(result_left.get_column(*it));
    }
        
    // Gather join-col of result 
    for (unsigned int i=0; i < joining_ind.size(); ++i)
    {
        result_joincol.push_back(result_left.get_column(l_joining_ind[i]));
    }
    
    // Gather the right non-join col of result
    for (int i=0; i < result_right.num_columns(); ++i)
    {
        result_rnonjoincol.push_back(result_right.get_column(i));
    }
 
    bool const check_bounds{ join_type != JoinType::INNER_JOIN };

    // Construct the left columns
    if (0 != lnonjoincol.size()) {
      cudf::table left_source_table(lnonjoincol);
      cudf::table left_destination_table(result_lnonjoincol);

      cudf::detail::gather(&left_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &left_destination_table, check_bounds);

      gdf_error update_err = nvcategory_gather_table(left_source_table,left_destination_table);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table throwing a GDF error");
    }
    
    // Construct the right columns
    if (0 != rnonjoincol.size()) {
      cudf::table right_source_table(rnonjoincol);
      cudf::table right_destination_table(result_rnonjoincol);

      cudf::detail::gather(&right_source_table,
                           static_cast<index_type const *>(right_indices->data),
                           &right_destination_table, check_bounds);
      gdf_error update_err = nvcategory_gather_table(right_source_table,right_destination_table);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table throwing a GDF error");
    }

    // Construct the joined columns
    if (0 != ljoin.num_columns() && joining_ind.size() > 0) {

      std::vector <gdf_column *> l_join;
      std::vector <gdf_column *> r_join;
      // Gather the columns which join into single column from joined columns
      for (unsigned int join_ind = 0; join_ind < joining_ind.size(); ++join_ind)
      {
          std::vector<int>::iterator itr = std::find(left_j_cols.begin(), left_j_cols.end(),
               l_joining_ind[join_ind]);

          int index = std::distance(left_j_cols.begin(), itr);

          l_join.push_back(const_cast<gdf_column*>(ljoin.get_column(index)));

          if (JoinType::FULL_JOIN == join_type)
          {
              r_join.push_back(const_cast<gdf_column*>(rjoin.get_column(index)));
          }
      }
      cudf::table join_source_table(l_join);
      cudf::table join_destination_table(result_joincol);

      // Gather valid rows from the right table
      // TODO: Revisit this, because it probably can be done more efficiently
      if (JoinType::FULL_JOIN == join_type) {
        cudf::table right_source_table(r_join);

        cudf::detail::gather(
            &right_source_table,
            static_cast<index_type const *>(right_indices->data),
            &join_destination_table, check_bounds);
      }

      cudf::detail::gather(&join_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &join_destination_table, check_bounds);
      gdf_error update_err = nvcategory_gather_table(join_source_table,join_destination_table);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table error");
    }
     
    CHECK_STREAM(0);
    POP_RANGE();
    return std::pair<cudf::table, cudf::table>(result_left, result_right);
}

template <JoinType join_type, typename index_type>
std::pair<cudf::table, cudf::table> join_call_compute_df(
                         cudf::table const& left, 
                         cudf::table const& right,
                         cudf::table const& left_on,
                         cudf::table const& right_on,
                         std::vector<std::pair<int, int>> const& joining_ind,
                         cudf::table *out_ind, 
                         gdf_context *join_context) {
 
  if (0 == left_on.num_rows() || 0 == right_on.num_rows() || left_on.num_rows() != right_on.num_rows())
  {
      return std::pair <cudf::table, cudf::table>(cudf::empty_like(left), cudf::empty_like(right));
  }
  CUDF_EXPECTS (0 != left.num_columns(), "Left table is empty");
  CUDF_EXPECTS (0 != right.num_columns(), "Right table is empty");
  CUDF_EXPECTS (nullptr != join_context, "Join context is invalid");

  std::vector<int> left_on_ind (left_on.num_rows());
  std::vector<int> right_on_ind (right_on.num_rows());
  std::vector<int> r_joining_ind (joining_ind.size());

  if (left_on.num_rows() > 0)
  {
      CUDA_TRY (hipMemcpy((void *)left_on_ind.data(), (void *)left_on.get_column(0)->data, 
                           sizeof(int)*left_on.num_rows(), hipMemcpyDeviceToHost));
  }
  if (right_on.num_rows() > 0)
  {
      CUDA_TRY (hipMemcpy((void *)right_on_ind.data(), (void *)right_on.get_column(0)->data, 
                           sizeof(int)*right_on.num_rows(), hipMemcpyDeviceToHost));
  }

  for (unsigned int i = 0; i < joining_ind.size(); ++i)
  {
      r_joining_ind [i] = joining_ind[i].second;
  }

  std::vector <gdf_column*> tmp_right_cols;
  std::vector<int> r_col_ind(right.num_columns());
  std::iota(std::begin(r_col_ind), std::end(r_col_ind), 0);
  std::sort(std::begin(r_joining_ind), std::end(r_joining_ind));
  std::vector <int> r_nonjoin_ind (right.num_columns() - r_joining_ind.size());

  // Gathering the indices that are not in join 
  std::set_difference(std::cbegin(r_col_ind), std::cend(r_col_ind),
                      std::cbegin(r_joining_ind), std::cend(r_joining_ind),
                      std::begin(r_nonjoin_ind));

  for (std::vector<int>::iterator it = r_nonjoin_ind.begin() ; it != r_nonjoin_ind.end(); ++it){
      tmp_right_cols.push_back(const_cast<gdf_column *> (right.get_column(*it)));
  }

  cudf::table tmp_right_table = (tmp_right_cols.size()>0)? cudf::table (tmp_right_cols) : cudf::table{};

  CUDF_EXPECTS(std::none_of(std::cbegin(left), std::cend(left), [](auto col) { return col->dtype == GDF_invalid; }), "Unsupported left column dtype");
  CUDF_EXPECTS(std::none_of(std::cbegin(right), std::cend(right), [](auto col) { return col->dtype == GDF_invalid; }), "Unsupported right column dtype");

  // Even though the resulting table might be empty, but the column should match the expected dtypes and other necessary information
  // So, there is a possibility that there will be lesser number of right columns, so the tmp_right_table.
  // If the inputs are empty, immediately return
  if ((0 == left.num_rows()) && (0 == right.num_rows())) {
      return std::pair <cudf::table, cudf::table>(cudf::empty_like(left), cudf::empty_like(tmp_right_table));
  }

  // If left join and the left table is empty, return immediately
  if ((JoinType::LEFT_JOIN == join_type) && (0 == left.num_rows())) {
      return std::pair <cudf::table, cudf::table>(cudf::empty_like(left), cudf::empty_like(tmp_right_table));
  }

  // If Inner Join and either table is empty, return immediately
  if ((JoinType::INNER_JOIN == join_type) &&
      ((0 == left.num_rows()) || (0 == right.num_rows()))) {
      return std::pair <cudf::table, cudf::table>(cudf::empty_like(left), cudf::empty_like(tmp_right_table));
  }

  //if the inputs are nvcategory we need to make the dictionaries comparable
  bool at_least_one_category_column = false;
  for(int join_column_index = 0; join_column_index < left_on.num_rows(); join_column_index++){
    at_least_one_category_column |= left.get_column(left_on_ind[join_column_index])->dtype == GDF_STRING_CATEGORY;
  }
  
  std::vector<gdf_column*> new_left_cols;
  std::vector<gdf_column*> new_right_cols;

  for (int i = 0; i < left.num_columns(); i++)
      new_left_cols.push_back (const_cast<gdf_column*>(left.get_column(i)));
  for (int i = 0; i < right.num_columns(); i++)
      new_right_cols.push_back (const_cast<gdf_column*>(right.get_column(i)));

  std::vector<gdf_column *> temp_columns_to_free;
  if(at_least_one_category_column){
    for(int join_column_index = 0; join_column_index < left_on.num_rows(); join_column_index++){
      if(left.get_column(left_on_ind[join_column_index])->dtype == GDF_STRING_CATEGORY){
        CUDF_EXPECTS(right.get_column(right_on_ind[join_column_index])->dtype == GDF_STRING_CATEGORY, "GDF type mismatch");

        gdf_column * left_original_column = new_left_cols[left_on_ind[join_column_index]];
        gdf_column * right_original_column = new_right_cols[right_on_ind[join_column_index]];

        gdf_column * new_left_column_ptr = new gdf_column{};
        gdf_column * new_right_column_ptr = new gdf_column{};

        temp_columns_to_free.push_back(new_left_column_ptr);
        temp_columns_to_free.push_back(new_right_column_ptr);

        gdf_column * input_join_columns_merge[2] = {left_original_column, right_original_column};
        gdf_column * new_join_columns[2] = {new_left_column_ptr,
            new_right_column_ptr};
        gdf_column_view(new_left_column_ptr, nullptr, nullptr, left_original_column->size, GDF_STRING_CATEGORY);
        gdf_column_view(new_right_column_ptr, nullptr, nullptr, right_original_column->size, GDF_STRING_CATEGORY);

        int col_width = cudf::byte_width(*new_left_column_ptr);
        RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->data), col_width * left_original_column->size, 0) ); // TODO: non-default stream?
        if(left_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(left_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_left_column_ptr->valid, left_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(left_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_left_column_ptr->valid = nullptr;
        }
        new_left_column_ptr->null_count = left_original_column->null_count;


        RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->data), col_width * right_original_column->size, 0) ); // TODO: non-default stream?
        if(right_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(right_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_right_column_ptr->valid, right_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(right_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_right_column_ptr->valid = nullptr;
        }
        new_right_column_ptr->null_count = right_original_column->null_count;
        gdf_error err = sync_column_categories(input_join_columns_merge,
            new_join_columns,
            2);

        CUDF_EXPECTS(GDF_SUCCESS == err, "GDF_ERROR");

        new_left_cols[left_on_ind[join_column_index]] = new_join_columns[0];
        new_right_cols[right_on_ind[join_column_index]] = new_join_columns[1];
      }
    }
  }

  cudf::table  updated_left_table(new_left_cols);
  cudf::table  updated_right_table(new_right_cols);

  using gdf_col_pointer =
      typename std::unique_ptr<gdf_column, std::function<void(gdf_column *)>>;
  auto gdf_col_deleter = [](gdf_column *col) {
    col->size = 0;
    if (col->data) {
      RMM_FREE(col->data, 0);
    }
    if (col->valid) {
      RMM_FREE(col->valid, 0);
    }
  };

  gdf_col_pointer l_index_temp, r_index_temp;
  gdf_column *left_index_out = nullptr;
  gdf_column *right_index_out = nullptr;

  if (nullptr != out_ind && out_ind->num_columns () > 0)
  {
      left_index_out = const_cast <gdf_column*>(out_ind->get_column(0));
      right_index_out = const_cast <gdf_column*>(out_ind->get_column(1));
  }
  else
  {
      l_index_temp = {new gdf_column{}, gdf_col_deleter};
      left_index_out = l_index_temp.get();

      r_index_temp = {new gdf_column{}, gdf_col_deleter};
      right_index_out = r_index_temp.get();
  }

  //get column pointers to join on
  std::vector<gdf_column*> ljoincol;
  std::vector<gdf_column*> rjoincol;
  for (int i = 0; i < left_on.num_rows(); ++i) {
      ljoincol.push_back(updated_left_table.get_column(left_on_ind[i]));
      rjoincol.push_back(updated_right_table.get_column(right_on_ind[i]));
  }
  cudf::table ljoin_ind_table(ljoincol);
  cudf::table rjoin_ind_table(rjoincol);
  join_call<join_type>(ljoin_ind_table, rjoin_ind_table,
            left_index_out, right_index_out,
            join_context);

  std::pair<cudf::table, cudf::table> result =
      construct_join_output_df<join_type, index_type>(
          ljoin_ind_table, rjoin_ind_table,
          updated_left_table, updated_right_table, 
          left_on, joining_ind, 
          left_index_out, right_index_out);
  l_index_temp.reset(nullptr);
  r_index_temp.reset(nullptr);

  //freeing up the temp column used to synch categories between columns
  for(unsigned int column_to_free = 0; column_to_free < temp_columns_to_free.size(); column_to_free++){
      gdf_column_free(temp_columns_to_free[column_to_free]);
      delete temp_columns_to_free[column_to_free];
  }

  CHECK_STREAM(0);
    
  return result;
}

std::pair<cudf::table, cudf::table> left_join(
                         cudf::table const& left,
                         cudf::table const& right,
                         cudf::table const& left_on,
                         cudf::table const& right_on,
                         std::vector<std::pair<int, int>> const& joining_ind,
                         cudf::table *out_ind,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::LEFT_JOIN, output_index_type>(
                     left,
                     right,
                     left_on,
                     right_on,
                     joining_ind,
                     out_ind,
                     join_context);
}

std::pair<cudf::table, cudf::table> inner_join(
                         cudf::table const& left,
                         cudf::table const& right,
                         cudf::table const& left_on,
                         cudf::table const& right_on,
                         std::vector<std::pair<int, int>> const& joining_ind,
                         cudf::table *out_ind,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::INNER_JOIN, output_index_type>(
                     left,
                     right,
                     left_on,
                     right_on,
                     joining_ind,
                     out_ind,
                     join_context);
}

std::pair<cudf::table, cudf::table> full_join(
                         cudf::table const& left,
                         cudf::table const& right,
                         cudf::table const& left_on,
                         cudf::table const& right_on,
                         std::vector<std::pair<int, int>> const& joining_ind,
                         cudf::table *out_ind,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::FULL_JOIN, output_index_type>(
                     left,
                     right,
                     left_on,
                     right_on,
                     joining_ind,
                     out_ind,
                     join_context);
}
}

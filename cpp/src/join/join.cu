/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/copying.hpp>
#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/error.hpp>

#include "hash_join.cuh"
#include "join_common_utils.hpp"
#include "nested_loop_join.cuh"

namespace cudf {
namespace detail {

/**
 * @brief Returns a vector with non-common indices which is set difference
 * between `[0, num_columns)` and index values in common_column_indices
 *
 * @param num_columns The number of columns, which represents column indices
 * from `[0, num_columns)` in a table
 * @param common_column_indices A vector of common indices which needs to be
 * excluded from `[0, num_columns)`
 *
 * @return vector A vector containing only the indices which are not present in
 * `common_column_indices`
 */
auto non_common_column_indices(size_type num_columns,
                               std::vector<size_type> const& common_column_indices)
{
  CUDF_EXPECTS(common_column_indices.size() <= static_cast<unsigned long>(num_columns),
               "Too many columns in common");
  std::vector<size_type> all_column_indices(num_columns);
  std::iota(std::begin(all_column_indices), std::end(all_column_indices), 0);
  std::vector<size_type> sorted_common_column_indices{common_column_indices};
  std::sort(std::begin(sorted_common_column_indices), std::end(sorted_common_column_indices));
  std::vector<size_type> non_common_column_indices(num_columns - common_column_indices.size());
  std::set_difference(std::cbegin(all_column_indices),
                      std::cend(all_column_indices),
                      std::cbegin(sorted_common_column_indices),
                      std::cend(sorted_common_column_indices),
                      std::begin(non_common_column_indices));
  return non_common_column_indices;
}

std::unique_ptr<table> get_empty_joined_table(
  table_view const& left,
  table_view const& right,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common)
{
  std::vector<size_type> right_columns_in_common(columns_in_common.size());
  std::transform(columns_in_common.begin(),
                 columns_in_common.end(),
                 right_columns_in_common.begin(),
                 [](auto& col) { return col.second; });
  std::unique_ptr<table> empty_left  = empty_like(left);
  std::unique_ptr<table> empty_right = empty_like(right);
  std::vector<size_type> right_non_common_indices =
    non_common_column_indices(right.num_columns(), right_columns_in_common);
  table_view tmp_right_table = (*empty_right).select(right_non_common_indices);
  table_view tmp_table{{*empty_left, tmp_right_table}};
  return std::make_unique<table>(tmp_table);
}

VectorPair concatenate_vector_pairs(VectorPair& a, VectorPair& b)
{
  CUDF_EXPECTS((a.first.size() == a.second.size()),
               "Mismatch between sizes of vectors in vector pair");
  CUDF_EXPECTS((b.first.size() == b.second.size()),
               "Mismatch between sizes of vectors in vector pair");
  if (a.first.size() == 0) {
    return b;
  } else if (b.first.size() == 0) {
    return a;
  }
  auto original_size = a.first.size();
  a.first.resize(a.first.size() + b.first.size());
  a.second.resize(a.second.size() + b.second.size());
  thrust::copy(b.first.begin(), b.first.end(), a.first.begin() + original_size);
  thrust::copy(b.second.begin(), b.second.end(), a.second.begin() + original_size);
  return a;
}

template <typename T>
struct valid_range {
  T start, stop;
  __host__ __device__ valid_range(const T begin, const T end) : start(begin), stop(end) {}

  __host__ __device__ __forceinline__ bool operator()(const T index)
  {
    return ((index >= start) && (index < stop));
  }
};

/**
 * @brief  Creates a table containing the complement of left join indices.
 * This table has two columns. The first one is filled with JoinNoneValue(-1)
 * and the second one contains values from 0 to right_table_row_count - 1
 * excluding those found in the right_indices column.
 *
 * @param right_indices Vector of indices
 * @param left_table_row_count Number of rows of left table
 * @param right_table_row_count Number of rows of right table
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Pair of vectors containing the left join indices complement
 */
std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>>
get_left_join_indices_complement(rmm::device_vector<size_type>& right_indices,
                                 size_type left_table_row_count,
                                 size_type right_table_row_count,
                                 hipStream_t stream)
{
  // Get array of indices that do not appear in right_indices

  // Vector allocated for unmatched result
  rmm::device_vector<size_type> right_indices_complement(right_table_row_count);

  // If left table is empty in a full join call then all rows of the right table
  // should be represented in the joined indices. This is an optimization since
  // if left table is empty and full join is called all the elements in
  // right_indices will be JoinNoneValue, i.e. -1. This if path should
  // produce exactly the same result as the else path but will be faster.
  if (left_table_row_count == 0) {
    thrust::sequence(rmm::exec_policy(stream)->on(stream),
                     right_indices_complement.begin(),
                     right_indices_complement.end(),
                     0);
  } else {
    // Assume all the indices in invalid_index_map are invalid
    rmm::device_vector<size_type> invalid_index_map(right_table_row_count, 1);
    // Functor to check for index validity since left joins can create invalid indices
    valid_range<size_type> valid(0, right_table_row_count);

    // invalid_index_map[index_ptr[i]] = 0 for i = 0 to right_table_row_count
    // Thus specifying that those locations are valid
    thrust::scatter_if(rmm::exec_policy(stream)->on(stream),
                       thrust::make_constant_iterator(0),
                       thrust::make_constant_iterator(0) + right_indices.size(),
                       right_indices.begin(),      // Index locations
                       right_indices.begin(),      // Stencil - Check if index location is valid
                       invalid_index_map.begin(),  // Output indices
                       valid);                     // Stencil Predicate
    size_type begin_counter = static_cast<size_type>(0);
    size_type end_counter   = static_cast<size_type>(right_table_row_count);

    // Create list of indices that have been marked as invalid
    size_type indices_count = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
                                              thrust::make_counting_iterator(begin_counter),
                                              thrust::make_counting_iterator(end_counter),
                                              invalid_index_map.begin(),
                                              right_indices_complement.begin(),
                                              thrust::identity<size_type>()) -
                              right_indices_complement.begin();
    right_indices_complement.resize(indices_count);
  }

  rmm::device_vector<size_type> left_invalid_indices(right_indices_complement.size(),
                                                     JoinNoneValue);

  return std::make_pair(std::move(left_invalid_indices), std::move(right_indices_complement));
}

std::unique_ptr<multimap_type> build_join_hash_table(table_device_view build_table,
                                                     hipStream_t stream)
{
  CUDF_EXPECTS(0 != build_table.num_columns(), "Selected build dataset is empty");
  CUDF_EXPECTS(0 != build_table.num_rows(), "Build side table has no rows");

  const size_type build_table_num_rows{build_table.num_rows()};
  size_t const hash_table_size = compute_hash_table_size(build_table_num_rows);
  auto hash_table              = multimap_type::create(hash_table_size,
                                          true,
                                          multimap_type::hasher(),
                                          multimap_type::key_equal(),
                                          multimap_type::allocator_type(),
                                          stream);

  row_hash hash_build{build_table};
  rmm::device_scalar<int> failure(0, 0);
  constexpr int block_size{DEFAULT_JOIN_BLOCK_SIZE};
  detail::grid_1d config(build_table_num_rows, block_size);
  build_hash_table<<<config.num_blocks, config.num_threads_per_block, 0, 0>>>(
    *hash_table, hash_build, build_table_num_rows, failure.data());
  // Check error code from the kernel
  if (failure.value() == 1) { CUDF_FAIL("Hash Table insert failure."); }

  return hash_table;
}

template <join_kind JoinKind>
std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>> probe_join_hash_table(
  table_device_view build_table,
  table_device_view probe_table,
  multimap_type const& hash_table,
  bool flip_join_indices,
  hipStream_t stream)
{
  size_type estimated_size = estimate_join_output_size<JoinKind, multimap_type>(
    build_table, probe_table, hash_table, stream);

  // If the estimated output size is zero, return immediately
  if (estimated_size == 0) {
    return std::make_pair(rmm::device_vector<size_type>{}, rmm::device_vector<size_type>{});
  }

  // Because we are approximating the number of joined elements, our approximation
  // might be incorrect and we might have underestimated the number of joined elements.
  // As such we will need to de-allocate memory and re-allocate memory to ensure
  // that the final output is correct.
  rmm::device_scalar<size_type> write_index(0, stream);
  size_type join_size{0};

  rmm::device_vector<size_type> left_indices;
  rmm::device_vector<size_type> right_indices;
  auto current_estimated_size = estimated_size;
  do {
    left_indices.resize(estimated_size);
    right_indices.resize(estimated_size);

    constexpr int block_size{DEFAULT_JOIN_BLOCK_SIZE};
    detail::grid_1d config(probe_table.num_rows(), block_size);
    write_index.set_value(0);

    row_hash hash_probe{probe_table};
    row_equality equality{probe_table, build_table};
    const auto& join_output_l =
      flip_join_indices ? right_indices.data().get() : left_indices.data().get();
    const auto& join_output_r =
      flip_join_indices ? left_indices.data().get() : right_indices.data().get();
    probe_hash_table<JoinKind, multimap_type, hash_value_type, block_size, DEFAULT_JOIN_CACHE_SIZE>
      <<<config.num_blocks, config.num_threads_per_block, 0, stream>>>(*hash_table,
                                                                       build_table,
                                                                       probe_table,
                                                                       hash_probe,
                                                                       equality,
                                                                       join_output_l,
                                                                       join_output_r,
                                                                       write_index.data(),
                                                                       estimated_size);

    CHECK_CUDA(stream);

    join_size              = write_index.value();
    current_estimated_size = estimated_size;
    estimated_size *= 2;
  } while ((current_estimated_size < join_size));

  left_indices.resize(join_size);
  right_indices.resize(join_size);
  return std::make_pair(std::move(left_indices), std::move(right_indices));
}

/**
 * @brief Computes the base join operation between two tables and returns the
 * output indices of left and right table as a combined table, i.e. if full
 * join is specified as the join type then left join is called.
 *
 * @throw cudf::logic_error if `left` or `right` table is empty
 * @throw cudf::logic_error if types do not match between joining columns
 *
 * @tparam JoinKind The type of join to be performed
 *
 * @param left  Table of left columns to join
 * @param right Table of right  columns to join
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Join output indices vector pair
 */
template <join_kind JoinKind>
std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>> get_base_join_indices(
  table_view const& build, table_view const& probe, bool flip_join_indices, hipStream_t stream)
{
  CUDF_EXPECTS(0 != build.num_columns(), "Selected build dataset is empty");
  CUDF_EXPECTS(0 != probe.num_columns(), "Selected probe dataset is empty");
  CUDF_EXPECTS(build.num_rows() <= probe.num_rows(), "Build dataset is bigger than probe dataset");
  CUDF_EXPECTS(std::equal(std::cbegin(build),
                          std::cend(build),
                          std::cbegin(probe),
                          std::cend(probe),
                          [](const auto& b, const auto& p) { return b.type() == p.type(); }),
               "Mismatch in joining column data types");

  constexpr join_kind BaseJoinKind =
    (JoinKind == join_kind::FULL_JOIN) ? join_kind::LEFT_JOIN : JoinKind;

  // Trivial left join case - exit early
  if (0 == build.num_rows() && BaseJoinKind == join_kind::LEFT_JOIN) {
    return get_trivial_left_join_indices(probe, flip_join_indices, stream);
  }

  auto build_table = table_device_view::create(build, stream);
  auto hash_table  = build_join_hash_table(*build_table, stream);

  auto probe_table = table_device_view::create(probe, stream);
  return probe_join_hash_table(*build_table, *probe_table, *hash_table, flip_join_indices, stream);
}

/**
 * @brief  Combines the non common left, common left and non common right
 * columns in the correct order to form the join output table.
 *
 * @param left_noncommon_cols Columns obtained by gathering non common left
 * columns.
 * @param left_noncommon_col_indices Output locations of non common left columns
 * in the final table output
 * @param left_common_cols Columns obtained by gathering common left
 * columns.
 * @param left_common_col_indices Output locations of common left columns in the
 * final table output
 * @param right_noncommon_cols Table obtained by gathering non common right
 * columns.
 *
 * @return Rearranged columns.
 */
std::vector<std::unique_ptr<column>> combine_join_columns(
  std::vector<std::unique_ptr<column>>&& left_noncommon_cols,
  std::vector<size_type> const& left_noncommon_col_indices,
  std::vector<std::unique_ptr<column>>&& left_common_cols,
  std::vector<size_type> const& left_common_col_indices,
  std::vector<std::unique_ptr<column>>&& right_noncommon_cols)
{
  std::vector<std::unique_ptr<column>> combined_cols(left_noncommon_cols.size() +
                                                     left_common_cols.size());
  for (size_t i = 0; i < left_noncommon_cols.size(); ++i) {
    combined_cols.at(left_noncommon_col_indices.at(i)) = std::move(left_noncommon_cols.at(i));
  }
  for (size_t i = 0; i < left_common_cols.size(); ++i) {
    combined_cols.at(left_common_col_indices.at(i)) = std::move(left_common_cols.at(i));
  }
  combined_cols.insert(combined_cols.end(),
                       std::make_move_iterator(right_noncommon_cols.begin()),
                       std::make_move_iterator(right_noncommon_cols.end()));
  return combined_cols;
}

/**
 * @brief  Gathers rows from `left` and `right` table and combines them into a
 * single table.
 *
 * @tparam JoinKind The type of join to be performed
 *
 * @param left Left input table
 * @param right Right input table
 * @param joined_indices Pair of vectors containing row indices from which
 * `left` and `right` tables are gathered. If any row index is out of bounds,
 * the contribution in the output `table` will be NULL.
 * @param columns_in_common is a vector of pairs of column indices
 * from tables `left` and `right` respectively, that are "in common".
 * For "common" columns, only a single output column will be produced.
 * For an inner or left join, the result will be gathered from the column in
 * `left`. For a full join, the result will be gathered from both common
 * columns in `left` and `right` and concatenated to form a single column.
 *
 * @return `table` containing the concatenation of rows from `left` and
 * `right` specified by `joined_indices`.
 * For any columns indicated by `columns_in_common`, only the corresponding
 * column in `left` will be included in the result. Final form would look like
 * `left(including common columns)+right(excluding common columns)`.
 */
template <join_kind JoinKind>
std::unique_ptr<table> construct_join_output_df(
  table_view const& left,
  table_view const& right,
  VectorPair& joined_indices,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  std::vector<size_type> left_common_col;
  left_common_col.reserve(columns_in_common.size());
  std::vector<size_type> right_common_col;
  right_common_col.reserve(columns_in_common.size());
  for (const auto& c : columns_in_common) {
    left_common_col.push_back(c.first);
    right_common_col.push_back(c.second);
  }
  std::vector<size_type> left_noncommon_col =
    non_common_column_indices(left.num_columns(), left_common_col);
  std::vector<size_type> right_noncommon_col =
    non_common_column_indices(right.num_columns(), right_common_col);

  bool const nullify_out_of_bounds{JoinKind != join_kind::INNER_JOIN};

  std::unique_ptr<table> common_table = std::make_unique<table>();
  // Construct the joined columns
  if (join_kind::FULL_JOIN == JoinKind) {
    auto complement_indices = get_left_join_indices_complement(
      joined_indices.second, left.num_rows(), right.num_rows(), stream);
    if (not columns_in_common.empty()) {
      auto common_from_right = detail::gather(right.select(right_common_col),
                                              complement_indices.second.begin(),
                                              complement_indices.second.end(),
                                              nullify_out_of_bounds,
                                              rmm::mr::get_default_resource(),
                                              stream);
      auto common_from_left  = detail::gather(left.select(left_common_col),
                                             joined_indices.first.begin(),
                                             joined_indices.first.end(),
                                             nullify_out_of_bounds,
                                             rmm::mr::get_default_resource(),
                                             stream);
      common_table           = cudf::detail::concatenate(
        {common_from_right->view(), common_from_left->view()}, mr, stream);
    }
    joined_indices = concatenate_vector_pairs(complement_indices, joined_indices);
  } else {
    if (not columns_in_common.empty()) {
      common_table = detail::gather(left.select(left_common_col),
                                    joined_indices.first.begin(),
                                    joined_indices.first.end(),
                                    nullify_out_of_bounds,
                                    mr,
                                    stream);
    }
  }

  // Construct the left non common columns
  std::unique_ptr<table> left_table = detail::gather(left.select(left_noncommon_col),
                                                     joined_indices.first.begin(),
                                                     joined_indices.first.end(),
                                                     nullify_out_of_bounds,
                                                     mr,
                                                     stream);

  std::unique_ptr<table> right_table = detail::gather(right.select(right_noncommon_col),
                                                      joined_indices.second.begin(),
                                                      joined_indices.second.end(),
                                                      nullify_out_of_bounds,
                                                      mr,
                                                      stream);

  return std::make_unique<table>(combine_join_columns(left_table->release(),
                                                      left_noncommon_col,
                                                      common_table->release(),
                                                      left_common_col,
                                                      right_table->release()));
}

/**
 * @brief  Performs join on the columns provided in `left` and `right` as per
 * the joining indices given in `left_on` and `right_on` and creates a single
 * table.
 *
 * @throw cudf::logic_error if `columns_in_common` contains a pair of indices
 * (`L`, `R`) where `L` does not exist in `left_on` or `R` does not exist in
 * `right_on`.
 * @throw cudf::logic_error if `columns_in_common` contains a pair of indices
 * (`L`, `R`) such that the location of `L` within `left_on` is not equal to
 * the location of `R` within `right_on`.
 * @throw cudf::logic_error if the number of elements in `left_on` or
 * `right_on` are not equal.
 * @throw cudf::logic_error if the number of columns in either `left` or
 * `right` table is 0 or exceeds MAX_JOIN_SIZE.
 * @throw std::out_of_range if elements of `left_on` or `right_on` exceed the
 * number of columns in the left or right table.
 *
 * @tparam JoinKind The type of join to be performed
 *
 * @param left The left table
 * @param right The right table
 * @param left_on The column's indices from `left` to join on.
 * Column `i` from `left_on` will be compared against column `i` of `right_on`.
 * @param right_on The column's indices from `right` to join on.
 * Column `i` from `right_on` will be compared with column `i` of `left_on`.
 * @param columns_in_common is a vector of pairs of column indices into
 * `left_on` and `right_on`, respectively, that are "in common". For "common"
 * columns, only a single output column will be produced, which is gathered
 * from `left_on` if it is left join or from intersection of `left_on` and
 * `right_on` if it is inner join or gathered from both `left_on` and
 * `right_on` if it is full join. Else, for every column in `left_on` and
 * `right_on`, an output column will be produced.
 * @param mr Device memory resource used to allocate the returned table's device memory
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Result of joining `left` and `right` tables on the columns specified
 * by `left_on` and `right_on`. The resulting table will be joined columns of
 * `left(including common columns)+right(excluding common columns)`.
 */
template <join_kind JoinKind>
std::unique_ptr<table> join_call_compute_df(
  table_view const& left,
  table_view const& right,
  std::vector<size_type> const& left_on,
  std::vector<size_type> const& right_on,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream = 0)
{
  CUDF_EXPECTS(0 != left.num_columns(), "Left table is empty");
  CUDF_EXPECTS(0 != right.num_columns(), "Right table is empty");
  CUDF_EXPECTS(left.num_rows() < MAX_JOIN_SIZE, "Left column size is too big");
  CUDF_EXPECTS(right.num_rows() < MAX_JOIN_SIZE, "Right column size is too big");

  CUDF_EXPECTS(left_on.size() == right_on.size(), "Mismatch in number of columns to be joined on");

  CUDF_EXPECTS(std::all_of(columns_in_common.begin(),
                           columns_in_common.end(),
                           [&left_on, &right_on](auto p) {
                             size_t lind =
                               std::find(left_on.begin(), left_on.end(), p.first) - left_on.begin();
                             size_t rind = std::find(right_on.begin(), right_on.end(), p.second) -
                                           right_on.begin();
                             return (lind != left_on.size()) && (rind != right_on.size()) &&
                                    (lind == rind);
                           }),
               "Invalid values passed to columns_in_common");

  if (is_trivial_join(left, right, left_on, right_on, JoinKind)) {
    return get_empty_joined_table(left, right, columns_in_common);
  }

  auto build             = right.select(right_on);
  auto probe             = left.select(left_on);
  bool flip_join_indices = false;

  // The `right` table is always used for building the hash map. We want to build the hash map
  // on the smaller table. Thus, if `left` is smaller than `right`, swap `left/right`.
  if (JoinKind == join_kind::INNER_JOIN && right.num_rows() > left.num_rows()) {
    std::swap(build, probe);
    flip_join_indices = true;
  }

  auto joined_indices = get_base_join_indices<JoinKind>(build, probe, flip_join_indices, stream);

  return construct_join_output_df<JoinKind>(
    probe, _build, joined_indices, columns_in_common, mr, stream);
}

}  // namespace detail

std::unique_ptr<table> inner_join(
  table_view const& left,
  table_view const& right,
  std::vector<size_type> const& left_on,
  std::vector<size_type> const& right_on,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_call_compute_df<::cudf::detail::join_kind::INNER_JOIN>(
    left, right, left_on, right_on, columns_in_common, mr);
}

std::unique_ptr<table> left_join(
  table_view const& left,
  table_view const& right,
  std::vector<size_type> const& left_on,
  std::vector<size_type> const& right_on,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_call_compute_df<::cudf::detail::join_kind::LEFT_JOIN>(
    left, right, left_on, right_on, columns_in_common, mr);
}

std::unique_ptr<table> full_join(
  table_view const& left,
  table_view const& right,
  std::vector<size_type> const& left_on,
  std::vector<size_type> const& right_on,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_call_compute_df<::cudf::detail::join_kind::FULL_JOIN>(
    left, right, left_on, right_on, columns_in_common, mr);
}

class hash_join_impl : public cudf::hash_join {
 public:
  hash_join_impl()                      = delete;
  hash_join_impl(hash_join_impl const&) = delete;
  hash_join_impl(hash_join_impl&&)      = delete;
  hash_join_impl& operator=(hash_join_impl const&) = delete;
  hash_join_impl& operator=(hash_join_impl&&) = delete;

 private:
  table_view _build, _build_selected;
  std::vector<size_type> _build_on;
  std::unique_ptr<table_device_view> _build_table;
  std::unique_ptr<multimap_type> _hash_table;

 public:
  explicit hash_join_impl(cudf::table_view const& build, std::vector<size_type> const& build_on)
    : _build(build),
      _build_selected(build.select(build_on)),
      _build_on(build_on),
      _hash_table(nullptr)
  {
    CUDF_EXPECTS(0 != _build.num_columns(), "Hash join build table is empty");
    CUDF_EXPECTS(_build.num_rows() < MAX_JOIN_SIZE,
                 "Hash join build column size is too big for hash join");

    if (0 == _build_on.size() || 0 == build.num_rows()) { return; }

    _build_table = table_device_view::create(_build_selected, 0);
    _hash_table  = build_join_hash_table(_build_table, 0)
  }

  ~hash_join_impl() {}

  std::unique_ptr<cudf::table> inner_join(
    cudf::table_view const& probe,
    std::vector<size_type> const& probe_on,
    std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
    rmm::mr::device_memory_resource* mr) override
  {
    CUDF_FUNC_RANGE();
    return compute_hash_join<join_kind::INNER_JOIN>(probe, probe_on, columns_in_common, mr);
  }

  std::unique_ptr<cudf::table> left_join(
    cudf::table_view const& probe,
    std::vector<size_type> const& probe_on,
    std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
    rmm::mr::device_memory_resource* mr) override
  {
    CUDF_FUNC_RANGE();
    return compute_hash_join<join_kind::LEFT_JOIN>(probe, probe_on, columns_in_common, mr);
  }

  std::unique_ptr<cudf::table> full_join(
    cudf::table_view const& probe,
    std::vector<size_type> const& probe_on,
    std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
    rmm::mr::device_memory_resource* mr) override
  {
    CUDF_FUNC_RANGE();
    return compute_hash_join<join_kind::LEFT_JOIN>(probe, probe_on, columns_in_common, mr);
  }

 private:
  template <join_kind JoinKind>
  std::enable_if_t<JoinKind != join_kind::FULL_JOIN, std::unique_ptr<table>> compute_hash_join(
    cudf::table_view const& probe,
    std::vector<size_type> const& probe_on,
    std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream = 0)
  {
    CUDF_EXPECTS(0 != probe.num_columns(), "Hash join probe table is empty");
    CUDF_EXPECTS(probe.num_rows() < MAX_JOIN_SIZE,
                 "Hash join probe column size is too big for hash join");
    CUDF_EXPECTS(_build_on.size() == probe_on.size(),
                 "Mismatch in number of columns to be joined on");

    CUDF_EXPECTS(std::all_of(columns_in_common.begin(),
                             columns_in_common.end(),
                             [&_build_on, &probe_on](auto p) {
                               size_t b = std::find(_build_on.begin(), _build_on.end(), p.first) -
                                          _build_on.begin();
                               size_t p = std::find(probe_on.begin(), probe_on.end(), p.second) -
                                          probe_on.begin();
                               return (b != _build_on.size()) && (p != probe_on.size()) && (b == p);
                             }),
                 "Invalid values passed to columns_in_common");

    if (is_trivial_join(probe, _build, probe_on, _build_on, JoinKind)) {
      return get_empty_joined_table(probe, _build, columns_in_common);
    }

    CUDF_EXPECTS(std::equal(std::cbegin(_build),
                            std::cend(_build),
                            std::cbegin(probe),
                            std::cend(probe),
                            [](const auto& b, const auto& p) { return b.type() == p.type(); }),
                 "Mismatch in joining column data types");

    auto joined_indices = probe_join_indices<JoinKind>(probe.select(probe_on), stream);

    return construct_join_output_df<JoinKind>(
      probe, right, joined_indices, columns_in_common, mr, stream);
  }

  template <join_kind JoinKind>
  std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>> probe_join_indices(
    cudf::table_view const& probe, hipStream_t stream)
  {
    // Trivial left join case - exit early
    if (!_hash_table && BaseJoinKind == join_kind::LEFT_JOIN) {
      return get_trivial_left_join_indices(probe, flip_join_indices, stream);
    }

    CUDF_EXPECTS(_hash_table, "Hash table of hash join is null.");

    auto probe_table = table_device_view::create(probe, stream);
    return probe_join_hash_table(*_build_table, *probe_table, *_hash_table, false, stream);
  }
};

std::unique_ptr<hash_join> hash_join::create(cudf::table_view const& build_table,
                                             std::vector<size_type> const& build_on)
{
  return std::make_unique<hash_join_impl>(build_table, build_on);
}

}  // namespace cudf

/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/copying.hpp>
#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>

#include "hash_join.cuh"

namespace cudf {
namespace detail {

/**
 * @brief Returns a vector with non-common indices which is set difference
 * between `[0, num_columns)` and index values in common_column_indices
 *
 * @param num_columns The number of columns, which represents column indices
 * from `[0, num_columns)` in a table
 * @param common_column_indices A vector of common indices which needs to be
 * excluded from `[0, num_columns)`
 *
 * @return vector A vector containing only the indices which are not present in
 * `common_column_indices`
 */
auto non_common_column_indices(size_type num_columns,
                               std::vector<size_type> const &common_column_indices)
{
  CUDF_EXPECTS(common_column_indices.size() <= static_cast<unsigned long>(num_columns),
               "Too many columns in common");
  std::vector<size_type> all_column_indices(num_columns);
  std::iota(std::begin(all_column_indices), std::end(all_column_indices), 0);
  std::vector<size_type> sorted_common_column_indices{common_column_indices};
  std::sort(std::begin(sorted_common_column_indices), std::end(sorted_common_column_indices));
  std::vector<size_type> non_common_column_indices(num_columns - common_column_indices.size());
  std::set_difference(std::cbegin(all_column_indices),
                      std::cend(all_column_indices),
                      std::cbegin(sorted_common_column_indices),
                      std::cend(sorted_common_column_indices),
                      std::begin(non_common_column_indices));
  return non_common_column_indices;
}

std::unique_ptr<table> get_empty_joined_table(
  table_view const &left,
  table_view const &right,
  std::vector<std::pair<size_type, size_type>> const &columns_in_common)
{
  std::vector<size_type> right_columns_in_common(columns_in_common.size());
  std::transform(columns_in_common.begin(),
                 columns_in_common.end(),
                 right_columns_in_common.begin(),
                 [](auto &col) { return col.second; });
  std::unique_ptr<table> empty_left  = empty_like(left);
  std::unique_ptr<table> empty_right = empty_like(right);
  std::vector<size_type> right_non_common_indices =
    non_common_column_indices(right.num_columns(), right_columns_in_common);
  table_view tmp_right_table = (*empty_right).select(right_non_common_indices);
  table_view tmp_table{{*empty_left, tmp_right_table}};
  return std::make_unique<table>(tmp_table);
}

VectorPair concatenate_vector_pairs(VectorPair &a, VectorPair &b)
{
  CUDF_EXPECTS((a.first.size() == a.second.size()),
               "Mismatch between sizes of vectors in vector pair");
  CUDF_EXPECTS((b.first.size() == b.second.size()),
               "Mismatch between sizes of vectors in vector pair");
  if (a.first.size() == 0) {
    return b;
  } else if (b.first.size() == 0) {
    return a;
  }
  auto original_size = a.first.size();
  a.first.resize(a.first.size() + b.first.size());
  a.second.resize(a.second.size() + b.second.size());
  thrust::copy(b.first.begin(), b.first.end(), a.first.begin() + original_size);
  thrust::copy(b.second.begin(), b.second.end(), a.second.begin() + original_size);
  return a;
}

template <typename T>
struct valid_range {
  T start, stop;
  __host__ __device__ valid_range(const T begin, const T end) : start(begin), stop(end) {}

  __host__ __device__ __forceinline__ bool operator()(const T index)
  {
    return ((index >= start) && (index < stop));
  }
};

/**
 * @brief  Creates a table containing the complement of left join indices.
 * This table has two columns. The first one is filled with JoinNoneValue(-1)
 * and the second one contains values from 0 to right_table_row_count - 1
 * excluding those found in the right_indices column.
 *
 * @param right_indices Vector of indices
 * @param left_table_row_count Number of rows of left table
 * @param right_table_row_count Number of rows of right table
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Pair of vectors containing the left join indices complement
 */
std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>>
get_left_join_indices_complement(rmm::device_vector<size_type> &right_indices,
                                 size_type left_table_row_count,
                                 size_type right_table_row_count,
                                 hipStream_t stream)
{
  // Get array of indices that do not appear in right_indices

  // Vector allocated for unmatched result
  rmm::device_vector<size_type> right_indices_complement(right_table_row_count);

  // If left table is empty in a full join call then all rows of the right table
  // should be represented in the joined indices. This is an optimization since
  // if left table is empty and full join is called all the elements in
  // right_indices will be JoinNoneValue, i.e. -1. This if path should
  // produce exactly the same result as the else path but will be faster.
  if (left_table_row_count == 0) {
    thrust::sequence(rmm::exec_policy(stream)->on(stream),
                     right_indices_complement.begin(),
                     right_indices_complement.end(),
                     0);
  } else {
    // Assume all the indices in invalid_index_map are invalid
    rmm::device_vector<size_type> invalid_index_map(right_table_row_count, 1);
    // Functor to check for index validity since left joins can create invalid indices
    valid_range<size_type> valid(0, right_table_row_count);

    // invalid_index_map[index_ptr[i]] = 0 for i = 0 to right_table_row_count
    // Thus specifying that those locations are valid
    thrust::scatter_if(rmm::exec_policy(stream)->on(stream),
                       thrust::make_constant_iterator(0),
                       thrust::make_constant_iterator(0) + right_indices.size(),
                       right_indices.begin(),      // Index locations
                       right_indices.begin(),      // Stencil - Check if index location is valid
                       invalid_index_map.begin(),  // Output indices
                       valid);                     // Stencil Predicate
    size_type begin_counter = static_cast<size_type>(0);
    size_type end_counter   = static_cast<size_type>(right_table_row_count);

    // Create list of indices that have been marked as invalid
    size_type indices_count = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
                                              thrust::make_counting_iterator(begin_counter),
                                              thrust::make_counting_iterator(end_counter),
                                              invalid_index_map.begin(),
                                              right_indices_complement.begin(),
                                              thrust::identity<size_type>()) -
                              right_indices_complement.begin();
    right_indices_complement.resize(indices_count);
  }

  rmm::device_vector<size_type> left_invalid_indices(right_indices_complement.size(),
                                                     JoinNoneValue);

  return std::make_pair(std::move(left_invalid_indices), std::move(right_indices_complement));
}

/**
 * @brief Builds the hash table based on the given `build_table`.
 *
 * @throw cudf::logic_error if the number of columns in `build` table is 0.
 * @throw cudf::logic_error if the number of rows in `build` table is 0.
 * @throw cudf::logic_error if insertion to the hash table fails.
 * @throw std::out_of_range if elements of `build_on` exceed the number of columns in the `build`
 * table.
 *
 * @param build_table Table of build side columns to join.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Built hash table.
 */
std::unique_ptr<multimap_type, std::function<void(multimap_type *)>> build_join_hash_table(
  cudf::table_device_view build_table, hipStream_t stream)
{
  CUDF_EXPECTS(0 != build_table.num_columns(), "Selected build dataset is empty");
  CUDF_EXPECTS(0 != build_table.num_rows(), "Build side table has no rows");

  const size_type build_table_num_rows{build_table.num_rows()};
  size_t const hash_table_size = compute_hash_table_size(build_table_num_rows);

  auto hash_table = multimap_type::create(hash_table_size,
                                          true,
                                          multimap_type::hasher(),
                                          multimap_type::key_equal(),
                                          multimap_type::allocator_type(),
                                          stream);

  row_hash hash_build{build_table};
  rmm::device_scalar<int> failure(0, 0);
  constexpr int block_size{DEFAULT_JOIN_BLOCK_SIZE};
  detail::grid_1d config(build_table_num_rows, block_size);
  build_hash_table<<<config.num_blocks, config.num_threads_per_block, 0, 0>>>(
    *hash_table, hash_build, build_table_num_rows, failure.data());
  // Check error code from the kernel
  if (failure.value() == 1) { CUDF_FAIL("Hash Table insert failure."); }

  return hash_table;
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table`,
 * and returns the output indices of `build_table` and `probe_table` as a combined table.
 *
 * @tparam JoinKind The type of join to be performed.
 *
 * @param build_table Table of build side columns to join.
 * @param probe_table Table of probe side columns to join.
 * @param hash_table Hash table built from `build_table`.
 * @param flip_join_indices Flag that indicates whether the left (probe) and right (build)
 * tables have been flipped, meaning the output indices should also be flipped.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Join output indices vector pair.
 */
template <join_kind JoinKind>
std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>> probe_join_hash_table(
  cudf::table_device_view build_table,
  cudf::table_device_view probe_table,
  multimap_type const &hash_table,
  bool flip_join_indices,
  null_equality compare_nulls,
  hipStream_t stream)
{
  size_type estimated_size = estimate_join_output_size<JoinKind, multimap_type>(
    build_table, probe_table, hash_table, compare_nulls, stream);

  // If the estimated output size is zero, return immediately
  if (estimated_size == 0) {
    return std::make_pair(rmm::device_vector<size_type>{}, rmm::device_vector<size_type>{});
  }

  // Because we are approximating the number of joined elements, our approximation
  // might be incorrect and we might have underestimated the number of joined elements.
  // As such we will need to de-allocate memory and re-allocate memory to ensure
  // that the final output is correct.
  rmm::device_scalar<size_type> write_index(0, stream);
  size_type join_size{0};

  rmm::device_vector<size_type> left_indices;
  rmm::device_vector<size_type> right_indices;
  auto current_estimated_size = estimated_size;
  do {
    left_indices.resize(estimated_size);
    right_indices.resize(estimated_size);

    constexpr int block_size{DEFAULT_JOIN_BLOCK_SIZE};
    detail::grid_1d config(probe_table.num_rows(), block_size);
    write_index.set_value(0);

    row_hash hash_probe{probe_table};
    row_equality equality{probe_table, build_table, compare_nulls == null_equality::EQUAL};
    const auto &join_output_l =
      flip_join_indices ? right_indices.data().get() : left_indices.data().get();
    const auto &join_output_r =
      flip_join_indices ? left_indices.data().get() : right_indices.data().get();
    probe_hash_table<JoinKind, multimap_type, block_size, DEFAULT_JOIN_CACHE_SIZE>
      <<<config.num_blocks, config.num_threads_per_block, 0, stream>>>(hash_table,
                                                                       build_table,
                                                                       probe_table,
                                                                       hash_probe,
                                                                       equality,
                                                                       join_output_l,
                                                                       join_output_r,
                                                                       write_index.data(),
                                                                       estimated_size);

    CHECK_CUDA(stream);

    join_size              = write_index.value();
    current_estimated_size = estimated_size;
    estimated_size *= 2;
  } while ((current_estimated_size < join_size));

  left_indices.resize(join_size);
  right_indices.resize(join_size);
  return std::make_pair(std::move(left_indices), std::move(right_indices));
}

/**
 * @brief  Combines the non common left, common left and non common right
 * columns in the correct order to form the join output table.
 *
 * @param left_noncommon_cols Columns obtained by gathering non common left
 * columns.
 * @param left_noncommon_col_indices Output locations of non common left columns
 * in the final table output
 * @param left_common_cols Columns obtained by gathering common left
 * columns.
 * @param left_common_col_indices Output locations of common left columns in the
 * final table output
 * @param right_noncommon_cols Table obtained by gathering non common right
 * columns.
 *
 * @return Rearranged columns.
 */
std::vector<std::unique_ptr<column>> combine_join_columns(
  std::vector<std::unique_ptr<column>> &&left_noncommon_cols,
  std::vector<size_type> const &left_noncommon_col_indices,
  std::vector<std::unique_ptr<column>> &&left_common_cols,
  std::vector<size_type> const &left_common_col_indices,
  std::vector<std::unique_ptr<column>> &&right_noncommon_cols)
{
  std::vector<std::unique_ptr<column>> combined_cols(left_noncommon_cols.size() +
                                                     left_common_cols.size());
  for (size_t i = 0; i < left_noncommon_cols.size(); ++i) {
    combined_cols.at(left_noncommon_col_indices.at(i)) = std::move(left_noncommon_cols.at(i));
  }
  for (size_t i = 0; i < left_common_cols.size(); ++i) {
    combined_cols.at(left_common_col_indices.at(i)) = std::move(left_common_cols.at(i));
  }
  combined_cols.insert(combined_cols.end(),
                       std::make_move_iterator(right_noncommon_cols.begin()),
                       std::make_move_iterator(right_noncommon_cols.end()));
  return combined_cols;
}

/**
 * @brief  Gathers rows from `left` and `right` table and combines them into a
 * single table.
 *
 * @tparam JoinKind The type of join to be performed
 *
 * @param left Left input table
 * @param right Right input table
 * @param joined_indices Pair of vectors containing row indices from which
 * `left` and `right` tables are gathered. If any row index is out of bounds,
 * the contribution in the output `table` will be NULL.
 * @param columns_in_common is a vector of pairs of column indices
 * from tables `left` and `right` respectively, that are "in common".
 * For "common" columns, only a single output column will be produced.
 * For an inner or left join, the result will be gathered from the column in
 * `left`. For a full join, the result will be gathered from both common
 * columns in `left` and `right` and concatenated to form a single column.
 *
 * @return `table` containing the concatenation of rows from `left` and
 * `right` specified by `joined_indices`.
 * For any columns indicated by `columns_in_common`, only the corresponding
 * column in `left` will be included in the result. Final form would look like
 * `left(including common columns)+right(excluding common columns)`.
 */
template <join_kind JoinKind>
std::unique_ptr<table> construct_join_output_df(
  table_view const &left,
  table_view const &right,
  VectorPair &joined_indices,
  std::vector<std::pair<size_type, size_type>> const &columns_in_common,
  rmm::mr::device_memory_resource *mr,
  hipStream_t stream)
{
  std::vector<size_type> left_common_col;
  left_common_col.reserve(columns_in_common.size());
  std::vector<size_type> right_common_col;
  right_common_col.reserve(columns_in_common.size());
  for (const auto &c : columns_in_common) {
    left_common_col.push_back(c.first);
    right_common_col.push_back(c.second);
  }
  std::vector<size_type> left_noncommon_col =
    non_common_column_indices(left.num_columns(), left_common_col);
  std::vector<size_type> right_noncommon_col =
    non_common_column_indices(right.num_columns(), right_common_col);

  bool const nullify_out_of_bounds{JoinKind != join_kind::INNER_JOIN};

  std::unique_ptr<table> common_table = std::make_unique<table>();
  // Construct the joined columns
  if (join_kind::FULL_JOIN == JoinKind) {
    auto complement_indices = get_left_join_indices_complement(
      joined_indices.second, left.num_rows(), right.num_rows(), stream);
    if (not columns_in_common.empty()) {
      auto common_from_right = detail::gather(right.select(right_common_col),
                                              complement_indices.second.begin(),
                                              complement_indices.second.end(),
                                              nullify_out_of_bounds,
                                              rmm::mr::get_default_resource(),
                                              stream);
      auto common_from_left  = detail::gather(left.select(left_common_col),
                                             joined_indices.first.begin(),
                                             joined_indices.first.end(),
                                             nullify_out_of_bounds,
                                             rmm::mr::get_default_resource(),
                                             stream);
      common_table           = cudf::detail::concatenate(
        {common_from_right->view(), common_from_left->view()}, mr, stream);
    }
    joined_indices = concatenate_vector_pairs(complement_indices, joined_indices);
  } else {
    if (not columns_in_common.empty()) {
      common_table = detail::gather(left.select(left_common_col),
                                    joined_indices.first.begin(),
                                    joined_indices.first.end(),
                                    nullify_out_of_bounds,
                                    mr,
                                    stream);
    }
  }

  // Construct the left non common columns
  std::unique_ptr<table> left_table = detail::gather(left.select(left_noncommon_col),
                                                     joined_indices.first.begin(),
                                                     joined_indices.first.end(),
                                                     nullify_out_of_bounds,
                                                     mr,
                                                     stream);

  std::unique_ptr<table> right_table = detail::gather(right.select(right_noncommon_col),
                                                      joined_indices.second.begin(),
                                                      joined_indices.second.end(),
                                                      nullify_out_of_bounds,
                                                      mr,
                                                      stream);

  return std::make_unique<table>(combine_join_columns(left_table->release(),
                                                      left_noncommon_col,
                                                      common_table->release(),
                                                      left_common_col,
                                                      right_table->release()));
}

hash_join_impl::hash_join_impl(cudf::table_view const &build,
                               std::vector<size_type> const &build_on)
  : _build(build),
    _build_selected(build.select(build_on)),
    _build_on(build_on),
    _hash_table(nullptr)
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(0 != _build.num_columns(), "Hash join build table is empty");
  CUDF_EXPECTS(_build.num_rows() < MAX_JOIN_SIZE, "Build column size is too big for hash join");

  if (_build_on.empty() || 0 == build.num_rows()) { return; }

  auto build_table = cudf::table_device_view::create(_build_selected);
  _hash_table      = build_join_hash_table(*build_table, 0);
}

std::unique_ptr<cudf::table> hash_join_impl::inner_join(
  cudf::table_view const &probe,
  std::vector<size_type> const &probe_on,
  std::vector<std::pair<cudf::size_type, cudf::size_type>> const &columns_in_common,
  cudf::hash_join::probe_output_side probe_output_side,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource *mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<join_kind::INNER_JOIN>(
    probe, probe_on, columns_in_common, probe_output_side, compare_nulls, mr);
}

std::unique_ptr<cudf::table> hash_join_impl::left_join(
  cudf::table_view const &probe,
  std::vector<size_type> const &probe_on,
  std::vector<std::pair<cudf::size_type, cudf::size_type>> const &columns_in_common,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource *mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<join_kind::LEFT_JOIN>(probe,
                                                 probe_on,
                                                 columns_in_common,
                                                 cudf::hash_join::probe_output_side::LEFT,
                                                 compare_nulls,
                                                 mr);
}

std::unique_ptr<cudf::table> hash_join_impl::full_join(
  cudf::table_view const &probe,
  std::vector<size_type> const &probe_on,
  std::vector<std::pair<cudf::size_type, cudf::size_type>> const &columns_in_common,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource *mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<join_kind::FULL_JOIN>(probe,
                                                 probe_on,
                                                 columns_in_common,
                                                 cudf::hash_join::probe_output_side::LEFT,
                                                 compare_nulls,
                                                 mr);
}

template <join_kind JoinKind>
std::unique_ptr<table> hash_join_impl::compute_hash_join(
  cudf::table_view const &probe,
  std::vector<size_type> const &probe_on,
  std::vector<std::pair<cudf::size_type, cudf::size_type>> const &columns_in_common,
  cudf::hash_join::probe_output_side probe_output_side,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource *mr,
  hipStream_t stream) const
{
  CUDF_EXPECTS(0 != probe.num_columns(), "Hash join probe table is empty");
  CUDF_EXPECTS(probe.num_rows() < MAX_JOIN_SIZE, "Probe column size is too big for hash join");
  CUDF_EXPECTS(_build_on.size() == probe_on.size(),
               "Mismatch in number of columns to be joined on");

  CUDF_EXPECTS(std::all_of(columns_in_common.begin(),
                           columns_in_common.end(),
                           [this, &probe_on](auto pair) {
                             size_t b = std::find(_build_on.begin(), _build_on.end(), pair.first) -
                                        _build_on.begin();
                             size_t p = std::find(probe_on.begin(), probe_on.end(), pair.second) -
                                        probe_on.begin();
                             return (b != _build_on.size()) && (p != probe_on.size()) && (b == p);
                           }),
               "Invalid values passed to columns_in_common");

  if (is_trivial_join(probe, _build, probe_on, _build_on, JoinKind)) {
    return get_empty_joined_table(probe, _build, columns_in_common);
  }

  auto probe_selected = probe.select(probe_on);
  CUDF_EXPECTS(std::equal(std::cbegin(_build_selected),
                          std::cend(_build_selected),
                          std::cbegin(probe_selected),
                          std::cend(probe_selected),
                          [](const auto &b, const auto &p) { return b.type() == p.type(); }),
               "Mismatch in joining column data types");

  bool probe_output_left{probe_output_side == cudf::hash_join::probe_output_side::LEFT};

  constexpr join_kind ProbeJoinKind =
    (JoinKind == join_kind::FULL_JOIN) ? join_kind::LEFT_JOIN : JoinKind;
  auto joined_indices =
    probe_join_indices<ProbeJoinKind>(probe_selected, !probe_output_left, compare_nulls, stream);
  auto actual_columns_in_common = columns_in_common;
  if (!probe_output_left) {
    std::for_each(actual_columns_in_common.begin(), actual_columns_in_common.end(), [](auto &pair) {
      std::swap(pair.first, pair.second);
    });
  }
  return construct_join_output_df<JoinKind>(probe_output_left ? probe : _build,
                                            probe_output_left ? _build : probe,
                                            joined_indices,
                                            actual_columns_in_common,
                                            mr,
                                            stream);
}

template <join_kind JoinKind>
std::enable_if_t<JoinKind != join_kind::FULL_JOIN,
                 std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>>>
hash_join_impl::probe_join_indices(cudf::table_view const &probe,
                                   bool flip_join_indices,
                                   null_equality compare_nulls,
                                   hipStream_t stream) const
{
  // Trivial left join case - exit early
  if (!_hash_table && JoinKind == join_kind::LEFT_JOIN) {
    return get_trivial_left_join_indices(probe, flip_join_indices, stream);
  }

  CUDF_EXPECTS(_hash_table, "Hash table of hash join is null.");

  auto build_table = cudf::table_device_view::create(_build_selected, stream);
  auto probe_table = cudf::table_device_view::create(probe, stream);
  return probe_join_hash_table<JoinKind>(
    *build_table, *probe_table, *_hash_table, flip_join_indices, compare_nulls, stream);
}

}  // namespace detail
}  // namespace cudf

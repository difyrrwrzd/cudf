/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "groupby.hpp"

#include <copying/scatter.hpp>
#include <table/legacy/device_table.cuh>
#include <table/legacy/device_table_row_operators.cuh>
#include <bitmask/legacy/bit_mask.cuh>
#include <utilities/column_utils.hpp>
#include <utilities/cuda_utils.hpp>

#include <cudf/copying.hpp>

#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/unique.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <algorithm>
#include <tuple>
#include <numeric>

namespace {

template <bool nullable = true>
struct transform_row_eq_comparator {
  row_equality_comparator<nullable> cmp;
  gdf_size_type const* sorted_order;

  CUDA_DEVICE_CALLABLE
  bool operator() (gdf_size_type lhs, gdf_size_type rhs) {
    return cmp(sorted_order[lhs], sorted_order[rhs]);
  }
};

struct permutation_label_setter {
  gdf_size_type * group_labels_ptr;
  gdf_size_type const* group_ids_ptr;

  CUDA_DEVICE_CALLABLE
  void operator() (gdf_size_type i) { 
    group_labels_ptr[group_ids_ptr[i]] = 1;
  }
};

} // namespace anonymous


namespace cudf {

namespace detail {

gdf_column const& groupby::key_sort_order() {
  if (_key_sorted_order)
    return *_key_sorted_order;

  _key_sorted_order = std::make_unique<gdf_column>(
    allocate_column(gdf_dtype_of<gdf_index_type>(),
                    _key_table.num_rows(),
                    false));

  if (_include_nulls ||
      !cudf::has_nulls(_key_table)) {  // SQL style
    gdf_context context{};
    context.flag_groupby_include_nulls = true;
    CUDF_TRY(gdf_order_by(_key_table.begin(), nullptr,
                          _key_table.num_columns(), _key_sorted_order.get(),
                          &context));
  } else {  // Pandas style
    auto key_cols_bitmask = row_bitmask(_key_table);

    gdf_column modified_first_key_col = *(_key_table.get_column(0));
    modified_first_key_col.valid =
        reinterpret_cast<gdf_valid_type*>(key_cols_bitmask.data().get());

    auto keys = const_cast<cudf::table*>(&_key_table);
    std::vector<gdf_column*> modified_key_cols_vect(keys->begin(), keys->end());
    modified_key_cols_vect[0] = &modified_first_key_col;
    cudf::table modified_key_col_table(modified_key_cols_vect.data(),
                                      modified_key_cols_vect.size());

    gdf_context temp_ctx;
    temp_ctx.flag_null_sort_behavior = GDF_NULL_AS_LARGEST;

    CUDF_TRY(gdf_order_by(modified_key_col_table.begin(), nullptr,
                          modified_key_col_table.num_columns(),
                          _key_sorted_order.get(), &temp_ctx));

    CUDF_TRY(gdf_count_nonzero_mask(
        reinterpret_cast<gdf_valid_type*>(key_cols_bitmask.data().get()),
        _key_table.num_rows(),
        &_num_keys));
  }

  return *_key_sorted_order;
}

rmm::device_vector<gdf_size_type> const& groupby::group_indices() {
  if (_group_ids)
    return *_group_ids;

  index_vector idx_data(_num_keys);

  auto counting_iter = thrust::make_counting_iterator<gdf_size_type>(0);
  auto device_input_table = device_table::create(_key_table);
  bool nullable = device_input_table.get()->has_nulls();
  auto sorted_order = reinterpret_cast<gdf_size_type*>(key_sort_order().data);
  decltype(idx_data.begin()) result_end;

  if (nullable) {
    auto comp = row_equality_comparator<true>(*device_input_table, true);
    result_end = thrust::unique_copy(
      thrust::device, counting_iter, counting_iter + _num_keys,
      idx_data.begin(), transform_row_eq_comparator<true>{comp, sorted_order});
  } else {
    auto comp = row_equality_comparator<false>(*device_input_table, true);
    result_end = thrust::unique_copy(
      thrust::device, counting_iter, counting_iter + _num_keys,
      idx_data.begin(), transform_row_eq_comparator<false>{comp, sorted_order});
  }

  gdf_size_type num_groups = thrust::distance(idx_data.begin(), result_end);
  _group_ids = std::make_unique<index_vector>(idx_data.begin(), idx_data.begin() + num_groups);

  return *_group_ids;
}

rmm::device_vector<gdf_size_type> const& groupby::group_labels() {
  if (_group_labels)
    return *_group_labels;

  // Get group labels for future use in segmented sorting
  _group_labels = std::make_unique<index_vector>(_num_keys);

  auto& group_labels = *_group_labels;
  thrust::fill(group_labels.begin(), group_labels.end(), 0);
  auto group_labels_ptr = group_labels.data().get();
  auto group_ids_ptr = group_indices().data().get();
  thrust::for_each_n(thrust::make_counting_iterator(1),
                    group_indices().size() - 1,
                    permutation_label_setter{group_labels_ptr, group_ids_ptr});
  thrust::inclusive_scan(thrust::device,
                        group_labels.begin(),
                        group_labels.end(),
                        group_labels.begin());

  return group_labels;
}

gdf_column const& groupby::unsorted_labels() {
  if (_unsorted_labels)
    return *_unsorted_labels;

  _unsorted_labels = std::make_unique<gdf_column>(
    allocate_column(gdf_dtype_of<gdf_size_type>(),
                    key_sort_order().size));
  auto& unsorted_labels = *_unsorted_labels;
  hipMemset(unsorted_labels.valid, 0,
              gdf_num_bitmask_elements(unsorted_labels.size));
  
  gdf_column group_labels_col{};
  gdf_column_view(&group_labels_col, 
                  const_cast<gdf_size_type*>(group_labels().data().get()), 
                  nullptr,
                  group_labels().size(), 
                  gdf_dtype_of<gdf_size_type>());
  cudf::table t_sorted_labels{&group_labels_col};
  cudf::table t_unsorted_labels{&unsorted_labels};
  cudf::detail::scatter(&t_sorted_labels,
                        reinterpret_cast<gdf_size_type*>(key_sort_order().data),
                        &t_unsorted_labels);
  return unsorted_labels;
}


std::pair<gdf_column, rmm::device_vector<gdf_size_type> >
groupby::sort_values(gdf_column const& val_col) {
  auto idx_col = allocate_column(gdf_dtype_of<gdf_index_type>(),
                                _key_table.num_rows(),
                                false);

  // We need a table constructor that can take const initializer list
  auto unsorted_val_col = const_cast<gdf_column*> (&val_col);
  auto unsorted_label_col = const_cast<gdf_column*> (&unsorted_labels());
  auto unsorted_table = cudf::table{unsorted_label_col, unsorted_val_col};

  gdf_context context{};
  context.flag_groupby_include_nulls = _include_nulls;
  gdf_order_by(unsorted_table.begin(),
              nullptr,
              unsorted_table.num_columns(), // always 2
              &idx_col,
              const_cast<gdf_context*>(&context));

  cudf::table unsorted_val_col_table{unsorted_val_col};
  auto sorted_val_col = allocate_like(val_col, _num_keys);
  cudf::table sorted_val_col_table{&sorted_val_col};
  cudf::gather(&unsorted_val_col_table,
              reinterpret_cast<gdf_size_type*>(idx_col.data),
              &sorted_val_col_table);
  gdf_column_free(&idx_col);

  // Get number of valid values in each group
  rmm::device_vector<gdf_size_type> val_group_sizes(group_indices().size());
  rmm::device_vector<gdf_size_type> d_bools(sorted_val_col.size);
  if ( is_nullable(sorted_val_col) ) {
    auto col_valid = reinterpret_cast<bit_mask::bit_mask_t*>(sorted_val_col.valid);

    thrust::transform(
      thrust::make_counting_iterator(static_cast<gdf_size_type>(0)),
      thrust::make_counting_iterator(sorted_val_col.size), d_bools.begin(),
      [col_valid] __device__ (gdf_size_type i) { return bit_mask::is_valid(col_valid, i); });
  } else {
    thrust::fill(d_bools.begin(), d_bools.end(), 1);
  }

  thrust::reduce_by_key(thrust::device,
                        group_labels().begin(),
                        group_labels().end(),
                        d_bools.begin(),
                        thrust::make_discard_iterator(),
                        val_group_sizes.begin());

  return std::make_pair(sorted_val_col, val_group_sizes);
}

cudf::table groupby::unique_keys() {
  auto uniq_key_table = cudf::allocate_like(_key_table, (gdf_size_type)group_indices().size());
  auto idx_data = reinterpret_cast<gdf_size_type*>(key_sort_order().data);
  auto transformed_group_ids = index_vector(group_indices().size());

  util::cuda::scoped_stream stream;
  auto exec = rmm::exec_policy(stream)->on(stream);

  thrust::transform(exec, group_indices().begin(), group_indices().end(),
                    transformed_group_ids.begin(),
    [=] __device__ (gdf_size_type i) { return idx_data[i]; } );
  hipStreamSynchronize(stream);
  
  cudf::gather(&_key_table,
              transformed_group_ids.data().get(),
              &uniq_key_table);
  return uniq_key_table;
}


} // namespace detail
  
} // namespace cudf

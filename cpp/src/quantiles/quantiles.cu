#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <memory>
#include <cudf/copying.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/types.hpp>
#include <quantiles/quantiles_util.hpp>
#include "cudf/utilities/traits.hpp"
#include "thrust/functional.h"
#include "thrust/iterator/counting_iterator.h"

using ScalarResult = double;

namespace cudf {
namespace experimental {
namespace detail {
namespace {

enum class extrema {
    min,
    max
};

template<extrema minmax, bool is_nullable>
size_type extrema(column_view const & in,
                  order order,
                  null_order null_order,
                  hipStream_t stream)
{
    std::vector<cudf::order> h_order{ order };
    std::vector<cudf::null_order> h_null_order{ null_order };
    rmm::device_vector<cudf::order> d_order( h_order );
    rmm::device_vector<cudf::null_order> d_null_order( h_null_order );
    table_view in_table({ in });
    auto in_table_d = table_device_view::create(in_table);
    auto it = thrust::make_counting_iterator<size_type>(0);

    auto comparator = row_lexicographic_comparator<is_nullable>(
        *in_table_d,
        *in_table_d,
        d_order.data().get(),
        d_null_order.data().get());

    auto extrema_id = minmax == extrema::min
        ? thrust::min_element(rmm::exec_policy(stream)->on(stream), it, it + in.size(), comparator)
        : thrust::max_element(rmm::exec_policy(stream)->on(stream), it, it + in.size(), comparator);

    return *extrema_id;
}

template<typename T>
std::unique_ptr<scalar>
pick(column_view const& in, size_type index) {
    auto result = get_array_value<ScalarResult>(in.begin<T>(), index);
    return std::make_unique<numeric_scalar<ScalarResult>>(result);
}

struct pick_functor
{
    template<typename T>
    typename std::enable_if_t<not std::is_arithmetic<T>::value, std::unique_ptr<scalar>>
    operator()(column_view const& in, size_type location) {
        CUDF_FAIL("non-arithmetic types are unsupported");
    }

    template<typename T>
    typename std::enable_if_t<std::is_arithmetic<T>::value, std::unique_ptr<scalar>>
    operator()(column_view const& in, size_type location) {
        return pick<T>(in, location);
    }
};

struct quantile_functor
{
    template<typename T>
    typename std::enable_if_t<not std::is_arithmetic<T>::value, std::unique_ptr<scalar>>
    operator()(column_view const& in,
               double quantile,
               interpolation interpolation,
               bool is_sorted,
               order order,
               null_order null_order,
               rmm::mr::device_memory_resource *mr =
               rmm::mr::get_default_resource(),
               hipStream_t stream = 0)
    {
        CUDF_FAIL("non-arithmetic types are unsupported");
    }

    template<typename T>
    typename std::enable_if_t<std::is_arithmetic<T>::value, std::unique_ptr<scalar>>
    operator()(column_view const& in,
               double quantile,
               interpolation interpolation,
               bool is_sorted,
               order order,
               null_order null_order,
               rmm::mr::device_memory_resource *mr =
                 rmm::mr::get_default_resource(),
               hipStream_t stream = 0)
    {
        auto null_offset = null_order == null_order::AFTER ? 0 : in.null_count();
        double result{};

        if (not is_sorted)
        {
            table_view const in_table { { in } };
            auto in_sortmap = sorted_order(in_table, { order }, { null_order });
            auto in_sortmap_begin = in_sortmap->view().begin<size_type>();
            auto in_begin = in.begin<T>() + null_offset;

            auto source = [&](size_type location) {
                auto idx = get_array_value<size_type>(in_sortmap_begin, location);
                return get_array_value<T>(in_begin, idx);
            };

            result = select_quantile<double>(source,
                                             in.size() + in.null_count(),
                                             quantile,
                                             interpolation);
        } else {
            auto in_begin = in.begin<T>() + null_offset;
            auto source = [&](size_type location) {
                return get_array_value<T>(in_begin, location);
            };

            result = select_quantile<double>(source,
                                             in.size() - in.null_count(),
                                             quantile,
                                             interpolation);
        }

        return std::make_unique<numeric_scalar<ScalarResult>>(result);
    }
};

} // anonymous namespace

std::unique_ptr<scalar>
quantile(column_view const& in,
         double quantile,
         interpolation interpolation,
         bool is_sorted,
         order order,
         null_order null_order)
{
        if (in.size() == in.null_count()) {
            return std::make_unique<numeric_scalar<ScalarResult>>(0, false);
        }

        if (in.size() == 1) {
            return type_dispatcher(in.type(), detail::pick_functor{}, in, 0);
        }

        // if (not is_sorted) {
        //     if (quantile <= 0.0) {
        //         auto idx = in.nullable()
        //             ? extrema<extrema::min, true>(in, order, null_order, 0)
        //             : extrema<extrema::min, false>(in, order, null_order, 0);

        //         return type_dispatcher(in.type(), detail::pick_functor{},
        //                                in, idx);
        //     }

        //     if (quantile >= 1.0) {
        //         auto idx = in.nullable()
        //             ? extrema<extrema::max, true>(in, order, null_order, 0)
        //             : extrema<extrema::max, false>(in, order, null_order, 0);

        //         return type_dispatcher(in.type(), detail::pick_functor{},
        //                                in, idx);
        //     }
        // }

        return type_dispatcher(in.type(), detail::quantile_functor{},
                               in,  quantile, interpolation, is_sorted, order, null_order);
}

} // namspace detail

std::vector<std::unique_ptr<scalar>>
quantiles(table_view const& in,
          double quantile,
          interpolation interpolation,
          bool is_sorted,
          std::vector<order> orders,
          std::vector<null_order> null_orders)
{
    std::vector<std::unique_ptr<scalar>> out(in.num_columns());
    for (size_type i = 0; i < in.num_columns(); i++) {
        out[i] = detail::quantile(in.column(i),
                                  quantile,
                                  interpolation,
                                  is_sorted,
                                  orders[i],
                                  null_orders[i]);
    }
    return out;
}

} // namespace experimental
} // namespace cudf

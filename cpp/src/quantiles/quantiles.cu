/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <memory>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <cudf/copying.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>
#include <quantiles/quantiles_util.hpp>

using ScalarResult = double;

namespace cudf {
namespace experimental {
namespace detail {
namespace {

struct quantile_functor
{
    template<typename T>
    std::enable_if_t<not std::is_arithmetic<T>::value, std::unique_ptr<scalar>>
    operator()(column_view const& input,
               double q,
               interpolation interp,
               bool is_sorted,
               cudf::order col_order,
               cudf::null_order col_null_order,
               rmm::mr::device_memory_resource *mr =
               rmm::mr::get_default_resource(),
               hipStream_t stream = 0)
    {
        CUDF_FAIL("non-arithmetic types are unsupported");
    }

    template<typename T>
    std::enable_if_t<std::is_arithmetic<T>::value, std::unique_ptr<scalar>>
    operator()(column_view const& input,
               double q,
               interpolation interp,
               bool is_sorted,
               cudf::order col_order,
               cudf::null_order col_null_order,
               rmm::mr::device_memory_resource *mr =
                 rmm::mr::get_default_resource(),
               hipStream_t stream = 0)
    {
        if (input.size() == 1){
            auto result = get_array_value<ScalarResult>(input.begin<T>(), 0);
            return std::make_unique<numeric_scalar<ScalarResult>>(result);
        }

        auto null_offset = col_null_order == cudf::null_order::AFTER ? 0 : input.null_count();
        double result{};

        if (not is_sorted)
        {
            table_view const in_table { { input } };
            auto in_sortmap = sorted_order(in_table, { col_order }, { col_null_order });
            auto in_sortmap_begin = in_sortmap->view().begin<size_type>();
            auto in_begin = input.begin<T>() + null_offset;

            auto source = [&](size_type location) {
                auto idx = detail::get_array_value<size_type>(in_sortmap_begin, location);
                return detail::get_array_value<T>(in_begin, idx);
            };

            result = select_quantile<double>(source,
                                             input.size() - input.null_count(),
                                             q,
                                             interp);
        } else {
            auto in_begin = input.begin<T>() + null_offset;
            auto source = [&](size_type location) {
                return detail::get_array_value<T>(in_begin, location);
            };

            result = select_quantile<double>(source,
                                             input.size() - input.null_count(),
                                             q,
                                             interp);
        }

        return std::make_unique<numeric_scalar<ScalarResult>>(result);
    }
};

} // anonymous namespace

std::unique_ptr<scalar>
quantile(column_view const& input,
         double q,
         interpolation interp,
         bool is_sorted,
         cudf::order col_order,
         cudf::null_order col_null_order)
{
        if (input.size() == input.null_count()) {
            return std::make_unique<numeric_scalar<ScalarResult>>(0, false);
        }

        return type_dispatcher(input.type(), detail::quantile_functor{},
                               input, q, interp, is_sorted, col_order, col_null_order);
}

} // namspace detail

std::vector<std::unique_ptr<scalar>>
quantiles(table_view const& input,
          double q,
          interpolation interp,
          bool col_is_sorted,
          std::vector<cudf::order> col_order,
          std::vector<cudf::null_order> col_null_order)
{
    std::vector<std::unique_ptr<scalar>> out(input.num_columns());
    for (size_type i = 0; i < input.num_columns(); i++) {
        out[i] = detail::quantile(input.column(i),
                                  q,
                                  interp,
                                  col_is_sorted,
                                  col_order[i],
                                  col_null_order[i]);
    }
    return out;
}

} // namespace experimental
} // namespace cudf

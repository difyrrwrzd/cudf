#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <memory>
#include <stdexcept>
#include <thrust/detail/execute_with_allocator.h>
#include <thrust/extrema.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/detail/par.h>
#include <thrust/transform.h>
#include <rmm/thrust_rmm_allocator.h>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/copying.hpp>
#include <cudf/legacy/reduction.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/sorting.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <quantiles/quantiles_util.hpp>

namespace cudf {
namespace experimental {
namespace detail {
namespace {

using TScalarResult = double;

struct quantile_index
{
    size_type lower;
    size_type higher;
    size_type nearest;
    double fraction;

    quantile_index(size_type count, double quantile)
    {
        quantile = std::min(std::max(quantile, 0.0), 1.0);

        double val = quantile * (count - 1);
        lower = std::floor(val);
        higher = static_cast<size_t>(std::ceil(val));
        nearest = static_cast<size_t>(std::nearbyint(val));
        fraction = val - lower;
    }
};

template<typename T>
std::unique_ptr<scalar>
select_quantile(T const * begin,
                size_t size,
                double quantile,
                interpolation interpolation)
{
    
    if (size < 2) {
        auto result_value = get_array_value<TScalarResult>(begin, 0);
        return std::make_unique<numeric_scalar<TScalarResult>>(result_value);
    }

    quantile_index idx(size, quantile);

    T a;
    T b;
    TScalarResult value;

    switch (interpolation) {
    case interpolation::LINEAR:
        a = get_array_value<T>(begin, idx.lower);
        b = get_array_value<T>(begin, idx.higher);
        value = interpolate::linear<TScalarResult>(a, b, idx.fraction);
        break;

    case interpolation::MIDPOINT:
        a = get_array_value<T>(begin, idx.lower);
        b = get_array_value<T>(begin, idx.higher);
        value = interpolate::midpoint<TScalarResult>(a, b);
        break;

    case interpolation::LOWER:
        value = get_array_value<TScalarResult>(begin, idx.lower);
        break;

    case interpolation::HIGHER:
        value = get_array_value<TScalarResult>(begin, idx.higher);
        break;

    case interpolation::NEAREST:
        value = get_array_value<TScalarResult>(begin, idx.nearest);
        break;

    default:
        throw new cudf::logic_error("not implemented");
    }

    return std::make_unique<numeric_scalar<TScalarResult>>(value);
}

enum class extrema {
    min,
    max
};


size_type extrema(column_view const & in,
          order order,
          null_order null_order,
          extrema minmax,
          hipStream_t stream)
{
    std::vector<cudf::order> h_order{ order };
    std::vector<cudf::null_order> h_null_order{ null_order };
    rmm::device_vector<cudf::order> d_order( h_order );
    rmm::device_vector<cudf::null_order> d_null_order( h_null_order );
    table_view in_table({ in });
    auto in_table_d = table_device_view::create(in_table);
    auto it = thrust::make_counting_iterator<size_type>(0);
    auto policy = rmm::exec_policy(stream);

    if (in.nullable()) {
        auto comparator = row_lexicographic_comparator<true>(
            *in_table_d,
            *in_table_d,
            d_order.data().get(),
            d_null_order.data().get());
        
        auto extrema_id = minmax == extrema::min
            ? thrust::min_element(policy->on(stream), it, it + in.size(), comparator)
            : thrust::max_element(policy->on(stream), it, it + in.size(), comparator);

        return *extrema_id;
    } else {
        auto comparator = row_lexicographic_comparator<false>(
            *in_table_d,
            *in_table_d,
            d_order.data().get(),
            d_null_order.data().get());
        
        auto extrema_idx = minmax == extrema::min
            ? thrust::min_element(policy->on(stream), it, it + in.size(), comparator)
            : thrust::max_element(policy->on(stream), it, it + in.size(), comparator);

        return *extrema_idx;
    }
}

template<typename T>
std::unique_ptr<scalar>
pick(column_view const& in, size_type index) {
    auto result = get_array_value<TScalarResult>(in.begin<T>(), index);
    return std::make_unique<numeric_scalar<TScalarResult>>(result);
}

struct quantile_functor
{
    template<typename T>
    typename std::enable_if_t<not std::is_arithmetic<T>::value, std::unique_ptr<scalar>>
    operator()(column_view const& in,
               double quantile,
               interpolation interpolation,
               bool is_sorted,
               order order,
               null_order null_order,
               rmm::mr::device_memory_resource *mr =
               rmm::mr::get_default_resource(),
               hipStream_t stream = 0)
    {
        CUDF_FAIL("non-arithmetic types are unsupported");
    }

    template<typename T>
    typename std::enable_if_t<std::is_arithmetic<T>::value, std::unique_ptr<scalar>>
    operator()(column_view const& in,
               double quantile,
               interpolation interpolation,
               bool is_sorted,
               order order,
               null_order null_order,
               rmm::mr::device_memory_resource *mr =
                 rmm::mr::get_default_resource(),
               hipStream_t stream = 0)
    {
        if (in.size() == 1) {
            return pick<T>(in, 0);
        }
    
        auto null_offset = null_order == null_order::AFTER ? 0 : in.null_count();
        
        if (not is_sorted)
        {
            if (quantile <= 0.0) {
                return pick<T>(in, extrema(in, order, null_order, extrema::min, stream));
            }
    
            if (quantile >= 1.0) {
                return pick<T>(in, extrema(in, order, null_order, extrema::max, stream));
            }
    
            table_view const in_table { { in } };
            auto sorted_idx = sorted_order(in_table, { order }, { null_order });

            // TODO: select_quantile can use the sortmap without gather.
            auto sorted = gather(in_table, sorted_idx->view());
            auto sorted_col = sorted->view().column(0);

            return select_quantile<T>(sorted_col.begin<T>() + null_offset,
                                      sorted_col.size() - sorted_col.null_count(),
                                      quantile,
                                      interpolation);
    
        } else {
            return select_quantile<T>(in.begin<T>() + null_offset,
                                      in.size() - in.null_count(),
                                      quantile,
                                      interpolation);
        }
    }
};

} // anonymous namespace

std::unique_ptr<scalar>
quantile(column_view const& in,
         double quantile,
         interpolation interpolation,
         bool is_sorted,
         order order,
         null_order null_order)
{
        if (in.size() == in.null_count()) {
            return std::make_unique<numeric_scalar<TScalarResult>>(0, false);
        }

        return type_dispatcher(in.type(), detail::quantile_functor{},
                               in,  quantile, interpolation, is_sorted, order, null_order);
}

} // namspace detail

std::vector<std::unique_ptr<scalar>>
quantiles(table_view const& in,
          double quantile,
          interpolation interpolation,
          bool is_sorted,
          std::vector<order> orders,
          std::vector<null_order> null_orders)
{
    std::vector<std::unique_ptr<scalar>> out(in.num_columns());
    for (size_type i = 0; i < in.num_columns(); i++) {
        out[i] = detail::quantile(in.column(i),
                                  quantile,
                                  interpolation,
                                  is_sorted,
                                  orders[i],
                                  null_orders[i]);
    }
    return out;
}

} // namespace experimental
} // namespace cudf

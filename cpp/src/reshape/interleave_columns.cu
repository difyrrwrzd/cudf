/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/copying.hpp>
#include <cudf/table/table_device_view.cuh>
#include "cudf/utilities/bit.hpp"

namespace cudf {

namespace experimental {

namespace detail {

template<typename TElement>
struct interleave_columns_selector
{
    table_device_view in;

    TElement __device__ operator()(size_type i)
    {
        auto col_num = i % in.num_columns();
        auto row_num = i / in.num_columns();
        column_device_view in_col = in.column(col_num);
        return in_col.element<TElement>(row_num);
    }
};

struct interleave_columns_validity_selector
{
    table_device_view in;
    size_type out_row_count;

    bitmask_type __device__ operator()(size_type i)
    {
        bitmask_type out = 0b00000000000000000000000000000000;

        const auto num_bits = cudf::detail::size_in_bits<bitmask_type>();

        for (size_type bit = 0; bit < num_bits; bit++) {
            size_type out_row = i * num_bits + bit;
            if (out_row >= out_row_count) {
                break;
            }

            out |= _select(out_row) << bit;
        }

        return out;
    }

private:

    bool __device__ _select(size_type out_row)
    {
        size_type in_col = out_row % in.num_columns();
        size_type in_row = out_row / in.num_columns();

        return bit_is_set(in.column(in_col).null_mask(), in_row);
    }
};

struct interleave_columns_functor
{
    template <typename TElement>
    std::enable_if_t<cudf::is_fixed_width<TElement>(), std::unique_ptr<cudf::column>>
    operator()(table_view const& in,
               mask_allocation_policy mask_policy,
               rmm::mr::device_memory_resource *mr,
               hipStream_t stream)
    {
        auto arch_column = in.column(0);
        auto size = in.num_columns() * in.num_rows();
        auto out = allocate_like(arch_column, size, mask_policy, mr);
        auto device_in = table_device_view::create(in);
        auto device_out = mutable_column_device_view::create(*out);
        auto counting_it = thrust::make_counting_iterator<size_type>(0);

        thrust::transform(rmm::exec_policy(stream)->on(stream),
                          counting_it,
                          counting_it + size,
                          device_out->data<TElement>(),
                          interleave_columns_selector<TElement>{*device_in});

        if (out->nullable())
        {
            thrust::transform(rmm::exec_policy(stream)->on(stream),
                              counting_it,
                              counting_it + 1,
                              device_out->null_mask(),
                              interleave_columns_validity_selector{*device_in, device_out->size()});
        }

        return out;
    }

    template <typename TElement>
    std::enable_if_t<not cudf::is_fixed_width<TElement>(), std::unique_ptr<cudf::column>>
    operator()(table_view const& in,
               mask_allocation_policy mask_policy,
               rmm::mr::device_memory_resource *mr,
               hipStream_t stream)
    {
        CUDF_FAIL("interleave_columns does not work for variable width types.");
    }
};

} // namespace detail

std::unique_ptr<column>
interleave_columns(table_view const& in,
                   rmm::mr::device_memory_resource *mr,
                   hipStream_t stream)
{
    CUDF_EXPECTS(in.num_columns() > 0, "input must have at least one column to determine dtype.");

    auto arch_column = in.column(0);

    if (in.num_columns() == 0)
    {
        return empty_like(arch_column);
    }
    auto dtype = arch_column.type();
    auto mask_policy = mask_allocation_policy::NEVER;

    for (auto &&col : in) {
        CUDF_EXPECTS(dtype == col.type(), "DTYPE mismatch");
        if (col.nullable()) {
            mask_policy = mask_allocation_policy::ALWAYS;
        }
    }

    auto out = type_dispatcher(dtype, detail::interleave_columns_functor{},
                               in, mask_policy,
                               mr, stream);

    return out;
}

} // namespace experimental

} // namespace cudf

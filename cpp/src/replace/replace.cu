#include "hip/hip_runtime.h"
/*
 * Copyright 2018 BlazingDB, Inc.

 *     Copyright 2018 Cristhian Alberto Gonzales Castillo <cristhian@blazingdb.com>
 *     Copyright 2018 Alexander Ocsa <alexander@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/device_ptr.h>
#include <thrust/find.h>
#include <thrust/execution_policy.h>
#include <hipcub/hipcub.hpp>

#include <cudf/legacy/copying.hpp>
#include <cudf/legacy/replace.hpp>
#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <utilities/cudf_utils.h>
#include <utilities/cuda_utils.hpp>
#include <utilities/column_utils.hpp>
#include <bitmask/legacy/legacy_bitmask.hpp>
#include <bitmask/legacy/bit_mask.cuh>

namespace {  // anonymous

using namespace cudf;

static constexpr int BLOCK_SIZE = 256;

/* --------------------------------------------------------------------------*/
/**
 * @brief Kernel that converts inputs from `in` to `out`  using the following
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in] column_device_view representing input data
 * @param[in] mutable_column_device_view representing output data. can be
 *            the same actual underlying buffer that in points to. 
 *
 * @returns
 */
/* ----------------------------------------------------------------------------*/
template <typename T>
__global__
void normalize_nans_and_zeros(column_device_view in, 
                              mutable_column_device_view out)
{
   int tid = threadIdx.x;
   int blkid = blockIdx.x;
   int blksz = blockDim.x;
   int gridsz = gridDim.x;

   int start = tid + blkid * blksz;
   int step = blksz * gridsz;

   // grid-stride
   for (int i=start; i<in.size(); i+=step) {
      if(!in.is_valid(i)){
         continue;
      }

      T el = in.element<T>(i);
      if(std::isnan(el)){
         out.element<T>(i) = std::numeric_limits<T>::quiet_NaN();
      } else if(el == (T)-0.0){
         out.element<T>(i) = (T)0.0;
      } else {
         out.element<T>(i) = el;
      }
   }
}                        

  /* --------------------------------------------------------------------------*/
  /**
   * @brief Functor called by the `type_dispatcher` in order to invoke and instantiate
   *        `normalize_nans_and_zeros` with the appropriate data types.
   */
  /* ----------------------------------------------------------------------------*/
struct normalize_nans_and_zeros_kernel_forwarder {
   // floats and doubles. what we really care about.
   template <typename T, std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
   void operator()(  column_device_view in,
                     mutable_column_device_view out,
                     hipStream_t stream)
   {
      util::cuda::grid_config_1d grid{in.size(), BLOCK_SIZE};
      normalize_nans_and_zeros<T><<<grid.num_blocks, BLOCK_SIZE, 0, stream>>>(in, out);
   }

   // if we get in here for anything but a float or double, that's a problem.
   template <typename T, std::enable_if_t<not std::is_floating_point<T>::value>* = nullptr>
   void operator()(  column_device_view in,
                     mutable_column_device_view out,
                     hipStream_t stream)
   {
      CUDF_FAIL("Unexpected non floating-point type.");      
   }   
};

} // end anonymous namespace

namespace cudf {
namespace detail {

std::unique_ptr<column> normalize_nans_and_zeros( column_view input,                                            
                                                  hipStream_t stream,
                                                  rmm::mr::device_memory_resource *mr)
{      
   CUDF_EXPECTS(input.head() != nullptr, "Null input data");
   if(input.size() == 0 || input.head() == nullptr){
      return make_numeric_column(input.type(), input.size(), ALL_VALID, stream, mr);
   }   
   CUDF_EXPECTS(input.type() == data_type(FLOAT32) || input.type() == data_type(FLOAT64), "Expects float or double input");

    // to device. unique_ptr which gets automatically cleaned up when we leave
   auto device_in = column_device_view::create(input);
   
   // ultimately, the output.
   auto out = make_numeric_column(input.type(), input.size(), ALL_VALID, stream, mr);
   // from device. unique_ptr which gets automatically cleaned up when we leave.
   auto device_out = mutable_column_device_view::create(*out);

   // invoke the actual kernel.  
  experimental::type_dispatcher(input.type(), 
                                normalize_nans_and_zeros_kernel_forwarder{},
                                *device_in,
                                *device_out,
                                stream);

   return out;                 
}                                                 

void normalize_nans_and_zeros(mutable_column_view in_out,
                              hipStream_t stream)
{   
   CUDF_EXPECTS(in_out.head() != nullptr, "Null input data");
   if(in_out.size() == 0 || in_out.head() == nullptr){
      return;
   }
   CUDF_EXPECTS(in_out.type() == data_type(FLOAT32) || in_out.type() == data_type(FLOAT64), "Expects float or double input");

   // wrapping the in_out data in a column_view so we can call the same lower level code.
   // that we use for the non in-place version.
   column_view input = in_out;

   // to device. unique_ptr which gets automatically cleaned up when we leave
   auto device_in = column_device_view::create(input);

   // from device. unique_ptr which gets automatically cleaned up when we leave.   
   auto device_out = mutable_column_device_view::create(in_out);

    // invoke the actual kernel.  
   cudf::experimental::type_dispatcher(input.type(), 
                                       normalize_nans_and_zeros_kernel_forwarder{},
                                       *device_in,
                                       *device_out,
                                       stream);
}

}  // namespace detail

/**
 * @brief Function that converts inputs from `input` using the following rule
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in] column_device_view representing input data
 * @param[in] device_memory_resource allocator for allocating output data 
 *
 * @returns new column
 */
std::unique_ptr<column> normalize_nans_and_zeros( column_view input,                                                                                                    
                                                  rmm::mr::device_memory_resource *mr)
{
   return detail::normalize_nans_and_zeros(input, 0, mr);
}

/**
 * @brief Function that processes values in-place from `in_out` using the following rule
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in, out] mutable_column_view representing input data. data is processed in-place
 *
 * @returns new column
 */
void normalize_nans_and_zeros(mutable_column_view in_out)
{
   return detail::normalize_nans_and_zeros(in_out, 0);
}

}  // namespace cudf


#include "hip/hip_runtime.h"
/*
 * Copyright 2018 BlazingDB, Inc.

 *     Copyright 2018 Cristhian Alberto Gonzales Castillo <cristhian@blazingdb.com>
 *     Copyright 2018 Alexander Ocsa <alexander@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/device_ptr.h>
#include <thrust/find.h>
#include <thrust/execution_policy.h>
#include <hipcub/hipcub.hpp>
#include <cudf/legacy/interop.hpp>
#include <cudf/copying.hpp>
#include <cudf/replace.hpp>
#include <cudf/detail/replace.hpp>
#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <cudf/types.hpp>
#include <utilities/error_utils.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <utilities/cudf_utils.h>
#include <utilities/cuda_utils.hpp>
#include <utilities/column_utils.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column.hpp>
#include <bitmask/legacy/legacy_bitmask.hpp>
#include <bitmask/legacy/bit_mask.cuh>

using bit_mask::bit_mask_t;

namespace{ //anonymous

static constexpr int warp_size = 32;
static constexpr int BLOCK_SIZE = 256;

// returns the block_sum using the given shared array of warp sums.
template <typename T>
__device__ T sum_warps(T* warp_smem)
{
  T block_sum = 0;

   if (threadIdx.x < warp_size) {
    T my_warp_sum = warp_smem[threadIdx.x];
    __shared__ typename hipcub::WarpReduce<T>::TempStorage temp_storage;
    block_sum = hipcub::WarpReduce<T>(temp_storage).Sum(my_warp_sum);
  }
  return block_sum;
}

// return the new_value for output column at index `idx`
template<class T, bool replacement_has_nulls>
__device__ auto get_new_value(gdf_size_type         idx,
                           const T* __restrict__ input_data,
                           const T* __restrict__ values_to_replace_begin,
                           const T* __restrict__ values_to_replace_end,
                           const T* __restrict__       d_replacement_values,
                           bit_mask_t const * __restrict__ replacement_valid)
   {
     auto found_ptr = thrust::find(thrust::seq, values_to_replace_begin,
                                      values_to_replace_end, input_data[idx]);
     T new_value{0};
     bool output_is_valid{true};

     if (found_ptr != values_to_replace_end) {
       auto d = thrust::distance(values_to_replace_begin, found_ptr);
       new_value = d_replacement_values[d];
       if (replacement_has_nulls) {
         output_is_valid = bit_mask::is_valid(replacement_valid, d);
       }
     } else {
       new_value = input_data[idx];
     }
     return thrust::make_pair(new_value, output_is_valid);
   }

  /* --------------------------------------------------------------------------*/
  /**
   * @brief Kernel that replaces elements from `output_data` given the following
   *        rule: replace all `values_to_replace[i]` in [values_to_replace_begin`,
   *        `values_to_replace_end`) present in `output_data` with `d_replacement_values[i]`.
   *
   * @tparam input_has_nulls `true` if output column has valid mask, `false` otherwise
   * @tparam replacement_has_nulls `true` if replacement_values column has valid mask, `false` otherwise
   * The input_has_nulls and replacement_has_nulls template parameters allows us to specialize
   * this kernel for the different scenario for performance without writing different kernel.
   *
   * @param[in] input_data Device array with the data to be modified
   * @param[in] input_valid Valid mask associated with input_data
   * @param[out] output_data Device array to store the data from input_data
   * @param[out] output_valid Valid mask associated with output_data
   * @param[out] output_valid_count #valid in output column
   * @param[in] nrows # rows in `output_data`
   * @param[in] values_to_replace_begin Device pointer to the beginning of the sequence
   * of old values to be replaced
   * @param[in] values_to_replace_end  Device pointer to the end of the sequence
   * of old values to be replaced
   * @param[in] d_replacement_values Device array with the new values
   * @param[in] replacement_valid Valid mask associated with d_replacement_values
   *
   * @returns
   */
  /* ----------------------------------------------------------------------------*/
  template <class T,
            bool input_has_nulls, bool replacement_has_nulls>
  __global__
  void replace_kernel(const T* __restrict__           input_data,
                      bit_mask_t const * __restrict__ input_valid,
                      T * __restrict__          output_data,
                      bit_mask_t * __restrict__ output_valid,
                      gdf_size_type * __restrict__    output_valid_count,
                      gdf_size_type                   nrows,
                      const T* __restrict__ values_to_replace_begin,
                      const T* __restrict__ values_to_replace_end,
                      const T* __restrict__           d_replacement_values,
                      bit_mask_t const * __restrict__ replacement_valid)
  {
  gdf_size_type i = blockIdx.x * blockDim.x + threadIdx.x;

  uint32_t active_mask = 0xffffffff;
  active_mask = __ballot_sync(active_mask, i < nrows);
  __shared__ uint32_t valid_sum[warp_size];

  // init shared memory for block valid counts
  if (input_has_nulls or replacement_has_nulls){
    if(threadIdx.x < warp_size) valid_sum[threadIdx.x] = 0;
    __syncthreads();
  }

  while (i < nrows) {
    bool output_is_valid = true;
    uint32_t bitmask = 0xffffffff;

    if (input_has_nulls) {
      bool const input_is_valid{bit_mask::is_valid(input_valid, i)};
      output_is_valid = input_is_valid;

      bitmask = __ballot_sync(active_mask, input_is_valid);

      if (input_is_valid) {
        thrust::tie(output_data[i], output_is_valid)  =
            get_new_value<T, replacement_has_nulls>(i, input_data,
                                      values_to_replace_begin,
                                      values_to_replace_end,
                                      d_replacement_values,
                                      replacement_valid);
      }

    } else {
       thrust::tie(output_data[i], output_is_valid) =
            get_new_value<T, replacement_has_nulls>(i, input_data,
                                      values_to_replace_begin,
                                      values_to_replace_end,
                                      d_replacement_values,
                                      replacement_valid);
    }

    /* output valid counts calculations*/
    if (input_has_nulls or replacement_has_nulls){

      bitmask &= __ballot_sync(active_mask, output_is_valid);

      if(0 == (threadIdx.x % warp_size)){
        output_valid[(int)(i/warp_size)] = bitmask;
        valid_sum[(int)(threadIdx.x / warp_size)] += __popc(bitmask);
      }
    }

    i += blockDim.x * gridDim.x;
    active_mask = __ballot_sync(active_mask, i < nrows);
  }
  if(input_has_nulls or replacement_has_nulls){
    __syncthreads(); // waiting for the valid counts of each warp to be ready

    // Compute total valid count for this block and add it to global count
    uint32_t block_valid_count = sum_warps<uint32_t>(valid_sum);

    // one thread computes and adds to output_valid_count
    if (threadIdx.x < warp_size && 0 == (threadIdx.x % warp_size)) {
      atomicAdd(output_valid_count, block_valid_count);
    }
  }
}

  /* --------------------------------------------------------------------------*/
  /**
   * @brief Functor called by the `type_dispatcher` in order to invoke and instantiate
   *        `replace_kernel` with the appropriate data types.
   */
  /* ----------------------------------------------------------------------------*/
  struct replace_kernel_forwarder {
    template <typename col_type>
    void operator()(cudf::column_view const& input_col,
                    cudf::column_view const& values_to_replace,
                    cudf::column_view const& replacement_values,
                    cudf::mutable_column_view& output,
                    hipStream_t stream = 0,
                    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource())
    {
      const bool input_has_nulls = input_col.has_nulls();
      const bool replacement_has_nulls = replacement_values.has_nulls();

      cudf::size_type *valid_count = nullptr;
      if (output.nullable()) {
        valid_count = reinterpret_cast<gdf_size_type*>(mr->allocate(sizeof(gdf_size_type), stream));
        CUDA_TRY(hipMemsetAsync(valid_count, 0, sizeof(gdf_size_type), stream));
      }

      cudf::util::cuda::grid_config_1d grid{output.size(), BLOCK_SIZE, 1};

      auto replace = replace_kernel<col_type, true, true>;

      if (input_has_nulls){
        if (replacement_has_nulls){
          replace = replace_kernel<col_type, true, true>;
        }else{
          replace = replace_kernel<col_type, true, false>;
        }
      }else{
        if (replacement_has_nulls){
          replace = replace_kernel<col_type, false, true>;
        }else{
          replace = replace_kernel<col_type, false, false>;
        }
      }
      replace<<<grid.num_blocks, BLOCK_SIZE, 0, stream>>>(
                                             input_col.data<col_type>(),
                                             input_col.null_mask(),
                                             output.data<col_type>(),
                                             output.null_mask(),
                                             valid_count,
                                             output.size(),
                                             values_to_replace.data<col_type>(),
                                             values_to_replace.data<col_type>() + replacement_values.size(),
                                             replacement_values.data<col_type>(),
                                             replacement_values.null_mask());

      if(valid_count != nullptr){
        cudf::size_type valids {0};
        CUDA_TRY(hipMemcpyAsync(&valids,
                                 valid_count,
                                 sizeof(cudf::size_type),
                                 hipMemcpyDefault,
                                 stream));
        output.set_null_count(output.size() - valids);
        mr->deallocate(valid_count, sizeof(cudf::size_type), stream);
      }
    }
  };

  template<>
  void replace_kernel_forwarder::operator()<cudf::string_view> (cudf::column_view const& input_col,
                                                                cudf::column_view const& values_to_replace,
                                                                cudf::column_view const& replacement_values,
                                                                cudf::mutable_column_view& output,
                                                                hipStream_t stream,
                                                                rmm::mr::device_memory_resource* mr) {
    CUDF_FAIL("Strings are not supported yet for replacement.");
  }

 } //end anonymous namespace

namespace cudf{
namespace detail {
  std::unique_ptr<cudf::column> find_and_replace_all(cudf::column_view const& input_col,
                                                     cudf::column_view const& values_to_replace,
                                                     cudf::column_view const& replacement_values,
                                                     hipStream_t stream,
                                                     rmm::mr::device_memory_resource* mr) {
    if (0 == input_col.size() )
    {
      return std::unique_ptr<cudf::column>(new cudf::column(input_col));
    }

    if (0 == values_to_replace.size() || 0 == replacement_values.size())
    {
      return std::unique_ptr<cudf::column>(new cudf::column(input_col));
    }

    CUDF_EXPECTS(values_to_replace.size() == replacement_values.size(),
                 "values_to_replace and replacement_values size mismatch.");
    CUDF_EXPECTS(input_col.type() == values_to_replace.type() &&
                 input_col.type() == replacement_values.type(),
                 "Columns type mismatch.");
    CUDF_EXPECTS(input_col.data<int32_t>() != nullptr, "Null input data.");
    CUDF_EXPECTS(values_to_replace.data<int32_t>() != nullptr && replacement_values.data<int32_t>() != nullptr,
                 "Null replace data.");
    CUDF_EXPECTS(values_to_replace.nullable() == false,
                 "Nulls are in values_to_replace column.");

    std::unique_ptr<column> output;
    if (input_col.nullable() || replacement_values.nullable()) {
      output = make_numeric_column(input_col.type(), input_col.size(), UNINITIALIZED, stream, mr);
    }
    else
      output = make_numeric_column(input_col.type(), input_col.size(), UNALLOCATED, stream, mr);

    cudf::mutable_column_view outputView = (*output).mutable_view();
    cudf::experimental::type_dispatcher(input_col.type(),
                                        replace_kernel_forwarder { },
                                        input_col,
                                        values_to_replace,
                                        replacement_values,
                                        outputView,
                                        stream,
                                        mr);

    CHECK_STREAM(stream);
    return output;
  }

} //end details
namespace experimental {
/* --------------------------------------------------------------------------*/
/**
 * @brief Replace elements from `input_col` according to the mapping `values_to_replace` to
 *        `replacement_values`, that is, replace all `values_to_replace[i]` present in `input_col`
 *        with `replacement_values[i]`.
 *
 * @param[in] col gdf_column with the data to be modified
 * @param[in] values_to_replace gdf_column with the old values to be replaced
 * @param[in] replacement_values gdf_column with the new values
 *
 * @returns output gdf_column with the modified data
 */
/* ----------------------------------------------------------------------------*/
  std::unique_ptr<cudf::column> find_and_replace_all(cudf::column_view const& input_col,
                                                     cudf::column_view const& values_to_replace,
                                                     cudf::column_view const& replacement_values,
                                                     rmm::mr::device_memory_resource* mr){
    return detail::find_and_replace_all(input_col, values_to_replace, replacement_values, 0, mr);
  }
} //end experimental
} //end cudf

namespace{ //anonymous

using bit_mask::bit_mask_t;

template <typename Type>
__global__
void replace_nulls_with_scalar(gdf_size_type size,
                               const Type* __restrict__ in_data,
                               const bit_mask_t* __restrict__ in_valid,
                               const Type* __restrict__ replacement,
                               Type* __restrict__ out_data)
{
  int tid = threadIdx.x;
  int blkid = blockIdx.x;
  int blksz = blockDim.x;
  int gridsz = gridDim.x;

  int start = tid + blkid * blksz;
  int step = blksz * gridsz;

  for (int i=start; i<size; i+=step) {
    out_data[i] = bit_mask::is_valid(in_valid, i)? in_data[i] : *replacement;
  }
}


template <typename Type>
__global__
void replace_nulls_with_column(gdf_size_type size,
                               Type const* __restrict__ in_data,
                               cudf::bitmask_type const* __restrict__ in_valid,
                               Type const* __restrict__ replacement,
                               Type* __restrict__ out_data)
{
  int tid = threadIdx.x;
  int blkid = blockIdx.x;
  int blksz = blockDim.x;
  int gridsz = gridDim.x;

  int start = tid + blkid * blksz;
  int step = blksz * gridsz;

  for (int i=start; i<size; i+=step) {
    out_data[i] = bit_mask::is_valid(in_valid, i)? in_data[i] : replacement[i];
  }
}


/* --------------------------------------------------------------------------*/
/**
 * @brief Functor called by the `type_dispatcher` in order to invoke and instantiate
 *        `replace_nulls` with the appropriate data types.
 */
/* ----------------------------------------------------------------------------*/
struct replace_nulls_column_kernel_forwarder {
  template <typename col_type>
  void operator()(cudf::column_view const& input,
                  cudf::column_view const& replacement,
                  cudf::mutable_column_view& output,
                  hipStream_t stream = 0)
  {
    cudf::size_type nrows = input.size();
    cudf::util::cuda::grid_config_1d grid{nrows, BLOCK_SIZE};

    replace_nulls_with_column<<<grid.num_blocks, BLOCK_SIZE, 0, stream>>>(nrows,
                                                                          input.data<col_type>(),
                                                                          input.null_mask(),
                                                                          replacement.data<col_type>(),
                                                                          output.data<col_type>());

  }
};

template<>
void replace_nulls_column_kernel_forwarder::operator ()<cudf::string_view>(cudf::column_view const& input,
                                                                           cudf::column_view const& replacement,
                                                                           cudf::mutable_column_view& output,
                                                                           hipStream_t stream){
  CUDF_FAIL("Strings not supported for replacement.");
}


/* --------------------------------------------------------------------------*/
/**
 * @brief Functor called by the `type_dispatcher` in order to invoke and instantiate
 *        `replace_nulls` with the appropriate data types.
 */
/* ----------------------------------------------------------------------------*/
struct replace_nulls_scalar_kernel_forwarder {
  template <typename col_type>
  void operator()(cudf::column_view const& input,
                  const void* replacement,
                  cudf::mutable_column_view& output,
                  hipStream_t stream = 0,
                  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource())
  {
    cudf::size_type nrows = input.size();
    cudf::util::cuda::grid_config_1d grid{nrows, BLOCK_SIZE};

    auto t_replacement = static_cast<const col_type*>(replacement);
    col_type* d_replacement = reinterpret_cast<col_type*>(mr->allocate(sizeof(col_type), stream));
    CUDA_TRY(hipMemcpyAsync(d_replacement, t_replacement, sizeof(col_type), hipMemcpyHostToDevice, stream));

    replace_nulls_with_scalar<<<grid.num_blocks, BLOCK_SIZE, 0, stream>>>(nrows,
                                                                          input.data<col_type>(),
                                                                          input.null_mask(),
                                                                          static_cast<const col_type*>(d_replacement),
                                                                          output.data<col_type>());
    mr->deallocate(d_replacement, sizeof(col_type), stream);
  }
};

template<>
void replace_nulls_scalar_kernel_forwarder::operator ()<cudf::string_view>(cudf::column_view const& input,
                                                                           const void* replacement,
                                                                           cudf::mutable_column_view& output,
                                                                           hipStream_t stream,
                                                                           rmm::mr::device_memory_resource* mr) {
  CUDF_FAIL("Strings not supported for replacement");
}


} //end anonymous namespace


namespace cudf {
namespace detail {

std::unique_ptr<cudf::column> replace_nulls(cudf::column_view const& input,
                                            cudf::column_view const& replacement,
                                            hipStream_t stream,
                                            rmm::mr::device_memory_resource* mr)
{
  if (input.size() == 0) {
    return std::unique_ptr<cudf::column>(new cudf::column(input));
  }

  CUDF_EXPECTS(nullptr != input.data<int32_t>(), "Null input data");

  if (input.nullable() == false || input.null_count() == 0) {
    return std::unique_ptr<cudf::column>(new cudf::column(input));
  }

  CUDF_EXPECTS(input.type() == replacement.type(), "Data type mismatch");
  CUDF_EXPECTS(replacement.size() == 1 || replacement.size() == input.size(), "Column size mismatch");
  CUDF_EXPECTS(nullptr != replacement.data<int32_t>(), "Null replacement data");
  CUDF_EXPECTS(replacement.nullable() == false || 0 == replacement.null_count(),
               "Invalid replacement data");

  std::unique_ptr<cudf::column> output = make_numeric_column(input.type(),
                                                             input.size(),
                                                             UNALLOCATED,
                                                             stream,
                                                             mr);
  cudf::mutable_column_view outputView = (*output).mutable_view();
  cudf::experimental::type_dispatcher(input.type(),
                                      replace_nulls_column_kernel_forwarder{},
                                      input,
                                      replacement,
                                      outputView,
                                      stream);
  return output;
}


std::unique_ptr<cudf::column> replace_nulls(cudf::column_view const& input,
                                            const gdf_scalar& replacement,
                                            hipStream_t stream,
                                            rmm::mr::device_memory_resource* mr)
{
  if (input.size() == 0) {
    return std::unique_ptr<cudf::column>(new cudf::column(input));
  }

  CUDF_EXPECTS(nullptr != input.data<int32_t>(), "Null input data");

  if (input.null_count() == 0 || input.nullable() == false) {
    return std::unique_ptr<cudf::column>(new cudf::column(input));
  }

  CUDF_EXPECTS(input.type() == cudf::legacy::gdf_dtype_to_data_type(replacement.dtype), "Data type mismatch");
  CUDF_EXPECTS(true == replacement.is_valid, "Invalid replacement data");

  std::unique_ptr<cudf::column> output = make_numeric_column(input.type(),
                                                             input.size(),
                                                             UNALLOCATED,
                                                             stream,
                                                             mr);
  cudf::mutable_column_view outputView = (*output).mutable_view();
  cudf::experimental::type_dispatcher(input.type(),
                                      replace_nulls_scalar_kernel_forwarder{},
                                      input,
                                      &(replacement.data),
                                      outputView,
                                      stream,
                                      mr);
  return output;
}

}  // namespace detail

namespace experimental {

std::unique_ptr<cudf::column> replace_nulls(cudf::column_view const& input,
                                            cudf::column_view const& replacement,
                                            rmm::mr::device_memory_resource* mr)
{
  return detail::replace_nulls(input, replacement, 0, mr);
}


std::unique_ptr<cudf::column> replace_nulls(cudf::column_view const& input,
                                            const gdf_scalar& replacement,
                                            rmm::mr::device_memory_resource* mr)
{
  return detail::replace_nulls(input, replacement, 0, mr);
}
} //end experimental
}  // namespace cudf

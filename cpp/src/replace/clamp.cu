/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/replace.hpp>
#include <cudf/detail/iterator.cuh>

namespace cudf {
namespace experimental {
namespace detail {
namespace {
struct dispatch_clamp {
    template <typename T, typename Transformer>
    void apply_transform (column_device_view  input,
                          mutable_column_device_view output,
                          Transformer trans,
                          hipStream_t stream)
    {
        if (input.nullable()){
            auto input_begin = cudf::experimental::detail::make_null_replacement_iterator<T>(input);
            thrust::transform(rmm::exec_policy(stream)->on(stream),
                              input_begin,
                              input_begin+input.size(),
                              detail::make_validity_iterator(input),
                              output.begin<T>(),
                              trans);
        } else {
            thrust::transform(rmm::exec_policy(stream)->on(stream),
                              input.begin<T>(),
                              input.end<T>(),
                              output.begin<T>(),
                              trans);
        }
    }

    template <typename T>
    std::enable_if_t<cudf::is_fixed_width<T>(), std::unique_ptr<cudf::column>>
    operator()(column_view const& input,
              scalar const& lo,
              scalar const& hi,
              rmm::mr::device_memory_resource* mr,
              hipStream_t stream) {
        using ScalarType = cudf::experimental::scalar_type_t<T>;
        auto lo_scalar = static_cast<ScalarType const&>(lo);
        auto hi_scalar = static_cast<ScalarType const&>(hi);
        auto lo_value = lo_scalar.value(stream);
        auto hi_value = hi_scalar.value(stream);
        auto output = detail::allocate_like(input, input.size(), mask_allocation_policy::RETAIN, mr, stream);
        // mask will not change
        if (input.nullable()){
            output->set_null_mask(copy_bitmask(input), input.null_count());
        }

        auto output_device_view  = cudf::mutable_column_device_view::create(output->mutable_view(), stream);
        auto input_device_view  = cudf::column_device_view::create(input, stream);

        if (lo.is_valid(stream) and hi.is_valid(stream)) {
            auto trans = [lo_value, hi_value] __device__ (T input, bool is_valid = true){
                if (is_valid) {
                    if (input < lo_value) {
                        return lo_value;
                    } else if (input > hi_value) {
                        return hi_value;
                    }
                }

                return input;
            };

            apply_transform<T>(*input_device_view, *output_device_view, trans, stream);
        } else if (not lo.is_valid(stream)) {
            auto trans = [hi_value] __device__ (T input, bool is_valid = true){
                if (is_valid and input > hi_value) {
                    return hi_value;
                }

                return input;
            };

            apply_transform<T>(*input_device_view, *output_device_view, trans, stream);
        } else {

            auto trans = [lo_value] __device__ (T input, bool is_valid = true){
                if (is_valid and input < lo_value) {
                    return lo_value;
                }

                return input;
            };

            apply_transform<T>(*input_device_view, *output_device_view, trans, stream);
        }
        
        return output;
    }
    
    template <typename T>
    std::enable_if_t<not cudf::is_fixed_width<T>(), std::unique_ptr<cudf::column>>
    operator()(column_view const& input,
              scalar const& lo,
              scalar const& hi,
              rmm::mr::device_memory_resource* mr,
              hipStream_t stream) {
        CUDF_FAIL("Clamp is not yet supporting non-fixed types");
    }

};
} //namespace

/**
 * @copydoc cudf::experimental::clamp
 *
 * @param[in] stream Optional stream on which to issue all memory allocations
 */
std::unique_ptr<column> clamp(column_view const& input,
                              scalar const& lo,
                              scalar const& hi,
                              rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                              hipStream_t stream = 0) {
    CUDF_EXPECTS(lo.type() == hi.type(), "mimatching types of scalars");
    CUDF_EXPECTS(lo.type() == input.type(), "mimatching types of scalar and input");

    if ((not lo.is_valid(stream) and not hi.is_valid(stream)) or 
        (input.is_empty())) {
        // There will be no change
        return std::make_unique<column>(input, stream, mr);
    }

    return cudf::experimental::type_dispatcher(input.type(), dispatch_clamp{},
                                               input, lo, hi,
                                               mr, stream);
}   

}// namespace detail

// clamp input at lo and hi
std::unique_ptr<column> clamp(column_view const& input,
                              scalar const& lo,
                              scalar const& hi,
                              rmm::mr::device_memory_resource* mr) {

    return detail::clamp(input, lo, hi, mr);
}

}// namespace experimental
}// namespace cudf

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/merge.h>

#include <algorithm>
#include <utility>
#include <vector>
#include <memory>
#include <type_traits>
#include <cmath> // for std::ceil()

#include <cudf/types.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/row_operators.cuh>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/thrust_rmm_allocator.h>
#include <cudf/utilities/bit.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/gather.cuh>


namespace cudf {
namespace experimental { 
namespace detail {

std::pair<std::unique_ptr<table>,
          std::vector<cudf::size_type>>
round_robin_partition(table_view const& input,
                      cudf::size_type num_partitions,
                      cudf::size_type start_partition = 0,
                      rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                      hipStream_t stream = 0)
{
  std::pair<std::unique_ptr<table>, std::vector<cudf::size_type>> ret_pair =
    std::make_pair(nullptr, std::vector<cudf::size_type>(num_partitions));
  
  auto nrows = input.num_rows();

  CUDF_EXPECTS( num_partitions > 1 && num_partitions < nrows, "Incorrect number of partitions. Must be greater than 1 and less than number of rows." );
  CUDF_EXPECTS( start_partition < num_partitions, "Incorrect start_partition index. Must be less than number of partitions." );
  
  auto n_pmax = nrows % num_partitions;//# partitions of max size
  size_t max_p_size = std::ceil( static_cast<double>(nrows) / static_cast<double>(num_partitions));// max size of partitions
  auto pmm = n_pmax * max_p_size;
  
  auto iter_begin =
    thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(0),
                                    [num_partitions, max_p_size, n_pmax, pmm] __device__ (auto indx){
                                      auto ipj = (indx <= pmm ? indx % max_p_size: (indx - pmm) % (max_p_size-1) );
                                      auto pij = (indx <= pmm ? indx / max_p_size: n_pmax + (indx - pmm) / (max_p_size-1) );
                                      return num_partitions * ipj + pij;
                                    });

  auto uniq_tbl = cudf::experimental::detail::gather(input,
                                                     iter_begin, iter_begin + nrows,
                                                     false, false, false,
                                                     mr,
                                                     stream);


  rmm::device_vector<cudf::size_type> d_partition_offsets(num_partitions, cudf::size_type{0});

  auto exec = rmm::exec_policy(stream);
  thrust::transform(exec->on(stream),
                    thrust::make_counting_iterator<cudf::size_type>(0), thrust::make_counting_iterator<cudf::size_type>(num_partitions),
                    d_partition_offsets.begin(),
                    [num_partitions, max_p_size, n_pmax, pmm, start_partition] __device__ (auto indx){
                      auto unshifted_index = (num_partitions + indx - start_partition) % num_partitions;
                      if ( unshifted_index <= n_pmax )
                        {
                          return unshifted_index * max_p_size;
                        }
                      else
                        {
                          return pmm + (unshifted_index - n_pmax) * (max_p_size-1);
                        }
                    });

 
  ret_pair.first = std::move(uniq_tbl);
  hipMemcpy(ret_pair.second.data(), d_partition_offsets.data().get(), sizeof(cudf::size_type)*num_partitions, hipMemcpyDeviceToHost);

  return ret_pair;
}
  
}  // namespace detail

std::pair<std::unique_ptr<cudf::experimental::table>, std::vector<cudf::size_type>>
round_robin_partition(table_view const& input,
                      cudf::size_type num_partitions,
                      cudf::size_type start_partition = 0,
                      rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource()) {
  
  return cudf::experimental::detail::round_robin_partition(input, num_partitions, start_partition, mr);
}
  
}  // namespace experimental
}  // namespace cudf

/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/detail/gather.cuh>
#include <cudf/lists/extract.hpp>

namespace cudf {
namespace lists {
namespace detail {

/**
 * @copydoc cudf::lists::extract_list_element
 *
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
std::unique_ptr<column> extract_list_element(lists_column_view lists_column,
                                             size_type index,
                                             hipStream_t stream,
                                             rmm::mr::device_memory_resource* mr)
{
  if (lists_column.size() == 0) return empty_like(lists_column.parent());
  auto offsets_column = lists_column.offsets();

  // create a column_view with attributes of the parent and data from the offsets
  column_view annotated_offsets(data_type{type_id::INT32},
                                lists_column.size() + 1,
                                offsets_column.data<int32_t>(),
                                lists_column.null_mask(),
                                lists_column.null_count(),
                                lists_column.offset());

  // create a gather map for extracting elements from the child column
  auto gather_map = make_fixed_width_column(
    data_type{type_id::INT32}, annotated_offsets.size() - 1, mask_state::UNALLOCATED, stream);
  auto d_gather_map = gather_map->mutable_view().data<int32_t>();
  auto child_column = lists_column.child();
  CUDF_EXPECTS(child_column.type().id() != type_id::LIST,
               "Nested lists not yet supported in extract_list_element");

  // build the gather map using the offsets and the provided index
  auto child_size = child_column.size();  // used for out-of-bounds condition
  auto d_column   = column_device_view::create(annotated_offsets, stream);
  auto d_offsets  = *d_column;
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(gather_map->size()),
                    d_gather_map,
                    [d_offsets, index, child_size] __device__(auto idx) {
                      if (d_offsets.is_null(idx)) return child_size;
                      auto offset = d_offsets.element<int32_t>(idx);
                      auto length = d_offsets.element<int32_t>(idx + 1) - offset;
                      return index < length ? index + offset : child_size;
                    });

  // call gather on the child column
  auto result = cudf::detail::gather(table_view({child_column}),
                                     d_gather_map,
                                     d_gather_map + gather_map->size(),
                                     true,  // nullify-out-of-bounds
                                     mr,
                                     stream)
                  ->release();
  return std::unique_ptr<column>(std::move(result.front()));
}

}  // namespace detail

/**
 * @copydoc cudf::lists::extract_list_element
 */
std::unique_ptr<column> extract_list_element(lists_column_view const& lists_column,
                                             size_type index,
                                             rmm::mr::device_memory_resource* mr)
{
  return detail::extract_list_element(lists_column, index, 0, mr);
}

}  // namespace lists
}  // namespace cudf

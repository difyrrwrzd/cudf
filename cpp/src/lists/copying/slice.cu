#include <thrust/iterator/counting_iterator.h>
#include <cudf/detail/copy_range.cuh>
#include <cudf/detail/gather.cuh>
#include <cudf/lists/lists_column_view.hpp>
#include <iostream>

namespace cudf {
namespace lists {
namespace detail {

std::unique_ptr<cudf::column> slice(lists_column_view const& lists,
                                    size_type start,
                                    size_type end,
                                    hipStream_t stream,
                                    rmm::mr::device_memory_resource* mr)
{
  size_type lists_count = lists.size();
  if (lists_count == 0) { return cudf::empty_like(lists.parent()); }

  lists_count       = end - start;
  auto offsets_data = lists.offsets().data<cudf::size_type>();
  cudf::size_type start_offset{0};
  cudf::size_type end_offset{0};

  CUDA_TRY(hipMemcpyAsync(
    &start_offset, offsets_data + start, sizeof(cudf::size_type), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipMemcpyAsync(
    &end_offset, offsets_data + end, sizeof(cudf::size_type), hipMemcpyDeviceToHost, stream));
  rmm::device_uvector<cudf::size_type> offsets_buffer(lists_count + 1, stream);

  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    offsets_data + start,
                    offsets_data + end + 1,
                    offsets_buffer.data(),
                    [start_offset] __device__(cudf::size_type i) { return i - start_offset; });

  std::unique_ptr<cudf::column> child =
    (lists.child().type() == cudf::data_type{type_id::LIST})
      ? slice(lists_column_view(lists.child()), start_offset, end_offset, stream, mr)
      : std::make_unique<cudf::column>(lists.child(), stream, mr);

  auto null_mask = cudf::copy_bitmask(lists.null_mask(), start_offset, end_offset, stream, mr);
  auto offsets   = std::make_unique<cudf::column>(
    cudf::data_type{cudf::type_id::INT32}, lists_count + 1, offsets_buffer.release());
  return make_lists_column(lists_count,
                           std::move(offsets),
                           std::move(child),
                           cudf::UNKNOWN_NULL_COUNT,
                           std::move(null_mask));
}
}  // namespace detail
}  // namespace lists
}  // namespace cudf

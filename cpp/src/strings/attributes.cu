/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_handler.hpp>
#include <cudf/strings/string_view.cuh>

#include <thrust/transform.h>

namespace cudf 
{
namespace strings
{

std::unique_ptr<cudf::column> characters_counts( strings_column_handler handler,
                                                 hipStream_t stream )
{
    size_type count = handler.size();
    auto execpol = rmm::exec_policy(stream);
    auto strings_column = column_device_view::create(handler.parent_column(),stream);
    auto d_column = *strings_column;
    // create output column
    auto result = std::make_unique<cudf::column>( data_type{INT32}, count,
        rmm::device_buffer(count * sizeof(int32_t), stream, handler.memory_resource()),
        rmm::device_buffer(d_column.null_mask(), gdf_valid_allocation_size(count),
                           stream, handler.memory_resource()),
        d_column.null_count());
    auto results_view = result->mutable_view();
    auto d_lengths = results_view.data<int32_t>();
    // set lengths
    thrust::transform( execpol->on(stream), 
        thrust::make_counting_iterator<int32_t>(0),
        thrust::make_counting_iterator<int32_t>(count),
        d_lengths,
        [d_column] __device__ (int32_t idx) {
            if( d_column.nullable() && d_column.is_null(idx) )
                return 0;
            return d_column.element<string_view>(idx).characters();
        });
    return result;
}

std::unique_ptr<cudf::column> bytes_counts( strings_column_handler handler,
                                            hipStream_t stream )
{
    size_type count = handler.size();
    auto execpol = rmm::exec_policy(stream);
    auto strings_column = column_device_view::create(handler.parent_column(),stream);
    auto d_column = *strings_column;
    // create output column
    auto result = std::make_unique<cudf::column>( data_type{INT32}, count,
        rmm::device_buffer(count * sizeof(int32_t), stream, handler.memory_resource()),
        rmm::device_buffer(d_column.null_mask(), gdf_valid_allocation_size(count),
                           stream, handler.memory_resource()),
        d_column.null_count());
    auto results_view = result->mutable_view();
    auto d_lengths = results_view.data<int32_t>();
    // set sizes
    thrust::transform( execpol->on(stream), 
        thrust::make_counting_iterator<int32_t>(0),
        thrust::make_counting_iterator<int32_t>(count),
        d_lengths,
        [d_column] __device__ (int32_t idx) {
            if( d_column.nullable() && d_column.is_null(idx) )
                return 0;
            return d_column.element<string_view>(idx).size();
        });
    return result;
}


} // namespace strings
} // namespace cudf
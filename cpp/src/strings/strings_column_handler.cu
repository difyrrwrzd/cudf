/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_factories.hpp>
#include <cudf/strings/strings_column_handler.hpp>
#include <cudf/strings/string_view.cuh>
#include <utilities/error_utils.hpp>

#include <thrust/for_each.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/transform_scan.h>

namespace cudf {

//
strings_column_handler::strings_column_handler( const column_view& strings_column, rmm::mr::device_memory_resource* mr )
    : _parent(strings_column), _mr(mr)
{
    CUDF_EXPECTS( _parent.type().id()==STRING, "string_column_handler only support strings");
    CUDF_EXPECTS( _parent.num_children()>0, "string column must have children"); // revisit this (all nulls column?)
}

size_type strings_column_handler::count() const
{
    return _parent.child(0).size();
}

const char* strings_column_handler::chars_data() const
{
    return _parent.child(1).data<char>();
}

const int32_t* strings_column_handler::offsets_data() const
{
    return _parent.child(0).data<int32_t>();
}

size_type strings_column_handler::chars_column_size() const
{
    return _parent.child(1).size();
}

const bitmask_type* strings_column_handler::null_mask() const
{
    return _parent.null_mask();
}

size_type strings_column_handler::null_count() const
{
    return _parent.null_count();
}

// print strings to stdout
void strings_column_handler::print( size_type start, size_type end,
                                    size_type max_width, const char* delimiter ) const
{
    size_type count = this->count();
    if( end < 0 || end > count )
        end = count;
    if( start < 0 )
        start = 0;
    if( start >= end )
        throw std::invalid_argument("invalid parameter value");
    count = end - start;

    // stick with the default stream for this odd/rare stdout function
    auto execpol = rmm::exec_policy(0);
    auto strings_column = column_device_view::create(_parent);
    auto d_column = *strings_column;
    auto d_offsets = offsets_data();
    auto d_strings = chars_data();

    // create output strings offsets
    rmm::device_vector<size_t> output_offsets(count,0);
    thrust::transform_inclusive_scan( execpol->on(0),
        thrust::make_counting_iterator<size_type>(start), thrust::make_counting_iterator<size_type>(end),
        output_offsets.begin(),
        [d_column, d_strings, max_width, d_offsets] __device__ (size_type idx) {
            if( d_column.nullable() && d_column.is_null(idx) )
                return 0;
            size_type offset = idx ? d_offsets[idx-1] : 0; // this logic will be a template
            size_type bytes = d_offsets[idx] - offset;     // specialization on element()
            string_view dstr( d_strings + offset, bytes ); // method of column_device_view
            if( (max_width > 0) && (dstr.characters() > max_width) )
                bytes = dstr.byte_offset_for(max_width);
            return bytes+1; // allow for null-terminator on non-null strings
        },
        thrust::plus<int32_t>());
    // build output buffer
    size_t buffer_size = output_offsets.back(); // last element has total size
    if( buffer_size == 0 )
    {
        printf("all %d strings are null\n", count);
        return;
    }
    rmm::device_vector<char> buffer(buffer_size,0); // allocate and pre-null-terminate
    char* d_buffer = buffer.data().get();
    // copy strings into output buffer
    size_t* d_output_offsets = output_offsets.data().get();
    thrust::for_each_n(execpol->on(0),
        thrust::make_counting_iterator<size_type>(0), count,
        [d_strings, start, d_offsets, d_output_offsets, d_buffer] __device__(size_type idx) {
            size_t output_offset = (idx ? d_output_offsets[idx-1] : 0);
            size_t length = d_output_offsets[idx] - output_offset; // bytes
            if( length ) // this is only 0 for nulls
            {
                idx += start;
                size_type offset = (idx ? d_offsets[idx-1]:0);
                memcpy(d_buffer + output_offset, d_strings + offset, length-1 );
            }
        });

    // copy output buffer to host
    std::vector<size_t> h_offsets(count);
    hipMemcpy( h_offsets.data(), d_output_offsets, count*sizeof(size_t), hipMemcpyDeviceToHost);
    std::vector<char> h_buffer(buffer_size);
    hipMemcpy( h_buffer.data(), d_buffer, buffer_size, hipMemcpyDeviceToHost );

    // print out the strings to stdout
    for( size_type idx=0; idx < count; ++idx )
    {
        size_t offset = (idx ? h_offsets[idx-1]:0);
        size_t length = h_offsets[idx] - offset;
        printf("%d:",idx);
        if( length )
            printf("[%s]", h_buffer.data()+offset);
        else
            printf("<null>");
        printf("%s",delimiter);
    }
}

// new strings column from subset of this strings instance
std::unique_ptr<cudf::column> strings_column_handler::sublist( size_type start, size_type end,
                                                               size_type step, hipStream_t stream )
{
    if( step <= 0 )
        step = 1;
    size_type count = this->count();
    if( end < 0 || end > count )
        end = count;
    if( start < 0 || start > end )
        throw std::invalid_argument("invalid start parameter");
    count = (end - start)/step +1;
    //
    auto execpol = rmm::exec_policy(stream);
    auto strings_column = column_device_view::create(_parent, stream);
    auto d_column = *strings_column;

    // build indices
    thrust::device_vector<size_type> indices(count);
    thrust::sequence( execpol->on(stream), indices.begin(), indices.end(), start, step );
    // should have a way to create a column_view with an existing memory buffer
    auto indices_column = make_numeric_column( data_type{INT32}, count, mask_state::UNALLOCATED, stream, _mr );
    auto indices_view = indices_column->mutable_view();
    hipMemcpyAsync( indices_view.data<int32_t>(), indices.data().get(), count*sizeof(int32_t), hipMemcpyDeviceToDevice, stream);

    return gather(indices_view);
}

// return new strings column with strings from this instance as specified by the indices
std::unique_ptr<cudf::column> strings_column_handler::gather( const column_view& indices, hipStream_t stream )
{
    size_type count = indices.size();
    auto d_indices = indices.data<int32_t>();

    auto execpol = rmm::exec_policy(stream);
    auto strings_column = column_device_view::create(_parent,stream);
    auto d_column = *strings_column;
    auto d_offsets = offsets_data();

    // build offsets column
    auto offsets_column = make_numeric_column( data_type{INT32}, count, mask_state::UNALLOCATED, stream, _mr );
    auto offsets_view = offsets_column->mutable_view();
    auto d_new_offsets = offsets_view.data<int32_t>();
    // create new offsets array
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<size_type>(0), thrust::make_counting_iterator<size_type>(count),
        d_new_offsets,
        [d_column, d_offsets, d_indices] __device__ (size_type idx) {
            size_type index = d_indices[idx];
            if( d_column.nullable() && d_column.is_null(index) )
                return 0;
            size_type offset = index ? d_offsets[index-1] : 0;
            return d_offsets[index] - offset;
        },
        thrust::plus<int32_t>());
    // build null mask
    size_type null_count = this->null_count();
    mask_state state = mask_state::UNINITIALIZED;
    if( null_count==0 )
        state = mask_state::UNALLOCATED;
    else if( null_count==count )
        state = mask_state::ALL_NULL;
    auto null_mask = create_null_mask(count, state, stream, _mr);
    if( (null_count > 0) && (null_count < count) )
    {
        uint8_t* d_null_mask = static_cast<uint8_t*>(null_mask.data());
        CUDA_TRY(hipMemsetAsync(d_null_mask, 0, null_mask.size(), stream));
        thrust::transform(execpol->on(stream),
            thrust::make_counting_iterator<size_type>(0), thrust::make_counting_iterator<size_type>(count/8),
            d_null_mask,
            [d_column, count] __device__(size_type byte_idx) {
                unsigned char byte = 0; // set one byte per thread -- init to all nulls
                for( size_type i=0; i < 8; ++i )
                {
                    size_type idx = i + (byte_idx*8);  // compute d_strs index
                    byte = byte >> 1;                  // shift until we are done
                    if( idx < count )                  // check boundary
                    {
                      if( d_column.is_null(idx) )
                          byte |= 128;               // string is not null, set high bit
                    }
                }
                return byte; //d_null_mask[byte_idx] = byte;
            });
    }

    // build chars column
    size_type bytes = thrust::device_pointer_cast(d_new_offsets)[count-1]; // this may not be stream friendly
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED, stream, _mr );
    auto chars_view = chars_column->mutable_view();
    auto d_chars = chars_view.data<int8_t>(); 
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), count,
        [d_column, d_indices, d_new_offsets, d_chars] __device__(size_type idx){
            // place individual strings
            if( d_column.nullable() && d_column.is_null(idx) )
                return;
            string_view dstr = d_column.element<string_view>(d_indices[idx]);
            size_type offset = (idx ? d_new_offsets[idx-1] : 0);
            memcpy(d_chars + offset, dstr.data(), dstr.size() );
        });

  // build children vector
  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(offsets_column));
  children.emplace_back(std::move(chars_column));

  return std::make_unique<column>(
      data_type{STRING}, 0, rmm::device_buffer{0,stream,_mr},
      null_mask, null_count,
      std::move(children));   
}

// return sorted version of the given strings column
std::unique_ptr<cudf::column> strings_column_handler::sort( sort_type stype, bool ascending, bool nullfirst, hipStream_t stream )
{
    auto execpol = rmm::exec_policy(stream);
    auto strings_column = column_device_view::create(_parent, stream);
    auto d_column = *strings_column;

    // lets sort indices
    size_type count = this->count();
    thrust::device_vector<size_type> indices(count);
    thrust::sequence( execpol->on(stream), indices.begin(), indices.end() );
    thrust::sort( execpol->on(stream), indices.begin(), indices.end(),
        [d_column, stype, ascending, nullfirst] __device__ (size_type lhs, size_type rhs) {
            bool lhs_null{d_column.nullable() && d_column.is_null(lhs)};
            bool rhs_null{d_column.nullable() && d_column.is_null(rhs)};
            if( lhs_null || rhs_null )
                return (nullfirst ? !rhs_null : !lhs_null);
            string_view lhs_str = d_column.element<string_view>(lhs);
            string_view rhs_str = d_column.element<string_view>(rhs);
            int cmp = lhs_str.compare(rhs_str);
            return (ascending ? (cmp<0) : (cmp>0));
        });

    // should have a way to create a column_view with an existing memory buffer
    auto d_indices = indices.data().get();
    // we will create an empty one and pass in this data for now
    auto indices_column = make_numeric_column( data_type{INT32}, count, mask_state::UNALLOCATED, stream, _mr );
    auto indices_view = indices_column->mutable_view();
    hipMemcpyAsync( indices_view.data<int32_t>(), d_indices, count*sizeof(int32_t), hipMemcpyDeviceToDevice, stream);

    // now build a new strings column from the indices
    return gather( indices_view );
}


}  // namespace cudf
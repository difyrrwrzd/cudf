/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/char_types/char_types.hpp>
#include "../utilities.hpp"
#include "../utilities.cuh"


namespace cudf
{
namespace strings
{

//
std::unique_ptr<cudf::column> is_characters_of_type( strings_column_view strings,
                                                     string_character_types types,
                                                     rmm::mr::device_memory_resource* mr,
                                                     hipStream_t stream)
{
    auto strings_count = strings.size();
    auto execpol = rmm::exec_policy(0);
    auto strings_column = column_device_view::create(strings.parent(),stream);
    auto d_column = *strings_column;

    // copy the null mask
    rmm::device_buffer null_mask;
    cudf::size_type null_count = d_column.null_count();
    if( d_column.nullable() ) // copy null_mask
        null_mask = rmm::device_buffer( d_column.null_mask(),
                                        gdf_valid_allocation_size(strings_count),
                                        stream, mr);

    // create output column
    // TODO: use BOOL8 type here when available
    auto results = std::make_unique<cudf::column>( cudf::data_type{cudf::INT8}, strings_count,
        rmm::device_buffer(strings_count * sizeof(int8_t), stream, mr),
        null_mask, null_count);
    auto results_view = results->mutable_view();
    auto d_results = results_view.data<int8_t>();
    //
    auto d_flags = detail::get_character_flags_table();
    // set the output values by checking the character types
    thrust::for_each_n(execpol->on(stream),
        thrust::make_counting_iterator<size_type>(0), strings_count,
        [d_column, d_flags, types, d_results] __device__(size_type idx){
            if( d_column.is_null(idx) )
                return;
            auto d_str = d_column.element<string_view>(idx);
            bool check = !d_str.empty(); // positive result requires at least one character
            for( auto itr = d_str.begin(); check && (itr != d_str.end()); ++itr )
            {
                auto code_point = detail::utf8_to_codepoint(*itr);
                // lookup flags in table by code-point
                auto flag = code_point <= 0x00FFFF ? d_flags[code_point] : 0;
                check = (types & flag) > 0;
            }
            d_results[idx] = static_cast<int8_t>(check);
        });
    //
    results->set_null_count(null_count);
    return results;
}

} // namespace strings
} // namespace cudf

/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/case.hpp>
#include <cudf/utilities/error.hpp>
#include <strings/char_types/is_flags.h>
#include <strings/utilities.hpp>
#include <strings/utilities.cuh>


namespace cudf
{
namespace strings
{
namespace detail
{
namespace { // anonym.
  enum class pass_step : int { SizeOnly = 0, ExecuteOp};

  template<typename modifier_functor,
           pass_step Pass = pass_step::SizeOnly>
  struct case_manip
  {
    //selective construction based on class template parameter
    //
    //for SFINAE to work need a memf template parameter, `p`,
    //which defaults to Pass, in order to make the SFINAE dependent on Pass,
    //which really is the intention here
    //
    //otherwise, no SFINAE is possible, because inside the class
    //Pass is already known (fixed);
    //
    //specialization for ExecuteOp:
    //
    template<pass_step p = Pass>
    case_manip(modifier_functor d_fctr,
               column_device_view const d_column,
               character_flags_table_type case_flag,
               character_flags_table_type const* d_flags,
               character_cases_table_type const* d_case_table,
               int32_t const* d_offsets,
               char* d_chars,
               typename std::enable_if_t<p == pass_step::ExecuteOp>* = nullptr):
      d_functor_(d_fctr),
      d_column_(d_column),
      case_flag_(case_flag),
      d_flags_(d_flags),
      d_case_table_(d_case_table),
      d_offsets_(d_offsets),
      d_chars_(d_chars)
    {
    }

    //specialization for SizeOnly:
    //
    template<pass_step p = Pass>
    case_manip(modifier_functor d_fctr,
               column_device_view const d_column,
               character_flags_table_type case_flag,
               character_flags_table_type const* d_flags,
               character_cases_table_type const* d_case_table,
               typename std::enable_if_t<p != pass_step::ExecuteOp>* = nullptr):
      d_functor_(d_fctr),
      d_column_(d_column),
      case_flag_(case_flag),
      d_flags_(d_flags),
      d_case_table_(d_case_table),
      d_offsets_(nullptr),
      d_chars_(nullptr)
    {
    }

    //same SFINAE mechanism as the one for cnstr.
    //to specialize operator();
    //specialization for ExecuteOp:
    //
    template<pass_step p = Pass>
    __device__
    int32_t operator()(size_type row_index,
                       typename std::enable_if_t<p == pass_step::ExecuteOp>* = nullptr)
    {
      if( d_column_.is_null(row_index) )
        return 0; // null string

      string_view d_str = d_column_.template element<string_view>(row_index);
      char* d_buffer = nullptr;
      d_buffer = d_chars_ + d_offsets_[row_index];

      for( auto itr = d_str.begin(); itr != d_str.end(); ++itr )
        {
          uint32_t code_point = detail::utf8_to_codepoint(*itr);
          detail::character_flags_table_type flag = code_point <= 0x00FFFF ? d_flags_[code_point] : 0;

          d_functor_(d_buffer, d_case_table_, case_flag_, code_point, flag);
        }

      return 0;
    }

    //specialization for SizeOnly:
    //
    template<pass_step p = Pass>
    __device__
    int32_t operator()(size_type row_index,
                       typename std::enable_if_t<p != pass_step::ExecuteOp>* = nullptr)
    {
      if( d_column_.is_null(row_index) )
        return 0; // null string
      
      int32_t bytes = 0;
      string_view d_str = d_column_.template element<string_view>(row_index);
      for( auto itr = d_str.begin(); itr != d_str.end(); ++itr )
        {
            uint32_t code_point = detail::utf8_to_codepoint(*itr);
            detail::character_flags_table_type flag = code_point <= 0x00FFFF ? d_flags_[code_point] : 0;
            if( flag & case_flag_ )
            {
              bytes += detail::bytes_in_char_utf8(detail::codepoint_to_utf8(d_case_table_[code_point]));
            }
            else
            {
              bytes += detail::bytes_in_char_utf8(*itr);
            }
        }
        return bytes;
    }
  private:
    modifier_functor d_functor_;
    column_device_view const d_column_;
    character_flags_table_type case_flag_; // flag to check with on each character
    character_flags_table_type const* d_flags_;
    character_cases_table_type const* d_case_table_;
    int32_t const* d_offsets_;
    char* d_chars_;
  };


  struct probe_capitalize
  {
    probe_capitalize(column_device_view const d_column,
                     character_flags_table_type const* d_flags,
                     character_cases_table_type const* d_case_table):
      d_column_(d_column),
      d_flags_(d_flags),
      d_case_table_(d_case_table)
    {  
    }
    
     __device__
     int32_t operator()(size_type idx) const {
      if( d_column_.is_null(idx) )
        return 0; // null string
      
      string_view d_str = d_column_.template element<string_view>(idx);
      int32_t bytes = 0;
      
      for( auto itr = d_str.begin(); itr != d_str.end(); ++itr ) {
        auto the_chr = *itr;
        uint32_t code_point = detail::utf8_to_codepoint(the_chr);
        detail::character_flags_table_type flag = code_point <= 0x00FFFF ? d_flags_[code_point] : 0;
        if( (itr == d_str.begin()) ? IS_LOWER(flag) : IS_UPPER(flag) )
          the_chr = detail::codepoint_to_utf8(d_case_table_[code_point]);
        bytes += detail::bytes_in_char_utf8(the_chr);
      }
      return bytes;
    }
  private:
    column_device_view const d_column_;
    character_flags_table_type const* d_flags_;
    character_cases_table_type const* d_case_table_;
  };

  struct execute_capitalize
  {
    execute_capitalize(column_device_view const d_column,
                       character_flags_table_type const* d_flags,
                       character_cases_table_type const* d_case_table,
                       int32_t const* d_offsets,
                       char* d_chars):
      d_column_(d_column),
      d_flags_(d_flags),
      d_case_table_(d_case_table),
      d_offsets_(d_offsets),
      d_chars_(d_chars)
    {
    }
    
    __device__
    int32_t operator()(size_type idx) {
      if( d_column_.is_null(idx) )
        return 0; // null string
      
      string_view d_str = d_column_.template element<string_view>(idx);
      char* d_buffer = d_chars_ + d_offsets_[idx];
      
      for( auto itr = d_str.begin(); itr != d_str.end(); ++itr ) {
        auto the_chr = *itr;
        uint32_t code_point = detail::utf8_to_codepoint(the_chr);
        detail::character_flags_table_type flag = code_point <= 0x00FFFF ? d_flags_[code_point] : 0;

        if( (itr == d_str.begin()) ? IS_LOWER(flag) : IS_UPPER(flag) )
          the_chr = detail::codepoint_to_utf8(d_case_table_[code_point]);
        d_buffer += detail::from_char_utf8(the_chr, d_buffer);
      }
      return 0;
    }
  private:
    column_device_view const d_column_;
    character_flags_table_type const* d_flags_;
    character_cases_table_type const* d_case_table_;
    int32_t const* d_offsets_;
    char* d_chars_;
  };
         
}//anonym.

template<typename device_probe_functor,
         typename device_modifier_functor>
std::unique_ptr<column> modify_strings( strings_column_view const& strings,
                                        character_flags_table_type case_flag,
                                        device_probe_functor d_probe_fctr,
                                        device_modifier_functor d_modifier_fctr,
                                        rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                        hipStream_t stream = 0)
{
  auto strings_count = strings.size();
  if( strings_count == 0 )
    return detail::make_empty_strings_column(mr,stream);

  auto execpol = rmm::exec_policy(stream);
  
  auto strings_column = column_device_view::create(strings.parent(),stream);
  auto d_column = *strings_column;

  // copy null mask
  rmm::device_buffer null_mask = copy_bitmask(strings.parent(),stream,mr);
  // get the lookup tables used for case conversion
  auto d_flags = get_character_flags_table();
  auto d_case_table = get_character_cases_table();  

  detail::case_manip<decltype(d_probe_fctr), pass_step::SizeOnly> cprobe{d_probe_fctr,
      d_column,
      case_flag,
      d_flags,
      d_case_table};


  // build offsets column -- calculate the size of each output string
  auto offsets_transformer_itr = thrust::make_transform_iterator( thrust::make_counting_iterator<size_type>(0), cprobe);
  auto offsets_column = detail::make_offsets_child_column(offsets_transformer_itr,
                                                          offsets_transformer_itr+strings_count,
                                                          mr, stream);
  auto offsets_view = offsets_column->view();
  auto d_new_offsets = offsets_view.template data<int32_t>();//not sure why this requires `.template` and the next one (`d_chars = ...`) doesn't

  // build the chars column -- convert characters based on case_flag parameter
  size_type bytes = thrust::device_pointer_cast(d_new_offsets)[strings_count];
  auto chars_column = strings::detail::create_chars_child_column( strings_count, d_column.null_count(), bytes, mr, stream );
  auto chars_view = chars_column->mutable_view();
  auto d_chars = chars_view.data<char>();

  detail::case_manip<device_modifier_functor, pass_step::ExecuteOp> cmanip{d_modifier_fctr,
      d_column,
      case_flag,
      d_flags,
      d_case_table,
      d_new_offsets,
      d_chars};
  
  thrust::for_each_n(execpol->on(stream),
                     thrust::make_counting_iterator<size_type>(0), strings_count, cmanip);
  
  //
  return make_strings_column(strings_count, std::move(offsets_column), std::move(chars_column),
                             d_column.null_count(), std::move(null_mask), stream, mr);
}

}//namespace detail

std::unique_ptr<column> capitalize( strings_column_view const& strings,
                                    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource())
{
  //TODO:
  //
  auto d_probe_fctr = [] __device__ (char* d_buffer,
                                     detail::character_cases_table_type const* d_case_table,
                                     detail::character_flags_table_type case_flag,
                                     uint32_t code_point,
                                     detail::character_flags_table_type flag){
    //purposely empty; used just to instantiate a sizeOnly `case_manip` that doesn't need a functor
  };

  
  //TODO:
  //
  auto d_modifier_fctr = [] __device__ (char* d_buffer,
                             detail::character_cases_table_type const* d_case_table,
                             detail::character_flags_table_type case_flag,
                             uint32_t code_point,
                             detail::character_flags_table_type flag){
    //TODO:
    //....
  };//nothing for now...

  detail::character_flags_table_type case_flag = IS_LOWER(0xFF);// <- ????? for now; TODO

  return detail::modify_strings(strings,
                                case_flag,
                                d_probe_fctr,
                                d_modifier_fctr,
                                mr);
}

std::unique_ptr<column> title( strings_column_view const& strings,
                               rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource())
{
  //TODO:
  //
  auto d_probe_fctr = [] __device__ (char* d_buffer,
                                     detail::character_cases_table_type const* d_case_table,
                                     detail::character_flags_table_type case_flag,
                                     uint32_t code_point,
                                     detail::character_flags_table_type flag){
    //purposely empty; used just to instantiate a sizeOnly `case_manip` that doesn't need a functor
  };

  
  //TODO:
  //
  auto d_modifier_fctr = [] __device__ (char* d_buffer,
                             detail::character_cases_table_type const* d_case_table,
                             detail::character_flags_table_type case_flag,
                             uint32_t code_point,
                             detail::character_flags_table_type flag){
    //TODO:
    //....
  };//nothing for now...

  detail::character_flags_table_type case_flag = IS_LOWER(0xFF);// <- ????? for now; TODO

  return detail::modify_strings(strings,
                                case_flag,
                                d_probe_fctr,
                                d_modifier_fctr,
                                mr);
}
  
}//namespace strings
}//namespace cudf

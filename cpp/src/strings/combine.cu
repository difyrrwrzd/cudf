/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <bitmask/valid_if.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <utilities/error_utils.hpp>
#include "./utilities.hpp"
#include "./utilities.cuh"

#include <algorithm>
#include <rmm/thrust_rmm_allocator.h>
#include <thrust/transform_scan.h>

namespace cudf
{
namespace strings
{

std::unique_ptr<cudf::column> concatenate( strings_column_view strings,
                                           strings_column_view others,
                                           const char* separator,
                                           const char* narep,
                                           hipStream_t stream,
                                           rmm::mr::device_memory_resource* mr )
{
    CUDF_EXPECTS( strings.size()==others.size(), "columns must be the same size");

    auto execpol = rmm::exec_policy(stream);
    size_type count = strings.size();

    if( !separator )
        separator = "";
    auto separator_ptr = detail::string_from_host(separator, stream);
    auto d_separator = *separator_ptr;
    auto narep_ptr = detail::string_from_host(narep, stream);
    string_view d_narep(nullptr,0);
    if( narep_ptr )
        d_narep = *narep_ptr;

    // create strings arrays
    auto strings_column_ptr = column_device_view::create(strings.parent(),stream);
    auto d_strings = *strings_column_ptr;
    auto others_column_ptr = column_device_view::create(others.parent(),stream);
    auto d_others = *others_column_ptr;

    // create resulting null mask
    auto valid_mask = valid_if( static_cast<const bit_mask_t*>(nullptr),
        [d_strings, d_others, d_narep] __device__ (size_type idx) {
            return !(((d_strings.nullable() && d_strings.is_null(idx)) ||
                     (d_others.nullable() && d_others.is_null(idx))) &&
                      d_narep.is_null());
        },
        count, stream );
    auto null_count = valid_mask.second;
    auto null_size = gdf_valid_allocation_size(count);
    rmm::device_buffer null_mask(valid_mask.first,null_size,stream,mr); // does deep copy
    RMM_TRY( RMM_FREE(valid_mask.first,stream) ); // TODO valid_if to return device_buffer in future

    // build offsets column
    auto offsets_column = make_numeric_column( data_type{INT32}, count+1, mask_state::UNALLOCATED,
                                               stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    auto d_results_offsets = offsets_view.data<int32_t>();
    // compute offsets
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<unsigned int>(0),
        thrust::make_counting_iterator<unsigned int>(count),
        d_results_offsets+1,
        [d_strings, d_others, d_separator, d_narep] __device__ (size_type idx) {
            string_view d_str1;
            if( d_strings.nullable() && d_strings.is_null(idx) )
                d_str1 = string_view(nullptr,0);
            else
                d_str1 = d_strings.element<string_view>(idx);
            string_view d_str2;
            if( d_others.nullable() && d_others.is_null(idx) )
                d_str2 = string_view(nullptr,0);
            else
                d_str2 = d_others.element<string_view>(idx);
            if( (d_str1.is_null() || d_str2.is_null()) && d_narep.is_null() )
                return 0; // null output case
            size_type bytes = 0;
            // left-side
            if( !d_str1.is_null() )
                bytes = d_str1.size_bytes();
            else if( !d_narep.is_null() )
                bytes = d_narep.size_bytes();
            // separator
            bytes += d_separator.size_bytes();
            if( !d_str2.is_null() )
                bytes += d_str2.size_bytes();
            else if( !d_narep.is_null() )
                bytes += d_narep.size_bytes();
            return bytes;
        },
        thrust::plus<int32_t>() );
    CUDA_TRY(hipMemsetAsync( d_results_offsets, 0, sizeof(*d_results_offsets), stream));

    // build chars column
    size_type bytes = thrust::device_pointer_cast(d_results_offsets)[count];
    if( (bytes==0) && (null_count < count) )
        bytes = 1; // all entries are empty strings
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED,
                                             stream, mr );
    auto chars_view = chars_column->mutable_view();
    auto d_results_chars = chars_view.data<char>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), count,
        [d_strings, d_others, d_separator, d_narep, d_results_offsets, d_results_chars] __device__(size_type idx){
            string_view d_str1;
            if( d_strings.nullable() && d_strings.is_null(idx) )
                d_str1 = string_view(nullptr,0);
            else
                d_str1 = d_strings.element<string_view>(idx);
            string_view d_str2;
            if( d_others.nullable() && d_others.is_null(idx) )
                d_str2 = string_view(nullptr,0);
            else
                d_str2 = d_others.element<string_view>(idx);
            if( (d_str1.is_null() || d_str2.is_null()) && d_narep.is_null() )
                return; // null -- nothing to do
            // concat the two strings with appropriate separator and narep
            size_type offset = d_results_offsets[idx];
            char* d_buffer = d_results_chars + offset;
            if( !d_str1.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_str1);
            else if( !d_narep.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_narep);
            if( !d_separator.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_separator);
            if( !d_str2.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_str2);
            else if( !d_narep.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_narep);
        });

    // build children vector
    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));

    return std::make_unique<column>(
        data_type{STRING}, count, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}

#if 0
std::unique_ptr<cudf::column> concatenate( std::vector<strings_column_view>& strings_columns,
                                           const char* separator,
                                           const char* narep,
                                           hipStream_t stream,
                                           rmm::mr::device_memory_resource* mr )
{
    auto num_columns = strings_columns.size();
    CUDF_EXPECTS( num_columns>1, "concatenate requires at least 2 columns");

    auto first_column = column_device_view::create(strings_columns[0].parent(),stream);
    auto num_strings = first_column->size();
    if( std::all_of(strings_columns.begin(),strings_columns.end(),
        [num_strings] (strings_column_view view) { return num_strings==view.size(); }) )
    {
        CUDF_FAIL( "concatenate requires all columns have an equal number of rows");
    }

    auto execpol = rmm::exec_policy(stream);
    if( !separator )
        separator = "";
    auto separator_ptr = detail::string_from_host(separator, stream);
    auto d_separator = *separator_ptr;
    auto narep_ptr = detail::string_from_host(narep, stream);
    string_view d_narep(nullptr,0);
    if( narep_ptr )
        d_narep = *narep_ptr;

    // Create device views from the strings columns.
    //
    // First calculate the size of memory needed to hold the
    // column_device_views. This is done by calling extent()
    // for each of the column_views of the strings_columns.
    size_type views_size_bytes =
        std::accumulate(strings_columns.begin(), strings_columns.end(), 0,
            [](size_type init, strings_column_view col) {
                return init + column_device_view::extent(col.parent());
            });
    // Allocate the device memory to be used in the device methods.
    // We need to pass this down when creating the column_device_views
    // so they can be resolved to point to any child objects.
    column_device_view* d_columns;
    RMM_TRY(RMM_ALLOC(&d_columns, views_size_bytes, stream));
    column_device_view* d_column = d_columns; // point to the first one
    // A buffer of CPU memory is created to hold the column_device_view
    // objects and then copied to device memory at the d_columns pointer.
    // But each column_device_view instance may have child objects which
    // require setting an internal device pointer before being copied from
    // CPU to device.
    {
        std::vector<int8_t> h_buffer(views_size_bytes);
        column_device_view* h_column = reinterpret_cast<column_device_view*>(h_buffer.data());
        // The beginning of the memory must be the fixed-sized column_device_view
        // objects in order for d_columns to be used as array. Therefore, any
        // child data is assigned to the end of this array.
        int8_t* h_end = (int8_t*)(h_column + num_columns);
        int8_t* d_end = (int8_t*)(d_column + num_columns);
        // Create the column_device_view from each column within the CPU memory
        // array. Any column child data should be copied into h_end and any
        // internal pointers should be set using d_end.
        for( auto itr=strings_columns.begin(); itr!=strings_columns.end(); ++itr )
        {
            auto col = itr->parent();
            // convert the column_view into column_device_view
            new(h_column) column_device_view(col,(ptrdiff_t)h_end,(ptrdiff_t)d_end);
            h_column++; // next element in array
            // point to the next chunk of memory for use of the children of the next column
            auto col_child_data_size = (column_device_view::extent(col) - sizeof(column_device_view));
            h_end += col_child_data_size;
            d_end += col_child_data_size;
        }
        CUDA_TRY(hipMemcpyAsync(d_columns, h_buffer.data(),
                                 views_size_bytes, hipMemcpyDefault, stream));
    }
    

    // create resulting null mask
    auto valid_mask = valid_if( static_cast<const bit_mask_t*>(nullptr),
        [d_strings, d_others, d_narep] __device__ (size_type idx) {
            return !(((d_strings.nullable() && d_strings.is_null(idx)) ||
                     (d_others.nullable() && d_others.is_null(idx))) &&
                      d_narep.is_null());
        },
        count, stream );
    auto null_count = valid_mask.second;
    auto null_size = gdf_valid_allocation_size(count);
    rmm::device_buffer null_mask(valid_mask.first,null_size,stream,mr); // does deep copy
    RMM_TRY( RMM_FREE(valid_mask.first,stream) ); // TODO valid_if to return device_buffer in future

    // build offsets column
    auto offsets_column = make_numeric_column( data_type{INT32}, count+1, mask_state::UNALLOCATED,
                                               stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    auto d_results_offsets = offsets_view.data<int32_t>();
    // compute offsets
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<unsigned int>(0),
        thrust::make_counting_iterator<unsigned int>(count),
        d_results_offsets+1,
        [d_strings, d_others, d_separator, d_narep] __device__ (size_type idx) {
            string_view d_str1;
            if( d_strings.nullable() && d_strings.is_null(idx) )
                d_str1 = string_view(nullptr,0);
            else
                d_str1 = d_strings.element<string_view>(idx);
            string_view d_str2;
            if( d_others.nullable() && d_others.is_null(idx) )
                d_str2 = string_view(nullptr,0);
            else
                d_str2 = d_others.element<string_view>(idx);
            if( (d_str1.is_null() || d_str2.is_null()) && d_narep.is_null() )
                return 0; // null output case
            size_type bytes = 0;
            // left-side
            if( !d_str1.is_null() )
                bytes = d_str1.size_bytes();
            else if( !d_narep.is_null() )
                bytes = d_narep.size_bytes();
            // separator
            bytes += d_separator.size_bytes();
            if( !d_str2.is_null() )
                bytes += d_str2.size_bytes();
            else if( !d_narep.is_null() )
                bytes += d_narep.size_bytes();
            return bytes;
        },
        thrust::plus<int32_t>() );
    CUDA_TRY(hipMemsetAsync( d_results_offsets, 0, sizeof(*d_results_offsets), stream));

    // build chars column
    size_type bytes = thrust::device_pointer_cast(d_results_offsets)[count];
    if( (bytes==0) && (null_count < count) )
        bytes = 1; // all entries are empty strings
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED,
                                             stream, mr );
    auto chars_view = chars_column->mutable_view();
    auto d_results_chars = chars_view.data<char>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), count,
        [d_strings, d_others, d_separator, d_narep, d_results_offsets, d_results_chars] __device__(size_type idx){
            string_view d_str1;
            if( d_strings.nullable() && d_strings.is_null(idx) )
                d_str1 = string_view(nullptr,0);
            else
                d_str1 = d_strings.element<string_view>(idx);
            string_view d_str2;
            if( d_others.nullable() && d_others.is_null(idx) )
                d_str2 = string_view(nullptr,0);
            else
                d_str2 = d_others.element<string_view>(idx);
            if( (d_str1.is_null() || d_str2.is_null()) && d_narep.is_null() )
                return; // null -- nothing to do
            // concat the two strings with appropriate separator and narep
            size_type offset = d_results_offsets[idx];
            char* d_buffer = d_results_chars + offset;
            if( !d_str1.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_str1);
            else if( !d_narep.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_narep);
            if( !d_separator.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_separator);
            if( !d_str2.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_str2);
            else if( !d_narep.is_null() )
                d_buffer = detail::copy_string(d_buffer, d_narep);
        });

    // build children vector
    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));

    return std::make_unique<column>(
        data_type{STRING}, count, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}
#endif

} // namespace strings
} // namespace cudf

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <bitmask/valid_if.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <utilities/error_utils.hpp>
#include "./utilities.hpp"
#include "./utilities.cuh"

#include <algorithm>
#include <numeric>
#include <rmm/thrust_rmm_allocator.h>
#include <thrust/transform_scan.h>
#include <thrust/logical.h>

namespace cudf
{
namespace strings
{

//
std::unique_ptr<cudf::column> concatenate( std::vector<strings_column_view>& strings_columns,
                                           const char* separator,
                                           const char* narep,
                                           hipStream_t stream,
                                           rmm::mr::device_memory_resource* mr )
{
    auto num_columns = strings_columns.size();
    CUDF_EXPECTS( num_columns>1, "concatenate requires at least 2 columns");

    auto first_column = column_device_view::create(strings_columns[0].parent(),stream);
    auto num_strings = first_column->size();
    if( !std::all_of(strings_columns.begin(),strings_columns.end(),
        [num_strings] (strings_column_view view) { return num_strings==view.size(); }) )
    {
        CUDF_FAIL( "concatenate requires all columns have an equal number of rows");
    }

    auto execpol = rmm::exec_policy(stream);
    if( !separator )
        separator = "";
    auto separator_ptr = detail::string_from_host(separator, stream);
    auto d_separator = *separator_ptr;
    auto narep_ptr = detail::string_from_host(narep, stream);
    string_view d_narep(nullptr,0);
    if( narep_ptr )
        d_narep = *narep_ptr;

    // Create device views from the strings columns.
    //
    // First calculate the size of memory needed to hold the
    // column_device_views. This is done by calling extent()
    // for each of the column_views of the strings_columns.
    size_type views_size_bytes =
        std::accumulate(strings_columns.begin(), strings_columns.end(), 0,
            [](size_type init, strings_column_view col) {
                return init + column_device_view::extent(col.parent());
            });
    // Allocate the device memory to be used in the device methods.
    // We need to pass this down when creating the column_device_views
    // so they can be resolved to point to any child objects.
    column_device_view* d_columns;
    RMM_TRY(RMM_ALLOC(&d_columns, views_size_bytes, stream));
    column_device_view* d_column = d_columns; // point to the first one
    // A buffer of CPU memory is created to hold the column_device_view
    // objects and then copied to device memory at the d_columns pointer.
    // But each column_device_view instance may have child objects which
    // require setting an internal device pointer before being copied from
    // CPU to device.
    {
        std::vector<int8_t> h_buffer(views_size_bytes);
        column_device_view* h_column = reinterpret_cast<column_device_view*>(h_buffer.data());
        // The beginning of the memory must be the fixed-sized column_device_view
        // objects in order for d_columns to be used as array. Therefore, any
        // child data is assigned to the end of this array.
        int8_t* h_end = (int8_t*)(h_column + num_columns);
        int8_t* d_end = (int8_t*)(d_column + num_columns);
        // Create the column_device_view from each column within the CPU memory
        // array. Any column child data should be copied into h_end and any
        // internal pointers should be set using d_end.
        for( auto itr=strings_columns.begin(); itr!=strings_columns.end(); ++itr )
        {
            auto col = itr->parent();
            // convert the column_view into column_device_view
            new(h_column) column_device_view(col,(ptrdiff_t)h_end,(ptrdiff_t)d_end);
            h_column++; // next element in array
            // point to the next chunk of memory for use of the children of the next column
            auto col_child_data_size = (column_device_view::extent(col) - sizeof(column_device_view));
            h_end += col_child_data_size;
            d_end += col_child_data_size;
        }
        CUDA_TRY(hipMemcpyAsync(d_columns, h_buffer.data(),
                                 views_size_bytes, hipMemcpyDefault, stream));
    }

    // create resulting null mask
    auto valid_mask = valid_if( static_cast<const bit_mask_t*>(nullptr),
        [d_columns, num_columns, d_narep] __device__ (size_type idx) {
            bool null_element = thrust::any_of( thrust::seq, d_columns, d_columns+num_columns,
                [idx] (column_device_view col) { return col.nullable() && col.is_null(idx);});
            return( !null_element || !d_narep.is_null() );
        },
        num_strings, stream );
    auto null_count = valid_mask.second;
    auto null_size = gdf_valid_allocation_size(num_strings);
    rmm::device_buffer null_mask(valid_mask.first,null_size,stream,mr); // does deep copy
    RMM_TRY( RMM_FREE(valid_mask.first,stream) ); // TODO valid_if to return device_buffer in future

    // build offsets column by computing sizes of each string in the output
    auto offsets_column = make_numeric_column( data_type{INT32}, num_strings+1, mask_state::UNALLOCATED,
                                               stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    auto d_results_offsets = offsets_view.data<int32_t>();
    //
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<unsigned int>(0),
        thrust::make_counting_iterator<unsigned int>(num_strings),
        d_results_offsets+1,
        [d_columns, num_columns, d_separator, d_narep] __device__ (size_type idx) {
            size_type bytes = 0;
            for( size_type col_idx=0; col_idx < num_columns; ++col_idx )
            {
                auto d_column = d_columns[col_idx];
                if( d_column.nullable() && d_column.is_null(idx) )
                {
                    if( d_narep.is_null() )
                        return 0; // null entry in result
                    bytes += d_narep.size_bytes();
                }
                else
                    bytes += d_column.element<string_view>(idx).size_bytes();
                // separator only in between elements
                if( col_idx+1 < num_columns )
                    bytes += d_separator.size_bytes();
            }
            return bytes;
        },
        thrust::plus<int32_t>() );
    // set first offset entry to zero
    CUDA_TRY(hipMemsetAsync( d_results_offsets, 0, sizeof(*d_results_offsets), stream));

    // create the chars column
    size_type bytes = thrust::device_pointer_cast(d_results_offsets)[num_strings];
    if( (bytes==0) && (null_count < num_strings) )
        bytes = 1; // all entries are empty strings

    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED,
                                             stream, mr );
    // fill the chars column
    auto chars_view = chars_column->mutable_view();
    auto d_results_chars = chars_view.data<char>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), num_strings,
        [d_columns, num_columns, d_separator, d_narep, d_results_offsets, d_results_chars] __device__(size_type idx){
            bool null_element = thrust::any_of( thrust::seq, d_columns, d_columns+num_columns,
                    [idx] (column_device_view col) { return col.nullable() && col.is_null(idx);});
            if( null_element && d_narep.is_null() )
                return; // do not write to buffer at all if any element is null
            size_type offset = d_results_offsets[idx];
            char* d_buffer = d_results_chars + offset;
            for( size_type col_idx=0; col_idx < num_columns; ++col_idx )
            {
                auto d_column = d_columns[col_idx];
                if( d_column.nullable() && d_column.is_null(idx) )
                    d_buffer = detail::copy_string(d_buffer, d_narep);
                else
                {
                    string_view d_str = d_column.element<string_view>(idx);
                    d_buffer = detail::copy_string(d_buffer, d_str);
                }
                // separator only in between elements
                if( col_idx+1 < num_columns )
                    d_buffer = detail::copy_string(d_buffer, d_separator);
            }
        });

    // build children vector
    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));

    return std::make_unique<column>(
        data_type{STRING}, num_strings, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}

/**---------------------------------------------------------------------------*
 * @brief Concatenates all strings in the column into one new string.
 * This provides the Pandas strings equivalent of join().
 *
 * @param strings Strings for this operation.
 * @param separator Null-terminated CPU string that should appear between each string.
 * @param narep Null-terminated CPU string that should represent any null strings found.
 * @param stream CUDA stream to use kernels in this method.
 * @param mr Resource for allocating device memory.
 * @return New column containing one string.
 *---------------------------------------------------------------------------**/
std::unique_ptr<cudf::column> join_strings( strings_column_view strings,
                                            const char* separator,
                                            const char* narep,
                                            hipStream_t stream,
                                            rmm::mr::device_memory_resource* mr )
{
    auto execpol = rmm::exec_policy(stream);
    if( !separator )
        separator = "";
    auto separator_ptr = detail::string_from_host(separator, stream);
    auto d_separator = *separator_ptr;
    auto narep_ptr = detail::string_from_host(narep, stream);
    string_view d_narep(nullptr,0);
    if( narep_ptr )
        d_narep = *narep_ptr;

    auto num_strings = strings.size();
    auto strings_column = column_device_view::create(strings.parent(),stream);
    auto d_column = *strings_column;
    auto d_offsets = strings.offsets().data<int32_t>();

    // create an offsets array for building the output memory layout
    rmm::device_vector<size_type> output_offsets(num_strings+1);
    auto d_output_offsets = output_offsets.data().get();
    // using inclusive-scan to compute last entry which is the total size
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(num_strings),
        d_output_offsets + 1,
        [d_column, d_separator, d_narep] __device__ (size_type idx) {
            size_type bytes = 0;
            if( d_column.nullable() && d_column.is_null(idx) )
            {
                if( d_narep.is_null() )
                    return 0; // skip nulls
                bytes += d_narep.size_bytes();
            }
            else
                bytes += d_column.element<string_view>(idx).size_bytes();
            if( (idx+1) < d_column.size() )
                bytes += d_separator.size_bytes();
            return bytes;
        },
        thrust::plus<int32_t>());
    // total size is the last entry
    size_type bytes = output_offsets.back();

     // build offsets column (only 1 string so 2 offset entries)
    auto offsets_column = make_numeric_column( data_type{INT32}, 2, mask_state::UNALLOCATED,
                                               stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    // set the first entry to 0 and the last entry to bytes
    int32_t new_offsets[] = {0, bytes};
    CUDA_TRY(hipMemcpyAsync(offsets_view.data<int32_t>(), new_offsets,
                             sizeof(new_offsets), hipMemcpyHostToDevice,stream));

    // build null mask
    // one entry so it is either all valid or all null
    size_type null_count = 0;
    rmm::device_buffer null_mask; // init to null null-mask
    if( strings.null_count()==num_strings )
    {
        null_mask = create_null_mask(1,cudf::ALL_NULL,stream,mr);
        null_count = 1;
    }
    else if( bytes==0 ) // If not all nulls and bytes is zero, then all strings are empty.
        bytes = 1;      // Still need 1 byte to make a valid/non-null chars column

    // build chars column
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED,
                                             stream, mr );
    auto chars_view = chars_column->mutable_view();
    auto d_chars = chars_view.data<char>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), num_strings,
        [d_column, d_separator, d_narep, d_output_offsets, d_chars] __device__(size_type idx){
            size_type offset = d_output_offsets[idx];
            char* d_buffer = d_chars + offset;
            if( d_column.nullable() && d_column.is_null(idx) )
            {
                if( d_narep.is_null() )
                    return; // do not write to buffer if element is null (including separator)
                d_buffer = detail::copy_string(d_buffer, d_narep);
            }
            else
            {
                string_view d_str = d_column.element<string_view>(idx);
                d_buffer = detail::copy_string(d_buffer, d_str);
            }
            if( (idx+1) < d_column.size() )
                d_buffer = detail::copy_string(d_buffer, d_separator);
        });

    // build children vector
    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));
    // return the single-string column
    return std::make_unique<column>(
        data_type{STRING}, 1, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}

} // namespace strings
} // namespace cudf

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/case.hpp>
#include <cudf/utilities/error.hpp>
#include "char_types/is_flags.h"
#include "./utilities.hpp"
#include "./utilities.cuh"

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>

namespace cudf
{
namespace strings
{
namespace
{

/**
 * @brief Used as template parameter to divide size calculation from
 * the actual string operation within a function.
 * Useful when most of the logic is identical for both passes.
 */
enum TwoPass
{
    SizeOnly = 0, ///< calculate the size only
    ExecuteOp     ///< run the string operation
};

/**
 * @brief Function logic for the substring API.
 * This will perform a substring operation on each string
 * using the provided start, stop, and step parameters.
 */
template <TwoPass Pass=SizeOnly>
struct upper_lower_fn
{
    const column_device_view d_column;
    detail::character_flags_table_type case_flag; // flag to check with on each character
    const detail::character_flags_table_type* d_flags;
    const detail::character_cases_table_type* d_case_table;
    const int32_t* d_offsets{};
    char* d_chars{};

    __device__ int32_t operator()(size_type idx)
    {
        if( d_column.is_null(idx) )
            return 0; // null string
        string_view d_str = d_column.template element<string_view>(idx);
        int32_t bytes = 0;
        char* d_buffer = nullptr;
        if( Pass==ExecuteOp )
            d_buffer = d_chars + d_offsets[idx];
        for( auto itr = d_str.begin(); itr != d_str.end(); ++itr )
        {
            uint32_t code_point = detail::utf8_to_codepoint(*itr);
            detail::character_flags_table_type flag = code_point <= 0x00FFFF ? d_flags[code_point] : 0;
            if( flag & case_flag )
            {
                if( Pass==SizeOnly )
                    bytes += detail::bytes_in_char_utf8(detail::codepoint_to_utf8(d_case_table[code_point]));
                else
                    d_buffer += detail::from_char_utf8(detail::codepoint_to_utf8(d_case_table[code_point]),d_buffer);
            }
            else
            {
                if( Pass==SizeOnly )
                    bytes += detail::bytes_in_char_utf8(*itr);
                else
                    d_buffer += detail::from_char_utf8(*itr, d_buffer);
            }
        }
        return bytes;
    }
};

/**
 * @brief Utility method for converting upper and lower case characters
 * in a strings column.
 *
 * @param strings Strings to convert.
 * @param case_flag The character type to convert (upper, lower, or both)
 * @param mr Memory resource to use for allocation.
 * @param stream Stream to use for any kernels launched.
 * @return New strings column with characters converted.
 */
std::unique_ptr<cudf::column> convert_case( strings_column_view strings,
                                            detail::character_flags_table_type case_flag,
                                            rmm::mr::device_memory_resource* mr,
                                            hipStream_t stream)
{
    auto strings_count = strings.size();
    if( strings_count == 0 )
        return detail::make_empty_strings_column(mr,stream);

    auto execpol = rmm::exec_policy(0);
    auto strings_column = column_device_view::create(strings.parent(),stream);
    auto d_column = *strings_column;

    rmm::device_buffer null_mask;
    cudf::size_type null_count = d_column.null_count();
    if( d_column.nullable() ) // copy null_mask
        null_mask = rmm::device_buffer( d_column.null_mask(),
                                        bitmask_allocation_size_bytes(strings_count),
                                        stream, mr);

    // get the lookup tables used for case conversion
    auto d_flags = detail::get_character_flags_table();
    auto d_case_table = detail::get_character_case_table();

    // build offsets column
    // calculate the size of each output string
    auto offsets_transformer_itr = thrust::make_transform_iterator( thrust::make_counting_iterator<size_type>(0),
        upper_lower_fn<SizeOnly>{d_column, case_flag, d_flags, d_case_table} );
    auto offsets_column = detail::make_offsets_child_column(offsets_transformer_itr,
                                               offsets_transformer_itr+strings_count,
                                               mr, stream);
    auto offsets_view = offsets_column->view();
    auto d_new_offsets = offsets_view.data<int32_t>();

    // build the chars column -- convert uppercase characters to lowercase
    size_type bytes = thrust::device_pointer_cast(d_new_offsets)[strings_count];
    auto chars_column = strings::detail::create_chars_child_column( strings_count, null_count, bytes, mr, stream );
    auto chars_view = chars_column->mutable_view();
    auto d_chars = chars_view.data<char>();
    thrust::for_each_n(execpol->on(stream),
        thrust::make_counting_iterator<size_type>(0), strings_count,
        upper_lower_fn<ExecuteOp>{d_column, case_flag, d_flags, d_case_table, d_new_offsets, d_chars} );
    //
    return make_strings_column(strings_count, std::move(offsets_column), std::move(chars_column),
                               null_count, std::move(null_mask), stream, mr);
}

} // namespace

// APIS
//
std::unique_ptr<cudf::column> to_lower( strings_column_view strings,
                                        rmm::mr::device_memory_resource* mr,
                                        hipStream_t stream )
{
    detail::character_flags_table_type case_flag = IS_UPPER(0xFF); // convert only uppercase characters
    return convert_case(strings,case_flag,mr,stream);
}

//
std::unique_ptr<cudf::column> to_upper( strings_column_view strings,
                                        rmm::mr::device_memory_resource* mr,
                                        hipStream_t stream )
{
    detail::character_flags_table_type case_flag = IS_LOWER(0xFF); // convert only lowercase characters
    return convert_case(strings,case_flag,mr,stream);
}

//
std::unique_ptr<cudf::column> swapcase( strings_column_view strings,
                                        rmm::mr::device_memory_resource* mr,
                                        hipStream_t stream )
{
    // convert only upper or lower case characters
    detail::character_flags_table_type case_flag = IS_LOWER(0xFF) | IS_UPPER(0xFF);
    return convert_case(strings,case_flag,mr,stream);
}

} // namespace strings
} // namespace cudf


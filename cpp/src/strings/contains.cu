/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/null_mask.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/char_types/char_types.hpp>
#include <cudf/wrappers/bool.hpp>
#include "./utilities.hpp"
#include "regex/regex.cuh"


namespace cudf
{
namespace strings
{
namespace detail
{
namespace
{

// This functor handles both contains() and match() to minimize the number
// of regex calls to find() to be inlined.
template<size_t stack_size>
struct contains_fn
{
    dreprog* prog;
    column_device_view d_strings;
    bool bmatch{false}; // do not make this a template parameter to keep compile times down

    __device__ cudf::experimental::bool8 operator()(size_type idx)
    {
        u_char data1[stack_size], data2[stack_size];
        prog->set_stack_mem(data1,data2);
        if( d_strings.is_null(idx) )
            return 0;
        string_view d_str = d_strings.element<string_view>(idx);
        int32_t begin = 0;
        int32_t end = bmatch ? 1 : d_str.length(); // 1=match only the beginning of the string
        return static_cast<experimental::bool8>(prog->find(idx,d_str,begin,end));
    }
};

//
std::unique_ptr<column> contains_util( strings_column_view const& strings,
                                       std::string const& pattern,
                                       bool beginning_only = false,
                                       rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                       hipStream_t stream = 0)
{
    auto strings_count = strings.size();
    auto strings_column = column_device_view::create(strings.parent(),stream);
    auto d_column = *strings_column;

    auto d_flags = detail::get_character_flags_table();
    // compile regex into device object
    std::vector<char32_t> pattern32 = string_to_char32_vector(pattern);
    dreprog* prog = dreprog::create_from(pattern32.data(),d_flags);

    // allocate regex working memory if necessary
    int regex_insts = prog->inst_counts();
    if( regex_insts > MAX_STACK_INSTS )
    {
        if( !prog->alloc_relists(strings_count) )
        {
            std::ostringstream message;
            message << "cuDF failure at: " __FILE__ ":" << __LINE__ << ": ";
            message << "number of instructions (" << prog->inst_counts() << ") ";
            message << "and number of strings (" << strings_count << ") ";
            message << "exceeds available memory";
            dreprog::destroy(prog);
            // throw std::invalid_argument(message.str());
            //CUDF_FAIL(message.str());
            throw cudf::logic_error(message.str());
        }
    }
    // copy the null mask
    rmm::device_buffer null_mask = copy_bitmask(strings.parent(),stream,mr);
    // create output column
    auto results = std::make_unique<cudf::column>( cudf::data_type{cudf::BOOL8}, strings_count,
        rmm::device_buffer(strings_count * sizeof(cudf::experimental::bool8), stream, mr),
        null_mask, strings.null_count());
    auto results_view = results->mutable_view();
    auto d_results = results_view.data<cudf::experimental::bool8>();

    //
    auto execpol = rmm::exec_policy(stream);
    if( (regex_insts > MAX_STACK_INSTS) || (regex_insts <= RX_SMALL_INSTS) )
        thrust::transform(execpol->on(stream),
            thrust::make_counting_iterator<size_type>(0),
            thrust::make_counting_iterator<size_type>(strings_count),
            d_results, contains_fn<RX_STACK_SMALL>{prog, d_column, beginning_only} );
    else if( regex_insts <= RX_MEDIUM_INSTS )
        thrust::transform(execpol->on(stream),
            thrust::make_counting_iterator<size_type>(0),
            thrust::make_counting_iterator<size_type>(strings_count),
            d_results, contains_fn<RX_STACK_MEDIUM>{prog, d_column, beginning_only} );
    else
        thrust::transform(execpol->on(stream),
            thrust::make_counting_iterator<size_type>(0),
            thrust::make_counting_iterator<size_type>(strings_count),
            d_results, contains_fn<RX_STACK_LARGE>{prog, d_column, beginning_only} );

    results->set_null_count(strings.null_count());
    return results;
}

} // namespace

std::unique_ptr<column> contains_re( strings_column_view const& strings,
                                     std::string const& pattern,
                                     rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                     hipStream_t stream = 0)
{
    return contains_util(strings, pattern, false, mr, stream);
}

std::unique_ptr<column> matches_re( strings_column_view const& strings,
                                    std::string const& pattern,
                                    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                    hipStream_t stream = 0)
{
    return contains_util(strings, pattern, true, mr, stream);
}

} // namespace detail

// external APIs

std::unique_ptr<column> contains_re( strings_column_view const& strings,
                                     std::string const& pattern,
                                     rmm::mr::device_memory_resource* mr)
{
    return detail::contains_re(strings, pattern, mr);
}

std::unique_ptr<column> matches_re( strings_column_view const& strings,
                                     std::string const& pattern,
                                     rmm::mr::device_memory_resource* mr)
{
    return detail::matches_re(strings, pattern, mr);
}

} // namespace strings
} // namespace cudf

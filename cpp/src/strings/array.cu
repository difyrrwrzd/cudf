/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <bitmask/valid_if.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_handler.hpp>
#include <cudf/strings/string_view.cuh>
#include "./utilities.h"

#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>

namespace cudf 
{
namespace strings
{

// new strings column from subset of this strings instance
std::unique_ptr<cudf::column> sublist( strings_column_handler handler,
                                       size_type start, size_type end,
                                       size_type step, hipStream_t stream )
{
    if( step <= 0 )
        step = 1;
    size_type count = handler.size();
    if( end < 0 || end > count )
        end = count;
    if( start < 0 || start > end )
        throw std::invalid_argument("invalid start parameter");
    count = (end - start)/step +1;
    //
    auto execpol = rmm::exec_policy(stream);
    // build indices
    thrust::device_vector<size_type> indices(count);
    thrust::sequence( execpol->on(stream), indices.begin(), indices.end(), start, step );
    // create a column_view as a wrapper of these indices
    column_view indices_view( data_type{INT32}, count, indices.data().get(), nullptr, 0 );
    // build a new strings column from the indices
    return gather(handler, indices_view);
}

// return new strings column with strings from this instance as specified by the indices
std::unique_ptr<cudf::column> gather( strings_column_handler handler,
                                      column_view gather_map, hipStream_t stream )
{
    size_type count = gather_map.size();
    auto d_indices = gather_map.data<int32_t>();

    auto execpol = rmm::exec_policy(stream);
    auto strings_column = column_device_view::create(handler.parent_column(),stream);
    auto d_column = *strings_column;
    auto d_offsets = handler.offsets_column().data<int32_t>();

    // build offsets column
    auto offsets_column = make_numeric_column( data_type{INT32}, count, mask_state::UNALLOCATED,
                                               stream, handler.memory_resource() );
    auto offsets_view = offsets_column->mutable_view();
    auto d_new_offsets = offsets_view.data<int32_t>();
    // create new offsets array
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(count),
        d_new_offsets,
        [d_column, d_offsets, d_indices] __device__ (size_type idx) {
            size_type index = d_indices[idx];
            if( d_column.nullable() && d_column.is_null(index) )
                return 0;
            size_type offset = index ? d_offsets[index-1] : 0;
            return d_offsets[index] - offset;
        },
        thrust::plus<int32_t>());

    // build null mask
    auto valid_mask = valid_if( static_cast<const bit_mask_t*>(nullptr),
        [d_column, d_indices] __device__ (size_type idx) {
            return !d_column.nullable() || !d_column.is_null(d_indices[idx]);
        },
        count, stream );
    auto null_count = valid_mask.second;
    auto null_size = gdf_valid_allocation_size(count);
    rmm::device_buffer null_mask(valid_mask.first,null_size); // does deep copy
    RMM_TRY( RMM_FREE(valid_mask.first,stream) ); // TODO valid_if to return device_buffer in future

    // build chars column
    size_type bytes = thrust::device_pointer_cast(d_new_offsets)[count-1]; // this may not be stream friendly
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED,
                                             stream, handler.memory_resource() );
    auto chars_view = chars_column->mutable_view();
    auto d_chars = chars_view.data<int8_t>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), count,
        [d_column, d_indices, d_new_offsets, d_chars] __device__(size_type idx){
            // place individual strings
            if( d_column.nullable() && d_column.is_null(idx) )
                return;
            string_view d_str = d_column.element<string_view>(d_indices[idx]);
            size_type offset = (idx ? d_new_offsets[idx-1] : 0);
            memcpy(d_chars + offset, d_str.data(), d_str.size() );
        });

  // build children vector
  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(offsets_column));
  children.emplace_back(std::move(chars_column));

  return std::make_unique<column>(
        data_type{STRING}, 0, rmm::device_buffer{0,stream,handler.memory_resource()},
        null_mask, null_count,
        std::move(children));
}

// return sorted version of the given strings column
std::unique_ptr<cudf::column> sort( strings_column_handler handler,
                                    strings_column_handler::sort_type stype,
                                    bool ascending, bool nullfirst, hipStream_t stream )
{
    auto execpol = rmm::exec_policy(stream);
    auto strings_column = column_device_view::create(handler.parent_column(), stream);
    auto d_column = *strings_column;

    // lets sort indices
    size_type count = handler.size();
    thrust::device_vector<size_type> indices(count);
    thrust::sequence( execpol->on(stream), indices.begin(), indices.end() );
    thrust::sort( execpol->on(stream), indices.begin(), indices.end(),
        [d_column, stype, ascending, nullfirst] __device__ (size_type lhs, size_type rhs) {
            bool lhs_null{d_column.nullable() && d_column.is_null(lhs)};
            bool rhs_null{d_column.nullable() && d_column.is_null(rhs)};
            if( lhs_null || rhs_null )
                return (nullfirst ? !rhs_null : !lhs_null);
            string_view lhs_str = d_column.element<string_view>(lhs);
            string_view rhs_str = d_column.element<string_view>(rhs);
            int cmp = lhs_str.compare(rhs_str);
            return (ascending ? (cmp<0) : (cmp>0));
        });

    // create a column_view as a wrapper of these indices
    column_view indices_view( data_type{INT32}, count, indices.data().get(), nullptr, 0 );
    // now build a new strings column from the indices
    return gather( handler, indices_view );
}

//
// s1 = ['a','b,'c','d']
// s2 = ['e','f']
// pos = [1,3]  -- must be the same length as s2
// s3 = s1.scatter(s2,pos)
// ['a','e','c','f']
//
std::unique_ptr<cudf::column> scatter( strings_column_handler handler,
                                       strings_column_handler strings,
                                       cudf::column_view scatter_map,
                                       hipStream_t stream )
{
    size_type elements = strings.size();
    CUDF_EXPECTS( elements==scatter_map.size(), "number of strings must match map size" );
    size_type count = handler.size();
    auto d_indices = scatter_map.data<int32_t>();
    auto execpol = rmm::exec_policy(stream);

    //
    rmm::device_buffer buffer = create_string_array_from_column(handler,stream);
    cudf::string_view* d_strings = reinterpret_cast<cudf::string_view*>(buffer.data());
    rmm::device_buffer map_buffer = create_string_array_from_column(strings,stream);
    cudf::string_view* d_map_strings = reinterpret_cast<cudf::string_view*>(map_buffer.data());
    thrust::scatter( execpol->on(stream), d_map_strings, d_map_strings+elements, d_indices, d_strings );

    // build offsets column
    auto offsets_column = make_numeric_column( data_type{INT32}, count, mask_state::UNALLOCATED,
                                               stream, handler.memory_resource() );
    auto offsets_view = offsets_column->mutable_view();
    auto d_offsets = offsets_view.data<int32_t>();
    // create new offsets array
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(count),
        d_offsets,
        [d_strings, d_offsets] __device__ (size_type idx) {
            return d_strings[idx].size();
        },
        thrust::plus<int32_t>());

    // build null mask
    auto valid_mask = valid_if( static_cast<const bit_mask_t*>(nullptr),
        [d_strings] __device__ (size_type idx) { return !d_strings[idx].is_null(); },
        count, stream );
    auto null_count = valid_mask.second;
    auto null_size = gdf_valid_allocation_size(count);
    rmm::device_buffer null_mask(valid_mask.first,null_size); // does deep copy
    RMM_TRY( RMM_FREE(valid_mask.first,stream) ); // TODO valid_if to return device_buffer in future

    // build chars column
    size_type bytes = thrust::device_pointer_cast(d_offsets)[count-1]; // this may not be stream friendly
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED,
                                             stream, handler.memory_resource() );
    auto chars_view = chars_column->mutable_view();
    auto d_chars = chars_view.data<int8_t>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), count,
        [d_strings, d_offsets, d_chars] __device__(size_type idx){
            cudf::string_view d_str = d_strings[idx];
            if( d_str.is_null() )
                return;
            size_type offset = (idx ? d_offsets[idx-1] : 0);
            memcpy(d_chars + offset, d_str.data(), d_str.size() );
        });

  // build children vector
  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(offsets_column));
  children.emplace_back(std::move(chars_column));

  return std::make_unique<column>(
        data_type{STRING}, 0, rmm::device_buffer{0,stream,handler.memory_resource()},
        null_mask, null_count,
        std::move(children));
}

//
// s1 = ['a','b,'c','d']
// pos = [1,3]
// s3 = s1.scatter('e',pos,2)
// ['a','e','c','e']
//
std::unique_ptr<cudf::column> scatter( strings_column_handler handler,
                                       const char* string,
                                       cudf::column_view scatter_map,
                                       hipStream_t stream )
{
//    size_type count = size();
//    size_type elements = scatter_map.size();
//    auto execpol = rmm::exec_policy(0);
//    // copy string to device
//    auto d_string = string_from_host(string);
//    cudf::string_view* d_replace = *d_string;
//    // create result output array
//    rmm::device_vector<custring_view*> results(count,nullptr);
//    auto d_results = results.data().get();
//    custring_view_array d_strings = pImpl->getStringsPtr();
//    thrust::copy( execpol->on(0), d_strings, d_strings+count, d_results );
//    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), elements,
//        [d_pos, count, d_repl, d_results] __device__ (unsigned int idx) {
//            int pos = d_pos[idx];
//            if( (pos >= 0) && (pos < count) )
//                d_results[pos] = d_repl;
//        });
//    // build resulting instance
//    NVStrings* rtn = new NVStrings(count);
//    NVStrings_init_from_custrings(rtn->pImpl, d_results, count);
//    if( !bdevmem )
//        RMM_FREE((void*)d_pos,0);
//    RMM_FREE((void*)d_repl,0);
    return nullptr;
}


} // namespace strings
} // namespace cudf
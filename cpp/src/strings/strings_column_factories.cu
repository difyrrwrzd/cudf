#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <bitmask/valid_if.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column.hpp>
#include <cudf/functions.h>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_factories.hpp>
#include <utilities/error_utils.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <thrust/for_each.h>


namespace cudf {

// Create a strings-type column from array of pointer/size pairs
std::unique_ptr<column> make_strings_column(
    const rmm::device_vector<thrust::pair<const char*,size_t>>& strings,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
{
    size_type count = strings.size();
    // maybe a separate factory for creating null strings-column
    CUDF_EXPECTS(count > 0, "must specify at least one pair");

    auto execpol = rmm::exec_policy(stream);
    auto d_strings = strings.data().get();

    // check total size is not too large for cudf column
    size_t bytes = thrust::transform_reduce( execpol->on(stream),
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(count),
        [d_strings] __device__ (size_t idx) {
            auto item = d_strings[idx];
            return item.first ? item.second : 0;
        },
        0, thrust::plus<size_t>());
    CUDF_EXPECTS( bytes < std::numeric_limits<size_type>::max(), "total size of strings is too large for cudf column" );

    // build offsets column
    auto offsets_column = make_numeric_column( data_type{INT32}, count, mask_state::UNALLOCATED, stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<size_type>(0), thrust::make_counting_iterator<size_type>(count),
        offsets_view.data<int32_t>(),
        [d_strings] __device__ (size_type idx) {
            thrust::pair<const char*,size_t> item = d_strings[idx];
            return ( item.first ? static_cast<int32_t>(item.second) : 0 );
        },
        thrust::plus<int32_t>() );

    // create null mask
    auto valid_mask = valid_if( static_cast<const bit_mask_t*>(nullptr),
        [d_strings] __device__ (size_type idx) { return d_strings[idx].first!=nullptr; },
        count, stream );
    auto null_count = valid_mask.second;
    rmm::device_buffer null_mask(valid_mask.first, gdf_valid_allocation_size(count),
                                 stream, mr);
    RMM_TRY( RMM_FREE(valid_mask.first,stream) ); // TODO valid_if to return device_buffer in future
    if( (bytes==0) && (null_count < count) )
        bytes = 1; // all entries are empty strings

    // build chars column
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED, stream, mr );
    auto chars_view = chars_column->mutable_view();
    auto d_chars = chars_view.data<char>();
    auto d_offsets = offsets_view.data<int32_t>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), count,
          [d_strings, d_offsets, d_chars] __device__(size_type idx){
              // place individual strings
              auto item = d_strings[idx];
              if( item.first )
              {
                  size_type offset = (idx ? d_offsets[idx-1] : 0);
                  memcpy(d_chars + offset, item.first, item.second );
              }
          });

    // build children vector
    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));

    // see column_view.cpp(45) to see why size must be 0 here
    return std::make_unique<column>(
        data_type{STRING}, 0, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}

// Create a strings-type column from array of chars and array of offsets.
std::unique_ptr<column> make_strings_column(
    const rmm::device_vector<char>& strings,
    const rmm::device_vector<size_type>& offsets,
    const rmm::device_vector<bitmask_type>& valid_mask,
    size_type null_count,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr )
{
    size_type count = offsets.size()-1;
    CUDF_EXPECTS( count > 0, "strings count must be greater than 0");
    CUDF_EXPECTS( null_count < count, "null strings column not yet supported");

    auto execpol = rmm::exec_policy(stream);
    size_type bytes = offsets.back() - offsets[0];
    CUDF_EXPECTS( bytes >=0, "invalid offsets vector");

    // build offsets column
    auto offsets_column = make_numeric_column( data_type{INT32}, count, mask_state::UNALLOCATED, stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    hipMemcpyAsync( offsets_view.data<int32_t>(), offsets.data().get()+1,
                     count*sizeof(int32_t),
                     hipMemcpyDeviceToHost, stream );

    // build null bitmask
    rmm::device_buffer null_mask;
    if( null_count )
        null_mask = rmm::device_buffer(valid_mask.data().get(),
                                       gdf_valid_allocation_size(count),
                                       stream, mr);

    // build chars column
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED, stream, mr );
    auto chars_view = chars_column->mutable_view();
    hipMemcpyAsync( chars_view.data<char>(), strings.data().get(), bytes,
                     hipMemcpyDeviceToHost, stream );

    // build children vector
    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));

    //
    return std::make_unique<column>(
        data_type{STRING}, 0, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}


}  // namespace cudf

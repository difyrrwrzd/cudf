#include "cudf.h"
#include "rmm/rmm.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.h"
#include "utilities/type_dispatcher.hpp"

#include <cub/device/device_scan.cuh>



template <class T>
struct Scan {
    static
    gdf_error call(const T *inp, T *out, size_t size, bool inclusive) {
        using hipcub::DeviceScan;

        auto scan_function = (inclusive? inclusive_sum : exclusive_sum);

        // Prepare temp storage
        void *temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        scan_function(temp_storage, temp_storage_bytes, inp, out, size);
        RMM_TRY( RMM_ALLOC(&temp_storage, temp_storage_bytes, 0) ); // TODO: non-default stream
        // Do scan
        scan_function(temp_storage, temp_storage_bytes, inp, out, size);
        // Cleanup
        RMM_TRY( RMM_FREE(temp_storage, 0) ); // TODO: non-default stream

        return GDF_SUCCESS;
    }

    static
    gdf_error exclusive_sum(void *&temp_storage, size_t &temp_storage_bytes,
                            const T *inp, T *out, size_t size) {
        hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_bytes,
                                      inp, out, size);
        CUDA_CHECK_LAST();
        return GDF_SUCCESS;
    }

    static
    gdf_error inclusive_sum(void *&temp_storage, size_t &temp_storage_bytes,
                            const T *inp, T *out, size_t size) {
        hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_bytes,
                                      inp, out, size);
        CUDA_CHECK_LAST();
        return GDF_SUCCESS;
    }
};

struct PrefixSumDispatcher {
    template <typename T,
        typename std::enable_if_t<std::is_arithmetic<T>::value>* = nullptr>
        gdf_error operator()(gdf_column *inp, gdf_column *out,
            int inclusive) {
        GDF_REQUIRE(inp->size == out->size, GDF_COLUMN_SIZE_MISMATCH);
        GDF_REQUIRE(inp->dtype == out->dtype, GDF_UNSUPPORTED_DTYPE);
        GDF_REQUIRE(!inp->valid || !inp->null_count, GDF_VALIDITY_UNSUPPORTED);
        GDF_REQUIRE(!out->valid || !out->null_count, GDF_VALIDITY_UNSUPPORTED);
        return Scan<T>::call((const T*)inp->data, (T*)out->data, inp->size,
            inclusive);
    }

    template <typename T,
        typename std::enable_if_t<!std::is_arithmetic<T>::value, T>* = nullptr>
        gdf_error operator()(gdf_column *inp, gdf_column *out,
            int inclusive) {
        return GDF_UNSUPPORTED_DTYPE;
    }
};

gdf_error gdf_prefixsum(gdf_column *inp, gdf_column *out,
    int inclusive)
{
    return cudf::type_dispatcher(inp->dtype, PrefixSumDispatcher(),
        inp, out, inclusive);
}

#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/iterator.cuh>

#include <rmm/rmm.h>
#include <cudf/utilities/error.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <utilities/device_atomics.cuh>
#include <cub/device/device_scan.cuh>
#include <cudf/reduction.hpp>


namespace cudf {
namespace experimental {

namespace detail {

template <typename T, typename Op>
struct Scan {
    static
    void call(const column_view input, mutable_column_view output,
                  bool inclusive, hipStream_t stream)
    {
        size_t size = input.size();

        //TODO use memory device_resource to allocate these memory
        // Prepare temp storage
        void *temp_storage = NULL;
        size_t temp_storage_bytes = 0;

        if (input.has_nulls()) {
          auto d_input = column_device_view::create(input, stream);
          auto it = make_null_replacement_iterator(*d_input, Op::template identity<T>());
          auto scan_function = (inclusive ? inclusive_scan<decltype(it)> : exclusive_scan<decltype(it)>);
          scan_function(temp_storage, temp_storage_bytes,
              it, output.data<T>(), size, stream);
          RMM_TRY(RMM_ALLOC(&temp_storage, temp_storage_bytes, stream));

          scan_function(temp_storage, temp_storage_bytes,
              it, output.data<T>(), size, stream);
        } else {
          auto it = input.data<T>();  //since scan is for arithmetic types only
          auto scan_function = (inclusive ? inclusive_scan<decltype(it)> : exclusive_scan<decltype(it)>);
          scan_function(temp_storage, temp_storage_bytes,
              it, output.data<T>(), size, stream);
          RMM_TRY(RMM_ALLOC(&temp_storage, temp_storage_bytes, stream));
          scan_function(temp_storage, temp_storage_bytes,
              it, output.data<T>(), size, stream);
        }

        // Cleanup
        RMM_TRY(RMM_FREE(temp_storage, stream));
    }

    template <typename InputIterator>
    static void exclusive_scan(void *&temp_storage, size_t &temp_storage_bytes,
        const InputIterator input, T* output, size_t size, hipStream_t stream)
    {
        //TODO benchmark between thrust and cub reduce and scan
        hipcub::DeviceScan::ExclusiveScan(temp_storage, temp_storage_bytes,
            input, output, Op{}, Op::template identity<T>(), size, stream);
        CUDA_CHECK_LAST();
    }

    template <typename InputIterator>
    static void inclusive_scan(void *&temp_storage, size_t &temp_storage_bytes,
        const InputIterator input, T* output, size_t size, hipStream_t stream)
    {
      hipcub::DeviceScan::InclusiveScan(temp_storage, temp_storage_bytes, input,
                                     output, Op{}, size, stream);
      CUDA_CHECK_LAST();
    }
};

template <typename Op>
struct PrefixSumDispatcher {
  private:
  // return true if T is arithmetic type (including cudf::experimental::bool8)
  template <typename T>
  static constexpr bool is_supported() {
    return std::is_arithmetic<T>::value;
  }

  public:
  template <typename T,
            typename std::enable_if_t<is_supported<T>(), T> * = nullptr>
  void operator()(const column_view& input, mutable_column_view& output,
                  bool inclusive, hipStream_t stream = 0)
  {
    CUDF_EXPECTS(input.size() == output.size(),
                 "input and output data size must be same");
    CUDF_EXPECTS(input.type() == output.type(),
                 "input and output data types must be same");

    CUDF_EXPECTS(input.nullable() == output.nullable(),
                 "Input column and Output column nullable mismatch (either one "
                 "cannot be nullable)");

    Scan<T, Op>::call(input, output, inclusive, stream);
    CUDF_EXPECTS(input.null_count() == output.null_count(),
                 "Input / output column null count mismatch");
  }

  template <typename T,
            typename std::enable_if_t<!is_supported<T>(), T> * = nullptr>
  void operator()(const column_view& input, mutable_column_view& output,
                  bool inclusive, hipStream_t stream = 0) {
    CUDF_FAIL("Non-arithmetic types not supported for `gdf_scan`");
  }
};

} // namespace detail

std::unique_ptr<column> scan(const column_view& input,
                             scan_op op, bool inclusive,
                             hipStream_t stream,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_numeric(input.type()), "Unexpected non-numeric type.");

  std::unique_ptr<column> output_column = make_numeric_column(
      input.type(), input.size(), 
      copy_bitmask(input, stream, mr), //copy bit mask
      input.null_count(), stream, mr);

  mutable_column_view output = output_column->mutable_view();

  switch (op) {
    case SCAN_SUM:
        cudf::experimental::type_dispatcher(input.type(),
              detail::PrefixSumDispatcher<cudf::DeviceSum>(), input, output, inclusive);
        break;
    case SCAN_MIN:
        cudf::experimental::type_dispatcher(input.type(),
            detail::PrefixSumDispatcher<cudf::DeviceMin>(), input, output, inclusive);
        break;
    case SCAN_MAX:
        cudf::experimental::type_dispatcher(input.type(),
            detail::PrefixSumDispatcher<cudf::DeviceMax>(), input, output, inclusive);
        break;
    case SCAN_PRODUCT:
        cudf::experimental::type_dispatcher(input.type(),
            detail::PrefixSumDispatcher<cudf::DeviceProduct>(), input, output, inclusive);
        break;
    default:
        CUDF_FAIL("The input enum `scan_op` is out of the range");
    }
  return output_column;
}

}  // namespace experimental
}  // namespace cudf

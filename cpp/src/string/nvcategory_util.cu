
#include "nvcategory_util.cuh"


#include <nvstrings/NVCategory.h>
#include <nvstrings/NVStrings.h>
#include "rmm/rmm.h"

#include "utilities/error_utils.hpp"
#include "utilities/nvtx/nvtx_utils.h"


gdf_error nvcategory_gather(gdf_column * column, NVCategory * nv_category){


	GDF_REQUIRE(nv_category != nullptr,GDF_INVALID_API_CALL );
	GDF_REQUIRE(column->dtype == GDF_STRING_CATEGORY,GDF_UNSUPPORTED_DTYPE);

	NVStrings * strings = nv_category->gather_strings(static_cast<nv_category_index_type *>(column->data),
			column->size,
			DEVICE_ALLOCATED);

	NVCategory * new_category = NVCategory::create_from_strings(*strings);
	new_category->get_values(static_cast<nv_category_index_type *>(column->data),
			DEVICE_ALLOCATED);

	//This is questionable behavior and should be reviewed by peers
	//Smart pointers would be lovely here
	if(column->dtype_info.category != nullptr){
		NVCategory::destroy(column->dtype_info.category);
	}
	column->dtype_info.category = new_category;

	NVStrings::destroy(strings);
	return GDF_SUCCESS;
}

gdf_error validate_categories(gdf_column * input_columns[], int num_columns, gdf_size_type & total_count){
	total_count = 0;
	for (int i = 0; i < num_columns; ++i) {
		gdf_column* current_column = input_columns[i];
		GDF_REQUIRE(current_column != nullptr,GDF_DATASET_EMPTY);
		GDF_REQUIRE(current_column->data != nullptr,GDF_DATASET_EMPTY);
		GDF_REQUIRE(current_column->dtype == GDF_STRING_CATEGORY,GDF_UNSUPPORTED_DTYPE);

		total_count += input_columns[i]->size;
	}
	return GDF_SUCCESS;
}

#include <iostream>
gdf_error concat_categories(gdf_column * input_columns[],gdf_column * output_column, int num_columns){

	gdf_size_type total_count;
	gdf_error err = validate_categories(input_columns,num_columns,total_count);
	GDF_REQUIRE(err == GDF_SUCCESS,err);
	GDF_REQUIRE(total_count <= output_column->size,GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output_column->dtype == GDF_STRING_CATEGORY,GDF_UNSUPPORTED_DTYPE);
	//TODO: we have no way to jsut copy a category this will fail if someone calls concat
	//on a single input
	GDF_REQUIRE(num_columns >= 2,GDF_DATASET_EMPTY);
	NVCategory * new_category = input_columns[0]->dtype_info.category;
	NVCategory * temp_category;

	for (int i = 1; i < num_columns; i++) {
		NVStrings * temp_strings = input_columns[i]->dtype_info.category->to_strings();
		temp_category = new_category->add_strings(*temp_strings); //this is the only way to add to a category and keep the dictionary sorted
		if(i > 1){
			//only destroy categoryy after first iteration
			NVCategory::destroy(new_category);

		}

		NVStrings::destroy(temp_strings);

		new_category = temp_category;
	}

	new_category->get_values(
			static_cast<nv_category_index_type *>(output_column->data),
			true);
	output_column->dtype_info.category = new_category;



	return GDF_SUCCESS;
}

gdf_error sync_column_categories(gdf_column * input_columns[],gdf_column * output_columns[], int num_columns){

	GDF_REQUIRE(num_columns > 0,GDF_DATASET_EMPTY);
	gdf_size_type total_count;

	gdf_error err = validate_categories(input_columns,num_columns,total_count);
	GDF_REQUIRE(GDF_SUCCESS == err, err);

	err = validate_categories(output_columns,num_columns,total_count);
	GDF_REQUIRE(GDF_SUCCESS == err, err);

	for(int column_index = 0; column_index < num_columns; column_index++){
		GDF_REQUIRE(input_columns[column_index]->size == output_columns[column_index]->size,GDF_COLUMN_SIZE_MISMATCH);
	}

	NVStrings** temp_strs = new NVStrings*[num_columns];

	for(int column_index = 0; column_index < num_columns; column_index++){
		temp_strs[column_index] = input_columns[column_index]->dtype_info.category->to_strings();
	}

	NVCategory** new_categories = new NVCategory*[num_columns];

	for(int column_index_x = 0; column_index_x < num_columns; column_index_x++)
		for(int column_index_y = 0; column_index_y < num_columns; column_index_y++){
			if(column_index_x != column_index_y){
				new_categories[column_index_x] = output_columns[column_index_x]->dtype_info.category->add_strings(*temp_strs[column_index_y]);
			}
		}

	std::vector<hipError_t> cuda_err(num_columns);
	for(int column_index = 0; column_index < num_columns; column_index++){
		if(output_columns[column_index]->dtype_info.category != nullptr){
			NVCategory::destroy(output_columns[column_index]->dtype_info.category);
		}

		output_columns[column_index]->dtype_info.category = new_categories[column_index];

		size_t size_to_copy = sizeof(nv_category_index_type) * output_columns[column_index]->size;
		cuda_err[column_index] = hipMemcpy(output_columns[column_index]->data,
			output_columns[column_index]->dtype_info.category->values_cptr(),
			size_to_copy,
			hipMemcpyDeviceToDevice);
	}

	for(int column_index = 0; column_index < num_columns; column_index++){
		GDF_REQUIRE(cuda_err[column_index] == hipSuccess, GDF_CUDA_ERROR);
	}

	return GDF_SUCCESS;
}

gdf_error free_nvcategory(gdf_column * column){
	NVCategory::destroy(column->dtype_info.category);
	column->dtype_info.category = nullptr;
	return GDF_SUCCESS;
}

/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/search.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/dictionary/detail/concatenate.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <thrust/binary_search.h>
#include <thrust/transform_scan.h>
#include <algorithm>
#include <rmm/device_uvector.hpp>
#include <vector>

namespace cudf {
namespace dictionary {
namespace detail {
namespace {

// keys and indices offsets for concatenated child columns
using offsets_pair = thrust::pair<size_type, size_type>;

struct compute_child_offsets {
  compute_child_offsets(std::vector<column_view> const& columns) : columns_ptrs{columns.size()}
  {
    std::transform(
      columns.begin(), columns.end(), columns_ptrs.begin(), [](auto& cv) { return &cv; });
  }

  data_type get_keys_type()
  {
    dictionary_column_view dict_view(**std::find_if(
      columns_ptrs.begin(), columns_ptrs.end(), [](auto pcv) { return pcv->size() > 0; }));
    return dict_view.keys().type();
  }

  rmm::device_uvector<offsets_pair> create_children_offsets(hipStream_t stream)
  {
    std::vector<offsets_pair> offsets(columns_ptrs.size());
    thrust::transform_exclusive_scan(
      thrust::host,
      columns_ptrs.begin(),
      columns_ptrs.end(),
      offsets.begin(),
      [](auto pcv) {
        dictionary_column_view view(*pcv);
        return offsets_pair{view.keys_size(), view.size()};
      },
      offsets_pair{0, 0},
      [](auto lhs, auto rhs) {
        return offsets_pair{lhs.first + rhs.first, lhs.second + rhs.second};
      });
    auto d_offsets = rmm::device_uvector<offsets_pair>(offsets.size(), stream);
    CUDA_TRY(hipMemcpyAsync(d_offsets.data(),
                             offsets.data(),
                             offsets.size() * sizeof(offsets_pair),
                             hipMemcpyHostToDevice,
                             stream));
    return d_offsets;
  }

 private:
  // thrust::transform_exclusive_scan does not compile with column_view
  std::vector<column_view const*> columns_ptrs;
};

/**
 * @brief Type-dispatch functor for remapping the old indices to new values based
 * on the new key-set.
 *
 * The dispatch is based on the key type.
 * The output column is the updated indices child for the new dictionary column.
 */
struct dispatch_compute_indices {
  template <typename Element>
  typename std::enable_if_t<cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(column_view const& all_keys,
             column_view const& all_indices,
             column_view const& new_keys,
             offsets_pair const* d_offsets,
             size_type const* d_map_to_keys,
             hipStream_t stream,
             rmm::mr::device_memory_resource* mr)
  {
    auto keys_view     = column_device_view::create(all_keys, stream);
    auto d_all_keys    = *keys_view;
    auto indices_view  = column_device_view::create(all_indices, stream);
    auto d_all_indices = *indices_view;
    auto all_itr       = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_type>(0),
      [d_all_keys, d_offsets, d_map_to_keys, d_all_indices] __device__(size_type idx) {
        if (d_all_indices.is_null(idx)) return Element{};
        size_type index =
          d_all_indices.template element<int32_t>(idx) + d_offsets[d_map_to_keys[idx]].first;
        return d_all_keys.template element<Element>(index);
      });
    auto new_keys_view = column_device_view::create(new_keys, stream);
    auto d_new_keys    = *new_keys_view;
    auto keys_itr      = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_type>(0),
      [d_new_keys] __device__(size_type idx) { return d_new_keys.template element<Element>(idx); });

    auto result = make_numeric_column(
      data_type{type_id::INT32}, all_indices.size(), mask_state::UNALLOCATED, stream, mr);
    auto d_result = result->mutable_view().data<int32_t>();
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                        keys_itr,
                        keys_itr + new_keys.size(),
                        all_itr,
                        all_itr + all_indices.size(),
                        d_result,
                        thrust::less<Element>());
    result->set_null_count(0);
    return result;
  }

  template <typename Element>
  typename std::enable_if_t<!cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(column_view const&,
             column_view const&,
             column_view const&,
             offsets_pair const*,
             size_type const*,
             hipStream_t stream,
             rmm::mr::device_memory_resource*)
  {
    CUDF_FAIL("list_view as keys for dictionary not supported");
  }
};

}  // namespace

std::unique_ptr<column> concatenate(std::vector<column_view> const& columns,
                                    hipStream_t stream,
                                    rmm::mr::device_memory_resource* mr)
{
  if (columns.size() == 0) return make_empty_column(data_type{type_id::DICTIONARY32});

  // concatenate the keys (and check the keys match)
  compute_child_offsets child_offsets_fn{columns};
  auto keys_type = child_offsets_fn.get_keys_type();
  std::vector<column_view> keys_views(columns.size());
  std::transform(columns.begin(), columns.end(), keys_views.begin(), [keys_type](auto cv) {
    auto dict_view = dictionary_column_view(cv);
    if (dict_view.size() == 0) return column_view{keys_type, 0, nullptr};
    auto keys = dict_view.keys();
    CUDF_EXPECTS(keys.type() == keys_type, "key types of each dictionary column must match");
    return keys;
  });
  auto all_keys = cudf::detail::concatenate(keys_views, rmm::mr::get_default_resource(), stream);

  // sort keys and remove duplicates
  auto table_keys = cudf::detail::drop_duplicates(table_view{{all_keys->view()}},
                                                  std::vector<size_type>{0},
                                                  duplicate_keep_option::KEEP_FIRST,
                                                  null_equality::EQUAL,
                                                  mr,
                                                  stream)
                      ->release();
  std::unique_ptr<column> keys_column(std::move(table_keys.front()));

  // concatenate the indices
  std::vector<column_view> indices_views(columns.size());
  std::transform(columns.begin(), columns.end(), indices_views.begin(), [](auto cv) {
    auto dict_view = dictionary_column_view(cv);
    if (dict_view.size() == 0) return column_view{data_type{type_id::INT32}, 0, nullptr};
    return dict_view.get_indices_annotated();  // includes validity mask and offset
  });
  auto all_indices       = cudf::detail::concatenate(indices_views, mr, stream);
  size_type indices_size = all_indices->size();

  // create vector of values to match old indices to the concatenated keys
  auto children_offsets = child_offsets_fn.create_children_offsets(stream);

  rmm::device_uvector<size_type> map_to_keys(indices_size, stream);
  auto pair_itr = thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(1),
                                                  [] __device__(size_type idx) {
                                                    return offsets_pair{0, idx};
                                                  });
  thrust::lower_bound(
    rmm::exec_policy(stream)->on(stream),
    children_offsets.begin() + 1,
    children_offsets.end(),
    pair_itr,
    pair_itr + indices_size + 1,
    map_to_keys.begin(),
    [] __device__(auto const& lhs, auto const& rhs) { return lhs.second < rhs.second; });

  // now recompute the indices values for the new keys_column
  auto indices_column = type_dispatcher(keys_type,
                                        dispatch_compute_indices{},
                                        all_keys->view(),     // old keys
                                        all_indices->view(),  // old indices
                                        keys_column->view(),  // new keys
                                        children_offsets.data(),
                                        map_to_keys.data(),
                                        stream,
                                        mr);

  // remove the bitmask from the indices_column
  auto null_count = all_indices->null_count();  // get before release()
  auto contents   = all_indices->release();     // all_indices will now be empty

  // finally, frankenstein the dictionary column
  return make_dictionary_column(std::move(keys_column),
                                std::move(indices_column),
                                std::move(*(contents.null_mask.release())),
                                null_count);
}

}  // namespace detail
}  // namespace dictionary
}  // namespace cudf

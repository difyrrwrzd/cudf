#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "transpose.hpp"
#include <cudf/table/table_device_view.cuh>
#include <utilities/nvtx/nvtx_utils.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/copying.hpp> // TODO use internal <copying/copy.hpp>

//#include <rmm/thrust_rmm_allocator.h>
//#include <cudf/cudf.h>
//#include <hipcub/hipcub.hpp>
//#include <memory>
//#include <algorithm>

namespace cudf {

namespace {

constexpr int WARP_SIZE = 32;
constexpr int MAX_GRID_SIZE = (1<<16)-1;

/**
 * @brief Transposes the values from ncols x nrows input columns to
 *  nrows x ncols output columns
 * 
 * @tparam T  Datatype of values pointed to by the views
 * @param input[in]  Device view of input columns' data
 * @param output[out]  Mutable device view of pre-allocated output columns' data
 */
template <typename T>
__global__
void gpu_transpose(table_device_view const input, mutable_table_device_view output)
{
  size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  size_type y = blockIdx.y * blockDim.y + threadIdx.y;

  size_type stride_x = blockDim.x * gridDim.x;
  size_type stride_y = blockDim.y * gridDim.y;

  for(size_type i = x; i < input.num_columns(); i += stride_x)
  {
    for(size_type j = y; j < input.num_rows(); j += stride_y)
    {
      output.column(j).element<T>(i) = input.column(i).element<T>(j);
    }
  }
}

/**
 * @brief Transposes the validity mask
 * 
 * @param[in] in_cols_valid  pointers to the validity mask of the input columns
 * @param[out] out_cols_valid  pointers to the pre-allocated validity mask of
 *  the output columns
 * @param[out] out_cols_null_count  array of per output-row null counts
 * @param[in] ncols  number of columns in input table
 * @param[in] nrows  number of rows in input table
 */
__global__
void gpu_transpose_valids(table_device_view const input, mutable_table_device_view output)
// TODO profile these two implementations
#if 1
{
  size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  size_type y = blockIdx.y * blockDim.y + threadIdx.y;

  size_type stride_x = blockDim.x * gridDim.x;
  size_type stride_y = blockDim.y * gridDim.y;

  for(size_type i = x; i < input.num_columns(); i += stride_x)
  {
    for(size_type j = y; j < input.num_rows(); j += stride_y)
    {
      if (input.column(i).is_valid(j)) {
        output.column(j).set_valid(i);
      } else {
        output.column(j).set_null(i);
      }
    }
  }
}
#else
{
  constexpr uint32_t BITS_PER_MASK{sizeof(bitmask_type) * 8};

  size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  size_type y = blockIdx.y * blockDim.y + threadIdx.y;

  size_type stride_x = blockDim.x * gridDim.x;
  size_type stride_y = blockDim.y * gridDim.y;

  size_type i = x;
  size_type j = y;
  auto active_threads = __ballot_sync(0xffffffff, i < input.num_columns());
  while(i < input.num_columns())
  {
    j = y;
    while(j < input.num_rows())
    {
      bool const input_is_valid{input.column(i).is_valid(j)};
      bitmask_type const result_mask{__ballot_sync(active_threads, input_is_valid)};

      bitmask_type* const __restrict__ out_mask32 = output.column(j).null_mask();

      cudf::size_type const out_location = i / BITS_PER_MASK;

      // Only one thread writes output
      if (0 == threadIdx.x % warpSize) {
        out_mask32[out_location] = result_mask;
        // TODO we can write the null count to an additional device buffer
        //int num_nulls = __popc(active_threads) - __popc(result_mask);
        //atomicAdd(out_cols_null_count + j, num_nulls);
      }
      
      j += stride_y;
    }
    i += stride_x;
    active_threads = __ballot_sync(active_threads, i < input.num_columns());
  }
}
#endif

// TODO: refactor and separate `valids` kernel launch into another function.
// Should not need to pass `has_null`
struct launch_kernel{
  template <typename T>
  void operator()(
    table_view const& input,
    mutable_table_view& output,
    bool has_null,
    hipStream_t stream)
  {
    // Copy input columns `data` and `valid` pointers to device
    auto device_input = table_device_view::create(input, stream);
    auto device_output = mutable_table_device_view::create(output, stream);

    dim3 dimBlock(WARP_SIZE, WARP_SIZE);
    dim3 dimGrid(std::min((input.num_columns() + WARP_SIZE - 1) / WARP_SIZE, MAX_GRID_SIZE),
                 std::min((input.num_rows() + WARP_SIZE - 1) / WARP_SIZE, MAX_GRID_SIZE));

    gpu_transpose<T><<<dimGrid, dimBlock, 0, stream>>>(*device_input, *device_output);

    if (has_null){
      gpu_transpose_valids<<<dimGrid, dimBlock, 0, stream>>>(*device_input, *device_output);

      // Force null counts to be recomputed next time they are queried
      for (auto& column : output) {
        column.set_null_count(UNKNOWN_NULL_COUNT);
      }
    }

    // Synchronize before return so we don't cut short the lifetime of our device_views
    CUDA_TRY(hipStreamSynchronize(stream));
  }
};

}  // namespace

namespace detail {

std::unique_ptr<experimental::table> transpose(table_view const& input,
  rmm::mr::device_memory_resource* mr, hipStream_t stream)
{
  auto const input_ncols = input.num_columns();
  auto const input_nrows = input.num_rows();

  // If there are no rows in the input, return successfully
  if (input_ncols == 0 || input_nrows == 0) {
    // NOTE this returns a table with the same shape as the input, not transposed
    // TODO this could take stream as a parameter
    return experimental::empty_like(input/*, stream*/);
  }

  // Check datatype homogeneity
  auto const dtype = input.column(0).type();
  for (auto const& col : input) {
    CUDF_EXPECTS(dtype == col.type(), "Column type mismatch");
  }

  // TODO does this need to support non-fixed-width tables?
  CUDF_EXPECTS(is_fixed_width(dtype), "Invalid, non-fixed-width type.");

  nvtx::range_push("CUDF_TRANSPOSE", nvtx::color::GREEN);

  // Check if there are nulls to be processed
  bool const has_null = has_nulls(input);
  auto const allocation_policy = has_null ? experimental::mask_allocation_policy::ALWAYS
    : experimental::mask_allocation_policy::NEVER;

  auto const& output_ncols = input_nrows;
  auto const& output_nrows = input_ncols;

  // Allocate output table with transposed shape
  std::vector<std::unique_ptr<column>> out_columns;
  out_columns.reserve(output_ncols);
  for (size_type i = 0; i < output_ncols; ++i) {
    // TODO this could take stream as a parameter
    out_columns.push_back(experimental::allocate_like(input.column(0), output_nrows,
      allocation_policy, mr/*, stream*/));
  }
  auto output = std::make_unique<experimental::table>(std::move(out_columns));
  auto output_view = output->mutable_view();

  experimental::type_dispatcher(dtype, launch_kernel{}, input, output_view, has_null, stream);

  nvtx::range_pop();
  return output;
}

}  // namespace detail

std::unique_ptr<experimental::table> transpose(table_view const& input,
                                 rmm::mr::device_memory_resource* mr)
{
  return detail::transpose(input, mr);
}

}  // namespace cudf

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/transpose.hpp>
#include <cudf/detail/transpose.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/detail/copy.hpp>

namespace cudf {

namespace {

constexpr int WARP_SIZE = 32;
constexpr int MAX_GRID_SIZE = (1<<16)-1;

/**
 * @brief Transposes the values from ncols x nrows input to nrows x ncols output 
 * 
 * @tparam T  Datatype of values pointed to by the views
 * @param input[in]  Device view of input columns' data
 * @param output[out]  Mutable device view of pre-allocated output columns' data
 */
template <typename T>
__global__
void gpu_transpose(table_device_view const input, mutable_table_device_view output)
{
  size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  size_type y = blockIdx.y * blockDim.y + threadIdx.y;

  size_type stride_x = blockDim.x * gridDim.x;
  size_type stride_y = blockDim.y * gridDim.y;

  for (size_type i = x; i < input.num_columns(); i += stride_x) {
    for (size_type j = y; j < input.num_rows(); j += stride_y) {
      output.column(j).element<T>(i) = input.column(i).element<T>(j);
    }
  }
}

/**
 * @brief Transposes the null mask from ncols x nrows input to nrows x ncols output
 * 
 * @tparam T  Datatype of values pointed to by the views
 * @param input[in]  Device view of input columns' data
 * @param output[out]  Mutable device view of pre-allocated output columns' data
 */
__global__
void gpu_transpose_valids(table_device_view const input, mutable_table_device_view output)
{
  constexpr cudf::size_type BITS_PER_MASK{sizeof(bitmask_type) * 8};

  size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  size_type y = blockIdx.y * blockDim.y + threadIdx.y;

  size_type stride_x = blockDim.x * gridDim.x;
  size_type stride_y = blockDim.y * gridDim.y;

  auto active_threads = 0xffffffff;
  for (size_type i = x; i < input.num_columns(); i += stride_x) {
    active_threads = __ballot_sync(active_threads, i < input.num_columns());

    for (size_type j = y; j < input.num_rows(); j += stride_y) {
      auto const result = __ballot_sync(active_threads, input.column(i).is_valid(j));

      // Only one thread writes output
      if (0 == threadIdx.x % WARP_SIZE) {
        output.column(j).set_mask_word(i / BITS_PER_MASK, result);
      }
    }
  }
}

struct launch_kernel {
  template <typename T>
  void operator()(
    table_view const& input,
    mutable_table_view& output,
    bool has_null,
    hipStream_t stream)
  {
    // Copy input columns `data` and `valid` pointers to device
    auto device_input = table_device_view::create(input, stream);
    auto device_output = mutable_table_device_view::create(output, stream);

    dim3 dimBlock(WARP_SIZE, WARP_SIZE);
    dim3 dimGrid(std::min((input.num_columns() + WARP_SIZE - 1) / WARP_SIZE, MAX_GRID_SIZE),
                 std::min((input.num_rows() + WARP_SIZE - 1) / WARP_SIZE, MAX_GRID_SIZE));

    gpu_transpose<T><<<dimGrid, dimBlock, 0, stream>>>(*device_input, *device_output);

    if (has_null) {
      gpu_transpose_valids<<<dimGrid, dimBlock, 0, stream>>>(*device_input, *device_output);

      // Force null counts to be recomputed next time they are queried
      for (auto& column : output) {
        column.set_null_count(UNKNOWN_NULL_COUNT);
      }
    }

    // Synchronize before return so we don't cut short the lifetime of our device_views
    CUDA_TRY(hipStreamSynchronize(stream));
  }
};

}  // namespace

namespace detail {

std::unique_ptr<experimental::table> transpose(table_view const& input,
  rmm::mr::device_memory_resource* mr, hipStream_t stream)
{
  auto const input_ncols = input.num_columns();
  auto const input_nrows = input.num_rows();

  // If there are no rows in the input, return successfully
  if (input_ncols == 0 || input_nrows == 0) {
    return std::make_unique<experimental::table>(std::vector<std::unique_ptr<column>>{});
  }

  // Check datatype homogeneity
  auto const dtype = input.column(0).type();
  for (auto const& col : input) {
    CUDF_EXPECTS(dtype == col.type(), "Column type mismatch");
  }

  // TODO does this need to support non-fixed-width tables?
  CUDF_EXPECTS(is_fixed_width(dtype), "Invalid, non-fixed-width type.");

  nvtx::range_push("CUDF_TRANSPOSE", nvtx::color::GREEN);

  // Check if there are nulls to be processed
  bool const has_null = has_nulls(input);
  auto const allocation_policy = has_null ? experimental::mask_allocation_policy::ALWAYS
    : experimental::mask_allocation_policy::NEVER;

  auto const& output_ncols = input_nrows;
  auto const& output_nrows = input_ncols;

  // Allocate output table with transposed shape
  std::vector<std::unique_ptr<column>> out_columns;
  out_columns.reserve(output_ncols);
  for (size_type i = 0; i < output_ncols; ++i) {
    out_columns.push_back(experimental::detail::allocate_like(input.column(0), output_nrows,
      allocation_policy, mr, stream));
  }
  auto output = std::make_unique<experimental::table>(std::move(out_columns));
  auto output_view = output->mutable_view();

  experimental::type_dispatcher(dtype, launch_kernel{}, input, output_view, has_null, stream);

  nvtx::range_pop();
  return output;
}

}  // namespace detail

std::unique_ptr<experimental::table> transpose(table_view const& input,
                                 rmm::mr::device_memory_resource* mr)
{
  return detail::transpose(input, mr);
}

}  // namespace cudf

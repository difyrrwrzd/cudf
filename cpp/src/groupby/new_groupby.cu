#include <cassert>
#include <thrust/fill.h>
#include <algorithm>

#include "cudf.h"
#include "types.hpp"
#include "copying.hpp"
#include "new_groupby.hpp"
#include "utilities/nvtx/nvtx_utils.h"
#include "utilities/error_utils.hpp"
#include "aggregation_operations.hpp"
#include "groupby/hash_groupby.cuh"
#include "string/nvcategory_util.hpp"
#include "sqls/sqls_rtti_comp.h"

namespace{
  /* --------------------------------------------------------------------------*/
  /**
   * @brief Verifies that a set gdf_columns contain non-null data buffers, and are all
   * of the same size.
   *
   *
   * TODO: remove when null support added.
   *
   * Also ensures that the columns do not contain any null values
   *
   * @param[in] first Pointer to first gdf_column in set
   * @param[in] last Pointer to one past the last column in set
   *
   * @returns GDF_DATASET_EMPTY if a column contains a null data buffer,
   * GDF_COLUMN_SIZE_MISMATCH if the columns are not of equal length,
   */
  /* ----------------------------------------------------------------------------*/
  gdf_error verify_columns(gdf_column * cols[], int num_cols)
  {
    GDF_REQUIRE((nullptr != cols[0]), GDF_DATASET_EMPTY);

    gdf_size_type const required_size{cols[0]->size};

    for(int i = 0; i < num_cols; ++i)
    {
      GDF_REQUIRE(nullptr != cols[i], GDF_DATASET_EMPTY);
      GDF_REQUIRE(nullptr != cols[i]->data, GDF_DATASET_EMPTY);
      GDF_REQUIRE(required_size == cols[i]->size, GDF_COLUMN_SIZE_MISMATCH );

      // TODO Remove when null support for hash-based groupby is added
      GDF_REQUIRE(nullptr == cols[i]->valid || 0 == cols[i]->null_count, GDF_VALIDITY_UNSUPPORTED);
    }
    return GDF_SUCCESS;
  }
} // anonymous namespace

/* --------------------------------------------------------------------------*/
/**
 * @brief  Groupby operation for an arbitrary number of key columns and an
 * arbitrary number of aggregation columns.
 *
 * "Groupby" is a reduce-by-key operation where rows in one or more "key" columns
 * act as the keys and one or more "aggregation" columns hold the values that will
 * be reduced.
 *
 * The output of the operation is the set of key columns that hold all the unique keys
 * from the input key columns and a set of aggregation columns that hold the specified
 * reduction among all identical keys.
 *
 * @param[in] in_key_columns[] The input key columns
 * @param[in] num_key_columns The number of input columns to groupby
 * @param[in] in_aggregation_columns[] The columns that will be aggregated
 * @param[in] num_aggregation_columns The number of columns that will be aggregated
 * @param[in] agg_ops[] The aggregation operations to perform. The number of aggregation
 * operations must be equal to the number of aggregation columns, such that agg_op[i]
 * will be applied to in_aggregation_columns[i]
 * @param[in,out] out_key_columns[] Preallocated buffers for the output key columns
 * columns
 * @param[in,out] out_aggregation_columns[] Preallocated buffers for the output
 * aggregation columns
 * @param[in] options Structure that controls behavior of groupby operation, i.e.,
 * sort vs. hash-based implementation, whether or not the output will be sorted,
 * etc. See definition of gdf_context.
 *
 * @returns GDF_SUCCESS upon succesful completion. Otherwise appropriate error code
 */
/* ----------------------------------------------------------------------------*/
gdf_error gdf_group_by(gdf_column* in_key_columns[],
                       int num_key_columns,
                       gdf_column* in_aggregation_columns[],
                       int num_aggregation_columns,
                       gdf_agg_op agg_ops[],
                       gdf_column* out_key_columns[],
                       gdf_column* out_aggregation_columns[],
                       gdf_context* options)
{

  // TODO: Remove when single pass multi-agg is implemented
  if(num_aggregation_columns > 1)
    assert(false && "Only 1 aggregation column currently supported.");

  // TODO: Remove when the `flag_method` member is removed from `gdf_context`
  if(GDF_SORT == options->flag_method)
    assert(false && "Sort-based groupby is no longer supported.");

  // Ensure inputs aren't null
  if( (0 == num_key_columns)
      || (0 == num_aggregation_columns)
      || (nullptr == in_key_columns)
      || (nullptr == in_aggregation_columns)
      || (nullptr == agg_ops)
      || (nullptr == out_key_columns)
      || (nullptr == out_aggregation_columns)
      || (nullptr == options))
  {
    return GDF_DATASET_EMPTY;
  }

  // Return immediately if inputs are empty
  GDF_REQUIRE(0 != in_key_columns[0]->size, GDF_SUCCESS);
  GDF_REQUIRE(0 != in_aggregation_columns[0]->size, GDF_SUCCESS);

  auto result = verify_columns(in_key_columns, num_key_columns);
  GDF_REQUIRE( GDF_SUCCESS == result, result );

  result = verify_columns(in_aggregation_columns, num_aggregation_columns);
  GDF_REQUIRE( GDF_SUCCESS == result, result );

  gdf_error gdf_error_code{GDF_SUCCESS};

  PUSH_RANGE("LIBGDF_GROUPBY", GROUPBY_COLOR);


  bool sort_result = false;

  if( 0 != options->flag_sort_result){
    sort_result = true;
  }

  // Check that user is not trying to sum or avg string columns
  for(int aggregation_index = 0; aggregation_index < num_aggregation_columns; aggregation_index++){
    if(( agg_ops[aggregation_index] == GDF_SUM ||
       agg_ops[aggregation_index] == GDF_AVG ) &&
       in_aggregation_columns[aggregation_index]->dtype == GDF_STRING_CATEGORY){
      return GDF_UNSUPPORTED_DTYPE;
    }

  }

  // TODO: Only a single aggregator supported right now
  gdf_agg_op op{agg_ops[0]};

  switch(op)
  {
    case GDF_MAX:
      {
        gdf_error_code = gdf_group_by_hash<max_op>(num_key_columns,
                                                   in_key_columns,
                                                   in_aggregation_columns[0],
                                                   out_key_columns,
                                                   out_aggregation_columns[0],
                                                   sort_result);
        break;
      }
    case GDF_MIN:
      {
        gdf_error_code = gdf_group_by_hash<min_op>(num_key_columns,
                                                   in_key_columns,
                                                   in_aggregation_columns[0],
                                                   out_key_columns,
                                                   out_aggregation_columns[0],
                                                   sort_result);
        break;
      }
    case GDF_SUM:
      {
        gdf_error_code = gdf_group_by_hash<sum_op>(num_key_columns,
                                                   in_key_columns,
                                                   in_aggregation_columns[0],
                                                   out_key_columns,
                                                   out_aggregation_columns[0],
                                                   sort_result);
        break;
      }
    case GDF_COUNT:
      {
        gdf_error_code = gdf_group_by_hash<count_op>(num_key_columns,
                                                   in_key_columns,
                                                   in_aggregation_columns[0],
                                                   out_key_columns,
                                                   out_aggregation_columns[0],
                                                   sort_result);
        break;
      }
    case GDF_AVG:
      {
        gdf_error_code = gdf_group_by_hash_avg(num_key_columns,
                                               in_key_columns,
                                               in_aggregation_columns[0],
                                               out_key_columns,
                                               out_aggregation_columns[0]);

        break;
      }
    default:
      std::cerr << "Unsupported aggregation method for hash-based groupby." << std::endl;
      gdf_error_code = GDF_UNSUPPORTED_METHOD;
  }

  GDF_REQUIRE(GDF_SUCCESS == gdf_error_code, gdf_error_code);

  // The following code handles propogating an NVCategory into columns which are of type nvcategory
  for(int key_index = 0; key_index < num_key_columns; key_index++){
    if(out_key_columns[key_index]->dtype == GDF_STRING_CATEGORY){
      gdf_error_code = nvcategory_gather(out_key_columns[key_index],
                                         static_cast<NVCategory *>(in_key_columns[key_index]->dtype_info.category));
      GDF_REQUIRE(GDF_SUCCESS == gdf_error_code, gdf_error_code);
    }
  }
  for(int out_column_index = 0; out_column_index < num_aggregation_columns; out_column_index++){
    if(out_aggregation_columns[out_column_index]->dtype == GDF_STRING_CATEGORY){
      gdf_error_code = nvcategory_gather(out_aggregation_columns[out_column_index],
                                         static_cast<NVCategory *>(in_aggregation_columns[out_column_index]->dtype_info.category));
      GDF_REQUIRE(GDF_SUCCESS == gdf_error_code, gdf_error_code);
    }
  }

  POP_RANGE();

  return gdf_error_code;
}

void gdf_unique_indices(cudf::table const& input_table,
                        gdf_index_type* unique_indices,
                        gdf_size_type* num_unique_indices,
                        gdf_context* context)
{
  CUDF_EXPECTS(nullptr != unique_indices, "unique_indices is null");
  CUDF_EXPECTS(nullptr != num_unique_indices, "num_unique_indices is null");

  gdf_size_type ncols = input_table.num_columns();
  gdf_size_type nrows = input_table.num_rows();

  rmm::device_vector<void*> d_cols(ncols);
  rmm::device_vector<int> d_types(ncols, 0);
  void** d_col_data = d_cols.data().get();
  int* d_col_types = d_types.data().get();

  bool nulls_are_smallest = (context->flag_null_sort_behavior == GDF_NULL_AS_SMALLEST);

  gdf_index_type* result_end;
  hipStream_t stream;
  hipStreamCreate(&stream);
  auto exec = rmm::exec_policy(stream)->on(stream);

  bool const have_nulls{ std::any_of(input_table.begin(), input_table.end(), [](gdf_column const* col){ return col->null_count > 0;}) };
  if (have_nulls){
    rmm::device_vector<gdf_valid_type*> d_valids(ncols);
    gdf_valid_type** d_valids_data = d_valids.data().get();

    soa_col_info(input_table.columns(), ncols, d_col_data, d_valids_data, d_col_types);

    LesserRTTI<gdf_size_type> comp(d_col_data, d_valids_data, d_col_types, nullptr, ncols, nulls_are_smallest);

    auto counting_iter = thrust::make_counting_iterator<gdf_size_type>(0);

    result_end = thrust::unique_copy(exec, counting_iter, counting_iter+nrows,
                              unique_indices,
                              [comp]  __device__(gdf_size_type key1, gdf_size_type key2){
                              return comp.equal_with_nulls(key1, key2);
                            });

  } else {
    soa_col_info(input_table.columns(), ncols, d_col_data, nullptr, d_col_types);

    LesserRTTI<gdf_size_type> comp(d_col_data, nullptr, d_col_types, nullptr, ncols, nulls_are_smallest);

    auto counting_iter = thrust::make_counting_iterator<gdf_size_type>(0);

    result_end = thrust::unique_copy(exec, counting_iter, counting_iter+nrows,
                              unique_indices,
                              [comp]  __device__(gdf_size_type key1, gdf_size_type key2){
                              return comp.equal(key1, key2);
                            });
  }

  gdf_size_type new_sz = thrust::distance(unique_indices, result_end);
  *num_unique_indices = new_sz;
  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);
}

void gdf_group_by_without_aggregations(cudf::table const& input_table,
                                      gdf_size_type num_key_cols,
                                      gdf_index_type const * key_col_indices,
                                      cudf::table* destination_table,
                                      gdf_index_type* group_start_indices,
                                      gdf_size_type* num_group_start_indices,
                                      gdf_context* context)
{
  CUDF_EXPECTS(nullptr != key_col_indices, "key_col_indices is null");
  CUDF_EXPECTS(0 < num_key_cols, "number of key colums should be greater than zero");
  CUDF_EXPECTS(nullptr != destination_table, "destination table is null");

  if (0 == input_table.num_rows()) {
    return;
  }

  CUDF_EXPECTS(input_table.num_columns() == destination_table->num_columns(),
              "Mismatched number of columns");

  gdf_size_type nrows = input_table.num_rows();

  bool group_by_keys_contain_nulls = false;
  gdf_column** raw_input_cols = input_table.columns();
  std::vector<gdf_column*> key_cols_vect(num_key_cols);
  for (gdf_size_type i = 0; i < num_key_cols; i++){
    key_cols_vect[i] = raw_input_cols[key_col_indices[i]];
    group_by_keys_contain_nulls = (group_by_keys_contain_nulls || key_cols_vect[i]->null_count > 0);
  }
  cudf::table key_col_table(key_cols_vect.data(), key_cols_vect.size());

  rmm::device_vector<gdf_size_type> sorted_indices(nrows);
  gdf_column sorted_indices_col;
  CUDF_TRY(gdf_column_view(&sorted_indices_col, (void*)(sorted_indices.data().get()),
                          nullptr, nrows, GDF_INT32));

  if (context->flag_groupby_include_nulls || !group_by_keys_contain_nulls){  // SQL style
    CUDF_TRY(gdf_order_by(key_col_table.columns(),
                          nullptr,
                          key_col_table.num_columns(),
                          &sorted_indices_col,
                          context));
  } else {  // Pandas style
    gdf_context temp_ctx;
    temp_ctx.flag_null_sort_behavior = GDF_NULL_AS_LARGEST_FOR_MULTISORT;

    CUDF_TRY(gdf_order_by(key_col_table.columns(),
                          nullptr,
                          key_col_table.num_columns(),
                          &sorted_indices_col,
                          &temp_ctx));

    // lets filter out all the nulls in the group by key column by:
    // we will take the data which has been sorted such that the nulls in the group by keys are all last
    // then using row_bitmask we can count how many rows have a null in the group by keys and use that 
    // to resize the data
    auto orderby_cols_bitmask = row_bitmask(key_col_table);
    int valid_count;
    CUDF_TRY(gdf_count_nonzero_mask(reinterpret_cast<gdf_valid_type*>(orderby_cols_bitmask.data().get()),
                                    nrows, &valid_count));
    
    for (gdf_size_type i = 0; i < input_table.num_columns(); ++i) {
      destination_table->get_column(i)->size = valid_count;
    }
  }

  // run gather operation to establish new order
  cudf::gather(&input_table, sorted_indices.data().get(), destination_table);

  for (gdf_size_type i = 0; i < num_key_cols; i++){
    key_cols_vect[i] = destination_table->get_column(key_col_indices[i]);
  }
  cudf::table key_col_sorted_table(key_cols_vect.data(), key_cols_vect.size());
  gdf_unique_indices(key_col_sorted_table, group_start_indices, num_group_start_indices, context);
}

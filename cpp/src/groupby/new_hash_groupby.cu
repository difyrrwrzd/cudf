#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf.h>
#include <bitmask/bit_mask.cuh>
#include <dataframe/device_table.cuh>
#include <groupby.hpp>
#include <hash/concurrent_unordered_map.cuh>
#include <types.hpp>
#include <utilities/device_atomics.cuh>
#include <utilities/release_assert.cuh>
#include <utilities/type_dispatcher.hpp>
#include "new_hash_groupby.hpp"

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/fill.h>
#include <type_traits>
#include <vector>

namespace cudf {
namespace detail {

namespace {

using namespace groupby;

struct identity_initializer {
  template <typename T>
  T get_identity(distributive_operators op) {
    switch (op) {
      case distributive_operators::SUM:
        return cudf::DeviceSum::identity<T>();
      case distributive_operators::MIN:
        return cudf::DeviceMin::identity<T>();
      case distributive_operators::MAX:
        return cudf::DeviceMax::identity<T>();
      case distributive_operators::COUNT:
        return cudf::DeviceSum::identity<T>();
      default:
        CUDF_FAIL("Invalid aggregation operation.");
    }
  }

  template <typename T>
  void operator()(gdf_column const& col, distributive_operators op,
                  hipStream_t stream = 0) {
    T* typed_data = static_cast<T*>(col.data);
    thrust::fill(rmm::exec_policy(stream)->on(stream), typed_data,
                 typed_data + col.size, get_identity<T>(op));

    // For COUNT operator, initialize column's bitmask to be all valid
    if ((nullptr != col.valid) and (distributive_operators::COUNT == op)) {
      CUDA_TRY(hipMemsetAsync(
          col.valid, 0xff,
          sizeof(gdf_valid_type) * gdf_valid_allocation_size(col.size),
          stream));
    }
  }
};

/**---------------------------------------------------------------------------*
 * @brief Initializes each column in a table with a corresponding identity value
 * of an aggregation operation.
 *
 * The `i`th column will be initialized with the identity value of the `i`th
 * aggregation operation.
 *
 * @note The validity bitmask for the column corresponding to a COUNT operator
 * will be initialized to all valid.
 *
 * @param table The table of columns to initialize.
 * @param operators The aggregation operations whose identity values will be
 *used to initialize the columns.
 *---------------------------------------------------------------------------**/
void initialize_with_identity(
    cudf::table const& table,
    std::vector<distributive_operators> const& operators,
    hipStream_t stream = 0) {
  // TODO: Initialize all the columns in a single kernel instead of invoking one
  // kernel per column
  for (gdf_size_type i = 0; i < table.num_columns(); ++i) {
    gdf_column const* col = table.get_column(i);
    cudf::type_dispatcher(col->dtype, identity_initializer{}, *col,
                          operators[i]);
  }
}

/**---------------------------------------------------------------------------*
 * @brief Determines accumulator type based on input type and operation.
 *
 * @tparam InputType The type of the input to the aggregation operation
 * @tparam op The aggregation operation performed
 * @tparam dummy Dummy for SFINAE
 *---------------------------------------------------------------------------**/
template <typename SourceType, distributive_operators op, typename dummy = void>
struct result_type {
  using type = void;
};

// Computing MIN of SourceType, use SourceType accumulator
template <typename SourceType>
struct result_type<SourceType, distributive_operators::MIN> {
  using type = SourceType;
};

// Computing MAX of SourceType, use SourceType accumulator
template <typename SourceType>
struct result_type<SourceType, distributive_operators::MAX> {
  using type = SourceType;
};

// Always use int64_t accumulator for COUNT
template <typename SourceType>
struct result_type<SourceType, distributive_operators::COUNT> {
  using type = int64_t;
};

// Summing integers of any type, always use int64_t accumulator
template <typename SourceType>
struct result_type<
    SourceType, distributive_operators::SUM,
    typename std::enable_if_t<std::is_integral<SourceType>::value>> {
  using type = int64_t;
};

// Summing float/doubles, use same type accumulator
template <typename SourceType>
struct result_type<
    SourceType, distributive_operators::SUM,
    typename std::enable_if_t<std::is_floating_point<SourceType>::value>> {
  using type = SourceType;
};

/**---------------------------------------------------------------------------*
 * @brief Error case for invalid combinations of SourceType and operator.
 *
 * For an invalid combination of SourceType and operator,
 * `result_type<SourceType, operator>::type` yields a `void` TargetType. This
 * specialization will be invoked for any invalid combination and cause a
 * runtime error.
 *---------------------------------------------------------------------------**/
template <typename TargetType, typename SourceType, typename Op,
          std::enable_if_t<std::is_void<TargetType>::value, int>* = nullptr>
__device__ inline void binary_op(gdf_column const& target,
                                 gdf_size_type target_index,
                                 gdf_column const& source,
                                 gdf_size_type source_index, Op&& op) {
  release_assert(false && "Invalid Source type and Aggregation combination.");
}

/**---------------------------------------------------------------------------*
 * @brief Performs inplace update of a target element via a binary operation
 * with a source element.
 *
 * Atomically performs `target[target_index] = target[target_index] op
 * source[source_index]`
 *
 * @tparam TargetType Type of the target element
 * @tparam SourceType Type of the source element
 * @tparam Op Type of the binary operation to perform
 * @param target Column containing target element
 * @param target_index Index of the target element
 * @param source Column containing source element
 * @param source_index Index of the source element
 * @param op The aggregation operation to perform
 *---------------------------------------------------------------------------**/
template <typename TargetType, typename SourceType, typename Op,
          std::enable_if_t<not std::is_void<TargetType>::value, int>* = nullptr>
__device__ inline void binary_op(gdf_column const& target,
                                 gdf_size_type target_index,
                                 gdf_column const& source,
                                 gdf_size_type source_index, Op&& op) {
  assert(gdf_dtype_of<TargetType>() == target.dtype);

  SourceType const& source_element{
      static_cast<SourceType const*>(source.data)[source_index]};

  cudf::genericAtomicOperation(
      &(static_cast<TargetType*>(target.data)[target_index]),
      static_cast<TargetType>(source_element), op);
}

/**---------------------------------------------------------------------------*
 * @brief Increments a target value by one.
 *
 * @tparam TargetType Target element's type
 * @param target The column containing the target element
 * @param target_index Index of the target element
 *---------------------------------------------------------------------------**/
template <typename TargetType>
__device__ inline void count_op(gdf_column const& target,
                                gdf_size_type target_index) {
  static_assert(std::is_integral<TargetType>::value,
                "TargetType of count operation must be integral.");
  assert(gdf_dtype_of<TargetType>() == target.dtype);
  cudf::genericAtomicOperation(
      &(static_cast<TargetType*>(target.data)[target_index]), TargetType{1},
      DeviceSum{});
}

/**---------------------------------------------------------------------------*
 * @brief Sets the specified bit in a column's validity bitmask.
 *
 * @note Setting a bit invokes an atomic operation. Therefore, to avoid
 * unnecessary/expensive atomic operations, the bit is only set if it is not
 * already set.
 *
 * @param target Column whose bitmask will be set
 * @param target_index Index of the bit to set
 *---------------------------------------------------------------------------**/
__device__ inline void set_valid_bit(gdf_column const& target,
                                     gdf_size_type target_index) {
  if (not gdf_is_valid(target.valid, target_index)) {
    bit_mask::set_bit_safe(
        reinterpret_cast<bit_mask::bit_mask_t*>(target.valid), target_index);
  }
}

struct elementwise_aggregator {
  template <typename SourceType>
  __device__ inline void operator()(gdf_column const& target,
                                    gdf_size_type target_index,
                                    gdf_column const& source,
                                    gdf_size_type source_index,
                                    distributive_operators op) {
    // TODO Can we avoid setting the target's valid bit for every binary
    // operation? Technically, it only needs to be set upon the first succesful
    // update of the target element.
    switch (op) {
      case distributive_operators::MIN: {
        using TargetType =
            typename result_type<SourceType, distributive_operators::MIN>::type;
        binary_op<TargetType, SourceType>(target, target_index, source,
                                          source_index, DeviceMin{});
        set_valid_bit(target, target_index);
        break;
      }
      case distributive_operators::MAX: {
        using TargetType =
            typename result_type<SourceType, distributive_operators::MAX>::type;
        binary_op<TargetType, SourceType>(target, target_index, source,
                                          source_index, DeviceMax{});
        set_valid_bit(target, target_index);
        break;
      }
      case distributive_operators::SUM: {
        using TargetType =
            typename result_type<SourceType, distributive_operators::SUM>::type;
        binary_op<TargetType, SourceType>(target, target_index, source,
                                          source_index, DeviceSum{});
        set_valid_bit(target, target_index);
        break;
      }
      case distributive_operators::COUNT: {
        using TargetType =
            typename result_type<SourceType,
                                 distributive_operators::COUNT>::type;
        count_op<TargetType>(target, target_index);
        break;
      }
      default:
        return;
    }
  }
};  // namespace

/**---------------------------------------------------------------------------*
 * @brief Performs an in-place update by performing elementwise aggregation
 * operations between a target and source row.
 *
 * For `i` in `[0, num_columns)`, each element in the target row is updated as:
 *
 *```
 * target_row[i] = target_row[i] op[i] source_row[i]
 *```
 * @note If a source element is NULL, the aggregation operation for
 * that column is skipped.
 *
 * @note If a target element is NULL, it is assumed that the value of the NULL
 * element is the identity value of the aggregation operation being performed.
 * The aggregation operation is performed between the source element and the
 * identity value, and the target element's bit is set to indicate it is no
 * longer NULL.
 *
 * @note It is assumed that the target element of a COUNT operation is *never*
 * NULL.
 *
 * @param target Table containing the target row
 * @param target_index Index of the target row
 * @param source Table cotaning the source row
 * @param source_index Index of the source row
 * @param ops Array of operators to perform between the elements of the
 * target and source rows
 *---------------------------------------------------------------------------**/
__device__ inline void aggregate_row(device_table const& target,
                                     gdf_size_type target_index,
                                     device_table const& source,
                                     gdf_size_type source_index,
                                     distributive_operators* ops) {
  thrust::for_each(
      thrust::seq, thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(target.num_columns()),
      [target, target_index, source, source_index, ops](gdf_size_type i) {
        if (gdf_is_valid(source.get_column(i)->valid, source_index)) {
          cudf::type_dispatcher(source.get_column(i)->dtype,
                                elementwise_aggregator{}, *target.get_column(i),
                                target_index, *source.get_column(i),
                                source_index, ops[i]);
        }
      });
}

struct type_mapper {
  template <typename InputT>
  gdf_dtype operator()(distributive_operators op) {
    switch (op) {
      case distributive_operators::MIN:
        return gdf_dtype_of<
            typename result_type<InputT, distributive_operators::MIN>::type>();
      case distributive_operators::MAX:
        return gdf_dtype_of<
            typename result_type<InputT, distributive_operators::MAX>::type>();
      case distributive_operators::SUM:
        return gdf_dtype_of<
            typename result_type<InputT, distributive_operators::SUM>::type>();
      case distributive_operators::COUNT:
        return gdf_dtype_of<typename result_type<
            InputT, distributive_operators::COUNT>::type>();
      default:
        return GDF_invalid;
    }
  }
};

/**---------------------------------------------------------------------------*
 * @brief Returns the output gdf_dtype to use for a combination of input
 * gdf_dtype and aggregation operation.
 *
 * @param input_type The type of the input aggregation column
 * @param op The aggregation operation
 * @return gdf_dtype Type to use for output aggregation column
 *---------------------------------------------------------------------------**/
gdf_dtype output_dtype(gdf_dtype input_type, distributive_operators op) {
  return cudf::type_dispatcher(input_type, type_mapper{}, op);
}
}  // namespace

std::tuple<cudf::table, cudf::table> hash_groupby(
    cudf::table const& keys, cudf::table const& values,
    std::vector<cudf::groupby::distributive_operators> const& operators,
    groupby::Options options, hipStream_t stream) {
  // The exact output size is unknown a priori, therefore, use the input size as
  // an upper bound
  gdf_size_type const output_size{keys.num_rows()};

  // Allocate output keys
  std::vector<gdf_dtype> key_dtypes(keys.num_columns());
  std::transform(keys.begin(), keys.end(), key_dtypes.begin(),
                 [](gdf_column const* col) { return col->dtype; });
  cudf::table output_keys{output_size, key_dtypes, true, stream};

  // Allocate/initialize output values
  // TODO: Move to function.
  std::vector<gdf_dtype> output_dtypes(values.num_columns());
  std::transform(
      values.begin(), values.end(), operators.begin(), output_dtypes.begin(),
      [](gdf_column const* input_col, groupby::distributive_operators op) {
        gdf_dtype t = output_dtype(input_col->dtype, op);
        CUDF_EXPECTS(
            t != GDF_invalid,
            "Invalid combination of input type and aggregation operation.");
        return t;
      });
  cudf::table output_values{output_size, output_dtypes, true, stream};
  initialize_with_identity(output_values, operators, stream);

  using map_type = concurrent_unordered_map<
      gdf_size_type, gdf_size_type, std::numeric_limits<gdf_size_type>::max(),
      default_hash<gdf_size_type>, equal_to<gdf_size_type>,
      legacy_allocator<thrust::pair<gdf_size_type, gdf_size_type>>>;

  std::unique_ptr<map_type> map =
      std::make_unique<map_type>(compute_hash_table_size(keys.num_rows()), 0);

  rmm::device_vector<groupby::distributive_operators> d_operators(operators);

  CHECK_STREAM(stream);

  return std::make_tuple(output_keys, output_values);
}

}  // namespace detail
}  // namespace cudf

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf.h>
#include <groupby.hpp>
#include <hash/concurrent_unordered_map.cuh>
#include <types.hpp>
#include <utilities/type_dispatcher.hpp>
#include "aggregation_operations.hpp"
#include "new_hash_groupby.hpp"

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/fill.h>
#include <vector>

namespace cudf {
namespace detail {

namespace {

struct identity_initializer {
  template <typename T>
  T get_identity(groupby::distributive_operators op) {
    switch (op) {
      case groupby::distributive_operators::SUM:
        return sum_op<T>::IDENTITY;
      case groupby::distributive_operators::MIN:
        return min_op<T>::IDENTITY;
      case groupby::distributive_operators::MAX:
        return max_op<T>::IDENTITY;
      case groupby::distributive_operators::COUNT:
        return sum_op<T>::IDENTITY;
      default:
        CUDF_FAIL("Invalid aggregation operation.");
    }
  }

  template <typename T>
  void operator()(gdf_column const& col, groupby::distributive_operators op,
                  hipStream_t stream = 0) {
    T* typed_data = static_cast<T*>(col.data);
    thrust::fill(rmm::exec_policy(stream)->on(stream), typed_data,
                 typed_data + col.size, get_identity<T>(op));
  }
};

/**---------------------------------------------------------------------------*
 * @brief Initializes each column in a table with a corresponding identity value
 * of an aggregation operation.
 *
 * The `i`th column will be initialized with the identity value of the `i`th
 * aggregation operation.
 *
 * @param table The table of columns to initialize.
 * @param operators The aggregation operations whose identity values will be
 *used to initialize the columns.
 *---------------------------------------------------------------------------**/
void initialize_with_identity(
    cudf::table const& table,
    std::vector<cudf::groupby::distributive_operators> const& operators,
    hipStream_t stream = 0) {
  // TODO: Initialize all the columns in a single kernel instead of invoking one
  // kernel per column
  for (gdf_size_type i = 0; i < table.num_columns(); ++i) {
    gdf_column const* col = table.get_column(i);
    cudf::type_dispatcher(col->dtype, identity_initializer{}, *col,
                          operators[i]);
  }
}
}  // namespace

std::tuple<cudf::table, cudf::table> hash_groupby(
    cudf::table const& keys, cudf::table const& values,
    std::vector<cudf::groupby::distributive_operators> const& operators,
    std::vector<gdf_dtype> const& output_dtypes, hipStream_t stream) {
  // Create the output key and value tables
  // The exact output size is unknown a priori, therefore, use the input size as
  // an upper bound
  std::vector<gdf_dtype> key_dtypes(keys.num_columns());
  std::transform(keys.begin(), keys.end(), key_dtypes.begin(),
                 [](gdf_column const* col) { return col->dtype; });
  cudf::table output_keys{keys.num_rows(), key_dtypes, true, stream};
  cudf::table output_values{keys.num_rows(), output_dtypes, true, stream};
  initialize_with_identity(output_values, operators, stream);

  using map_type = concurrent_unordered_map<
      gdf_size_type, gdf_size_type, std::numeric_limits<gdf_size_type>::max(),
      default_hash<gdf_size_type>, equal_to<gdf_size_type>,
      legacy_allocator<thrust::pair<gdf_size_type, gdf_size_type> > >;

  std::unique_ptr<map_type> map =
      std::make_unique<map_type>(compute_hash_table_size(keys.num_rows()), 0);

  CHECK_STREAM(stream);

  return std::make_tuple(output_keys, output_values);
}

}  // namespace detail
}  // namespace cudf

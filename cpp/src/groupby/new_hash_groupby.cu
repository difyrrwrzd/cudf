#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf.h>
#include <bitmask/bit_mask.cuh>
#include <dataframe/device_table.cuh>
#include <groupby.hpp>
#include <hash/concurrent_unordered_map.cuh>
#include <types.hpp>
#include <utilities/device_atomics.cuh>
#include <utilities/release_assert.cuh>
#include <utilities/type_dispatcher.hpp>
#include "new_hash_groupby.hpp"

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/fill.h>
#include <type_traits>
#include <vector>

namespace cudf {
namespace detail {

namespace {

using namespace groupby;

struct identity_initializer {
  template <typename T>
  T get_identity(distributive_operators op) {
    switch (op) {
      case distributive_operators::SUM:
        return cudf::DeviceSum::identity<T>();
      case distributive_operators::MIN:
        return cudf::DeviceMin::identity<T>();
      case distributive_operators::MAX:
        return cudf::DeviceMax::identity<T>();
      case distributive_operators::COUNT:
        return cudf::DeviceSum::identity<T>();
      default:
        CUDF_FAIL("Invalid aggregation operation.");
    }
  }

  template <typename T>
  void operator()(gdf_column const& col, distributive_operators op,
                  hipStream_t stream = 0) {
    T* typed_data = static_cast<T*>(col.data);
    thrust::fill(rmm::exec_policy(stream)->on(stream), typed_data,
                 typed_data + col.size, get_identity<T>(op));
  }
};

/**---------------------------------------------------------------------------*
 * @brief Initializes each column in a table with a corresponding identity value
 * of an aggregation operation.
 *
 * The `i`th column will be initialized with the identity value of the `i`th
 * aggregation operation.
 *
 * @param table The table of columns to initialize.
 * @param operators The aggregation operations whose identity values will be
 *used to initialize the columns.
 *---------------------------------------------------------------------------**/
void initialize_with_identity(
    cudf::table const& table,
    std::vector<distributive_operators> const& operators,
    hipStream_t stream = 0) {
  // TODO: Initialize all the columns in a single kernel instead of invoking one
  // kernel per column
  for (gdf_size_type i = 0; i < table.num_columns(); ++i) {
    gdf_column const* col = table.get_column(i);
    cudf::type_dispatcher(col->dtype, identity_initializer{}, *col,
                          operators[i]);
  }
}

/**---------------------------------------------------------------------------*
 * @brief Determines accumulator type based on input type and operation.
 *
 * @tparam InputType The type of the input to the aggregation operation
 * @tparam op The aggregation operation performed
 * @tparam dummy Dummy for SFINAE
 *---------------------------------------------------------------------------**/
template <typename InputType, distributive_operators op, typename dummy = void>
struct result_type {
  using type = void;
};

// Computing MIN of T, use T accumulator
template <typename T>
struct result_type<T, distributive_operators::MIN> {
  using type = T;
};

// Computing MAX of T, use T accumulator
template <typename T>
struct result_type<T, distributive_operators::MAX> {
  using type = T;
};

// Counting T, always use int64_t accumulator
template <typename T>
struct result_type<T, distributive_operators::COUNT> {
  using type = int64_t;
};

// Summing integers of any type, always use int64_t
template <typename T>
struct result_type<T, distributive_operators::SUM,
                   typename std::enable_if<std::is_integral<T>::value>::type> {
  using type = int64_t;
};

// Summing float/doubles, use same type
template <typename T>
struct result_type<
    T, distributive_operators::SUM,
    typename std::enable_if<std::is_floating_point<T>::value>::type> {
  using type = T;
};

/**---------------------------------------------------------------------------*
 * @brief Error case for invalid combinations of SourceType and operator.
 *
 * For an invalid combination of SourceType and operator,
 * `result_type<SourceType, operator>::type` yields a `void` TargetType. This
 * specialization will be invoked for any invalid combination and cause a
 *runtime error.
 *---------------------------------------------------------------------------**/
template <typename TargetType, typename SourceType, typename Op,
          std::enable_if_t<std::is_void<TargetType>::value, int>* = nullptr>
__device__ inline void binary_op(gdf_column const& target,
                                 gdf_size_type target_index,
                                 gdf_column const& source,
                                 gdf_size_type source_index, Op&& op) {
  release_assert(false && "Invalid type and aggregation combination.");
}

/**---------------------------------------------------------------------------*
 * @brief Performs a binary operation between two elements in two columns.
 *
 * @tparam TargetType
 * @tparam SourceType
 * @tparam Op
 * @tparam nullptr
 * @param target
 * @param target_index
 * @param source
 * @param source_index
 * @param op
 *---------------------------------------------------------------------------**/
template <typename TargetType, typename SourceType, typename Op,
          std::enable_if_t<not std::is_void<TargetType>::value, int>* = nullptr>
__device__ inline void binary_op(gdf_column const& target,
                                 gdf_size_type target_index,
                                 gdf_column const& source,
                                 gdf_size_type source_index, Op&& op) {
  if (gdf_is_valid(source.valid, source_index)) {
    cudf::genericAtomicOperation(
        &(static_cast<TargetType*>(target.data)[target_index]),
        static_cast<TargetType>(*static_cast<SourceType*>(source.data)), op);

    // TODO Inefficient to always check the target's validity bit
    // We only need to set the target's validity bit on the first
    // succesful update of the target element
    if (not gdf_is_valid(target.valid, target_index)) {
      bit_mask::set_bit_safe(
          reinterpret_cast<bit_mask::bit_mask_t*>(target.valid), target_index);
    }
  }
}

/**---------------------------------------------------------------------------*
 * @brief
 *
 * @tparam TargetType
 * @param target
 * @param target_index
 * @param source_is_valid
 *---------------------------------------------------------------------------**/
template <typename TargetType>
__device__ inline void count_op(gdf_column const& target,
                                gdf_size_type target_index,
                                bool source_is_valid) {
  if (source_is_valid) {
    cudf::genericAtomicOperation(
        &(static_cast<TargetType*>(target.data)[target_index]), TargetType{1},
        DeviceSum{});
  }
  // For COUNT, the output can never be NULL. The count of a columns of
  // all NULLs is just zero. Therefore, always set the output validity
  // bit
  if (not gdf_is_valid(target.valid, target_index)) {
    bit_mask::set_bit_safe(
        reinterpret_cast<bit_mask::bit_mask_t*>(target.valid), target_index);
  }
}

struct aggregate_elements {
  template <typename SourceType>
  __device__ inline void operator()(gdf_column const& target,
                                    gdf_size_type target_index,
                                    gdf_column const& source,
                                    gdf_size_type source_index,
                                    distributive_operators op) {
    switch (op) {
      case distributive_operators::MIN: {
        using TargetType =
            typename result_type<SourceType, distributive_operators::MIN>::type;

        binary_op<TargetType, SourceType>(target, target_index, source,
                                          source_index, DeviceMin{});
        break;
      }
      case distributive_operators::MAX: {
        using TargetType =
            typename result_type<SourceType, distributive_operators::MAX>::type;
        binary_op<TargetType, SourceType>(target, target_index, source,
                                          source_index, DeviceMax{});
        break;
      }
      case distributive_operators::SUM: {
        using TargetType =
            typename result_type<SourceType, distributive_operators::SUM>::type;
        binary_op<TargetType, SourceType>(target, target_index, source,
                                          source_index, DeviceSum{});
        break;
      }
      case distributive_operators::COUNT: {
        using TargetType =
            typename result_type<SourceType,
                                 distributive_operators::COUNT>::type;
        count_op<TargetType>(target, target_index,
                             gdf_is_valid(source.valid, source_index));
        break;
      }
      default:
        return;
    }
  }
};  // namespace

/**---------------------------------------------------------------------------*
 * @brief Updates a target row by performing a set of aggregation operations
 * between it and a source row.
 *
 * @param target Table containing the target row
 * @param target_index Index of the target row
 * @param source Table cotaning the source row
 * @param source_index Index of the source row
 * @param ops Array of operators to perform between the elements of the
 * target and source rows
 *---------------------------------------------------------------------------**/
__device__ inline void aggregate_row(device_table const& target,
                                     gdf_size_type target_index,
                                     device_table const& source,
                                     gdf_size_type source_index,
                                     distributive_operators* ops) {
  thrust::for_each(
      thrust::seq, thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(target.num_columns()),
      [target, target_index, source, source_index, ops](gdf_size_type i) {
        cudf::type_dispatcher(source.get_column(i)->dtype, aggregate_elements{},
                              *target.get_column(i), target_index,
                              *source.get_column(i), source_index, ops[i]);
      });
}

struct type_mapper {
  template <typename InputT>
  gdf_dtype operator()(distributive_operators op) {
    switch (op) {
      case distributive_operators::MIN:
        return gdf_dtype_of<
            typename result_type<InputT, distributive_operators::MIN>::type>();
      case distributive_operators::MAX:
        return gdf_dtype_of<
            typename result_type<InputT, distributive_operators::MAX>::type>();
      case distributive_operators::SUM:
        return gdf_dtype_of<
            typename result_type<InputT, distributive_operators::SUM>::type>();
      case distributive_operators::COUNT:
        return gdf_dtype_of<typename result_type<
            InputT, distributive_operators::COUNT>::type>();
      default:
        return GDF_invalid;
    }
  }
};

/**---------------------------------------------------------------------------*
 * @brief Determines the output gdf_dtype that should be used for a given input
 * gdf_dtype and operator.
 *
 * @param input_type The type of the input aggregation column
 * @param op The aggregation operation
 * @return gdf_dtype Type to use for output aggregation column
 *---------------------------------------------------------------------------**/
gdf_dtype output_dtype(gdf_dtype input_type, distributive_operators op) {
  return cudf::type_dispatcher(input_type, type_mapper{}, op);
}
}  // namespace

std::tuple<cudf::table, cudf::table> hash_groupby(
    cudf::table const& keys, cudf::table const& values,
    std::vector<cudf::groupby::distributive_operators> const& operators,
    groupby::Options options, hipStream_t stream) {
  // The exact output size is unknown a priori, therefore, use the input size as
  // an upper bound
  gdf_size_type const output_size{keys.num_rows()};

  // Allocate output keys
  std::vector<gdf_dtype> key_dtypes(keys.num_columns());
  std::transform(keys.begin(), keys.end(), key_dtypes.begin(),
                 [](gdf_column const* col) { return col->dtype; });
  cudf::table output_keys{output_size, key_dtypes, true, stream};

  // Allocate/initialize output values
  std::vector<gdf_dtype> output_dtypes(values.num_columns());
  std::transform(
      values.begin(), values.end(), operators.begin(), output_dtypes.begin(),
      [](gdf_column const* input_col, groupby::distributive_operators op) {
        gdf_dtype t = output_dtype(input_col->dtype, op);
        CUDF_EXPECTS(
            t != GDF_invalid,
            "Invalid combination of input type and aggregation operation.");
        return t;
      });
  cudf::table output_values{output_size, output_dtypes, true, stream};
  initialize_with_identity(output_values, operators, stream);

  using map_type = concurrent_unordered_map<
      gdf_size_type, gdf_size_type, std::numeric_limits<gdf_size_type>::max(),
      default_hash<gdf_size_type>, equal_to<gdf_size_type>,
      legacy_allocator<thrust::pair<gdf_size_type, gdf_size_type>>>;

  std::unique_ptr<map_type> map =
      std::make_unique<map_type>(compute_hash_table_size(keys.num_rows()), 0);

  rmm::device_vector<groupby::distributive_operators> d_operators(operators);

  CHECK_STREAM(stream);

  return std::make_tuple(output_keys, output_values);
}

}  // namespace detail
}  // namespace cudf

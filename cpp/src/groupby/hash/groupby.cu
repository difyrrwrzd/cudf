#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/cudf.h>
#include <bitmask/bit_mask.cuh>
#include <cudf/binaryop.hpp>
#include <cudf/bitmask.hpp>
#include <cudf/copying.hpp>
#include <cudf/groupby.hpp>
#include <cudf/table.hpp>
#include <hash/concurrent_unordered_map.cuh>
#include <string/nvcategory_util.hpp>
#include <table/device_table.cuh>
#include <table/device_table_row_operators.cuh>
#include <utilities/column_utils.hpp>
#include <utilities/cuda_utils.hpp>
#include <utilities/device_atomics.cuh>
#include <utilities/release_assert.cuh>
#include <utilities/type_dispatcher.hpp>
#include "groupby.hpp"
#include "groupby_kernels.cuh"
#include "type_info.hpp"

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/fill.h>
#include <algorithm>
#include <map>
#include <set>
#include <type_traits>
#include <unordered_map>
#include <vector>

namespace cudf {
namespace groupby {
namespace hash {
namespace {
using AggRequestType = std::pair<gdf_column*, operators>;
/**---------------------------------------------------------------------------*
 * @brief Verifies the requested aggregation is valid for the type of the value
 * column.
 *
 * Given a table of values and a set of operators, verifies that `ops[i]` is
 * valid to perform on `column[i]`.
 *
 * @throw cudf::logic_error if an invalid combination of value type and operator
 * is requested.
 *
 * @param values The table of columns
 * @param ops The aggregation operators
 *---------------------------------------------------------------------------**/
void verify_operators(table const& values, std::vector<operators> const& ops) {
  CUDF_EXPECTS(static_cast<gdf_size_type>(ops.size()) == values.num_columns(),
               "Size mismatch between ops and value columns");
  for (gdf_size_type i = 0; i < values.num_columns(); ++i) {
    // TODO Add more checks here, i.e., can't compute sum of non-arithemtic
    // types
    if ((ops[i] == SUM) and
        (values.get_column(i)->dtype == GDF_STRING_CATEGORY)) {
      CUDF_FAIL(
          "Cannot compute SUM aggregation of GDF_STRING_CATEGORY column.");
    }
  }
}

/**---------------------------------------------------------------------------*
 * @brief Deteremines target gdf_dtypes to use for combinations of source
 * gdf_dtypes and aggregation operations.
 *
 * Given vectors of source gdf_dtypes and corresponding aggregation operations
 * to be performed on that type, returns a vector of gdf_dtypes to use to store
 * the result of the aggregation operations.
 *
 * @param source_dtypes The source types
 * @param op The aggregation operations
 * @return Target gdf_dtypes to use for the target aggregation columns
 *---------------------------------------------------------------------------**/
inline std::vector<gdf_dtype> target_dtypes(
    std::vector<gdf_dtype> const& source_dtypes,
    std::vector<operators> const& ops) {
  std::vector<gdf_dtype> output_dtypes(source_dtypes.size());

  std::transform(
      source_dtypes.begin(), source_dtypes.end(), ops.begin(),
      output_dtypes.begin(), [](gdf_dtype source_dtype, operators op) {
        gdf_dtype t =
            cudf::type_dispatcher(source_dtype, target_type_mapper{}, op);
        CUDF_EXPECTS(
            t != GDF_invalid,
            "Invalid combination of input type and aggregation operation.");
        return t;
      });

  return output_dtypes;
}

/**---------------------------------------------------------------------------*
 * @brief Dispatched functor to initialize a column with the identity of an
 *aggregation operation.
 *---------------------------------------------------------------------------**/
struct identity_initializer {
  template <typename T>
  T get_identity(operators op) {
    switch (op) {
      case SUM:
        return corresponding_functor_t<SUM>::identity<T>();
      case MIN:
        return corresponding_functor_t<MIN>::identity<T>();
      case MAX:
        return corresponding_functor_t<MAX>::identity<T>();
      case COUNT:
        return corresponding_functor_t<COUNT>::identity<T>();
      default:
        CUDF_FAIL("Invalid aggregation operation.");
    }
  }

  template <typename T>
  void operator()(gdf_column const& col, operators op,
                  hipStream_t stream = 0) {
    T* typed_data = static_cast<T*>(col.data);
    thrust::fill(rmm::exec_policy(stream)->on(stream), typed_data,
                 typed_data + col.size, get_identity<T>(op));

    // For COUNT operator, initialize column's bitmask to be all valid
    if ((nullptr != col.valid) and (COUNT == op)) {
      CUDA_TRY(hipMemsetAsync(
          col.valid, 0xff,
          sizeof(gdf_valid_type) * gdf_valid_allocation_size(col.size),
          stream));
    }
  }
};

/**---------------------------------------------------------------------------*
 * @brief Initializes each column in a table with a corresponding identity value
 * of an aggregation operation.
 *
 * The `i`th column will be initialized with the identity value of the `i`th
 * aggregation operation.
 *
 * @note The validity bitmask (if not `nullptr`) for the column corresponding to
 * a COUNT operator will be initialized to all valid.
 *
 * @param table The table of columns to initialize.
 * @param operators The aggregation operations whose identity values will be
 *used to initialize the columns.
 *---------------------------------------------------------------------------**/
void initialize_with_identity(cudf::table const& table,
                              std::vector<operators> const& ops,
                              hipStream_t stream = 0) {
  // TODO: Initialize all the columns in a single kernel instead of invoking one
  // kernel per column
  for (gdf_size_type i = 0; i < table.num_columns(); ++i) {
    gdf_column const* col = table.get_column(i);
    cudf::type_dispatcher(col->dtype, identity_initializer{}, *col, ops[i]);
  }
}

/**---------------------------------------------------------------------------*
 * @brief Compacts any GDF_STRING_CATEGORY columns in the output keys or values.
 *
 * After the groupby operation, any GDF_STRING_CATEGORY column in either the
 * keys or values may reference only a subset of the strings in the original
 * input category. This function will create a new associated NVCategory object
 * for the output GDF_STRING_CATEGORY columns whose dictionary contains only the
 * strings referenced in the output result.
 *
 * @param[in] input_keys The set of input key columns
 * @param[in/out] output_keys The set of output key columns
 * @param[in] input_values The set of input value columns
 * @param[in/out] output_values The set of output value columns
 *---------------------------------------------------------------------------**/
void update_nvcategories(table const& input_keys, table& output_keys,
                         table const& input_values, table& output_values) {
  nvcategory_gather_table(input_keys, output_keys);
  nvcategory_gather_table(input_values, output_values);
}

template <bool keys_have_nulls, bool values_have_nulls>
auto build_aggregation_map(table const& input_keys, table const& input_values,
                           device_table const& d_input_keys,
                           device_table const& d_input_values,
                           std::vector<operators> const& ops, Options options,
                           hipStream_t stream) {
  gdf_size_type constexpr unused_key{std::numeric_limits<gdf_size_type>::max()};
  gdf_size_type constexpr unused_value{
      std::numeric_limits<gdf_size_type>::max()};
  CUDF_EXPECTS(input_keys.num_rows() < unused_key,
               "Groupby input size too large.");

  // The exact output size is unknown a priori, therefore, use the input size as
  // an upper bound
  gdf_size_type const output_size_estimate{input_keys.num_rows()};

  cudf::table sparse_output_values{
      output_size_estimate, target_dtypes(column_dtypes(input_values), ops),
      values_have_nulls, false, stream};

  initialize_with_identity(sparse_output_values, ops, stream);

  auto d_sparse_output_values =
      device_table::create(sparse_output_values, stream);
  rmm::device_vector<operators> d_ops(ops);

  // If we ignore null keys, then nulls are not equivalent
  bool const null_keys_are_equal{not options.ignore_null_keys};
  bool const skip_key_rows_with_nulls{keys_have_nulls and
                                      not null_keys_are_equal};

  row_hasher<keys_have_nulls> hasher{d_input_keys};
  row_equality_comparator<keys_have_nulls> rows_equal{
      d_input_keys, d_input_keys, null_keys_are_equal};

  using map_type =
      concurrent_unordered_map<gdf_size_type, gdf_size_type, decltype(hasher),
                               decltype(rows_equal)>;

  auto map =
      std::make_unique<map_type>(compute_hash_table_size(input_keys.num_rows()),
                                 unused_key, unused_value, hasher, rows_equal);

  cudf::util::cuda::grid_config_1d grid_params{input_keys.num_rows(), 256};

  if (skip_key_rows_with_nulls) {
    auto row_bitmask{cudf::row_bitmask(input_keys, stream)};
    build_aggregation_map<true, values_have_nulls>
        <<<grid_params.num_blocks, grid_params.num_threads_per_block, 0,
           stream>>>(map.get(), d_input_keys, d_input_values,
                     *d_sparse_output_values, d_ops.data().get(),
                     row_bitmask.data().get());
  } else {
    build_aggregation_map<false, values_have_nulls>
        <<<grid_params.num_blocks, grid_params.num_threads_per_block, 0,
           stream>>>(map.get(), d_input_keys, d_input_values,
                     *d_sparse_output_values, d_ops.data().get(), nullptr);
  }
  CHECK_STREAM(stream);

  return std::make_pair(std::move(map), sparse_output_values);
}

template <bool keys_have_nulls, bool values_have_nulls, typename Map>
auto extract_results(table const& input_keys, table const& input_values,
                     device_table const& d_input_keys,
                     table const& sparse_output_values, Map* map,
                     hipStream_t stream) {
  cudf::table output_keys{cudf::allocate_like(input_keys, true, stream)};
  cudf::table output_values{
      cudf::allocate_like(sparse_output_values, true, stream)};

  auto d_sparse_output_values =
      device_table::create(sparse_output_values, stream);

  auto d_output_keys = device_table::create(output_keys, stream);
  auto d_output_values = device_table::create(output_values, stream);

  gdf_size_type* d_result_size{nullptr};
  RMM_TRY(RMM_ALLOC(&d_result_size, sizeof(gdf_size_type), stream));
  CUDA_TRY(hipMemsetAsync(d_result_size, 0, sizeof(gdf_size_type), stream));

  cudf::util::cuda::grid_config_1d grid_params{input_keys.num_rows(), 256};

  extract_groupby_result<keys_have_nulls, values_have_nulls>
      <<<grid_params.num_blocks, grid_params.num_threads_per_block, 0,
         stream>>>(map, d_input_keys, *d_output_keys, *d_sparse_output_values,
                   *d_output_values, d_result_size);

  CHECK_STREAM(stream);

  gdf_size_type result_size{-1};
  CUDA_TRY(hipMemcpyAsync(&result_size, d_result_size, sizeof(gdf_size_type),
                           hipMemcpyDeviceToHost, stream));

  // Update size and null count of output columns
  auto update_column = [result_size](gdf_column* col) {
    col->size = result_size;
    set_null_count(*col);
    return col;
  };

  std::transform(output_keys.begin(), output_keys.end(), output_keys.begin(),
                 update_column);
  std::transform(output_values.begin(), output_values.end(),
                 output_values.begin(), update_column);

  return std::make_pair(output_keys, output_values);
}

/**---------------------------------------------------------------------------*
 * @brief Converts a set of "compound" aggregation requests into a set of
 *"simple" aggregation requests that can be used to satisfy the compound
 *request.
 *
 * An "aggregation request" is the combination of an input `gdf_column`
 * an aggregation operation to perform on that column.
 *
 * A "compound" aggregation request is a requested aggregation operation that
 * can only be satisfied by first computing 1 or more "simple" aggregation
 * requests, and then transforming the result of the simple aggregation request
 * into the requested compound aggregation.
 *
 * For example, `AVG` is a "compound" aggregation. The request to compute an AVG
 * on a column can be satisfied via the `COUNT` and `SUM` "simple" aggregation
 * operations.
 *
 * @param compound_requests The set of compound aggregation requests
 * @return The set of corresponding simple aggregation requests that can be used
 * to satisfy the compound requests
 *---------------------------------------------------------------------------**/
std::vector<AggRequestType> compound_to_simple(
    std::vector<AggRequestType> const& compound_requests) {
  // Contructs a mapping of every value column to the minimal set of simple
  // ops to be performed on that column
  std::unordered_map<gdf_column*, std::set<operators>> columns_to_ops;
  std::for_each(
      compound_requests.begin(), compound_requests.end(),
      [&columns_to_ops](std::pair<gdf_column const*, operators> pair) {
        gdf_column* col = const_cast<gdf_column*>(pair.first);
        auto op = pair.second;
        // AVG requires computing a COUNT and SUM aggregation and then doing
        // elementwise division
        if (op == AVG) {
          columns_to_ops[col].insert(COUNT);
          columns_to_ops[col].insert(SUM);
        } else {
          columns_to_ops[col].insert(op);
        }
      });

  // Create minimal set of columns and simple operators
  std::vector<std::pair<gdf_column*, operators>> simple_requests;
  for (auto& p : columns_to_ops) {
    auto col = p.first;
    std::set<operators>& ops = p.second;
    while (not ops.empty()) {
      simple_requests.emplace_back(col, *ops.begin());
      ops.erase(ops.begin());
    }
  }
  return simple_requests;
}

/**---------------------------------------------------------------------------*
 * @brief Computes the `AVG` aggregation of a column by doing element-wise
 * division of the corresponding `SUM` and `COUNT` aggregation columns.
 *
 * @param sum The result of a `SUM` aggregation request
 * @param count The result of a `COUNT` aggregation request
 * @param stream Stream on which to perform operation
 * @return gdf_column* New column containing the result of elementwise division
 * of the sum and count columns
 *---------------------------------------------------------------------------**/
gdf_column* compute_average(gdf_column sum, gdf_column count,
                            hipStream_t stream) {
  CUDF_EXPECTS(sum.size == count.size,
               "Size mismatch between sum and count columns.");
  gdf_column* avg = new gdf_column{};
  avg->dtype = GDF_FLOAT64;
  RMM_TRY(RMM_ALLOC(&avg->data, sizeof(double) * sum.size, stream));
  if (cudf::has_nulls(sum) or cudf::has_nulls(count)) {
    RMM_TRY(RMM_ALLOC(
        &avg->valid,
        sizeof(gdf_size_type) * gdf_valid_allocation_size(sum.size), stream));
  }
  cudf::binary_operation(avg, &sum, &count, GDF_DIV);
  return avg;
}

/**---------------------------------------------------------------------------*
 * @brief Computes the results of a set of aggregation requests from a set of
 * computed simple requests.
 *
 * Given a set of pre-computed results for simple aggregation requests, computes
 * the results of a set of (potentially compound) requests. If the simple
 * aggregation request neccessary to compute the original request is not
 *present, an exception is thrown.
 *
 * @param original_requests[in] The original set of potentially compound
 * aggregation requests
 * @param simple_requests[in] Set of simple requests generated from the original
 * requests
 * @param simple_outputs[in] Set of output aggregation columns corresponding to
 *the simple requests
 * @param stream[in] CUDA stream on which to execute
 * @return table Set of columns satisfying each of the original requests
 *---------------------------------------------------------------------------**/
table compute_original_requests(
    std::vector<AggRequestType> const& original_requests,
    std::vector<AggRequestType> const& simple_requests, table simple_outputs,
    hipStream_t stream) {
  // Maps the requested simple aggregation to the resulting output column
  std::map<AggRequestType, gdf_column*> simple_requests_to_outputs;

  for (std::size_t i = 0; i < simple_requests.size(); ++i) {
    simple_requests_to_outputs[simple_requests[i]] =
        simple_outputs.get_column(i);
  }

  std::vector<gdf_column*> final_value_columns;

  // Iterate requests. For any compound request, compute the compound result
  // from the corresponding simple requests
  for (auto const& req : original_requests) {
    if (req.second == AVG) {
      auto found = simple_requests_to_outputs.find({req.first, SUM});
      CUDF_EXPECTS(found != simple_requests_to_outputs.end(),
                   "SUM request missing.");
      gdf_column* sum = found->second;

      found = simple_requests_to_outputs.find({req.first, COUNT});
      CUDF_EXPECTS(found != simple_requests_to_outputs.end(),
                   "COUNT request missing.");
      gdf_column* count = found->second;

      final_value_columns.push_back(compute_average(*sum, *count, stream));
    } else {
      // For non-compound requests, append the result to the final output
      // and remove it from the map
      auto found = simple_requests_to_outputs.find(req);
      CUDF_EXPECTS(found != simple_requests_to_outputs.end(),
                   "Aggregation missing!");
      final_value_columns.push_back(found->second);
      simple_requests_to_outputs.erase(req);
    }
  }

  // Any remaining columns in the `simple_outputs` are intermediary columns used
  // to satisfy a compound request that should be deleted.
  for (auto& p : simple_requests_to_outputs) {
    gdf_column_free(p.second);
    delete p.second;
  }

  return cudf::table{final_value_columns};
}

/**---------------------------------------------------------------------------*
 * @brief Computes the groupby operation for a set of keys, values, and
 * operators using a hash-based implementation.
 *
 * The algorithm has two primary steps:
 * 1.) Build a hash map
 * 2.) Extract the non-empty entries from the hash table
 *
 * 1.) The hash map is built by inserting every row `i` from the `keys` and
 * `values` tables as a single (key,value) pair. When the pair is inserted, if
 * the key was not already present in the map, then the corresponding value is
 * simply copied to the output. If the key was already present in the map,
 * then the inserted `values` row is aggregated with the existing row. This
 * aggregation is done for every element `j` in the row by applying aggregation
 * operation `j` between the new and existing element.
 *
 * This process yields a hash map and table holding the resulting aggregation
 * rows. The aggregation output table is sparse, i.e., not every row is
 * populated. This is because the size of the output is not known a priori, and
 * so the output aggregation table is allocated to be as large as the input (the
 * upper bound of the output size).
 *
 * 2.) The final result is materialized by extracting the non-empty keys from
 * the hash map and the non-empty rows from the sparse output aggregation table.
 * Every non-empty key and value row is appended to the output key and value
 * tables.
 *
 * @tparam keys_have_nulls Indicates keys have one or more null values
 * @tparam values_have_nulls Indicates values have one or more null values
 * @param keys Table whose rows are used as keys of the groupby
 * @param values Table whose rows are aggregated in the groupby
 * @param ops Set of aggregation operations to perform for each element in a row
 * in the values table
 * @param options Options to control behavior of the groupby operation
 * @param stream CUDA stream on which all memory allocations and kernels will be
 * executed
 * @return A pair of the output keys table and output values table
 *---------------------------------------------------------------------------**/
template <bool keys_have_nulls, bool values_have_nulls>
auto compute_hash_groupby(cudf::table const& keys, cudf::table const& values,
                          std::vector<operators> const& ops, Options options,
                          hipStream_t stream) {
  CUDF_EXPECTS(values.num_columns() == static_cast<gdf_size_type>(ops.size()),
               "Size mismatch between number of value columns and number of "
               "aggregations.");

  // An "aggregation request" is the combination of a `gdf_column*` to a column
  // of values, and an aggregation operation enum indicating the aggregation
  // requested to be performed on the column
  std::vector<AggRequestType> original_requests(values.num_columns());
  std::transform(values.begin(), values.end(), ops.begin(),
                 original_requests.begin(),
                 [](gdf_column const* col, operators op) {
                   return std::make_pair(const_cast<gdf_column*>(col), op);
                 });

  // Some aggregations are "compound", meaning they need be satisfied via the
  // composition of 1 or more "simple" aggregation requests. For example, AVG is
  // satisfied via the division of the SUM by the COUNT aggregation. We
  // translate these compound requests into simple requests, and compute the
  // groupby operation for these simple requests. Later, we translate the simple
  // requests back to compound request results.
  std::vector<AggRequestType> simple_requests =
      compound_to_simple(original_requests);

  std::vector<gdf_column*> simple_values_columns;
  std::vector<operators> simple_operators;
  for (auto const& p : simple_requests) {
    simple_values_columns.push_back(const_cast<gdf_column*>(p.first));
    simple_operators.push_back(p.second);
  }

  cudf::table simple_values_table{simple_values_columns};

  auto const d_input_keys = device_table::create(keys);
  auto const d_input_values = device_table::create(simple_values_table);

  auto result = build_aggregation_map<keys_have_nulls, values_have_nulls>(
      keys, values, *d_input_keys, *d_input_values, simple_operators, options,
      stream);

  auto const map{std::move(result.first)};
  cudf::table const sparse_output_values{result.second};

  cudf::table output_keys;
  cudf::table simple_output_values;
  std::tie(output_keys, simple_output_values) =
      extract_results<keys_have_nulls, values_have_nulls>(
          keys, values, *d_input_keys, sparse_output_values, map.get(), stream);

  // If any of the original requests were compound, compute them from the
  // results of simple aggregation requests
  cudf::table final_output_values = compute_original_requests(
      original_requests, simple_requests, simple_output_values, stream);

  return std::make_pair(output_keys, final_output_values);
}

/**---------------------------------------------------------------------------*
 * @brief Returns appropriate callable instantiation of `compute_hash_groupby`
 * based on presence of null values in keys and values.
 *
 * @param keys The groupby key columns
 * @param values The groupby value columns
 * @return Instantiated callable of compute_hash_groupby
 *---------------------------------------------------------------------------**/
auto groupby_null_specialization(table const& keys, table const& values) {
  if (cudf::has_nulls(keys)) {
    if (cudf::has_nulls(values)) {
      return compute_hash_groupby<true, true>;
    } else {
      return compute_hash_groupby<true, false>;
    }
  } else {
    if (cudf::has_nulls(values)) {
      return compute_hash_groupby<false, true>;
    } else {
      return compute_hash_groupby<false, false>;
    }
  }
}

}  // namespace
namespace detail {

std::pair<cudf::table, cudf::table> groupby(cudf::table const& keys,
                                            cudf::table const& values,
                                            std::vector<operators> const& ops,
                                            Options options,
                                            hipStream_t stream) {
  // TODO Handle Empty inputs
  CUDF_EXPECTS(keys.num_rows() == values.num_rows(),
               "Size mismatch between number of rows in keys and values.");

  verify_operators(values, ops);

  auto compute_groupby = groupby_null_specialization(keys, values);

  cudf::table output_keys;
  cudf::table output_values;
  std::tie(output_keys, output_values) =
      compute_groupby(keys, values, ops, options, stream);

  update_nvcategories(keys, output_keys, values, output_values);

  return std::make_pair(output_keys, output_values);
}
}  // namespace detail

std::pair<cudf::table, cudf::table> groupby(cudf::table const& keys,
                                            cudf::table const& values,
                                            std::vector<operators> const& ops,
                                            Options options) {
  return detail::groupby(keys, values, ops, options);
}
}  // namespace hash
}  // namespace groupby
}  // namespace cudf

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-20, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <groupby/common/utils.hpp>
#include <groupby/hash/groupby_kernels.cuh>

#include <hash/concurrent_unordered_map.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/groupby.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/row_operators.cuh>
#include <cudf/types.hpp>
#include <cudf/aggregation.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/detail/aggregation/result_cache.hpp>
#include <cudf/utilities/traits.hpp>

#include <memory>
#include <utility>

namespace cudf {
namespace experimental {
namespace groupby {
namespace detail {
namespace hash {
namespace {
// This is a temporary fix due to compiler bug and we can resort back to
// constexpr once cuda 10.2 becomes RAPIDS's minimum compiler version
#if 0
/**
 * @brief List of aggregation operations that can be computed with a hash-based
 * implementation.
 */
constexpr std::array<aggregation::Kind, 5> hash_aggregations{
    aggregation::SUM, aggregation::MIN, aggregation::MAX, aggregation::COUNT,
    aggregation::ARGMIN, aggregation::ARGMAX};

template <class T, size_t N>
constexpr bool array_contains(std::array<T, N> const& haystack, T needle) {
  for (auto i = 0u; i < N; ++i) {
    if (haystack[i] == needle) return true;
  }
  return false;
}
#endif

/**
 * @brief Indicates whether the specified aggregation operation can be computed
 * with a hash-based implementation.
 *
 * @param t The aggregation operation to verify
 * @return true `t` is valid for a hash based groupby
 * @return false `t` is invalid for a hash based groupby
 */
bool constexpr is_hash_aggregation(aggregation::Kind t) {
  // this is a temporary fix due to compiler bug and we can resort back to
  // constexpr once cuda 10.2 becomes RAPIDS's minimum compiler version
  // return array_contains(hash_aggregations, t);
  return (t == aggregation::SUM) or (t == aggregation::MIN) or
         (t == aggregation::MAX) or (t == aggregation::COUNT) or
         (t == aggregation::ARGMIN) or (t == aggregation::ARGMAX);
}

// flatten aggs to filter in single pass aggs 
std::tuple<table_view, std::vector<aggregation::Kind>, std::vector<size_t> >
flatten_single_pass_aggs(std::vector<aggregation_request> const& requests) {
  std::vector<column_view> columns;
  std::vector<aggregation::Kind> agg_kinds;
  std::vector<size_t> col_ids;

  for (size_t i = 0; i < requests.size(); i++) {
    auto const& request = requests[i];
    auto const& agg_v = request.aggregations;

    auto insert_agg = [&agg_kinds, &columns, &col_ids, &request, i]
    (aggregation::Kind k) {
      agg_kinds.push_back(k);
      columns.push_back(request.values);
      col_ids.push_back(i);
    };

    std::for_each(agg_v.begin(), agg_v.end(),
      [&columns, &agg_kinds, &request, &col_ids, insert_agg] 
      (std::unique_ptr<aggregation> const& agg) {

        if (is_hash_aggregation(agg->kind)) {
          if (is_fixed_width(request.values.type())) {
            insert_agg(agg->kind);
          } else if (request.values.type().id() == type_id::STRING) {
            // For string type, only MIN and MAX are supported
            if (agg->kind == aggregation::MIN) {
              insert_agg(aggregation::ARGMIN);
            } else if (agg->kind == aggregation::MAX) {
              insert_agg(aggregation::ARGMAX);
            }
          }
        }
      });
  }
  return std::make_tuple(table_view(columns), 
                         std::move(agg_kinds), std::move(col_ids));
}

/**
 * @brief Gather sparse results into dense using @p gather_map and add to 
 * @p dense_cache
 */
void sparse_to_dense_results(
    std::vector<aggregation_request> const& requests,
    experimental::detail::result_cache const& sparse_results,
    experimental::detail::result_cache& dense_results,
    rmm::device_vector<size_type> const& gather_map,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
{
  for (size_t i = 0; i < requests.size(); i++) {
    auto const& agg_v = requests[i].aggregations;
    auto const& col = requests[i].values;

    // Given an aggregation, this will get the result from sparse_results and 
    // convert and return dense, compacted result
    auto to_dense_agg_result =
    [&sparse_results, &gather_map, i, mr, stream]
    (auto const& agg) {
      auto s = sparse_results.get_result(i, agg);
      auto dense_result_table = 
        experimental::detail::gather(
          table_view({s}),
          gather_map.begin(),
          gather_map.end(),
          false, false, false, mr, stream);
      auto dense_result = std::move(dense_result_table->release()[0]);
      return dense_result;
    };

    // Enables conversion of ARGMIN/ARGMAX into MIN/MAX
    auto transformed_result =
    [&col, to_dense_agg_result, mr, stream]
    (auto const& agg_kind) {
      auto tranformed_agg = std::make_unique<aggregation>(agg_kind);
      auto argmax_result = to_dense_agg_result(tranformed_agg);
      auto transformed_result = experimental::detail::gather(
        table_view({col}), *argmax_result, false, false, false, mr, stream);
      return std::move(transformed_result->release()[0]);
    };

    std::for_each(agg_v.begin(), agg_v.end(),
      [&sparse_results, &dense_results, to_dense_agg_result, transformed_result,
       &col, i]
      (auto const& agg) {
        if (col.type().id() == type_id::STRING) {
          if (agg->kind == aggregation::MAX) {
            dense_results.add_result(i, agg,
              transformed_result(aggregation::ARGMAX));
          }
          else if (agg->kind == aggregation::MIN) {
            dense_results.add_result(i, agg,
              transformed_result(aggregation::ARGMIN));
          }
        }
        else if (sparse_results.has_result(i, agg)) {
          dense_results.add_result(i, agg, to_dense_agg_result(agg));
        }
      });
  }
}

/**
 * @brief Construct hash map that uses row comparator and row hasher on 
 * @p d_keys table and stores indices
 */
template <bool keys_have_nulls>
auto create_hash_map(table_device_view const& d_keys, bool ignore_null_keys,
                     hipStream_t stream = 0)
{
  size_type constexpr unused_key{std::numeric_limits<size_type>::max()};
  size_type constexpr unused_value{std::numeric_limits<size_type>::max()};

  using map_type =
      concurrent_unordered_map<size_type, size_type,
                              row_hasher<default_hash, keys_have_nulls>,
                              row_equality_comparator<keys_have_nulls>>;

  using allocator_type = typename map_type::allocator_type;

  bool const null_keys_are_equal{not ignore_null_keys};

  row_hasher<default_hash, keys_have_nulls> hasher{d_keys};
  row_equality_comparator<keys_have_nulls> rows_equal{
      d_keys, d_keys, null_keys_are_equal};

  return map_type::create(compute_hash_table_size(d_keys.num_rows()),
                            unused_key, unused_value, hasher, rows_equal,
                            allocator_type(), stream);
}

/**
 * @brief Computes all aggregations from @p requests that require a single pass
 * over the data and stores the results in @p sparse_results
 */
template <bool keys_have_nulls, typename Map>
void compute_single_pass_aggs(table_view const& keys,
                              std::vector<aggregation_request> const& requests,
                              experimental::detail::result_cache& sparse_results,
                              Map& map, bool ignore_null_keys,
                              hipStream_t stream)
{
  // flatten the aggs to a table that can be operated on by aggregate_row
  table_view flattened_values;
  std::vector<aggregation::Kind> aggs;
  std::vector<size_t> col_ids;
  std::tie(flattened_values, aggs, col_ids) = flatten_single_pass_aggs(requests);

  // make table that will hold sparse results
  std::vector<std::unique_ptr<column>> sparse_columns;
  for (size_t i = 0; i < aggs.size(); i++) {
    auto const& col = flattened_values.column(i);
    bool nullable = (aggs[i] == aggregation::COUNT) ? false : col.has_nulls();
    auto mask_state = (nullable) ? ALL_NULL : UNALLOCATED;
    
    sparse_columns.emplace_back(make_fixed_width_column(
      experimental::detail::target_type(col.type(), aggs[i]),
      col.size(), mask_state, stream));
  }
  table sparse_table(std::move(sparse_columns));
  mutable_table_view table_view = sparse_table.mutable_view();
  experimental::detail::initialize_with_identity(table_view, aggs, stream);

  // prepare to launch kernel to do the actual aggregation
  auto d_sparse_table = mutable_table_device_view::create(sparse_table);
  auto d_values = table_device_view::create(flattened_values);
  rmm::device_vector<aggregation::Kind> d_aggs(aggs);

  bool skip_key_rows_with_nulls = keys_have_nulls and ignore_null_keys;

  experimental::detail::grid_1d grid(keys.num_rows(), 256);
  if (skip_key_rows_with_nulls) {
    auto row_bitmask{bitmask_and(keys, rmm::mr::get_default_resource(), stream)};
    hash::compute_single_pass_aggs<true>
      <<<grid.num_blocks, grid.num_threads_per_block, 0, stream>>>(
        map, keys.num_rows(), *d_values, *d_sparse_table, d_aggs.data().get(),
        static_cast<bitmask_type*>(row_bitmask.data()));
  } else {
    hash::compute_single_pass_aggs<false>
      <<<grid.num_blocks, grid.num_threads_per_block, 0, stream>>>(
        map, keys.num_rows(), *d_values, *d_sparse_table, d_aggs.data().get(),
        nullptr);
  }

  // Add results back to sparse_results cache
  auto sparse_result_cols = sparse_table.release();
  for (size_t i = 0; i < aggs.size(); i++) {
    sparse_results.add_result(col_ids[i],
                              std::make_unique<aggregation>(aggs[i]),
                              std::move(sparse_result_cols[i]));
  }
}

template <bool keys_have_nulls>
auto groupby_null_templated(
    table_view const& keys, std::vector<aggregation_request> const& requests,
    experimental::detail::result_cache& cache,
    bool ignore_null_keys, hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
{
  auto d_keys = table_device_view::create(keys);
  auto map = create_hash_map<keys_have_nulls>(*d_keys, ignore_null_keys, stream);

  // Cache of sparse results where the location of aggregate value in each
  // column is indexed by the hash map
  experimental::detail::result_cache sparse_results(requests.size());

  // Compute all single pass aggs first
  compute_single_pass_aggs<keys_have_nulls>(
    keys, requests, sparse_results, *map, ignore_null_keys, stream);

  // Now continue with remaining multi-pass aggs
  // <placeholder>

  // Extract the populated indices from the hash map and create a gather map.
  // Gathering using this map from sparse results will give dense results.
  rmm::device_vector<size_type> gather_map(keys.num_rows());
  rmm::device_scalar<size_type> num_groups(0);
  experimental::detail::grid_1d grid(keys.num_rows(), 256);
  extract_gather_map<<<grid.num_blocks, grid.num_threads_per_block, 0, stream>>>
    (*map, gather_map.data().get(), num_groups.data());
  gather_map.resize(num_groups.value());

  // Compact all results from sparse_results and insert into cache
  sparse_to_dense_results(requests, sparse_results, cache, gather_map, stream, mr);

  // Extract unique keys and return
  auto unique_keys = experimental::detail::gather(
    keys, gather_map.begin(), gather_map.end(),
    false, false, false, mr, stream);
  return unique_keys;
}

}  // namespace

/**
 * @brief Indicates if a set of aggregation requests can be satisfied with a
 * hash-based groupby implementation.
 *
 * @param keys The table of keys
 * @param requests The set of columns to aggregate and the aggregations to
 * perform
 * @return true A hash-based groupby should be used
 * @return false A hash-based groupby should not be used
 */
bool can_use_hash_groupby(table_view const& keys,
                      std::vector<aggregation_request> const& requests) {
  return std::all_of(
      requests.begin(), requests.end(), [](aggregation_request const& r) {
        return std::all_of(
            r.aggregations.begin(), r.aggregations.end(),
            [](auto const& a) { return is_hash_aggregation(a->kind); });
      });
}

// Hash-based groupby
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby(
    table_view const& keys, std::vector<aggregation_request> const& requests,
    bool ignore_null_keys, hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
{
  experimental::detail::result_cache cache(requests.size());

  std::unique_ptr<table> unique_keys;
  if (has_nulls(keys)) {
    unique_keys = groupby_null_templated<true>(keys, requests, cache, 
                                               ignore_null_keys, stream, mr);
  } else {
    unique_keys = groupby_null_templated<false>(keys, requests, cache,
                                                ignore_null_keys, stream, mr);
  }

  return std::make_pair(std::move(unique_keys), extract_results(requests, cache));  
}
}  // namespace hash
}  // namespace detail
}  // namespace groupby
}  // namespace experimental
}  // namespace cudf

/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/types.hpp>

#include <thrust/gather.h>

namespace cudf {
namespace experimental {
namespace groupby {
namespace detail {

std::unique_ptr<column>
group_nth_element(column_view const &values,
                  column_view const &group_sizes,
                  rmm::device_vector<size_type> const &group_labels,
                  rmm::device_vector<size_type> const &group_offsets,
                  size_type num_groups, size_type n,
                  include_nulls _include_nulls,
                  rmm::mr::device_memory_resource *mr, hipStream_t stream) {

  CUDF_EXPECTS(static_cast<size_t>(values.size()) == group_labels.size(),
               "Size of values column should be same as that of group labels");

  if (num_groups == 0) {
    return experimental::empty_like(values);
  }

  auto output = make_numeric_column(
      data_type{experimental::type_to_id<size_type>()}, num_groups,
      mask_state::UNALLOCATED, stream);
  mutable_column_view output_view = output->mutable_view();
  auto exec = rmm::exec_policy(stream)->on(stream);

  // include nulls (equivalent to pandas nth(dropna=None) but return nulls for n
  if (_include_nulls == include_nulls::YES || !values.has_nulls()) {
    // Returns index of nth value.
    thrust::transform(exec, group_sizes.begin<size_type>(),
                      group_sizes.end<size_type>(),
                      group_offsets.begin(), output_view.begin<size_type>(),
                      [n, out_of_bounds = values.size()] __device__(
                          auto group_size, auto group_offset) {
                        bool nth_within_group =
                            (n < 0) ? group_size >= (-n) : group_size > n;
                        if (nth_within_group)
                          return group_offset + ((n < 0) ? group_size + n : n);
                        else
                          return out_of_bounds;
                      });
  } else { // skip nulls (equivalent to pandas nth(dropna='any'))
    // Returns index of nth value.
    thrust::fill(exec, output->mutable_view().begin<size_type>(),
                 output->mutable_view().end<size_type>(),
                 values.size()); // for out of bounds
    auto values_view = column_device_view::create(values);
    auto bitmask_iterator = thrust::make_transform_iterator(
        experimental::detail::make_validity_iterator(*values_view),
        [] __device__(auto b) { return static_cast<size_type>(b); });
    rmm::device_vector<size_type> intra_group_index(values.size());
    // intra group index for valids only.
    thrust::exclusive_scan_by_key(exec, group_labels.begin(),
                                  group_labels.end(), bitmask_iterator,
                                  intra_group_index.begin());
    // gather the valid index == n
    thrust::scatter_if(
        exec, thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(0) + values.size(),
        group_labels.begin(),                         // map
        thrust::make_counting_iterator<size_type>(0), // stencil
        output->mutable_view().begin<size_type>(),
        [n, bitmask_iterator,
         intra_group_index =
             intra_group_index.begin()] __device__(auto i) -> bool {
          return (bitmask_iterator[i] && intra_group_index[i] == n);
        });
  }
  bool nullify_out_of_bounds = thrust::transform_reduce(
      exec, group_sizes.begin<size_type>(),
      group_sizes.end<size_type>(),
      [n] __device__(const size_type group_size) {
        bool nth_within_group = (n < 0) ? group_size >= (-n) : group_size > n;
        return !nth_within_group;
      },
      false, thrust::logical_or<bool>{});
  auto output_table =
      experimental::detail::gather(table_view{{values}}, output->view(), false,
                                   nullify_out_of_bounds, false, mr, stream);
  return std::make_unique<column>(std::move(output_table->get_column(0)));
}
} // namespace detail
} // namespace groupby
} // namespace experimental
} // namespace cudf

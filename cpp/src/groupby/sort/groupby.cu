/*
 * Copyright 2019 BlazingDB, Inc.
 *     Copyright 2019 Alexander Ocsa <alexander@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cassert>
#include <thrust/fill.h>
#include <tuple>


#include <cudf/cudf.h>
#include <bitmask/legacy/bit_mask.cuh>
#include <cudf/copying.hpp>
#include <cudf/groupby.hpp>
#include <cudf/legacy/bitmask.hpp>
#include <cudf/legacy/table.hpp>
#include <cudf/utilities/legacy/nvcategory_util.hpp>
#include <table/legacy/device_table.cuh>
#include <table/legacy/device_table_row_operators.cuh>
#include <utilities/column_utils.hpp>
#include <utilities/cuda_utils.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
 
#include "../common/util.hpp"
#include "groupby.hpp"
#include "groupby_kernels.cuh"  

using namespace cudf::groupby::common;

namespace cudf {
namespace groupby {
namespace sort {

namespace {   

cudf::table compose_inputs(cudf::table input_table, gdf_column* col) {
  std::vector<gdf_column*> output(input_table.num_columns());  
  std::transform(input_table.begin(), input_table.end(), output.begin(), [](const gdf_column *item){
    return (gdf_column *)item;
  }); 
  output.push_back(col);

  gdf_column **group_by_input_key = output.data();
  return cudf::table{group_by_input_key, input_table.num_columns() + 1};
}

cudf::table compose_output_keys(cudf::table input_table) {
  std::vector<gdf_column*> output(input_table.num_columns() - 1);  
  std::transform(input_table.begin(), input_table.end() - 1, output.begin(), [](const gdf_column *item){
    return (gdf_column *)item;
  }); 
  return cudf::table {output};
}

rmm::device_vector<gdf_size_type> get_last_column (cudf::table current_table) {
  auto num_column = current_table.num_columns();
  gdf_column * sorted_column = current_table.get_column(num_column - 1);
  rmm::device_vector<gdf_size_type> returned_vector(current_table.num_rows());
  hipMemcpy(returned_vector.data().get(), sorted_column->data, sorted_column->size * sizeof(gdf_size_type), hipMemcpyDeviceToDevice); 
  return returned_vector;
}

std::pair<cudf::table, gdf_column> compute_sort_groupby_wo_agg(cudf::table const& input_keys, 
                            Options options,
                            rmm::device_vector<gdf_size_type> &d_sorted_indices,
                            hipStream_t stream) {
  gdf_context context;
  auto ignore_null_keys = options.ignore_null_keys;
  if (not ignore_null_keys) { // SQL
    context.flag_groupby_include_nulls = true;
    context.flag_null_sort_behavior = GDF_NULL_AS_LARGEST;
  } else { // PANDAS
    context.flag_groupby_include_nulls = false;
    context.flag_null_sort_behavior = GDF_NULL_AS_LARGEST;
  }

  std::vector<int> groupby_col_indices;
  for (gdf_size_type i = 0; i < input_keys.num_columns(); i++)
    groupby_col_indices.push_back(i);

  cudf::table sorted_keys_table;
  gdf_column group_indices_col;
  
  auto nrows = input_keys.num_rows();
  rmm::device_vector<gdf_size_type> d_seq_indices_values(nrows);
  thrust::sequence(d_seq_indices_values.begin(), d_seq_indices_values.end(), 0, 1);

  gdf_column seq_indices_col{};
  CUDF_TRY(gdf_column_view(&seq_indices_col,
                           (void *)(d_seq_indices_values.data().get()), nullptr,
                           nrows, GDF_INT32));

  auto input_table = compose_inputs(input_keys, &seq_indices_col);
  std::tie(sorted_keys_table,
                        group_indices_col) = gdf_group_by_without_aggregations(input_table,
                                                                          groupby_col_indices.size(),
                                                                          groupby_col_indices.data(),
                                                                          &context);
  cudf::table output_keys = compose_output_keys(sorted_keys_table);
  d_sorted_indices = get_last_column(sorted_keys_table); 
  return std::make_pair(output_keys, group_indices_col);
}

template <bool keys_have_nulls, bool values_have_nulls>
auto compute_sort_groupby(cudf::table const& input_keys, cudf::table const& input_values,
                          std::vector<operators> const& ops, Options options,
                          hipStream_t stream) {
  cudf::table sorted_keys_table;
  gdf_column group_indices_col;

  rmm::device_vector<gdf_size_type> d_sorted_indices;
  std::tie(sorted_keys_table,
                          group_indices_col) = compute_sort_groupby_wo_agg(input_keys, options, d_sorted_indices, stream);

  if (sorted_keys_table.num_rows() == 0) {
    return std::make_pair(
        cudf::empty_like(input_keys),
        cudf::table(0, target_dtypes(column_dtypes(input_values), ops), column_dtype_infos(input_values)));
  }
  cudf::table output_values{
      group_indices_col.size, target_dtypes(column_dtypes(input_values), ops),
      column_dtype_infos(input_values), values_have_nulls, false, stream};

  initialize_with_identity(output_values, ops, stream);

  auto d_input_keys = device_table::create(sorted_keys_table);
  auto d_input_values = device_table::create(input_values);
  auto d_output_values = device_table::create(output_values, stream);
  rmm::device_vector<operators> d_ops(ops);
 
  auto row_bitmask = cudf::row_bitmask(sorted_keys_table, stream);

  cudf::util::cuda::grid_config_1d grid_params{sorted_keys_table.num_rows(), 256};

  cudf::groupby::sort::aggregate_all_rows<keys_have_nulls, values_have_nulls><<<
      grid_params.num_blocks, grid_params.num_threads_per_block, 0, stream>>>(
      *d_input_keys, *d_input_values, *d_output_values, d_sorted_indices.data().get(), 
      (gdf_index_type *)group_indices_col.data, group_indices_col.size,
      d_ops.data().get(), row_bitmask.data().get());

  cudf::table destination_table(group_indices_col.size,
                                cudf::column_dtypes(sorted_keys_table),
                                cudf::column_dtype_infos(sorted_keys_table),
                                keys_have_nulls);
  
  cudf::gather(&sorted_keys_table, (gdf_index_type *)group_indices_col.data,
               &destination_table); 

  // TODO: destroy temporal tables, and temporal gdf_columns! 
  return std::make_pair(destination_table, output_values);
}

/**---------------------------------------------------------------------------*
 * @brief Returns appropriate callable instantiation of `compute_sort_groupby`
 * based on presence of null values in keys and values.
 *
 * @param keys The groupby key columns
 * @param values The groupby value columns
 * @return Instantiated callable of compute_sort_groupby
 *---------------------------------------------------------------------------**/
auto groupby_null_specialization(table const& keys, table const& values) {
  if (cudf::has_nulls(keys)) {
    if (cudf::has_nulls(values)) {
      return compute_sort_groupby<true, true>;
    } else {
      return compute_sort_groupby<true, false>;
    }
  } else {
    if (cudf::has_nulls(values)) {
      return compute_sort_groupby<false, true>;
    } else {
      return compute_sort_groupby<false, false>;
    }
  }
}
} // anonymous namespace

namespace detail {

std::pair<cudf::table, cudf::table> groupby(cudf::table const &keys,
                                            cudf::table const &values,
                                            std::vector<operators> const &ops,
                                            Options options,
                                            hipStream_t stream) {
  CUDF_EXPECTS(keys.num_rows() == values.num_rows(),
               "Size mismatch between number of rows in keys and values.");

  verify_operators(values, ops);

  // Empty inputs
  if (keys.num_rows() == 0) {
    return std::make_pair(
        cudf::empty_like(keys),
        cudf::table(0, target_dtypes(column_dtypes(values), ops), column_dtype_infos(values)));
  }

 auto compute_groupby = groupby_null_specialization(keys, values);

  cudf::table output_keys;
  cudf::table output_values;
  std::tie(output_keys, output_values) =
      compute_groupby(keys, values, ops, options, stream);

  update_nvcategories(keys, output_keys, values, output_values);
  return std::make_pair(output_keys, output_values);
}

} // namespace detail

std::pair<cudf::table, cudf::table> groupby(cudf::table const &keys,
                                            cudf::table const &values,
                                            std::vector<operators> const &ops,
                                            Options options) {
  return detail::groupby(keys, values, ops, options);
}

} // END: namespace sort
} // END: namespace groupby
} // END: namespace cudf

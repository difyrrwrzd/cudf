/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "sort_helper.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/row_operators.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/scatter.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/detail/copy.hpp>

#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/unique.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/constant_iterator.h>

#include <algorithm>
#include <tuple>
#include <numeric>


namespace {

/**
 * @brief Compares two `table` rows for equality as if the table were
 * ordered according to a specified permutation map.
 *
 */
template <bool nullable = true>
struct permuted_row_equality_comparator {
  cudf::experimental::row_equality_comparator<nullable> _comparator;
  cudf::size_type const *_map;

  /**
   * @brief Construct a permuted_row_equality_comparator.
   *
   * @param t The `table` whose rows will be compared
   * @param map The permutation map that specifies the effective ordering of
   *`t`. Must be the same size as `t.num_rows()`
   */
  permuted_row_equality_comparator(cudf::table_device_view const &t,
                                   cudf::size_type const *map)
      : _comparator(t, t, true), _map{map} {}

  /**
   * @brief Returns true if the two rows at the specified indices in the permuted
   * order are equivalent.
   *
   * For example, comparing rows `i` and `j` is
   * equivalent to comparing rows `map[i]` and `map[j]` in the original table.
   *
   * @param lhs The index of the first row
   * @param rhs The index of the second row
   * @returns if the two specified rows in the permuted order are equivalent
   */
  CUDA_DEVICE_CALLABLE
  bool operator()(cudf::size_type lhs, cudf::size_type rhs) {
    return _comparator(_map[lhs], _map[rhs]);
  }
};

} // namespace anonymous


namespace cudf {
namespace experimental {
namespace groupby {
namespace detail {
namespace sort {

size_type helper::num_keys() {
  if (_num_keys > -1)
    return _num_keys;

  if (not _include_nulls and has_nulls(_keys)) {
    // The number of rows w/o null values `n` is indicated by number of valid bits
    // in the row bitmask. When `include_nulls == false`, then only rows `[0, n)` 
    // in the sorted order are considered for grouping. 
    _num_keys = keys_bitmask_column().size() - keys_bitmask_column().null_count();
  } else {
    _num_keys = _keys.num_rows();
  }

  return _num_keys; 
}

column_view helper::key_sort_order(hipStream_t stream) {
  if (_key_sorted_order)
    return _key_sorted_order->view();

  if (_keys_pre_sorted) {
    _key_sorted_order = make_numeric_column(data_type(type_to_id<size_type>()),
                          _keys.num_rows(), mask_state::UNALLOCATED, stream);

    auto d_key_sorted_order = _key_sorted_order->mutable_view().data<size_type>();

    thrust::sequence(rmm::exec_policy(stream)->on(stream), 
                     d_key_sorted_order,
                     d_key_sorted_order + _key_sorted_order->size(), 0);

    return _key_sorted_order->view();
  }

  if (_include_nulls || !cudf::has_nulls(_keys)) {  // SQL style
    _key_sorted_order = cudf::experimental::detail::sorted_order(_keys, {},
      std::vector<null_order>(_keys.num_columns(), null_order::AFTER),
      rmm::mr::get_default_resource(), stream);
  } else {  // Pandas style
    // Temporarily prepend the keys table with a column that indicates the 
    // presence of a null value within a row. This allows moving all rows that 
    // contain a null value to the end of the sorted order. 

    auto augmented_keys = table_view({ 
      table_view( {keys_bitmask_column()} ),
      _keys });

    _key_sorted_order = cudf::experimental::detail::sorted_order(
      augmented_keys, {},
      std::vector<null_order>(_keys.num_columns(), null_order::AFTER),
      rmm::mr::get_default_resource(), stream);

    // All rows with one or more null values are at the end of the resulting sorted order.
  }

  return _key_sorted_order->view();
}

helper::index_vector const& helper::group_offsets(hipStream_t stream) {
  if (_group_offsets)
    return *_group_offsets;

  _group_offsets = std::make_unique<index_vector>(num_keys());

  auto device_input_table = table_device_view::create(_keys, stream);
  auto sorted_order = key_sort_order().data<size_type>();
  decltype(_group_offsets->begin()) result_end;
  auto exec = rmm::exec_policy(stream);

  if (has_nulls(_keys)) {
    result_end = thrust::unique_copy(exec->on(stream),
      thrust::make_counting_iterator<size_type>(0),
      thrust::make_counting_iterator<size_type>(num_keys()),
      _group_offsets->begin(),
      permuted_row_equality_comparator<true>(*device_input_table, sorted_order));
  } else {
    result_end = thrust::unique_copy(exec->on(stream), 
      thrust::make_counting_iterator<size_type>(0),
      thrust::make_counting_iterator<size_type>(num_keys()),
      _group_offsets->begin(),
      permuted_row_equality_comparator<false>(*device_input_table, sorted_order));
  }

  size_type num_groups = thrust::distance(_group_offsets->begin(), result_end);
  _group_offsets->resize(num_groups);

  return *_group_offsets;
}

helper::index_vector const& helper::group_labels(hipStream_t stream) {
  if (_group_labels)
    return *_group_labels;

  // Get group labels for future use in segmented sorting
  _group_labels = std::make_unique<index_vector>(num_keys());

  auto& group_labels = *_group_labels;
  auto exec = rmm::exec_policy(stream);
  thrust::scatter(exec->on(stream),
    thrust::make_constant_iterator(1, decltype(num_groups())(1)), 
    thrust::make_constant_iterator(1, num_groups()), 
    group_offsets().begin() + 1, 
    group_labels.begin());
 
  thrust::inclusive_scan(exec->on(stream),
                        group_labels.begin(),
                        group_labels.end(),
                        group_labels.begin());

  return group_labels;
}

column_view helper::unsorted_keys_labels(hipStream_t stream) {
  if (_unsorted_keys_labels)
    return _unsorted_keys_labels->view();

  column_ptr temp_labels = make_numeric_column(
                              data_type(type_to_id<size_type>()),
                              key_sort_order().size(),
                              mask_state::ALL_NULL, stream);
  
  auto group_labels_view = cudf::column_view(
                              data_type(type_to_id<size_type>()),
                              group_labels().size(),
                              group_labels().data().get());

  std::unique_ptr<table> t_unsorted_keys_labels = 
    cudf::experimental::detail::scatter(
      table_view({group_labels_view}), key_sort_order(), 
      table_view({temp_labels->view()}),
      false, rmm::mr::get_default_resource(), stream);

  _unsorted_keys_labels = std::move(t_unsorted_keys_labels->release()[0]);

  return _unsorted_keys_labels->view();
}

column_view helper::keys_bitmask_column(hipStream_t stream) {
  if (_keys_bitmask_column)
    return _keys_bitmask_column->view();

  // TODO (dm): port row_bitmask
  // _keys_bitmask_column = 
  //   ( new bitmask_vector(row_bitmask(_keys, stream)));

  return _keys_bitmask_column->view();
}
  
helper::index_vector helper::count_valids_in_groups(
  column const& grouped_values,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  // Get number of valid values in each group
  helper::index_vector val_group_sizes(num_groups());
  auto col_view = column_device_view::create(grouped_values.view());
  auto d_col_view = col_view.get();
  
  auto bitmask_iterator = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0), 
    [d_col_view] __device__ (size_type i) -> int { 
      return d_col_view->is_valid(i);
    });

  thrust::reduce_by_key(rmm::exec_policy(stream)->on(stream),
                        group_labels().begin(),
                        group_labels().end(),
                        bitmask_iterator,
                        thrust::make_discard_iterator(),
                        val_group_sizes.begin());

}

std::pair<helper::column_ptr, helper::index_vector >
helper::sorted_values_and_num_valids(column_view const& values,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  auto sorted_values = this->sorted_values(values, mr, stream);
  auto val_group_sizes = count_valids_in_groups(*sorted_values, mr, stream); 
  return std::make_pair(sorted_values, val_group_sizes);
}

helper::column_ptr helper::sorted_values(column_view const& values, 
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  column_ptr values_sort_order = cudf::experimental::detail::sorted_order(
    table_view({unsorted_keys_labels(), values}), {},
    std::vector<null_order>(2, null_order::AFTER), mr, stream);

  // Zero-copy slice this sort order so that its new size is num_keys()
  column_view gather_map = cudf::experimental::detail::slice(
    values_sort_order->view(), 0, num_keys() );

  auto sorted_values_table = cudf::experimental::detail::gather(
    table_view({values}), gather_map, false, false, false, mr, stream);

  return std::move(sorted_values_table->release()[0]);
}


std::unique_ptr<table> helper::unique_keys(
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  auto idx_data = key_sort_order().data<size_type>();
  auto transformed_group_ids = index_vector(num_groups());

  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    group_offsets().begin(), group_offsets().end(),
                    transformed_group_ids.begin(),
    [=] __device__ (size_type i) { return idx_data[i]; } );

  auto gather_map = cudf::column_view(data_type(type_to_id<size_type>()),
    num_groups(), transformed_group_ids.data().get());

  // TODO (dm): replace this with iterator based gather when it's made so that 
  //            gather_map is not required to be generated
  return cudf::experimental::detail::gather(_keys, gather_map,
                                            false, false, false, mr, stream);
}


}  // namespace sort
}  // namespace detail
}  // namespace groupby
}  // namespace experimental
}  // namespace cudf

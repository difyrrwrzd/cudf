#include "hip/hip_runtime.h"
#include <cudf/cudf.h>
#include <cudf/column/column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/copying.hpp>
#include <cudf/detail/utilities/cuda.cuh>

namespace cudf {

namespace experimental {

namespace detail {

namespace {

template <size_type block_size, typename T, bool has_validity>
__launch_bounds__(block_size)
__global__
void copy_in_place_kernel( column_device_view const in,
                           mutable_column_device_view out)
{
   const size_type tid = threadIdx.x + blockIdx.x * block_size;
   const int warp_id = tid / cudf::experimental::detail::warp_size;
   const size_type warps_per_grid = gridDim.x * block_size / cudf::experimental::detail::warp_size;      

   // begin/end indices for the column data
   size_type begin = 0;
   size_type end = in.size();
   // warp indices.  since 1 warp == 32 threads == sizeof(bit_mask_t) * 8,
   // each warp will process one (32 bit) of the validity mask via
   // __ballot_sync()
   size_type warp_begin = cudf::word_index(begin);
   size_type warp_end = cudf::word_index(end-1);      

   // lane id within the current warp   
   const int lane_id = threadIdx.x % cudf::experimental::detail::warp_size;
   
   // current warp.
   size_type warp_cur = warp_begin + warp_id;   
   size_type index = tid;
   while(warp_cur <= warp_end){
      bool in_range = (index >= begin && index < end);
            
      bool valid = true;
      if(has_validity){
         valid = in_range && in.is_valid(index);
      }
      if(in_range){
         out.element<T>(index) = in.element<T>(index);
      }
      
      // update validity      
      if(has_validity){
         // the final validity mask for this warp 
         int warp_mask = __ballot_sync(0xFFFF'FFFF, valid && in_range);
         // only one guy in the warp needs to update the mask and count
         if(lane_id == 0){            
            out.set_mask_word(warp_cur, warp_mask);            
         }
      }            

      // next grid
      warp_cur += warps_per_grid;
      index += block_size * gridDim.x;
   }      
}


static constexpr size_t split_align = 8;

template<typename T>
struct column_buf_size_functor_impl {
   void operator()(column_view const& c, size_t& running_data_size, size_t& running_validity_size)
   {
      running_data_size += cudf::util::div_rounding_up_safe(c.size() * sizeof(T), split_align) * split_align;      
      if(c.nullable()){
         running_validity_size += cudf::bitmask_allocation_size_bytes(c.size(), split_align);         
      }
   }
};

template<>
struct column_buf_size_functor_impl<string_view> {
   void operator()(column_view const& c, size_t& running_data_size, size_t& running_validity_size)
   {
      CUDF_FAIL("contiguous_split for strings not implemented yet");
   };
};

struct column_buf_size_functor {
   template<typename T>
   void operator()(column_view const& c, size_t& running_data_size, size_t& running_validity_size)
   {
      column_buf_size_functor_impl<T> sizer{};
      sizer(c, running_data_size, running_validity_size);
   }
};


template<typename T>
struct column_copy_functor_impl {
   void operator()(column_view const& in, char*& dst, std::vector<column_view>& out_cols)
   {      
      // there's some unnecessary recomputation of sizes happening here, but it really shouldn't affect much.
      size_t data_size = 0;
      size_t validity_size = 0;      
      column_buf_size_functor_impl<T>{}(in, data_size, validity_size);

      // outgoing pointers
      char* data = dst;
      bitmask_type* validity = validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + data_size);

      // increment working buffer
      dst += (data_size + validity_size);      

      // custom copy kernel (which should probably just be an in-place copy() function in cudf.
      cudf::size_type num_els = cudf::util::round_up_safe(in.size(), cudf::experimental::detail::warp_size);
      constexpr int block_size = 256;
      cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};      
      
      // so there's a significant performance issue that comes up. our incoming column_view objects
      // are the result of a slice.  because of this, they have an UNKNOWN_NULL_COUNT.  because of that,
      // calling column_device_view::create() will cause a recompute of the count, which ends up being
      // extremely slow because a.) the typical use case here will involve huge numbers of calls and
      // b.) the count recompute involves tons of device allocs and memcopies.
      //
      // so to get around this, I am manually constructing a fake-ish view here where the null
      // count is arbitrarily bashed to 0.            
      //
      column_view   in_wrapped{in.type(), in.size(), in.head<T>(), 
                               in.null_mask(), in.null_mask() == nullptr ? UNKNOWN_NULL_COUNT : 0,
                               in.offset() };
      mutable_column_view  mcv{in.type(), in.size(), data, 
                               validity, validity == nullptr ? UNKNOWN_NULL_COUNT : 0 };      
      if(in.nullable()){               
         copy_in_place_kernel<block_size, T, true><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped), 
                           *mutable_column_device_view::create(mcv));         
      } else {
         copy_in_place_kernel<block_size, T, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped), 
                           *mutable_column_device_view::create(mcv));
      }
      mcv.set_null_count(cudf::UNKNOWN_NULL_COUNT);

      out_cols.push_back(mcv);
   }
};

template<>
struct column_copy_functor_impl<string_view> {
   void operator()(column_view const& in, char*& dst, std::vector<column_view>& out_cols) 
   {
      CUDF_FAIL("contiguous_split for strings not implemented yet");
   };
};

struct column_copy_functor {
   template<typename T>   
   void operator()(column_view const& in, char*& dst, std::vector<column_view>& out_cols)
   {
      column_copy_functor_impl<T> fn{};      
      fn(in, dst, out_cols);
   }
};

contiguous_split_result alloc_and_copy(cudf::table_view const& t, rmm::mr::device_memory_resource* mr, hipStream_t stream)
{
   size_t data_size = 0;
   size_t validity_size = 0;      

   // compute sizes
   std::for_each(t.begin(), t.end(), [&data_size, &validity_size](cudf::column_view const& c){
      cudf::experimental::type_dispatcher(c.type(), column_buf_size_functor{}, c, data_size, validity_size);
   });

   // allocate 
   auto device_buf = std::make_unique<rmm::device_buffer>(rmm::device_buffer{data_size + validity_size, stream, mr});   
   char *buf = static_cast<char*>(device_buf->data());

   // copy
   std::vector<column_view> out_cols;
   out_cols.reserve(t.num_columns());
   std::for_each(t.begin(), t.end(), [&out_cols, &buf](cudf::column_view const& c){
      cudf::experimental::type_dispatcher(c.type(), column_copy_functor{}, c, buf, out_cols);
   });

   return contiguous_split_result{cudf::table_view{out_cols}, std::move(device_buf)};
}

}; // anonymous namespace

std::vector<contiguous_split_result> contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr,
                                                      hipStream_t stream)
{    
   auto subtables = cudf::experimental::split(input, splits);      

   std::vector<contiguous_split_result> result;
   std::transform(subtables.begin(), subtables.end(), std::back_inserter(result), [mr, stream](table_view const& t) { 
      return alloc_and_copy(t, mr, stream);
   });
   
   return result;
}

}; // namespace detail

std::vector<contiguous_split_result> contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr)
{    
   return cudf::experimental::detail::contiguous_split(input, splits, mr, (hipStream_t)0);   
}

}; // namespace experimental

}; // namespace cudf
#include "hip/hip_runtime.h"
#include <cudf/cudf.h>
#include <cudf/column/column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/copying.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/utilities/bit.hpp>

#include <numeric>

namespace cudf {

namespace experimental {

namespace detail {

namespace {

template <typename S>
__device__ inline S round_up_safe_nothrow(S number_to_round, S modulus) {
    auto remainder = number_to_round % modulus;
    if (remainder == 0) { return number_to_round; }
    auto rounded_up = number_to_round - remainder + modulus;    
    return rounded_up;
}

// Computes required allocation size of a bitmask
__device__ std::size_t bitmask_allocation_size_bytes_nothrow(size_type number_of_bits,
                                          std::size_t padding_boundary) {  
  auto necessary_bytes =
      cudf::util::div_rounding_up_safe<size_type>(number_of_bits, CHAR_BIT);

  auto padded_bytes =
      padding_boundary * cudf::util::div_rounding_up_safe<size_type>(
                             necessary_bytes, padding_boundary);
  return padded_bytes;
}


/**
 * @brief Copies contents of `in` to `out`.  Copies validity if present
 * but does not compute null count.
 *  
 * @param in column_view to copy from
 * @param out mutable_column_view to copy to.
 */
template <size_type block_size, typename T, bool has_validity>
__launch_bounds__(block_size)
__global__
void copy_in_place_kernel( column_device_view const in,
                           size_type validity_size,
                           mutable_column_device_view out,
                           T val_subtract)
{
   const size_type tid = threadIdx.x + blockIdx.x * block_size;
   const int warp_id = tid / cudf::experimental::detail::warp_size;
   const size_type warps_per_grid = gridDim.x * block_size / cudf::experimental::detail::warp_size;      

   // begin/end indices for the column data
   size_type begin = 0;      
   size_type end = in.size();   
   size_type validity_end = validity_size;   
   // warp indices.  since 1 warp == 32 threads == sizeof(bit_mask_t) * 8,
   // each warp will process one (32 bit) of the validity mask via
   // __ballot_sync()
   size_type warp_begin = cudf::word_index(begin);   
   size_type warp_end = cudf::word_index(end-1);         

   // lane id within the current warp   
   const int lane_id = threadIdx.x % cudf::experimental::detail::warp_size;
   
   // current warp.
   size_type warp_cur = warp_begin + warp_id;   
   size_type index = tid;
   while(warp_cur <= warp_end){
      bool validity_in_range = (index >= begin && index < validity_end);
      bool valid = true;      
      if(has_validity){         
         valid = validity_in_range && in.is_valid(index);
      }

      bool in_range = (index >= begin && index < end);
      if(in_range){         
         out.element<T>(index) = in.element<T>(index) - val_subtract;
      }
      
      // update validity      
      if(has_validity && validity_in_range){
         // the final validity mask for this warp 
         int warp_mask = __ballot_sync(0xFFFF'FFFF, valid && validity_in_range);
         // only one guy in the warp needs to update the mask and count
         if(lane_id == 0){            
            out.set_mask_word(warp_cur, warp_mask);            
         }
      }            

      // next grid
      warp_cur += warps_per_grid;
      index += block_size * gridDim.x;
   }
}

// align all column size allocations to this boundary so that all output column buffers
// start at that alignment.
static constexpr size_t split_align = 64;

struct column_split_info {
   size_t   data_size;     // size of the data
   size_t   validity_size; // validity vector size
   
   size_t   offsets_size;  // (strings only) size of offset column
   size_t   chars_size;    // (strings only) # of chars in the column
   size_t   chars_offset;  // (strings only) offset from head of chars data
};

/**
 * @brief Functor called by the `type_dispatcher` to incrementally compute total
 * memory buffer size needed to allocate a contiguous copy of all columns within
 * a source table. 
 */
struct column_buffer_size_functor {
   template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
   size_t operator()(column_view const& c, column_split_info &split_info)
   {
      // this has already been precomputed in an earlier step      
      return split_info.data_size + split_info.validity_size + split_info.offsets_size;
   }

   template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
   size_t operator()(column_view const& c, column_split_info &split_info)
   {      
      split_info.data_size = cudf::util::round_up_safe(c.size() * sizeof(T), split_align);  
      split_info.validity_size = (c.nullable() ? cudf::bitmask_allocation_size_bytes(c.size(), split_align) : 0);
      return split_info.data_size + split_info.validity_size;
   }
};

/**
 * @brief Functor called by the `type_dispatcher` to copy a column into a contiguous
 * buffer of output memory. 
 * 
 * Used for copying each column in a source table into one contiguous buffer of memory.
 */
struct column_copy_functor {
   template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
   void operator()(column_view const& in, column_split_info const& split_info, char*& dst, std::vector<column_view>& out_cols)
   {            
      strings_column_view strings_c(in);      

      // outgoing pointers
      char* chars_buf = dst;
      bitmask_type* validity_buf = split_info.validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + split_info.data_size);
      char* offsets_buf = dst + split_info.data_size + split_info.validity_size;

      // increment working buffer
      dst += (split_info.data_size + split_info.validity_size + split_info.offsets_size);

      // 2 kernel calls. 1 to copy offsets and validity, and another to copy chars            
      
      // copy offsets and validity
      column_view offsets_col = strings_c.offsets();
      mutable_column_view temp_offsets_and_validity{
                              offsets_col.type(), offsets_col.size(), offsets_buf,
                              validity_buf, validity_buf == nullptr ? UNKNOWN_NULL_COUNT : 0,
                              0 };
      {         
         // contruct a column which wraps the validity vector and the offsets from the child column. 
         // this is weird but it removes an extra kernel call. however, since the length of the offsets column
         // is always 1 greater than the # of strings, the validity vector will be short by 1. the kernel will have to
         // compensate for that. 
         CUDF_EXPECTS(in.size() == offsets_col.size()-1, "Expected offsets column to be the same size as parent");
         CUDF_EXPECTS(in.offset() == offsets_col.offset(), "Expected offsets column offset to be the same as parent");
         CUDF_EXPECTS(offsets_col.type() == cudf::data_type(INT32), "Expected offsets column type to be int32");
         column_view in_offsets_and_validity{
                                 offsets_col.type(), offsets_col.size(), offsets_col.head<int32_t>(),
                                 in.null_mask(), in.null_mask() == nullptr ? UNKNOWN_NULL_COUNT : 0,
                                 in.offset()};
         
         cudf::size_type num_els = cudf::util::round_up_safe(strings_c.offsets().size(), cudf::experimental::detail::warp_size);
         constexpr int block_size = 256;
         cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};         
         if(in.nullable()){
            copy_in_place_kernel<block_size, size_type, true><<<grid.num_blocks, block_size, 0, 0>>>(
                              *column_device_view::create(in_offsets_and_validity), 
                              in.size(),  // validity vector length
                              *mutable_column_device_view::create(temp_offsets_and_validity), split_info.chars_offset);
         } else {
            copy_in_place_kernel<block_size, size_type, false><<<grid.num_blocks, block_size, 0, 0>>>(
                              *column_device_view::create(in_offsets_and_validity),
                              in.size(),  // validity vector length
                              *mutable_column_device_view::create(temp_offsets_and_validity), split_info.chars_offset);
         }
      }

      // get the chars column directly instead of calling .chars(), since .chars() will end up
      // doing gpu work we specifically want to avoid.
      column_view chars_col = in.child(strings_column_view::chars_column_index);

      // copy chars
      mutable_column_view out_chars{chars_col.type(), static_cast<size_type>(split_info.chars_size), chars_buf};      
      {         
         CUDF_EXPECTS(!chars_col.nullable(), "Expected input chars column to not be nullable");
         CUDF_EXPECTS(chars_col.offset() == 0, "Expected input chars column to have an offset of 0");
         column_view in_chars{ chars_col.type(), static_cast<size_type>(split_info.chars_size), chars_col.data<char>() + split_info.chars_offset };
                                 
         cudf::size_type num_els = cudf::util::round_up_safe(static_cast<size_type>(split_info.chars_size), cudf::experimental::detail::warp_size);
         constexpr int block_size = 256;
         cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};         
         copy_in_place_kernel<block_size, char, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_chars),
                           split_info.chars_size,
                           *mutable_column_device_view::create(out_chars), 0);
      }

      // construct output string column_view.  offsets and validity have been glued together so
      // we have to rearrange things a bit.      
      column_view out_offsets{strings_c.offsets().type(), strings_c.offsets().size(), offsets_buf};
      
      out_cols.push_back(column_view(in.type(), in.size(), nullptr,
                                     validity_buf, UNKNOWN_NULL_COUNT, 0,
                                     { out_offsets, out_chars }));
   }

   template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
   void operator()(column_view const& in, column_split_info const& split_info, char*& dst, std::vector<column_view>& out_cols)
   {     
      // outgoing pointers
      char* data = dst;
      bitmask_type* validity = split_info.validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + split_info.data_size);

      // increment working buffer
      dst += (split_info.data_size + split_info.validity_size);

      // custom copy kernel (which should probably just be an in-place copy() function in cudf.
      cudf::size_type num_els = cudf::util::round_up_safe(in.size(), cudf::experimental::detail::warp_size);
      constexpr int block_size = 256;
      cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};
      
      // so there's a significant performance issue that comes up. our incoming column_view objects
      // are the result of a slice.  because of this, they have an UNKNOWN_NULL_COUNT.  because of that,
      // calling column_device_view::create() will cause a recompute of the count, which ends up being
      // extremely slow because a.) the typical use case here will involve huge numbers of calls and
      // b.) the count recompute involves tons of device allocs and memcopies.
      //
      // so to get around this, I am manually constructing a fake-ish view here where the null
      // count is arbitrarily bashed to 0.            
      //            
      // Remove this hack once rapidsai/cudf#3600 is fixed.
      column_view   in_wrapped{in.type(), in.size(), in.head<T>(), 
                               in.null_mask(), in.null_mask() == nullptr ? UNKNOWN_NULL_COUNT : 0,
                               in.offset() };
      mutable_column_view  mcv{in.type(), in.size(), data, 
                               validity, validity == nullptr ? UNKNOWN_NULL_COUNT : 0 };      
      if(in.nullable()){               
         copy_in_place_kernel<block_size, T, true><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped), 
                           in.size(),
                           *mutable_column_device_view::create(mcv), 0);         
      } else {
         copy_in_place_kernel<block_size, T, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped), 
                           in.size(),
                           *mutable_column_device_view::create(mcv), 0);
      }
      mcv.set_null_count(cudf::UNKNOWN_NULL_COUNT);                 

      out_cols.push_back(mcv);
   }
};

/**
 * @brief Creates a contiguous_split_result object which contains a deep-copy of the input
 * table_view into a single contiguous block of memory. 
 * 
 * The table_view contained within the contiguous_split_result will pass an expect_tables_equal()
 * call with the input table.  The memory referenced by the table_view and its internal column_views
 * is entirely contained in single block of memory.
 */
contiguous_split_result alloc_and_copy(cudf::table_view const& t, thrust::device_vector<column_split_info>& device_split_info, rmm::mr::device_memory_resource* mr, hipStream_t stream)      
{            
   // preprocess column sizes for string columns.  the idea here is this:
   // - determining string lengths involves reaching into device memory to look at offsets, which is slow.
   // - contiguous_split() is typically used in situations with very large numbers of output columns, exaggerating
   //   the problem.
   // - so rather than reaching into device memory once per column (in column_buffer_size_functor), 
   //   we are doing it once per split (for all string columns in the split).  For an example case of a table with 
   //   512 columns split 256 ways, that reduces our number of trips to/from the gpu from 128k -> 256
   
   // build a list of all the offset columns and their indices for all input string columns and put them on the gpu
   //
   // i'm using this pair structure instead of thrust::tuple because using tuple somehow causes the cudf::column_device_view
   // default constructor to get called (compiler error) when doing the assignment to device_offset_columns below
   thrust::host_vector<thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>> offset_columns;
   offset_columns.reserve(t.num_columns());  // worst case
   size_type column_index = 0;
   std::for_each(t.begin(), t.end(), [&offset_columns, &column_index](cudf::column_view const& c){
      if(c.type().id() == STRING){
         // constructing device view from the offsets column only, because doing so for the entire
         // strings_column_view will result in memory allocation/hipMemcpy() calls, which would
         // defeat the whole purpose of this step.
         cudf::column_device_view cdv((strings_column_view(c)).offsets(), 0, 0);
         offset_columns.push_back(thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>(
                  thrust::pair<size_type, bool>(column_index, c.nullable()), cdv));
      }
      column_index++;
   });   
   thrust::device_vector<thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>> device_offset_columns = offset_columns;   
     
   // compute column sizes for all string columns   
   auto *sizes_p = device_split_info.data().get();   
   thrust::for_each(rmm::exec_policy(stream)->on(stream), device_offset_columns.begin(), device_offset_columns.end(),
      [sizes_p] __device__ (auto column_info){
         size_type                  col_index = column_info.first.first;
         bool                       include_validity = column_info.first.second;
         cudf::column_device_view   col = column_info.second;
         size_type                  num_elements = col.size()-1;

         size_t align = split_align;

         auto num_chars = col.data<int32_t>()[num_elements] - col.data<int32_t>()[0];         
         sizes_p[col_index].data_size = round_up_safe_nothrow(static_cast<size_t>(num_chars), align);         
         // can't use cudf::bitmask_allocation_size_bytes() because it throws
         sizes_p[col_index].validity_size = include_validity ? bitmask_allocation_size_bytes_nothrow(num_elements, align) : 0;                  
         // can't use cudf::util::round_up_safe() because it throws
         sizes_p[col_index].offsets_size = round_up_safe_nothrow(col.size() * sizeof(size_type), align);
         sizes_p[col_index].chars_size = num_chars;
         sizes_p[col_index].chars_offset = col.data<int32_t>()[0];
      }
   );
   
   // copy sizes back from gpu. entries from non-string columns are uninitialized at this point.
   thrust::host_vector<column_split_info> split_info = device_split_info;  
     
   // compute the rest of the column sizes (non-string columns, and total buffer size)
   size_t total_size = 0;
   column_index = 0;
   std::for_each(t.begin(), t.end(), [&total_size, &column_index, &split_info](cudf::column_view const& c){   
      total_size += cudf::experimental::type_dispatcher(c.type(), column_buffer_size_functor{}, c, split_info[column_index]);
      column_index++;
   });

   // allocate
   auto device_buf = std::make_unique<rmm::device_buffer>(total_size, stream, mr);
   char *buf = static_cast<char*>(device_buf->data());

   // copy (this would be cleaner with a std::transform, but there's an nvcc compiler issue in the way)   
   std::vector<column_view> out_cols;
   out_cols.reserve(t.num_columns());
   column_index = 0;   
   std::for_each(t.begin(), t.end(), [&out_cols, &buf, &column_index, &split_info](cudf::column_view const& c){
      cudf::experimental::type_dispatcher(c.type(), column_copy_functor{}, c, split_info[column_index], buf, out_cols);
      column_index++;
   });   
   
   return contiguous_split_result{cudf::table_view{out_cols}, std::move(device_buf)};   
}

}; // anonymous namespace

std::vector<contiguous_split_result> contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr,
                                                      hipStream_t stream)
{   
   auto subtables = cudf::experimental::split(input, splits);

   // optimization : for large #'s of splits this allocation can dominate total time
   //                spent if done inside alloc_and_copy().  so we'll allocate it once
   //                and reuse it.
   // 
   //                benchmark:        1 GB data, 10 columns, 256 splits.
   //                no optimization:  106 ms (8 GB/s)
   //                optimization:     20 ms (48 GB/s)
   thrust::device_vector<column_split_info> device_split_info(input.num_columns());

   std::vector<contiguous_split_result> result;
   std::transform(subtables.begin(), subtables.end(), std::back_inserter(result), [mr, stream, &device_split_info](table_view const& t) { 
      return alloc_and_copy(t, device_split_info, mr, stream);
   });

   return result;
}

}; // namespace detail

std::vector<contiguous_split_result> contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr)
{    
   return cudf::experimental::detail::contiguous_split(input, splits, mr, (hipStream_t)0);   
}

}; // namespace experimental

}; // namespace cudf

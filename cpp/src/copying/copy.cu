/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/wrappers/bool.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/copy_if_else.cuh>

namespace cudf {
namespace experimental {
namespace detail {

// column/column, scalar/column
template <typename Left, typename Right>
std::unique_ptr<column> copy_if_else( Left const& lhs, Right const& rhs, 
                                      column_view const& boolean_mask, bool invert_mask,
                                      rmm::mr::device_memory_resource *mr,
                                      hipStream_t stream)
{
   CUDF_EXPECTS(lhs.type() == rhs.type(), "Both inputs must be of the same type");
   CUDF_EXPECTS(not boolean_mask.has_nulls(), "Boolean mask must not contain null values.");
   CUDF_EXPECTS(boolean_mask.type() == data_type(BOOL8), "Boolean mask column must be of type BOOL8");   
   CUDF_EXPECTS(boolean_mask.size() == lhs.size(), "Boolean mask column must be the same size as lhs and rhs columns");
   if (lhs.size() == 0) {
       return cudf::experimental::empty_like(lhs);
   }

   auto bool_mask_device_p = column_device_view::create(boolean_mask);
   column_device_view bool_mask_device = *bool_mask_device_p;

   if(invert_mask){
      auto filter = [bool_mask_device] __device__ (cudf::size_type i) { return !bool_mask_device.element<cudf::experimental::bool8>(i); };
      return copy_if_else(lhs, rhs, filter, mr, stream);
   }
   auto filter = [bool_mask_device] __device__ (cudf::size_type i) { return bool_mask_device.element<cudf::experimental::bool8>(i); };
   return copy_if_else(lhs, rhs, filter, mr, stream);
}

// scalar/scalar.
std::unique_ptr<column> copy_if_else( scalar const& lhs, scalar const& rhs,
                                      column_view const& boolean_mask,
                                      rmm::mr::device_memory_resource *mr,
                                      hipStream_t stream)
{
   CUDF_EXPECTS(lhs.type() == rhs.type(), "Both inputs must be of the same type");
   CUDF_EXPECTS(not boolean_mask.has_nulls(), "Boolean mask must not contain null values.");
   CUDF_EXPECTS(boolean_mask.type() == data_type(BOOL8), "Boolean mask column must be of type BOOL8");

   auto bool_mask_device_p = column_device_view::create(boolean_mask);
   column_device_view bool_mask_device = *bool_mask_device_p;
   auto filter = [bool_mask_device] __device__ (cudf::size_type i) { return bool_mask_device.element<cudf::experimental::bool8>(i); };

   return copy_if_else(lhs, rhs, boolean_mask.size(), filter, mr, stream);
}

}  // namespace detail

std::unique_ptr<column> copy_if_else( column_view const& lhs, column_view const& rhs, column_view const& boolean_mask,
                                      rmm::mr::device_memory_resource *mr)
{
   CUDF_EXPECTS(lhs.size() == rhs.size(), "Both columns must be of the size");
   CUDF_EXPECTS(boolean_mask.size() == lhs.size(), "Boolean mask column must be the same size as lhs and rhs columns");
   return detail::copy_if_else(lhs, rhs, boolean_mask, false, mr, 0);
}

std::unique_ptr<column> copy_if_else( scalar const& lhs, column_view const& rhs, column_view const& boolean_mask,
                                      rmm::mr::device_memory_resource *mr)
{
   CUDF_EXPECTS(boolean_mask.size() == rhs.size(), "Boolean mask column must be the same size as rhs column"); 
   return detail::copy_if_else(lhs, rhs, boolean_mask, false, mr, 0);
}

std::unique_ptr<column> copy_if_else( column_view const& lhs, scalar const& rhs, column_view const& boolean_mask,
                                      rmm::mr::device_memory_resource *mr)
{
   CUDF_EXPECTS(boolean_mask.size() == lhs.size(), "Boolean mask column must be the same size as lhs column");
   return detail::copy_if_else(rhs, lhs, boolean_mask, true, mr, 0);
}

std::unique_ptr<column> copy_if_else( scalar const& lhs, scalar const& rhs, column_view const& boolean_mask,
                                      rmm::mr::device_memory_resource *mr)
{
   return detail::copy_if_else(lhs, rhs, boolean_mask, mr, 0);
}

} // namespace experimental
} // namespace cudf

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "gather.cuh"
#include <cudf/copying.hpp>
#include <cudf/filling.hpp>
#include <cudf/cudf.h>
#include <rmm/thrust_rmm_allocator.h>
#include <utilities/cudf_utils.h>
#include <cudf/legacy/table.hpp>

#include <copying/gather.hpp>

#include <cudf/types.h>
#include <utilities/bit_util.cuh>
#include <utilities/cuda_utils.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <cudf/utilities/legacy/nvcategory_util.hpp>
#include <utilities/column_utils.hpp>
#include <bitmask/legacy/bit_mask.cuh>

using bit_mask::bit_mask_t;

namespace cudf {
namespace detail {


template <typename value_type, typename InputIterator>
struct scatter_to_gather {
  scatter_to_gather(InputIterator first, InputIterator last, value_type default_value):
    first(first), last(last), default_value(default_value) {}

  value_type operator()(value_type in) {
    InputIterator found = thrust::find(first, last, in);
    if (found == last) {
      return default_value;
    }
    else {
      return thrust::distance(first, found);
    }
  }

  InputIterator first, last;
  value_type default_value;
};

struct dispatch_map_type {
template <typename map_type, std::enable_if_t<std::is_integral<map_type>::value>* = nullptr>
void operator()(table const* source_table,
		const gdf_column scatter_map,
		table *destination_table) {
  map_type const * typed_scatter_map = static_cast<map_type const*>(scatter_map.data);

  // Turn the scatter_map[] into a gather_map[] and then call gather(...).
  auto gather_map = thrust::make_transform_iterator(
      thrust::make_counting_iterator(0),
      scatter_to_gather<map_type, map_type const*>(
	  typed_scatter_map,
	  typed_scatter_map+source_table->num_rows(),
	  -1));;

  detail::gather<decltype(gather_map)>(source_table, gather_map, destination_table, false, true, true, false);

}

template <typename map_type, std::enable_if_t<not std::is_integral<map_type>::value>* = nullptr>
void operator()(table const* source_table,
    const gdf_column scatter_map,
    table *destination_table) {
  CUDF_FAIL("Scatter map must be an integral type.");
}

};

void scatter(table const* source_table, gdf_column const scatter_map, table* destination_table) {
  const gdf_size_type num_source_rows = source_table->num_rows();
  const gdf_size_type num_destination_rows = destination_table->num_rows();
  
  CUDF_EXPECTS(nullptr != source_table, "source table is null");
  CUDF_EXPECTS(nullptr != destination_table, "destination table is null");
  
  if (0 == source_table->num_rows()) {
    return;
  }

  type_dispatcher(scatter_map.dtype, dispatch_map_type{},
		  source_table,
		  scatter_map,
		  destination_table);
}

void scatter(table const* source_table, gdf_index_type const scatter_map[],
            table* destination_table) {
  gdf_column scatter_map_column{};
  gdf_column_view(&scatter_map_column,
		  const_cast<gdf_index_type*>(scatter_map),
		  nullptr,
		  source_table->num_rows(),
		  gdf_dtype_of<gdf_index_type>());
  scatter(source_table, scatter_map_column, destination_table);
}

template<bool mark_true>
__global__ void marking_bitmask_kernel(
    bit_mask_t* destination_mask,
    gdf_size_type num_destination_rows,
    const gdf_index_type scatter_map[],
    gdf_size_type num_scatter_rows
){
  
  gdf_index_type row = threadIdx.x + blockIdx.x * blockDim.x;
 
  while (row < num_scatter_rows) {

    const gdf_index_type output_row = scatter_map[row];

    if(mark_true){
      bit_mask::set_bit_safe(destination_mask, output_row);
    }else{
      bit_mask::clear_bit_safe(destination_mask, output_row);
    }

    row += blockDim.x * gridDim.x;
  }
}

struct scalar_scatterer {
  /**---------------------------------------------------------------------------*
   * @brief Type-dispatched function to scatter from one scalar to a table based
   * on a `scatter_map`.
   *
   * @tparam ColumnType Dispatched type for the column being scattered 
   * @param source The scalar to scatter to
   * @param scatter_map Array of indices that maps the source element to destination
   * elements
   * @param destination_column The column to gather into
   * @param stream Optional CUDA stream on which to execute kernels
   *---------------------------------------------------------------------------**/
  template <typename ColumnType>
  void operator()(gdf_scalar const& source,
                  gdf_index_type const scatter_map[], const gdf_size_type num_scatter_rows,
                  gdf_column* destination_column, hipStream_t stream = 0) {
    
    const ColumnType source_data {
        *reinterpret_cast<ColumnType const*>(&source.data) };
    ColumnType* destination_data {
        reinterpret_cast<ColumnType*>(destination_column->data) };

    thrust::constant_iterator<ColumnType> const_iter(source_data);
    thrust::scatter(rmm::exec_policy(stream)->on(stream), const_iter,
                     const_iter + num_scatter_rows, scatter_map, 
                     destination_data);
    
    CHECK_STREAM(stream);
  
  }
};

void scalar_scatter(const std::vector<gdf_scalar>& source, 
                    gdf_index_type const scatter_map[],
                    gdf_size_type num_scatter_rows, table* destination_table){
 
  CUDF_EXPECTS(source.size() == (size_t)destination_table->num_columns(),
    "scalar vector and destination table size mismatch.");

  const int n_cols = source.size();

  std::vector<cudf::util::cuda::scoped_stream> v_streams(2*n_cols);

  // data part
  for(int i = 0; i < n_cols; i++){
    CUDF_EXPECTS(source[i].dtype == destination_table->get_column(i)->dtype,
        "source/destination data type mismatch.");
    CUDF_EXPECTS(source[i].dtype != GDF_STRING_CATEGORY,
        "Scalar scatter currently does not support GDF_STRING_CATEGORY.");
    type_dispatcher(source[i].dtype, scalar_scatterer{}, source[i], 
        scatter_map, num_scatter_rows, destination_table->get_column(i), v_streams[i]);
  }

  constexpr int block_size = 256;  
  const int grid_size = cudf::util::cuda::grid_config_1d(num_scatter_rows, block_size).num_blocks;
  
  // bitmask part
  for(int i = 0; i < n_cols; i++){
    gdf_column* dest_col = destination_table->get_column(i);
    if(dest_col->valid){
      bit_mask_t* dest_valid = reinterpret_cast<bit_mask_t*>(dest_col->valid);
      auto bitmask_kernel = source[i].is_valid ?
        marking_bitmask_kernel<true> : marking_bitmask_kernel<false>;
      bitmask_kernel<<<grid_size, block_size, 0, v_streams[i+n_cols]>>>
        (dest_valid, dest_col->size, scatter_map, num_scatter_rows);
      set_null_count(*dest_col);
    }
  }

}

}  // namespace detail

table scatter(table const& source, gdf_column const scatter_map,
	      table const& target) {

  const gdf_size_type n_cols = target.num_columns();

  table output = copy(target);
  for(int i = 0; i < n_cols; ++i){
    // Allocate bitmask for each column
    if(cudf::has_nulls(*source.get_column(i)) && !is_nullable(*target.get_column(i))){

      gdf_size_type valid_size = gdf_valid_allocation_size(target.get_column(i)->size);
      RMM_TRY(RMM_ALLOC(&output.get_column(i)->valid, valid_size, 0));

      gdf_size_type valid_size_set = gdf_num_bitmask_elements(target.get_column(i)->size);
      CUDA_TRY(hipMemset(output.get_column(i)->valid, 0xff, valid_size_set));

    }
  }

  detail::scatter(&source, scatter_map, &output);
  nvcategory_gather_table(output, output);

  return output;

}


table scatter(table const& source, gdf_index_type const scatter_map[], 
    table const& target) {
  
  const gdf_size_type n_cols = target.num_columns();

  table output = copy(target);
  for(int i = 0; i < n_cols; ++i){
    // Allocate bitmask for each column
    if(cudf::has_nulls(*source.get_column(i)) && !is_nullable(*target.get_column(i))){
      
      gdf_size_type valid_size = gdf_valid_allocation_size(target.get_column(i)->size);
      RMM_TRY(RMM_ALLOC(&output.get_column(i)->valid, valid_size, 0));
      
      gdf_size_type valid_size_set = gdf_num_bitmask_elements(target.get_column(i)->size);
      CUDA_TRY(hipMemset(output.get_column(i)->valid, 0xff, valid_size_set));
    
    }
  }

  detail::scatter(&source, scatter_map, &output);
  nvcategory_gather_table(output, output);

  return output;

}

table scatter(std::vector<gdf_scalar> const& source, 
              gdf_index_type const scatter_map[],
              gdf_size_type num_scatter_rows, table const& target){

  const gdf_size_type n_cols = target.num_columns();

  table output = copy(target);
  for(int i = 0; i < n_cols; ++i){
    // Allocate bitmask for each column
    if(source[i].is_valid == false && !is_nullable(*target.get_column(i))){
      
      gdf_size_type valid_size = gdf_valid_allocation_size(target.get_column(i)->size);
      RMM_TRY(RMM_ALLOC(&output.get_column(i)->valid, valid_size, 0));
    	
      gdf_size_type valid_size_set = gdf_num_bitmask_elements(target.get_column(i)->size);
      CUDA_TRY(hipMemset(output.get_column(i)->valid, 0xff, valid_size_set));
    
    }
  }

  detail::scalar_scatter(source, scatter_map, num_scatter_rows, &output);
  
  return output;
}


}  // namespace cudf

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <memory>
#include <cudf/copying.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/detail/fill.hpp>
#include <cudf/detail/gather.hpp>
#include "cudf/column/column_device_view.cuh"
#include "cudf/null_mask.hpp"
#include "cudf/types.hpp"
#include "cudf/utilities/traits.hpp"
#include "cudf/utilities/type_dispatcher.hpp"
#include "hip/driver_types.h"
#include "rmm/device_scalar.hpp"
#include "rmm/thrust_rmm_allocator.h"
#include "thrust/detail/copy.h"
#include "thrust/execution_policy.h"
#include "thrust/for_each.h"
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/iterator/transform_iterator.h>
#include <cudf/detail/valid_if.cuh>
#include <cudf/scalar/scalar_device_view.cuh>

namespace cudf {
namespace experimental {
namespace {

template<typename T>
struct value_functor {
    column_device_view const input;
    size_type size;
    size_type offset;
    T const* fill;

    T __device__ operator()(size_type idx) {
        auto src_idx = idx - offset;
        return src_idx < 0 || src_idx >= size
            ? *fill
            : input.element<T>(src_idx);
    }
};

struct validity_functor {
    column_device_view const input;
    size_type size;
    size_type offset;
    bool const* fill;

    bool __device__ operator()(size_type idx) {
        auto src_idx = idx - offset;
        return src_idx < 0 || src_idx >= size
            ? *fill
            : input.is_valid(src_idx);
    }
};

struct functor {

    template<typename T, typename... Args>
    std::enable_if_t<not cudf::is_fixed_width<T>(), std::unique_ptr<column>>
    operator()(Args&&... args)
    {
        throw cudf::logic_error("shift does not support non-fixed-width types.");
    }

    template<typename T>
    std::enable_if_t<cudf::is_fixed_width<T>(), std::unique_ptr<column>>
    operator()(column_view const& input,
               size_type offset,
               scalar const& fill_value,
               rmm::mr::device_memory_resource *mr,
               hipStream_t stream)
    {
        using ScalarType = cudf::experimental::scalar_type_t<T>;
        auto& scalar = static_cast<ScalarType const&>(fill_value);

        auto device_input = column_device_view::create(input);
        auto output = allocate_like(input, mask_allocation_policy::NEVER);
        auto device_output = mutable_column_device_view::create(*output);

        auto index_begin = thrust::make_counting_iterator<size_type>(0);
        auto index_end = thrust::make_counting_iterator<size_type>(input.size());

        auto func_value = value_functor<T>{*device_input,
                                           input.size(),
                                           offset,
                                           scalar.data() };

        if (scalar.is_valid() && not input.nullable())
        {
            thrust::transform(rmm::exec_policy(stream)->on(stream),
                              index_begin,
                              index_end,
                              device_output->data<T>(),
                              func_value);

            return output;
        }


        auto func_validity = validity_functor{*device_input,
                                              input.size(),
                                              offset,
                                              scalar.validity_data()};

        thrust::transform_if(rmm::exec_policy(stream)->on(stream),
                             index_begin,
                             index_end,
                             device_output->data<T>(),
                             func_value,
                             func_validity);

        auto mask_pair = detail::valid_if(index_begin, index_end, func_validity);

        output->set_null_mask(std::move(std::get<0>(mask_pair)));
        output->set_null_count(std::get<1>(mask_pair));

        return output;
    }
};

} // anonymous namespace

std::unique_ptr<table> shift(table_view const& input,
                             size_type offset,
                             std::vector<std::unique_ptr<scalar>> const& fill_values,
                             rmm::mr::device_memory_resource *mr,
                             hipStream_t stream)
{
    if (input.num_rows() == 0) {
        return empty_like(input);
    }


    CUDF_EXPECTS(input.num_columns() == static_cast<size_type>(fill_values.size()),
                 "");

    for (size_type i = 0; i < input.num_columns(); ++i) {
        CUDF_EXPECTS(input.column(i).type() == fill_values[i]->type(),
                 "");
    }

    auto output_columns = std::vector<std::unique_ptr<column>>{};

    for (auto col = 0; col < input.num_columns(); col++) {
        auto input_column = input.column(col);
        auto const& fill_value = fill_values[col];
        output_columns.push_back(type_dispatcher(input_column.type(), functor{},
                                                 input_column, offset, *fill_value,
                                                 mr, stream));
    }

    return std::make_unique<table>(std::move(output_columns));
}

} // namespace experimental
} // namespace cudf

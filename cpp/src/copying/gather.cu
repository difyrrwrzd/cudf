#include "hip/hip_runtime.h"
#include "gather.cuh"
#include <cudf/types.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/copying.hpp>
#include <utilities/legacy/error_utils.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/count.h>

#include <memory>


namespace cudf {
namespace experimental {
namespace detail {


struct dispatch_map_type {
  template <typename map_type, std::enable_if_t<std::is_integral<map_type>::value>* = nullptr>
  std::unique_ptr<table> operator()(table_view const& source_table,
				    column_view const& gather_map,
				    size_type num_destination_rows, bool check_bounds,
				    bool ignore_out_of_bounds,
				    bool allow_negative_indices = false)
  {
    std::unique_ptr<table> destination_table;

    if (check_bounds) {
      cudf::size_type begin = (allow_negative_indices) ? -source_table.num_rows() : 0;
      CUDF_EXPECTS(
	  num_destination_rows == thrust::count_if(
	      rmm::exec_policy()->on(0),
	      gather_map.begin<map_type>(),
	      gather_map.end<map_type>(),
	      bounds_checker<map_type>{begin, source_table.num_rows()}),
	  "Index out of bounds.");
    }

    if (allow_negative_indices) {
      destination_table =
	gather(source_table,
	       thrust::make_transform_iterator(
					       gather_map.begin<map_type>(),
					       index_converter<map_type>{source_table.num_rows()}),
	       thrust::make_transform_iterator(
					       gather_map.end<map_type>(),
					       index_converter<map_type>{source_table.num_rows()}),
	       check_bounds,
	       ignore_out_of_bounds,
	       allow_negative_indices
	     );
    }
    else {
      destination_table =
	gather(source_table,
	       gather_map.begin<map_type>(),
	       gather_map.end<map_type>(),
	       check_bounds,
	       ignore_out_of_bounds,
	       allow_negative_indices
	       );
    }

    return destination_table;
  }

  template <typename map_type, std::enable_if_t<not std::is_integral<map_type>::value>* = nullptr>
  std::unique_ptr<table> operator()(table_view const& source_table, column_view const& gather_map,
				    size_type num_destination_rows, bool check_bounds,
				    bool ignore_out_of_bounds, bool allow_negative_indices = false) {
    CUDF_FAIL("Gather map must be an integral type.");
  }
};

std::unique_ptr<table> gather(table_view const& source_table, column_view const& gather_map,
			      bool check_bounds = false, bool ignore_out_of_bounds = false,
			      bool allow_negative_indices = false,
			      rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource()) {


  CUDF_EXPECTS(gather_map.has_nulls() == false, "gather_map contains nulls");

  std::unique_ptr<table> destination_table =
    cudf::experimental::type_dispatcher(gather_map.type(), dispatch_map_type{},
					source_table, gather_map,
					gather_map.size(),
					check_bounds, ignore_out_of_bounds,
					allow_negative_indices);

  return destination_table;
}


}  // namespace detail

std::unique_ptr<table> gather(table_view const& source_table, column_view const& gather_map,
			      bool check_bounds, rmm::mr::device_memory_resource* mr) {
  return detail::gather(source_table, gather_map, check_bounds, false, true, mr);
}

}  // namespace exp
}  // namespace cudf

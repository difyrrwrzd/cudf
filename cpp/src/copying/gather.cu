#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/cudf.h>
#include <utilities/cudf_utils.h>
#include <cudf/copying.hpp>
#include <utilities/bit_util.cuh>
#include <utilities/type_dispatcher.hpp>

#include <cudf/table.hpp>
#include <string/nvcategory_util.hpp>

#include <algorithm>

#include <thrust/gather.h>
#include <table/device_table.cuh>

#include <hipcub/hipcub.hpp>
#include <utilities/column_utils.hpp>
#include <utilities/cuda_utils.hpp>
#include <string/nvcategory_util.hpp>
#include <nvstrings/NVCategory.h>
#include <nvstrings/NVStrings.h>
using bit_mask::bit_mask_t;

namespace cudf {
namespace detail {

constexpr int warp_size = 32;
  
/**---------------------------------------------------------------------------*
 * @brief Function object to check if an index is within the bounds [begin,
 * end).
 *
 *---------------------------------------------------------------------------**/
struct bounds_checker {
  gdf_index_type const begin;
  gdf_index_type const end;

  __device__ bounds_checker(gdf_index_type begin_, gdf_index_type end_)
      : begin{begin_}, end{end_} {}

  __device__ __forceinline__ bool operator()(gdf_index_type const index) {
    return ((index >= begin) && (index < end));
  }
};

template <class BitType, int lane = 0>
__device__ __inline__ gdf_size_type single_lane_reduce(BitType f) {
  static __shared__ gdf_size_type smem[warp_size];

  int lane_id = (threadIdx.x % warp_size);
  int warp_id = (threadIdx.x / warp_size);

  // Assuming one lane of each warp holds the value that we want to perform
  // reduction
  if (lane_id == lane) {
    smem[warp_id] = __popc(f);
  }
  __syncthreads();

  if (warp_id == 0) {
    // Here I am assuming maximum block size is 1024 and 1024 / 32 = 32
    // so one single warp is enough to do the reduction over different warps
    f = (lane_id < (blockDim.x / warp_size)) ? smem[lane_id] : 0;

    __shared__ typename hipcub::WarpReduce<gdf_size_type>::TempStorage temp_storage;
    f = hipcub::WarpReduce<gdf_size_type>(temp_storage).Sum(f);
  }

  return f;
}

template <bool check_bounds>
__global__ void gather_bitmask_kernel(const bit_mask_t* const* source_valid,
                                      gdf_size_type num_source_rows,
                                      const gdf_index_type* gather_map,
                                      bit_mask_t** destination_valid,
                                      gdf_size_type num_destination_rows,
                                      gdf_size_type* d_count,
                                      gdf_size_type num_columns) {
  for (gdf_index_type i = 0; i < num_columns; i++) {
    const bit_mask_t* source_valid_col = source_valid[i];
    bit_mask_t* destination_valid_col = destination_valid[i];

    const bool src_has_nulls = source_valid_col != nullptr;
    const bool dest_has_nulls = destination_valid_col != nullptr;

    if (dest_has_nulls) {
      gdf_index_type destination_row_base = blockIdx.x * blockDim.x;

      gdf_size_type valid_count_accumulate = 0;

      while (destination_row_base < num_destination_rows) {
        gdf_index_type destination_row = destination_row_base + threadIdx.x;

        const bool thread_active = destination_row < num_destination_rows;
        gdf_index_type source_row =
            thread_active ? gather_map[destination_row] : 0;

        const uint32_t active_threads =
            __ballot_sync(0xffffffff, thread_active);

        bool source_bit_is_valid;
        if (check_bounds) {
          if (0 <= source_row && source_row < num_source_rows) {
            source_bit_is_valid =
                src_has_nulls ? bit_mask::is_valid(source_valid_col, source_row)
                              : true;
          } else {
            // If gather_map does not include this row we should just keep the
            // original value,
            source_bit_is_valid =
                bit_mask::is_valid(destination_valid_col, destination_row);
          }
        } else {
          source_bit_is_valid =
              src_has_nulls ? bit_mask::is_valid(source_valid_col, source_row)
                            : true;
        }

        // Use ballot to find all valid bits in this warp and create the output
        // bitmask element
        const uint32_t valid_warp =
            __ballot_sync(active_threads, source_bit_is_valid);

        const gdf_index_type valid_index =
            cudf::util::detail::bit_container_index<bit_mask_t>(
                destination_row);
        // Only one thread writes output
        if (0 == threadIdx.x % warp_size && thread_active) {
          destination_valid_col[valid_index] = valid_warp;
        }
        valid_count_accumulate += single_lane_reduce(valid_warp);

        destination_row_base += blockDim.x * gridDim.x;
      }
      if (threadIdx.x == 0) {
        atomicAdd(d_count + i, valid_count_accumulate);
      }
    }
  }
}

/**---------------------------------------------------------------------------*
 * @brief Function object for gathering a type-erased
 * gdf_column. To be used with the cudf::type_dispatcher.
 *
 *---------------------------------------------------------------------------**/
struct column_gatherer {
  /**---------------------------------------------------------------------------*
   * @brief Type-dispatched function to gather from one column to another based
   * on a `gather_map`.
   *
   * @tparam ColumnType Dispatched type for the column being gathered
   * @param source_column The column to gather from
   * @param gather_map Array of indices that maps source elements to destination
   * elements
   * @param destination_column The column to gather into
   * @param check_bounds Optionally perform bounds checking on the values of
   * `gather_map`
   * @param stream Optional CUDA stream on which to execute kernels
   *---------------------------------------------------------------------------**/
  template <typename ColumnType>
  void operator()(gdf_column const* source_column,
                  gdf_index_type const gather_map[],
                  gdf_column* destination_column, bool check_bounds,
                  hipStream_t stream, bool merge_nvstring_category = false) {
    ColumnType const* const source_data{
        static_cast<ColumnType const*>(source_column->data)};
    ColumnType* destination_data{
        static_cast<ColumnType*>(destination_column->data)};

    gdf_size_type const num_destination_rows{destination_column->size};

    // If gathering in-place or scattering nvstring 
    // (in which case the merge_nvstring_category should be set to true) allocate 
    // temporary buffers to hold intermediate results
    bool const merge_category = std::is_same<ColumnType, nvstring_category>::value && merge_nvstring_category;
    bool const in_place = !merge_category && (source_data == destination_data);
    
    if(merge_category){
      // merge the categories.
      gdf_column temp_src = cudf::copy(*source_column);
      gdf_column copy_src = cudf::copy(*source_column);
      gdf_column temp_dest = cudf::copy(*destination_column);
      gdf_column copy_dest = cudf::copy(*destination_column);
      gdf_column* input_columns[2] = {&temp_src, &temp_dest};
      gdf_column* output_columns[2] = {&copy_src, &copy_dest};
      
      CUDF_EXPECTS(GDF_SUCCESS ==
        sync_column_categories(input_columns, output_columns, 2),
        "Failed to synchronize NVCategory");

      if (check_bounds) {
        thrust::gather_if(rmm::exec_policy(stream)->on(stream), gather_map,
                        gather_map + num_destination_rows, gather_map,
                        static_cast<ColumnType*>(copy_src.data), static_cast<ColumnType*>(copy_dest.data),
                        bounds_checker{0, source_column->size});
      } else {
        thrust::gather(rmm::exec_policy(stream)->on(stream), gather_map,
                     gather_map + num_destination_rows, static_cast<ColumnType*>(copy_src.data),
                     static_cast<ColumnType*>(copy_dest.data));
      }
      
      CUDF_EXPECTS(GDF_SUCCESS ==
        clear_column_categories(copy_dest, *destination_column), "Failed to clear NVCategory");

      gdf_column_free(&temp_src);
      gdf_column_free(&copy_src);
      gdf_column_free(&temp_dest);
      gdf_column_free(&copy_dest);
      return;
    }

    if (in_place) {
      RMM_TRY(RMM_ALLOC(&destination_data,
                        sizeof(ColumnType) * num_destination_rows, stream));
    }
    if (check_bounds) {
      thrust::gather_if(rmm::exec_policy(stream)->on(stream), gather_map,
                        gather_map + num_destination_rows, gather_map,
                        source_data, destination_data,
                        bounds_checker{0, source_column->size});
    } else {
      thrust::gather(rmm::exec_policy(stream)->on(stream), gather_map,
                     gather_map + num_destination_rows, source_data,
                     destination_data);
    }

    // Copy temporary buffers used for in-place gather to destination column
    if (in_place) {
      thrust::copy(rmm::exec_policy(stream)->on(stream), destination_data,
                   destination_data + num_destination_rows,
                   static_cast<ColumnType*>(destination_column->data));
      RMM_TRY(RMM_FREE(destination_data, stream));
    }

    CHECK_STREAM(stream);
  }
};

void gather(table const* source_table, gdf_index_type const gather_map[],
            table* destination_table, bool check_bounds, bool merge_nvstring_category) {
  CUDF_EXPECTS(nullptr != source_table, "source table is null");
  CUDF_EXPECTS(nullptr != destination_table, "destination table is null");

  // If the destination is empty, return immediately as there is nothing to
  // gather
  if (0 == destination_table->num_rows()) {
    return;
  }

  CUDF_EXPECTS(nullptr != gather_map, "gather_map is null");
  CUDF_EXPECTS(source_table->num_columns() == destination_table->num_columns(),
               "Mismatched number of columns");
  const gdf_size_type n_cols = source_table->num_columns();

  // We create (n_cols+1) streams for the (n_cols+1) kernels we are gonna
  // launch.
  std::vector<util::cuda::scoped_stream> v_stream(n_cols);

  for (gdf_size_type i = 0; i < n_cols; i++) {
    // Perform sanity checks
    gdf_column* dest_col = destination_table->get_column(i);
    const gdf_column* src_col = source_table->get_column(i);

    CUDF_EXPECTS(src_col->dtype == dest_col->dtype, "Column type mismatch");

    // If source table has 0 rows it is okay to have null buffers
    CUDF_EXPECTS(src_col->data != nullptr || source_table->num_rows() == 0,
                 "Missing source data buffer.");
    CUDF_EXPECTS(dest_col->data != nullptr, "Missing source data buffer.");

    // The data gather for n columns will be put on the first n streams
    cudf::type_dispatcher(src_col->dtype, column_gatherer{}, src_col,
                          gather_map, dest_col, check_bounds, v_stream[i], merge_nvstring_category);

    if(cudf::is_nullable(*src_col)){
       CUDF_EXPECTS(cudf::is_nullable(*dest_col), "Missing destination null mask.");
    }
  }

  rmm::device_vector<gdf_size_type> d_count_vec(n_cols, 0);

  std::vector<bit_mask_t*> h_bit_src(n_cols);
  std::vector<bit_mask_t*> h_bit_dest(n_cols);

  std::vector<rmm::device_vector<bit_mask_t>> vec_temp_bit(n_cols);

  for (gdf_size_type i = 0; i < n_cols; i++) {
    const gdf_column* dest_col = destination_table->get_column(i);
    h_bit_src[i] =
        reinterpret_cast<bit_mask_t*>(source_table->get_column(i)->valid);
    // Allocate inplace buffer
    if (cudf::is_nullable(*dest_col) &&
        dest_col->valid == source_table->get_column(i)->valid) {
      vec_temp_bit[i].resize(dest_col->size);
      h_bit_dest[i] = vec_temp_bit[i].data().get();
    } else {
      h_bit_dest[i] = reinterpret_cast<bit_mask_t*>(dest_col->valid);
    }
  }

  // In the following we allocate the device array thats hold the valid
  // bits.
  rmm::device_vector<bit_mask_t*> d_bit_src(n_cols);
  rmm::device_vector<bit_mask_t*> d_bit_dest(n_cols);
  CUDA_TRY(hipMemcpy(d_bit_src.data().get(), h_bit_src.data(),
                        n_cols * sizeof(bit_mask_t*), hipMemcpyHostToDevice));
  CUDA_TRY(hipMemcpy(d_bit_dest.data().get(), h_bit_dest.data(),
                        n_cols * sizeof(bit_mask_t*), hipMemcpyHostToDevice));

  auto bitmask_kernel =
      check_bounds ? gather_bitmask_kernel<true> : gather_bitmask_kernel<false>;

  int gather_grid_size;
  int gather_block_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&gather_grid_size,
                                              &gather_block_size, bitmask_kernel));

  bitmask_kernel<<<gather_grid_size, gather_block_size>>>(
      d_bit_src.data().get(), source_table->num_rows(), gather_map, d_bit_dest.data().get(),
      destination_table->num_rows(), d_count_vec.data().get(), n_cols);

  std::vector<gdf_size_type> h_count(n_cols);
  CUDA_TRY(hipMemcpy(h_count.data(), d_count_vec.data().get(),
                           sizeof(gdf_size_type) * n_cols,
                           hipMemcpyDeviceToHost));

  for (gdf_size_type i = 0; i < destination_table->num_columns(); i++) {
    gdf_column* dest_col = destination_table->get_column(i);
    if (is_nullable(*dest_col)) {
      // Copy temp buffer content back to column
      if (dest_col->valid == source_table->get_column(i)->valid) {
        gdf_size_type num_bitmask_elements =
            gdf_num_bitmask_elements(dest_col->size);
        CUDA_TRY(hipMemcpy(dest_col->valid, h_bit_dest[i],
                                 num_bitmask_elements, hipMemcpyDeviceToDevice));
      }
      dest_col->null_count = dest_col->size - h_count[i];
    } else {
      dest_col->null_count = 0;
    }
  }

}

}  // namespace detail

void gather(table const* source_table, gdf_index_type const gather_map[],
            table* destination_table) {
  detail::gather(source_table, gather_map, destination_table, false, false);
  nvcategory_gather_table(*source_table, *destination_table);
}

}  // namespace cudf

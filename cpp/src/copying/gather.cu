#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <utilities/legacy/error_utils.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/count.h>

#include <memory>

namespace cudf {
namespace experimental {
namespace detail {

struct dispatch_map_type {
  template <typename map_type,
            std::enable_if_t<std::is_integral<map_type>::value and
                             not std::is_same<map_type, bool>::value>* = nullptr>
  std::unique_ptr<table> operator()(
    table_view const& source_table,
    column_view const& gather_map,
    size_type num_destination_rows,
    bounds check_bounds,
    out_of_bounds oob,
    negative_indices neg_indices        = negative_indices::NOT_ALLOWED,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
    hipStream_t stream                 = 0)
  {
    if (check_bounds == bounds::CHECK) {
      cudf::size_type begin = neg_indices == negative_indices::ALLOW ? -source_table.num_rows() : 0;
      CUDF_EXPECTS(num_destination_rows ==
                     thrust::count_if(rmm::exec_policy()->on(0),
                                      gather_map.begin<map_type>(),
                                      gather_map.end<map_type>(),
                                      bounds_checker<map_type>{begin, source_table.num_rows()}),
                   "Index out of bounds.");
    }

    if (neg_indices == negative_indices::ALLOW) {
      auto idx_converter = index_converter<map_type>{source_table.num_rows()};
      return gather(source_table,
                    thrust::make_transform_iterator(gather_map.begin<map_type>(), idx_converter),
                    thrust::make_transform_iterator(gather_map.end<map_type>(), idx_converter),
                    oob == out_of_bounds::IGNORE,
                    mr,
                    stream);
    } else {
      return gather(source_table,
                    gather_map.begin<map_type>(),
                    gather_map.end<map_type>(),
                    oob == out_of_bounds::IGNORE,
                    mr,
                    stream);
    }
  }

  // TODO args
  template <typename map_type,
            std::enable_if_t<not std::is_integral<map_type>::value or
                             std::is_same<map_type, bool>::value>* = nullptr>
  std::unique_ptr<table> operator()(
    table_view const& source_table,
    column_view const& gather_map,
    size_type num_destination_rows,
    bounds check_bounds,
    out_of_bounds oob,
    negative_indices neg_indices        = negative_indices::NOT_ALLOWED,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
    hipStream_t stream                 = 0)
  {
    CUDF_FAIL("Gather map must be an integral type.");
  }
};  // namespace detail

std::unique_ptr<table> gather(table_view const& source_table,
                              column_view const& gather_map,
                              bounds check_bounds,
                              out_of_bounds oob,
                              negative_indices neg_indices,
                              rmm::mr::device_memory_resource* mr,
                              hipStream_t stream)
{
  CUDF_EXPECTS(gather_map.has_nulls() == false, "gather_map contains nulls");

  std::unique_ptr<table> destination_table =
    cudf::experimental::type_dispatcher(gather_map.type(),
                                        dispatch_map_type{},
                                        source_table,
                                        gather_map,
                                        gather_map.size(),
                                        check_bounds,
                                        oob,
                                        neg_indices,
                                        mr,
                                        stream);

  return destination_table;
}

}  // namespace detail

std::unique_ptr<table> gather(table_view const& source_table,
                              column_view const& gather_map,
                              bool check_bounds,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::gather(source_table,
                        gather_map,
                        check_bounds ? detail::bounds::CHECK : detail::bounds::NO_CHECK,
                        detail::out_of_bounds::DONT_IGNORE,
                        detail::negative_indices::ALLOW,
                        mr);
}

}  // namespace experimental
}  // namespace cudf

#include "hip/hip_runtime.h"
#include "gather.cuh"
#include <cudf/types.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/copying.hpp>
#include <utilities/error_utils.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/count.h>

#include <memory>


namespace cudf {
namespace experimental {
namespace detail {


struct dispatch_map_type {
  template <typename map_type, std::enable_if_t<std::is_integral<map_type>::value>* = nullptr>
  void operator()(table_view const& source_table, column_view const& gather_map,
		  mutable_table_view& destination_table, bool check_bounds,
		  bool ignore_out_of_bounds, bool allow_negative_indices = false)
  {

    map_type const * typed_gather_map = gather_map.data<map_type>();

    if (check_bounds) {
      cudf::size_type begin = (allow_negative_indices) ? -source_table.num_rows() : 0;
      CUDF_EXPECTS(
	  destination_table.num_rows() == thrust::count_if(
	      rmm::exec_policy()->on(0),
	      typed_gather_map,
	      typed_gather_map + destination_table.num_rows(),
	      bounds_checker<map_type>{begin, source_table.num_rows()}),
	  "Index out of bounds.");
    }

    if (allow_negative_indices) {
      gather(source_table,
	     thrust::make_transform_iterator(
		 typed_gather_map,
		 index_converter<map_type,index_conversion::NEGATIVE_TO_POSITIVE>{source_table.num_rows()}),
	     destination_table,
	     check_bounds,
	     ignore_out_of_bounds,
	     allow_negative_indices
	     );
    }
    else {
      gather(source_table,
	     thrust::make_transform_iterator(
		 typed_gather_map,
		 index_converter<map_type>{source_table.num_rows()}),
	     destination_table,
	     check_bounds,
	     ignore_out_of_bounds,
	     allow_negative_indices
	     );
    }
  }

  template <typename map_type, std::enable_if_t<not std::is_integral<map_type>::value>* = nullptr>
  void operator()(table_view const& source_table, column_view const& gather_map,
                  mutable_table_view& destination_table, bool check_bounds,
		  bool ignore_out_of_bounds, bool allow_negative_indices = false) {
   CUDF_FAIL("Gather map must be an integral type.");
  }
};

std::unique_ptr<table> gather(table_view const& source_table, column_view const& gather_map,
			      bool check_bounds = false, bool ignore_out_of_bounds = false,
			      bool allow_negative_indices = false,
			      rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource()) {

  std::vector<std::unique_ptr<column>> columns(source_table.num_columns());
  std::transform(source_table.begin(), source_table.end(), columns.begin(),
		 [&](column_view in_col) {
		   return allocate_like(in_col);
		 });

  std::unique_ptr<table> destination_table = std::make_unique<table>(std::move(columns));
  
  CUDF_EXPECTS(gather_map.has_nulls() == false, "gather_map contains nulls");
  CUDF_EXPECTS(source_table.num_columns() == destination_table->num_columns(),
               "Mismatched number of columns");

  mutable_table_view v {destination_table->mutable_view()};
  cudf::experimental::type_dispatcher(gather_map.type(), dispatch_map_type{},
				      source_table, gather_map, v,
				      check_bounds, ignore_out_of_bounds,
				      allow_negative_indices);

  return destination_table;
}


}  // namespace detail

std::unique_ptr<table> gather(table_view const& source_table, column_view const& gather_map,
			      bool check_bounds = false, bool allow_negative_indices = false,
			      rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource()) {

  return detail::gather(source_table, gather_map, check_bounds, false, true, mr);

}
    
}  // namespace exp
}  // namespace cudf

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "./utilities.h"

#include <cudf/column/column_factories.hpp>
#include <tests/utilities/column_utilities.cuh>

#include <cstring>
#include <thrust/execution_policy.h>
#include <thrust/equal.h>

#include <gmock/gmock.h>

namespace cudf {
namespace test {

// TODO investigate replace with strings_column_wrapper when available
std::unique_ptr<cudf::column> create_strings_column( const std::vector<const char*>& h_strings )
{
    cudf::size_type memsize = 0;
    for( auto itr=h_strings.begin(); itr!=h_strings.end(); ++itr )
        memsize += *itr ? (cudf::size_type)strlen(*itr) : 0;
    if( memsize==0 && h_strings.size() )
        memsize = 1; // prevent vectors from being null in all empty-string case
    cudf::size_type count = (cudf::size_type)h_strings.size();
    thrust::host_vector<char> h_buffer(memsize);
    thrust::device_vector<char> d_buffer(memsize);
    thrust::host_vector<thrust::pair<const char*,size_type> > strings(count);
    cudf::size_type offset = 0;
    for( cudf::size_type idx=0; idx < count; ++idx )
    {
        const char* str = h_strings[idx];
        if( !str )
            strings[idx] = thrust::pair<const char*,size_type>{nullptr,0};
        else
        {
            cudf::size_type length = (cudf::size_type)strlen(str);
            memcpy( h_buffer.data() + offset, str, length );
            strings[idx] = thrust::pair<const char*,size_type>{d_buffer.data().get()+offset,length};
            offset += length;
        }
    }
    rmm::device_vector<thrust::pair<const char*,size_type>> d_strings(strings);
    hipMemcpy( d_buffer.data().get(), h_buffer.data(), memsize, hipMemcpyHostToDevice );
    return cudf::make_strings_column( d_strings );
}

// TODO replace with column_wrapper when available
std::unique_ptr<cudf::column> create_numeric_column( const std::vector<int32_t>& h_values, const std::vector<bitmask_type>& h_nulls, cudf::size_type null_count )
{
    cudf::size_type count = static_cast<cudf::size_type>(h_values.size());
    rmm::device_buffer d_data{count*sizeof(int32_t)};
    hipMemcpy( d_data.data(), h_values.data(), count*sizeof(int32_t), hipMemcpyHostToDevice );
    rmm::device_buffer d_nulls;
    if( null_count )
    {
        d_nulls = rmm::device_buffer{ static_cast<size_t>(gdf_valid_allocation_size(count)) };
        hipMemcpy( d_nulls.data(), h_nulls.data(), gdf_valid_allocation_size(count), hipMemcpyHostToDevice );
    }
    return std::make_unique<cudf::column>( cudf::data_type{cudf::INT32},
                                           count, d_data,
                                           d_nulls, null_count );
}

void expect_strings_equal(cudf::column_view strings_column, const std::vector<const char*>& h_expected )
{
    auto results_view = cudf::strings_column_view(strings_column);
    auto d_expected = cudf::test::create_strings_column(h_expected);
    auto expected_view = cudf::strings_column_view(d_expected->view());
    cudf::test::expect_columns_equal(results_view.parent(), d_expected->view());
}

void expect_strings_empty(cudf::column_view strings_column)
{
    EXPECT_EQ(STRING, strings_column.type().id());
    EXPECT_EQ(0,strings_column.size());
    EXPECT_EQ(0,strings_column.null_count());
    EXPECT_EQ(0,strings_column.num_children());
}

}  // namespace test
}  // namespace cudf

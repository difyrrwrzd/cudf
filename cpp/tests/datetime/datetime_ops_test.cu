/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/datetime.hpp>
#include <cudf/utilities/chrono.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>

#include <tests/utilities/type_lists.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/timestamp_utilities.cuh>

#include <tests/utilities/legacy/cudf_test_utils.cuh>


#include <gmock/gmock.h>

template <typename T>
struct DatetimeOpsTest : public cudf::test::BaseFixture {
  hipStream_t stream() { return hipStream_t(0); }
  cudf::size_type size() { return cudf::size_type(10); }
  cudf::data_type type() { return cudf::data_type{cudf::experimental::type_to_id<T>()}; }
};

template <typename Element>
void print_column(cudf::column_view col) {
  print_typed_column<Element>(
    col.data<Element>(),
    (gdf_valid_type*) col.null_mask(),
    col.size(),
    1);
}

// TYPED_TEST_CASE(DatetimeOpsTest, cudf::test::TimestampTypes);

TYPED_TEST_CASE(DatetimeOpsTest, cudf::test::Types<cudf::timestamp_D>);

TYPED_TEST(DatetimeOpsTest, TimestampDurationsMatchPrimitiveRepresentation) {

  using namespace cudf::test;
  using namespace simt::std::chrono;
  using Rep = typename TypeParam::rep;
  using Period = typename TypeParam::period;

  auto start = milliseconds(-2500000000000); // Sat, 11 Oct 1890 19:33:20 GMT
  auto stop_ = milliseconds( 2500000000000); // Mon, 22 Mar 2049 04:26:40 GMT
  auto test_timestamps = generate_timestamps<Rep, Period>(this->size(),
                                                          time_point_ms(start),
                                                          time_point_ms(stop_));

  auto timestamp_col = cudf::make_timestamp_column(this->type(), this->size(),
                                                   cudf::mask_state::UNALLOCATED,
                                                   this->stream(), this->mr());

  cudf::mutable_column_view timestamp_view = *timestamp_col;
  // auto timestamp_dev_view = cudf::column_device_view::create(timestamp_view);

  CUDA_TRY(hipMemcpy(timestamp_view.data<Rep>(),
    thrust::raw_pointer_cast(test_timestamps.data()),
    test_timestamps.size() * sizeof(Rep), hipMemcpyDefault));

  print_column<Rep>(timestamp_view);

  auto expected_years = fixed_width_column_wrapper<int16_t>{1890, 1906, 1922, 1938, 1954, 1970, 1985, 2001, 2017, 2033};
  auto actual_years = *cudf::datetime::extract_year(timestamp_view);
  print_column<int16_t>(expected_years);
  print_column<int16_t>(actual_years);
  expect_columns_equal(expected_years, actual_years);

  // auto expected_months = fixed_width_column_wrapper<int16_t>{9, 7, 5, 3, 1, 0, 10, 8, 6, 4};
  // auto actual_months = *cudf::datetime::extract_month(timestamp_view);
  // expect_columns_equal(expected_months, actual_months);

  // auto expected_days = fixed_width_column_wrapper<int16_t>{11, 16, 20, 24, 26, 1, 5, 9, 14, 18};
  // auto actual_days = *cudf::datetime::extract_day(timestamp_view);
  // expect_columns_equal(expected_days, actual_days);

  // auto expected_weekdays = fixed_width_column_wrapper<int16_t>{6, 4, 2, 0, 5, 4, 2, 0, 5, 3};
  // auto actual_weekdays = *cudf::datetime::extract_weekday(timestamp_view);
  // expect_columns_equal(expected_weekdays, actual_weekdays);

  // auto expected_hours = fixed_width_column_wrapper<int16_t>{19, 20, 21, 22, 23, 0, 0, 1, 2, 3};
  // auto actual_hours = *cudf::datetime::extract_hour(timestamp_view);
  // expect_columns_equal(expected_hours, actual_hours);

  // auto expected_minutes = fixed_width_column_wrapper<int16_t>{33, 26, 20, 13, 6, 0, 53, 46, 40, 33};
  // auto actual_minutes = *cudf::datetime::extract_minute(timestamp_view);
  // expect_columns_equal(expected_minutes, actual_minutes);

  // auto expected_seconds = fixed_width_column_wrapper<int16_t>{20, 40, 0, 20, 40, 0, 20, 40, 0, 20};
  // auto actual_seconds = *cudf::datetime::extract_second(timestamp_view);
  // expect_columns_equal(expected_seconds, actual_seconds);
}

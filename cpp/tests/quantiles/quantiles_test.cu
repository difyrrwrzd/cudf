#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/scalar/scalar.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/quantiles.hpp>
#include <limits>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/scalar_utilities.hpp>
#include <tests/utilities/type_list_utilities.hpp>
#include <tests/utilities/type_lists.hpp>
#include <type_traits>
#include "cudf/scalar/scalar_factories.hpp"
#include "cudf/utilities/error.hpp"
#include "cudf/utilities/legacy/wrapper_types.hpp"
#include "cudf/utilities/traits.hpp"
#include "cudf/wrappers/bool.hpp"
#include "cudf/wrappers/timestamps.hpp"

using std::vector;
using cudf::experimental::bool8;
using cudf::null_order;
using cudf::order;
using cudf::test::expect_scalars_equal;
using cudf::test::fixed_width_column_wrapper;

using q_res = cudf::numeric_scalar<double>;

// template<>
// struct precision<float> {
//     constexpr static double tolerance = 1.0e-7;
// };

// ----- test data -------------------------------------------------------------

namespace testdata {

    struct q_expect
    {
        q_expect(double quantile):
            quantile(quantile),
            higher(0, false), lower(0, false), linear(0, false), midpoint(0, false), nearest(0, false) { }

        q_expect(double quantile,
                 double higher, double lower, double linear, double midpoint, double nearest):
            quantile(quantile),
            higher(higher), lower(lower), linear(linear), midpoint(midpoint), nearest(nearest) { }
    
        double quantile;
        q_res higher;
        q_res lower;
        q_res linear;
        q_res midpoint;
        q_res nearest;
    };

template<typename T>
struct test_case {
    fixed_width_column_wrapper<T> column;
    vector<q_expect> expectations;
    cudf::order_info col_order;
};

// empty

template<typename T>
test_case<T>
empty() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ }),
        {
            q_expect{ -1.0 },
            q_expect{  0.0 },
            q_expect{  0.5 },
            q_expect{  1.0 },
            q_expect{  2.0 }
        }
    };
}

// interpolate_center

template<typename T>
test_case<T>
interpolate_center() {
    auto low = std::numeric_limits<T>::lowest();
    auto max = std::numeric_limits<T>::max();
    auto mid_d = std::is_floating_point<T>::value ? 0.0 : -0.5;

    // int64_t is internally casted to a double, meaning the lerp center point
    // is float-like.
    auto lin_d = std::is_floating_point<T>::value ||
                 std::is_same<T, int64_t>::value ? 0.0 : -0.5;
    auto max_d = static_cast<double>(max);
    auto low_d = static_cast<double>(low);
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ low, max }),
        {
            q_expect{ 0.50, max_d, low_d, lin_d, mid_d, low_d }
        }
    };
}

template<>
test_case<bool8>
interpolate_center() {
    auto low = std::numeric_limits<bool8>::lowest();
    auto max = std::numeric_limits<bool8>::max();
    auto mid_d = 0.5;
    auto low_d = static_cast<double>(low);
    auto max_d = static_cast<double>(max);
    return test_case<bool8> {
        fixed_width_column_wrapper<bool8> ({ low, max }),
        {
            q_expect{ 0.5, max_d, low_d, mid_d, mid_d, low_d }
        }
    };
}

// interpolate_extrema_high

template<typename T>
test_case<T>
interpolate_extrema_high() {
    T max = std::numeric_limits<T>::max();
    T low = max - 2;
    auto low_d = static_cast<double>(low);
    auto max_d = static_cast<double>(max);
    auto exact_d = static_cast<double>(max - 1);
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ low, max }),
        {
            q_expect{ 0.50, max_d, low_d, exact_d, exact_d, low_d }
        }
    };
}

template<>
test_case<bool8>
interpolate_extrema_high<bool8>() {
    return interpolate_center<bool8>();
}

// interpolate_extrema_low

template<typename T>
test_case<T>
interpolate_extrema_low() {
    T lowest = std::numeric_limits<T>::lowest();
    T a = lowest;
    T b = lowest + 2;
    auto a_d = static_cast<double>(a);
    auto b_d = static_cast<double>(b);
    auto exact_d = static_cast<double>(a + 1);
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ a, b }),
        {
            q_expect{ 0.50, b_d, a_d, exact_d, exact_d, a_d }
        }
    };
}

template<>
test_case<bool8>
interpolate_extrema_low<bool8>() {
    return interpolate_center<bool8>();
}

// sorted_ascending_null_before

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
sorted_ascending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 2, 3, 4, 5, 6, 7, 8, 9 },
                                       { 0, 0, 0, 0, 0, 1, 1, 1, 1 }),
        {
            q_expect{ 0.00, 6, 6, 6, 6, 6 },
            q_expect{ 0.75, 9, 8, 8.25, 8.5, 8 },
            q_expect{ 1.00, 9, 9, 9, 9, 9 }
        },
        { true, cudf::order::ASCENDING, cudf::null_order::BEFORE }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
sorted_ascending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 2, 3, 4, 5, 6, 7, 8, 9 },
                                       { 0, 0, 0, 0, 0, 1, 1, 1, 1 }),
        {
            q_expect{ 0.00, 6, 6, 6, 6, 6 },
            q_expect{ 0.50, 8, 7, 7.5, 7.5, 8 },
            q_expect{ 1.00, 9, 9, 9, 9, 9 }
        },
        { true, cudf::order::ASCENDING, cudf::null_order::BEFORE }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
sorted_ascending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 0, 1, },
                                       { 0, 1, 1, }),
        {
            q_expect{ 0.00, 0, 0, 0, 0, 0 },
            q_expect{ 0.50, 1, 0, 0.5, 0.5, 0 },
            q_expect{ 1.50, 1, 1, 1, 1, 1 }
        },
        { true, cudf::order::ASCENDING, cudf::null_order::BEFORE }
    };
}

// sorted_descending_null_after

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
sorted_descending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 9, 8, 7, 6, 5, 4, 3, 2, 1 },
                                       { 1, 1, 1, 1, 0, 0, 0, 0, 0 }),
        {
            q_expect{ 0.00, 6, 6, 6, 6, 6 },
            q_expect{ 0.75, 9, 8, 8.25, 8.5, 8 },
            q_expect{ 1.00, 9, 9, 9, 9, 9 }
        },
        { true, cudf::order::DESCENDING, cudf::null_order::AFTER }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
sorted_descending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 9, 8, 7, 6, 5, 4, 3, 2, 1 },
                                       { 1, 1, 1, 1, 0, 0, 0, 0, 0 }),
        {
            q_expect{ 0.00, 6, 6, 6, 6, 6 },
            q_expect{ 0.50, 8, 7, 7.5, 7.5, 8 },
            q_expect{ 1.00, 9, 9, 9, 9, 9 }
        },
        { true, cudf::order::DESCENDING, cudf::null_order::AFTER }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
sorted_descending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 0, 1, },
                                       { 1, 1, 0, }),
        {
            q_expect{ 0.50, 1, 0, 0.5, 0.5, 0 }
        },
        { true, cudf::order::DESCENDING, cudf::null_order::AFTER }
    };
}

// single

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
single() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 7.309999942779541 }),
        {
            q_expect{ -1.0, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, },
            q_expect{  0.0, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, },
            q_expect{  1.0, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, },
        }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
single() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1 }),
        {
            q_expect{ 0.7, 1, 1, 1, 1, 1 }
        }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
single() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1 }),
        {
            q_expect{ 0.7, 1.0, 1.0, 1.0, 1.0, 1.0 }
        }
    };
}

// all_invalid

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
all_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6.8, 0.15, 3.4, 4.17, 2.13, 1.11, -1.01, 0.8, 5.7 },
                                       { 0,      0,   0,    0,    0,    0,     0,   0,   0 }),
        {
            q_expect{ -1.0 },
            q_expect{  0.0 },
            q_expect{  0.5 },
            q_expect{  1.0 },
            q_expect{  2.0 }
        }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
all_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6, 0, 3, 4, 2, 1, -1, 1, 6 },
                                       { 0, 0, 0, 0, 0, 0,  0, 0, 0}),
        {
            q_expect{ 0.7 }
        }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
all_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 0, 1, 1, 0, 1, 0, 1, 1 }, 
                                       { 0, 0, 0, 0, 0, 0, 0, 0, 0 }),
        {
            q_expect{ 0.7 }
        }
    };
}

// some invalid

template<typename T>
std::enable_if_t<std::is_same<T, double>::value, test_case<T>>
some_invalid() {
    T high = 0.16;
    T low = -1.024;
    T mid = -0.432;
    T lin = -0.432;
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6.8, high, 3.4, 4.17, 2.13, 1.11, low, 0.8, 5.7 },
                                       { 0,      1,   0,    0,    0,    0,   1,   0,   0 }),
        {
            q_expect{ -1.0, low,  low,  low,  low,  low },
            q_expect{  0.0, low,  low,  low,  low,  low },
            q_expect{  0.5, high, low,  lin,  mid,  low },
            q_expect{  1.0, high, high, high, high, high },
            q_expect{  2.0, high, high, high, high, high }
        }
    };
}

template<typename T>
std::enable_if_t<std::is_same<T, float>::value, test_case<T>>
some_invalid() {
    T high = 0.16;
    T low = -1.024;
    double mid = -0.43200002610683441;
    double lin = -0.43200002610683441;
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6.8, high, 3.4, 4.17, 2.13, 1.11, low, 0.8, 5.7 },
                                       { 0,      1,   0,    0,    0,    0,   1,   0,   0 }),
        {
            q_expect{ -1.0, low,  low,  low,  low,  low },
            q_expect{  0.0, low,  low,  low,  low,  low },
            q_expect{  0.5, high, low,  lin,  mid,  low },
            q_expect{  1.0, high, high, high, high, high },
            q_expect{  2.0, high, high, high, high, high }
        }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
some_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6, 0, 3, 4, 2, 1, -1, 1, 6 },
                                       { 0, 0, 1, 0, 0, 0,  0, 0, 1}),
        {
            q_expect{ 0.0, 3.0, 3.0, 3.0, 3.0, 3.0 },
            q_expect{ 0.5, 6.0, 3.0, 4.5, 4.5, 3.0 },
            q_expect{ 1.0, 6.0, 6.0, 6.0, 6.0, 6.0 }
        }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
some_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 0, 1, 1, 0, 1, 0, 1, 1 },
                                       { 0, 0, 1, 0, 1, 0, 0, 0, 0}),
        {
            q_expect{ 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 },
            q_expect{ 0.5, 1.0, 0.0, 0.5, 0.5, 0.0 },
            q_expect{ 1.0, 1.0, 1.0, 1.0, 1.0, 1.0 }
        }
    };
}

// unsorted

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
unsorted() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6.8, 0.15, 3.4, 4.17, 2.13, 1.11, -1.00, 0.8, 5.7 }),
        {
            q_expect{ 0.0, -1.00, -1.00, -1.00, -1.00, -1.00 },
        }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
unsorted() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6, 0, 3, 4, 2, 1, -1, 1, 6 }),
        {
            q_expect{ 0.0, -1, -1, -1, -1, -1 }
        }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
unsorted() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 0, 0, 1, 1, 0, 1, 1, 0, 1 }),
        {
            q_expect{ 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,}
        }
    };
}

} // namespace testdata

// =============================================================================
// ----- helper functions ------------------------------------------------------

template<typename T>
void test(testdata::test_case<T> test_case) {
    using namespace cudf::experimental;

    cudf::table_view in_table { { test_case.column } };

    for (auto & expected : test_case.expectations) {

        auto actual_higher = quantiles(in_table, expected.quantile, interpolation::HIGHER, { test_case.col_order });
        expect_scalars_equal(expected.higher, *actual_higher[0]);

        auto actual_lower = quantiles(in_table, expected.quantile, interpolation::LOWER, { test_case.col_order });
        expect_scalars_equal(expected.lower, *actual_lower[0]);

        auto actual_linear = quantiles(in_table, expected.quantile, interpolation::LINEAR, { test_case.col_order });
        expect_scalars_equal(expected.linear, *actual_linear[0]);

        auto actual_midpoint = quantiles(in_table, expected.quantile, interpolation::MIDPOINT, { test_case.col_order });
        expect_scalars_equal(expected.midpoint, *actual_midpoint[0]);

        auto actual_nearest = quantiles(in_table, expected.quantile, interpolation::NEAREST, { test_case.col_order });
        expect_scalars_equal(expected.nearest, *actual_nearest[0]);
    }
}

// =============================================================================
// ----- tests -----------------------------------------------------------------

template <typename T>
struct QuantilesTest : public cudf::test::BaseFixture {
};

using TestTypes = cudf::test::NumericTypes;
// using TestTypes = cudf::test::Types<int8_t>;
// using TestTypes = cudf::test::Types<int8_t, int16_t, int32_t, int64_t>;
// using TestTypes = cudf::test::Types<float, double>;
// using TestTypes = cudf::test::Types<cudf::experimental::bool8>;

TYPED_TEST_CASE(QuantilesTest, TestTypes);

TYPED_TEST(QuantilesTest, TestEmpty)
{
    test(testdata::empty<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestSingle)
{
    test(testdata::single<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestSomeElementsInvalid)
{
    test(testdata::some_invalid<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestAllElementsInvalid)
{
    test(testdata::all_invalid<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestUnsorted)
{
    test(testdata::unsorted<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestInterpolateCenter)
{
    test(testdata::interpolate_center<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestInterpolateExtremaHigh)
{
    test(testdata::interpolate_extrema_high<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestInterpolateExtremaLow)
{
    test(testdata::interpolate_extrema_low<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestSortedAscendingNullBefore)
{
    test(testdata::sorted_ascending_null_after<TypeParam>());
}


TYPED_TEST(QuantilesTest, TestSortedDescendingNullAfter)
{
    test(testdata::sorted_descending_null_after<TypeParam>());
}

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* Proof the concept for iterator driven aggregations to reuse the logic

   The concepts:
   1. computes the aggregation by given iterators
   2. computes by using cub and thrust with same function parameters
   3. accepts nulls and group_by with same function parameters

    CUB:  https://nvlabs.github.io/cub/structcub_1_1_device_reduce.html#aa4adabeb841b852a7a5ecf4f99a2daeb
    Thrust: https://thrust.github.io/doc/group__reductions.html#ga43eea9a000f912716189687306884fc7
*/

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <bitset>
#include <cstdint>
#include <iostream>
#include <numeric>
#include <iterator>

#include <utilities/cudf_utils.h> // need for CUDA_HOST_DEVICE_CALLABLE
#include <utilities/device_atomics.cuh> // need for device operators.

#include <tests/utilities/column_wrapper.cuh>
#include <tests/utilities/cudf_test_fixtures.h>

#include <cub/device/device_reduce.cuh>
#include <thrust/device_vector.h>
#include <thrust/transform.h>

void gen_nullbitmap(std::vector<gdf_valid_type>& v, std::vector<bool>& host_bools)
{
    int length = host_bools.size();
    auto n_bytes = gdf_valid_allocation_size(length);

    v.resize(n_bytes);
    // TODO: generic
    for(int i=0; i<length; i++)
    {
        int pos = i/8;
        int bit_index = i%8;
        if( bit_index == 0)v[pos] = 0;
        if( host_bools[i] )v[pos] += (1 << bit_index);
    }
}


template <typename T>
class IteratorWithNulls : public std::iterator<std::random_access_iterator_tag, T>
{
public:
//    using difference_type = std::iterator<std::random_access_iterator_tag, T>::difference_type;
    using difference_type = gdf_size_type;

    // Discussion: std::iterator is deprecated in C++17
    CUDA_HOST_DEVICE_CALLABLE
    IteratorWithNulls(const T* _data, const gdf_valid_type* _valid, T _identity)
    : data(_data), valid(_valid), identity(_identity), index(0)
    {};

    CUDA_HOST_DEVICE_CALLABLE
    IteratorWithNulls(const IteratorWithNulls& ref)
    : data(ref.data), valid(ref.valid), identity(ref.identity), index(ref.index)
    {};

    CUDA_HOST_DEVICE_CALLABLE
    T operator*() const
    {
//        return (gdf_is_valid(valid, index))? data[index] : identity;
        T val = (gdf_is_valid(valid, index))? data[index] : identity;
        printf("val(%d, %d) = %d\n", data[index], gdf_is_valid(valid, index), val);
        return val;

};

    CUDA_HOST_DEVICE_CALLABLE
    T operator[](const difference_type length) const
    {
        gdf_size_type id = index + length;
        return (gdf_is_valid(valid, id))? data[id] : identity;
    };

    CUDA_HOST_DEVICE_CALLABLE
    bool operator==(const IteratorWithNulls& others)
    {//printf("cmp (%d, %d)\n", index, others.index);
        return index == others.index; };

    CUDA_HOST_DEVICE_CALLABLE
    bool operator!=(const IteratorWithNulls& others)
    { return !(*this == others); };

    CUDA_HOST_DEVICE_CALLABLE
    IteratorWithNulls& operator+=(const difference_type length)
    { //printf("operator+ (%d, %d)\n", index, length);
        index += length;
        return *this; };

    CUDA_HOST_DEVICE_CALLABLE
    IteratorWithNulls& operator-=(const difference_type length)
    {return (*this -= length ); };

    CUDA_HOST_DEVICE_CALLABLE
    IteratorWithNulls& operator++() { return (*this += 1);};

    CUDA_HOST_DEVICE_CALLABLE
    IteratorWithNulls operator++(int) {IteratorWithNulls retval = *this; ++(*this); return retval;}

    CUDA_HOST_DEVICE_CALLABLE
    IteratorWithNulls operator+(const difference_type length) {
        IteratorWithNulls tmp(*this);
        return (tmp += length);
    };


    CUDA_HOST_DEVICE_CALLABLE
    difference_type operator-(const IteratorWithNulls& others)
    {
        return (index - others.index );
    };

protected:
    const T *data;
    const gdf_valid_type *valid;
    const T identity;

    gdf_size_type index; // variables
};




template <typename T>
struct IteratorTest : public GdfTest
{
    // iterator test case which uses cub
    template <typename InputIterator, typename T_output>
    void iterator_test_cub(T_output expected, InputIterator d_in, int num_items)
    {
        T init = T{0};
        thrust::device_vector<T> dev_result(1);

        void     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0;

        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, dev_result.begin(), num_items,
            cudf::DeviceSum{}, init);
        // Allocate temporary storage
        RMM_TRY(RMM_ALLOC(&d_temp_storage, temp_storage_bytes, 0));

        // Run reduction
        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, dev_result.begin(), num_items,
            cudf::DeviceSum{}, init);

        evaluate(expected, dev_result, "cub test");
    }

    // iterator test case which uses thrust
    template <typename InputIterator, typename T_output>
    void iterator_test_thrust(T_output expected, InputIterator d_in, int num_items)
    {
        T init = T{0};

        InputIterator d_in_last =  d_in + num_items;

        T result = thrust::reduce(d_in, d_in_last, init, cudf::DeviceSum{});

        EXPECT_EQ(expected, result) << "thrust test";
    }

    void evaluate(T expected, thrust::device_vector<T> &dev_result, const char* msg=nullptr)
    {
        thrust::host_vector<T>  hos_result(dev_result);

        EXPECT_EQ(expected, dev_result[0]) << msg ;
        std::cout << "expected <" << msg << "> = " << expected << std::endl;
    }
};

using TestingTypes = ::testing::Types<
    int32_t
>;

TYPED_TEST_CASE(IteratorTest, TestingTypes);


// tests for non-null iterator (pointer of device array)
TYPED_TEST(IteratorTest, non_null_iterator)
{
    using T = int32_t;
    std::vector<T> hos_array({0, 6, 0, -14, 13, 64, -13, -20, 45});
    thrust::device_vector<T> dev_array(hos_array);

    T expected_value = std::accumulate(hos_array.begin(), hos_array.end(), T{0});

    auto it_dev = dev_array.begin();
    this->iterator_test_cub(expected_value, it_dev, dev_array.size());
    this->iterator_test_thrust(expected_value, it_dev, dev_array.size());
}

// tests for null iterator (column with null bitmap)
TYPED_TEST(IteratorTest, null_iterator)
{
    using T = int32_t;
    T init = T{0};

    std::vector<T> hos_array({0, 6, 0, -14, 13, 64, -13, -20, 45});
    thrust::device_vector<T> dev_array(hos_array);

    std::vector<bool> host_bools({1, 1, 0, 1, 1, 1, 0, 1, 1});
    std::vector<gdf_valid_type> host_nulls;
    gen_nullbitmap(host_nulls, host_bools);
    thrust::device_vector<gdf_valid_type> dev_nulls(host_nulls);

    EXPECT_EQ(hos_array.size(), host_bools.size());

    std::vector<T> replaced_array(hos_array.size());
    std::transform(hos_array.begin(), hos_array.end(), host_bools.begin(),
        replaced_array.begin(), [&](T x, bool b) { return (b)? x : init; } );
    T expected_value = std::accumulate(replaced_array.begin(), replaced_array.end(), init);
    std::cout << "expected <null_iterator> = " << expected_value << std::endl;

    if(0)
    {  // check host side `IteratorWithNulls`.
        IteratorWithNulls<T> it_hos(hos_array.data(), host_nulls.data(), init);
        T expected_value_host = std::accumulate(it_hos, it_hos + hos_array.size(), T{0});
        EXPECT_EQ(expected_value, expected_value_host) << "CPU iterator test";
    }

    // create device side `IteratorWithNulls`.
    IteratorWithNulls<T> it_dev(
        static_cast<const T*>( dev_array.data().get() ),
        static_cast<const gdf_valid_type*>( dev_nulls.data().get() ),
        init);

    this->iterator_test_cub(expected_value, it_dev, dev_array.size());
//    this->iterator_test_thrust(expected_value, it_dev, dev_array.size());

}

// tests for group_by iterator
TYPED_TEST(IteratorTest, group_by_iterator)
{
    // TBD.
}


// tests for group_by iterator
TYPED_TEST(IteratorTest, group_by_iterator_null)
{
    // Discussion: how to do if all of values are nulls ?
    // maybe need to exclude null values first ? (it also gives `count` of a column value in the group)

    // TBD.
}
#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* Proof the concept of iterator driven aggregations to reuse the logic

   The concepts:
   1. computes the aggregation by given iterators
   2. computes by using cub and thrust with same function parameters
   3. accepts nulls and group_by with same function parameters

    CUB reduction:  https://nvlabs.github.io/cub/structcub_1_1_device_reduce.html#aa4adabeb841b852a7a5ecf4f99a2daeb
    Thrust reduction: https://thrust.github.io/doc/group__reductions.html#ga43eea9a000f912716189687306884fc7

    Thrust iterators: https://thrust.github.io/doc/group__fancyiterator.html
*/

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <bitset>
#include <cstdint>
#include <iostream>
#include <numeric>

#include <reduction.hpp>

#include <tests/utilities/cudf_test_fixtures.h>
#include <tests/utilities/column_wrapper.cuh>
#include <tests/utilities/scalar_wrapper.cuh>

#include <iterator/iterator.cuh>    // include iterator header

#include <cub/device/device_reduce.cuh>
#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/iterator_adaptor.h>


// ---------------------------------------------------------------------------
// __host__ option is required for thrust::host operation
struct HostDeviceSum {
    template<typename T>
    __device__ __host__
    T operator() (const T &lhs, const T &rhs) {
        return lhs + rhs;
    }

    template<typename T>
    static constexpr T identity() { return T{0}; }
};


template <typename T>
T random_int(T min, T max)
{
  static unsigned seed = 13377331;
  static std::mt19937 engine{seed};
  static std::uniform_int_distribution<T> uniform{min, max};

  return uniform(engine);
}

bool random_bool()
{
  static unsigned seed = 13377331;
  static std::mt19937 engine{seed};
  static std::uniform_int_distribution<int> uniform{0, 1};

  return static_cast<bool>( uniform(engine) );
}


// ---------------------------------------------------------------------------


template <typename T>
struct IteratorTest : public GdfTest
{
    // iterator test case which uses cub
    template <typename InputIterator, typename T_output>
    void iterator_test_cub(T_output expected, InputIterator d_in, int num_items)
    {
        T_output init{0};
        thrust::device_vector<T_output> dev_result(1, init);

        void     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0;

        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, dev_result.begin(), num_items,
            HostDeviceSum{}, init);
        // Allocate temporary storage
        RMM_TRY(RMM_ALLOC(&d_temp_storage, temp_storage_bytes, 0));

        // Run reduction
        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, dev_result.begin(), num_items,
            HostDeviceSum{}, init);

        evaluate(expected, dev_result, "cub test");
    }

    // iterator test case which uses thrust
    template <typename InputIterator, typename T_output>
    void iterator_test_thrust(T_output expected, InputIterator d_in, int num_items)
    {
        T_output init{0};
        InputIterator d_in_last =  d_in + num_items;
        EXPECT_EQ( thrust::distance(d_in, d_in_last), num_items);

        T_output result = thrust::reduce(thrust::device, d_in, d_in_last, init, HostDeviceSum{});
        EXPECT_EQ(expected, result) << "thrust test";
    }

    // iterator test case which uses thrust
    template <typename InputIterator, typename T_output>
    void iterator_test_thrust_host(T_output expected, InputIterator d_in, int num_items)
    {
        T_output init{0};
        InputIterator d_in_last =  d_in + num_items;
        EXPECT_EQ( thrust::distance(d_in, d_in_last), num_items);

        T_output result = thrust::reduce(thrust::host, d_in, d_in_last, init, HostDeviceSum{});
        EXPECT_EQ(expected, result) << "thrust host test";
    }

    template <typename T_output>
    void evaluate(T_output expected, thrust::device_vector<T_output> &dev_result, const char* msg=nullptr)
    {
        thrust::host_vector<T_output>  hos_result(dev_result);

        EXPECT_EQ(expected, hos_result[0]) << msg ;
        std::cout << "Done: expected <" << msg << "> = " << hos_result[0] << std::endl;
    }
};

using TestingTypes = ::testing::Types<
    int32_t
>;

TYPED_TEST_CASE(IteratorTest, TestingTypes);


// tests for non-null iterator (pointer of device array)
TYPED_TEST(IteratorTest, non_null_iterator)
{
    using T = int32_t;
    std::vector<T> hos_array({0, 6, 0, -14, 13, 64, -13, -20, 45});
    thrust::device_vector<T> dev_array(hos_array);

    // calculate the expected value by CPU.
    T expected_value = std::accumulate(hos_array.begin(), hos_array.end(), T{0});

    // driven by iterator as a pointer of device array.
    auto it_dev = dev_array.begin();
    this->iterator_test_cub(expected_value, it_dev, dev_array.size());
    this->iterator_test_thrust(expected_value, it_dev, dev_array.size());

    this->iterator_test_thrust_host(expected_value, hos_array.begin(), hos_array.size());
}



/* tests for null input iterator (column with null bitmap)
   Actually, we can use cub for reduction with nulls without creating custom kernel or multiple steps.
   we may accelarate the reduction for a column using cub
*/
TYPED_TEST(IteratorTest, null_iterator)
{
    using T = int32_t;
    T init = T{0};
    std::vector<bool> host_bools({1, 1, 0, 1, 1, 1, 0, 1, 1});

    // create a column with bool vector
    cudf::test::column_wrapper<T> w_col({0, 6, 0, -14, 13, 64, -13, -20, 45},
        [&](gdf_index_type row) { return host_bools[row]; });

    // copy back data and valid arrays
    auto hos = w_col.to_host();

    // calculate the expected value by CPU.
    std::vector<T> replaced_array(w_col.size());
    std::transform(std::get<0>(hos).begin(), std::get<0>(hos).end(), host_bools.begin(),
        replaced_array.begin(), [&](T x, bool b) { return (b)? x : init; } );
    T expected_value = std::accumulate(replaced_array.begin(), replaced_array.end(), init);
    std::cout << "expected <null_iterator> = " << expected_value << std::endl;

    // CPU test
    auto it_hos = cudf::make_iterator_with_nulls(std::get<0>(hos).data(), std::get<1>(hos).data(), init);
    this->iterator_test_thrust_host(expected_value, it_hos, w_col.size());

    // GPU test
    auto it_dev = cudf::make_iterator_with_nulls(static_cast<T*>( w_col.get()->data ), w_col.get()->valid, init);
    this->iterator_test_thrust(expected_value, it_dev, w_col.size());
    this->iterator_test_cub(expected_value, it_dev, w_col.size());

    std::cout << "test done." << std::endl;
}

/* tests up cast reduction with null iterator
*/
TYPED_TEST(IteratorTest, null_iterator_upcast)
{
    const int column_size{1000};
    using T = int8_t;
    using T_upcast = int64_t;
    T init{0};

    std::vector<bool> host_bools(column_size);
    std::generate(host_bools.begin(), host_bools.end(),
        []() { return static_cast<bool>( random_bool() ); } );

    cudf::test::column_wrapper<T> w_col(
        column_size,
        [](gdf_index_type row) { return T{random_int<T>(-128, 127)}; },
        [&](gdf_index_type row) { return host_bools[row]; } );


    // copy back data and valid arrays
    auto hos = w_col.to_host();

    // calculate the expected value by CPU.
    std::vector<T> replaced_array(w_col.size());
    std::transform(std::get<0>(hos).begin(), std::get<0>(hos).end(), host_bools.begin(),
        replaced_array.begin(), [&](T x, bool b) { return (b)? x : init; } );
    T_upcast expected_value = std::accumulate(replaced_array.begin(), replaced_array.end(), T_upcast{0});
    std::cout << "expected <null_iterator> = " << expected_value << std::endl;

    // CPU test
    auto it_hos = cudf::make_iterator_with_nulls<T, T_upcast>(std::get<0>(hos).data(), std::get<1>(hos).data(), T{0});
    this->iterator_test_thrust_host(expected_value, it_hos, w_col.size());

    // GPU test
    auto it_dev = cudf::make_iterator_with_nulls<T, T_upcast>(static_cast<T*>( w_col.get()->data ), w_col.get()->valid, T{0});
    this->iterator_test_thrust(expected_value, it_dev, w_col.size());
    this->iterator_test_cub(expected_value, it_dev, w_col.size());

    std::cout << "test done." << std::endl;
}



/* tests for square input iterator
*/
TYPED_TEST(IteratorTest, null_iterator_square)
{
    const int column_size{1000};
    using T = int8_t;
    using T_upcast = int64_t;
    T init{0};

    std::vector<bool> host_bools(column_size);
    std::generate(host_bools.begin(), host_bools.end(),
        []() { return static_cast<bool>( random_bool() ); } );

    cudf::test::column_wrapper<T> w_col(
        column_size,
        [](gdf_index_type row) { return T{random_int<T>(-128, 127)}; },
        [&](gdf_index_type row) { return host_bools[row]; } );

    // copy back data and valid arrays
    auto hos = w_col.to_host();

    // calculate the expected value by CPU.
    std::vector<T_upcast> replaced_array(w_col.size());
    std::transform(std::get<0>(hos).begin(), std::get<0>(hos).end(), host_bools.begin(),
        replaced_array.begin(), [&](T x, bool b) { return (b)?  x*x : init; } );
    T_upcast expected_value = std::accumulate(replaced_array.begin(), replaced_array.end(), T_upcast{0});
    std::cout << "expected <null_iterator> = " << expected_value << std::endl;

    // CPU test
    auto it_hos = cudf::make_iterator_with_nulls<T, T_upcast, ColumnOutputSquared<T_upcast>>
        (std::get<0>(hos).data(), std::get<1>(hos).data(), T{0});
    this->iterator_test_thrust_host(expected_value, it_hos, w_col.size());

    // GPU test
    auto it_dev = cudf::make_iterator_with_nulls<T, T_upcast, ColumnOutputSquared<T_upcast>>
        (static_cast<T*>( w_col.get()->data ), w_col.get()->valid, T{0});
    this->iterator_test_thrust(expected_value, it_dev, w_col.size());
    this->iterator_test_cub(expected_value, it_dev, w_col.size());

    std::cout << "test done." << std::endl;
}



/*
    tests for indexed access

    this was used by old implementation of group_by.

    This won't be used with the newer implementation
     (a.k.a. Single pass, distributive groupby https://github.com/rapidsai/cudf/pull/1478)
    distributive groupby uses atomic operation to accumulate.

    For group_by.cumsum() (scan base group_by) may not be single pass scan.
    There is a possiblity that this process may be used for group_by.cumsum().
*/
TYPED_TEST(IteratorTest, indexed_iterator)
{
    using T = int32_t;
    using T_index = gdf_index_type;

    std::vector<T> hos_array({0, 6, 0, -14, 13, 64, -13, -20, 45});
    thrust::device_vector<T> dev_array(hos_array);

    std::vector<T_index> hos_indices({0, 1, 3, 5}); // sorted indices belongs to a group
    thrust::device_vector<T_index> dev_indices(hos_indices);

    // calculate the expected value by CPU.
    T expected_value = std::accumulate(hos_indices.begin(), hos_indices.end(), T{0},
        [&](T acc, T_index id){ return (acc + hos_array[id]); } );
    std::cout << "expected <group_by_iterator> = " << expected_value << std::endl;


    using T_input = ColumnInput<ColumnOutputSingle<T>, T, false>;

    // pass `dev_indices` as base iterator of `column_input_iterator`.
    T_input col(dev_array.data().get());
    column_input_iterator<T, T_input, T_index*> it_dev(col, dev_indices.data().get());

    // reduction using thrust
    this->iterator_test_thrust(expected_value, it_dev, dev_indices.size());
    // reduction using cub
    this->iterator_test_cub(expected_value, it_dev, dev_indices.size());

    // pass `dev_indices` as base iterator of `column_input_iterator`.
    T_input col_hos(hos_array.data());
    column_input_iterator<T, T_input, T_index*> it_host(col_hos, hos_indices.data());
    this->iterator_test_thrust_host(expected_value, it_host, hos_indices.size());
}

TYPED_TEST(IteratorTest, large_size_reduction)
{
    using T = int32_t;

    const int column_size{1000000};
    const T init{0};

    std::vector<bool> host_bools(column_size);
    std::generate(host_bools.begin(), host_bools.end(),
        []() { return static_cast<bool>( random_bool() ); } );

    cudf::test::column_wrapper<TypeParam> w_col(
        column_size,
        [](gdf_index_type row) { return T{random_int(-128, 128)}; },
        [&](gdf_index_type row) { return host_bools[row]; } );

    // copy back data and valid arrays
    auto hos = w_col.to_host();

    // calculate by cudf::reduction
    std::vector<T> replaced_array(w_col.size());
    std::transform(std::get<0>(hos).begin(), std::get<0>(hos).end(), host_bools.begin(),
        replaced_array.begin(), [&](T x, bool b) { return (b)? x : init; } );
    T expected_value = std::accumulate(replaced_array.begin(), replaced_array.end(), init);
    std::cout << "expected <null_iterator> = " << expected_value << std::endl;

    // CPU test
    auto it_hos = cudf::make_iterator_with_nulls(std::get<0>(hos).data(), std::get<1>(hos).data(), init);
    this->iterator_test_thrust_host(expected_value, it_hos, w_col.size());

    // GPU test
    auto it_dev = cudf::make_iterator_with_nulls(static_cast<T*>( w_col.get()->data ), w_col.get()->valid, init);
    this->iterator_test_thrust(expected_value, it_dev, w_col.size());
    this->iterator_test_cub(expected_value, it_dev, w_col.size());


    // compare with cudf::reduction
    cudf::test::scalar_wrapper<T> result =
        cudf::reduction(w_col, GDF_REDUCTION_SUM, GDF_INT32);

    EXPECT_EQ(expected_value, result.value());

}


// test for mixed output value using `ColumnOutputMix`
// it wpuld be useful for `var`, `std` operation
TYPED_TEST(IteratorTest, mixed_output)
{
    using T = int32_t;
    using T_upcast = int64_t;

    const int column_size{5000};
    const T init{0};

    std::vector<bool> host_bools(column_size);
    std::generate(host_bools.begin(), host_bools.end(),
        []() { return static_cast<bool>( random_bool() ); } );

    cudf::test::column_wrapper<TypeParam> w_col(
        column_size,
        [](gdf_index_type row) { return T{random_int(-128, 128)}; },
        [&](gdf_index_type row) { return host_bools[row]; } );

    // copy back data and valid arrays
    auto hos = w_col.to_host();

    // calculate expected values by CPU
    ColumnOutputMixed<T_upcast> expected_value;

    expected_value.count = w_col.size() - w_col.null_count();

    std::vector<T> replaced_array(w_col.size());
    std::transform(std::get<0>(hos).begin(), std::get<0>(hos).end(), host_bools.begin(),
        replaced_array.begin(), [&](T x, bool b) { return (b)? x : init; } );

    expected_value.count = w_col.size() - w_col.null_count();
    expected_value.value = std::accumulate(replaced_array.begin(), replaced_array.end(), T_upcast{0});
    expected_value.value_squared = std::accumulate(replaced_array.begin(), replaced_array.end(), T_upcast{0},
        [](T acc, T i) { return acc + i * i; });

    std::cout << "expected <mixed_output> = " << expected_value << std::endl;

    // CPU test
    auto it_hos = cudf::make_iterator_with_nulls<T, ColumnOutputMixed<T_upcast>, ColumnOutputMixed<T_upcast>>
        (std::get<0>(hos).data(), std::get<1>(hos).data(), T{0});
    this->iterator_test_thrust_host(expected_value, it_hos, w_col.size());

    // GPU test
    auto it_dev = cudf::make_iterator_with_nulls<T, ColumnOutputMixed<T_upcast>, ColumnOutputMixed<T_upcast>>
        (static_cast<T*>( w_col.get()->data ), w_col.get()->valid, init);
    this->iterator_test_thrust(expected_value, it_dev, w_col.size());
    this->iterator_test_cub(expected_value, it_dev, w_col.size());

    { // ColumnOutputMixedNoCount test
        using T_helper = ColumnOutputMixed<T_upcast, false>;

        T_helper expected_value_no_count;
        expected_value_no_count.value = expected_value.value;
        expected_value_no_count.value_squared = expected_value.value_squared;
        expected_value_no_count.count = 0;


        auto it_hos = cudf::make_iterator_with_nulls<T, T_helper, T_helper>
            (std::get<0>(hos).data(), std::get<1>(hos).data(), T{0});
        this->iterator_test_thrust_host(expected_value_no_count, it_hos, w_col.size());

        // GPU test
        auto it_dev = cudf::make_iterator_with_nulls<T, T_helper, T_helper>
            (static_cast<T*>( w_col.get()->data ), w_col.get()->valid, init);
        this->iterator_test_thrust(expected_value_no_count, it_dev, w_col.size());
        this->iterator_test_cub(expected_value_no_count, it_dev, w_col.size());
    }
}


#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/cudf.h>
#include <tests/utilities/base_fixture.hpp>
#include <cudf/unary.hpp>
#include <cudf/column/column_factories.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <tests/utilities/type_lists.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <cudf/legacy/interop.hpp>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <vector>


template <typename T>
cudf::test::fixed_width_column_wrapper<T> create_fixed_columns(cudf::size_type start, cudf::size_type size, bool nullable) {
    auto iter = cudf::test::make_counting_transform_iterator(start, [](auto i) { return T(i);});

    if(not nullable) {
        return cudf::test::fixed_width_column_wrapper<T> (iter, iter + size);
    } else {
        auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i%2==0? true:false; });
        return  cudf::test::fixed_width_column_wrapper<T> (iter, iter + size, valids);
    }

}

template <typename T>
cudf::test::fixed_width_column_wrapper<T> create_expected_columns(cudf::size_type size, bool nullable, bool nulls_to_be) {

    if(not nullable) {
        auto iter = cudf::test::make_counting_transform_iterator(0, [nulls_to_be](auto i) { return not nulls_to_be;});
        return cudf::test::fixed_width_column_wrapper<T> (iter, iter + size);
    } else {
        auto iter = cudf::test::make_counting_transform_iterator(0, [nulls_to_be](auto i) { return i%2==0? not nulls_to_be: nulls_to_be; });
        return cudf::test::fixed_width_column_wrapper<T> (iter, iter + size);
    }
}

template <typename T>
struct cudf_logical_test : public cudf::test::BaseFixture {};

TYPED_TEST_CASE(cudf_logical_test, cudf::test::NumericTypes);

TYPED_TEST(cudf_logical_test, LogicalNot)
{
    cudf::size_type colSize = 1000;
    std::vector<TypeParam> h_input_v(colSize, false);
    std::vector<cudf::experimental::bool8> h_expect_v(colSize);

    std::transform(
        std::cbegin(h_input_v),
        std::cend(h_input_v),
        std::begin(h_expect_v),
        [] (TypeParam e) -> cudf::bool8 {
            return static_cast<cudf::bool8>(!e);
        });

    cudf::test::fixed_width_column_wrapper<TypeParam>                 input    (std::cbegin(h_input_v),  std::cend(h_input_v));
    cudf::test::fixed_width_column_wrapper<cudf::experimental::bool8> expected (std::cbegin(h_expect_v), std::cend(h_expect_v));

    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::NOT);

    cudf::test::expect_columns_equal(expected, output->view());
}

TYPED_TEST(cudf_logical_test, SimpleLogicalNot)
{
    cudf::test::fixed_width_column_wrapper<TypeParam>                 input    {{ true,  true,  true,  true  }};
    cudf::test::fixed_width_column_wrapper<cudf::experimental::bool8> expected {{ false, false, false, false }};
    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::NOT);
    cudf::test::expect_columns_equal(expected, output->view());
}

template <typename T>
struct cudf_math_test : public cudf::test::BaseFixture {};

TYPED_TEST_CASE(cudf_math_test, cudf::test::NumericTypes);

TYPED_TEST(cudf_math_test, ABS)
{
    using T = TypeParam;

    cudf::size_type colSize = 1000;
    std::vector<T> h_input_v(colSize);
    std::vector<T> h_expect_v(colSize);

    std::iota(
        std::begin(h_input_v),
        std::end(h_input_v),
        -1 * colSize);

    std::transform(
        std::cbegin(h_input_v),
        std::cend(h_input_v),
        std::begin(h_expect_v),
        [] (auto e) { return std::abs(e); });

    cudf::test::fixed_width_column_wrapper<T> input    (std::cbegin(h_input_v),  std::cend(h_input_v));
    cudf::test::fixed_width_column_wrapper<T> expected (std::cbegin(h_expect_v), std::cend(h_expect_v));

    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::ABS);

    cudf::test::expect_columns_equal(expected, output->view());
}

TYPED_TEST(cudf_math_test, SQRT)
{
    using T = TypeParam;

    cudf::size_type colSize = 175;
    std::vector<T> h_input_v(colSize);
    std::vector<T> h_expect_v(colSize);

    std::generate(
        std::begin(h_input_v),
        std::end(h_input_v),
        [i = 0] () mutable { ++i; return i * i; });

    std::transform(
        std::cbegin(h_input_v),
        std::cend(h_input_v),
        std::begin(h_expect_v),
        [] (auto e) { return std::sqrt(static_cast<float>(e)); });

    cudf::test::fixed_width_column_wrapper<T> input    (std::cbegin(h_input_v),  std::cend(h_input_v));
    cudf::test::fixed_width_column_wrapper<T> expected (std::cbegin(h_expect_v), std::cend(h_expect_v));

    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::SQRT);

    cudf::test::expect_columns_equal(expected, output->view());
}

TYPED_TEST(cudf_math_test, SimpleABS)
{
    cudf::test::fixed_width_column_wrapper<TypeParam> input    {{ -2, -1, 1, 2 }};
    cudf::test::fixed_width_column_wrapper<TypeParam> expected {{  2,  1, 1, 2 }};
    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::ABS);
    cudf::test::expect_columns_equal(expected, output->view());
}

TYPED_TEST(cudf_math_test, SimpleSQRT)
{
    cudf::test::fixed_width_column_wrapper<TypeParam> input    {{ 1, 4, 9, 16 }};
    cudf::test::fixed_width_column_wrapper<TypeParam> expected {{ 1, 2, 3, 4  }};
    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::SQRT);
    cudf::test::expect_columns_equal(expected, output->view());
}

template <typename T>
struct cudf_math_with_floating_point_test : public cudf::test::BaseFixture {};

using floating_point_type_list = ::testing::Types<float, double>;

TYPED_TEST_CASE(cudf_math_with_floating_point_test, floating_point_type_list);

TYPED_TEST(cudf_math_with_floating_point_test, SimpleSIN)
{
    cudf::test::fixed_width_column_wrapper<TypeParam> input    {{ 0.0 }};
    cudf::test::fixed_width_column_wrapper<TypeParam> expected {{ 0.0 }};
    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::SIN);
    cudf::test::expect_columns_equal(expected, output->view());
}

TYPED_TEST(cudf_math_with_floating_point_test, SimpleCOS)
{
    cudf::test::fixed_width_column_wrapper<TypeParam> input    {{ 0.0 }};
    cudf::test::fixed_width_column_wrapper<TypeParam> expected {{ 1.0 }};
    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::COS);
    cudf::test::expect_columns_equal(expected, output->view());
}

TYPED_TEST(cudf_math_with_floating_point_test, SimpleFLOOR)
{
    cudf::test::fixed_width_column_wrapper<TypeParam> input    {{ 1.1, 3.3, 5.5, 7.7 }};
    cudf::test::fixed_width_column_wrapper<TypeParam> expected {{ 1.0, 3.0, 5.0, 7.0 }};
    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::FLOOR);
    cudf::test::expect_columns_equal(expected, output->view());
}

TYPED_TEST(cudf_math_with_floating_point_test, SimpleCEIL)
{
    cudf::test::fixed_width_column_wrapper<TypeParam> input    {{ 1.1, 3.3, 5.5, 7.7 }};
    cudf::test::fixed_width_column_wrapper<TypeParam> expected {{ 2.0, 4.0, 6.0, 8.0 }};
    auto output = cudf::experimental::unary_operation(input, cudf::experimental::unary_op::CEIL);
    cudf::test::expect_columns_equal(expected, output->view());
}

template <typename T>
struct IsNull : public cudf::test::BaseFixture {};

TYPED_TEST_CASE(IsNull, cudf::test::NumericTypes);

TYPED_TEST(IsNull, AllValid)
{
    using T = TypeParam;

    cudf::size_type start = 0;
    cudf::size_type size = 10;
    cudf::test::fixed_width_column_wrapper<T> col = create_fixed_columns<T>(start, size, false);
    cudf::test::fixed_width_column_wrapper<cudf::experimental::bool8> expected = create_expected_columns<cudf::experimental::bool8>(size, false, true);

    std::unique_ptr<cudf::column> got = cudf::experimental::is_null(col);

    cudf::test::expect_columns_equal(expected, got->view());
}

TYPED_TEST(IsNull, WithInvalids)
{
    using T = TypeParam;

    cudf::size_type start = 0;
    cudf::size_type size = 10;
    cudf::test::fixed_width_column_wrapper<T> col = create_fixed_columns<T>(start, size, true);
    cudf::test::fixed_width_column_wrapper<cudf::experimental::bool8> expected = create_expected_columns<cudf::experimental::bool8>(size, true, true);

    std::unique_ptr<cudf::column> got = cudf::experimental::is_null(col);

    cudf::test::expect_columns_equal(expected, got->view());
}

TYPED_TEST(IsNull, EmptyColumns)
{
    using T = TypeParam;

    cudf::size_type start = 0;
    cudf::size_type size = 0;
    cudf::test::fixed_width_column_wrapper<T> col = create_fixed_columns<T>(start, size, true);
    cudf::test::fixed_width_column_wrapper<cudf::experimental::bool8> expected = create_expected_columns<cudf::experimental::bool8>(size, true, true);

    std::unique_ptr<cudf::column> got = cudf::experimental::is_null(col);

    cudf::test::expect_columns_equal(expected, got->view());
}

template <typename T>
struct IsNotNull : public cudf::test::BaseFixture {};

TYPED_TEST_CASE(IsNotNull, cudf::test::NumericTypes);

TYPED_TEST(IsNotNull, AllValid)
{
    using T = TypeParam;

    cudf::size_type start = 0;
    cudf::size_type size = 10;
    cudf::test::fixed_width_column_wrapper<T> col = create_fixed_columns<T>(start, size, false);
    cudf::test::fixed_width_column_wrapper<cudf::experimental::bool8> expected = create_expected_columns<cudf::experimental::bool8>(size, false, false);

    std::unique_ptr<cudf::column> got = cudf::experimental::is_valid(col);

    cudf::test::expect_columns_equal(expected, got->view());
}

TYPED_TEST(IsNotNull, WithInvalids)
{
    using T = TypeParam;

    cudf::size_type start = 0;
    cudf::size_type size = 10;
    cudf::test::fixed_width_column_wrapper<T> col = create_fixed_columns<T>(start, size, true);
    cudf::test::fixed_width_column_wrapper<cudf::experimental::bool8> expected = create_expected_columns<cudf::experimental::bool8>(size, true, false);

    std::unique_ptr<cudf::column> got = cudf::experimental::is_valid(col);

    cudf::test::expect_columns_equal(expected, got->view());
}

TYPED_TEST(IsNotNull, EmptyColumns)
{
    using T = TypeParam;

    cudf::size_type start = 0;
    cudf::size_type size = 0;
    cudf::test::fixed_width_column_wrapper<T> col = create_fixed_columns<T>(start, size, true);
    cudf::test::fixed_width_column_wrapper<cudf::experimental::bool8> expected = create_expected_columns<cudf::experimental::bool8>(size, true, false);

    std::unique_ptr<cudf::column> got = cudf::experimental::is_valid(col);

    cudf::test::expect_columns_equal(expected, got->view());
}


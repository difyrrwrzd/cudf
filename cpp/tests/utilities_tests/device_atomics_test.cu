#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/column_wrapper.cuh>
#include <tests/utilities/cudf_test_fixtures.h>
#include <utilities/wrapper_types.hpp>
#include <utilities/device_atomics.cuh>

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <bitset>
#include <cstdint>
#include <random>

template<typename T>
__global__
void gpu_atomic_test(T *result, T *data, size_t size)
{
    size_t id   = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = blockDim.x * gridDim.x;

    for (; id < size; id += step) {
        atomicAdd(&result[0], data[id]);
        atomicMin(&result[1], data[id]);
        atomicMax(&result[2], data[id]);
        atomicAdd(&result[3], data[id]);
    }
}

template<typename T, typename BinaryOp>
__device__
T atomic_op(T* addr, T const & value, BinaryOp op)
{
    T old_value = *addr;
    T assumed;

    do {
        assumed  = old_value;
        const T new_value = op(old_value, value);

        old_value = atomicCAS(addr, assumed, new_value);
    } while (assumed != old_value);

    return old_value;
}

template<typename T>
__global__
void gpu_atomicCAS_test(T *result, T *data, size_t size)
{
    size_t id   = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = blockDim.x * gridDim.x;

    for (; id < size; id += step) {
        atomic_op(&result[0], data[id], cudf::DeviceSum{});
        atomic_op(&result[1], data[id], cudf::DeviceMin{});
        atomic_op(&result[2], data[id], cudf::DeviceMax{});
        atomic_op(&result[3], data[id], cudf::DeviceSum{});
    }
}

// TODO: remove these explicit instantiation for kernels
// At TYPED_TEST, the kernel for TypeParam of `wrapper` types won't be instantiated
// because `TypeParam` is a private member of class ::testing::Test
// then kenrel call failed by `hipErrorInvalidDeviceFunction`

template  __global__ void gpu_atomic_test<cudf::date32>(cudf::date32 *result, cudf::date32 *data, size_t size);
template  __global__ void gpu_atomic_test<cudf::date64>(cudf::date64 *result, cudf::date64 *data, size_t size);
template  __global__ void gpu_atomic_test<cudf::category>(cudf::category *result, cudf::category *data, size_t size);
template  __global__ void gpu_atomic_test<cudf::timestamp>(cudf::timestamp *result, cudf::timestamp *data, size_t size);

template  __global__ void gpu_atomicCAS_test<cudf::date32>(cudf::date32 *result, cudf::date32 *data, size_t size);
template  __global__ void gpu_atomicCAS_test<cudf::date64>(cudf::date64 *result, cudf::date64 *data, size_t size);
template  __global__ void gpu_atomicCAS_test<cudf::category>(cudf::category *result, cudf::category *data, size_t size);
template  __global__ void gpu_atomicCAS_test<cudf::timestamp>(cudf::timestamp *result, cudf::timestamp *data, size_t size);

// ---------------------------------------------

template <typename T>
struct AtomicsTest : public GdfTest {

    void atomic_test(bool call_atomicCAS)
    {
        std::vector<int> v({6, -14, 13, 64, -13, -20, 45});
        int exact[3];
        exact[0] = std::accumulate(v.begin(), v.end(), 0);
        exact[1] = *( std::min_element(v.begin(), v.end()) );
        exact[2] = *( std::max_element(v.begin(), v.end()) );
        size_t vec_size = v.size();

        // std::vector<T> v_type({6, -14, 13, 64, -13, -20, 45}));
        // use transform from std::vector<int> instead.
        std::vector<T> v_type(vec_size);
        std::transform(v.begin(), v.end(), v_type.begin(),
            [](int x) { T t(x) ; return t; } );

        std::vector<T> result_init(4);
        result_init[0] = T{0};
        result_init[1] = std::numeric_limits<T>::max();
        result_init[2] = std::numeric_limits<T>::min();
        result_init[3] = T{0};

        thrust::device_vector<T> dev_result(result_init);
        thrust::device_vector<T> dev_data(v_type);

        hipDeviceSynchronize();
        CUDA_CHECK_LAST();

        if( call_atomicCAS ){
            gpu_atomicCAS_test<T> <<<1, vec_size>>> (
                reinterpret_cast<T*>( dev_result.data().get() ),
                reinterpret_cast<T*>( dev_data.data().get() ),
                vec_size);
        }else{
            gpu_atomic_test<T> <<<1, vec_size>>> (
                reinterpret_cast<T*>( dev_result.data().get() ),
                reinterpret_cast<T*>( dev_data.data().get() ),
                vec_size);
        }

        hipDeviceSynchronize();
        CUDA_CHECK_LAST();

        thrust::host_vector<T> host_result(dev_result);
        hipDeviceSynchronize();

        CUDA_CHECK_LAST();

        EXPECT_EQ(host_result[0], T(exact[0])) << "atomicAdd test failed";
        EXPECT_EQ(host_result[1], T(exact[1])) << "atomicMin test failed";
        EXPECT_EQ(host_result[2], T(exact[2])) << "atomicMax test failed";
        EXPECT_EQ(host_result[3], T(exact[0])) << "atomicAdd test(2) failed";
    }
};

using TestingTypes = ::testing::Types<
    int8_t, int16_t, int32_t, int64_t, float, double,
    cudf::date32, cudf::date64, cudf::timestamp, cudf::category
    >;

TYPED_TEST_CASE(AtomicsTest, TestingTypes);

// tests for atomicAdd/Min/Max
TYPED_TEST(AtomicsTest, atomicOps)
{
    this->atomic_test(false);
}

// tests for atomicCAS
TYPED_TEST(AtomicsTest, atomicCAS)
{
    this->atomic_test(true);
}



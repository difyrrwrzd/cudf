#include "hip/hip_runtime.h"
//#include <nvstrings/NVCategory.h>
//#include <nvstrings/NVStrings.h>

#include <cudf/cudf.h>
#include <cudf/types.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <cudf/merge.hpp>
#include <rmm/thrust_rmm_allocator.h>
#include <cudf/column/column_factories.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <tests/utilities/type_lists.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/legacy/cudf_test_utils.cuh>

#include <cassert>
#include <vector>
#include <memory>
#include <algorithm>
#include <limits>
#include <initializer_list>

#include <gtest/gtest.h>

template <typename T>
class MergeTest_ : public cudf::test::BaseFixture {};

//TODO: confirm if the legacy test_types below can be replaced
//just by cudf::test::NumericTypes
//
//legacy:
//{
///using test_types =
///  ::testing::Types<int8_t, int16_t, int32_t, int64_t, float, double>
//,cudf::bool8>; //column_wrapper failure //, cudf::nvstring_category>; //string not ready

//TYPED_TEST_CASE(MergeTest_, test_types);
//}

TYPED_TEST_CASE(MergeTest_, cudf::test::NumericTypes);

TYPED_TEST(MergeTest_, MismatchedNumColumns) {
    using columnFactoryT = cudf::test::fixed_width_column_wrapper<TypeParam>;
    
    //TODO: figure out solution around compiler error
    //when trying to instantiate
    //fixed_width_column_wrapper<cudf::bool8>
    //
    columnFactoryT leftColWrap1{{0,1,2,3}}; 
    columnFactoryT rightColWrap1{{0,1,2,3}};
    columnFactoryT rightColWrap2{{0,1,2,3}};

    std::vector<cudf::size_type> key_cols{0};
    std::vector<cudf::order> column_order {cudf::order::ASCENDING};
    std::vector<cudf::null_order> null_precedence{};

    cudf::table_view left_view{{leftColWrap1}};
    cudf::table_view right_view{{rightColWrap1, rightColWrap2}};

    EXPECT_THROW(cudf::experimental::merge(left_view,
                                           right_view,
                                           key_cols,
                                           column_order,
                                           null_precedence), cudf::logic_error);
}


/*
TYPED_TEST(MergeTest_, MismatchedColumnDypes) {
    cudf::size_type inputRows = 4;

    cudf::test::column_wrapper<int32_t> leftColWrap1(inputRows, [](cudf::size_type row) { return row; });

    cudf::test::column_wrapper<double> rightColWrap1(inputRows, [](cudf::size_type row) { return row; });

    std::vector<cudf::size_type> sortByCols = {0};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC};

    EXPECT_THROW(cudf::merge(cudf::table{leftColWrap1.get()},
                            cudf::table{rightColWrap1.get()},
                            sortByCols,
                            orderByTypes), cudf::logic_error);
}

TYPED_TEST(MergeTest_, EmptyKeyColumns) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 4;

    auto leftColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    auto rightColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    std::vector<cudf::size_type> sortByCols;
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC};

    EXPECT_THROW(cudf::merge(cudf::table{leftColWrap1.get()},
                            cudf::table{rightColWrap1.get()},
                            sortByCols,
                            orderByTypes), cudf::logic_error);
}

TYPED_TEST(MergeTest_, TooManyKeyColumns) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 4;

    auto leftColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    auto rightColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    std::vector<cudf::size_type> sortByCols = {0, 1};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC};

    EXPECT_THROW(cudf::merge(cudf::table{leftColWrap1.get()},
                            cudf::table{rightColWrap1.get()},
                            sortByCols,
                            orderByTypes), cudf::logic_error);
}

TYPED_TEST(MergeTest_, EmptyOrderTypes) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 4;

    auto leftColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    auto rightColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    std::vector<cudf::size_type> sortByCols = {0};
    std::vector<order_by_type> orderByTypes;

    EXPECT_THROW(cudf::merge(cudf::table{leftColWrap1.get()},
                            cudf::table{rightColWrap1.get()},
                            sortByCols,
                            orderByTypes), cudf::logic_error);
}

TYPED_TEST(MergeTest_, TooManyOrderTypes) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 4;

    auto leftColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    auto rightColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    std::vector<cudf::size_type> sortByCols = {0};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC, GDF_ORDER_DESC};

    EXPECT_THROW(cudf::merge(cudf::table{leftColWrap1.get()},
                            cudf::table{rightColWrap1.get()},
                            sortByCols,
                            orderByTypes), cudf::logic_error);
}

TYPED_TEST(MergeTest_, MismatchedKeyColumnsAndOrderTypes) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 4;

    auto leftColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });
    auto leftColWrap2 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    auto rightColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });
    auto rightColWrap2 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    std::vector<cudf::size_type> sortByCols = {0, 1};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC};

    EXPECT_THROW(cudf::merge(cudf::table{leftColWrap1.get(), leftColWrap2.get()},
                            cudf::table{rightColWrap1.get(), rightColWrap2.get()},
                            sortByCols,
                            orderByTypes), cudf::logic_error);
}

TYPED_TEST(MergeTest_, MergeWithEmptyColumn) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 50000;
    inputRows = (cudf::detail::unwrap(std::numeric_limits<TypeParam>::max()) < inputRows ? 40 : inputRows);

    auto leftColWrap1 = columnFactory.make(inputRows, [](cudf::size_type row) { return row; });

    auto rightColWrap1 = columnFactory.make(0, [](cudf::size_type row) { return 0; });

    std::vector<cudf::size_type> sortByCols = {0};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC};

    cudf::table outputTable;
    EXPECT_NO_THROW(outputTable = cudf::merge(cudf::table{leftColWrap1.get()},
                                            cudf::table{rightColWrap1.get()},
                                            sortByCols,
                                            orderByTypes));
    
    const cudf::size_type outputRows = leftColWrap1.size() + rightColWrap1.size();
    auto expectedDataWrap1 = columnFactory.make(outputRows, [](cudf::size_type row) { return row; });

    EXPECT_TRUE(gdf_equal_columns(*expectedDataWrap1.get(), *outputTable.get_column(0)));
}

TYPED_TEST(MergeTest_, Merge1KeyColumns) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 50000;
    inputRows = (cudf::detail::unwrap(std::numeric_limits<TypeParam>::max()) < inputRows ? 40 : inputRows);

    auto leftColWrap1 = columnFactory.make(inputRows,
                                            [](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return 1;
                                                else return 2 * row; 
                                            });
    auto leftColWrap2 = columnFactory.make(inputRows,
                                            [](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return 0;
                                                else return row;
                                            });

    auto rightColWrap1 = columnFactory.make(inputRows,
                                            [](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return 0;
                                                else return 2 * row + 1;
                                            });
    auto rightColWrap2 = columnFactory.make(inputRows,
                                            [](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return 0;
                                                else return row;
                                            });

    std::vector<cudf::size_type> sortByCols = {0};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC};

    cudf::table outputTable;
    EXPECT_NO_THROW(outputTable = cudf::merge(cudf::table{leftColWrap1.get(), leftColWrap2.get()},
                                            cudf::table{rightColWrap1.get(), rightColWrap2.get()},
                                            sortByCols,
                                            orderByTypes));

    const cudf::size_type outputRows = leftColWrap1.size() + rightColWrap1.size();
    auto expectedDataWrap1 = columnFactory.make(outputRows,
                                                [=](cudf::size_type row)->cudf::size_type {
                                                    if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return row >= outputRows / 2;
                                                    else return row;
                                                });
    auto expectedDataWrap2 = columnFactory.make(outputRows,
                                                [](cudf::size_type row)->cudf::size_type {
                                                    if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return 0;
                                                    else return row / 2; 
                                                });

    EXPECT_TRUE(gdf_equal_columns(*expectedDataWrap1.get(), *outputTable.get_column(0)));
    EXPECT_TRUE(gdf_equal_columns(*expectedDataWrap2.get(), *outputTable.get_column(1)));
}

TYPED_TEST(MergeTest_, Merge2KeyColumns) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 50000;
    inputRows = (cudf::detail::unwrap(std::numeric_limits<TypeParam>::max()) < inputRows ? 40 : inputRows);

    auto leftColWrap1 = columnFactory.make(inputRows,
                                            [=](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return row >= inputRows / 2;
                                                else return row;
                                            });
    auto leftColWrap2 = columnFactory.make(inputRows,
                                            [=](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return (row / (inputRows / 4)) % 2 == 0;
                                                else return 2 * row;
                                            });

    auto rightColWrap1 = columnFactory.make(inputRows,
                                            [=](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return row >= inputRows / 2;
                                                else return row;
                                            });
    auto rightColWrap2 = columnFactory.make(inputRows,
                                            [=](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return (row / (inputRows / 4)) % 2 == 0;
                                                else return 2 * row + 1;
                                            });

    std::vector<cudf::size_type> sortByCols = {0, 1};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC, GDF_ORDER_DESC};

    cudf::table outputTable;
    EXPECT_NO_THROW(outputTable = cudf::merge(cudf::table{leftColWrap1.get(), leftColWrap2.get()},
                                            cudf::table{rightColWrap1.get(), rightColWrap2.get()},
                                            sortByCols,
                                            orderByTypes));

    const cudf::size_type outputRows = leftColWrap1.size() + rightColWrap1.size();
    auto expectedDataWrap1 = columnFactory.make(outputRows,
                                                [=](cudf::size_type row)->cudf::size_type {
                                                    if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return row >= outputRows / 2;
                                                    else return row / 2;
                                                });
    auto expectedDataWrap2 = columnFactory.make(outputRows,
                                                [=](cudf::size_type row)->cudf::size_type {
                                                    if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return (row / (outputRows / 4)) % 2 == 0;
                                                    else return row % 2 == 0 ? row + 1 : row - 1;
                                                });

    EXPECT_TRUE(gdf_equal_columns(*expectedDataWrap1.get(), *outputTable.get_column(0)));
    EXPECT_TRUE(gdf_equal_columns(*expectedDataWrap2.get(), *outputTable.get_column(1)));
}

TYPED_TEST(MergeTest_, Merge1KeyNullColumns) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 50000;
    inputRows = (cudf::detail::unwrap(std::numeric_limits<TypeParam>::max()) < inputRows ? 40 : inputRows);

    // data: 0  2  4  6 | valid: 1 1 1 0
    auto leftColWrap1 = columnFactory.make(inputRows,
                                            [](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return 0;
                                                else return 2 * row;
                                            },
                                            [=](cudf::size_type row) { return row < inputRows - 1; });

    // data: 1  3  5  7 | valid: 1 1 1 0
    auto rightColWrap1 = columnFactory.make(inputRows,
                                            [](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return 1;
                                                else return 2 * row + 1;
                                            },
                                            [=](cudf::size_type row) { return row < inputRows - 1; });

    std::vector<cudf::size_type> sortByCols = {0};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC};

    cudf::table outputTable;
    EXPECT_NO_THROW(outputTable = cudf::merge(cudf::table{leftColWrap1.get()},
                                            cudf::table{rightColWrap1.get()},
                                            sortByCols,
                                            orderByTypes));

    const cudf::size_type outputRows = leftColWrap1.size() + rightColWrap1.size();
    // data: 0 1 2 3 4 5 6 7 | valid: 1 1 1 1 1 1 0 0
    const cudf::size_type column1TotalNulls = leftColWrap1.null_count() + rightColWrap1.null_count();
    auto expectedDataWrap1 = columnFactory.make(outputRows,
                                                [=](cudf::size_type row)->cudf::size_type {
                                                    if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return row >= (outputRows - column1TotalNulls) / 2;
                                                    else return row;
                                                },
                                                [=](cudf::size_type row) { return row < (outputRows - column1TotalNulls); });

    EXPECT_TRUE(gdf_equal_columns(*expectedDataWrap1.get(), *outputTable.get_column(0)));
}

TYPED_TEST(MergeTest_, Merge2KeyNullColumns) {
    cudf::test::column_wrapper_factory<TypeParam> columnFactory;

    cudf::size_type inputRows = 50000;
    inputRows = (cudf::detail::unwrap(std::numeric_limits<TypeParam>::max()) < inputRows ? 40 : inputRows);

    // data: 0 1 2 3 | valid: 1 1 1 1
    auto leftColWrap1 = columnFactory.make(inputRows,
                                            [=](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return row >= inputRows / 2;
                                                else return row;
                                            });
    // data: 0 2 4 6 | valid: 1 1 1 1
    auto leftColWrap2 = columnFactory.make(inputRows,
                                            [=](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return (row / (inputRows / 4)) % 2 == 0;
                                                else return 2 * row;
                                            },
                                            [](cudf::size_type row) { return true; });

    // data: 0 1 2 3 | valid: 1 1 1 1
    auto rightColWrap1 = columnFactory.make(inputRows,
                                            [=](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return row >= inputRows / 2;
                                                else return row;
                                            });
    // data: 0 1 2 3 | valid: 0 0 0 0
    auto rightColWrap2 = columnFactory.make(inputRows,
                                            [=](cudf::size_type row)->cudf::size_type {
                                                if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return (row / (inputRows / 4)) % 2 == 0;
                                                else return row;
                                            },
                                            [](cudf::size_type row) { return false; });

    std::vector<cudf::size_type> sortByCols = {0, 1};
    std::vector<order_by_type> orderByTypes = {GDF_ORDER_ASC, GDF_ORDER_DESC};

    cudf::table outputTable;
    EXPECT_NO_THROW(outputTable = cudf::merge(cudf::table{leftColWrap1.get(), leftColWrap2.get()},
                                            cudf::table{rightColWrap1.get(), rightColWrap2.get()},
                                            sortByCols,
                                            orderByTypes));

    const cudf::size_type outputRows = leftColWrap1.size() + rightColWrap1.size();
    // data: 0 0 1 1 2 2 3 3 | valid: 1 1 1 1 1 1 1 1
    auto expectedDataWrap1 = columnFactory.make(outputRows,
                                                [=](cudf::size_type row)->cudf::size_type {
                                                    if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return row >= outputRows / 2;
                                                    else return row / 2;
                                                },
                                                [](cudf::size_type row) { return true; });
    // data: 0 0 2 1 4 2 6 3 | valid: 0 1 0 1 0 1 0 1
    auto expectedDataWrap2 = columnFactory.make(outputRows,
                                                [=](cudf::size_type row)->cudf::size_type {
                                                    if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return (row / (outputRows / 8)) % 2 == 0;
                                                    else return row % 2 != 0 ? 2 * (row / 2) : (row / 2);
                                                },
                                                [=](cudf::size_type row) {
                                                    if(cudf::gdf_dtype_of<TypeParam>() == GDF_BOOL8) return (row / (outputRows / 4)) % 2 == 1;
                                                    else return row % 2 != 0;
                                                });

    EXPECT_TRUE(gdf_equal_columns(*expectedDataWrap1.get(), *outputTable.get_column(0)));
    EXPECT_TRUE(gdf_equal_columns(*expectedDataWrap2.get(), *outputTable.get_column(1)));
}
*/

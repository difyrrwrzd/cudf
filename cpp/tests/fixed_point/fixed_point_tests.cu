#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/type_lists.hpp>
#include "tests/utilities/column_utilities.hpp"
#include "tests/utilities/table_utilities.hpp"

#include <cudf/binaryop.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/fixed_point/fixed_point.hpp>
#include <cudf/reduction.hpp>
#include <cudf/replace.hpp>
#include <cudf/reshape.hpp>
#include <cudf/sorting.hpp>
#include <cudf/unary.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include "cudf/search.hpp"
#include "cudf/types.hpp"

#include <algorithm>
#include <limits>
#include <numeric>
#include <type_traits>
#include <vector>

using namespace numeric;

struct FixedPointTest : public cudf::test::BaseFixture {
};

template <typename T>
struct FixedPointTestBothReps : public cudf::test::BaseFixture {
};

using RepresentationTypes = ::testing::Types<int32_t, int64_t>;

TYPED_TEST_CASE(FixedPointTestBothReps, RepresentationTypes);

TYPED_TEST(FixedPointTestBothReps, SimpleDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{1.234567, scale_type{0}};
  decimalXX num1{1.234567, scale_type{-1}};
  decimalXX num2{1.234567, scale_type{-2}};
  decimalXX num3{1.234567, scale_type{-3}};
  decimalXX num4{1.234567, scale_type{-4}};
  decimalXX num5{1.234567, scale_type{-5}};
  decimalXX num6{1.234567, scale_type{-6}};

  EXPECT_EQ(1, static_cast<double>(num0));
  EXPECT_EQ(1.2, static_cast<double>(num1));
  EXPECT_EQ(1.23, static_cast<double>(num2));
  EXPECT_EQ(1.235, static_cast<double>(num3));    // rounds up
  EXPECT_EQ(1.2346, static_cast<double>(num4));   // rounds up
  EXPECT_EQ(1.23457, static_cast<double>(num5));  // rounds up
  EXPECT_EQ(1.234567, static_cast<double>(num6));
}

TYPED_TEST(FixedPointTestBothReps, SimpleNegativeDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{-1.234567, scale_type{0}};
  decimalXX num1{-1.234567, scale_type{-1}};
  decimalXX num2{-1.234567, scale_type{-2}};
  decimalXX num3{-1.234567, scale_type{-3}};
  decimalXX num4{-1.234567, scale_type{-4}};
  decimalXX num5{-1.234567, scale_type{-5}};
  decimalXX num6{-1.234567, scale_type{-6}};

  EXPECT_EQ(-1, static_cast<double>(num0));
  EXPECT_EQ(-1.2, static_cast<double>(num1));
  EXPECT_EQ(-1.23, static_cast<double>(num2));
  EXPECT_EQ(-1.235, static_cast<double>(num3));    // rounds up
  EXPECT_EQ(-1.2346, static_cast<double>(num4));   // rounds up
  EXPECT_EQ(-1.23457, static_cast<double>(num5));  // rounds up
  EXPECT_EQ(-1.234567, static_cast<double>(num6));
}

TYPED_TEST(FixedPointTestBothReps, PaddedDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX a{1.1, scale_type{-1}};
  decimalXX b{1.01, scale_type{-2}};
  decimalXX c{1.001, scale_type{-3}};
  decimalXX d{1.0001, scale_type{-4}};
  decimalXX e{1.00001, scale_type{-5}};
  decimalXX f{1.000001, scale_type{-6}};

  decimalXX x{1.000123, scale_type{-8}};
  decimalXX y{0.000123, scale_type{-8}};

  EXPECT_EQ(1.1, static_cast<double>(a));
  EXPECT_EQ(1.01, static_cast<double>(b));
  EXPECT_EQ(1.001, static_cast<double>(c));
  EXPECT_EQ(1.0001, static_cast<double>(d));
  EXPECT_EQ(1.00001, static_cast<double>(e));
  EXPECT_EQ(1.000001, static_cast<double>(f));

  EXPECT_TRUE(1.000123 - static_cast<double>(x) < std::numeric_limits<double>::epsilon());
  EXPECT_EQ(0.000123, static_cast<double>(y));
}

TYPED_TEST(FixedPointTestBothReps, SimpleBinaryFPConstruction)
{
  using binary_fp = fixed_point<TypeParam, Radix::BASE_2>;

  binary_fp num0{10, scale_type{0}};
  binary_fp num1{10, scale_type{1}};
  binary_fp num2{10, scale_type{2}};
  binary_fp num3{10, scale_type{3}};
  binary_fp num4{10, scale_type{4}};

  binary_fp num5{1.24, scale_type{0}};
  binary_fp num6{1.24, scale_type{-1}};
  binary_fp num7{1.32, scale_type{-2}};
  binary_fp num8{1.41, scale_type{-3}};
  binary_fp num9{1.45, scale_type{-4}};

  EXPECT_EQ(10, static_cast<double>(num0));
  EXPECT_EQ(10, static_cast<double>(num1));
  EXPECT_EQ(12, static_cast<double>(num2));
  EXPECT_EQ(8, static_cast<double>(num3));
  EXPECT_EQ(16, static_cast<double>(num4));

  EXPECT_EQ(1, static_cast<double>(num5));
  EXPECT_EQ(1, static_cast<double>(num6));
  EXPECT_EQ(1.25, static_cast<double>(num7));
  EXPECT_EQ(1.375, static_cast<double>(num8));
  EXPECT_EQ(1.4375, static_cast<double>(num9));
}

TYPED_TEST(FixedPointTestBothReps, MoreSimpleBinaryFPConstruction)
{
  using binary_fp = fixed_point<TypeParam, Radix::BASE_2>;

  binary_fp num0{1.25, scale_type{-2}};
  binary_fp num1{2.1, scale_type{-4}};

  EXPECT_EQ(1.25, static_cast<double>(num0));
  EXPECT_EQ(2.125, static_cast<double>(num1));
}

TYPED_TEST(FixedPointTestBothReps, SimpleDecimalXXMath)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ONE{1, scale_type{-2}};
  decimalXX TWO{2, scale_type{-2}};
  decimalXX THREE{3, scale_type{-2}};
  decimalXX SIX{6, scale_type{-2}};

  EXPECT_TRUE(ONE + ONE == TWO);

  EXPECT_EQ(ONE + ONE, TWO);
  EXPECT_EQ(ONE * TWO, TWO);
  EXPECT_EQ(THREE * TWO, SIX);
  EXPECT_EQ(THREE - TWO, ONE);
  EXPECT_EQ(TWO / ONE, TWO);
  EXPECT_EQ(SIX / TWO, THREE);

  decimalXX a{1.23, scale_type{-2}};
  decimalXX b{0, scale_type{0}};

  EXPECT_EQ(a + b, a);
  EXPECT_EQ(a - b, a);
}

TYPED_TEST(FixedPointTestBothReps, ComparisonOperators)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ONE{1, scale_type{-1}};
  decimalXX TWO{2, scale_type{-2}};
  decimalXX THREE{3, scale_type{-3}};
  decimalXX SIX{6, scale_type{-4}};

  EXPECT_TRUE(ONE + ONE >= TWO);

  EXPECT_TRUE(ONE + ONE <= TWO);
  EXPECT_TRUE(ONE * TWO < THREE);
  EXPECT_TRUE(THREE * TWO > THREE);
  EXPECT_TRUE(THREE - TWO >= ONE);
  EXPECT_TRUE(TWO / ONE < THREE);
  EXPECT_TRUE(SIX / TWO >= ONE);
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXTrickyDivision)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ONE_1{1, scale_type{1}};
  decimalXX SIX_0{6, scale_type{0}};
  decimalXX SIX_1{6, scale_type{1}};
  decimalXX TEN_0{10, scale_type{0}};
  decimalXX TEN_1{10, scale_type{1}};
  decimalXX SIXTY_1{60, scale_type{1}};

  EXPECT_EQ(static_cast<int32_t>(ONE_1), 0);   // round(1 / 10) = 0
  EXPECT_EQ(static_cast<int32_t>(SIX_1), 10);  // round(6 / 10) = 10
  EXPECT_EQ(static_cast<int32_t>(TEN_0), 10);
  EXPECT_EQ(static_cast<int32_t>(SIXTY_1), 60);

  EXPECT_EQ(SIXTY_1 / TEN_0, TEN_1);
  EXPECT_EQ(SIXTY_1 / TEN_1, SIX_0);

  decimalXX A{34.56, scale_type{-2}};
  decimalXX B{1.234, scale_type{-3}};
  decimalXX C{1, scale_type{-2}};

  EXPECT_EQ(static_cast<int32_t>(A / B), 30);
  EXPECT_EQ(static_cast<int32_t>((A * C) / B), 28);

  decimalXX n{28, scale_type{1}};
  EXPECT_EQ(static_cast<int32_t>(n), 30);
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXRounding)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ZERO_0{0, scale_type{0}};
  decimalXX ZERO_1{4, scale_type{1}};
  decimalXX THREE_0{3, scale_type{0}};
  decimalXX FOUR_0{4, scale_type{0}};
  decimalXX FIVE_0{5, scale_type{0}};
  decimalXX TEN_0{10, scale_type{0}};
  decimalXX TEN_1{5, scale_type{1}};

  decimalXX FOURTEEN_0{14, scale_type{0}};
  decimalXX FIFTEEN_0{15, scale_type{0}};

  EXPECT_EQ(ZERO_0, ZERO_1);
  EXPECT_EQ(TEN_0, TEN_1);

  EXPECT_EQ(ZERO_1 + TEN_1, TEN_1);
  EXPECT_EQ(FOUR_0 + TEN_1, FOURTEEN_0);
  EXPECT_TRUE(ZERO_0 == ZERO_1);
  EXPECT_TRUE(FIVE_0 != TEN_1);
  EXPECT_TRUE(FIVE_0 + FIVE_0 + FIVE_0 == FIFTEEN_0);
  EXPECT_TRUE(FIVE_0 + FIVE_0 + FIVE_0 != TEN_1);
  EXPECT_TRUE(FIVE_0 * THREE_0 == FIFTEEN_0);
  EXPECT_TRUE(FIVE_0 * THREE_0 != TEN_1);
}

TYPED_TEST(FixedPointTestBothReps, ArithmeticWithDifferentScales)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX a{1, scale_type{0}};
  decimalXX b{1.2, scale_type{-1}};
  decimalXX c{1.23, scale_type{-2}};
  decimalXX d{1.111, scale_type{-3}};

  decimalXX x{2.2, scale_type{-1}};
  decimalXX y{3.43, scale_type{-2}};
  decimalXX z{4.541, scale_type{-3}};

  decimalXX xx{0.2, scale_type{-1}};
  decimalXX yy{0.03, scale_type{-2}};
  decimalXX zz{0.119, scale_type{-3}};

  EXPECT_EQ(a + b, x);
  EXPECT_EQ(a + b + c, y);
  EXPECT_EQ(a + b + c + d, z);
  EXPECT_EQ(b - a, xx);
  EXPECT_EQ(c - b, yy);
  EXPECT_EQ(c - d, zz);
}

TYPED_TEST(FixedPointTestBothReps, RescaledTest)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{1, scale_type{0}};
  decimalXX num1{1.2, scale_type{-1}};
  decimalXX num2{1.23, scale_type{-2}};
  decimalXX num3{1.235, scale_type{-3}};
  decimalXX num4{1.2346, scale_type{-4}};
  decimalXX num5{1.23457, scale_type{-5}};
  decimalXX num6{1.234567, scale_type{-6}};

  EXPECT_EQ(num0, num6.rescaled(scale_type{0}));
  EXPECT_EQ(num1, num6.rescaled(scale_type{-1}));
  EXPECT_EQ(num2, num6.rescaled(scale_type{-2}));
  EXPECT_EQ(num3, num6.rescaled(scale_type{-3}));
  EXPECT_EQ(num4, num6.rescaled(scale_type{-4}));
  EXPECT_EQ(num5, num6.rescaled(scale_type{-5}));
}

TYPED_TEST(FixedPointTestBothReps, RescaledRounding)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{1500, scale_type{0}};
  decimalXX num1{1499, scale_type{0}};
  decimalXX num2{-1499, scale_type{0}};
  decimalXX num3{-1500, scale_type{0}};

  EXPECT_EQ(2000, static_cast<TypeParam>(num0.rescaled(scale_type{3})));
  EXPECT_EQ(1000, static_cast<TypeParam>(num1.rescaled(scale_type{3})));
  EXPECT_EQ(-1000, static_cast<TypeParam>(num2.rescaled(scale_type{3})));
  EXPECT_EQ(-2000, static_cast<TypeParam>(num3.rescaled(scale_type{3})));
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXThrust)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  std::vector<decimalXX> vec1(1000);
  std::vector<int32_t> vec2(1000);

  std::iota(std::begin(vec1), std::end(vec1), decimalXX{0, scale_type{-2}});
  std::iota(std::begin(vec2), std::end(vec2), 0);

  auto const res1 =
    thrust::reduce(std::cbegin(vec1), std::cend(vec1), decimalXX{0, scale_type{-2}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), 0);

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  std::vector<int32_t> vec3(vec1.size());

  thrust::transform(std::cbegin(vec1), std::cend(vec1), std::begin(vec3), [](auto const& e) {
    return static_cast<int32_t>(e);
  });

  EXPECT_EQ(vec2, vec3);
}

TEST_F(FixedPointTest, OverflowDecimal32)
{
  // This flag is needed to avoid warnings with ASSERT_DEATH
  ::testing::FLAGS_gtest_death_test_style = "threadsafe";

  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

#if defined(__CUDACC_DEBUG__)

  decimal32 num0{2, scale_type{-9}};
  decimal32 num1{-2, scale_type{-9}};

  ASSERT_DEATH(num0 + num0, ".*");
  ASSERT_DEATH(num1 - num0, ".*");

  decimal32 min{std::numeric_limits<int32_t>::min(), scale_type{0}};
  decimal32 max{std::numeric_limits<int32_t>::max(), scale_type{0}};
  decimal32 NEG_ONE{-1, scale_type{0}};
  decimal32 ONE{1, scale_type{0}};
  decimal32 TWO{2, scale_type{0}};

  ASSERT_DEATH(min / NEG_ONE, ".*");
  ASSERT_DEATH(max * TWO, ".*");
  ASSERT_DEATH(min * TWO, ".*");
  ASSERT_DEATH(max + ONE, ".*");
  ASSERT_DEATH(max - NEG_ONE, ".*");
  ASSERT_DEATH(min - ONE, ".*");
  ASSERT_DEATH(max - NEG_ONE, ".*");

#endif
}

TEST_F(FixedPointTest, OverflowDecimal64)
{
  // This flag is needed to avoid warnings with ASSERT_DEATH
  ::testing::FLAGS_gtest_death_test_style = "threadsafe";

  using decimal64 = fixed_point<int64_t, Radix::BASE_10>;

#if defined(__CUDACC_DEBUG__)

  decimal64 num0{5, scale_type{-18}};
  decimal64 num1{-5, scale_type{-18}};

  ASSERT_DEATH(num0 + num0, ".*");
  ASSERT_DEATH(num1 - num0, ".*");

  decimal64 min{std::numeric_limits<int64_t>::min(), scale_type{0}};
  decimal64 max{std::numeric_limits<int64_t>::max(), scale_type{0}};
  decimal64 NEG_ONE{-1, scale_type{0}};
  decimal64 ONE{1, scale_type{0}};
  decimal64 TWO{2, scale_type{0}};

  ASSERT_DEATH(min / NEG_ONE, ".*");
  ASSERT_DEATH(max * TWO, ".*");
  ASSERT_DEATH(min * TWO, ".*");
  ASSERT_DEATH(max + ONE, ".*");
  ASSERT_DEATH(max - NEG_ONE, ".*");
  ASSERT_DEATH(min - ONE, ".*");
  ASSERT_DEATH(max - NEG_ONE, ".*");

#endif
}

template <typename ValueType, typename Binop>
void integer_vector_test(ValueType const initial_value,
                         int32_t const size,
                         int32_t const scale,
                         Binop binop)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  std::vector<decimal32> vec1(size);
  std::vector<ValueType> vec2(size);

  std::iota(std::begin(vec1), std::end(vec1), decimal32{initial_value, scale_type{scale}});
  std::iota(std::begin(vec2), std::end(vec2), initial_value);

  auto const res1 =
    std::accumulate(std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{scale}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), static_cast<ValueType>(0));

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  std::vector<ValueType> vec3(vec1.size());

  std::transform(std::cbegin(vec1), std::cend(vec1), std::begin(vec3), [](auto const& e) {
    return static_cast<int32_t>(e);
  });

  EXPECT_EQ(vec2, vec3);
}

TEST_F(FixedPointTest, Decimal32IntVector)
{
  integer_vector_test(0, 10, -2, std::plus<>());
  integer_vector_test(0, 1000, -2, std::plus<>());

  integer_vector_test(1, 10, 0, std::multiplies<>());
  integer_vector_test(2, 20, 0, std::multiplies<>());
}

template <typename ValueType, typename Binop>
void float_vector_test(ValueType const initial_value,
                       int32_t const size,
                       int32_t const scale,
                       Binop binop)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  std::vector<decimal32> vec1(size);
  std::vector<ValueType> vec2(size);

  std::iota(std::begin(vec1), std::end(vec1), decimal32{initial_value, scale_type{scale}});
  std::iota(std::begin(vec2), std::end(vec2), initial_value);

  auto equal = std::equal(
    std::cbegin(vec1), std::cend(vec1), std::cbegin(vec2), [](auto const& a, auto const& b) {
      return static_cast<double>(a) - b <= std::numeric_limits<ValueType>::epsilon();
    });

  EXPECT_TRUE(equal);
}

TEST_F(FixedPointTest, Decimal32FloatVector)
{
  float_vector_test(0.1, 1000, -2, std::plus<>());
  float_vector_test(0.15, 1000, -2, std::plus<>());

  float_vector_test(0.1, 10, -2, std::multiplies<>());
  float_vector_test(0.15, 20, -2, std::multiplies<>());
}

struct cast_to_int32_fn {
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;
  int32_t __host__ __device__ operator()(decimal32 fp) { return static_cast<int32_t>(fp); }
};

TEST_F(FixedPointTest, DecimalXXThrustOnDevice)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  thrust::device_vector<decimal32> vec1(1000, decimal32{1, scale_type{-2}});

  auto const sum = thrust::reduce(
    rmm::exec_policy(0)->on(0), std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{-2}});

  EXPECT_EQ(static_cast<int32_t>(sum), 1000);

  // TODO: Once nvbugs/1990211 is fixed (ExclusiveSum initial_value = 0 bug)
  //       change inclusive scan to run on device (avoid copying to host)
  thrust::host_vector<decimal32> vec1_host = vec1;

  thrust::inclusive_scan(std::cbegin(vec1_host), std::cend(vec1_host), std::begin(vec1_host));

  vec1 = vec1_host;

  std::vector<int32_t> vec2(1000);
  std::iota(std::begin(vec2), std::end(vec2), 1);

  auto const res1 = thrust::reduce(
    rmm::exec_policy(0)->on(0), std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{-2}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), 0);

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  thrust::device_vector<int32_t> vec3(1000);

  thrust::transform(rmm::exec_policy(0)->on(0),
                    std::cbegin(vec1),
                    std::cend(vec1),
                    std::begin(vec3),
                    cast_to_int32_fn{});

  thrust::host_vector<int32_t> vec3_host = vec3;

  EXPECT_EQ(vec2, vec3);
}

template <typename T>
using wrapper = cudf::test::fixed_width_column_wrapper<T>;

TYPED_TEST(FixedPointTestBothReps, FixedPointSortedOrderGather)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto const ZERO  = decimalXX{0, scale_type{0}};
  auto const ONE   = decimalXX{1, scale_type{0}};
  auto const TWO   = decimalXX{2, scale_type{0}};
  auto const THREE = decimalXX{3, scale_type{0}};
  auto const FOUR  = decimalXX{4, scale_type{0}};

  auto const input_vec  = std::vector<decimalXX>{TWO, ONE, ZERO, FOUR, THREE};
  auto const index_vec  = std::vector<cudf::size_type>{2, 1, 0, 4, 3};
  auto const sorted_vec = std::vector<decimalXX>{ZERO, ONE, TWO, THREE, FOUR};

  auto const input_col  = wrapper<decimalXX>(input_vec.begin(), input_vec.end());
  auto const index_col  = wrapper<cudf::size_type>(index_vec.begin(), index_vec.end());
  auto const sorted_col = wrapper<decimalXX>(sorted_vec.begin(), sorted_vec.end());

  auto const sorted_table = cudf::table_view{{sorted_col}};
  auto const input_table  = cudf::table_view{{input_col}};

  auto const indices = cudf::sorted_order(input_table);
  auto const sorted  = cudf::gather(input_table, indices->view());

  cudf::test::expect_columns_equal(index_col, indices->view());
  cudf::test::expect_tables_equal(sorted_table, sorted->view());
}

TYPED_TEST(FixedPointTestBothReps, FixedPointBinaryOpAdd)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto const sz = std::size_t{1000};

  auto vec1       = std::vector<decimalXX>(sz);
  auto const vec2 = std::vector<decimalXX>(sz, decimalXX{1, scale_type{-1}});
  auto expected   = std::vector<decimalXX>(sz);

  std::iota(std::begin(vec1), std::end(vec1), decimalXX{});

  std::transform(std::cbegin(vec1),
                 std::cend(vec1),
                 std::cbegin(vec2),
                 std::begin(expected),
                 std::plus<decimalXX>());

  auto const lhs          = wrapper<decimalXX>(vec1.begin(), vec1.end());
  auto const rhs          = wrapper<decimalXX>(vec2.begin(), vec2.end());
  auto const expected_col = wrapper<decimalXX>(expected.begin(), expected.end());

  auto const result = cudf::binary_operation(
    lhs, rhs, cudf::binary_operator::ADD, static_cast<cudf::column_view>(lhs).type());

  cudf::test::expect_columns_equal(expected_col, result->view());
}

TYPED_TEST(FixedPointTestBothReps, FixedPointBinaryOpMultiply)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto const sz = std::size_t{1000};

  auto vec1       = std::vector<decimalXX>(sz);
  auto const vec2 = std::vector<decimalXX>(sz, decimalXX{1, scale_type{-1}});
  auto expected   = std::vector<decimalXX>(sz);

  std::iota(std::begin(vec1), std::end(vec1), decimalXX{});

  std::transform(std::cbegin(vec1),
                 std::cend(vec1),
                 std::cbegin(vec2),
                 std::begin(expected),
                 std::multiplies<decimalXX>());

  auto const lhs          = wrapper<decimalXX>(vec1.begin(), vec1.end());
  auto const rhs          = wrapper<decimalXX>(vec2.begin(), vec2.end());
  auto const expected_col = wrapper<decimalXX>(expected.begin(), expected.end());

  auto const result = cudf::binary_operation(
    lhs, rhs, cudf::binary_operator::MUL, static_cast<cudf::column_view>(lhs).type());

  cudf::test::expect_columns_equal(expected_col, result->view());
}

TYPED_TEST(FixedPointTestBothReps, FixedPointConcatentate)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto vec = std::vector<decimalXX>(1000);
  std::iota(std::begin(vec), std::end(vec), decimalXX{});

  auto const a = wrapper<decimalXX>(vec.begin(), /***/ vec.begin() + 300);
  auto const b = wrapper<decimalXX>(vec.begin() + 300, vec.begin() + 700);
  auto const c = wrapper<decimalXX>(vec.begin() + 700, vec.end());

  auto const fixed_point_columns = std::vector<cudf::column_view>{a, b, c};
  auto const results             = cudf::concatenate(fixed_point_columns);
  auto const expected            = wrapper<decimalXX>(vec.begin(), vec.end());

  cudf::test::expect_columns_equal(*results, expected);
}

TYPED_TEST(FixedPointTestBothReps, FixedPointReplace)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto const ONE = decimalXX{1, scale_type{0}};
  auto const TWO = decimalXX{2, scale_type{0}};
  auto const sz  = std::size_t{1000};

  auto vec1       = std::vector<decimalXX>(sz);
  auto const vec2 = std::vector<decimalXX>(sz, TWO);

  std::generate(vec1.begin(), vec1.end(), [&, i = 0]() mutable { return ++i % 2 ? ONE : TWO; });

  auto const to_replace  = std::vector<decimalXX>{ONE};
  auto const replacement = std::vector<decimalXX>{TWO};

  auto const input_w       = wrapper<decimalXX>(vec1.begin(), vec1.end());
  auto const to_replace_w  = wrapper<decimalXX>(to_replace.begin(), to_replace.end());
  auto const replacement_w = wrapper<decimalXX>(replacement.begin(), replacement.end());
  auto const expected_w    = wrapper<decimalXX>(vec2.begin(), vec2.end());

  auto const result = cudf::find_and_replace_all(input_w, to_replace_w, replacement_w);

  cudf::test::expect_columns_equal(*result, expected_w);
}

TYPED_TEST(FixedPointTestBothReps, FixedPointLowerBound)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto vec = std::vector<decimalXX>(1000);
  std::iota(std::begin(vec), std::end(vec), decimalXX{});

  auto const values = wrapper<decimalXX>{decimalXX{200, scale_type{0}},
                                         decimalXX{400, scale_type{0}},
                                         decimalXX{600, scale_type{0}},
                                         decimalXX{800, scale_type{0}}};
  auto const expect = wrapper<cudf::size_type>{200, 400, 600, 800};
  auto const column = wrapper<decimalXX>(vec.begin(), vec.end());

  auto result = cudf::lower_bound({cudf::table_view{{column}}},
                                  {cudf::table_view{{values}}},
                                  {cudf::order::ASCENDING},
                                  {cudf::null_order::BEFORE});

  expect_columns_equal(*result, expect);
}

TYPED_TEST(FixedPointTestBothReps, FixedPointUpperBound)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto vec = std::vector<decimalXX>(1000);
  std::iota(std::begin(vec), std::end(vec), decimalXX{});

  auto const values = wrapper<decimalXX>{decimalXX{200, scale_type{0}},
                                         decimalXX{400, scale_type{0}},
                                         decimalXX{600, scale_type{0}},
                                         decimalXX{800, scale_type{0}}};
  auto const expect = wrapper<cudf::size_type>{201, 401, 601, 801};
  auto const column = wrapper<decimalXX>(vec.begin(), vec.end());

  auto result = cudf::upper_bound({cudf::table_view{{column}}},
                                  {cudf::table_view{{values}}},
                                  {cudf::order::ASCENDING},
                                  {cudf::null_order::BEFORE});

  expect_columns_equal(*result, expect);
}

TYPED_TEST(FixedPointTestBothReps, FixedPointInterleave)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  for (int i = 0; i > -4; --i) {
    auto const ONE   = decimalXX{1, scale_type{i}};
    auto const TWO   = decimalXX{2, scale_type{i}};
    auto const THREE = decimalXX{3, scale_type{i}};
    auto const FOUR  = decimalXX{4, scale_type{i}};

    auto const a = wrapper<decimalXX>({ONE, THREE});
    auto const b = wrapper<decimalXX>({TWO, FOUR});

    auto const input    = cudf::table_view{std::vector<cudf::column_view>{a, b}};
    auto const expected = wrapper<decimalXX>({ONE, TWO, THREE, FOUR});
    auto const actual   = cudf::interleave_columns(input);

    expect_columns_equal(expected, actual->view());
  }
}

TYPED_TEST(FixedPointTestBothReps, FixedPointReductionProduct)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto const ONE   = decimalXX{1, scale_type{0}};
  auto const TWO   = decimalXX{2, scale_type{0}};
  auto const THREE = decimalXX{3, scale_type{0}};
  auto const FOUR  = decimalXX{4, scale_type{0}};
  auto const _24   = decimalXX{24, scale_type{0}};

  auto const in       = std::vector<decimalXX>{ONE, TWO, THREE, FOUR};
  auto const column   = wrapper<decimalXX>(in.cbegin(), in.cend());
  auto const expected = std::accumulate(in.cbegin(), in.cend(), ONE, std::multiplies<decimalXX>());
  auto const out_type = static_cast<cudf::column_view>(column).type();

  auto const result        = cudf::reduce(column, cudf::make_product_aggregation(), out_type);
  auto const result_scalar = static_cast<cudf::scalar_type_t<decimalXX>*>(result.get());

  EXPECT_EQ(result_scalar->value(), expected);
  EXPECT_EQ(result_scalar->value(), _24);
}

TYPED_TEST(FixedPointTestBothReps, FixedPointReductionSum)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto const ZERO  = decimalXX{0, scale_type{0}};
  auto const ONE   = decimalXX{1, scale_type{0}};
  auto const TWO   = decimalXX{2, scale_type{0}};
  auto const THREE = decimalXX{3, scale_type{0}};
  auto const FOUR  = decimalXX{4, scale_type{0}};
  auto const TEN   = decimalXX{10, scale_type{0}};

  auto const in       = std::vector<decimalXX>{ONE, TWO, THREE, FOUR};
  auto const column   = wrapper<decimalXX>(in.cbegin(), in.cend());
  auto const expected = std::accumulate(in.cbegin(), in.cend(), ZERO, std::plus<decimalXX>());
  auto const out_type = static_cast<cudf::column_view>(column).type();

  auto const result        = cudf::reduce(column, cudf::make_sum_aggregation(), out_type);
  auto const result_scalar = static_cast<cudf::scalar_type_t<decimalXX>*>(result.get());

  EXPECT_EQ(result_scalar->value(), expected);
  EXPECT_EQ(result_scalar->value(), TEN);
}

CUDF_TEST_PROGRAM_MAIN()

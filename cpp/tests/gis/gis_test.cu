#include <bitmask/BitMask.cuh>
#include "gtest/gtest.h"
#include <tests/utilities/cudf_test_utils.cuh>
#include <tests/utilities/cudf_test_fixtures.h>  // GdfTest
#include <cudf.h>     // create_gdf_column
#include <vector>

template <typename T>
struct GISTest : public GdfTest 
{
    std::vector<T> polygon_lats;
    std::vector<T> polygon_lons;
    std::vector<T> point_lats;
    std::vector<T> point_lons;
    std::vector<int32_t> inside_polygon;
    size_t total_points;

    gdf_col_pointer gdf_col_ptr_polygon_lats;
    gdf_col_pointer gdf_col_ptr_polygon_lons;
    gdf_col_pointer gdf_col_ptr_point_lats;
    gdf_col_pointer gdf_col_ptr_point_lons;
    gdf_col_pointer gdf_col_ptr_inside_points;

    gdf_column* gdf_raw_polygon_lats;
    gdf_column* gdf_raw_polygon_lons;
    gdf_column* gdf_raw_point_lats;
    gdf_column* gdf_raw_point_lons;
    gdf_column* gdf_raw_inside_polygon;

    // TODO: comment params
    void create_input(const std::initializer_list<T> &column_polygon_lats_list,
        const std::initializer_list<T> &column_polygon_lons_list,
        const std::initializer_list<T> &column_point_lats_list,
        const std::initializer_list<T> &column_point_lons_list,
        bool print = false)
    {
        polygon_lats = column_polygon_lats_list;
        polygon_lons = column_polygon_lons_list;
        point_lats = column_point_lats_list;
        point_lons = column_point_lons_list;

        if (polygon_lats.size() != polygon_lons.size())
        {
            std::cerr << "Polygon size doesn't match." << std::endl;
            return;
        }
        
        if (point_lats.size() != point_lons.size())
        {
            std::cerr << "Points size doesn't match." << std::endl;
            return;
        }

        total_points = point_lats.size();
        inside_polygon.resize(total_points, -1);

        gdf_col_ptr_polygon_lats = create_gdf_column(polygon_lats);
        gdf_col_ptr_polygon_lons = create_gdf_column(polygon_lons);
        gdf_col_ptr_point_lats = create_gdf_column(point_lats);
        gdf_col_ptr_point_lons = create_gdf_column(point_lons);
        gdf_col_ptr_inside_points = create_gdf_column(inside_polygon);

        gdf_raw_polygon_lats = gdf_col_ptr_polygon_lats.get();
        gdf_raw_polygon_lons = gdf_col_ptr_polygon_lons.get();
        gdf_raw_point_lats = gdf_col_ptr_point_lats.get();
        gdf_raw_point_lons = gdf_col_ptr_point_lons.get();
        gdf_raw_inside_polygon = gdf_col_ptr_inside_points.get();

        if(print)
        {
            std::cout << "\nSize of polygon: " << polygon_lats.size() << std::endl;
            std::cout << "Number of points: " << point_lats.size() << std::endl;
        }
    }

    // TODO: Implement  pip host
    std::vector<int32_t> compute_reference_pip(bool print = false)
    {
        // todo: pip host
        std::vector<int32_t> h_inside_polygon(total_points, 0);//default 0


        if(print)
        {
            std::cout << "\nReference result: " << std::endl;
            print_vector(h_inside_polygon);
            std::cout << std::endl;;
        }

        return h_inside_polygon;
    }

    std::vector<int32_t> compute_gdf_pip(bool print = false)
    {
        gdf_point_in_polygon(gdf_raw_polygon_lats, gdf_raw_polygon_lons, gdf_raw_point_lats, gdf_raw_point_lons, gdf_raw_inside_polygon);
      
        size_t output_size = gdf_raw_point_lats->size;
        std::vector<int32_t> host_inside_polygon(output_size);
      
        EXPECT_EQ(hipMemcpy(host_inside_polygon.data(), gdf_raw_inside_polygon->data, output_size * sizeof(int32_t), hipMemcpyDeviceToHost), hipSuccess);

        if(print)
        {
            std::cout << "\nGDF result: " << std::endl;
            print_vector(host_inside_polygon);
            std::cout << std::endl;;
        }

        return host_inside_polygon;
    }

    // TODO: Function to check the range for latitude and longitude
    
};


using Types = testing::Types<float, double>;

TYPED_TEST_CASE(GISTest, Types);

TYPED_TEST(GISTest, InsidePolygon)
{
    // Latitudes polygon, longitudes polygon, latitudes of query points, longitudes of query points, print = false
    this->create_input({0.0, 1.0, 1.0, 0.0, 0.0}, {0.0, 0.0, 1.0, 1.0, 0.0}, {0.4, 0.5, 0.2, 0.6}, {0.2, 0.6, 0.5, 0.8}, false);

    std::vector<int32_t> reference_pip_result = this->compute_reference_pip(true);
    std::vector<int32_t> gdf_pip_result = this->compute_gdf_pip(true);

    ASSERT_EQ(reference_pip_result.size(), gdf_pip_result.size()) << "Size of gdf result doesn't match";

    // Compare the GDF and reference solutions
    for(size_t i = 0; i < reference_pip_result.size(); ++i) {
        EXPECT_EQ(reference_pip_result[i], gdf_pip_result[i]);
    }
}

TYPED_TEST(GISTest, OutsidePolygon)
{
    this->create_input({0.0, 1.0, 1.0, 0.0, 0.0}, {0.0, 0.0, 1.0, 1.0, 0.0}, {-0.4, -0.5, -0.2}, {0.2, 0.6, 0.5}, false);

    std::vector<int32_t> reference_pip_result = this->compute_reference_pip(true);
    std::vector<int32_t> gdf_pip_result = this->compute_gdf_pip(true);

    ASSERT_EQ(reference_pip_result.size(), gdf_pip_result.size()) << "Size of gdf result doesn't match";

    // Compare the GDF and reference solutions
    for(size_t i = 0; i < reference_pip_result.size(); ++i) {
        EXPECT_EQ(reference_pip_result[i], gdf_pip_result[i]);
    }
}

TYPED_TEST(GISTest, EmptyPolygon)
{

}
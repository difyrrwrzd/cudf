#include "gtest/gtest.h"
#include <tests/utilities/cudf_test_utils.cuh>
#include <tests/utilities/cudf_test_fixtures.h>
#include <tests/utilities/column_wrapper.cuh>
#include <vector>

template <typename T>
struct GISTest : public GdfTest 
{
    std::vector<T> polygon_lats, polygon_lons, point_lats, point_lons;
    gdf_column *gdf_raw_polygon_lats, *gdf_raw_polygon_lons, *gdf_raw_point_lats, *gdf_raw_point_lons;

    void create_input(const std::initializer_list<T> &column_polygon_lats_list,
        const std::initializer_list<T> &column_polygon_lons_list,
        const std::initializer_list<T> &column_point_lats_list,
        const std::initializer_list<T> &column_point_lons_list,
        bool print = false)
    {
        polygon_lats = column_polygon_lats_list;
        polygon_lons = column_polygon_lons_list;
        point_lats = column_point_lats_list;
        point_lons = column_point_lons_list;

        EXPECT_EQ( polygon_lats.size(), polygon_lons.size() ) << "TEST: Polygon size doesn't match.";
        EXPECT_EQ( point_lats.size(), point_lons.size() ) << "TEST: Points size doesn't match." ;

        if(print)
        {
            std::cout << "\nSize of polygon: " << polygon_lats.size() << std::endl;
            std::cout << "Number of points: " << point_lats.size() << std::endl;
        }
    }

    T orientation(T p1_x, T p1_y, T p2_x, T p2_y, T p3_x, T p3_y)
    {
	    return ((p2_y - p1_y) * (p3_x - p2_x) - (p2_x - p1_x) * (p3_y - p2_y));
    }

    std::vector<int8_t> compute_reference_pip(bool print = false)
    {   
        size_t total_points = polygon_lats.size();
        std::vector<int8_t> h_inside_polygon(total_points, -1);

        for (size_t id_point = 0; id_point < total_points; ++id_point)
        {
            T point_lat = point_lats[id_point];
            T point_lon = point_lons[id_point];
            int count = 0;

            for (size_t poly_idx = 0; poly_idx < polygon_lats.size() - 1; ++poly_idx)
            {
                if(polygon_lons[poly_idx] <= point_lon && point_lon < polygon_lons[poly_idx + 1])
                {
                    if (orientation(polygon_lons[poly_idx], polygon_lats[poly_idx], polygon_lons[poly_idx + 1], polygon_lats[poly_idx + 1], point_lon, point_lat) > 0)
                    {
                        count++;
                    }
                }
                else if (point_lon <= polygon_lons[poly_idx] && polygon_lons[poly_idx + 1] < point_lon) 
                {
                    if (orientation(polygon_lons[poly_idx], polygon_lats[poly_idx], polygon_lons[poly_idx + 1], polygon_lats[poly_idx + 1], point_lon, point_lat) > 0)
                    {
                        count++;
                    }
                }
            }

            if ((count > 0) && (count % 2 == 0)) h_inside_polygon[id_point] = 1;
		    else h_inside_polygon[id_point] = 0;
        }

        if(print)
        {
            std::cout << "\nReference result: " << std::endl;
            print_vector(h_inside_polygon);
            std::cout << std::endl;;
        }

        return h_inside_polygon;
    }

    std::vector<int8_t> compute_gdf_pip(bool print = false)
    {
        // column_wrapper for tests
        cudf::test::column_wrapper<T> polygon_lat_wrapp{polygon_lats};
        cudf::test::column_wrapper<T> polygon_lon_wrapp{polygon_lons};
        cudf::test::column_wrapper<T> point_lat_wrapp{point_lats};
        cudf::test::column_wrapper<T> point_lon_wrapp{point_lons};

        gdf_raw_polygon_lats = polygon_lat_wrapp.get();
        gdf_raw_polygon_lons = polygon_lon_wrapp.get();
        gdf_raw_point_lats = point_lat_wrapp.get();
        gdf_raw_point_lons = point_lon_wrapp.get();

        gdf_column* inside_polygon_column = gdf_point_in_polygon(gdf_raw_polygon_lats, gdf_raw_polygon_lons, gdf_raw_point_lats, gdf_raw_point_lons);

        size_t total_points = polygon_lats.size();
        std::vector<int8_t> host_inside_polygon(total_points);
      
        EXPECT_EQ(hipMemcpy(host_inside_polygon.data(), inside_polygon_column->data, total_points * sizeof(int8_t), hipMemcpyDeviceToHost), hipSuccess);

        if(print)
        {
            std::cout << "\nGDF result: " << std::endl;
            print_vector(host_inside_polygon);
            std::cout << std::endl;;
        }
    
        return host_inside_polygon;
    }

    // TODO: Function to check the range for latitude and longitude  
};

using Types = testing::Types<double>;

TYPED_TEST_CASE(GISTest, Types);

TYPED_TEST(GISTest, InsidePolygon)
{
    // Latitudes polygon, longitudes polygon, latitudes of query points, longitudes of query points, print = false
    this->create_input({0.0, 1.0, 1.0, 0.0, 0.0}, {0.0, 0.0, 1.0, 1.0, 0.0}, {0.4, 0.5, 0.2, 0.6, 0.32, 0.78}, {0.2, 0.6, 0.5, 0.8, 0.41, 0.63}, false);

    std::vector<int8_t> reference_pip_result = this->compute_reference_pip(true);
    std::vector<int8_t> gdf_pip_result = this->compute_gdf_pip(true);

    ASSERT_EQ(reference_pip_result.size(), gdf_pip_result.size()) << "Size of gdf result doesn't match with reference result";

    // Compare the GDF and reference solutions
    for(size_t i = 0; i < reference_pip_result.size(); ++i) {
        EXPECT_EQ(reference_pip_result[i], gdf_pip_result[i]);
        
    }   
}

TYPED_TEST(GISTest, OutsidePolygon)
{
    this->create_input({0.0, 1.0, 1.0, 0.0, 0.0}, {0.0, 0.0, 1.0, 1.0, 0.0}, {-0.4, -0.5, -0.2, 1.25, 5.36}, {-0.2, 0.6, 0.5, 0.22, 8.21}, false);

    std::vector<int8_t> reference_pip_result = this->compute_reference_pip(false);
    std::vector<int8_t> gdf_pip_result = this->compute_gdf_pip(false);

    ASSERT_EQ(reference_pip_result.size(), gdf_pip_result.size()) << "Size of gdf result doesn't match with reference result";

    // Compare the GDF and reference solutions
    for(size_t i = 0; i < reference_pip_result.size(); ++i) {
        EXPECT_EQ(reference_pip_result[i], gdf_pip_result[i]);
    }
}
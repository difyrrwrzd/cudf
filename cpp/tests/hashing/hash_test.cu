#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/hashing.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/type_lists.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>

class HashTest : public cudf::test::BaseFixture {};

TEST_F(HashTest, MultiValue)
{
  using cudf::test::fixed_width_column_wrapper;
  using cudf::test::strings_column_wrapper;
  using cudf::test::expect_columns_equal;
  using cudf::experimental::bool8;

  auto const strings_col = strings_column_wrapper(
    {"",
    "The quick brown fox",
    "jumps over the lazy dog.",
    "All work and no play makes Jack a dull boy",
    "!\"#$%&\'()*+,-./0123456789:;<=>?@[\\]^_`{|}~"});

  using limits = std::numeric_limits<int32_t>;
  auto const ints_col = fixed_width_column_wrapper<int32_t>(
    {0, 100, -100, limits::min(), limits::max()});

  // Different truthy values should be equal
  auto const bools_col1 = fixed_width_column_wrapper<bool8>({0, 1, 1, 1, 0});
  auto const bools_col2 = fixed_width_column_wrapper<bool8>({0, 1, 2, 255, 0});

  using ts = cudf::timestamp_s;
  auto const secs_col = fixed_width_column_wrapper<ts>(
    {ts::duration::zero(), 100, -100, ts::duration::min(), ts::duration::max()});

  auto const input1 = cudf::table_view(
    {strings_col, ints_col, bools_col1, secs_col});
  auto const input2 = cudf::table_view(
    {strings_col, ints_col, bools_col2, secs_col});

  auto const output1 = cudf::hash(input1);
  auto const output2 = cudf::hash(input2);

  expect_columns_equal(output1->view(), output2->view());
}

TEST_F(HashTest, MultiValueNulls)
{
  using cudf::test::fixed_width_column_wrapper;
  using cudf::test::strings_column_wrapper;
  using cudf::test::expect_columns_equal;
  using cudf::experimental::bool8;

  // Nulls with different values should be equal
  auto const strings_col1 = strings_column_wrapper(
    {"",
    "The quick brown fox",
    "jumps over the lazy dog.",
    "All work and no play makes Jack a dull boy",
    "!\"#$%&\'()*+,-./0123456789:;<=>?@[\\]^_`{|}~"},
    {0, 1, 1, 0, 1});
  auto const strings_col2 = strings_column_wrapper(
    {"different but null",
    "The quick brown fox",
    "jumps over the lazy dog.",
    "I am Jack's complete lack of null value",
    "!\"#$%&\'()*+,-./0123456789:;<=>?@[\\]^_`{|}~"},
    {0, 1, 1, 0, 1});

  // Nulls with different values should be equal
  using limits = std::numeric_limits<int32_t>;
  auto const ints_col1 = fixed_width_column_wrapper<int32_t>(
    {0, 100, -100, limits::min(), limits::max()}, {1, 0, 0, 1, 1});
  auto const ints_col2 = fixed_width_column_wrapper<int32_t>(
    {0, -200, 200, limits::min(), limits::max()}, {1, 0, 0, 1, 1});

  // Nulls with different values should be equal
  // Different truthy values should be equal
  auto const bools_col1 = fixed_width_column_wrapper<bool8>(
    {0, 1, 0, 1, 1}, {1, 1, 0, 0, 1});
  auto const bools_col2 = fixed_width_column_wrapper<bool8>(
    {0, 2, 1, 0, 255}, {1, 1, 0, 0, 1});

  // Nulls with different values should be equal
  using ts = cudf::timestamp_s;
  auto const secs_col1 = fixed_width_column_wrapper<ts>(
    {ts::duration::zero(), 100, -100, ts::duration::min(), ts::duration::max()},
    {1, 0, 0, 1, 1});
  auto const secs_col2 = fixed_width_column_wrapper<ts>(
    {ts::duration::zero(), -200, 200, ts::duration::min(), ts::duration::max()},
    {1, 0, 0, 1, 1});

  auto const input1 = cudf::table_view(
    {strings_col1, ints_col1, bools_col1, secs_col1});
  auto const input2 = cudf::table_view(
    {strings_col2, ints_col2, bools_col2, secs_col2});

  auto const output1 = cudf::hash(input1);
  auto const output2 = cudf::hash(input2);

  expect_columns_equal(output1->view(), output2->view());
}

/*template <typename T>
class HashTestTyped : public cudf::test::BaseFixture {};

TYPED_TEST_CASE(HashTestTyped, cudf::test::FixedWidthTypes);

TYPED_TEST(HashTestTyped, SingleValue)
{
}*/

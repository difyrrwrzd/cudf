#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/column_wrapper.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/replace.hpp>

#include <cudf/cudf.h>

#include <gtest/gtest.h>

// This is the main test feature
template <class T>
struct ReplaceTest : public GdfTest
{

  ReplaceTest()
  {    
  }

  ~ReplaceTest()
  {
  }
};

// ugly.  only here to get around strict-aliasing compiler warnings
uint32_t hard_cast_float(float val)
{
   void *v_val = &val;
   return *((uint32_t*)v_val);
}
uint64_t hard_cast_double(double val)
{
   void *v_val = &val;
   return *((uint64_t*)v_val);
}

// Test for normalize_nans_and_nulls
TEST(ReplaceTest, NormalizeNansAndZeros)
{      
   int num_els = 8;   
   
   // floats
   {
      float els[] = { 32.5f, -0.0f, 111.0f, -NAN, NAN, 1.0f, 0.0f, 54.3f };   

      // copy the data to mutable device column
      auto test_data = cudf::make_numeric_column(cudf::data_type(cudf::FLOAT32), num_els, cudf::ALL_VALID, 0);      
      auto view = test_data->mutable_view();
      hipMemcpy(view.head(), els, sizeof(float) * num_els, hipMemcpyHostToDevice);
         
      cudf::normalize_nans_and_zeros(view);

      // get the data back
      hipMemcpy(els, view.head(), sizeof(float) * num_els, hipMemcpyDeviceToHost);      
      
      // can't compare nans and -nans directly since they will always be equal, so we'll compare against
      // bit patterns.       
      uint32_t nan = hard_cast_float(NAN);            
      EXPECT_TRUE(hard_cast_float(els[3]) == nan);
      EXPECT_TRUE(hard_cast_float(els[4]) == nan);
      EXPECT_TRUE(els[1] == 0.0f);
   }
   
   // doubles
   {
      double dels[] = { 32.5, -0.0, 111.0, -NAN, NAN, 1.0, 0.0, 54.3 };   

      // copy the data to mutable device column
      auto test_data = cudf::make_numeric_column(cudf::data_type(cudf::FLOAT64), num_els, cudf::ALL_VALID, 0);      
      auto view = test_data->mutable_view();
      hipMemcpy(view.head(), dels, sizeof(double) * num_els, hipMemcpyHostToDevice);
         
      cudf::normalize_nans_and_zeros(view);

      // get the data back
      hipMemcpy(dels, view.head(), sizeof(double) * num_els, hipMemcpyDeviceToHost);      
      
      // can't compare nans and -nans directly since they will always be equal, so we'll compare against
      // bit patterns.       
      uint64_t nan = hard_cast_double(NAN);            
      EXPECT_TRUE(hard_cast_double(dels[3]) == nan);
      EXPECT_TRUE(hard_cast_double(dels[4]) == nan);
      EXPECT_TRUE(dels[1] == 0.0);
   }
}
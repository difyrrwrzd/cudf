#include "hip/hip_runtime.h"
#include <gdf/gdf.h>
#include <gdf/errorutils.h>
#include <gdf/utils.h>
#include <vector>
#include <cassert>
#include <hipcub/hipcub.hpp>


using valid32_t = uint32_t;

// To account for if gdf_valid_type is not a 4 byte type,
// compute the RATIO of the number of bytes in gdf_valid_type
// to the 4 byte type being used for casting
constexpr size_t RATIO = sizeof(valid32_t) / sizeof(gdf_valid_type);

constexpr int block_size = 256;

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Counts the number of valid bits for the specified number of rows
   in the host vector of gdf_valid_type masks
 * 
 * @Param masks The host vector of masks whose bits will be counted
 * @Param num_rows The number of bits to count
 * 
 * @Returns  The number of valid bits in [0, num_rows) in the host vector of masks
 */
/* ----------------------------------------------------------------------------*/
size_t count_valid_bits_host(std::vector<gdf_valid_type> const & masks, const int num_rows)
{
  if((0 == num_rows) || (0 == masks.size())){
    return 0;
  }

  size_t count{0};

  // Count the valid bits for all masks except the last one
  for(size_t i = 0; i < (masks.size() - 1); ++i)
  {
    gdf_valid_type current_mask = masks[i];

    while(current_mask > 0)
    {
      current_mask &= (current_mask-1) ;
      count++;
    }
  }

  // Only count the bits in the last mask that correspond to rows
  int num_rows_last_mask = num_rows % GDF_VALID_BITSIZE;

  if(num_rows_last_mask == 0)
    num_rows_last_mask = GDF_VALID_BITSIZE;

  gdf_valid_type last_mask = *(masks.end() - 1);
  for(int i = 0; (i < num_rows_last_mask) && (last_mask > 0); ++i)
  {
    count += (last_mask & gdf_valid_type(1));
    last_mask >>= 1;
  }

  return count;
}


__global__ 
void count_valid_bits(valid32_t const * const __restrict__ masks32, const int num_masks32, int * const __restrict__ global_count)
{
  typedef hipcub::BlockReduce<int, block_size> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int cur_mask = threadIdx.x + blockIdx.x * gridDim.x;

  int my_count = 0;
  while(cur_mask < num_masks32)
  {
    my_count += __popc(masks32[cur_mask]);

    cur_mask += blockDim.x * gridDim.x;
  }

  int block_count = BlockReduce(temp_storage).Sum(my_count);

  if(threadIdx.x == 0)
    atomicAdd(global_count, block_count);
}

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Counts the number of valid bits for the specified number of rows
   in a validity bitmask.
 * 
 * @Param[in] masks The validity bitmask buffer in device memory
 * @Param[in] num_rows The number of bits to count
 * @Param[out] count The number of valid bits in the buffer from [0, num_rows)
 * 
 * @Returns  GDF_SUCCESS upon successful completion 
 */
/* ----------------------------------------------------------------------------*/
gdf_error gdf_count_nonzero_mask(gdf_valid_type const * masks, int num_rows, int * count)
{

  // Why am I getting an unused function warning error if I don't do this?
  gdf_is_valid(nullptr, 0);

  if((nullptr == masks) || (nullptr == count)){return GDF_DATASET_EMPTY;}
  if(0 == num_rows) {return GDF_SUCCESS;}

  assert(sizeof(valid32_t) >= sizeof(gdf_valid_type));

  // Number of gdf_valid_types in the validity bitmask
  const size_t num_masks = gdf_get_num_chars_bitmask(num_rows);

  // Number of 4 byte types in the validity bit mask 
  size_t num_masks32 = num_masks / RATIO;

  // If the total number of masks is not a multiple of the RATIO 
  // between the original mask type and the 4 byte masks type, then 
  // these "remainder" masks cannot be proccessed in the transform_reduce
  // and must be handled separately
  hipStream_t copy_stream;
  size_t num_remainder_masks = num_masks % RATIO;

  // If there are no remainder masks, the last mask still
  // needs to be handled separately
  if(0 == num_remainder_masks){
    num_remainder_masks = RATIO;
    num_masks32--;
  }

  std::vector<gdf_valid_type> remainder_masks(num_remainder_masks); 
  if(remainder_masks.size() > 0)
  {
    CUDA_TRY(hipStreamCreate(&copy_stream));

    // Copy the remainder masks to the host
    // FIXME: Is this endian safe?
    const gdf_valid_type * first_remainder_mask = &(masks[num_masks - num_remainder_masks]);

    CUDA_TRY( hipMemcpyAsync(remainder_masks.data(), 
                              first_remainder_mask, 
                              num_remainder_masks * sizeof(gdf_valid_type), 
                              hipMemcpyDeviceToHost,
                              copy_stream) );
  }

  int * d_count32;
  hipStream_t count_stream;
  if(num_masks32 > 0)
  {
    CUDA_TRY(hipStreamCreate(&count_stream));
    // Cast validity buffer to 4 byte type
    valid32_t const * masks32 = reinterpret_cast<valid32_t const *>(masks);

    CUDA_TRY(hipMalloc(&d_count32, sizeof(int)));
    CUDA_TRY(hipMemsetAsync(d_count32, 0, sizeof(int),count_stream));

    const int grid_size = (num_masks32 + block_size - 1)/block_size;
    count_valid_bits<<<grid_size, block_size,0,count_stream>>>(masks32, num_masks32, d_count32);

    CUDA_TRY( hipGetLastError() );
  }

  // Count the number of valid bits in the remainder masks
  size_t remainder_count = 0;
  if(remainder_masks.size() > 0)
  {
    CUDA_TRY(hipStreamSynchronize(copy_stream));
    CUDA_TRY(hipStreamDestroy(copy_stream));
    remainder_count = count_valid_bits_host(remainder_masks, num_rows);
  }

  int count32{0};
  if(num_masks32 > 0)
  {
    CUDA_TRY(hipMemcpyAsync(&count32, d_count32, sizeof(int), hipMemcpyDeviceToHost,count_stream));
    CUDA_TRY(hipStreamSynchronize(count_stream));
    CUDA_TRY(hipStreamDestroy(count_stream));
  }

  // The final count of valid bits is the sum of the result from the
  // transform_reduce and the remainder masks
  *count = (count32 + remainder_count);

  return GDF_SUCCESS;
}




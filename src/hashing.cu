#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gdf/gdf.h>
#include <gdf/errorutils.h>

#include "join/joining.h"
#include "gdf_table.cuh"
#include "hashmap/hash_functions.cuh"

constexpr int HASH_KERNEL_BLOCK_SIZE = 256;
constexpr int HASH_KERNEL_ROWS_PER_THREAD = 1;

// convert to int dtype with the same size
gdf_dtype to_int_dtype(gdf_dtype type)
{
  switch (type) {
    case GDF_INT8:
    case GDF_INT16:
    case GDF_INT32:
    case GDF_INT64:
      return type;
    case GDF_FLOAT32:
      return GDF_INT32;
    case GDF_FLOAT64:
      return GDF_INT64;
    default:
      return GDF_invalid;
  }
}

__device__ __inline__
uint32_t hashed(void *ptr, int int_dtype, int index)
{
  // TODO: add switch to select the right hash class, currently we only support Murmur3 anyways
  switch (int_dtype) {
  case GDF_INT8:  { default_hash<int8_t> hasher; return hasher(((int8_t*)ptr)[index]); }
  case GDF_INT16: { default_hash<int16_t> hasher; return hasher(((int16_t*)ptr)[index]); }
  case GDF_INT32: { default_hash<int32_t> hasher; return hasher(((int32_t*)ptr)[index]); }
  case GDF_INT64: { default_hash<int64_t> hasher; return hasher(((int64_t*)ptr)[index]); }
  default:
    return 0;
  }
}

template<typename size_type>
__device__ __inline__
void hash_combine(size_type &seed, const uint32_t hash_val)
{
  seed ^= hash_val + 0x9e3779b9 + (seed<<6) + (seed>>2);
}

// one thread handles multiple rows
// d_col_data[i]: column's data (on device)
// d_col_int_dtype[i]: column's dtype (converted to int) 
__global__ void hash_cols(int num_rows, int num_cols, void **d_col_data, gdf_dtype *d_col_int_dtype, int *d_output)
{
  for (int row = threadIdx.x + blockIdx.x * blockDim.x; row < num_rows; row += blockDim.x * gridDim.x) {
    uint32_t seed = 0;
    for (int col = 0; col < num_cols; col++) {
      uint32_t hash_val = hashed(d_col_data[col], d_col_int_dtype[col], row);
      hash_combine(seed, hash_val);
    }
    d_output[row] = seed;
  }
}

gdf_error gdf_hash(int num_cols, gdf_column **input, gdf_hash_func hash, gdf_column *output)
{
  // check that all columns have the same size
  for (int i = 0; i < num_cols; i++)
    if (i > 0 && input[i]->size != input[i-1]->size) return GDF_COLUMN_SIZE_MISMATCH;
  // check that the output dtype is int32
  // TODO: do we need to support int64 as well?
  if (output->dtype != GDF_INT32) return GDF_UNSUPPORTED_DTYPE;
  int64_t num_rows = input[0]->size;

  // copy data pointers to device
  void **d_col_data, **h_col_data;
  hipMalloc(&d_col_data, num_cols * sizeof(void*));
  hipHostMalloc(&h_col_data, num_cols * sizeof(void*));
  for (int i = 0; i < num_cols; i++)
    h_col_data[i] = input[i]->data;
  hipMemcpy(d_col_data, h_col_data, num_cols * sizeof(void*), hipMemcpyDefault);

  // copy dtype (converted to int) to device
  gdf_dtype *d_col_int_dtype, *h_col_int_dtype;
  hipMalloc(&d_col_int_dtype, num_cols * sizeof(gdf_dtype));
  hipHostMalloc(&h_col_int_dtype, num_cols * sizeof(gdf_dtype));
  for (int i = 0; i < num_cols; i++)
    h_col_int_dtype[i] = to_int_dtype(input[i]->dtype);
  hipMemcpy(d_col_int_dtype, h_col_int_dtype, num_cols * sizeof(gdf_dtype), hipMemcpyDefault);

  // launch a kernel
  const int rows_per_block = HASH_KERNEL_BLOCK_SIZE * HASH_KERNEL_ROWS_PER_THREAD;
  const int64_t grid = (num_rows + rows_per_block-1) / rows_per_block;
  hash_cols<<<grid, HASH_KERNEL_BLOCK_SIZE>>>(num_rows, num_cols, d_col_data, d_col_int_dtype, (int32_t*)output->data);

  // TODO: do we need to synchronize here
  hipDeviceSynchronize();
  CUDA_CHECK_LAST();

  // free temp memory
  hipFree(d_col_data);
  hipHostFree(h_col_data);
  hipFree(d_col_int_dtype);
  hipHostFree(h_col_int_dtype);

  return GDF_SUCCESS;

}


template < template <typename> class hash_function,
           typename size_type>
void hash_partition_gdf_table(gdf_table<size_type> const & input_table,
                              gdf_table<size_type> const & table_to_hash,
                              const size_type num_partitions,
                              size_type partition_offsets[],
                              gdf_table<size_type> & partitioned_output)
{

}

/* --------------------------------------------------------------------------*/
/**
 * @brief Computes the hash values of the specified rows in the input columns and
 * bins the hash values into the desired number of partitions. Rearranges the input
 * columns such that rows with hash values in the same bin are contiguous.
 *
 * @Param[in] num_input_cols The number of columns in the input columns
 * @Param[in] input[] The input set of columns
 * @Param[in] columns_to_hash[] Indices of the columns in the input set to hash
 * @Param[in] num_cols_to_hash The number of columns to hash
 * @Param[in] num_partitions The number of partitions to rearrange the input rows into
 * @Param[out] partitioned_output Preallocated gdf_columns to hold the rearrangement
 * of the input columns into the desired number of partitions
 * @Param[out] partition_offsets Preallocated array the size of the number of 
 * partitions. Where partition_offsets[i] indicates the starting position 
 * of partition 'i'
 * @Param[in] hash The hash function to use
 *
 * @Returns  If the operation was successful, returns GDF_SUCCESS
 */
/* ----------------------------------------------------------------------------*/
gdf_error gdf_hash_partition(int num_input_cols,
                             gdf_column * input[],
                             int columns_to_hash[],
                             int num_cols_to_hash,
                             int num_partitions,
                             gdf_column * partitioned_output[],
                             int partition_offsets[],
                             gdf_hash_func hash)
{
  // Use int until gdf API is updated to use something other than int
  // for ordinal variables
  using size_type = int;

  // Ensure all the inputs are non-zero and not null
  if((0 == num_input_cols) 
      || (0 == num_cols_to_hash)
      || (0 == num_partitions)
      || (nullptr == input) 
      || (nullptr == partitioned_output)
      || (nullptr == columns_to_hash)
      || (nullptr == partition_offsets))
  {
    return GDF_INVALID_API_CALL;
  }

  // check that the columns data are not null, have matching types,
  // and the same number of rows
  for (size_type i = 0; i < num_input_cols; i++) {
    if( (nullptr == input[i]->data) 
        || (nullptr == partitioned_output[i]->data))
      return GDF_DATASET_EMPTY;

    if(input[i]->dtype != partitioned_output[i]->dtype) 
      return GDF_PARTITION_DTYPE_MISMATCH;

    if((input[0]->size != input[i]->size) 
        || (input[0]->size != partitioned_output[i]->size))
      return GDF_COLUMN_SIZE_MISMATCH;
  }

  // Wrap input and output columns in gdf_table
  std::unique_ptr< const gdf_table<size_type> > input_table{new gdf_table<size_type>(num_input_cols, input)};
  std::unique_ptr< gdf_table<size_type> > output_table{new gdf_table<size_type>(num_input_cols, partitioned_output)};

  // Create vector of pointers to columns that will be hashed
  std::vector<gdf_column *> gdf_columns_to_hash(num_cols_to_hash);
  for(size_type i = 0; i < num_cols_to_hash; ++i)
  {
    gdf_columns_to_hash[i] = input[columns_to_hash[i]];
  }
  // Create a separate table of the columns to be hashed
  std::unique_ptr< const gdf_table<size_type> > table_to_hash {new gdf_table<size_type>(num_cols_to_hash, 
                                                                                        gdf_columns_to_hash.data())};
  switch(hash)
  {
    case GDF_HASH_MURMUR3:
      {
        hash_partition_gdf_table<MurmurHash3_32>(*input_table, 
                                                 *table_to_hash,
                                                 num_partitions,
                                                 partition_offsets,
                                                 *output_table);
        break;
      }
    default:
      return GDF_INVALID_HASH_FUNCTION;
  }

  return GDF_SUCCESS;
}


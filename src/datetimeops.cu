
#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>


#include <hip/hip_runtime.h>
#include <vector>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/iterator_adaptor.h>
#include <thrust/device_vector.h>




/*
 * Brainstorming
 *
 * functions needed:
 * extract_datetime_year
 * extract_datetime_month
 * extract_datetime_day
 * extract_datetime_hour
 * extract_datetime_minute
 * extract_datetime_second
 *
 * input formats:
 * date64
 * date32
 * timestamp with second, milisecond, microsecond, nanosecond
 *
 * these functions depend on starting dateformat may need to just implement them all?
 *
 *
 */


/* original unittime to date functions
 *
 * int unixDate = unixTime/86400000;
		int totalDays = 719469 + unixDate;

		int year = 400*totalDays/146097;
		totalDays -= (365*year + year/4 - year/100 + year/400);
		int month = (totalDays*5 + 457)/153;

		int day = totalDays - ((153*month - 457)/5);

		if (day == 0){
			month--;
			day = totalDays - ((153*month - 457)/5);
		}

		if(month > 12){
			month -= 12;
			year++;
		}

		another one more complex
 *
 * from http://howardhinnant.github.io/date_algorithms.html
 * int z = unixTime/86400000;
 * z += 719468;
    const Int era = (z >= 0 ? z : z - 146096) / 146097;
    const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
    const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
    const Int y = static_cast<Int>(yoe) + era * 400;
    const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
    const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
    const unsigned d = doy - (153*mp+2)/5 + 1;                             // [1, 31]
    const unsigned m = mp + (mp < 10 ? 3 : -9);                            // [1, 12]
    return std::tuple<Int, unsigned, unsigned>(y + (m <= 2), m, d);
 */






struct gdf_extract_datetime_year_date64_op : public thrust::unary_function<int64_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		const int z = ((unixTime >= 0 ? unixTime : unixTime - 86399999) / 86400000) + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
		const unsigned d = doy - (153*mp+2)/5 + 1;                             // [1, 31]
		const unsigned m = mp + (mp < 10 ? 3 : -9);                            // [1, 12]
		if (m <= 2)
			return y + 1;
		else
			return y;
	}
};

struct gdf_extract_datetime_month_date64_op : public thrust::unary_function<int64_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		const int z = ((unixTime >= 0 ? unixTime : unixTime - 86399999) / 86400000) + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
		const unsigned d = doy - (153*mp+2)/5 + 1;                             // [1, 31]
		return mp + (mp < 10 ? 3 : -9);                            // [1, 12]

	}
};

struct gdf_extract_datetime_day_date64_op : public thrust::unary_function<int64_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		const int z = ((unixTime >= 0 ? unixTime : unixTime - 86399999) / 86400000) + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
		return doy - (153*mp+2)/5 + 1;                             // [1, 31]
	}
};

/*

1528996790 unix time
5:19:50 pm UTC | Thursday, June 14, 2018

1528935590000
'2018-06-14 00:19:50'
 */

struct gdf_extract_datetime_hour_date64_op : public thrust::unary_function<int64_t, int16_t>
{
	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		return (unixTime % 86400000)/3600000;
	}
};

struct gdf_extract_datetime_minute_date64_op : public thrust::unary_function<int64_t, int16_t>
{
	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		return (unixTime % 3600000)/60000 ;
	}
};

struct gdf_extract_datetime_second_date64_op : public thrust::unary_function<int64_t, int16_t>
{
	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		return (unixTime % 60000)/1000;
	}
};

struct gdf_extract_datetime_millisecond_date64_op : public thrust::unary_function<int64_t, int16_t>
{
	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		return unixTime % 1000;
	}
};

struct gdf_extract_datetime_year_date32_op : public thrust::unary_function<int32_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int32_t unixDate) // unixDate is days since the UNIX epoch
	{
		int totalDays = 719469 + unixDate;
		int year = 400*totalDays/146097;
		totalDays -= (365*year + year/4 - year/100 + year/400);
		int month = (totalDays*5 + 457)/153;
		if ((totalDays - ((153*month - 457)/5)) == 0){
			month--;
		}
		if(month > 12){
			year++;
		}
		return year;
	}
};

struct gdf_extract_datetime_month_date32_op : public thrust::unary_function<int32_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int32_t unixDate) // unixDate is days since the UNIX epoch
	{
		int totalDays = 719469 + unixDate;
		int year = 400*totalDays/146097;
		totalDays -= (365*year + year/4 - year/100 + year/400);
		int month = (totalDays*5 + 457)/153;
		if ((totalDays - ((153*month - 457)/5)) == 0){
			month--;
		}
		if(month > 12){
			month -= 12;
		}
		return month;
	}
};

struct gdf_extract_datetime_day_date32_op : public thrust::unary_function<int32_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int32_t unixDate) // unixDate is days since the UNIX epoch
	{
		int totalDays = 719469 + unixDate;

		int year = 400*totalDays/146097;
		totalDays -= (365*year + year/4 - year/100 + year/400);
		int month = (totalDays*5 + 457)/153;

		int day = totalDays - ((153*month - 457)/5);

		if (day == 0){
			month--;
			day = totalDays - ((153*month - 457)/5);
		}
		return day;
	}
};





gdf_error gdf_extract_datetime_year(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);  // WSM do we want extracted values to be other than 16-bit?

	hipStream_t stream;
	hipStreamCreate(&stream);

	// WSM DO WE NEED TO DO THIS? DO WE WANT TO DO IT THIS WAY?
	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_year_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	}else if (input->dtype == GDF_DATE32) {
		thrust::device_ptr<int32_t> input_ptr((int32_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_year_date32_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);
	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_month(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);  // WSM do we want extracted values to be other than 16-bit?

	hipStream_t stream;
	hipStreamCreate(&stream);

	// WSM DO WE NEED TO DO THIS? DO WE WANT TO DO IT THIS WAY?
	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_month_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	}else if (input->dtype == GDF_DATE32) {
		thrust::device_ptr<int32_t> input_ptr((int32_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_month_date32_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);
	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_day(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);  // WSM do we want extracted values to be other than 16-bit?

	hipStream_t stream;
	hipStreamCreate(&stream);

	// WSM DO WE NEED TO DO THIS? DO WE WANT TO DO IT THIS WAY?
	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_day_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	}else if (input->dtype == GDF_DATE32) {
		thrust::device_ptr<int32_t> input_ptr((int32_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_day_date32_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);
	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_hour(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);  // WSM do we want extracted values to be other than 16-bit?
	GDF_REQUIRE(input->dtype != GDF_DATE32, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	// WSM DO WE NEED TO DO THIS? DO WE WANT TO DO IT THIS WAY?
	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_hour_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_minute(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);  // WSM do we want extracted values to be other than 16-bit?
	GDF_REQUIRE(input->dtype != GDF_DATE32, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	// WSM DO WE NEED TO DO THIS? DO WE WANT TO DO IT THIS WAY?
	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_minute_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_second(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);  // WSM do we want extracted values to be other than 16-bit?
	GDF_REQUIRE(input->dtype != GDF_DATE32, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	// WSM DO WE NEED TO DO THIS? DO WE WANT TO DO IT THIS WAY?
	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_second_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}




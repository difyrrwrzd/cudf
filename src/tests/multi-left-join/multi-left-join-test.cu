#include <cstdlib>
#include <iostream>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/gather.h>

#include "gtest/gtest.h"
#include "gmock/gmock.h"
#include <gdf/gdf.h>
#include <gdf/cffi/functions.h>

#include <moderngpu/kernel_sortedsearch.hxx>
#include <moderngpu/kernel_mergesort.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_load_balance.hxx>
#include "../../joining.h"


using namespace testing;
using namespace std;
using namespace mgpu;


gdf_column
create_gdf_column(thrust::device_vector<int> &d) {
    gdf_column c = {thrust::raw_pointer_cast(d.data()), nullptr, d.size(), GDF_INT32, TIME_UNIT_NONE};
    return c;
}

gdf_column
create_gdf_column(mem_t<int> &d) {
      gdf_column c = {d.data(), nullptr, d.size(), GDF_INT32, TIME_UNIT_NONE};
          return c;
}

std::vector<int> host_vec(thrust::device_vector<int> &dev_vec) {
    std::vector<int> data(dev_vec.size());
    thrust::copy(dev_vec.begin(), dev_vec.end(), data.begin());
    return data;
}

gdf_error
call_gdf_test(
        std::array<thrust::device_vector<int>, 3> &l,
        std::array<thrust::device_vector<int>, 3> &r,
        thrust::device_vector<int> &out_left_pos,
        thrust::device_vector<int> &out_right_pos,
        const int index) {
    std::vector<int> l0{0, 0, 4, 5, 5};
    std::vector<int> l1{1, 2, 2, 3, 4};
    std::vector<int> l2{1, 1, 3, 1, 2};
    std::vector<int> r0{0, 0, 2, 3, 5};
    std::vector<int> r1{1, 2, 3, 3, 4};
    std::vector<int> r2{3, 3, 2, 1, 1};

    thrust::device_vector<int> dl0 = l0; thrust::swap(dl0, l[0]);
    thrust::device_vector<int> dl1 = l1; thrust::swap(dl1, l[1]);
    thrust::device_vector<int> dl2 = l2; thrust::swap(dl2, l[2]);
    thrust::device_vector<int> dr0 = r0; thrust::swap(dr0, r[0]);
    thrust::device_vector<int> dr1 = r1; thrust::swap(dr1, r[1]);
    thrust::device_vector<int> dr2 = r2; thrust::swap(dr2, r[2]);

    gdf_column gdl0 = create_gdf_column(l[0]);
    gdf_column gdl1 = create_gdf_column(l[1]);
    gdf_column gdl2 = create_gdf_column(l[2]);

    gdf_column gdr0 = create_gdf_column(r[0]);
    gdf_column gdr1 = create_gdf_column(r[1]);
    gdf_column gdr2 = create_gdf_column(r[2]);

    gdf_column* gl[3] = {&gdl0, &gdl1, &gdl2};
    gdf_column* gr[3] = {&gdr0, &gdr1, &gdr2};
    gdf_join_result_type *out;
    gdf_error err = gdf_multi_left_join_generic(index, gl, gr, &out);

    size_t len = gdf_join_result_size(out);
    size_t hlen = len/2;
    int* out_ptr = reinterpret_cast<int*>(gdf_join_result_data(out));
    thrust::device_vector<int> out_data(out_ptr, out_ptr + len);

    thrust::sort_by_key(out_data.begin() + hlen, out_data.end(), out_data.begin());
    thrust::sort_by_key(out_data.begin(), out_data.begin() + hlen, out_data.begin() + hlen);
    out_left_pos.resize(hlen);
    out_right_pos.resize(hlen);

    thrust::copy(out_data.begin(), out_data.begin() + out_left_pos.size(), out_left_pos.begin());
    thrust::copy(out_data.begin() + out_right_pos.size(), out_data.end(), out_right_pos.begin());
    return err;
}

TEST(gdf_multi_left_join_TEST, case1) {
    std::array<thrust::device_vector<int>, 3> l;
    std::array<thrust::device_vector<int>, 3> r;
    thrust::device_vector<int> l_pos;
    thrust::device_vector<int> r_pos;
    auto err = call_gdf_test(l, r, l_pos, r_pos, 1);
    thrust::device_vector<int> map_out(l_pos.size());

    EXPECT_THAT(host_vec(l_pos), ElementsAre(0, 0, 1, 1, 2, 3, 4));
    EXPECT_THAT(host_vec(r_pos), ElementsAre(0, 1, 0, 1, -1, 4, 4));

    thrust::gather(l_pos.begin(), l_pos.end(), l[0].begin(), map_out.begin());
    EXPECT_THAT(host_vec(map_out), ElementsAre(0, 0, 0, 0, 4, 5, 5));

    ASSERT_EQ(err, GDF_SUCCESS);
}

TEST(gdf_multi_left_join_TEST, case2) {
    std::array<thrust::device_vector<int>, 3> l;
    std::array<thrust::device_vector<int>, 3> r;
    thrust::device_vector<int> l_pos;
    thrust::device_vector<int> r_pos;
    auto err = call_gdf_test(l, r, l_pos, r_pos, 2);
    thrust::device_vector<int> map_out(l_pos.size());

    EXPECT_THAT(host_vec(l_pos), ElementsAre(0, 1, 2, 3, 4));

    {
        thrust::gather(l_pos.begin(), l_pos.end(), l[0].begin(), map_out.begin());
        EXPECT_THAT(host_vec(map_out), ElementsAre(0, 0, 4, 5, 5));
    }

    {
        thrust::gather(l_pos.begin(), l_pos.end(), l[1].begin(), map_out.begin());
        EXPECT_THAT(host_vec(map_out), ElementsAre(1, 2, 2, 3, 4));
    }

    ASSERT_EQ(err, GDF_SUCCESS);
}

TEST(gdf_multi_left_join_TEST, case3) {
    std::array<thrust::device_vector<int>, 3> l;
    std::array<thrust::device_vector<int>, 3> r;
    thrust::device_vector<int> l_pos;
    thrust::device_vector<int> r_pos;
    auto err = call_gdf_test(l, r, l_pos, r_pos, 2);
    thrust::device_vector<int> map_out(l_pos.size());

    EXPECT_THAT(host_vec(l_pos), ElementsAre(0, 1, 2, 3, 4));

    {
        thrust::gather(l_pos.begin(), l_pos.end(), l[0].begin(), map_out.begin());
        EXPECT_THAT(host_vec(map_out), ElementsAre(0, 0, 4, 5, 5));
    }

    {
        thrust::gather(l_pos.begin(), l_pos.end(), l[1].begin(), map_out.begin());
        EXPECT_THAT(host_vec(map_out), ElementsAre(1, 2, 2, 3, 4));
    }

    {
        thrust::gather(l_pos.begin(), l_pos.end(), l[2].begin(), map_out.begin());
        EXPECT_THAT(host_vec(map_out), ElementsAre(1, 1, 3, 1, 2));
    }

    ASSERT_EQ(err, GDF_SUCCESS);
}


TEST(gdf_foo_sample_TEST, case1) {
    standard_context_t context;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds=0,sortms=0,hashms = 0;
    for (int countSize=1e3; countSize <=1e8; countSize*=10){ 
        int countA=countSize;
        int countB=countSize;
        for(int maxkey=1e4; maxkey<=1e8; maxkey*=10){
            mem_t<int> dataA = fill_random(0, maxkey, countA, false, context);
            mem_t<int> dataB = fill_random(0, maxkey, countB, false, context);
            hipEventRecord(start); 
            mergesort(dataA.data(), countA, less_t<int>(), context);
            mergesort(dataB.data(), countB, less_t<int>(), context);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&sortms, start, stop);
            // printf("Sorting, %10d, %10d, %8.5f, %10.0f\n", 
            printf("Sorting,%d,%d,%f,%f\n", 
                countA, maxkey, sortms, float(1000*countA)/(float)sortms);
            mem_t<int> common;
            hipEventRecord(start);
            //common = inner_join(dataA.data(), countA, dataB.data(), countB, less_t<int>() , context);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            // printf("time: %f  - common elements %d\n", milliseconds,common.size());
            mem_t<int> dataA1 = fill_random(0, maxkey, countA, false, context);
            mem_t<int> dataA2 = fill_random(0, maxkey, countA, false, context);
            mem_t<int> dataA3 = fill_random(0, maxkey, countA, false, context);
            mem_t<int> dataB1 = fill_random(0, maxkey, countB, false, context);
            mem_t<int> dataB2 = fill_random(0, maxkey, countB, false, context);
            mem_t<int> dataB3 = fill_random(0, maxkey, countB, false, context);
            gdf_column gdl0 = create_gdf_column(dataA1);
            gdf_column gdl1 = create_gdf_column(dataA2);
            gdf_column gdl2 = create_gdf_column(dataA3);
            gdf_column gdr0 = create_gdf_column(dataB1);
            gdf_column gdr1 = create_gdf_column(dataB2);
            gdf_column gdr2 = create_gdf_column(dataB3);
            gdf_column* gl[3] = {&gdl0, &gdl1, &gdl2};
            gdf_column* gr[3] = {&gdr0, &gdr1, &gdr2};
            gdf_join_result_type *out;
            // gdf_error err = gdf_multi_left_join_generic(1, gl, gr, &out);
            hipEventRecord(start); 
            gdf_error err = gdf_multi_left_join_generic(1, gl, gr, &out);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&hashms, start, stop);
            // printf("Hashing, %10d, %10d, %8.5f, %10.0f\n", 
            printf("Hashing,%d,%d,%f,%f\n", 
                countA,  maxkey, hashms, float(1000*countA)/(float)hashms);
                          
            // gdf_error err = gdf_inner_join_i32(&gdl0, &gdr0 , &out);
        }
    }
}


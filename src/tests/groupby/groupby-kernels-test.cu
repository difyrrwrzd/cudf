#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdlib>
#include <iostream>
#include <vector>
#include <unordered_map>
#include <random>

#include <thrust/device_vector.h>
#include <thrust/unique.h>
#include <thrust/sort.h>

#include "gtest/gtest.h"
#include <../../src/groupby/hash/groupby_kernels.cuh>
#include <../../src/groupby/hash/groupby_compute_api.h>
#include <../../src/groupby/hash/aggregation_operations.cuh>



/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  This file is for unit testing all functions and kernels that are below the
 * public libgdf groupby API.
 */
/* ----------------------------------------------------------------------------*/



// This is necessary to do a parametrized typed-test over multiple template arguments
template <typename Key, typename Value, template <typename T> class Aggregation_Operator>
struct KeyValueTypes
{
  using key_type = Key;
  using value_type = Value;
  using op_type = Aggregation_Operator<value_type>;
};

// A new instance of this class will be created for each *TEST(GroupByTest, ...)
// Put all repeated stuff for each test here
template <class T>
struct GroupByTest : public testing::Test 
{
  using key_type = typename T::key_type;
  using value_type = typename T::value_type;
  using op_type = typename T::op_type;
  using map_type = concurrent_unordered_map<key_type, value_type, std::numeric_limits<key_type>::max()>;
  using size_type = typename map_type::size_type;

  std::unique_ptr<map_type> the_map;

  const key_type unused_key = std::numeric_limits<key_type>::max();
  const value_type unused_value = op_type::IDENTITY;

  size_type hash_table_size;
  size_type input_size;

  const int THREAD_BLOCK_SIZE{256};

  std::vector<key_type> groupby_column;
  std::vector<value_type> aggregation_column;

  thrust::device_vector<key_type> d_groupby_column;
  thrust::device_vector<value_type> d_aggregation_column;

  key_type *d_groupby_result{nullptr};
  value_type *d_aggregation_result{nullptr};


  GroupByTest(const size_type _hash_table_size = 10000)
    : hash_table_size(_hash_table_size), the_map(new map_type(_hash_table_size, op_type::IDENTITY))
  {
  }

  ~GroupByTest()
  {
    hipFree(d_groupby_result);
    hipFree(d_aggregation_result);
  }

  std::pair<key_type*, value_type*>
  create_input(const int num_keys, const int num_values_per_key, const int max_key = RAND_MAX, const int max_value = RAND_MAX, bool print = false, const int ratio = 1) 
  {

    input_size = num_keys * num_values_per_key;

    hash_table_size = ratio * input_size;

    this->the_map.reset(new map_type(hash_table_size, unused_value));

    groupby_column.reserve(input_size);
    aggregation_column.reserve(input_size);

    // Always use the same seed so the random sequence is the same each time
    std::srand(0);

    for(int i = 0; i < num_keys; ++i )
    {
      // Create random key
      key_type current_key = std::rand() % max_key;

      // Don't use unused_key
      while(current_key == this->unused_key)
      {
        current_key = std::rand();
      }

      // For the current key, generate random values
      for(int j = 0; j < num_values_per_key; ++j)
      {
        value_type current_value = std::rand() % max_value;

        // Don't use unused_value
        while(current_value == this->unused_value)
        {
          current_value = std::rand() % max_value;
        }

        // Store current key and value
        groupby_column.push_back(current_key);
        aggregation_column.push_back(current_value);

      }
    }

    d_groupby_column = groupby_column;
    d_aggregation_column = aggregation_column;

    return std::make_pair(thrust::raw_pointer_cast(d_groupby_column.data()), 
                          thrust::raw_pointer_cast(d_aggregation_column.data()));
  }

  template <class aggregation_operation>
  std::map<key_type, value_type> compute_reference_solution(bool print = false)
  {
    std::map<key_type, value_type> expected_values;

    aggregation_operation op;

    for(size_t i = 0; i < groupby_column.size(); ++i){

      key_type current_key = groupby_column[i];
      value_type current_value = aggregation_column[i];

      // Use a STL map to keep track of the aggregation for each key
      auto found = expected_values.find(current_key);

      // Key doesn't exist yet, insert it
      if(found == expected_values.end())
      {
        // To support operations like `count`, on the first insert, perform the
        // operation on the new value and the operation's identity value and store the result
        current_value = op(current_value, aggregation_operation::IDENTITY);

        expected_values.insert(std::make_pair(current_key,current_value)); 

        if(print)
          std::cout << "First Insert of Key: " << current_key << " value: " << current_value << std::endl;
      }
      // Key exists, update the value with the operator
      else
      {
        value_type new_value = op(current_value, found->second);
        if(print)
          std::cout << "Insert of Key: " << current_key << " inserting value: " << current_value 
            << " storing: " << new_value << std::endl;
        found->second = new_value;
      }
    }

    return expected_values;
  }

  void build_aggregation_table_device(std::pair<key_type*, value_type*> input)
  {

    const dim3 grid_size ((this->input_size + this->THREAD_BLOCK_SIZE - 1) / this->THREAD_BLOCK_SIZE, 1, 1);
    const dim3 block_size (this->THREAD_BLOCK_SIZE, 1, 1);

    key_type * d_group = input.first;
    value_type * d_agg = input.second;

    hipDeviceSynchronize();
    build_aggregation_table<<<grid_size, block_size>>>((this->the_map).get(), d_group, d_agg, this->input_size, op_type());
    hipDeviceSynchronize(); 

  }

  void verify_aggregation_table(std::map<key_type, value_type> const & expected_values){

    for(auto const &expected : expected_values)
    {
      key_type test_key = expected.first;

      value_type expected_value = expected.second;

      auto found = this->the_map->find(test_key);

      ASSERT_NE(this->the_map->end(), found) << "Key is: " << test_key;

      value_type test_value = found->second;

      if(std::is_integral<value_type>::value){
        EXPECT_EQ(expected_value, test_value) << "Key is: " << test_key;
      }
      else if(std::is_same<value_type, float>::value){
        EXPECT_FLOAT_EQ(expected_value, test_value) << "Key is: " << test_key;
      }
      else if(std::is_same<value_type, double>::value){
        EXPECT_DOUBLE_EQ(expected_value, test_value) << "Key is: " << test_key;
      }
      else{
        std::cout << "Unhandled value type.\n";
      }
    }
  }

  size_t extract_groupby_result_device()
  {

    const dim3 grid_size ((this->hash_table_size + this->THREAD_BLOCK_SIZE - 1) / this->THREAD_BLOCK_SIZE, 1, 1);
    const dim3 block_size (this->THREAD_BLOCK_SIZE, 1, 1);

    // TODO: Find a more efficient way to size the output buffer.
    // In general, input_size is going to be larger than the actual
    // size of the result.
    hipMallocManaged(&d_groupby_result, input_size * sizeof(key_type));
    hipMallocManaged(&d_aggregation_result, input_size * sizeof(value_type));

    // This variable is used by the threads to coordinate where they should write 
    // to the output buffers
    unsigned int * global_write_index{nullptr}; 
    hipMallocManaged(&global_write_index, sizeof(unsigned int));
    *global_write_index = 0;

    hipDeviceSynchronize();
    extract_groupby_result<<<grid_size, block_size>>>((this->the_map).get(), 
                                                      (this->the_map)->size(), 
                                                      d_groupby_result, 
                                                      d_aggregation_result, 
                                                      global_write_index );
    hipDeviceSynchronize();

    size_t result_size = *global_write_index;

    // Return the actual size of the result
    return result_size;

  }

  void verify_groupby_result(size_t computed_result_size, std::map<key_type, value_type> const & expected_values )
  {

    ASSERT_NE(nullptr, d_groupby_result);
    ASSERT_NE(nullptr, d_aggregation_result);

    // The size of the result should be equal to the number of unique keys
    const auto begin = this->d_groupby_column.begin();
    const auto end = this->d_groupby_column.end();
    thrust::sort(begin, end);
    size_t unique_count = thrust::unique(begin, end) - begin;
    ASSERT_EQ(unique_count, computed_result_size);

    // Prefetch groupby and aggregation result to host to improve performance
    hipMemPrefetchAsync(d_groupby_result, input_size * sizeof(key_type), hipCpuDeviceId);
    hipMemPrefetchAsync(d_aggregation_result, input_size * sizeof(value_type), hipCpuDeviceId);

    // Verify that every <key,value> in the computed result is present in the reference solution
    for(size_type i = 0; i < expected_values.size(); ++i)
    {
      key_type groupby_key = d_groupby_result[i];
      value_type aggregation_value = d_aggregation_result[i];

      auto found = expected_values.find(groupby_key);

      ASSERT_NE(expected_values.end(), found) << "key: " << groupby_key;

      EXPECT_EQ(found->first, groupby_key) << "index: " << i;

      if(std::is_integral<value_type>::value){
        EXPECT_EQ(found->second, aggregation_value) << "key: " << groupby_key << " index: " << i;
      }
      else if(std::is_same<value_type, float>::value){
        EXPECT_FLOAT_EQ(found->second, aggregation_value) << "key: " << groupby_key << " index: " << i;
      }
      else if(std::is_same<value_type, double>::value){
        EXPECT_DOUBLE_EQ(found->second, aggregation_value) << "key: " << groupby_key << " index: " << i;
      }
      else{
        std::cout << "Unhandled value type.\n";
      }
    }

  }

  unsigned int groupby(const key_type * const groupby_column, const value_type * const aggregation_column)
  {

    // TODO: Find a more efficient way to size the output buffer.
    // In general, input_size is going to be larger than the actual
    // size of the result.
    hipMallocManaged(&d_groupby_result, input_size * sizeof(key_type));
    hipMallocManaged(&d_aggregation_result, input_size * sizeof(value_type));

    size_type result_size{0};

    GroupbyHash(groupby_column,
                aggregation_column,
                input_size,
                d_groupby_result,
                d_aggregation_result,
                &result_size,
                op_type());

    return result_size;
  }

};

// Google Test can only do a parameterized typed-test over a single type, so we have
// to nest multiple types inside of the KeyValueTypes struct above
// KeyValueTypes<type1, type2> implies key_type = type1, value_type = type2
// This list is the types across which Google Test will run our tests
typedef ::testing::Types< 
                            KeyValueTypes<int32_t, int32_t, max_op>,
                            KeyValueTypes<int32_t, float, max_op>,
                            KeyValueTypes<int32_t, double, max_op>,
                            KeyValueTypes<int32_t, int64_t, max_op>,
                            KeyValueTypes<int32_t, uint64_t, max_op>,
                            KeyValueTypes<int64_t, int32_t, max_op>,
                            KeyValueTypes<int64_t, float, max_op>,
                            KeyValueTypes<int64_t, double, max_op>,
                            KeyValueTypes<int64_t, int64_t, max_op>,
                            KeyValueTypes<int64_t, uint64_t, max_op>,
                            KeyValueTypes<uint64_t, int32_t, max_op>,
                            KeyValueTypes<uint64_t, float, max_op>,
                            KeyValueTypes<uint64_t, double, max_op>,
                            KeyValueTypes<uint64_t, int64_t, max_op>,
                            KeyValueTypes<uint64_t, uint64_t, max_op>,
                            KeyValueTypes<int32_t, int32_t, min_op>,
                            KeyValueTypes<int32_t, float, min_op>,
                            KeyValueTypes<int32_t, double, min_op>,
                            KeyValueTypes<int32_t, int64_t, min_op>,
                            KeyValueTypes<int32_t, uint64_t, min_op>,
                            KeyValueTypes<uint64_t, int32_t, min_op>,
                            KeyValueTypes<uint64_t, float, min_op>,
                            KeyValueTypes<uint64_t, double, min_op>,
                            KeyValueTypes<uint64_t, int64_t, min_op>,
                            KeyValueTypes<uint64_t, uint64_t, min_op>,
                            KeyValueTypes<int32_t, int32_t, count_op>,
                            KeyValueTypes<int32_t, float, count_op>,
                            KeyValueTypes<int32_t, double, count_op>,
                            KeyValueTypes<int32_t, int64_t, count_op>,
                            KeyValueTypes<int32_t, uint64_t, count_op>,
                            KeyValueTypes<uint64_t, int32_t, count_op>,
                            KeyValueTypes<uint64_t, float, count_op>,
                            KeyValueTypes<uint64_t, double, count_op>,
                            KeyValueTypes<uint64_t, int64_t, count_op>,
                            KeyValueTypes<uint64_t, uint64_t, count_op>,
                            KeyValueTypes<int32_t, int32_t, sum_op>,
                            //KeyValueTypes<int32_t, float, sum_op>, // TODO: Tests for SUM on single precision floats currently fail due to numerical stability issues
                            KeyValueTypes<int32_t, double, sum_op>,
                            KeyValueTypes<int32_t, int64_t, sum_op>,
                            KeyValueTypes<int32_t, uint64_t, sum_op>,
                            KeyValueTypes<uint64_t, double, sum_op>,
                            KeyValueTypes<uint64_t, double, sum_op>,
                            KeyValueTypes<uint64_t, int64_t, sum_op>,
                            KeyValueTypes<uint64_t, uint64_t, sum_op>
                            > Implementations;

  TYPED_TEST_CASE(GroupByTest, Implementations);




TYPED_TEST(GroupByTest, AggregationTestDeviceAllSame)
{
  const int num_keys = 1;
  const int num_values_per_key = 1<<12;

  auto input = this->create_input(num_keys, num_values_per_key);

  // When you have a templated member function of a templated class, the preceeding 'template' keyword is required
  // See: https://stackoverflow.com/questions/16508743/error-expected-expression-in-this-template-code
  using aggregation_op = typename GroupByTest<TypeParam>::op_type;
  auto expected_values = this->template compute_reference_solution<aggregation_op>();

  this->build_aggregation_table_device(input);
  this->verify_aggregation_table(expected_values);

  size_t computed_result_size = this->extract_groupby_result_device();
  this->verify_groupby_result(computed_result_size, expected_values);
}

// TODO Update the create_input function to ensure all keys are actually unique
TYPED_TEST(GroupByTest, AggregationTestDeviceAllUnique)
{
  const int num_keys = 1<<12;
  const int num_values_per_key = 1;
  auto input = this->create_input(num_keys, num_values_per_key);
  // When you have a templated member function of a templated class, the preceeding 'template' keyword is required
  // See: https://stackoverflow.com/questions/16508743/error-expected-expression-in-this-template-code
  using aggregation_op = typename GroupByTest<TypeParam>::op_type;
  auto expected_values = this->template compute_reference_solution<aggregation_op>();

  this->build_aggregation_table_device(input);
  this->verify_aggregation_table(expected_values);

  size_t computed_result_size = this->extract_groupby_result_device();
  this->verify_groupby_result(computed_result_size,expected_values);
}

TYPED_TEST(GroupByTest, AggregationTestDeviceWarpSame)
{
  const int num_keys = 1<<12;
  const int num_values_per_key = 32;

  auto input = this->create_input(num_keys, num_values_per_key);
  // When you have a templated member function of a templated class, the preceeding 'template' keyword is required
  // See: https://stackoverflow.com/questions/16508743/error-expected-expression-in-this-template-code
  using aggregation_op = typename GroupByTest<TypeParam>::op_type;
  auto expected_values = this->template compute_reference_solution<aggregation_op>();

  this->build_aggregation_table_device(input);
  this->verify_aggregation_table(expected_values);

  size_t computed_result_size = this->extract_groupby_result_device();
  this->verify_groupby_result(computed_result_size,expected_values);
}

TYPED_TEST(GroupByTest, AggregationTestDeviceBlockSame)
{
  const int num_keys = 1<<8;
  const int num_values_per_key = this->THREAD_BLOCK_SIZE;
  auto input = this->create_input(num_keys, num_values_per_key);
  // When you have a templated member function of a templated class, the preceeding 'template' keyword is required
  // See: https://stackoverflow.com/questions/16508743/error-expected-expression-in-this-template-code
  using aggregation_op = typename GroupByTest<TypeParam>::op_type;
  auto expected_values = this->template compute_reference_solution<aggregation_op>();

  this->build_aggregation_table_device(input);
  this->verify_aggregation_table(expected_values);

  size_t computed_result_size = this->extract_groupby_result_device();
  this->verify_groupby_result(computed_result_size, expected_values);
}

TYPED_TEST(GroupByTest, GroupByHash)
{
  const int num_keys = 1<<12;
  const int num_values_per_key = 1;

  auto input = this->create_input(num_keys, num_values_per_key);
  // When you have a templated member function of a templated class, the preceeding 'template' keyword is required
  // See: https://stackoverflow.com/questions/16508743/error-expected-expression-in-this-template-code
  using aggregation_op = typename GroupByTest<TypeParam>::op_type;
  auto expected_values = this->template compute_reference_solution<aggregation_op>();

  const size_t computed_result_size = this->groupby(input.first, input.second);
  this->verify_groupby_result(computed_result_size,expected_values);
}



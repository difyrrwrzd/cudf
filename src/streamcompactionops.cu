#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>


#include <hip/hip_runtime.h>
#include <vector>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <thrust/iterator/counting_iterator.h>

#include <thrust/execution_policy.h>
#include <thrust/iterator/iterator_adaptor.h>
#include <thrust/iterator/transform_iterator.h>

//std lib
#include <map>




//wow the freaking example from iterator_adaptpr, what a break right!
template<typename Iterator>
class repeat_iterator
		: public thrust::iterator_adaptor<
		  repeat_iterator<Iterator>, // the first template parameter is the name of the iterator we're creating
		  Iterator                   // the second template parameter is the name of the iterator we're adapting
		  // we can use the default for the additional template parameters
		  >
{
public:
	// shorthand for the name of the iterator_adaptor we're deriving from
	typedef thrust::iterator_adaptor<
			repeat_iterator<Iterator>,
			Iterator
			> super_t;
	__host__ __device__
	repeat_iterator(const Iterator &x, int n) : super_t(x), begin(x), n(n) {}
	// befriend thrust::iterator_core_access to allow it access to the private interface below
	friend class thrust::iterator_core_access;
private:
	// repeat each element of the adapted range n times
	unsigned int n;
	// used to keep track of where we began
	const Iterator begin;
	// it is private because only thrust::iterator_core_access needs access to it
	__host__ __device__
	typename super_t::reference dereference() const
	{
		return *(begin + (this->base() - begin) / n);
	}
};



typedef repeat_iterator<thrust::detail::normal_iterator<thrust::device_ptr<gdf_valid_type> > > gdf_valid_iterator;

size_t get_number_of_bytes_for_valid (size_t column_size) {
    return sizeof(gdf_valid_type) * (column_size + GDF_VALID_BITSIZE - 1) / GDF_VALID_BITSIZE;
}


// note: functor inherits from unary_function
struct modulus_bit_width : public thrust::unary_function<gdf_size_type,gdf_size_type>
{
	size_t n_bytes;
	size_t column_size;
	
	modulus_bit_width (size_t b_nytes, size_t column_size) {
		this->n_bytes = n_bytes;
		this->column_size = column_size;
	}
	__host__ __device__
	gdf_size_type operator()(gdf_size_type x) const
	{
		int col_position = x / 8;	
        int length_col = n_bytes != col_position+1 ? GDF_VALID_BITSIZE : column_size - GDF_VALID_BITSIZE * (n_bytes - 1);
		//return x % GDF_VALID_BITSIZE;
		return (length_col - 1) - (x % 8);
		// x << 
	}
};

// note: functor inherits from unary_function
struct shift_operator : public thrust::unary_function<gdf_valid_type,gdf_valid_type>
{
	int num_bits;
	shift_operator (int num_bits) {
		this->num_bits = num_bits;
	}
	
	__host__ __device__
	gdf_valid_type operator()(gdf_valid_type x) const
	{
		return x << num_bits;
	}
};

typedef thrust::transform_iterator<modulus_bit_width, thrust::counting_iterator<gdf_size_type> > bit_position_iterator;


template<typename stencil_type>
struct is_stencil_true
{
	__host__ __device__
	bool operator()(const thrust::tuple<stencil_type, gdf_valid_iterator::value_type, bit_position_iterator::value_type> value)
	{
		gdf_size_type position = thrust::get<2>(value);

		return ((thrust::get<1>(value) >> position) & 1) && (thrust::get<0>(value) != 0);
	}
};

struct is_bit_set
{
	__host__ __device__
	bool operator()(const thrust::tuple< gdf_valid_iterator::value_type, bit_position_iterator::value_type> value)
	{
		gdf_size_type position = thrust::get<1>(value);

		return ((thrust::get<0>(value) >> position) & 1);
	}
};
/*
	//before
	for(int i = 0; i < column->size; i++){
		int col_position = i / 8;
		int bit_offset = i % 8;
	}
	//now 
	for(int i = 0; i < column->size; i++) {
        int col_position =  i / 8;
		// i = col_position * 8
        int length_col = n_bytes != col_position+1 ? GDF_VALID_BITSIZE : column->size - GDF_VALID_BITSIZE * (n_bytes - 1);
        int bit_offset =  (length_col - 1) - (i % 8);
    }
*/
struct bit_mask_pack_op : public thrust::unary_function<int64_t,gdf_valid_type>
{
	__host__ __device__
		gdf_valid_type operator()(const int64_t expanded)
		{
			gdf_valid_type result = 0;
			for(int i = 0; i < GDF_VALID_BITSIZE; i++){
				// 0, 8, 16, ....,48,  56
				unsigned char byte = (expanded >> ( (GDF_VALID_BITSIZE - 1 - i )  * 8));
				result |= (byte & 1) << i;
			}
			return (result);
		}
};

/*
damn im passing out, ok how im going to do this, ill derefence the fucking entire character, offsetting with the iterator i make however the fuck that works right?
		then i will make it so that i zip a counting iterator as well and use that counting iterator to figure out which bit to and
		i think something like a repeat iterator or somethign else will suffce

 */

std::map<gdf_dtype, int16_t> column_type_width = {{GDF_INT8, sizeof(int8_t)}, {GDF_INT16, sizeof(int16_t)},{GDF_INT32, sizeof(int32_t)}, {GDF_INT64, sizeof(int64_t)},
		{GDF_FLOAT32, sizeof(float)}, {GDF_FLOAT64, sizeof(double)} };
//because applying a stencil only needs to know the WIDTH of a type for copying to output, we won't be making a bunch of templated version to store this but rather
//storing a map from gdf_type to width
gdf_error gpu_apply_stencil(gdf_column *lhs, gdf_column * stencil, gdf_column * output){
	//OK: add a rquire here that output and lhs are the same size
	GDF_REQUIRE(output->size == lhs->size, GDF_COLUMN_SIZE_MISMATCH);


	//find the width in bytes of this data type
	auto searched_item = column_type_width.find(lhs->dtype);
	int16_t width = searched_item->second; //width in bytes

	searched_item = column_type_width.find(stencil->dtype);
	int16_t stencil_width= searched_item->second; //width in bytes

	hipStream_t stream;
	hipStreamCreate(&stream);

	size_t n_bytes = get_number_of_bytes_for_valid(stencil->size);

	bit_position_iterator bit_position_iter(thrust::make_counting_iterator<gdf_size_type>(0), modulus_bit_width(n_bytes, stencil->size));
	gdf_valid_iterator valid_iterator(thrust::detail::make_normal_iterator(thrust::device_pointer_cast(stencil->valid)),GDF_VALID_BITSIZE);
	//TODO: can probably make this happen with some kind of iterator so it can work on any width size

	//zip the stencil and the valid iterator together
	typedef thrust::tuple<thrust::detail::normal_iterator<thrust::device_ptr<int8_t> >,gdf_valid_iterator, bit_position_iterator > zipped_stencil_tuple;
	typedef thrust::zip_iterator<zipped_stencil_tuple> zipped_stencil_iterator;

	//what kind of shit is that you might wonder?
	//well basically we are zipping up an iterator to the stencil, one to the bit masks, and one which lets us get the bit position based on our index
	zipped_stencil_iterator zipped_stencil_iter(
			thrust::make_tuple(
					thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int8_t * )stencil->data)),
					valid_iterator,
					thrust::make_transform_iterator<modulus_bit_width, thrust::counting_iterator<gdf_size_type> >(
							thrust::make_counting_iterator<gdf_size_type>(0),
							modulus_bit_width(n_bytes, stencil->size))
			));

	//NOTE!!!! the output column is getting set to a specific size  but we are NOT compacting the allocation,
	//whoever calls that should handle that
	if(width == 1){
		thrust::detail::normal_iterator<thrust::device_ptr<int8_t> > input_start =
				thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int8_t *) lhs->data));
		thrust::detail::normal_iterator<thrust::device_ptr<int8_t> > output_start =
				thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int8_t *) output->data));
		thrust::detail::normal_iterator<thrust::device_ptr<int8_t> > output_end =
				thrust::copy_if(thrust::cuda::par.on(stream),input_start,input_start + lhs->size,zipped_stencil_iter,output_start,is_stencil_true<thrust::detail::normal_iterator<thrust::device_ptr<int8_t> >::value_type >());
		output->size = output_end - output_start;
	}else if(width == 2){
		thrust::detail::normal_iterator<thrust::device_ptr<int16_t> > input_start =
				thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int16_t *) lhs->data));
		thrust::detail::normal_iterator<thrust::device_ptr<int16_t> > output_start =
				thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int16_t *) output->data));
		thrust::detail::normal_iterator<thrust::device_ptr<int16_t> > output_end =
				thrust::copy_if(thrust::cuda::par.on(stream),input_start,input_start + lhs->size,zipped_stencil_iter,output_start,is_stencil_true<thrust::detail::normal_iterator<thrust::device_ptr<int8_t> >::value_type >());
		output->size = output_end - output_start;
	}else if(width == 4){
		thrust::detail::normal_iterator<thrust::device_ptr<int32_t> > input_start =
				thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int32_t *) lhs->data));
		thrust::detail::normal_iterator<thrust::device_ptr<int32_t> > output_start =
				thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int32_t *) output->data));
		thrust::detail::normal_iterator<thrust::device_ptr<int32_t> > output_end =
				thrust::copy_if(thrust::cuda::par.on(stream),input_start,input_start + lhs->size,zipped_stencil_iter,output_start,is_stencil_true<thrust::detail::normal_iterator<thrust::device_ptr<int8_t> >::value_type >());
		output->size = output_end - output_start;
	}else if(width == 8){
		thrust::detail::normal_iterator<thrust::device_ptr<int64_t> > input_start =
				thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int64_t *) lhs->data));
		thrust::detail::normal_iterator<thrust::device_ptr<int64_t> > output_start =
				thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int64_t *) output->data));
		thrust::detail::normal_iterator<thrust::device_ptr<int64_t> > output_end =
				thrust::copy_if(thrust::cuda::par.on(stream),input_start,input_start + lhs->size,zipped_stencil_iter,output_start,is_stencil_true<thrust::detail::normal_iterator<thrust::device_ptr<int8_t> >::value_type >());
		output->size = output_end - output_start;
	}

	gdf_size_type num_values = lhs->size;
	//TODO:BRING OVER THE BITMASK!!!
	//need to store a prefix sum
	//align to size 8
	thrust::device_vector<gdf_valid_type> valid_bit_mask; //we are expanding the bit mask to an int8 because I can't envision an algorithm that operates on the bitmask that
	if(num_values % GDF_VALID_BITSIZE != 0){
		valid_bit_mask.resize(num_values + (GDF_VALID_BITSIZE - (num_values % GDF_VALID_BITSIZE))); //align this allocation on GDF_VALID_BITSIZE so we don't have to bounds check
	}else{
		valid_bit_mask.resize(num_values);
	}

	// doesn't require the use for a prefix sum which will have size 8 * num rows which is much larger than this

	typedef thrust::tuple<gdf_valid_iterator, bit_position_iterator > mask_tuple;
	typedef thrust::zip_iterator<mask_tuple> zipped_mask;


	zipped_mask  zipped_mask_iter(
			thrust::make_tuple(
					valid_iterator,
					thrust::make_transform_iterator<modulus_bit_width, thrust::counting_iterator<gdf_size_type> >(
							thrust::make_counting_iterator<gdf_size_type>(0),
							modulus_bit_width(n_bytes, stencil->size))
			)
	);

	typedef thrust::transform_iterator<is_bit_set, zipped_mask > bit_set_iterator;
	bit_set_iterator bit_set_iter = thrust::make_transform_iterator<is_bit_set,zipped_mask>(
			zipped_mask_iter,
			is_bit_set()
	);

	//copy the bitmask to device_vector of int8
	thrust::copy(thrust::cuda::par.on(stream), bit_set_iter, bit_set_iter + num_values, valid_bit_mask.begin());

	//remove the values that don't pass the stencil
	thrust::remove_if(thrust::cuda::par.on(stream),valid_bit_mask.begin(), valid_bit_mask.begin() + num_values,zipped_stencil_iter, is_stencil_true<thrust::detail::normal_iterator<thrust::device_ptr<int8_t> >::value_type >());

	//recompact the values and store them in the output bitmask
	//we can group them into pieces of 8 because we aligned this earlier on when we made the device_vector
	thrust::detail::normal_iterator<thrust::device_ptr<int64_t> > valid_bit_mask_group_8_iter =
			thrust::detail::make_normal_iterator(thrust::device_pointer_cast((int64_t *) valid_bit_mask.data().get()));


	//you may notice that we can write out more bytes than our valid_num_bytes, this only happens when we are not aligned to  GDF_VALID_BITSIZE bytes, becasue the
	//arrow standard requires 64 byte alignment, this is a safe assumption to make
	thrust::transform(thrust::cuda::par.on(stream), valid_bit_mask_group_8_iter, valid_bit_mask_group_8_iter + ((num_values + GDF_VALID_BITSIZE - 1) / GDF_VALID_BITSIZE),
			thrust::detail::make_normal_iterator(thrust::device_pointer_cast(output->valid)),bit_mask_pack_op());

	hipStreamSynchronize(stream);

	hipStreamDestroy(stream);

	return GDF_SUCCESS;

}

size_t  valid_left_length(gdf_column *column) {
    int  n_bytes = get_number_of_bytes_for_valid(column->size);
    size_t length = column->size - GDF_VALID_BITSIZE * (n_bytes - 1);
    if (n_bytes == 1 ) {
        length = column->size;
    }
    return  length;
}

struct valid_array_iterator{
    gdf_column* column;
    size_t iter;
    size_t n_bytes;
    size_t init_length;
    gdf_valid_type init_value; 
	size_t number_of_calls;
    valid_array_iterator(gdf_column* column, gdf_valid_type init, size_t init_size, size_t  init_index = 1) {
        this->column = column;
        this->n_bytes =  sizeof(int8_t) * (column->size + GDF_VALID_BITSIZE - 1) / GDF_VALID_BITSIZE;
        this->init_value = init;
        this->init_length = init_size;
        this->iter = init_index;
		this->number_of_calls = 0;
    }

    template <typename Functor>
    void for_each(Functor output_functor) {
        gdf_valid_type prev = this->init_value;
        size_t prev_length = this->init_length;

        gdf_valid_type current;
        size_t current_length;
        std::tie(current, current_length) = next_node();

        size_t length = column->size - GDF_VALID_BITSIZE * (n_bytes - 1);
        while (true) {
            auto result = concat_bins(prev, current, prev_length, current_length, last_with_too_many_bits(), length);
            output_functor(result, iter);
			number_of_calls++;
            auto result_size = prev_length + current_length;
            if ( !has_next() )
                break;
            prev_length = this->init_length;
            prev = this->column->valid[iter - 1];
            std::tie(current, current_length) = next_node();
        }
        if (last_with_too_many_bits()) {
            auto len = length - current_length;
            auto result = this->column->valid[n_bytes - 1];
            result = result << current_length;
            result = result >> current_length;
            output_functor(result, iter + 1);
			number_of_calls++;
        }
    }
    bool last_with_too_many_bits() {
        size_t length = column->size - GDF_VALID_BITSIZE * (n_bytes - 1);
        if (iter == n_bytes) { // the last one
            // the last one has to many bits
            if (this->init_length + length > GDF_VALID_BITSIZE) {
                return true;
            }
        }
        return false;
    }

    std::tuple<gdf_valid_type, size_t> next_node() {
        gdf_valid_type valid;
        size_t length = column->size - GDF_VALID_BITSIZE * (n_bytes - 1);
        if (iter == n_bytes - 1) { // the last one
            valid = this->column->valid[iter];
            // the last one has to many bits
            if (this->init_length + length > GDF_VALID_BITSIZE) {
                length = GDF_VALID_BITSIZE - this->init_length;
            }
        }
        else {
            length = GDF_VALID_BITSIZE - this->init_length;
            valid = this->column->valid[iter] >> this->init_length;
        }
        iter++;
        return std::make_tuple(valid, length);
    }

	auto concat_bins (gdf_valid_type A, gdf_valid_type B, int len_a, int len_b, bool has_next = false, size_t right_length = -1) -> gdf_valid_type  {
		A = A << len_b;
		if (!has_next) {
			B = B << len_a;
			B = B >> len_a;
		} else {
			B = B >> right_length - len_b;
		}
		return  (A | B);
	}
    bool has_next() {
        return iter < n_bytes;
    }
};

gdf_valid_type * gdf_valid_from_device(gdf_column* column, hipStream_t &stream) {
    gdf_valid_type * host_valid_out;
    size_t n_bytes = get_number_of_bytes_for_valid(column->size);
    host_valid_out = new gdf_valid_type[n_bytes];
    hipMemcpyAsync(host_valid_out, column->valid, n_bytes, hipMemcpyDeviceToHost, stream);
    return host_valid_out;
}

void gdf_copy_valid_from_host_to_device (gdf_column *column, gdf_column *device,  size_t lnbytes, size_t n_bytes, hipStream_t &stream) {
    gdf_valid_type *host_valid = column->valid;
    hipMemcpyAsync(device->valid + sizeof(gdf_valid_type) * (lnbytes - 1), host_valid, n_bytes, hipMemcpyHostToDevice, stream);
}

gdf_error gpu_concat(gdf_column *lhs, gdf_column *rhs, gdf_column *output)
{
	GDF_REQUIRE( (lhs->dtype == output->dtype ) && ( rhs->dtype == output->dtype), GDF_VALIDITY_MISSING);
	GDF_REQUIRE(output->size == lhs->size + rhs->size, GDF_COLUMN_SIZE_MISMATCH);
	hipStream_t stream;
	hipStreamCreate(&stream);

	//@todo: check if  lsh->dtype is NOT GDF_invalid
	int type_width = column_type_width[ lhs->dtype ];

	//copy data 
	hipMemcpyAsync(output->data, lhs->data, type_width * lhs->size, hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync( (void *)( (int8_t*) (output->data) + type_width * lhs->size), rhs->data, type_width * rhs->size, hipMemcpyDeviceToDevice, stream);
	
	int lnbytes = get_number_of_bytes_for_valid(lhs->size);
	int rnbytes = get_number_of_bytes_for_valid(rhs->size);
  
	if (lnbytes > 1) {
		hipMemcpyAsync(output->valid, lhs->valid, sizeof(gdf_valid_type) * (lnbytes - 1), hipMemcpyDeviceToDevice, stream);
	}
	int last_char_index = sizeof(gdf_valid_type) * lnbytes - 1;
	gdf_valid_type* left_char = new gdf_valid_type[1];
	hipError_t error = hipMemcpyAsync(left_char, &lhs->valid[last_char_index], sizeof(gdf_valid_type), hipMemcpyDeviceToHost, stream);
	size_t len_prev = valid_left_length(lhs);

	if (lnbytes == 0) {
        hipMemcpyAsync(output->valid, rhs->valid, sizeof(gdf_valid_type) * rnbytes, hipMemcpyDeviceToDevice, stream);
    }
    else if (rhs->size > 0) {
		gdf_column rhs_host = *rhs;
        rhs_host.valid = gdf_valid_from_device(rhs, stream);
        gdf_valid_type * host_output_valid = new gdf_valid_type[rnbytes];
		valid_array_iterator iter(&rhs_host, *left_char, len_prev, 0);
		iter.for_each( [&host_output_valid, &lnbytes] (gdf_valid_type result, size_t iter) {
 			std::memcpy ( host_output_valid + sizeof(gdf_valid_type) * (iter - 1) , &result, sizeof(gdf_valid_type));
        });
		hipMemcpyAsync(output->valid + sizeof(gdf_valid_type) * (lnbytes - 1), host_output_valid, iter.number_of_calls, hipMemcpyHostToDevice, stream);
	
		delete [] host_output_valid;
		delete [] left_char;
	} else if (lnbytes == 1){
		hipMemcpyAsync(output->valid, lhs->valid, sizeof(gdf_valid_type), hipMemcpyDeviceToDevice, stream);
    }
	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);
	return GDF_SUCCESS;
}


#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>


#include <hip/hip_runtime.h>
#include <vector>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>


/*
 * bit_mask_null_counts Generated using the following code

#include <iostream>


int main()
{
	for (int i = 0 ; i != 256 ; i++) {
		int count = 0;
		for (int p = 0 ; p != 8 ; p++) {
			if (i & (1 << p)) {
				count++;
			}
		}
		std::cout<<(8-count)<<", ";
	}
	std::cout<<std::endl;
}
 */
std::vector<gdf_valid_type> bit_mask_null_counts = { 8, 7, 7, 6, 7, 6, 6, 5, 7, 6, 6, 5, 6, 5, 5, 4, 7, 6, 6, 5, 6, 5, 5, 4, 6, 5, 5, 4, 5, 4, 4, 3, 7, 6, 6, 5, 6, 5, 5, 4, 6, 5, 5, 4, 5, 4, 4, 3, 6, 5, 5, 4, 5, 4, 4, 3, 5, 4, 4, 3, 4, 3, 3, 2, 7, 6, 6, 5, 6, 5, 5, 4, 6, 5, 5, 4, 5, 4, 4, 3, 6, 5, 5, 4, 5, 4, 4, 3, 5, 4, 4, 3, 4, 3, 3, 2, 6, 5, 5, 4, 5, 4, 4, 3, 5, 4, 4, 3, 4, 3, 3, 2, 5, 4, 4, 3, 4, 3, 3, 2, 4, 3, 3, 2, 3, 2, 2, 1, 7, 6, 6, 5, 6, 5, 5, 4, 6, 5, 5, 4, 5, 4, 4, 3, 6, 5, 5, 4, 5, 4, 4, 3, 5, 4, 4, 3, 4, 3, 3, 2, 6, 5, 5, 4, 5, 4, 4, 3, 5, 4, 4, 3, 4, 3, 3, 2, 5, 4, 4, 3, 4, 3, 3, 2, 4, 3, 3, 2, 3, 2, 2, 1, 6, 5, 5, 4, 5, 4, 4, 3, 5, 4, 4, 3, 4, 3, 3, 2, 5, 4, 4, 3, 4, 3, 3, 2, 4, 3, 3, 2, 3, 2, 2, 1, 5, 4, 4, 3, 4, 3, 3, 2, 4, 3, 3, 2, 3, 2, 2, 1, 4, 3, 3, 2, 3, 2, 2, 1, 3, 2, 2, 1, 2, 1, 1, 0 };

unsigned char gdf_num_bits_zero_after_pos(unsigned char number, int pos){
	//if pos == 0 then its aligned
	if(pos == 0){
		return 0;
	}
	unsigned char count = 0;
	for (int p = pos ; p != 8 ; p++) {
		if (number & (number << p)) {
			count++;
		}
	}
	return (8 - pos) - count;
}

template <typename LeftType,typename RightType,typename ResultType >
struct gdf_equals_op : public thrust::binary_function< LeftType, RightType, ResultType>
{
	__host__ __device__
	ResultType operator()(LeftType x, RightType y)
	{
		return x == y;
	}
};

template <typename LeftType,typename RightType,typename ResultType >
struct gdf_not_equals_op : public thrust::binary_function< LeftType, RightType, ResultType>
{
	__host__ __device__
	ResultType operator()(LeftType x, RightType y)
	{
		return x != y;
	}
};

template <typename LeftType,typename RightType,typename ResultType >
struct gdf_greater_than_op : public thrust::binary_function< LeftType, RightType, ResultType>
{
	__host__ __device__
	ResultType operator()(LeftType x, RightType y)
	{
		return x > y;
	}
};

template <typename LeftType,typename RightType,typename ResultType >
struct gdf_greater_than_or_equals_op : public thrust::binary_function< LeftType, RightType, ResultType>
{
	__host__ __device__
	ResultType operator()(LeftType x, RightType y)
	{
		return x >= y;
	}
};

template <typename LeftType,typename RightType,typename ResultType >
struct gdf_less_than_op : public thrust::binary_function< LeftType, RightType, ResultType>
{
	__host__ __device__
	ResultType operator()(LeftType x, RightType y)
	{
		return x > y;
	}
};

template <typename LeftType,typename RightType,typename ResultType >
struct gdf_less_than_or_equals_op : public thrust::binary_function< LeftType, RightType, ResultType>
{
	__host__ __device__
	ResultType operator()(LeftType x, RightType y)
	{
		return x >= y;
	}
};






/**
 * @brief takes two columns data and their valid bitmasks and performs a comparison operation returning a column of type bool
 *
 * Takes two thrust::iterator_adaptor implemented iterators and performs a filter operation on them that it outputs into a third thust::iterator_adaptor dervied iterator.
 * We are not making assumptions about what kind of data is being passed into it for these pointers so
 *
 * @param begin_left an iterator that implements thrust::iterator_adaptor
 * @param begin_right an iterator that implements thrust::iterator_adaptor
 * @param result an iterator that implements thrust::iterator_adaptor
 * @param operation an enum telling us what kind of comparision operation we are trying to do
 * @param num_values the number of rows in our columns
 * @param valid_left left column null bitmask (1 = not null)
 * @param valid_right right column null bitmask
 * @param valid_out output column null bitmask
 * @param left_null_count tells us if there are any nulls in the left column
 * @param right_null_count tells us if there are any nulls in the right column
 * @param
 */
template<typename IteratorTypeLeft, typename IteratorTypeRight, typename IteratorTypeResult,
class LeftType = typename IteratorTypeLeft::value_type, class RightType = typename IteratorTypeRight::value_type, class ResultType = typename IteratorTypeResult::value_type>
void gpu_filter_op(IteratorTypeLeft begin_left, IteratorTypeRight begin_right, IteratorTypeResult result, gdf_comparison_operator operation, gdf_size_type num_values,
		gdf_valid_type * valid_left, gdf_valid_type * valid_right, gdf_valid_type * valid_out, gdf_size_type left_null_count, gdf_size_type right_null_count, gdf_size_type & out_null_count, hipStream_t stream){

	//TODO: be able to pass in custom comparison operators so we can handle types that have not implemented these oeprators

	IteratorTypeLeft end_left = begin_left + num_values;


	//regardless of nulls we perform the same operation
	//the nulls we are going to and together
	if (operation == GDF_EQUALS) {

		gdf_equals_op<LeftType, RightType, ResultType> op;
		thrust::transform(thrust::cuda::par.on(stream), begin_left, end_left, begin_right, result, op);

	} else if (operation == GDF_NOT_EQUALS) {
		gdf_not_equals_op<LeftType, RightType, ResultType> op;
		thrust::transform(thrust::cuda::par.on(stream), begin_left, end_left, begin_right, result, op);

	} else if (operation == GDF_GREATER_THAN_OR_EQUALS) {
		gdf_greater_than_or_equals_op<LeftType, RightType, ResultType> op;
		thrust::transform(thrust::cuda::par.on(stream), begin_left, end_left, begin_right, result, op);

	} else if (operation == GDF_GREATER_THAN) {
		gdf_greater_than_op<LeftType, RightType, ResultType> op;
		thrust::transform(thrust::cuda::par.on(stream), begin_left, end_left, begin_right, result, op);

	} else if (operation == GDF_LESS_THAN) {
		gdf_less_than_op<LeftType, RightType, ResultType> op;
		thrust::transform(thrust::cuda::par.on(stream), begin_left, end_left, begin_right, result, op);

	} else if (operation == GDF_LESS_THAN_OR_EQUALS) {
		gdf_less_than_or_equals_op<LeftType, RightType, ResultType> op;
		thrust::transform(thrust::cuda::par.on(stream),begin_left, end_left, begin_right, result, op);

	}

	gdf_size_type num_chars_bitmask = ( ( num_values +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	//TODO: if we could make sure that these things aligned on 8 byte boundaries we could probable do this more efficiently as an unsigned long long
	if((left_null_count == 0) && (right_null_count == 0) && false){
		thrust::device_ptr<gdf_valid_type> valid_out_ptr = thrust::device_pointer_cast(valid_out);
		gdf_valid_type max_char = 255;
		thrust::fill(thrust::cuda::par.on(stream),thrust::detail::make_normal_iterator(valid_out_ptr),thrust::detail::make_normal_iterator(valid_out_ptr + num_chars_bitmask),max_char);
		//we have no nulls so set all the bits in gdf_valid_type to 1
		out_null_count = 0;

	}else{


		thrust::device_ptr<gdf_valid_type> valid_out_ptr = thrust::device_pointer_cast(valid_out);
		thrust::device_ptr<gdf_valid_type> valid_left_ptr = thrust::device_pointer_cast(valid_left);
		//here we are basically figuring out what is the last pointed to unsigned char that can contain part of the bitmask
		thrust::device_ptr<gdf_valid_type> valid_left_end_ptr = thrust::device_pointer_cast(valid_left + num_chars_bitmask );
		thrust::device_ptr<gdf_valid_type> valid_right_ptr = thrust::device_pointer_cast(valid_right);

		//TODO:: I am assuming
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(valid_left_ptr),
				thrust::detail::make_normal_iterator(valid_left_end_ptr), thrust::detail::make_normal_iterator(valid_right_ptr),
				thrust::detail::make_normal_iterator(valid_out_ptr), thrust::bit_and<gdf_valid_type>());

		//figure out how to count nulls from the bitmask on gpu
		//a fast algorithm would create a look up table that stores how many bits are set for each value
		//then we make a thrust::permutation iterator based on those values, we can hold it in gpu memory if we want o rmake it part of initializing
		//we can also copy it in adhoc its really not that big at all


		char * last_char = new char[1];
		hipError_t error = hipMemcpyAsync(last_char,valid_out + ( num_chars_bitmask-1),sizeof(gdf_valid_type),hipMemcpyDeviceToHost,stream);


		thrust::device_vector<gdf_valid_type> bit_mask_null_counts_device(bit_mask_null_counts);

		//this permutation iterator makes it so that each char basically gets replaced with its number of null counts
		//so if you sum up this perm iterator you add up all of the counts for null values per unsigned char
		thrust::permutation_iterator<thrust::device_vector<gdf_valid_type>::iterator,thrust::detail::normal_iterator<thrust::device_ptr<gdf_valid_type> > >
		null_counts_iter( bit_mask_null_counts_device.begin(),thrust::detail::make_normal_iterator(valid_out_ptr));

		//you will notice that we subtract the number of zeros we found in the last character
		out_null_count = thrust::reduce(null_counts_iter, null_counts_iter + num_chars_bitmask) - gdf_num_bits_zero_after_pos(*last_char,num_values % GDF_VALID_BITSIZE );

		delete[] last_char;

	}

	hipStreamSynchronize(stream);

}



gdf_error gpu_comparison(gdf_column *lhs, gdf_column *rhs, gdf_column *output,gdf_comparison_operator operation){
	GDF_REQUIRE(lhs->size == rhs->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(lhs->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	//TODO: consider adding more requirements like that the columns be well defined in their type
	//I commented this out because I am not sure if we want to require the output be an int8
	//GDF_REQUIRE(output->dtype == GDF_INT8,GDF_UNSUPPORTED_DTYPE);

	// SO... I know the follow code looks, questionable, but the other option is to have a shitload of function definitions
	// given that our gdf_columns very conveniently carry around their types with them, this seems to be to be simpler
	// than having tons of function definitions. it also makes it so much nicer to just type gpu_filter(lhs,rhs,output);
	// also we are making it so that we can send any types here, the only one which is debatable I feel is output which
	// we could decide to always have be an int8 since the output is a boolean



	hipStream_t stream;
	hipStreamCreate(&stream);



	if(lhs->dtype == GDF_INT8){
		thrust::device_ptr<int8_t> left_ptr((int8_t *) lhs->data);
		if(rhs->dtype == GDF_INT8){
			thrust::device_ptr<int8_t> right_ptr((int8_t *) rhs->data);
			if(output->dtype == GDF_INT8){

				thrust::device_ptr<int8_t> out_ptr((int8_t *) output->data);
				gpu_filter_op(
						thrust::detail::make_normal_iterator(left_ptr),thrust::detail::make_normal_iterator(right_ptr),
						thrust::detail::make_normal_iterator(out_ptr),operation,lhs->size,lhs->valid,rhs->valid,output->valid,
						lhs->null_count,rhs->null_count,output->null_count,stream
				);
			}else if(output->dtype == GDF_INT16){
				thrust::device_ptr<int16_t> out_ptr((int16_t *) output->data);
				gpu_filter_op(
						thrust::detail::make_normal_iterator(left_ptr),thrust::detail::make_normal_iterator(right_ptr),
						thrust::detail::make_normal_iterator(out_ptr),operation,lhs->size,lhs->valid,rhs->valid,output->valid,
						lhs->null_count,rhs->null_count,output->null_count,stream
				);

			}else if(output->dtype == GDF_INT32){
				thrust::device_ptr<int32_t> out_ptr((int32_t *) output->data);
				gpu_filter_op(
						thrust::detail::make_normal_iterator(left_ptr),thrust::detail::make_normal_iterator(right_ptr),
						thrust::detail::make_normal_iterator(out_ptr),operation,lhs->size,lhs->valid,rhs->valid,output->valid,
						lhs->null_count,rhs->null_count,output->null_count,stream
				);

			}else if(output->dtype == GDF_INT64){
				thrust::device_ptr<int64_t> out_ptr((int64_t *) output->data);
				gpu_filter_op(
						thrust::detail::make_normal_iterator(left_ptr),thrust::detail::make_normal_iterator(right_ptr),
						thrust::detail::make_normal_iterator(out_ptr),operation,lhs->size,lhs->valid,rhs->valid,output->valid,
						lhs->null_count,rhs->null_count,output->null_count,stream
				);

			}else if(output->dtype == GDF_FLOAT32){
				thrust::device_ptr<float> out_ptr((float *) output->data);
				gpu_filter_op(
						thrust::detail::make_normal_iterator(left_ptr),thrust::detail::make_normal_iterator(right_ptr),
						thrust::detail::make_normal_iterator(out_ptr),operation,lhs->size,lhs->valid,rhs->valid,output->valid,
						lhs->null_count,rhs->null_count,output->null_count,stream
				);

			}else if(output->dtype == GDF_FLOAT64){
				thrust::device_ptr<double> out_ptr((double *) output->data);
				gpu_filter_op(
						thrust::detail::make_normal_iterator(left_ptr),thrust::detail::make_normal_iterator(right_ptr),
						thrust::detail::make_normal_iterator(out_ptr),operation,lhs->size,lhs->valid,rhs->valid,output->valid,
						lhs->null_count,rhs->null_count,output->null_count,stream
				);
			}

		}else if(rhs->dtype == GDF_INT16){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}
	}else if(lhs->dtype == GDF_INT16){
		if(rhs->dtype == GDF_INT8){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT16){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}
	}else if(lhs->dtype == GDF_INT32){
		if(rhs->dtype == GDF_INT8){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT16){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}
	}else if(lhs->dtype == GDF_INT64){
		if(rhs->dtype == GDF_INT8){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT16){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}
	}else if(lhs->dtype == GDF_FLOAT32){
		if(rhs->dtype == GDF_INT8){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT16){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}
	}else if(lhs->dtype == GDF_FLOAT64){
		if(rhs->dtype == GDF_INT8){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT16){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_INT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT32){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}else if(rhs->dtype == GDF_FLOAT64){
			if(output->dtype == GDF_INT8){

			}else if(output->dtype == GDF_INT16){

			}else if(output->dtype == GDF_INT32){

			}else if(output->dtype == GDF_INT64){

			}else if(output->dtype == GDF_FLOAT32){

			}else if(output->dtype == GDF_FLOAT64){

			}

		}
	}












hipStreamSynchronize(stream);
hipStreamDestroy(stream);



return GDF_SUCCESS;
}



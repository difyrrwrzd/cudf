#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <vector>

#include <thrust/device_vector.h>

#include "gtest/gtest.h"
#include <gdf/gdf.h>
#include <gdf/cffi/functions.h>
#include <../../src/hashmap/concurrent_unordered_map.cuh>


// This is necessary to do a parametrized typed-test over multiple template arguments
template <typename Key, typename Value>
struct KeyValueTypes
{
  using key_type = Key;
  using value_type = Value;
};


// A new instance of this class will be created for each *TEST(MapTest, ...)
// Put all repeated stuff for each test here
template <class T>
struct MapTest : public testing::Test 
{
  using key_type = typename T::key_type;
  using value_type = typename T::value_type;
  using map_type = concurrent_unordered_map<key_type, value_type, std::numeric_limits<key_type>::max()>;

  std::unique_ptr<map_type> the_map;

  const key_type unused_key = std::numeric_limits<key_type>::max();
  const value_type unused_value = std::numeric_limits<value_type>::max();

  const int size;


  MapTest(const int hash_table_size = 100)
    : size(hash_table_size), the_map(new map_type(hash_table_size))
  {
  }

  ~MapTest(){
  }


};

// Google Test can only do a parameterized typed-test over a single type, so we have
// to nest multiple types inside of the KeyValueTypes struct above
// KeyValueTypes<type1, type2> implies key_type = type1, value_type = type2
// This list is the types across which Google Test will run our tests
typedef ::testing::Types< KeyValueTypes<int,int>,
                          KeyValueTypes<int,float>,
                          KeyValueTypes<int,double>,
                          KeyValueTypes<int,long long int>,
                          KeyValueTypes<int,unsigned long long int>,
                          KeyValueTypes<unsigned long long int, int>,
                          KeyValueTypes<unsigned long long int, float>,
                          KeyValueTypes<unsigned long long int, double>,
                          KeyValueTypes<unsigned long long int, long long int>,
                          KeyValueTypes<unsigned long long int, unsigned long long int>
                          > Implementations;

TYPED_TEST_CASE(MapTest, Implementations);

TYPED_TEST(MapTest, InitialState)
{
  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  auto begin = this->the_map->begin();
  auto end = this->the_map->end();
  EXPECT_NE(begin,end);

}

TYPED_TEST(MapTest, CheckUnusedValues){

  EXPECT_EQ(this->the_map->get_unused_key(), this->unused_key);

  auto begin = this->the_map->begin();
  EXPECT_EQ(begin->first, this->unused_key);
  EXPECT_EQ(begin->second, this->unused_value);
}

TYPED_TEST(MapTest, Insert)
{
  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  const int NUM_PAIRS{this->size};

  // Generate a list of pairs (key, value) to insert into map
  std::vector<thrust::pair<key_type, value_type>> pairs(NUM_PAIRS);
  std::generate(pairs.begin(), pairs.end(), 
                [] () {static int i = 0; return thrust::make_pair(i,(i++)*10);});

  // Insert every pair into the map
  for(const auto& it : pairs){
    this->the_map->insert(it);
  }

  // Make sure all the pairs are in the map
  for(const auto& it : pairs){
    auto found = this->the_map->find(it.first);
    EXPECT_NE(found, this->the_map->end());
    EXPECT_EQ(found->first, it.first);
    EXPECT_EQ(found->second, it.second);
  }

}

TYPED_TEST(MapTest, MaxAggregationTestHost)
{

  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  thrust::pair<key_type, value_type> first_pair{0,0};
  thrust::pair<key_type, value_type> second_pair{0,10};
  thrust::pair<key_type, value_type> third_pair{0,5};

  auto max = [](value_type a, value_type b) { return (a >= b ? a : b); };

  this->the_map->insert(first_pair, max);
  auto found = this->the_map->find(0);
  EXPECT_EQ(0, found->second);

  this->the_map->insert(second_pair, max);
  found = this->the_map->find(0);
  EXPECT_EQ(10, found->second);

  this->the_map->insert(third_pair, max);
  found = this->the_map->find(0);
  EXPECT_EQ(10, found->second);

  this->the_map->insert(thrust::make_pair(0,11), max);
  found = this->the_map->find(0);
  EXPECT_EQ(11, found->second);

  this->the_map->insert(thrust::make_pair(7, 42), max);
  found = this->the_map->find(7);
  EXPECT_EQ(42, found->second);

  this->the_map->insert(thrust::make_pair(7, 62), max);
  found = this->the_map->find(7);
  EXPECT_EQ(62, found->second);

  this->the_map->insert(thrust::make_pair(7, 42), max);
  found = this->the_map->find(7);
  EXPECT_EQ(62, found->second);

  found = this->the_map->find(0);
  EXPECT_EQ(11, found->second);

}


template<typename map_type, typename Aggregation_Operator>
__global__ void build_table(map_type * const the_map,
                            const typename map_type::key_type * const input_keys,
                            const typename map_type::mapped_type * const input_values,
                            const typename map_type::size_type input_size,
                            Aggregation_Operator op)
{

  using size_type = typename map_type::size_type;

  size_type i = threadIdx.x + blockIdx.x * blockDim.x;

  while( i < input_size ){
    const auto p = thrust::make_pair(input_keys[i], input_values[i]);
    the_map->insert(p, op);
    i += blockDim.x * gridDim.x;
  }

}

// Have to use a functor instead of a device lambda because
// you can't create a device lambda inside of a Google Test
// because the macro expands into a private member function
// and you can't have a device lambda inside a private member
// function
template<typename value_type>
struct max_op
{
  __host__ __device__
  value_type operator()(value_type a, value_type b)
  {
    return (a > b? a : b);
  }
};


TYPED_TEST(MapTest, MaxAggregationTestDevice)
{
  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;
  using size_type = typename MapTest<TypeParam>::map_type::size_type;

  std::vector<key_type>     keys   {0, 0, 0, 5, 5, 5, 10, 10, 10, 11, 12, 13};
  std::vector<value_type>   values {0, 1, 2, 5, 5, 4, 13, 12, 11,  6, 97, 42};

  thrust::device_vector<key_type> d_keys(keys);
  thrust::device_vector<value_type> d_values(values);

  key_type *k = thrust::raw_pointer_cast(d_keys.data());
  value_type *v = thrust::raw_pointer_cast(d_values.data());

  size_type input_size = keys.size();

  hipDeviceSynchronize();
  build_table<<<1,256>>>((this->the_map).get(), k, v, input_size, max_op<value_type>());
  hipDeviceSynchronize(); 

  auto found = this->the_map->find(0);
  EXPECT_EQ(2, found->second);

  found = this->the_map->find(5);
  EXPECT_EQ(5, found->second);

  found = this->the_map->find(10);
  EXPECT_EQ(13, found->second);

  found = this->the_map->find(11);
  EXPECT_EQ(6, found->second);

  found = this->the_map->find(13);
  EXPECT_EQ(42, found->second);

  found = this->the_map->find(12);
  EXPECT_EQ(97, found->second);

}


int main(int argc, char * argv[]){
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}

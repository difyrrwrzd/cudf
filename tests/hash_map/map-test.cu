#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <vector>

#include <thrust/device_vector.h>

#include "gtest/gtest.h"
#include <gdf/gdf.h>
#include <gdf/cffi/functions.h>
#include <../../src/hashmap/concurrent_unordered_map.cuh>

// This is necessary to do a parametrized typed-test over multiple template arguments
template <typename Key, typename Value>
struct KeyValueTypes
{
  using key_type = Key;
  using value_type = Value;
};


// A new instance of this class will be created for each *TEST(MapTest, ...)
// Put all repeated stuff for each test here
template <class T>
class MapTest : public testing::Test 
{
public:
  using key_type = typename T::key_type;
  using value_type = typename T::value_type;


  concurrent_unordered_map<key_type, 
                                value_type, 
                                std::numeric_limits<key_type>::max() > the_map;

  const key_type unused_key = std::numeric_limits<key_type>::max();
  const value_type unused_value = std::numeric_limits<value_type>::max();

  const int size;


  MapTest(const int hash_table_size = 100)
    : the_map(hash_table_size), size(hash_table_size)
  {


  }

  ~MapTest(){
  }


};

// Google Test can only do a parameterized typed-test over a single type, so we have
// to nest multiple types inside of the KeyValueTypes struct above
// KeyValueTypes<type1, type2> implies key_type = type1, value_type = type2
// This list is the types across which Google Test will run our tests
typedef ::testing::Types< KeyValueTypes<int,int>, 
                          KeyValueTypes<int,float>, 
                          KeyValueTypes<int,double>,
                          KeyValueTypes<int,long long int>,
                          KeyValueTypes<int,unsigned long long int>,
                          KeyValueTypes<unsigned long long int, int>,
                          KeyValueTypes<unsigned long long int, float>,
                          KeyValueTypes<unsigned long long int, double>,
                          KeyValueTypes<unsigned long long int, long long int>,
                          KeyValueTypes<unsigned long long int, unsigned long long int>
                          > Implementations;

TYPED_TEST_CASE(MapTest, Implementations);

TYPED_TEST(MapTest, InitialState)
{
  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  auto begin = this->the_map.begin();
  auto end = this->the_map.end();
  EXPECT_NE(begin,end);

}

TYPED_TEST(MapTest, CheckUnusedValues){

  EXPECT_EQ(this->the_map.get_unused_key(), this->unused_key);

  auto begin = this->the_map.begin();
  EXPECT_EQ(begin->first, this->unused_key);
  EXPECT_EQ(begin->second, this->unused_value);
}

TYPED_TEST(MapTest, Insert)
{
  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  const int NUM_PAIRS{this->size};

  // Generate a list of pairs (key, value) to insert into map
  std::vector<thrust::pair<key_type, value_type>> pairs(NUM_PAIRS);
  std::generate(pairs.begin(), pairs.end(), 
                [] () {static int i = 0; return thrust::make_pair(i,(i++)*10);});

  // Insert every pair into the map
  for(const auto& it : pairs){
    this->the_map.insert(it);
  }

  // Make sure all the pairs are in the map
  for(const auto& it : pairs){
    auto found = this->the_map.find(it.first);
    EXPECT_NE(found, this->the_map.end());
    EXPECT_EQ(found->first, it.first);
    EXPECT_EQ(found->second, it.second);
  }

}

TYPED_TEST(MapTest, MaxAggregationTestHost)
{

  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  thrust::pair<key_type, value_type> first_pair{0,0};
  thrust::pair<key_type, value_type> second_pair{0,10};
  thrust::pair<key_type, value_type> third_pair{0,5};

  auto max = [](value_type a, value_type b) { return std::max<value_type>(a,b); };

  this->the_map.insert(first_pair, max);
  auto found = this->the_map.find(0);
  EXPECT_EQ(0, found->second);

  this->the_map.insert(second_pair, max);
  found = this->the_map.find(0);
  EXPECT_EQ(10, found->second);

  this->the_map.insert(third_pair, max);
  found = this->the_map.find(0);
  EXPECT_EQ(10, found->second);

  this->the_map.insert(thrust::make_pair(0,11), max);
  found = this->the_map.find(0);
  EXPECT_EQ(11, found->second);


  this->the_map.insert(thrust::make_pair(7, 42), max);
  found = this->the_map.find(7);
  EXPECT_EQ(42, found->second);

  this->the_map.insert(thrust::make_pair(7, 62), max);
  found = this->the_map.find(7);
  EXPECT_EQ(62, found->second);

  this->the_map.insert(thrust::make_pair(7, 42), max);
  found = this->the_map.find(7);
  EXPECT_EQ(62, found->second);

  found = this->the_map.find(0);
  EXPECT_EQ(11, found->second);

}


int main(int argc, char * argv[]){
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}

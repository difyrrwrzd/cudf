#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <vector>
#include <unordered_map>

#include <thrust/device_vector.h>

#include "gtest/gtest.h"
#include <gdf/gdf.h>
#include <gdf/cffi/functions.h>
#include <../../src/hashmap/concurrent_unordered_map.cuh>


// This is necessary to do a parametrized typed-test over multiple template arguments
template <typename Key, typename Value>
struct KeyValueTypes
{
  using key_type = Key;
  using value_type = Value;
};

// Have to use a functor instead of a device lambda because
// you can't create a device lambda inside of a Google Test
// because the macro expands into a private member function
// and you can't have a device lambda inside a private member
// function
template<typename value_type>
  struct max_op
  {
    __host__ __device__
    value_type operator()(value_type a, value_type b)
    {
      return (a > b? a : b);
    }
  };


// A new instance of this class will be created for each *TEST(MapTest, ...)
// Put all repeated stuff for each test here
template <class T>
struct MapTest : public testing::Test 
{
  using key_type = typename T::key_type;
  using value_type = typename T::value_type;
  using map_type = concurrent_unordered_map<key_type, value_type, std::numeric_limits<key_type>::max()>;

  std::unique_ptr<map_type> the_map;

  const key_type unused_key = std::numeric_limits<key_type>::max();
  const value_type unused_value = std::numeric_limits<value_type>::max();

  const int size;

  const int THREAD_BLOCK_SIZE{256};

  std::vector<key_type> keys;
  std::vector<value_type> values;
  thrust::device_vector<key_type> d_keys;
  thrust::device_vector<value_type> d_values;

  std::unordered_map<key_type, value_type> expected_values;

  MapTest(const int hash_table_size = 10000)
    : size(hash_table_size), the_map(new map_type(hash_table_size))
  {
  }

  int create_input(const int num_unique_keys, const int num_values_per_key, const int max_key = RAND_MAX, const int max_value = RAND_MAX)
  {


    const int TOTAL_PAIRS = num_unique_keys * num_values_per_key;

    this->the_map.reset(new map_type(2*TOTAL_PAIRS));

    keys.reserve(TOTAL_PAIRS);
    values.reserve(TOTAL_PAIRS);

    // Always use the same seed so the random sequence is the same each time
    std::srand(0);

    for(int i = 0; i < num_unique_keys; ++i )
    {
      // Create random key
      key_type current_key = std::rand() % max_key;

      // Don't use unused_key
      while(current_key == this->unused_key)
      {
        current_key = std::rand();
      }

      // For the current key, generate random values
      for(int j = 0; j < num_values_per_key; ++j)
      {
        value_type current_value = std::rand() % max_value;

        // Don't use unused_value
        while(current_value == this->unused_value)
        {
          current_value = std::rand();
        }

        // Store current key and value
        keys.push_back(current_key);
        values.push_back(current_value);

        // Use a STL map to keep track of the max value for each key
        auto found = expected_values.find(current_key);

        // Key doesn't exist yet, insert it
        if(found == expected_values.end())
        {
          expected_values.insert(std::make_pair(current_key,current_value));
        }
        // Key exists, update the value with the max
        else
        {
          max_op<value_type> op;
          value_type new_value = op(found->second, current_value);
          found->second = new_value;
        }
      }
    }

    d_keys = keys;
    d_values = values;
    return TOTAL_PAIRS;
  }

  void check_answer(){

    for(auto const &k : this->expected_values)
    {
      key_type test_key = k.first;

      value_type expected_value = k.second;

      auto found = this->the_map->find(test_key);

      ASSERT_NE(this->the_map->end(), found);

      value_type test_value = found->second;

      EXPECT_EQ(expected_value, test_value) << "Key is: " << test_key;
    }
  }

  ~MapTest(){
  }


};

// Google Test can only do a parameterized typed-test over a single type, so we have
// to nest multiple types inside of the KeyValueTypes struct above
// KeyValueTypes<type1, type2> implies key_type = type1, value_type = type2
// This list is the types across which Google Test will run our tests
typedef ::testing::Types< KeyValueTypes<int,int>
                         // KeyValueTypes<int,float>,
                         // KeyValueTypes<int,double>,
                         // KeyValueTypes<int,long long int>,
                         // KeyValueTypes<int,unsigned long long int>,
                         // KeyValueTypes<unsigned long long int, int>,
                         // KeyValueTypes<unsigned long long int, float>,
                         // KeyValueTypes<unsigned long long int, double>,
                         // KeyValueTypes<unsigned long long int, long long int>,
                         // KeyValueTypes<unsigned long long int, unsigned long long int>
                          > Implementations;

TYPED_TEST_CASE(MapTest, Implementations);

/*
TYPED_TEST(MapTest, InitialState)
{
  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  auto begin = this->the_map->begin();
  auto end = this->the_map->end();
  EXPECT_NE(begin,end);

}

TYPED_TEST(MapTest, CheckUnusedValues){

  EXPECT_EQ(this->the_map->get_unused_key(), this->unused_key);

  auto begin = this->the_map->begin();
  EXPECT_EQ(begin->first, this->unused_key);
  EXPECT_EQ(begin->second, this->unused_value);
}

TYPED_TEST(MapTest, Insert)
{
  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  const int NUM_PAIRS{this->size};

  // Generate a list of pairs (key, value) to insert into map
  std::vector<thrust::pair<key_type, value_type>> pairs(NUM_PAIRS);
  std::generate(pairs.begin(), pairs.end(), 
                [] () {static int i = 0; return thrust::make_pair(i,(i++)*10);});

  // Insert every pair into the map
  for(const auto& it : pairs){
    this->the_map->insert(it);
  }

  // Make sure all the pairs are in the map
  for(const auto& it : pairs){
    auto found = this->the_map->find(it.first);
    ASSERT_NE(found, this->the_map->end());
    EXPECT_EQ(found->first, it.first);
    EXPECT_EQ(found->second, it.second);
  }

}

TYPED_TEST(MapTest, MaxAggregationTestHost)
{

  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;

  thrust::pair<key_type, value_type> first_pair{0,0};
  thrust::pair<key_type, value_type> second_pair{0,10};
  thrust::pair<key_type, value_type> third_pair{0,5};

  auto max = [](value_type a, value_type b) { return (a >= b ? a : b); };

  this->the_map->insert(first_pair, max);
  auto found = this->the_map->find(0);
  EXPECT_EQ(0, found->second);

  this->the_map->insert(second_pair, max);
  found = this->the_map->find(0);
  EXPECT_EQ(10, found->second);

  this->the_map->insert(third_pair, max);
  found = this->the_map->find(0);
  EXPECT_EQ(10, found->second);

  this->the_map->insert(thrust::make_pair(0,11), max);
  found = this->the_map->find(0);
  EXPECT_EQ(11, found->second);

  this->the_map->insert(thrust::make_pair(7, 42), max);
  found = this->the_map->find(7);
  EXPECT_EQ(42, found->second);

  this->the_map->insert(thrust::make_pair(7, 62), max);
  found = this->the_map->find(7);
  EXPECT_EQ(62, found->second);

  this->the_map->insert(thrust::make_pair(7, 42), max);
  found = this->the_map->find(7);
  EXPECT_EQ(62, found->second);

  found = this->the_map->find(0);
  EXPECT_EQ(11, found->second);

}
*/


template<typename map_type, typename Aggregation_Operator>
__global__ void build_table(map_type * const the_map,
                            const typename map_type::key_type * const input_keys,
                            const typename map_type::mapped_type * const input_values,
                            const typename map_type::size_type input_size,
                            Aggregation_Operator op)
{

  using size_type = typename map_type::size_type;

  size_type i = threadIdx.x + blockIdx.x * blockDim.x;

  while( i < input_size ){
    const auto p = thrust::make_pair(input_keys[i], input_values[i]);
    the_map->insert(p, op);
    i += blockDim.x * gridDim.x;
  }

}



TYPED_TEST(MapTest, MaxAggregationTestDevice)
{
  using key_type = typename TypeParam::key_type;
  using value_type = typename TypeParam::value_type;
  using size_type = typename MapTest<TypeParam>::map_type::size_type;

  const size_type input_size = this->create_input(512, 256*256);

  key_type *k = thrust::raw_pointer_cast(this->d_keys.data());
  value_type *v = thrust::raw_pointer_cast(this->d_values.data());

  const dim3 grid_size ((input_size + this->THREAD_BLOCK_SIZE -1) / this->THREAD_BLOCK_SIZE,1,1);
  const dim3 block_size (this->THREAD_BLOCK_SIZE, 1, 1);

  std::cout << "Input Size: " << input_size << " Grid Size: " << grid_size.x << " Block Size: " << block_size.x << std::endl;

  hipDeviceSynchronize();
  build_table<<<grid_size, block_size>>>((this->the_map).get(), k, v, input_size, max_op<value_type>());
  hipDeviceSynchronize(); 

  this->check_answer();

}


int main(int argc, char * argv[]){
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
